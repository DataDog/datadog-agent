/* AUTO-GENERATED, DO NOT CHANGE */

// To regenerate, run `make heavy-sample.cu` in this directory

#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void kernel_0(float * var_0_0, float * var_0_1, float * var_0_2, float * var_0_3, float * var_0_4, float * var_0_5, float * var_0_6, float * var_0_7, float * var_0_8, float * var_0_9) {
	__shared__ float myVar[1024];
	myVar[0] = 48.682422 * myVar[threadIdx.x];
	myVar[6] = 12.630602 * myVar[threadIdx.x];
	myVar[1] = 28.516594 * myVar[threadIdx.x];
	myVar[8] = 21.327449 * myVar[threadIdx.x];
	myVar[2] = 11.235985 * myVar[threadIdx.x];
	myVar[6] = 9.634747 * myVar[threadIdx.x];
	myVar[0] = 6.104667 * myVar[threadIdx.x];
	myVar[0] = 19.813955 * myVar[threadIdx.x];
	myVar[2] = 47.226601 * myVar[threadIdx.x];
	myVar[6] = 38.647377 * myVar[threadIdx.x];
	var_0_0[0] = myVar[0];
	var_0_1[1] = myVar[1];
	var_0_2[2] = myVar[2];
	var_0_3[3] = myVar[3];
	var_0_4[4] = myVar[4];
	var_0_5[5] = myVar[5];
	var_0_6[6] = myVar[6];
	var_0_7[7] = myVar[7];
	var_0_8[8] = myVar[8];
	var_0_9[9] = myVar[9];
	
}

__global__ void kernel_1(float * var_1_0, float * var_1_1, float * var_1_2, float * var_1_3, float * var_1_4, float * var_1_5, float * var_1_6, float * var_1_7, float * var_1_8, float * var_1_9) {
	__shared__ float myVar[1024];
	myVar[7] = 10.498290 * myVar[threadIdx.x];
	myVar[9] = 48.573285 * myVar[threadIdx.x];
	myVar[7] = 40.453661 * myVar[threadIdx.x];
	myVar[9] = 39.308308 * myVar[threadIdx.x];
	myVar[5] = 45.143141 * myVar[threadIdx.x];
	myVar[9] = 21.797042 * myVar[threadIdx.x];
	myVar[4] = 41.584658 * myVar[threadIdx.x];
	myVar[7] = 20.129875 * myVar[threadIdx.x];
	myVar[0] = 3.211955 * myVar[threadIdx.x];
	myVar[5] = 27.658041 * myVar[threadIdx.x];
	var_1_0[0] = myVar[0];
	var_1_1[1] = myVar[1];
	var_1_2[2] = myVar[2];
	var_1_3[3] = myVar[3];
	var_1_4[4] = myVar[4];
	var_1_5[5] = myVar[5];
	var_1_6[6] = myVar[6];
	var_1_7[7] = myVar[7];
	var_1_8[8] = myVar[8];
	var_1_9[9] = myVar[9];
	
}

__global__ void kernel_2(float * var_2_0, float * var_2_1, float * var_2_2, float * var_2_3, float * var_2_4, float * var_2_5, float * var_2_6, float * var_2_7, float * var_2_8, float * var_2_9) {
	__shared__ float myVar[1024];
	myVar[6] = 0.409518 * myVar[threadIdx.x];
	myVar[4] = 8.690601 * myVar[threadIdx.x];
	myVar[9] = 38.152384 * myVar[threadIdx.x];
	myVar[3] = 47.490931 * myVar[threadIdx.x];
	myVar[0] = 43.900222 * myVar[threadIdx.x];
	myVar[3] = 44.705960 * myVar[threadIdx.x];
	myVar[1] = 32.840704 * myVar[threadIdx.x];
	myVar[1] = 34.670584 * myVar[threadIdx.x];
	myVar[6] = 0.856672 * myVar[threadIdx.x];
	myVar[6] = 13.523540 * myVar[threadIdx.x];
	var_2_0[0] = myVar[0];
	var_2_1[1] = myVar[1];
	var_2_2[2] = myVar[2];
	var_2_3[3] = myVar[3];
	var_2_4[4] = myVar[4];
	var_2_5[5] = myVar[5];
	var_2_6[6] = myVar[6];
	var_2_7[7] = myVar[7];
	var_2_8[8] = myVar[8];
	var_2_9[9] = myVar[9];
	
}

__global__ void kernel_3(float * var_3_0, float * var_3_1, float * var_3_2, float * var_3_3, float * var_3_4, float * var_3_5, float * var_3_6, float * var_3_7, float * var_3_8, float * var_3_9) {
	__shared__ float myVar[1024];
	myVar[0] = 25.753676 * myVar[threadIdx.x];
	myVar[7] = 37.243147 * myVar[threadIdx.x];
	myVar[1] = 39.496028 * myVar[threadIdx.x];
	myVar[8] = 40.283404 * myVar[threadIdx.x];
	myVar[8] = 9.798944 * myVar[threadIdx.x];
	myVar[0] = 9.330874 * myVar[threadIdx.x];
	myVar[3] = 12.683786 * myVar[threadIdx.x];
	myVar[7] = 16.171586 * myVar[threadIdx.x];
	myVar[4] = 44.389549 * myVar[threadIdx.x];
	myVar[3] = 13.795409 * myVar[threadIdx.x];
	var_3_0[0] = myVar[0];
	var_3_1[1] = myVar[1];
	var_3_2[2] = myVar[2];
	var_3_3[3] = myVar[3];
	var_3_4[4] = myVar[4];
	var_3_5[5] = myVar[5];
	var_3_6[6] = myVar[6];
	var_3_7[7] = myVar[7];
	var_3_8[8] = myVar[8];
	var_3_9[9] = myVar[9];
	
}

__global__ void kernel_4(float * var_4_0, float * var_4_1, float * var_4_2, float * var_4_3, float * var_4_4, float * var_4_5, float * var_4_6, float * var_4_7, float * var_4_8, float * var_4_9) {
	__shared__ float myVar[1024];
	myVar[6] = 31.317364 * myVar[threadIdx.x];
	myVar[9] = 29.208017 * myVar[threadIdx.x];
	myVar[3] = 49.461131 * myVar[threadIdx.x];
	myVar[5] = 20.083331 * myVar[threadIdx.x];
	myVar[4] = 47.012036 * myVar[threadIdx.x];
	myVar[1] = 20.070240 * myVar[threadIdx.x];
	myVar[3] = 28.171519 * myVar[threadIdx.x];
	myVar[6] = 39.276712 * myVar[threadIdx.x];
	myVar[6] = 11.861790 * myVar[threadIdx.x];
	myVar[2] = 15.870188 * myVar[threadIdx.x];
	var_4_0[0] = myVar[0];
	var_4_1[1] = myVar[1];
	var_4_2[2] = myVar[2];
	var_4_3[3] = myVar[3];
	var_4_4[4] = myVar[4];
	var_4_5[5] = myVar[5];
	var_4_6[6] = myVar[6];
	var_4_7[7] = myVar[7];
	var_4_8[8] = myVar[8];
	var_4_9[9] = myVar[9];
	
}

__global__ void kernel_5(float * var_5_0, float * var_5_1, float * var_5_2, float * var_5_3, float * var_5_4, float * var_5_5, float * var_5_6, float * var_5_7, float * var_5_8, float * var_5_9) {
	__shared__ float myVar[1024];
	myVar[6] = 19.442937 * myVar[threadIdx.x];
	myVar[1] = 22.960740 * myVar[threadIdx.x];
	myVar[3] = 25.491718 * myVar[threadIdx.x];
	myVar[9] = 30.849896 * myVar[threadIdx.x];
	myVar[2] = 49.195293 * myVar[threadIdx.x];
	myVar[5] = 36.407166 * myVar[threadIdx.x];
	myVar[6] = 49.075702 * myVar[threadIdx.x];
	myVar[0] = 22.936021 * myVar[threadIdx.x];
	myVar[4] = 38.690914 * myVar[threadIdx.x];
	myVar[0] = 25.462527 * myVar[threadIdx.x];
	var_5_0[0] = myVar[0];
	var_5_1[1] = myVar[1];
	var_5_2[2] = myVar[2];
	var_5_3[3] = myVar[3];
	var_5_4[4] = myVar[4];
	var_5_5[5] = myVar[5];
	var_5_6[6] = myVar[6];
	var_5_7[7] = myVar[7];
	var_5_8[8] = myVar[8];
	var_5_9[9] = myVar[9];
	
}

__global__ void kernel_6(float * var_6_0, float * var_6_1, float * var_6_2, float * var_6_3, float * var_6_4, float * var_6_5, float * var_6_6, float * var_6_7, float * var_6_8, float * var_6_9) {
	__shared__ float myVar[1024];
	myVar[8] = 40.910830 * myVar[threadIdx.x];
	myVar[3] = 44.110014 * myVar[threadIdx.x];
	myVar[1] = 21.377741 * myVar[threadIdx.x];
	myVar[2] = 22.050224 * myVar[threadIdx.x];
	myVar[3] = 44.650340 * myVar[threadIdx.x];
	myVar[9] = 44.102511 * myVar[threadIdx.x];
	myVar[7] = 5.207397 * myVar[threadIdx.x];
	myVar[7] = 36.209409 * myVar[threadIdx.x];
	myVar[9] = 44.929571 * myVar[threadIdx.x];
	myVar[8] = 49.088663 * myVar[threadIdx.x];
	var_6_0[0] = myVar[0];
	var_6_1[1] = myVar[1];
	var_6_2[2] = myVar[2];
	var_6_3[3] = myVar[3];
	var_6_4[4] = myVar[4];
	var_6_5[5] = myVar[5];
	var_6_6[6] = myVar[6];
	var_6_7[7] = myVar[7];
	var_6_8[8] = myVar[8];
	var_6_9[9] = myVar[9];
	
}

__global__ void kernel_7(float * var_7_0, float * var_7_1, float * var_7_2, float * var_7_3, float * var_7_4, float * var_7_5, float * var_7_6, float * var_7_7, float * var_7_8, float * var_7_9) {
	__shared__ float myVar[1024];
	myVar[8] = 28.669346 * myVar[threadIdx.x];
	myVar[0] = 39.255807 * myVar[threadIdx.x];
	myVar[9] = 29.531385 * myVar[threadIdx.x];
	myVar[5] = 30.978964 * myVar[threadIdx.x];
	myVar[2] = 12.881451 * myVar[threadIdx.x];
	myVar[1] = 31.567788 * myVar[threadIdx.x];
	myVar[0] = 15.197734 * myVar[threadIdx.x];
	myVar[4] = 49.744884 * myVar[threadIdx.x];
	myVar[0] = 36.741280 * myVar[threadIdx.x];
	myVar[3] = 12.612324 * myVar[threadIdx.x];
	var_7_0[0] = myVar[0];
	var_7_1[1] = myVar[1];
	var_7_2[2] = myVar[2];
	var_7_3[3] = myVar[3];
	var_7_4[4] = myVar[4];
	var_7_5[5] = myVar[5];
	var_7_6[6] = myVar[6];
	var_7_7[7] = myVar[7];
	var_7_8[8] = myVar[8];
	var_7_9[9] = myVar[9];
	
}

__global__ void kernel_8(float * var_8_0, float * var_8_1, float * var_8_2, float * var_8_3, float * var_8_4, float * var_8_5, float * var_8_6, float * var_8_7, float * var_8_8, float * var_8_9) {
	__shared__ float myVar[1024];
	myVar[4] = 35.305747 * myVar[threadIdx.x];
	myVar[3] = 22.985684 * myVar[threadIdx.x];
	myVar[3] = 41.990318 * myVar[threadIdx.x];
	myVar[5] = 6.845127 * myVar[threadIdx.x];
	myVar[3] = 28.615930 * myVar[threadIdx.x];
	myVar[5] = 37.194092 * myVar[threadIdx.x];
	myVar[5] = 49.266076 * myVar[threadIdx.x];
	myVar[7] = 33.359126 * myVar[threadIdx.x];
	myVar[9] = 16.949092 * myVar[threadIdx.x];
	myVar[1] = 35.605362 * myVar[threadIdx.x];
	var_8_0[0] = myVar[0];
	var_8_1[1] = myVar[1];
	var_8_2[2] = myVar[2];
	var_8_3[3] = myVar[3];
	var_8_4[4] = myVar[4];
	var_8_5[5] = myVar[5];
	var_8_6[6] = myVar[6];
	var_8_7[7] = myVar[7];
	var_8_8[8] = myVar[8];
	var_8_9[9] = myVar[9];
	
}

__global__ void kernel_9(float * var_9_0, float * var_9_1, float * var_9_2, float * var_9_3, float * var_9_4, float * var_9_5, float * var_9_6, float * var_9_7, float * var_9_8, float * var_9_9) {
	__shared__ float myVar[1024];
	myVar[6] = 6.178335 * myVar[threadIdx.x];
	myVar[8] = 16.840668 * myVar[threadIdx.x];
	myVar[1] = 43.176823 * myVar[threadIdx.x];
	myVar[3] = 31.393618 * myVar[threadIdx.x];
	myVar[4] = 44.561645 * myVar[threadIdx.x];
	myVar[8] = 0.559624 * myVar[threadIdx.x];
	myVar[1] = 14.828130 * myVar[threadIdx.x];
	myVar[9] = 4.181657 * myVar[threadIdx.x];
	myVar[1] = 8.504773 * myVar[threadIdx.x];
	myVar[5] = 13.515462 * myVar[threadIdx.x];
	var_9_0[0] = myVar[0];
	var_9_1[1] = myVar[1];
	var_9_2[2] = myVar[2];
	var_9_3[3] = myVar[3];
	var_9_4[4] = myVar[4];
	var_9_5[5] = myVar[5];
	var_9_6[6] = myVar[6];
	var_9_7[7] = myVar[7];
	var_9_8[8] = myVar[8];
	var_9_9[9] = myVar[9];
	
}

__global__ void kernel_10(float * var_10_0, float * var_10_1, float * var_10_2, float * var_10_3, float * var_10_4, float * var_10_5, float * var_10_6, float * var_10_7, float * var_10_8, float * var_10_9) {
	__shared__ float myVar[1024];
	myVar[8] = 7.882892 * myVar[threadIdx.x];
	myVar[2] = 34.916764 * myVar[threadIdx.x];
	myVar[5] = 0.152698 * myVar[threadIdx.x];
	myVar[7] = 29.688767 * myVar[threadIdx.x];
	myVar[1] = 41.704095 * myVar[threadIdx.x];
	myVar[7] = 36.150542 * myVar[threadIdx.x];
	myVar[6] = 5.765721 * myVar[threadIdx.x];
	myVar[2] = 48.071225 * myVar[threadIdx.x];
	myVar[5] = 48.565510 * myVar[threadIdx.x];
	myVar[0] = 3.512245 * myVar[threadIdx.x];
	var_10_0[0] = myVar[0];
	var_10_1[1] = myVar[1];
	var_10_2[2] = myVar[2];
	var_10_3[3] = myVar[3];
	var_10_4[4] = myVar[4];
	var_10_5[5] = myVar[5];
	var_10_6[6] = myVar[6];
	var_10_7[7] = myVar[7];
	var_10_8[8] = myVar[8];
	var_10_9[9] = myVar[9];
	
}

__global__ void kernel_11(float * var_11_0, float * var_11_1, float * var_11_2, float * var_11_3, float * var_11_4, float * var_11_5, float * var_11_6, float * var_11_7, float * var_11_8, float * var_11_9) {
	__shared__ float myVar[1024];
	myVar[9] = 4.789338 * myVar[threadIdx.x];
	myVar[4] = 36.930728 * myVar[threadIdx.x];
	myVar[9] = 16.710979 * myVar[threadIdx.x];
	myVar[9] = 20.257562 * myVar[threadIdx.x];
	myVar[7] = 40.874004 * myVar[threadIdx.x];
	myVar[7] = 19.896021 * myVar[threadIdx.x];
	myVar[4] = 42.319657 * myVar[threadIdx.x];
	myVar[0] = 44.471235 * myVar[threadIdx.x];
	myVar[1] = 15.504836 * myVar[threadIdx.x];
	myVar[2] = 2.378930 * myVar[threadIdx.x];
	var_11_0[0] = myVar[0];
	var_11_1[1] = myVar[1];
	var_11_2[2] = myVar[2];
	var_11_3[3] = myVar[3];
	var_11_4[4] = myVar[4];
	var_11_5[5] = myVar[5];
	var_11_6[6] = myVar[6];
	var_11_7[7] = myVar[7];
	var_11_8[8] = myVar[8];
	var_11_9[9] = myVar[9];
	
}

__global__ void kernel_12(float * var_12_0, float * var_12_1, float * var_12_2, float * var_12_3, float * var_12_4, float * var_12_5, float * var_12_6, float * var_12_7, float * var_12_8, float * var_12_9) {
	__shared__ float myVar[1024];
	myVar[3] = 33.044254 * myVar[threadIdx.x];
	myVar[8] = 36.172918 * myVar[threadIdx.x];
	myVar[9] = 22.500201 * myVar[threadIdx.x];
	myVar[9] = 42.259863 * myVar[threadIdx.x];
	myVar[0] = 25.286195 * myVar[threadIdx.x];
	myVar[2] = 18.583546 * myVar[threadIdx.x];
	myVar[4] = 37.845654 * myVar[threadIdx.x];
	myVar[6] = 23.263653 * myVar[threadIdx.x];
	myVar[6] = 0.334531 * myVar[threadIdx.x];
	myVar[8] = 48.571792 * myVar[threadIdx.x];
	var_12_0[0] = myVar[0];
	var_12_1[1] = myVar[1];
	var_12_2[2] = myVar[2];
	var_12_3[3] = myVar[3];
	var_12_4[4] = myVar[4];
	var_12_5[5] = myVar[5];
	var_12_6[6] = myVar[6];
	var_12_7[7] = myVar[7];
	var_12_8[8] = myVar[8];
	var_12_9[9] = myVar[9];
	
}

__global__ void kernel_13(float * var_13_0, float * var_13_1, float * var_13_2, float * var_13_3, float * var_13_4, float * var_13_5, float * var_13_6, float * var_13_7, float * var_13_8, float * var_13_9) {
	__shared__ float myVar[1024];
	myVar[5] = 2.329164 * myVar[threadIdx.x];
	myVar[2] = 27.468162 * myVar[threadIdx.x];
	myVar[2] = 31.282914 * myVar[threadIdx.x];
	myVar[2] = 25.963372 * myVar[threadIdx.x];
	myVar[6] = 18.401266 * myVar[threadIdx.x];
	myVar[9] = 19.282669 * myVar[threadIdx.x];
	myVar[9] = 25.353553 * myVar[threadIdx.x];
	myVar[5] = 3.507961 * myVar[threadIdx.x];
	myVar[5] = 8.984620 * myVar[threadIdx.x];
	myVar[1] = 3.687348 * myVar[threadIdx.x];
	var_13_0[0] = myVar[0];
	var_13_1[1] = myVar[1];
	var_13_2[2] = myVar[2];
	var_13_3[3] = myVar[3];
	var_13_4[4] = myVar[4];
	var_13_5[5] = myVar[5];
	var_13_6[6] = myVar[6];
	var_13_7[7] = myVar[7];
	var_13_8[8] = myVar[8];
	var_13_9[9] = myVar[9];
	
}

__global__ void kernel_14(float * var_14_0, float * var_14_1, float * var_14_2, float * var_14_3, float * var_14_4, float * var_14_5, float * var_14_6, float * var_14_7, float * var_14_8, float * var_14_9) {
	__shared__ float myVar[1024];
	myVar[1] = 31.522549 * myVar[threadIdx.x];
	myVar[6] = 47.020477 * myVar[threadIdx.x];
	myVar[9] = 41.012974 * myVar[threadIdx.x];
	myVar[8] = 13.093407 * myVar[threadIdx.x];
	myVar[8] = 31.646809 * myVar[threadIdx.x];
	myVar[6] = 48.843660 * myVar[threadIdx.x];
	myVar[6] = 22.924309 * myVar[threadIdx.x];
	myVar[7] = 26.845918 * myVar[threadIdx.x];
	myVar[3] = 9.920997 * myVar[threadIdx.x];
	myVar[6] = 33.245486 * myVar[threadIdx.x];
	var_14_0[0] = myVar[0];
	var_14_1[1] = myVar[1];
	var_14_2[2] = myVar[2];
	var_14_3[3] = myVar[3];
	var_14_4[4] = myVar[4];
	var_14_5[5] = myVar[5];
	var_14_6[6] = myVar[6];
	var_14_7[7] = myVar[7];
	var_14_8[8] = myVar[8];
	var_14_9[9] = myVar[9];
	
}

__global__ void kernel_15(float * var_15_0, float * var_15_1, float * var_15_2, float * var_15_3, float * var_15_4, float * var_15_5, float * var_15_6, float * var_15_7, float * var_15_8, float * var_15_9) {
	__shared__ float myVar[1024];
	myVar[7] = 40.622161 * myVar[threadIdx.x];
	myVar[7] = 19.946032 * myVar[threadIdx.x];
	myVar[5] = 21.594293 * myVar[threadIdx.x];
	myVar[6] = 4.439377 * myVar[threadIdx.x];
	myVar[9] = 42.537210 * myVar[threadIdx.x];
	myVar[6] = 49.677309 * myVar[threadIdx.x];
	myVar[5] = 43.046848 * myVar[threadIdx.x];
	myVar[4] = 28.071790 * myVar[threadIdx.x];
	myVar[2] = 16.273522 * myVar[threadIdx.x];
	myVar[1] = 14.347631 * myVar[threadIdx.x];
	var_15_0[0] = myVar[0];
	var_15_1[1] = myVar[1];
	var_15_2[2] = myVar[2];
	var_15_3[3] = myVar[3];
	var_15_4[4] = myVar[4];
	var_15_5[5] = myVar[5];
	var_15_6[6] = myVar[6];
	var_15_7[7] = myVar[7];
	var_15_8[8] = myVar[8];
	var_15_9[9] = myVar[9];
	
}

__global__ void kernel_16(float * var_16_0, float * var_16_1, float * var_16_2, float * var_16_3, float * var_16_4, float * var_16_5, float * var_16_6, float * var_16_7, float * var_16_8, float * var_16_9) {
	__shared__ float myVar[1024];
	myVar[3] = 0.721046 * myVar[threadIdx.x];
	myVar[8] = 29.682274 * myVar[threadIdx.x];
	myVar[7] = 45.508895 * myVar[threadIdx.x];
	myVar[3] = 16.352109 * myVar[threadIdx.x];
	myVar[1] = 7.590435 * myVar[threadIdx.x];
	myVar[8] = 29.519743 * myVar[threadIdx.x];
	myVar[5] = 33.387906 * myVar[threadIdx.x];
	myVar[5] = 29.884450 * myVar[threadIdx.x];
	myVar[8] = 33.424171 * myVar[threadIdx.x];
	myVar[3] = 26.802417 * myVar[threadIdx.x];
	var_16_0[0] = myVar[0];
	var_16_1[1] = myVar[1];
	var_16_2[2] = myVar[2];
	var_16_3[3] = myVar[3];
	var_16_4[4] = myVar[4];
	var_16_5[5] = myVar[5];
	var_16_6[6] = myVar[6];
	var_16_7[7] = myVar[7];
	var_16_8[8] = myVar[8];
	var_16_9[9] = myVar[9];
	
}

__global__ void kernel_17(float * var_17_0, float * var_17_1, float * var_17_2, float * var_17_3, float * var_17_4, float * var_17_5, float * var_17_6, float * var_17_7, float * var_17_8, float * var_17_9) {
	__shared__ float myVar[1024];
	myVar[9] = 24.796966 * myVar[threadIdx.x];
	myVar[9] = 44.205041 * myVar[threadIdx.x];
	myVar[1] = 5.699130 * myVar[threadIdx.x];
	myVar[6] = 0.715713 * myVar[threadIdx.x];
	myVar[4] = 19.104383 * myVar[threadIdx.x];
	myVar[2] = 23.502298 * myVar[threadIdx.x];
	myVar[6] = 4.815490 * myVar[threadIdx.x];
	myVar[0] = 23.821100 * myVar[threadIdx.x];
	myVar[8] = 17.668747 * myVar[threadIdx.x];
	myVar[6] = 17.090312 * myVar[threadIdx.x];
	var_17_0[0] = myVar[0];
	var_17_1[1] = myVar[1];
	var_17_2[2] = myVar[2];
	var_17_3[3] = myVar[3];
	var_17_4[4] = myVar[4];
	var_17_5[5] = myVar[5];
	var_17_6[6] = myVar[6];
	var_17_7[7] = myVar[7];
	var_17_8[8] = myVar[8];
	var_17_9[9] = myVar[9];
	
}

__global__ void kernel_18(float * var_18_0, float * var_18_1, float * var_18_2, float * var_18_3, float * var_18_4, float * var_18_5, float * var_18_6, float * var_18_7, float * var_18_8, float * var_18_9) {
	__shared__ float myVar[1024];
	myVar[4] = 3.345577 * myVar[threadIdx.x];
	myVar[7] = 37.659292 * myVar[threadIdx.x];
	myVar[3] = 21.886083 * myVar[threadIdx.x];
	myVar[2] = 33.293481 * myVar[threadIdx.x];
	myVar[1] = 4.373278 * myVar[threadIdx.x];
	myVar[3] = 36.263312 * myVar[threadIdx.x];
	myVar[5] = 48.536571 * myVar[threadIdx.x];
	myVar[4] = 42.266164 * myVar[threadIdx.x];
	myVar[2] = 38.843900 * myVar[threadIdx.x];
	myVar[0] = 17.701493 * myVar[threadIdx.x];
	var_18_0[0] = myVar[0];
	var_18_1[1] = myVar[1];
	var_18_2[2] = myVar[2];
	var_18_3[3] = myVar[3];
	var_18_4[4] = myVar[4];
	var_18_5[5] = myVar[5];
	var_18_6[6] = myVar[6];
	var_18_7[7] = myVar[7];
	var_18_8[8] = myVar[8];
	var_18_9[9] = myVar[9];
	
}

__global__ void kernel_19(float * var_19_0, float * var_19_1, float * var_19_2, float * var_19_3, float * var_19_4, float * var_19_5, float * var_19_6, float * var_19_7, float * var_19_8, float * var_19_9) {
	__shared__ float myVar[1024];
	myVar[2] = 9.784314 * myVar[threadIdx.x];
	myVar[4] = 31.513838 * myVar[threadIdx.x];
	myVar[1] = 40.367282 * myVar[threadIdx.x];
	myVar[8] = 41.773924 * myVar[threadIdx.x];
	myVar[0] = 30.935291 * myVar[threadIdx.x];
	myVar[6] = 3.365774 * myVar[threadIdx.x];
	myVar[2] = 33.805888 * myVar[threadIdx.x];
	myVar[1] = 11.250243 * myVar[threadIdx.x];
	myVar[1] = 16.077158 * myVar[threadIdx.x];
	myVar[7] = 16.428407 * myVar[threadIdx.x];
	var_19_0[0] = myVar[0];
	var_19_1[1] = myVar[1];
	var_19_2[2] = myVar[2];
	var_19_3[3] = myVar[3];
	var_19_4[4] = myVar[4];
	var_19_5[5] = myVar[5];
	var_19_6[6] = myVar[6];
	var_19_7[7] = myVar[7];
	var_19_8[8] = myVar[8];
	var_19_9[9] = myVar[9];
	
}

__global__ void kernel_20(float * var_20_0, float * var_20_1, float * var_20_2, float * var_20_3, float * var_20_4, float * var_20_5, float * var_20_6, float * var_20_7, float * var_20_8, float * var_20_9) {
	__shared__ float myVar[1024];
	myVar[5] = 12.700576 * myVar[threadIdx.x];
	myVar[3] = 41.213208 * myVar[threadIdx.x];
	myVar[5] = 9.692584 * myVar[threadIdx.x];
	myVar[4] = 32.773133 * myVar[threadIdx.x];
	myVar[4] = 1.923570 * myVar[threadIdx.x];
	myVar[6] = 16.914192 * myVar[threadIdx.x];
	myVar[3] = 36.856016 * myVar[threadIdx.x];
	myVar[9] = 23.362504 * myVar[threadIdx.x];
	myVar[0] = 36.527513 * myVar[threadIdx.x];
	myVar[0] = 35.543537 * myVar[threadIdx.x];
	var_20_0[0] = myVar[0];
	var_20_1[1] = myVar[1];
	var_20_2[2] = myVar[2];
	var_20_3[3] = myVar[3];
	var_20_4[4] = myVar[4];
	var_20_5[5] = myVar[5];
	var_20_6[6] = myVar[6];
	var_20_7[7] = myVar[7];
	var_20_8[8] = myVar[8];
	var_20_9[9] = myVar[9];
	
}

__global__ void kernel_21(float * var_21_0, float * var_21_1, float * var_21_2, float * var_21_3, float * var_21_4, float * var_21_5, float * var_21_6, float * var_21_7, float * var_21_8, float * var_21_9) {
	__shared__ float myVar[1024];
	myVar[4] = 33.888910 * myVar[threadIdx.x];
	myVar[3] = 42.863040 * myVar[threadIdx.x];
	myVar[2] = 21.880141 * myVar[threadIdx.x];
	myVar[8] = 34.265026 * myVar[threadIdx.x];
	myVar[8] = 0.629872 * myVar[threadIdx.x];
	myVar[9] = 3.620120 * myVar[threadIdx.x];
	myVar[1] = 27.851501 * myVar[threadIdx.x];
	myVar[4] = 10.533276 * myVar[threadIdx.x];
	myVar[9] = 11.981865 * myVar[threadIdx.x];
	myVar[2] = 28.404232 * myVar[threadIdx.x];
	var_21_0[0] = myVar[0];
	var_21_1[1] = myVar[1];
	var_21_2[2] = myVar[2];
	var_21_3[3] = myVar[3];
	var_21_4[4] = myVar[4];
	var_21_5[5] = myVar[5];
	var_21_6[6] = myVar[6];
	var_21_7[7] = myVar[7];
	var_21_8[8] = myVar[8];
	var_21_9[9] = myVar[9];
	
}

__global__ void kernel_22(float * var_22_0, float * var_22_1, float * var_22_2, float * var_22_3, float * var_22_4, float * var_22_5, float * var_22_6, float * var_22_7, float * var_22_8, float * var_22_9) {
	__shared__ float myVar[1024];
	myVar[3] = 42.561043 * myVar[threadIdx.x];
	myVar[1] = 5.375239 * myVar[threadIdx.x];
	myVar[0] = 12.570711 * myVar[threadIdx.x];
	myVar[8] = 48.561617 * myVar[threadIdx.x];
	myVar[8] = 46.902203 * myVar[threadIdx.x];
	myVar[3] = 18.228843 * myVar[threadIdx.x];
	myVar[2] = 19.245598 * myVar[threadIdx.x];
	myVar[0] = 13.722794 * myVar[threadIdx.x];
	myVar[5] = 27.891088 * myVar[threadIdx.x];
	myVar[8] = 29.828456 * myVar[threadIdx.x];
	var_22_0[0] = myVar[0];
	var_22_1[1] = myVar[1];
	var_22_2[2] = myVar[2];
	var_22_3[3] = myVar[3];
	var_22_4[4] = myVar[4];
	var_22_5[5] = myVar[5];
	var_22_6[6] = myVar[6];
	var_22_7[7] = myVar[7];
	var_22_8[8] = myVar[8];
	var_22_9[9] = myVar[9];
	
}

__global__ void kernel_23(float * var_23_0, float * var_23_1, float * var_23_2, float * var_23_3, float * var_23_4, float * var_23_5, float * var_23_6, float * var_23_7, float * var_23_8, float * var_23_9) {
	__shared__ float myVar[1024];
	myVar[6] = 3.447684 * myVar[threadIdx.x];
	myVar[9] = 3.640567 * myVar[threadIdx.x];
	myVar[3] = 49.837136 * myVar[threadIdx.x];
	myVar[4] = 10.540101 * myVar[threadIdx.x];
	myVar[1] = 8.881791 * myVar[threadIdx.x];
	myVar[1] = 22.540074 * myVar[threadIdx.x];
	myVar[7] = 38.443411 * myVar[threadIdx.x];
	myVar[6] = 25.384029 * myVar[threadIdx.x];
	myVar[3] = 48.160367 * myVar[threadIdx.x];
	myVar[9] = 13.455500 * myVar[threadIdx.x];
	var_23_0[0] = myVar[0];
	var_23_1[1] = myVar[1];
	var_23_2[2] = myVar[2];
	var_23_3[3] = myVar[3];
	var_23_4[4] = myVar[4];
	var_23_5[5] = myVar[5];
	var_23_6[6] = myVar[6];
	var_23_7[7] = myVar[7];
	var_23_8[8] = myVar[8];
	var_23_9[9] = myVar[9];
	
}

__global__ void kernel_24(float * var_24_0, float * var_24_1, float * var_24_2, float * var_24_3, float * var_24_4, float * var_24_5, float * var_24_6, float * var_24_7, float * var_24_8, float * var_24_9) {
	__shared__ float myVar[1024];
	myVar[4] = 15.322267 * myVar[threadIdx.x];
	myVar[9] = 18.603918 * myVar[threadIdx.x];
	myVar[0] = 35.903317 * myVar[threadIdx.x];
	myVar[9] = 31.488988 * myVar[threadIdx.x];
	myVar[7] = 3.880680 * myVar[threadIdx.x];
	myVar[9] = 17.495960 * myVar[threadIdx.x];
	myVar[5] = 6.304608 * myVar[threadIdx.x];
	myVar[2] = 20.825722 * myVar[threadIdx.x];
	myVar[9] = 10.871806 * myVar[threadIdx.x];
	myVar[1] = 6.955122 * myVar[threadIdx.x];
	var_24_0[0] = myVar[0];
	var_24_1[1] = myVar[1];
	var_24_2[2] = myVar[2];
	var_24_3[3] = myVar[3];
	var_24_4[4] = myVar[4];
	var_24_5[5] = myVar[5];
	var_24_6[6] = myVar[6];
	var_24_7[7] = myVar[7];
	var_24_8[8] = myVar[8];
	var_24_9[9] = myVar[9];
	
}

__global__ void kernel_25(float * var_25_0, float * var_25_1, float * var_25_2, float * var_25_3, float * var_25_4, float * var_25_5, float * var_25_6, float * var_25_7, float * var_25_8, float * var_25_9) {
	__shared__ float myVar[1024];
	myVar[0] = 28.724806 * myVar[threadIdx.x];
	myVar[4] = 4.714734 * myVar[threadIdx.x];
	myVar[5] = 46.098177 * myVar[threadIdx.x];
	myVar[0] = 44.913593 * myVar[threadIdx.x];
	myVar[6] = 47.402892 * myVar[threadIdx.x];
	myVar[4] = 2.686918 * myVar[threadIdx.x];
	myVar[5] = 35.819338 * myVar[threadIdx.x];
	myVar[2] = 7.862932 * myVar[threadIdx.x];
	myVar[2] = 32.582440 * myVar[threadIdx.x];
	myVar[1] = 45.237856 * myVar[threadIdx.x];
	var_25_0[0] = myVar[0];
	var_25_1[1] = myVar[1];
	var_25_2[2] = myVar[2];
	var_25_3[3] = myVar[3];
	var_25_4[4] = myVar[4];
	var_25_5[5] = myVar[5];
	var_25_6[6] = myVar[6];
	var_25_7[7] = myVar[7];
	var_25_8[8] = myVar[8];
	var_25_9[9] = myVar[9];
	
}

__global__ void kernel_26(float * var_26_0, float * var_26_1, float * var_26_2, float * var_26_3, float * var_26_4, float * var_26_5, float * var_26_6, float * var_26_7, float * var_26_8, float * var_26_9) {
	__shared__ float myVar[1024];
	myVar[6] = 13.791539 * myVar[threadIdx.x];
	myVar[1] = 48.158467 * myVar[threadIdx.x];
	myVar[7] = 29.527633 * myVar[threadIdx.x];
	myVar[6] = 19.880055 * myVar[threadIdx.x];
	myVar[8] = 25.291974 * myVar[threadIdx.x];
	myVar[8] = 46.013702 * myVar[threadIdx.x];
	myVar[2] = 25.500842 * myVar[threadIdx.x];
	myVar[9] = 15.082102 * myVar[threadIdx.x];
	myVar[3] = 49.431983 * myVar[threadIdx.x];
	myVar[8] = 37.754153 * myVar[threadIdx.x];
	var_26_0[0] = myVar[0];
	var_26_1[1] = myVar[1];
	var_26_2[2] = myVar[2];
	var_26_3[3] = myVar[3];
	var_26_4[4] = myVar[4];
	var_26_5[5] = myVar[5];
	var_26_6[6] = myVar[6];
	var_26_7[7] = myVar[7];
	var_26_8[8] = myVar[8];
	var_26_9[9] = myVar[9];
	
}

__global__ void kernel_27(float * var_27_0, float * var_27_1, float * var_27_2, float * var_27_3, float * var_27_4, float * var_27_5, float * var_27_6, float * var_27_7, float * var_27_8, float * var_27_9) {
	__shared__ float myVar[1024];
	myVar[2] = 49.098625 * myVar[threadIdx.x];
	myVar[6] = 15.201905 * myVar[threadIdx.x];
	myVar[6] = 8.158726 * myVar[threadIdx.x];
	myVar[1] = 29.073564 * myVar[threadIdx.x];
	myVar[4] = 22.325693 * myVar[threadIdx.x];
	myVar[9] = 15.360091 * myVar[threadIdx.x];
	myVar[9] = 31.617193 * myVar[threadIdx.x];
	myVar[9] = 26.942423 * myVar[threadIdx.x];
	myVar[7] = 4.814359 * myVar[threadIdx.x];
	myVar[9] = 8.658239 * myVar[threadIdx.x];
	var_27_0[0] = myVar[0];
	var_27_1[1] = myVar[1];
	var_27_2[2] = myVar[2];
	var_27_3[3] = myVar[3];
	var_27_4[4] = myVar[4];
	var_27_5[5] = myVar[5];
	var_27_6[6] = myVar[6];
	var_27_7[7] = myVar[7];
	var_27_8[8] = myVar[8];
	var_27_9[9] = myVar[9];
	
}

__global__ void kernel_28(float * var_28_0, float * var_28_1, float * var_28_2, float * var_28_3, float * var_28_4, float * var_28_5, float * var_28_6, float * var_28_7, float * var_28_8, float * var_28_9) {
	__shared__ float myVar[1024];
	myVar[9] = 32.363108 * myVar[threadIdx.x];
	myVar[0] = 26.294276 * myVar[threadIdx.x];
	myVar[3] = 4.833897 * myVar[threadIdx.x];
	myVar[8] = 27.514344 * myVar[threadIdx.x];
	myVar[4] = 10.111906 * myVar[threadIdx.x];
	myVar[6] = 27.445743 * myVar[threadIdx.x];
	myVar[0] = 20.299435 * myVar[threadIdx.x];
	myVar[3] = 32.876743 * myVar[threadIdx.x];
	myVar[4] = 21.097365 * myVar[threadIdx.x];
	myVar[2] = 29.296228 * myVar[threadIdx.x];
	var_28_0[0] = myVar[0];
	var_28_1[1] = myVar[1];
	var_28_2[2] = myVar[2];
	var_28_3[3] = myVar[3];
	var_28_4[4] = myVar[4];
	var_28_5[5] = myVar[5];
	var_28_6[6] = myVar[6];
	var_28_7[7] = myVar[7];
	var_28_8[8] = myVar[8];
	var_28_9[9] = myVar[9];
	
}

__global__ void kernel_29(float * var_29_0, float * var_29_1, float * var_29_2, float * var_29_3, float * var_29_4, float * var_29_5, float * var_29_6, float * var_29_7, float * var_29_8, float * var_29_9) {
	__shared__ float myVar[1024];
	myVar[4] = 3.609021 * myVar[threadIdx.x];
	myVar[9] = 33.072502 * myVar[threadIdx.x];
	myVar[3] = 31.548664 * myVar[threadIdx.x];
	myVar[1] = 49.996068 * myVar[threadIdx.x];
	myVar[5] = 29.369450 * myVar[threadIdx.x];
	myVar[6] = 3.544996 * myVar[threadIdx.x];
	myVar[1] = 2.933794 * myVar[threadIdx.x];
	myVar[3] = 40.278806 * myVar[threadIdx.x];
	myVar[3] = 39.109933 * myVar[threadIdx.x];
	myVar[3] = 49.954560 * myVar[threadIdx.x];
	var_29_0[0] = myVar[0];
	var_29_1[1] = myVar[1];
	var_29_2[2] = myVar[2];
	var_29_3[3] = myVar[3];
	var_29_4[4] = myVar[4];
	var_29_5[5] = myVar[5];
	var_29_6[6] = myVar[6];
	var_29_7[7] = myVar[7];
	var_29_8[8] = myVar[8];
	var_29_9[9] = myVar[9];
	
}

__global__ void kernel_30(float * var_30_0, float * var_30_1, float * var_30_2, float * var_30_3, float * var_30_4, float * var_30_5, float * var_30_6, float * var_30_7, float * var_30_8, float * var_30_9) {
	__shared__ float myVar[1024];
	myVar[5] = 37.387502 * myVar[threadIdx.x];
	myVar[6] = 2.144567 * myVar[threadIdx.x];
	myVar[7] = 23.431729 * myVar[threadIdx.x];
	myVar[5] = 17.514979 * myVar[threadIdx.x];
	myVar[0] = 24.510694 * myVar[threadIdx.x];
	myVar[0] = 6.914233 * myVar[threadIdx.x];
	myVar[1] = 43.205739 * myVar[threadIdx.x];
	myVar[7] = 40.285453 * myVar[threadIdx.x];
	myVar[5] = 19.025275 * myVar[threadIdx.x];
	myVar[3] = 8.837556 * myVar[threadIdx.x];
	var_30_0[0] = myVar[0];
	var_30_1[1] = myVar[1];
	var_30_2[2] = myVar[2];
	var_30_3[3] = myVar[3];
	var_30_4[4] = myVar[4];
	var_30_5[5] = myVar[5];
	var_30_6[6] = myVar[6];
	var_30_7[7] = myVar[7];
	var_30_8[8] = myVar[8];
	var_30_9[9] = myVar[9];
	
}

__global__ void kernel_31(float * var_31_0, float * var_31_1, float * var_31_2, float * var_31_3, float * var_31_4, float * var_31_5, float * var_31_6, float * var_31_7, float * var_31_8, float * var_31_9) {
	__shared__ float myVar[1024];
	myVar[7] = 12.833705 * myVar[threadIdx.x];
	myVar[1] = 28.441507 * myVar[threadIdx.x];
	myVar[7] = 13.415374 * myVar[threadIdx.x];
	myVar[2] = 8.406161 * myVar[threadIdx.x];
	myVar[5] = 17.512429 * myVar[threadIdx.x];
	myVar[6] = 42.466597 * myVar[threadIdx.x];
	myVar[5] = 34.852249 * myVar[threadIdx.x];
	myVar[0] = 39.042674 * myVar[threadIdx.x];
	myVar[5] = 32.211152 * myVar[threadIdx.x];
	myVar[4] = 31.657029 * myVar[threadIdx.x];
	var_31_0[0] = myVar[0];
	var_31_1[1] = myVar[1];
	var_31_2[2] = myVar[2];
	var_31_3[3] = myVar[3];
	var_31_4[4] = myVar[4];
	var_31_5[5] = myVar[5];
	var_31_6[6] = myVar[6];
	var_31_7[7] = myVar[7];
	var_31_8[8] = myVar[8];
	var_31_9[9] = myVar[9];
	
}

__global__ void kernel_32(float * var_32_0, float * var_32_1, float * var_32_2, float * var_32_3, float * var_32_4, float * var_32_5, float * var_32_6, float * var_32_7, float * var_32_8, float * var_32_9) {
	__shared__ float myVar[1024];
	myVar[6] = 33.097889 * myVar[threadIdx.x];
	myVar[0] = 8.568342 * myVar[threadIdx.x];
	myVar[3] = 37.312544 * myVar[threadIdx.x];
	myVar[8] = 2.731467 * myVar[threadIdx.x];
	myVar[0] = 7.537503 * myVar[threadIdx.x];
	myVar[2] = 31.249875 * myVar[threadIdx.x];
	myVar[4] = 15.036837 * myVar[threadIdx.x];
	myVar[3] = 1.455600 * myVar[threadIdx.x];
	myVar[6] = 20.962517 * myVar[threadIdx.x];
	myVar[7] = 11.834914 * myVar[threadIdx.x];
	var_32_0[0] = myVar[0];
	var_32_1[1] = myVar[1];
	var_32_2[2] = myVar[2];
	var_32_3[3] = myVar[3];
	var_32_4[4] = myVar[4];
	var_32_5[5] = myVar[5];
	var_32_6[6] = myVar[6];
	var_32_7[7] = myVar[7];
	var_32_8[8] = myVar[8];
	var_32_9[9] = myVar[9];
	
}

__global__ void kernel_33(float * var_33_0, float * var_33_1, float * var_33_2, float * var_33_3, float * var_33_4, float * var_33_5, float * var_33_6, float * var_33_7, float * var_33_8, float * var_33_9) {
	__shared__ float myVar[1024];
	myVar[1] = 49.841332 * myVar[threadIdx.x];
	myVar[8] = 20.504063 * myVar[threadIdx.x];
	myVar[9] = 41.076575 * myVar[threadIdx.x];
	myVar[6] = 21.032054 * myVar[threadIdx.x];
	myVar[0] = 40.220464 * myVar[threadIdx.x];
	myVar[7] = 9.936741 * myVar[threadIdx.x];
	myVar[8] = 41.653157 * myVar[threadIdx.x];
	myVar[0] = 11.531191 * myVar[threadIdx.x];
	myVar[8] = 17.733310 * myVar[threadIdx.x];
	myVar[3] = 22.221154 * myVar[threadIdx.x];
	var_33_0[0] = myVar[0];
	var_33_1[1] = myVar[1];
	var_33_2[2] = myVar[2];
	var_33_3[3] = myVar[3];
	var_33_4[4] = myVar[4];
	var_33_5[5] = myVar[5];
	var_33_6[6] = myVar[6];
	var_33_7[7] = myVar[7];
	var_33_8[8] = myVar[8];
	var_33_9[9] = myVar[9];
	
}

__global__ void kernel_34(float * var_34_0, float * var_34_1, float * var_34_2, float * var_34_3, float * var_34_4, float * var_34_5, float * var_34_6, float * var_34_7, float * var_34_8, float * var_34_9) {
	__shared__ float myVar[1024];
	myVar[9] = 40.016089 * myVar[threadIdx.x];
	myVar[0] = 7.827281 * myVar[threadIdx.x];
	myVar[5] = 47.266293 * myVar[threadIdx.x];
	myVar[5] = 30.054875 * myVar[threadIdx.x];
	myVar[2] = 39.705856 * myVar[threadIdx.x];
	myVar[0] = 26.049503 * myVar[threadIdx.x];
	myVar[2] = 7.311032 * myVar[threadIdx.x];
	myVar[7] = 26.354148 * myVar[threadIdx.x];
	myVar[5] = 7.888674 * myVar[threadIdx.x];
	myVar[8] = 30.327730 * myVar[threadIdx.x];
	var_34_0[0] = myVar[0];
	var_34_1[1] = myVar[1];
	var_34_2[2] = myVar[2];
	var_34_3[3] = myVar[3];
	var_34_4[4] = myVar[4];
	var_34_5[5] = myVar[5];
	var_34_6[6] = myVar[6];
	var_34_7[7] = myVar[7];
	var_34_8[8] = myVar[8];
	var_34_9[9] = myVar[9];
	
}

__global__ void kernel_35(float * var_35_0, float * var_35_1, float * var_35_2, float * var_35_3, float * var_35_4, float * var_35_5, float * var_35_6, float * var_35_7, float * var_35_8, float * var_35_9) {
	__shared__ float myVar[1024];
	myVar[4] = 4.548860 * myVar[threadIdx.x];
	myVar[0] = 48.596612 * myVar[threadIdx.x];
	myVar[2] = 19.079956 * myVar[threadIdx.x];
	myVar[8] = 31.379061 * myVar[threadIdx.x];
	myVar[3] = 18.408245 * myVar[threadIdx.x];
	myVar[7] = 29.562698 * myVar[threadIdx.x];
	myVar[5] = 11.288055 * myVar[threadIdx.x];
	myVar[3] = 28.110632 * myVar[threadIdx.x];
	myVar[3] = 18.387617 * myVar[threadIdx.x];
	myVar[1] = 36.910730 * myVar[threadIdx.x];
	var_35_0[0] = myVar[0];
	var_35_1[1] = myVar[1];
	var_35_2[2] = myVar[2];
	var_35_3[3] = myVar[3];
	var_35_4[4] = myVar[4];
	var_35_5[5] = myVar[5];
	var_35_6[6] = myVar[6];
	var_35_7[7] = myVar[7];
	var_35_8[8] = myVar[8];
	var_35_9[9] = myVar[9];
	
}

__global__ void kernel_36(float * var_36_0, float * var_36_1, float * var_36_2, float * var_36_3, float * var_36_4, float * var_36_5, float * var_36_6, float * var_36_7, float * var_36_8, float * var_36_9) {
	__shared__ float myVar[1024];
	myVar[4] = 38.709092 * myVar[threadIdx.x];
	myVar[1] = 17.538780 * myVar[threadIdx.x];
	myVar[3] = 27.188513 * myVar[threadIdx.x];
	myVar[2] = 19.507238 * myVar[threadIdx.x];
	myVar[9] = 42.973725 * myVar[threadIdx.x];
	myVar[6] = 30.387322 * myVar[threadIdx.x];
	myVar[9] = 11.370702 * myVar[threadIdx.x];
	myVar[2] = 20.046934 * myVar[threadIdx.x];
	myVar[3] = 23.269483 * myVar[threadIdx.x];
	myVar[7] = 42.634197 * myVar[threadIdx.x];
	var_36_0[0] = myVar[0];
	var_36_1[1] = myVar[1];
	var_36_2[2] = myVar[2];
	var_36_3[3] = myVar[3];
	var_36_4[4] = myVar[4];
	var_36_5[5] = myVar[5];
	var_36_6[6] = myVar[6];
	var_36_7[7] = myVar[7];
	var_36_8[8] = myVar[8];
	var_36_9[9] = myVar[9];
	
}

__global__ void kernel_37(float * var_37_0, float * var_37_1, float * var_37_2, float * var_37_3, float * var_37_4, float * var_37_5, float * var_37_6, float * var_37_7, float * var_37_8, float * var_37_9) {
	__shared__ float myVar[1024];
	myVar[9] = 5.358901 * myVar[threadIdx.x];
	myVar[6] = 37.276176 * myVar[threadIdx.x];
	myVar[7] = 38.499256 * myVar[threadIdx.x];
	myVar[5] = 0.677148 * myVar[threadIdx.x];
	myVar[2] = 17.141034 * myVar[threadIdx.x];
	myVar[5] = 5.427960 * myVar[threadIdx.x];
	myVar[9] = 5.819996 * myVar[threadIdx.x];
	myVar[4] = 24.209951 * myVar[threadIdx.x];
	myVar[2] = 45.153299 * myVar[threadIdx.x];
	myVar[6] = 13.056218 * myVar[threadIdx.x];
	var_37_0[0] = myVar[0];
	var_37_1[1] = myVar[1];
	var_37_2[2] = myVar[2];
	var_37_3[3] = myVar[3];
	var_37_4[4] = myVar[4];
	var_37_5[5] = myVar[5];
	var_37_6[6] = myVar[6];
	var_37_7[7] = myVar[7];
	var_37_8[8] = myVar[8];
	var_37_9[9] = myVar[9];
	
}

__global__ void kernel_38(float * var_38_0, float * var_38_1, float * var_38_2, float * var_38_3, float * var_38_4, float * var_38_5, float * var_38_6, float * var_38_7, float * var_38_8, float * var_38_9) {
	__shared__ float myVar[1024];
	myVar[8] = 5.460291 * myVar[threadIdx.x];
	myVar[1] = 25.222137 * myVar[threadIdx.x];
	myVar[5] = 17.176304 * myVar[threadIdx.x];
	myVar[7] = 28.634038 * myVar[threadIdx.x];
	myVar[6] = 23.609900 * myVar[threadIdx.x];
	myVar[3] = 41.332861 * myVar[threadIdx.x];
	myVar[8] = 29.642004 * myVar[threadIdx.x];
	myVar[7] = 19.468654 * myVar[threadIdx.x];
	myVar[1] = 38.410628 * myVar[threadIdx.x];
	myVar[8] = 24.252108 * myVar[threadIdx.x];
	var_38_0[0] = myVar[0];
	var_38_1[1] = myVar[1];
	var_38_2[2] = myVar[2];
	var_38_3[3] = myVar[3];
	var_38_4[4] = myVar[4];
	var_38_5[5] = myVar[5];
	var_38_6[6] = myVar[6];
	var_38_7[7] = myVar[7];
	var_38_8[8] = myVar[8];
	var_38_9[9] = myVar[9];
	
}

__global__ void kernel_39(float * var_39_0, float * var_39_1, float * var_39_2, float * var_39_3, float * var_39_4, float * var_39_5, float * var_39_6, float * var_39_7, float * var_39_8, float * var_39_9) {
	__shared__ float myVar[1024];
	myVar[7] = 4.699386 * myVar[threadIdx.x];
	myVar[4] = 42.780262 * myVar[threadIdx.x];
	myVar[2] = 46.730611 * myVar[threadIdx.x];
	myVar[1] = 17.028525 * myVar[threadIdx.x];
	myVar[2] = 26.071464 * myVar[threadIdx.x];
	myVar[3] = 1.573222 * myVar[threadIdx.x];
	myVar[6] = 43.866070 * myVar[threadIdx.x];
	myVar[3] = 39.808741 * myVar[threadIdx.x];
	myVar[0] = 10.624138 * myVar[threadIdx.x];
	myVar[6] = 46.929066 * myVar[threadIdx.x];
	var_39_0[0] = myVar[0];
	var_39_1[1] = myVar[1];
	var_39_2[2] = myVar[2];
	var_39_3[3] = myVar[3];
	var_39_4[4] = myVar[4];
	var_39_5[5] = myVar[5];
	var_39_6[6] = myVar[6];
	var_39_7[7] = myVar[7];
	var_39_8[8] = myVar[8];
	var_39_9[9] = myVar[9];
	
}

__global__ void kernel_40(float * var_40_0, float * var_40_1, float * var_40_2, float * var_40_3, float * var_40_4, float * var_40_5, float * var_40_6, float * var_40_7, float * var_40_8, float * var_40_9) {
	__shared__ float myVar[1024];
	myVar[5] = 28.462100 * myVar[threadIdx.x];
	myVar[5] = 16.902711 * myVar[threadIdx.x];
	myVar[2] = 24.259712 * myVar[threadIdx.x];
	myVar[0] = 34.166913 * myVar[threadIdx.x];
	myVar[4] = 49.967410 * myVar[threadIdx.x];
	myVar[7] = 49.559763 * myVar[threadIdx.x];
	myVar[9] = 25.396087 * myVar[threadIdx.x];
	myVar[4] = 19.431114 * myVar[threadIdx.x];
	myVar[7] = 27.760430 * myVar[threadIdx.x];
	myVar[4] = 5.094379 * myVar[threadIdx.x];
	var_40_0[0] = myVar[0];
	var_40_1[1] = myVar[1];
	var_40_2[2] = myVar[2];
	var_40_3[3] = myVar[3];
	var_40_4[4] = myVar[4];
	var_40_5[5] = myVar[5];
	var_40_6[6] = myVar[6];
	var_40_7[7] = myVar[7];
	var_40_8[8] = myVar[8];
	var_40_9[9] = myVar[9];
	
}

__global__ void kernel_41(float * var_41_0, float * var_41_1, float * var_41_2, float * var_41_3, float * var_41_4, float * var_41_5, float * var_41_6, float * var_41_7, float * var_41_8, float * var_41_9) {
	__shared__ float myVar[1024];
	myVar[2] = 22.832298 * myVar[threadIdx.x];
	myVar[1] = 32.084364 * myVar[threadIdx.x];
	myVar[7] = 26.671853 * myVar[threadIdx.x];
	myVar[8] = 7.974848 * myVar[threadIdx.x];
	myVar[2] = 29.369853 * myVar[threadIdx.x];
	myVar[8] = 32.925229 * myVar[threadIdx.x];
	myVar[5] = 28.874093 * myVar[threadIdx.x];
	myVar[1] = 29.357745 * myVar[threadIdx.x];
	myVar[2] = 30.595407 * myVar[threadIdx.x];
	myVar[0] = 46.058006 * myVar[threadIdx.x];
	var_41_0[0] = myVar[0];
	var_41_1[1] = myVar[1];
	var_41_2[2] = myVar[2];
	var_41_3[3] = myVar[3];
	var_41_4[4] = myVar[4];
	var_41_5[5] = myVar[5];
	var_41_6[6] = myVar[6];
	var_41_7[7] = myVar[7];
	var_41_8[8] = myVar[8];
	var_41_9[9] = myVar[9];
	
}

__global__ void kernel_42(float * var_42_0, float * var_42_1, float * var_42_2, float * var_42_3, float * var_42_4, float * var_42_5, float * var_42_6, float * var_42_7, float * var_42_8, float * var_42_9) {
	__shared__ float myVar[1024];
	myVar[2] = 3.880864 * myVar[threadIdx.x];
	myVar[4] = 35.747074 * myVar[threadIdx.x];
	myVar[2] = 15.077994 * myVar[threadIdx.x];
	myVar[4] = 7.648367 * myVar[threadIdx.x];
	myVar[3] = 48.654527 * myVar[threadIdx.x];
	myVar[2] = 22.623383 * myVar[threadIdx.x];
	myVar[2] = 47.879960 * myVar[threadIdx.x];
	myVar[2] = 5.522035 * myVar[threadIdx.x];
	myVar[5] = 1.406982 * myVar[threadIdx.x];
	myVar[3] = 32.108976 * myVar[threadIdx.x];
	var_42_0[0] = myVar[0];
	var_42_1[1] = myVar[1];
	var_42_2[2] = myVar[2];
	var_42_3[3] = myVar[3];
	var_42_4[4] = myVar[4];
	var_42_5[5] = myVar[5];
	var_42_6[6] = myVar[6];
	var_42_7[7] = myVar[7];
	var_42_8[8] = myVar[8];
	var_42_9[9] = myVar[9];
	
}

__global__ void kernel_43(float * var_43_0, float * var_43_1, float * var_43_2, float * var_43_3, float * var_43_4, float * var_43_5, float * var_43_6, float * var_43_7, float * var_43_8, float * var_43_9) {
	__shared__ float myVar[1024];
	myVar[9] = 48.492659 * myVar[threadIdx.x];
	myVar[8] = 23.671270 * myVar[threadIdx.x];
	myVar[8] = 38.490300 * myVar[threadIdx.x];
	myVar[4] = 2.131732 * myVar[threadIdx.x];
	myVar[1] = 36.505205 * myVar[threadIdx.x];
	myVar[8] = 39.658574 * myVar[threadIdx.x];
	myVar[5] = 6.777877 * myVar[threadIdx.x];
	myVar[1] = 27.597590 * myVar[threadIdx.x];
	myVar[4] = 10.845351 * myVar[threadIdx.x];
	myVar[5] = 24.901491 * myVar[threadIdx.x];
	var_43_0[0] = myVar[0];
	var_43_1[1] = myVar[1];
	var_43_2[2] = myVar[2];
	var_43_3[3] = myVar[3];
	var_43_4[4] = myVar[4];
	var_43_5[5] = myVar[5];
	var_43_6[6] = myVar[6];
	var_43_7[7] = myVar[7];
	var_43_8[8] = myVar[8];
	var_43_9[9] = myVar[9];
	
}

__global__ void kernel_44(float * var_44_0, float * var_44_1, float * var_44_2, float * var_44_3, float * var_44_4, float * var_44_5, float * var_44_6, float * var_44_7, float * var_44_8, float * var_44_9) {
	__shared__ float myVar[1024];
	myVar[8] = 28.626900 * myVar[threadIdx.x];
	myVar[1] = 15.559386 * myVar[threadIdx.x];
	myVar[9] = 13.209298 * myVar[threadIdx.x];
	myVar[6] = 37.720059 * myVar[threadIdx.x];
	myVar[7] = 17.716526 * myVar[threadIdx.x];
	myVar[3] = 4.130992 * myVar[threadIdx.x];
	myVar[5] = 22.501120 * myVar[threadIdx.x];
	myVar[0] = 26.947997 * myVar[threadIdx.x];
	myVar[3] = 23.235711 * myVar[threadIdx.x];
	myVar[0] = 1.034861 * myVar[threadIdx.x];
	var_44_0[0] = myVar[0];
	var_44_1[1] = myVar[1];
	var_44_2[2] = myVar[2];
	var_44_3[3] = myVar[3];
	var_44_4[4] = myVar[4];
	var_44_5[5] = myVar[5];
	var_44_6[6] = myVar[6];
	var_44_7[7] = myVar[7];
	var_44_8[8] = myVar[8];
	var_44_9[9] = myVar[9];
	
}

__global__ void kernel_45(float * var_45_0, float * var_45_1, float * var_45_2, float * var_45_3, float * var_45_4, float * var_45_5, float * var_45_6, float * var_45_7, float * var_45_8, float * var_45_9) {
	__shared__ float myVar[1024];
	myVar[6] = 16.404295 * myVar[threadIdx.x];
	myVar[2] = 4.287836 * myVar[threadIdx.x];
	myVar[0] = 6.790351 * myVar[threadIdx.x];
	myVar[5] = 8.390972 * myVar[threadIdx.x];
	myVar[5] = 35.574646 * myVar[threadIdx.x];
	myVar[9] = 29.376300 * myVar[threadIdx.x];
	myVar[5] = 24.313347 * myVar[threadIdx.x];
	myVar[1] = 46.508907 * myVar[threadIdx.x];
	myVar[4] = 10.751607 * myVar[threadIdx.x];
	myVar[5] = 13.335187 * myVar[threadIdx.x];
	var_45_0[0] = myVar[0];
	var_45_1[1] = myVar[1];
	var_45_2[2] = myVar[2];
	var_45_3[3] = myVar[3];
	var_45_4[4] = myVar[4];
	var_45_5[5] = myVar[5];
	var_45_6[6] = myVar[6];
	var_45_7[7] = myVar[7];
	var_45_8[8] = myVar[8];
	var_45_9[9] = myVar[9];
	
}

__global__ void kernel_46(float * var_46_0, float * var_46_1, float * var_46_2, float * var_46_3, float * var_46_4, float * var_46_5, float * var_46_6, float * var_46_7, float * var_46_8, float * var_46_9) {
	__shared__ float myVar[1024];
	myVar[6] = 6.164319 * myVar[threadIdx.x];
	myVar[3] = 39.749101 * myVar[threadIdx.x];
	myVar[1] = 32.019275 * myVar[threadIdx.x];
	myVar[4] = 22.489652 * myVar[threadIdx.x];
	myVar[4] = 24.629295 * myVar[threadIdx.x];
	myVar[6] = 6.320353 * myVar[threadIdx.x];
	myVar[3] = 22.544241 * myVar[threadIdx.x];
	myVar[4] = 26.402154 * myVar[threadIdx.x];
	myVar[8] = 20.717110 * myVar[threadIdx.x];
	myVar[4] = 36.832258 * myVar[threadIdx.x];
	var_46_0[0] = myVar[0];
	var_46_1[1] = myVar[1];
	var_46_2[2] = myVar[2];
	var_46_3[3] = myVar[3];
	var_46_4[4] = myVar[4];
	var_46_5[5] = myVar[5];
	var_46_6[6] = myVar[6];
	var_46_7[7] = myVar[7];
	var_46_8[8] = myVar[8];
	var_46_9[9] = myVar[9];
	
}

__global__ void kernel_47(float * var_47_0, float * var_47_1, float * var_47_2, float * var_47_3, float * var_47_4, float * var_47_5, float * var_47_6, float * var_47_7, float * var_47_8, float * var_47_9) {
	__shared__ float myVar[1024];
	myVar[3] = 27.788791 * myVar[threadIdx.x];
	myVar[6] = 29.835578 * myVar[threadIdx.x];
	myVar[1] = 10.718828 * myVar[threadIdx.x];
	myVar[4] = 8.423091 * myVar[threadIdx.x];
	myVar[0] = 18.408419 * myVar[threadIdx.x];
	myVar[3] = 34.166867 * myVar[threadIdx.x];
	myVar[1] = 33.818438 * myVar[threadIdx.x];
	myVar[9] = 38.649392 * myVar[threadIdx.x];
	myVar[2] = 38.995460 * myVar[threadIdx.x];
	myVar[1] = 7.026142 * myVar[threadIdx.x];
	var_47_0[0] = myVar[0];
	var_47_1[1] = myVar[1];
	var_47_2[2] = myVar[2];
	var_47_3[3] = myVar[3];
	var_47_4[4] = myVar[4];
	var_47_5[5] = myVar[5];
	var_47_6[6] = myVar[6];
	var_47_7[7] = myVar[7];
	var_47_8[8] = myVar[8];
	var_47_9[9] = myVar[9];
	
}

__global__ void kernel_48(float * var_48_0, float * var_48_1, float * var_48_2, float * var_48_3, float * var_48_4, float * var_48_5, float * var_48_6, float * var_48_7, float * var_48_8, float * var_48_9) {
	__shared__ float myVar[1024];
	myVar[8] = 34.125071 * myVar[threadIdx.x];
	myVar[7] = 5.222487 * myVar[threadIdx.x];
	myVar[2] = 36.672181 * myVar[threadIdx.x];
	myVar[9] = 12.274317 * myVar[threadIdx.x];
	myVar[6] = 9.177071 * myVar[threadIdx.x];
	myVar[7] = 5.821057 * myVar[threadIdx.x];
	myVar[8] = 1.231224 * myVar[threadIdx.x];
	myVar[3] = 49.790522 * myVar[threadIdx.x];
	myVar[8] = 39.761171 * myVar[threadIdx.x];
	myVar[4] = 22.404854 * myVar[threadIdx.x];
	var_48_0[0] = myVar[0];
	var_48_1[1] = myVar[1];
	var_48_2[2] = myVar[2];
	var_48_3[3] = myVar[3];
	var_48_4[4] = myVar[4];
	var_48_5[5] = myVar[5];
	var_48_6[6] = myVar[6];
	var_48_7[7] = myVar[7];
	var_48_8[8] = myVar[8];
	var_48_9[9] = myVar[9];
	
}

__global__ void kernel_49(float * var_49_0, float * var_49_1, float * var_49_2, float * var_49_3, float * var_49_4, float * var_49_5, float * var_49_6, float * var_49_7, float * var_49_8, float * var_49_9) {
	__shared__ float myVar[1024];
	myVar[9] = 16.530505 * myVar[threadIdx.x];
	myVar[2] = 15.127651 * myVar[threadIdx.x];
	myVar[0] = 30.241751 * myVar[threadIdx.x];
	myVar[4] = 32.781389 * myVar[threadIdx.x];
	myVar[7] = 39.703450 * myVar[threadIdx.x];
	myVar[3] = 20.524503 * myVar[threadIdx.x];
	myVar[2] = 9.988706 * myVar[threadIdx.x];
	myVar[0] = 31.878672 * myVar[threadIdx.x];
	myVar[8] = 23.459937 * myVar[threadIdx.x];
	myVar[8] = 46.195898 * myVar[threadIdx.x];
	var_49_0[0] = myVar[0];
	var_49_1[1] = myVar[1];
	var_49_2[2] = myVar[2];
	var_49_3[3] = myVar[3];
	var_49_4[4] = myVar[4];
	var_49_5[5] = myVar[5];
	var_49_6[6] = myVar[6];
	var_49_7[7] = myVar[7];
	var_49_8[8] = myVar[8];
	var_49_9[9] = myVar[9];
	
}

__global__ void kernel_50(float * var_50_0, float * var_50_1, float * var_50_2, float * var_50_3, float * var_50_4, float * var_50_5, float * var_50_6, float * var_50_7, float * var_50_8, float * var_50_9) {
	__shared__ float myVar[1024];
	myVar[4] = 5.325346 * myVar[threadIdx.x];
	myVar[6] = 15.725661 * myVar[threadIdx.x];
	myVar[0] = 13.795713 * myVar[threadIdx.x];
	myVar[4] = 37.816785 * myVar[threadIdx.x];
	myVar[0] = 24.448054 * myVar[threadIdx.x];
	myVar[8] = 13.825842 * myVar[threadIdx.x];
	myVar[6] = 3.172842 * myVar[threadIdx.x];
	myVar[0] = 20.339939 * myVar[threadIdx.x];
	myVar[0] = 38.466321 * myVar[threadIdx.x];
	myVar[4] = 1.731809 * myVar[threadIdx.x];
	var_50_0[0] = myVar[0];
	var_50_1[1] = myVar[1];
	var_50_2[2] = myVar[2];
	var_50_3[3] = myVar[3];
	var_50_4[4] = myVar[4];
	var_50_5[5] = myVar[5];
	var_50_6[6] = myVar[6];
	var_50_7[7] = myVar[7];
	var_50_8[8] = myVar[8];
	var_50_9[9] = myVar[9];
	
}

__global__ void kernel_51(float * var_51_0, float * var_51_1, float * var_51_2, float * var_51_3, float * var_51_4, float * var_51_5, float * var_51_6, float * var_51_7, float * var_51_8, float * var_51_9) {
	__shared__ float myVar[1024];
	myVar[6] = 19.079411 * myVar[threadIdx.x];
	myVar[9] = 12.770786 * myVar[threadIdx.x];
	myVar[6] = 45.832591 * myVar[threadIdx.x];
	myVar[2] = 23.565949 * myVar[threadIdx.x];
	myVar[1] = 13.269062 * myVar[threadIdx.x];
	myVar[4] = 29.815152 * myVar[threadIdx.x];
	myVar[2] = 47.923472 * myVar[threadIdx.x];
	myVar[2] = 25.084106 * myVar[threadIdx.x];
	myVar[1] = 9.889331 * myVar[threadIdx.x];
	myVar[1] = 25.405339 * myVar[threadIdx.x];
	var_51_0[0] = myVar[0];
	var_51_1[1] = myVar[1];
	var_51_2[2] = myVar[2];
	var_51_3[3] = myVar[3];
	var_51_4[4] = myVar[4];
	var_51_5[5] = myVar[5];
	var_51_6[6] = myVar[6];
	var_51_7[7] = myVar[7];
	var_51_8[8] = myVar[8];
	var_51_9[9] = myVar[9];
	
}

__global__ void kernel_52(float * var_52_0, float * var_52_1, float * var_52_2, float * var_52_3, float * var_52_4, float * var_52_5, float * var_52_6, float * var_52_7, float * var_52_8, float * var_52_9) {
	__shared__ float myVar[1024];
	myVar[1] = 14.796697 * myVar[threadIdx.x];
	myVar[5] = 29.243528 * myVar[threadIdx.x];
	myVar[1] = 11.022113 * myVar[threadIdx.x];
	myVar[2] = 42.219422 * myVar[threadIdx.x];
	myVar[3] = 8.393879 * myVar[threadIdx.x];
	myVar[6] = 36.936989 * myVar[threadIdx.x];
	myVar[2] = 19.929292 * myVar[threadIdx.x];
	myVar[6] = 37.384822 * myVar[threadIdx.x];
	myVar[3] = 46.113482 * myVar[threadIdx.x];
	myVar[9] = 32.640692 * myVar[threadIdx.x];
	var_52_0[0] = myVar[0];
	var_52_1[1] = myVar[1];
	var_52_2[2] = myVar[2];
	var_52_3[3] = myVar[3];
	var_52_4[4] = myVar[4];
	var_52_5[5] = myVar[5];
	var_52_6[6] = myVar[6];
	var_52_7[7] = myVar[7];
	var_52_8[8] = myVar[8];
	var_52_9[9] = myVar[9];
	
}

__global__ void kernel_53(float * var_53_0, float * var_53_1, float * var_53_2, float * var_53_3, float * var_53_4, float * var_53_5, float * var_53_6, float * var_53_7, float * var_53_8, float * var_53_9) {
	__shared__ float myVar[1024];
	myVar[2] = 10.261608 * myVar[threadIdx.x];
	myVar[5] = 28.951232 * myVar[threadIdx.x];
	myVar[6] = 24.001826 * myVar[threadIdx.x];
	myVar[0] = 43.966242 * myVar[threadIdx.x];
	myVar[7] = 46.266413 * myVar[threadIdx.x];
	myVar[7] = 19.628547 * myVar[threadIdx.x];
	myVar[5] = 3.449005 * myVar[threadIdx.x];
	myVar[7] = 13.980082 * myVar[threadIdx.x];
	myVar[6] = 47.656687 * myVar[threadIdx.x];
	myVar[3] = 14.673002 * myVar[threadIdx.x];
	var_53_0[0] = myVar[0];
	var_53_1[1] = myVar[1];
	var_53_2[2] = myVar[2];
	var_53_3[3] = myVar[3];
	var_53_4[4] = myVar[4];
	var_53_5[5] = myVar[5];
	var_53_6[6] = myVar[6];
	var_53_7[7] = myVar[7];
	var_53_8[8] = myVar[8];
	var_53_9[9] = myVar[9];
	
}

__global__ void kernel_54(float * var_54_0, float * var_54_1, float * var_54_2, float * var_54_3, float * var_54_4, float * var_54_5, float * var_54_6, float * var_54_7, float * var_54_8, float * var_54_9) {
	__shared__ float myVar[1024];
	myVar[7] = 35.271961 * myVar[threadIdx.x];
	myVar[3] = 46.033162 * myVar[threadIdx.x];
	myVar[2] = 19.404058 * myVar[threadIdx.x];
	myVar[0] = 19.280636 * myVar[threadIdx.x];
	myVar[7] = 13.225660 * myVar[threadIdx.x];
	myVar[9] = 23.648565 * myVar[threadIdx.x];
	myVar[4] = 1.204307 * myVar[threadIdx.x];
	myVar[7] = 20.344610 * myVar[threadIdx.x];
	myVar[5] = 43.198196 * myVar[threadIdx.x];
	myVar[2] = 10.681342 * myVar[threadIdx.x];
	var_54_0[0] = myVar[0];
	var_54_1[1] = myVar[1];
	var_54_2[2] = myVar[2];
	var_54_3[3] = myVar[3];
	var_54_4[4] = myVar[4];
	var_54_5[5] = myVar[5];
	var_54_6[6] = myVar[6];
	var_54_7[7] = myVar[7];
	var_54_8[8] = myVar[8];
	var_54_9[9] = myVar[9];
	
}

__global__ void kernel_55(float * var_55_0, float * var_55_1, float * var_55_2, float * var_55_3, float * var_55_4, float * var_55_5, float * var_55_6, float * var_55_7, float * var_55_8, float * var_55_9) {
	__shared__ float myVar[1024];
	myVar[4] = 19.916123 * myVar[threadIdx.x];
	myVar[4] = 22.751341 * myVar[threadIdx.x];
	myVar[5] = 44.696533 * myVar[threadIdx.x];
	myVar[5] = 24.933806 * myVar[threadIdx.x];
	myVar[9] = 25.149382 * myVar[threadIdx.x];
	myVar[5] = 9.417759 * myVar[threadIdx.x];
	myVar[1] = 17.649512 * myVar[threadIdx.x];
	myVar[6] = 19.933094 * myVar[threadIdx.x];
	myVar[6] = 7.024863 * myVar[threadIdx.x];
	myVar[1] = 27.755281 * myVar[threadIdx.x];
	var_55_0[0] = myVar[0];
	var_55_1[1] = myVar[1];
	var_55_2[2] = myVar[2];
	var_55_3[3] = myVar[3];
	var_55_4[4] = myVar[4];
	var_55_5[5] = myVar[5];
	var_55_6[6] = myVar[6];
	var_55_7[7] = myVar[7];
	var_55_8[8] = myVar[8];
	var_55_9[9] = myVar[9];
	
}

__global__ void kernel_56(float * var_56_0, float * var_56_1, float * var_56_2, float * var_56_3, float * var_56_4, float * var_56_5, float * var_56_6, float * var_56_7, float * var_56_8, float * var_56_9) {
	__shared__ float myVar[1024];
	myVar[9] = 6.687973 * myVar[threadIdx.x];
	myVar[7] = 31.218800 * myVar[threadIdx.x];
	myVar[8] = 14.682340 * myVar[threadIdx.x];
	myVar[6] = 32.591882 * myVar[threadIdx.x];
	myVar[5] = 0.628655 * myVar[threadIdx.x];
	myVar[5] = 29.086831 * myVar[threadIdx.x];
	myVar[5] = 38.344642 * myVar[threadIdx.x];
	myVar[5] = 48.892267 * myVar[threadIdx.x];
	myVar[1] = 5.500571 * myVar[threadIdx.x];
	myVar[9] = 31.552227 * myVar[threadIdx.x];
	var_56_0[0] = myVar[0];
	var_56_1[1] = myVar[1];
	var_56_2[2] = myVar[2];
	var_56_3[3] = myVar[3];
	var_56_4[4] = myVar[4];
	var_56_5[5] = myVar[5];
	var_56_6[6] = myVar[6];
	var_56_7[7] = myVar[7];
	var_56_8[8] = myVar[8];
	var_56_9[9] = myVar[9];
	
}

__global__ void kernel_57(float * var_57_0, float * var_57_1, float * var_57_2, float * var_57_3, float * var_57_4, float * var_57_5, float * var_57_6, float * var_57_7, float * var_57_8, float * var_57_9) {
	__shared__ float myVar[1024];
	myVar[0] = 42.942852 * myVar[threadIdx.x];
	myVar[7] = 16.922594 * myVar[threadIdx.x];
	myVar[5] = 25.290475 * myVar[threadIdx.x];
	myVar[3] = 42.944496 * myVar[threadIdx.x];
	myVar[8] = 36.049624 * myVar[threadIdx.x];
	myVar[3] = 21.299058 * myVar[threadIdx.x];
	myVar[7] = 35.597965 * myVar[threadIdx.x];
	myVar[2] = 10.644784 * myVar[threadIdx.x];
	myVar[6] = 48.275254 * myVar[threadIdx.x];
	myVar[2] = 24.570567 * myVar[threadIdx.x];
	var_57_0[0] = myVar[0];
	var_57_1[1] = myVar[1];
	var_57_2[2] = myVar[2];
	var_57_3[3] = myVar[3];
	var_57_4[4] = myVar[4];
	var_57_5[5] = myVar[5];
	var_57_6[6] = myVar[6];
	var_57_7[7] = myVar[7];
	var_57_8[8] = myVar[8];
	var_57_9[9] = myVar[9];
	
}

__global__ void kernel_58(float * var_58_0, float * var_58_1, float * var_58_2, float * var_58_3, float * var_58_4, float * var_58_5, float * var_58_6, float * var_58_7, float * var_58_8, float * var_58_9) {
	__shared__ float myVar[1024];
	myVar[0] = 32.557784 * myVar[threadIdx.x];
	myVar[4] = 31.142459 * myVar[threadIdx.x];
	myVar[1] = 44.341997 * myVar[threadIdx.x];
	myVar[2] = 39.714522 * myVar[threadIdx.x];
	myVar[4] = 42.604394 * myVar[threadIdx.x];
	myVar[7] = 15.058580 * myVar[threadIdx.x];
	myVar[2] = 25.976174 * myVar[threadIdx.x];
	myVar[1] = 30.940931 * myVar[threadIdx.x];
	myVar[1] = 16.873948 * myVar[threadIdx.x];
	myVar[2] = 10.993214 * myVar[threadIdx.x];
	var_58_0[0] = myVar[0];
	var_58_1[1] = myVar[1];
	var_58_2[2] = myVar[2];
	var_58_3[3] = myVar[3];
	var_58_4[4] = myVar[4];
	var_58_5[5] = myVar[5];
	var_58_6[6] = myVar[6];
	var_58_7[7] = myVar[7];
	var_58_8[8] = myVar[8];
	var_58_9[9] = myVar[9];
	
}

__global__ void kernel_59(float * var_59_0, float * var_59_1, float * var_59_2, float * var_59_3, float * var_59_4, float * var_59_5, float * var_59_6, float * var_59_7, float * var_59_8, float * var_59_9) {
	__shared__ float myVar[1024];
	myVar[3] = 1.311810 * myVar[threadIdx.x];
	myVar[5] = 28.465090 * myVar[threadIdx.x];
	myVar[0] = 15.562939 * myVar[threadIdx.x];
	myVar[5] = 18.741216 * myVar[threadIdx.x];
	myVar[4] = 33.144149 * myVar[threadIdx.x];
	myVar[7] = 33.103929 * myVar[threadIdx.x];
	myVar[1] = 22.436713 * myVar[threadIdx.x];
	myVar[6] = 5.993131 * myVar[threadIdx.x];
	myVar[4] = 10.973600 * myVar[threadIdx.x];
	myVar[2] = 17.460804 * myVar[threadIdx.x];
	var_59_0[0] = myVar[0];
	var_59_1[1] = myVar[1];
	var_59_2[2] = myVar[2];
	var_59_3[3] = myVar[3];
	var_59_4[4] = myVar[4];
	var_59_5[5] = myVar[5];
	var_59_6[6] = myVar[6];
	var_59_7[7] = myVar[7];
	var_59_8[8] = myVar[8];
	var_59_9[9] = myVar[9];
	
}

__global__ void kernel_60(float * var_60_0, float * var_60_1, float * var_60_2, float * var_60_3, float * var_60_4, float * var_60_5, float * var_60_6, float * var_60_7, float * var_60_8, float * var_60_9) {
	__shared__ float myVar[1024];
	myVar[7] = 26.745334 * myVar[threadIdx.x];
	myVar[0] = 44.518574 * myVar[threadIdx.x];
	myVar[7] = 1.071710 * myVar[threadIdx.x];
	myVar[4] = 28.570562 * myVar[threadIdx.x];
	myVar[9] = 40.192279 * myVar[threadIdx.x];
	myVar[2] = 8.392118 * myVar[threadIdx.x];
	myVar[7] = 37.779091 * myVar[threadIdx.x];
	myVar[3] = 44.950181 * myVar[threadIdx.x];
	myVar[1] = 29.133288 * myVar[threadIdx.x];
	myVar[1] = 3.291740 * myVar[threadIdx.x];
	var_60_0[0] = myVar[0];
	var_60_1[1] = myVar[1];
	var_60_2[2] = myVar[2];
	var_60_3[3] = myVar[3];
	var_60_4[4] = myVar[4];
	var_60_5[5] = myVar[5];
	var_60_6[6] = myVar[6];
	var_60_7[7] = myVar[7];
	var_60_8[8] = myVar[8];
	var_60_9[9] = myVar[9];
	
}

__global__ void kernel_61(float * var_61_0, float * var_61_1, float * var_61_2, float * var_61_3, float * var_61_4, float * var_61_5, float * var_61_6, float * var_61_7, float * var_61_8, float * var_61_9) {
	__shared__ float myVar[1024];
	myVar[7] = 36.374968 * myVar[threadIdx.x];
	myVar[7] = 47.836531 * myVar[threadIdx.x];
	myVar[3] = 28.497043 * myVar[threadIdx.x];
	myVar[1] = 3.867084 * myVar[threadIdx.x];
	myVar[0] = 33.422697 * myVar[threadIdx.x];
	myVar[4] = 9.390457 * myVar[threadIdx.x];
	myVar[3] = 34.073638 * myVar[threadIdx.x];
	myVar[6] = 31.175615 * myVar[threadIdx.x];
	myVar[0] = 29.532395 * myVar[threadIdx.x];
	myVar[7] = 9.283403 * myVar[threadIdx.x];
	var_61_0[0] = myVar[0];
	var_61_1[1] = myVar[1];
	var_61_2[2] = myVar[2];
	var_61_3[3] = myVar[3];
	var_61_4[4] = myVar[4];
	var_61_5[5] = myVar[5];
	var_61_6[6] = myVar[6];
	var_61_7[7] = myVar[7];
	var_61_8[8] = myVar[8];
	var_61_9[9] = myVar[9];
	
}

__global__ void kernel_62(float * var_62_0, float * var_62_1, float * var_62_2, float * var_62_3, float * var_62_4, float * var_62_5, float * var_62_6, float * var_62_7, float * var_62_8, float * var_62_9) {
	__shared__ float myVar[1024];
	myVar[1] = 14.789948 * myVar[threadIdx.x];
	myVar[5] = 14.691171 * myVar[threadIdx.x];
	myVar[8] = 13.428209 * myVar[threadIdx.x];
	myVar[1] = 43.424723 * myVar[threadIdx.x];
	myVar[8] = 11.275440 * myVar[threadIdx.x];
	myVar[4] = 27.078670 * myVar[threadIdx.x];
	myVar[5] = 39.230396 * myVar[threadIdx.x];
	myVar[0] = 2.988316 * myVar[threadIdx.x];
	myVar[1] = 24.087731 * myVar[threadIdx.x];
	myVar[9] = 30.846373 * myVar[threadIdx.x];
	var_62_0[0] = myVar[0];
	var_62_1[1] = myVar[1];
	var_62_2[2] = myVar[2];
	var_62_3[3] = myVar[3];
	var_62_4[4] = myVar[4];
	var_62_5[5] = myVar[5];
	var_62_6[6] = myVar[6];
	var_62_7[7] = myVar[7];
	var_62_8[8] = myVar[8];
	var_62_9[9] = myVar[9];
	
}

__global__ void kernel_63(float * var_63_0, float * var_63_1, float * var_63_2, float * var_63_3, float * var_63_4, float * var_63_5, float * var_63_6, float * var_63_7, float * var_63_8, float * var_63_9) {
	__shared__ float myVar[1024];
	myVar[5] = 3.936064 * myVar[threadIdx.x];
	myVar[3] = 47.100185 * myVar[threadIdx.x];
	myVar[0] = 37.955791 * myVar[threadIdx.x];
	myVar[8] = 48.851432 * myVar[threadIdx.x];
	myVar[2] = 10.548980 * myVar[threadIdx.x];
	myVar[6] = 22.418456 * myVar[threadIdx.x];
	myVar[6] = 32.476558 * myVar[threadIdx.x];
	myVar[1] = 12.657882 * myVar[threadIdx.x];
	myVar[3] = 41.171619 * myVar[threadIdx.x];
	myVar[8] = 14.120089 * myVar[threadIdx.x];
	var_63_0[0] = myVar[0];
	var_63_1[1] = myVar[1];
	var_63_2[2] = myVar[2];
	var_63_3[3] = myVar[3];
	var_63_4[4] = myVar[4];
	var_63_5[5] = myVar[5];
	var_63_6[6] = myVar[6];
	var_63_7[7] = myVar[7];
	var_63_8[8] = myVar[8];
	var_63_9[9] = myVar[9];
	
}

__global__ void kernel_64(float * var_64_0, float * var_64_1, float * var_64_2, float * var_64_3, float * var_64_4, float * var_64_5, float * var_64_6, float * var_64_7, float * var_64_8, float * var_64_9) {
	__shared__ float myVar[1024];
	myVar[7] = 45.565801 * myVar[threadIdx.x];
	myVar[6] = 25.062463 * myVar[threadIdx.x];
	myVar[8] = 1.728582 * myVar[threadIdx.x];
	myVar[9] = 40.347319 * myVar[threadIdx.x];
	myVar[2] = 15.007933 * myVar[threadIdx.x];
	myVar[8] = 35.658580 * myVar[threadIdx.x];
	myVar[6] = 45.459833 * myVar[threadIdx.x];
	myVar[5] = 18.062262 * myVar[threadIdx.x];
	myVar[2] = 8.765494 * myVar[threadIdx.x];
	myVar[6] = 11.171619 * myVar[threadIdx.x];
	var_64_0[0] = myVar[0];
	var_64_1[1] = myVar[1];
	var_64_2[2] = myVar[2];
	var_64_3[3] = myVar[3];
	var_64_4[4] = myVar[4];
	var_64_5[5] = myVar[5];
	var_64_6[6] = myVar[6];
	var_64_7[7] = myVar[7];
	var_64_8[8] = myVar[8];
	var_64_9[9] = myVar[9];
	
}

__global__ void kernel_65(float * var_65_0, float * var_65_1, float * var_65_2, float * var_65_3, float * var_65_4, float * var_65_5, float * var_65_6, float * var_65_7, float * var_65_8, float * var_65_9) {
	__shared__ float myVar[1024];
	myVar[5] = 30.269819 * myVar[threadIdx.x];
	myVar[8] = 32.043282 * myVar[threadIdx.x];
	myVar[5] = 45.355472 * myVar[threadIdx.x];
	myVar[8] = 33.669889 * myVar[threadIdx.x];
	myVar[4] = 23.793983 * myVar[threadIdx.x];
	myVar[4] = 18.830421 * myVar[threadIdx.x];
	myVar[4] = 48.601843 * myVar[threadIdx.x];
	myVar[8] = 5.349688 * myVar[threadIdx.x];
	myVar[6] = 40.942829 * myVar[threadIdx.x];
	myVar[0] = 15.357022 * myVar[threadIdx.x];
	var_65_0[0] = myVar[0];
	var_65_1[1] = myVar[1];
	var_65_2[2] = myVar[2];
	var_65_3[3] = myVar[3];
	var_65_4[4] = myVar[4];
	var_65_5[5] = myVar[5];
	var_65_6[6] = myVar[6];
	var_65_7[7] = myVar[7];
	var_65_8[8] = myVar[8];
	var_65_9[9] = myVar[9];
	
}

__global__ void kernel_66(float * var_66_0, float * var_66_1, float * var_66_2, float * var_66_3, float * var_66_4, float * var_66_5, float * var_66_6, float * var_66_7, float * var_66_8, float * var_66_9) {
	__shared__ float myVar[1024];
	myVar[5] = 25.266525 * myVar[threadIdx.x];
	myVar[6] = 13.470612 * myVar[threadIdx.x];
	myVar[8] = 3.439291 * myVar[threadIdx.x];
	myVar[5] = 42.202740 * myVar[threadIdx.x];
	myVar[5] = 18.426540 * myVar[threadIdx.x];
	myVar[0] = 46.380957 * myVar[threadIdx.x];
	myVar[0] = 49.348087 * myVar[threadIdx.x];
	myVar[0] = 41.588064 * myVar[threadIdx.x];
	myVar[2] = 41.296533 * myVar[threadIdx.x];
	myVar[6] = 34.181203 * myVar[threadIdx.x];
	var_66_0[0] = myVar[0];
	var_66_1[1] = myVar[1];
	var_66_2[2] = myVar[2];
	var_66_3[3] = myVar[3];
	var_66_4[4] = myVar[4];
	var_66_5[5] = myVar[5];
	var_66_6[6] = myVar[6];
	var_66_7[7] = myVar[7];
	var_66_8[8] = myVar[8];
	var_66_9[9] = myVar[9];
	
}

__global__ void kernel_67(float * var_67_0, float * var_67_1, float * var_67_2, float * var_67_3, float * var_67_4, float * var_67_5, float * var_67_6, float * var_67_7, float * var_67_8, float * var_67_9) {
	__shared__ float myVar[1024];
	myVar[0] = 38.271522 * myVar[threadIdx.x];
	myVar[8] = 31.755713 * myVar[threadIdx.x];
	myVar[9] = 7.291866 * myVar[threadIdx.x];
	myVar[1] = 14.571830 * myVar[threadIdx.x];
	myVar[3] = 11.520106 * myVar[threadIdx.x];
	myVar[9] = 5.117608 * myVar[threadIdx.x];
	myVar[5] = 21.261513 * myVar[threadIdx.x];
	myVar[6] = 20.912550 * myVar[threadIdx.x];
	myVar[3] = 8.134773 * myVar[threadIdx.x];
	myVar[2] = 0.225429 * myVar[threadIdx.x];
	var_67_0[0] = myVar[0];
	var_67_1[1] = myVar[1];
	var_67_2[2] = myVar[2];
	var_67_3[3] = myVar[3];
	var_67_4[4] = myVar[4];
	var_67_5[5] = myVar[5];
	var_67_6[6] = myVar[6];
	var_67_7[7] = myVar[7];
	var_67_8[8] = myVar[8];
	var_67_9[9] = myVar[9];
	
}

__global__ void kernel_68(float * var_68_0, float * var_68_1, float * var_68_2, float * var_68_3, float * var_68_4, float * var_68_5, float * var_68_6, float * var_68_7, float * var_68_8, float * var_68_9) {
	__shared__ float myVar[1024];
	myVar[5] = 29.808839 * myVar[threadIdx.x];
	myVar[2] = 32.892619 * myVar[threadIdx.x];
	myVar[0] = 37.725041 * myVar[threadIdx.x];
	myVar[3] = 19.956411 * myVar[threadIdx.x];
	myVar[7] = 15.362055 * myVar[threadIdx.x];
	myVar[6] = 35.670678 * myVar[threadIdx.x];
	myVar[4] = 24.520880 * myVar[threadIdx.x];
	myVar[4] = 4.538341 * myVar[threadIdx.x];
	myVar[7] = 15.441425 * myVar[threadIdx.x];
	myVar[4] = 47.349828 * myVar[threadIdx.x];
	var_68_0[0] = myVar[0];
	var_68_1[1] = myVar[1];
	var_68_2[2] = myVar[2];
	var_68_3[3] = myVar[3];
	var_68_4[4] = myVar[4];
	var_68_5[5] = myVar[5];
	var_68_6[6] = myVar[6];
	var_68_7[7] = myVar[7];
	var_68_8[8] = myVar[8];
	var_68_9[9] = myVar[9];
	
}

__global__ void kernel_69(float * var_69_0, float * var_69_1, float * var_69_2, float * var_69_3, float * var_69_4, float * var_69_5, float * var_69_6, float * var_69_7, float * var_69_8, float * var_69_9) {
	__shared__ float myVar[1024];
	myVar[6] = 15.561522 * myVar[threadIdx.x];
	myVar[3] = 28.889930 * myVar[threadIdx.x];
	myVar[0] = 42.692009 * myVar[threadIdx.x];
	myVar[8] = 48.031363 * myVar[threadIdx.x];
	myVar[0] = 40.455473 * myVar[threadIdx.x];
	myVar[5] = 17.495201 * myVar[threadIdx.x];
	myVar[2] = 45.045797 * myVar[threadIdx.x];
	myVar[3] = 41.532599 * myVar[threadIdx.x];
	myVar[6] = 44.325313 * myVar[threadIdx.x];
	myVar[5] = 17.036809 * myVar[threadIdx.x];
	var_69_0[0] = myVar[0];
	var_69_1[1] = myVar[1];
	var_69_2[2] = myVar[2];
	var_69_3[3] = myVar[3];
	var_69_4[4] = myVar[4];
	var_69_5[5] = myVar[5];
	var_69_6[6] = myVar[6];
	var_69_7[7] = myVar[7];
	var_69_8[8] = myVar[8];
	var_69_9[9] = myVar[9];
	
}

__global__ void kernel_70(float * var_70_0, float * var_70_1, float * var_70_2, float * var_70_3, float * var_70_4, float * var_70_5, float * var_70_6, float * var_70_7, float * var_70_8, float * var_70_9) {
	__shared__ float myVar[1024];
	myVar[4] = 43.822753 * myVar[threadIdx.x];
	myVar[3] = 45.664721 * myVar[threadIdx.x];
	myVar[5] = 18.776748 * myVar[threadIdx.x];
	myVar[2] = 31.794177 * myVar[threadIdx.x];
	myVar[8] = 5.619331 * myVar[threadIdx.x];
	myVar[5] = 12.781326 * myVar[threadIdx.x];
	myVar[0] = 0.490978 * myVar[threadIdx.x];
	myVar[6] = 23.569952 * myVar[threadIdx.x];
	myVar[5] = 35.965219 * myVar[threadIdx.x];
	myVar[3] = 46.658023 * myVar[threadIdx.x];
	var_70_0[0] = myVar[0];
	var_70_1[1] = myVar[1];
	var_70_2[2] = myVar[2];
	var_70_3[3] = myVar[3];
	var_70_4[4] = myVar[4];
	var_70_5[5] = myVar[5];
	var_70_6[6] = myVar[6];
	var_70_7[7] = myVar[7];
	var_70_8[8] = myVar[8];
	var_70_9[9] = myVar[9];
	
}

__global__ void kernel_71(float * var_71_0, float * var_71_1, float * var_71_2, float * var_71_3, float * var_71_4, float * var_71_5, float * var_71_6, float * var_71_7, float * var_71_8, float * var_71_9) {
	__shared__ float myVar[1024];
	myVar[4] = 44.703040 * myVar[threadIdx.x];
	myVar[0] = 3.950518 * myVar[threadIdx.x];
	myVar[7] = 19.348442 * myVar[threadIdx.x];
	myVar[2] = 30.184186 * myVar[threadIdx.x];
	myVar[5] = 18.387463 * myVar[threadIdx.x];
	myVar[1] = 28.608455 * myVar[threadIdx.x];
	myVar[9] = 25.229606 * myVar[threadIdx.x];
	myVar[2] = 6.512417 * myVar[threadIdx.x];
	myVar[6] = 21.735847 * myVar[threadIdx.x];
	myVar[2] = 9.153115 * myVar[threadIdx.x];
	var_71_0[0] = myVar[0];
	var_71_1[1] = myVar[1];
	var_71_2[2] = myVar[2];
	var_71_3[3] = myVar[3];
	var_71_4[4] = myVar[4];
	var_71_5[5] = myVar[5];
	var_71_6[6] = myVar[6];
	var_71_7[7] = myVar[7];
	var_71_8[8] = myVar[8];
	var_71_9[9] = myVar[9];
	
}

__global__ void kernel_72(float * var_72_0, float * var_72_1, float * var_72_2, float * var_72_3, float * var_72_4, float * var_72_5, float * var_72_6, float * var_72_7, float * var_72_8, float * var_72_9) {
	__shared__ float myVar[1024];
	myVar[0] = 27.560063 * myVar[threadIdx.x];
	myVar[6] = 34.444489 * myVar[threadIdx.x];
	myVar[4] = 5.275940 * myVar[threadIdx.x];
	myVar[8] = 10.500783 * myVar[threadIdx.x];
	myVar[1] = 6.446349 * myVar[threadIdx.x];
	myVar[6] = 28.947571 * myVar[threadIdx.x];
	myVar[3] = 44.292461 * myVar[threadIdx.x];
	myVar[0] = 24.047115 * myVar[threadIdx.x];
	myVar[0] = 29.227834 * myVar[threadIdx.x];
	myVar[0] = 1.828860 * myVar[threadIdx.x];
	var_72_0[0] = myVar[0];
	var_72_1[1] = myVar[1];
	var_72_2[2] = myVar[2];
	var_72_3[3] = myVar[3];
	var_72_4[4] = myVar[4];
	var_72_5[5] = myVar[5];
	var_72_6[6] = myVar[6];
	var_72_7[7] = myVar[7];
	var_72_8[8] = myVar[8];
	var_72_9[9] = myVar[9];
	
}

__global__ void kernel_73(float * var_73_0, float * var_73_1, float * var_73_2, float * var_73_3, float * var_73_4, float * var_73_5, float * var_73_6, float * var_73_7, float * var_73_8, float * var_73_9) {
	__shared__ float myVar[1024];
	myVar[9] = 18.401364 * myVar[threadIdx.x];
	myVar[1] = 18.918785 * myVar[threadIdx.x];
	myVar[2] = 12.418437 * myVar[threadIdx.x];
	myVar[2] = 22.731725 * myVar[threadIdx.x];
	myVar[9] = 7.621444 * myVar[threadIdx.x];
	myVar[7] = 20.529277 * myVar[threadIdx.x];
	myVar[7] = 2.812689 * myVar[threadIdx.x];
	myVar[2] = 34.598437 * myVar[threadIdx.x];
	myVar[0] = 5.511729 * myVar[threadIdx.x];
	myVar[4] = 5.773224 * myVar[threadIdx.x];
	var_73_0[0] = myVar[0];
	var_73_1[1] = myVar[1];
	var_73_2[2] = myVar[2];
	var_73_3[3] = myVar[3];
	var_73_4[4] = myVar[4];
	var_73_5[5] = myVar[5];
	var_73_6[6] = myVar[6];
	var_73_7[7] = myVar[7];
	var_73_8[8] = myVar[8];
	var_73_9[9] = myVar[9];
	
}

__global__ void kernel_74(float * var_74_0, float * var_74_1, float * var_74_2, float * var_74_3, float * var_74_4, float * var_74_5, float * var_74_6, float * var_74_7, float * var_74_8, float * var_74_9) {
	__shared__ float myVar[1024];
	myVar[6] = 6.423541 * myVar[threadIdx.x];
	myVar[1] = 45.527027 * myVar[threadIdx.x];
	myVar[2] = 9.083565 * myVar[threadIdx.x];
	myVar[8] = 33.581670 * myVar[threadIdx.x];
	myVar[3] = 25.234802 * myVar[threadIdx.x];
	myVar[3] = 5.522352 * myVar[threadIdx.x];
	myVar[6] = 47.150690 * myVar[threadIdx.x];
	myVar[7] = 46.448180 * myVar[threadIdx.x];
	myVar[8] = 12.366391 * myVar[threadIdx.x];
	myVar[6] = 7.389587 * myVar[threadIdx.x];
	var_74_0[0] = myVar[0];
	var_74_1[1] = myVar[1];
	var_74_2[2] = myVar[2];
	var_74_3[3] = myVar[3];
	var_74_4[4] = myVar[4];
	var_74_5[5] = myVar[5];
	var_74_6[6] = myVar[6];
	var_74_7[7] = myVar[7];
	var_74_8[8] = myVar[8];
	var_74_9[9] = myVar[9];
	
}

__global__ void kernel_75(float * var_75_0, float * var_75_1, float * var_75_2, float * var_75_3, float * var_75_4, float * var_75_5, float * var_75_6, float * var_75_7, float * var_75_8, float * var_75_9) {
	__shared__ float myVar[1024];
	myVar[9] = 49.933159 * myVar[threadIdx.x];
	myVar[9] = 45.530373 * myVar[threadIdx.x];
	myVar[1] = 21.071016 * myVar[threadIdx.x];
	myVar[9] = 19.223554 * myVar[threadIdx.x];
	myVar[2] = 8.624469 * myVar[threadIdx.x];
	myVar[5] = 21.207931 * myVar[threadIdx.x];
	myVar[3] = 49.200149 * myVar[threadIdx.x];
	myVar[4] = 1.103956 * myVar[threadIdx.x];
	myVar[7] = 13.565424 * myVar[threadIdx.x];
	myVar[2] = 19.014096 * myVar[threadIdx.x];
	var_75_0[0] = myVar[0];
	var_75_1[1] = myVar[1];
	var_75_2[2] = myVar[2];
	var_75_3[3] = myVar[3];
	var_75_4[4] = myVar[4];
	var_75_5[5] = myVar[5];
	var_75_6[6] = myVar[6];
	var_75_7[7] = myVar[7];
	var_75_8[8] = myVar[8];
	var_75_9[9] = myVar[9];
	
}

__global__ void kernel_76(float * var_76_0, float * var_76_1, float * var_76_2, float * var_76_3, float * var_76_4, float * var_76_5, float * var_76_6, float * var_76_7, float * var_76_8, float * var_76_9) {
	__shared__ float myVar[1024];
	myVar[1] = 13.045966 * myVar[threadIdx.x];
	myVar[9] = 38.295206 * myVar[threadIdx.x];
	myVar[1] = 16.186648 * myVar[threadIdx.x];
	myVar[9] = 39.560001 * myVar[threadIdx.x];
	myVar[5] = 32.686363 * myVar[threadIdx.x];
	myVar[6] = 29.674898 * myVar[threadIdx.x];
	myVar[2] = 29.036698 * myVar[threadIdx.x];
	myVar[4] = 18.136690 * myVar[threadIdx.x];
	myVar[7] = 37.859740 * myVar[threadIdx.x];
	myVar[5] = 39.406620 * myVar[threadIdx.x];
	var_76_0[0] = myVar[0];
	var_76_1[1] = myVar[1];
	var_76_2[2] = myVar[2];
	var_76_3[3] = myVar[3];
	var_76_4[4] = myVar[4];
	var_76_5[5] = myVar[5];
	var_76_6[6] = myVar[6];
	var_76_7[7] = myVar[7];
	var_76_8[8] = myVar[8];
	var_76_9[9] = myVar[9];
	
}

__global__ void kernel_77(float * var_77_0, float * var_77_1, float * var_77_2, float * var_77_3, float * var_77_4, float * var_77_5, float * var_77_6, float * var_77_7, float * var_77_8, float * var_77_9) {
	__shared__ float myVar[1024];
	myVar[3] = 13.892454 * myVar[threadIdx.x];
	myVar[4] = 15.717451 * myVar[threadIdx.x];
	myVar[2] = 45.974949 * myVar[threadIdx.x];
	myVar[2] = 26.309925 * myVar[threadIdx.x];
	myVar[4] = 42.529810 * myVar[threadIdx.x];
	myVar[8] = 26.425277 * myVar[threadIdx.x];
	myVar[3] = 32.305462 * myVar[threadIdx.x];
	myVar[1] = 8.888775 * myVar[threadIdx.x];
	myVar[5] = 30.602284 * myVar[threadIdx.x];
	myVar[3] = 47.799063 * myVar[threadIdx.x];
	var_77_0[0] = myVar[0];
	var_77_1[1] = myVar[1];
	var_77_2[2] = myVar[2];
	var_77_3[3] = myVar[3];
	var_77_4[4] = myVar[4];
	var_77_5[5] = myVar[5];
	var_77_6[6] = myVar[6];
	var_77_7[7] = myVar[7];
	var_77_8[8] = myVar[8];
	var_77_9[9] = myVar[9];
	
}

__global__ void kernel_78(float * var_78_0, float * var_78_1, float * var_78_2, float * var_78_3, float * var_78_4, float * var_78_5, float * var_78_6, float * var_78_7, float * var_78_8, float * var_78_9) {
	__shared__ float myVar[1024];
	myVar[2] = 5.436115 * myVar[threadIdx.x];
	myVar[3] = 10.161375 * myVar[threadIdx.x];
	myVar[8] = 46.097263 * myVar[threadIdx.x];
	myVar[3] = 0.070710 * myVar[threadIdx.x];
	myVar[3] = 5.119169 * myVar[threadIdx.x];
	myVar[2] = 46.053299 * myVar[threadIdx.x];
	myVar[4] = 13.764615 * myVar[threadIdx.x];
	myVar[6] = 37.530775 * myVar[threadIdx.x];
	myVar[7] = 1.313748 * myVar[threadIdx.x];
	myVar[7] = 17.369290 * myVar[threadIdx.x];
	var_78_0[0] = myVar[0];
	var_78_1[1] = myVar[1];
	var_78_2[2] = myVar[2];
	var_78_3[3] = myVar[3];
	var_78_4[4] = myVar[4];
	var_78_5[5] = myVar[5];
	var_78_6[6] = myVar[6];
	var_78_7[7] = myVar[7];
	var_78_8[8] = myVar[8];
	var_78_9[9] = myVar[9];
	
}

__global__ void kernel_79(float * var_79_0, float * var_79_1, float * var_79_2, float * var_79_3, float * var_79_4, float * var_79_5, float * var_79_6, float * var_79_7, float * var_79_8, float * var_79_9) {
	__shared__ float myVar[1024];
	myVar[1] = 48.598773 * myVar[threadIdx.x];
	myVar[4] = 23.854089 * myVar[threadIdx.x];
	myVar[1] = 44.375970 * myVar[threadIdx.x];
	myVar[7] = 47.116124 * myVar[threadIdx.x];
	myVar[6] = 46.363602 * myVar[threadIdx.x];
	myVar[7] = 38.127347 * myVar[threadIdx.x];
	myVar[3] = 37.386352 * myVar[threadIdx.x];
	myVar[4] = 19.948166 * myVar[threadIdx.x];
	myVar[8] = 41.688540 * myVar[threadIdx.x];
	myVar[2] = 30.193864 * myVar[threadIdx.x];
	var_79_0[0] = myVar[0];
	var_79_1[1] = myVar[1];
	var_79_2[2] = myVar[2];
	var_79_3[3] = myVar[3];
	var_79_4[4] = myVar[4];
	var_79_5[5] = myVar[5];
	var_79_6[6] = myVar[6];
	var_79_7[7] = myVar[7];
	var_79_8[8] = myVar[8];
	var_79_9[9] = myVar[9];
	
}


int main(void) {
	
	float * h_var_0_0 = (float *)malloc(sizeof(float *));
	float * d_var_0_0;
	hipMalloc((void **)&d_var_0_0, sizeof(float *));
	
	float * h_var_0_1 = (float *)malloc(sizeof(float *));
	float * d_var_0_1;
	hipMalloc((void **)&d_var_0_1, sizeof(float *));
	
	float * h_var_0_2 = (float *)malloc(sizeof(float *));
	float * d_var_0_2;
	hipMalloc((void **)&d_var_0_2, sizeof(float *));
	
	float * h_var_0_3 = (float *)malloc(sizeof(float *));
	float * d_var_0_3;
	hipMalloc((void **)&d_var_0_3, sizeof(float *));
	
	float * h_var_0_4 = (float *)malloc(sizeof(float *));
	float * d_var_0_4;
	hipMalloc((void **)&d_var_0_4, sizeof(float *));
	
	float * h_var_0_5 = (float *)malloc(sizeof(float *));
	float * d_var_0_5;
	hipMalloc((void **)&d_var_0_5, sizeof(float *));
	
	float * h_var_0_6 = (float *)malloc(sizeof(float *));
	float * d_var_0_6;
	hipMalloc((void **)&d_var_0_6, sizeof(float *));
	
	float * h_var_0_7 = (float *)malloc(sizeof(float *));
	float * d_var_0_7;
	hipMalloc((void **)&d_var_0_7, sizeof(float *));
	
	float * h_var_0_8 = (float *)malloc(sizeof(float *));
	float * d_var_0_8;
	hipMalloc((void **)&d_var_0_8, sizeof(float *));
	
	float * h_var_0_9 = (float *)malloc(sizeof(float *));
	float * d_var_0_9;
	hipMalloc((void **)&d_var_0_9, sizeof(float *));
	
	float * h_var_1_0 = (float *)malloc(sizeof(float *));
	float * d_var_1_0;
	hipMalloc((void **)&d_var_1_0, sizeof(float *));
	
	float * h_var_1_1 = (float *)malloc(sizeof(float *));
	float * d_var_1_1;
	hipMalloc((void **)&d_var_1_1, sizeof(float *));
	
	float * h_var_1_2 = (float *)malloc(sizeof(float *));
	float * d_var_1_2;
	hipMalloc((void **)&d_var_1_2, sizeof(float *));
	
	float * h_var_1_3 = (float *)malloc(sizeof(float *));
	float * d_var_1_3;
	hipMalloc((void **)&d_var_1_3, sizeof(float *));
	
	float * h_var_1_4 = (float *)malloc(sizeof(float *));
	float * d_var_1_4;
	hipMalloc((void **)&d_var_1_4, sizeof(float *));
	
	float * h_var_1_5 = (float *)malloc(sizeof(float *));
	float * d_var_1_5;
	hipMalloc((void **)&d_var_1_5, sizeof(float *));
	
	float * h_var_1_6 = (float *)malloc(sizeof(float *));
	float * d_var_1_6;
	hipMalloc((void **)&d_var_1_6, sizeof(float *));
	
	float * h_var_1_7 = (float *)malloc(sizeof(float *));
	float * d_var_1_7;
	hipMalloc((void **)&d_var_1_7, sizeof(float *));
	
	float * h_var_1_8 = (float *)malloc(sizeof(float *));
	float * d_var_1_8;
	hipMalloc((void **)&d_var_1_8, sizeof(float *));
	
	float * h_var_1_9 = (float *)malloc(sizeof(float *));
	float * d_var_1_9;
	hipMalloc((void **)&d_var_1_9, sizeof(float *));
	
	float * h_var_2_0 = (float *)malloc(sizeof(float *));
	float * d_var_2_0;
	hipMalloc((void **)&d_var_2_0, sizeof(float *));
	
	float * h_var_2_1 = (float *)malloc(sizeof(float *));
	float * d_var_2_1;
	hipMalloc((void **)&d_var_2_1, sizeof(float *));
	
	float * h_var_2_2 = (float *)malloc(sizeof(float *));
	float * d_var_2_2;
	hipMalloc((void **)&d_var_2_2, sizeof(float *));
	
	float * h_var_2_3 = (float *)malloc(sizeof(float *));
	float * d_var_2_3;
	hipMalloc((void **)&d_var_2_3, sizeof(float *));
	
	float * h_var_2_4 = (float *)malloc(sizeof(float *));
	float * d_var_2_4;
	hipMalloc((void **)&d_var_2_4, sizeof(float *));
	
	float * h_var_2_5 = (float *)malloc(sizeof(float *));
	float * d_var_2_5;
	hipMalloc((void **)&d_var_2_5, sizeof(float *));
	
	float * h_var_2_6 = (float *)malloc(sizeof(float *));
	float * d_var_2_6;
	hipMalloc((void **)&d_var_2_6, sizeof(float *));
	
	float * h_var_2_7 = (float *)malloc(sizeof(float *));
	float * d_var_2_7;
	hipMalloc((void **)&d_var_2_7, sizeof(float *));
	
	float * h_var_2_8 = (float *)malloc(sizeof(float *));
	float * d_var_2_8;
	hipMalloc((void **)&d_var_2_8, sizeof(float *));
	
	float * h_var_2_9 = (float *)malloc(sizeof(float *));
	float * d_var_2_9;
	hipMalloc((void **)&d_var_2_9, sizeof(float *));
	
	float * h_var_3_0 = (float *)malloc(sizeof(float *));
	float * d_var_3_0;
	hipMalloc((void **)&d_var_3_0, sizeof(float *));
	
	float * h_var_3_1 = (float *)malloc(sizeof(float *));
	float * d_var_3_1;
	hipMalloc((void **)&d_var_3_1, sizeof(float *));
	
	float * h_var_3_2 = (float *)malloc(sizeof(float *));
	float * d_var_3_2;
	hipMalloc((void **)&d_var_3_2, sizeof(float *));
	
	float * h_var_3_3 = (float *)malloc(sizeof(float *));
	float * d_var_3_3;
	hipMalloc((void **)&d_var_3_3, sizeof(float *));
	
	float * h_var_3_4 = (float *)malloc(sizeof(float *));
	float * d_var_3_4;
	hipMalloc((void **)&d_var_3_4, sizeof(float *));
	
	float * h_var_3_5 = (float *)malloc(sizeof(float *));
	float * d_var_3_5;
	hipMalloc((void **)&d_var_3_5, sizeof(float *));
	
	float * h_var_3_6 = (float *)malloc(sizeof(float *));
	float * d_var_3_6;
	hipMalloc((void **)&d_var_3_6, sizeof(float *));
	
	float * h_var_3_7 = (float *)malloc(sizeof(float *));
	float * d_var_3_7;
	hipMalloc((void **)&d_var_3_7, sizeof(float *));
	
	float * h_var_3_8 = (float *)malloc(sizeof(float *));
	float * d_var_3_8;
	hipMalloc((void **)&d_var_3_8, sizeof(float *));
	
	float * h_var_3_9 = (float *)malloc(sizeof(float *));
	float * d_var_3_9;
	hipMalloc((void **)&d_var_3_9, sizeof(float *));
	
	float * h_var_4_0 = (float *)malloc(sizeof(float *));
	float * d_var_4_0;
	hipMalloc((void **)&d_var_4_0, sizeof(float *));
	
	float * h_var_4_1 = (float *)malloc(sizeof(float *));
	float * d_var_4_1;
	hipMalloc((void **)&d_var_4_1, sizeof(float *));
	
	float * h_var_4_2 = (float *)malloc(sizeof(float *));
	float * d_var_4_2;
	hipMalloc((void **)&d_var_4_2, sizeof(float *));
	
	float * h_var_4_3 = (float *)malloc(sizeof(float *));
	float * d_var_4_3;
	hipMalloc((void **)&d_var_4_3, sizeof(float *));
	
	float * h_var_4_4 = (float *)malloc(sizeof(float *));
	float * d_var_4_4;
	hipMalloc((void **)&d_var_4_4, sizeof(float *));
	
	float * h_var_4_5 = (float *)malloc(sizeof(float *));
	float * d_var_4_5;
	hipMalloc((void **)&d_var_4_5, sizeof(float *));
	
	float * h_var_4_6 = (float *)malloc(sizeof(float *));
	float * d_var_4_6;
	hipMalloc((void **)&d_var_4_6, sizeof(float *));
	
	float * h_var_4_7 = (float *)malloc(sizeof(float *));
	float * d_var_4_7;
	hipMalloc((void **)&d_var_4_7, sizeof(float *));
	
	float * h_var_4_8 = (float *)malloc(sizeof(float *));
	float * d_var_4_8;
	hipMalloc((void **)&d_var_4_8, sizeof(float *));
	
	float * h_var_4_9 = (float *)malloc(sizeof(float *));
	float * d_var_4_9;
	hipMalloc((void **)&d_var_4_9, sizeof(float *));
	
	float * h_var_5_0 = (float *)malloc(sizeof(float *));
	float * d_var_5_0;
	hipMalloc((void **)&d_var_5_0, sizeof(float *));
	
	float * h_var_5_1 = (float *)malloc(sizeof(float *));
	float * d_var_5_1;
	hipMalloc((void **)&d_var_5_1, sizeof(float *));
	
	float * h_var_5_2 = (float *)malloc(sizeof(float *));
	float * d_var_5_2;
	hipMalloc((void **)&d_var_5_2, sizeof(float *));
	
	float * h_var_5_3 = (float *)malloc(sizeof(float *));
	float * d_var_5_3;
	hipMalloc((void **)&d_var_5_3, sizeof(float *));
	
	float * h_var_5_4 = (float *)malloc(sizeof(float *));
	float * d_var_5_4;
	hipMalloc((void **)&d_var_5_4, sizeof(float *));
	
	float * h_var_5_5 = (float *)malloc(sizeof(float *));
	float * d_var_5_5;
	hipMalloc((void **)&d_var_5_5, sizeof(float *));
	
	float * h_var_5_6 = (float *)malloc(sizeof(float *));
	float * d_var_5_6;
	hipMalloc((void **)&d_var_5_6, sizeof(float *));
	
	float * h_var_5_7 = (float *)malloc(sizeof(float *));
	float * d_var_5_7;
	hipMalloc((void **)&d_var_5_7, sizeof(float *));
	
	float * h_var_5_8 = (float *)malloc(sizeof(float *));
	float * d_var_5_8;
	hipMalloc((void **)&d_var_5_8, sizeof(float *));
	
	float * h_var_5_9 = (float *)malloc(sizeof(float *));
	float * d_var_5_9;
	hipMalloc((void **)&d_var_5_9, sizeof(float *));
	
	float * h_var_6_0 = (float *)malloc(sizeof(float *));
	float * d_var_6_0;
	hipMalloc((void **)&d_var_6_0, sizeof(float *));
	
	float * h_var_6_1 = (float *)malloc(sizeof(float *));
	float * d_var_6_1;
	hipMalloc((void **)&d_var_6_1, sizeof(float *));
	
	float * h_var_6_2 = (float *)malloc(sizeof(float *));
	float * d_var_6_2;
	hipMalloc((void **)&d_var_6_2, sizeof(float *));
	
	float * h_var_6_3 = (float *)malloc(sizeof(float *));
	float * d_var_6_3;
	hipMalloc((void **)&d_var_6_3, sizeof(float *));
	
	float * h_var_6_4 = (float *)malloc(sizeof(float *));
	float * d_var_6_4;
	hipMalloc((void **)&d_var_6_4, sizeof(float *));
	
	float * h_var_6_5 = (float *)malloc(sizeof(float *));
	float * d_var_6_5;
	hipMalloc((void **)&d_var_6_5, sizeof(float *));
	
	float * h_var_6_6 = (float *)malloc(sizeof(float *));
	float * d_var_6_6;
	hipMalloc((void **)&d_var_6_6, sizeof(float *));
	
	float * h_var_6_7 = (float *)malloc(sizeof(float *));
	float * d_var_6_7;
	hipMalloc((void **)&d_var_6_7, sizeof(float *));
	
	float * h_var_6_8 = (float *)malloc(sizeof(float *));
	float * d_var_6_8;
	hipMalloc((void **)&d_var_6_8, sizeof(float *));
	
	float * h_var_6_9 = (float *)malloc(sizeof(float *));
	float * d_var_6_9;
	hipMalloc((void **)&d_var_6_9, sizeof(float *));
	
	float * h_var_7_0 = (float *)malloc(sizeof(float *));
	float * d_var_7_0;
	hipMalloc((void **)&d_var_7_0, sizeof(float *));
	
	float * h_var_7_1 = (float *)malloc(sizeof(float *));
	float * d_var_7_1;
	hipMalloc((void **)&d_var_7_1, sizeof(float *));
	
	float * h_var_7_2 = (float *)malloc(sizeof(float *));
	float * d_var_7_2;
	hipMalloc((void **)&d_var_7_2, sizeof(float *));
	
	float * h_var_7_3 = (float *)malloc(sizeof(float *));
	float * d_var_7_3;
	hipMalloc((void **)&d_var_7_3, sizeof(float *));
	
	float * h_var_7_4 = (float *)malloc(sizeof(float *));
	float * d_var_7_4;
	hipMalloc((void **)&d_var_7_4, sizeof(float *));
	
	float * h_var_7_5 = (float *)malloc(sizeof(float *));
	float * d_var_7_5;
	hipMalloc((void **)&d_var_7_5, sizeof(float *));
	
	float * h_var_7_6 = (float *)malloc(sizeof(float *));
	float * d_var_7_6;
	hipMalloc((void **)&d_var_7_6, sizeof(float *));
	
	float * h_var_7_7 = (float *)malloc(sizeof(float *));
	float * d_var_7_7;
	hipMalloc((void **)&d_var_7_7, sizeof(float *));
	
	float * h_var_7_8 = (float *)malloc(sizeof(float *));
	float * d_var_7_8;
	hipMalloc((void **)&d_var_7_8, sizeof(float *));
	
	float * h_var_7_9 = (float *)malloc(sizeof(float *));
	float * d_var_7_9;
	hipMalloc((void **)&d_var_7_9, sizeof(float *));
	
	float * h_var_8_0 = (float *)malloc(sizeof(float *));
	float * d_var_8_0;
	hipMalloc((void **)&d_var_8_0, sizeof(float *));
	
	float * h_var_8_1 = (float *)malloc(sizeof(float *));
	float * d_var_8_1;
	hipMalloc((void **)&d_var_8_1, sizeof(float *));
	
	float * h_var_8_2 = (float *)malloc(sizeof(float *));
	float * d_var_8_2;
	hipMalloc((void **)&d_var_8_2, sizeof(float *));
	
	float * h_var_8_3 = (float *)malloc(sizeof(float *));
	float * d_var_8_3;
	hipMalloc((void **)&d_var_8_3, sizeof(float *));
	
	float * h_var_8_4 = (float *)malloc(sizeof(float *));
	float * d_var_8_4;
	hipMalloc((void **)&d_var_8_4, sizeof(float *));
	
	float * h_var_8_5 = (float *)malloc(sizeof(float *));
	float * d_var_8_5;
	hipMalloc((void **)&d_var_8_5, sizeof(float *));
	
	float * h_var_8_6 = (float *)malloc(sizeof(float *));
	float * d_var_8_6;
	hipMalloc((void **)&d_var_8_6, sizeof(float *));
	
	float * h_var_8_7 = (float *)malloc(sizeof(float *));
	float * d_var_8_7;
	hipMalloc((void **)&d_var_8_7, sizeof(float *));
	
	float * h_var_8_8 = (float *)malloc(sizeof(float *));
	float * d_var_8_8;
	hipMalloc((void **)&d_var_8_8, sizeof(float *));
	
	float * h_var_8_9 = (float *)malloc(sizeof(float *));
	float * d_var_8_9;
	hipMalloc((void **)&d_var_8_9, sizeof(float *));
	
	float * h_var_9_0 = (float *)malloc(sizeof(float *));
	float * d_var_9_0;
	hipMalloc((void **)&d_var_9_0, sizeof(float *));
	
	float * h_var_9_1 = (float *)malloc(sizeof(float *));
	float * d_var_9_1;
	hipMalloc((void **)&d_var_9_1, sizeof(float *));
	
	float * h_var_9_2 = (float *)malloc(sizeof(float *));
	float * d_var_9_2;
	hipMalloc((void **)&d_var_9_2, sizeof(float *));
	
	float * h_var_9_3 = (float *)malloc(sizeof(float *));
	float * d_var_9_3;
	hipMalloc((void **)&d_var_9_3, sizeof(float *));
	
	float * h_var_9_4 = (float *)malloc(sizeof(float *));
	float * d_var_9_4;
	hipMalloc((void **)&d_var_9_4, sizeof(float *));
	
	float * h_var_9_5 = (float *)malloc(sizeof(float *));
	float * d_var_9_5;
	hipMalloc((void **)&d_var_9_5, sizeof(float *));
	
	float * h_var_9_6 = (float *)malloc(sizeof(float *));
	float * d_var_9_6;
	hipMalloc((void **)&d_var_9_6, sizeof(float *));
	
	float * h_var_9_7 = (float *)malloc(sizeof(float *));
	float * d_var_9_7;
	hipMalloc((void **)&d_var_9_7, sizeof(float *));
	
	float * h_var_9_8 = (float *)malloc(sizeof(float *));
	float * d_var_9_8;
	hipMalloc((void **)&d_var_9_8, sizeof(float *));
	
	float * h_var_9_9 = (float *)malloc(sizeof(float *));
	float * d_var_9_9;
	hipMalloc((void **)&d_var_9_9, sizeof(float *));
	
	float * h_var_10_0 = (float *)malloc(sizeof(float *));
	float * d_var_10_0;
	hipMalloc((void **)&d_var_10_0, sizeof(float *));
	
	float * h_var_10_1 = (float *)malloc(sizeof(float *));
	float * d_var_10_1;
	hipMalloc((void **)&d_var_10_1, sizeof(float *));
	
	float * h_var_10_2 = (float *)malloc(sizeof(float *));
	float * d_var_10_2;
	hipMalloc((void **)&d_var_10_2, sizeof(float *));
	
	float * h_var_10_3 = (float *)malloc(sizeof(float *));
	float * d_var_10_3;
	hipMalloc((void **)&d_var_10_3, sizeof(float *));
	
	float * h_var_10_4 = (float *)malloc(sizeof(float *));
	float * d_var_10_4;
	hipMalloc((void **)&d_var_10_4, sizeof(float *));
	
	float * h_var_10_5 = (float *)malloc(sizeof(float *));
	float * d_var_10_5;
	hipMalloc((void **)&d_var_10_5, sizeof(float *));
	
	float * h_var_10_6 = (float *)malloc(sizeof(float *));
	float * d_var_10_6;
	hipMalloc((void **)&d_var_10_6, sizeof(float *));
	
	float * h_var_10_7 = (float *)malloc(sizeof(float *));
	float * d_var_10_7;
	hipMalloc((void **)&d_var_10_7, sizeof(float *));
	
	float * h_var_10_8 = (float *)malloc(sizeof(float *));
	float * d_var_10_8;
	hipMalloc((void **)&d_var_10_8, sizeof(float *));
	
	float * h_var_10_9 = (float *)malloc(sizeof(float *));
	float * d_var_10_9;
	hipMalloc((void **)&d_var_10_9, sizeof(float *));
	
	float * h_var_11_0 = (float *)malloc(sizeof(float *));
	float * d_var_11_0;
	hipMalloc((void **)&d_var_11_0, sizeof(float *));
	
	float * h_var_11_1 = (float *)malloc(sizeof(float *));
	float * d_var_11_1;
	hipMalloc((void **)&d_var_11_1, sizeof(float *));
	
	float * h_var_11_2 = (float *)malloc(sizeof(float *));
	float * d_var_11_2;
	hipMalloc((void **)&d_var_11_2, sizeof(float *));
	
	float * h_var_11_3 = (float *)malloc(sizeof(float *));
	float * d_var_11_3;
	hipMalloc((void **)&d_var_11_3, sizeof(float *));
	
	float * h_var_11_4 = (float *)malloc(sizeof(float *));
	float * d_var_11_4;
	hipMalloc((void **)&d_var_11_4, sizeof(float *));
	
	float * h_var_11_5 = (float *)malloc(sizeof(float *));
	float * d_var_11_5;
	hipMalloc((void **)&d_var_11_5, sizeof(float *));
	
	float * h_var_11_6 = (float *)malloc(sizeof(float *));
	float * d_var_11_6;
	hipMalloc((void **)&d_var_11_6, sizeof(float *));
	
	float * h_var_11_7 = (float *)malloc(sizeof(float *));
	float * d_var_11_7;
	hipMalloc((void **)&d_var_11_7, sizeof(float *));
	
	float * h_var_11_8 = (float *)malloc(sizeof(float *));
	float * d_var_11_8;
	hipMalloc((void **)&d_var_11_8, sizeof(float *));
	
	float * h_var_11_9 = (float *)malloc(sizeof(float *));
	float * d_var_11_9;
	hipMalloc((void **)&d_var_11_9, sizeof(float *));
	
	float * h_var_12_0 = (float *)malloc(sizeof(float *));
	float * d_var_12_0;
	hipMalloc((void **)&d_var_12_0, sizeof(float *));
	
	float * h_var_12_1 = (float *)malloc(sizeof(float *));
	float * d_var_12_1;
	hipMalloc((void **)&d_var_12_1, sizeof(float *));
	
	float * h_var_12_2 = (float *)malloc(sizeof(float *));
	float * d_var_12_2;
	hipMalloc((void **)&d_var_12_2, sizeof(float *));
	
	float * h_var_12_3 = (float *)malloc(sizeof(float *));
	float * d_var_12_3;
	hipMalloc((void **)&d_var_12_3, sizeof(float *));
	
	float * h_var_12_4 = (float *)malloc(sizeof(float *));
	float * d_var_12_4;
	hipMalloc((void **)&d_var_12_4, sizeof(float *));
	
	float * h_var_12_5 = (float *)malloc(sizeof(float *));
	float * d_var_12_5;
	hipMalloc((void **)&d_var_12_5, sizeof(float *));
	
	float * h_var_12_6 = (float *)malloc(sizeof(float *));
	float * d_var_12_6;
	hipMalloc((void **)&d_var_12_6, sizeof(float *));
	
	float * h_var_12_7 = (float *)malloc(sizeof(float *));
	float * d_var_12_7;
	hipMalloc((void **)&d_var_12_7, sizeof(float *));
	
	float * h_var_12_8 = (float *)malloc(sizeof(float *));
	float * d_var_12_8;
	hipMalloc((void **)&d_var_12_8, sizeof(float *));
	
	float * h_var_12_9 = (float *)malloc(sizeof(float *));
	float * d_var_12_9;
	hipMalloc((void **)&d_var_12_9, sizeof(float *));
	
	float * h_var_13_0 = (float *)malloc(sizeof(float *));
	float * d_var_13_0;
	hipMalloc((void **)&d_var_13_0, sizeof(float *));
	
	float * h_var_13_1 = (float *)malloc(sizeof(float *));
	float * d_var_13_1;
	hipMalloc((void **)&d_var_13_1, sizeof(float *));
	
	float * h_var_13_2 = (float *)malloc(sizeof(float *));
	float * d_var_13_2;
	hipMalloc((void **)&d_var_13_2, sizeof(float *));
	
	float * h_var_13_3 = (float *)malloc(sizeof(float *));
	float * d_var_13_3;
	hipMalloc((void **)&d_var_13_3, sizeof(float *));
	
	float * h_var_13_4 = (float *)malloc(sizeof(float *));
	float * d_var_13_4;
	hipMalloc((void **)&d_var_13_4, sizeof(float *));
	
	float * h_var_13_5 = (float *)malloc(sizeof(float *));
	float * d_var_13_5;
	hipMalloc((void **)&d_var_13_5, sizeof(float *));
	
	float * h_var_13_6 = (float *)malloc(sizeof(float *));
	float * d_var_13_6;
	hipMalloc((void **)&d_var_13_6, sizeof(float *));
	
	float * h_var_13_7 = (float *)malloc(sizeof(float *));
	float * d_var_13_7;
	hipMalloc((void **)&d_var_13_7, sizeof(float *));
	
	float * h_var_13_8 = (float *)malloc(sizeof(float *));
	float * d_var_13_8;
	hipMalloc((void **)&d_var_13_8, sizeof(float *));
	
	float * h_var_13_9 = (float *)malloc(sizeof(float *));
	float * d_var_13_9;
	hipMalloc((void **)&d_var_13_9, sizeof(float *));
	
	float * h_var_14_0 = (float *)malloc(sizeof(float *));
	float * d_var_14_0;
	hipMalloc((void **)&d_var_14_0, sizeof(float *));
	
	float * h_var_14_1 = (float *)malloc(sizeof(float *));
	float * d_var_14_1;
	hipMalloc((void **)&d_var_14_1, sizeof(float *));
	
	float * h_var_14_2 = (float *)malloc(sizeof(float *));
	float * d_var_14_2;
	hipMalloc((void **)&d_var_14_2, sizeof(float *));
	
	float * h_var_14_3 = (float *)malloc(sizeof(float *));
	float * d_var_14_3;
	hipMalloc((void **)&d_var_14_3, sizeof(float *));
	
	float * h_var_14_4 = (float *)malloc(sizeof(float *));
	float * d_var_14_4;
	hipMalloc((void **)&d_var_14_4, sizeof(float *));
	
	float * h_var_14_5 = (float *)malloc(sizeof(float *));
	float * d_var_14_5;
	hipMalloc((void **)&d_var_14_5, sizeof(float *));
	
	float * h_var_14_6 = (float *)malloc(sizeof(float *));
	float * d_var_14_6;
	hipMalloc((void **)&d_var_14_6, sizeof(float *));
	
	float * h_var_14_7 = (float *)malloc(sizeof(float *));
	float * d_var_14_7;
	hipMalloc((void **)&d_var_14_7, sizeof(float *));
	
	float * h_var_14_8 = (float *)malloc(sizeof(float *));
	float * d_var_14_8;
	hipMalloc((void **)&d_var_14_8, sizeof(float *));
	
	float * h_var_14_9 = (float *)malloc(sizeof(float *));
	float * d_var_14_9;
	hipMalloc((void **)&d_var_14_9, sizeof(float *));
	
	float * h_var_15_0 = (float *)malloc(sizeof(float *));
	float * d_var_15_0;
	hipMalloc((void **)&d_var_15_0, sizeof(float *));
	
	float * h_var_15_1 = (float *)malloc(sizeof(float *));
	float * d_var_15_1;
	hipMalloc((void **)&d_var_15_1, sizeof(float *));
	
	float * h_var_15_2 = (float *)malloc(sizeof(float *));
	float * d_var_15_2;
	hipMalloc((void **)&d_var_15_2, sizeof(float *));
	
	float * h_var_15_3 = (float *)malloc(sizeof(float *));
	float * d_var_15_3;
	hipMalloc((void **)&d_var_15_3, sizeof(float *));
	
	float * h_var_15_4 = (float *)malloc(sizeof(float *));
	float * d_var_15_4;
	hipMalloc((void **)&d_var_15_4, sizeof(float *));
	
	float * h_var_15_5 = (float *)malloc(sizeof(float *));
	float * d_var_15_5;
	hipMalloc((void **)&d_var_15_5, sizeof(float *));
	
	float * h_var_15_6 = (float *)malloc(sizeof(float *));
	float * d_var_15_6;
	hipMalloc((void **)&d_var_15_6, sizeof(float *));
	
	float * h_var_15_7 = (float *)malloc(sizeof(float *));
	float * d_var_15_7;
	hipMalloc((void **)&d_var_15_7, sizeof(float *));
	
	float * h_var_15_8 = (float *)malloc(sizeof(float *));
	float * d_var_15_8;
	hipMalloc((void **)&d_var_15_8, sizeof(float *));
	
	float * h_var_15_9 = (float *)malloc(sizeof(float *));
	float * d_var_15_9;
	hipMalloc((void **)&d_var_15_9, sizeof(float *));
	
	float * h_var_16_0 = (float *)malloc(sizeof(float *));
	float * d_var_16_0;
	hipMalloc((void **)&d_var_16_0, sizeof(float *));
	
	float * h_var_16_1 = (float *)malloc(sizeof(float *));
	float * d_var_16_1;
	hipMalloc((void **)&d_var_16_1, sizeof(float *));
	
	float * h_var_16_2 = (float *)malloc(sizeof(float *));
	float * d_var_16_2;
	hipMalloc((void **)&d_var_16_2, sizeof(float *));
	
	float * h_var_16_3 = (float *)malloc(sizeof(float *));
	float * d_var_16_3;
	hipMalloc((void **)&d_var_16_3, sizeof(float *));
	
	float * h_var_16_4 = (float *)malloc(sizeof(float *));
	float * d_var_16_4;
	hipMalloc((void **)&d_var_16_4, sizeof(float *));
	
	float * h_var_16_5 = (float *)malloc(sizeof(float *));
	float * d_var_16_5;
	hipMalloc((void **)&d_var_16_5, sizeof(float *));
	
	float * h_var_16_6 = (float *)malloc(sizeof(float *));
	float * d_var_16_6;
	hipMalloc((void **)&d_var_16_6, sizeof(float *));
	
	float * h_var_16_7 = (float *)malloc(sizeof(float *));
	float * d_var_16_7;
	hipMalloc((void **)&d_var_16_7, sizeof(float *));
	
	float * h_var_16_8 = (float *)malloc(sizeof(float *));
	float * d_var_16_8;
	hipMalloc((void **)&d_var_16_8, sizeof(float *));
	
	float * h_var_16_9 = (float *)malloc(sizeof(float *));
	float * d_var_16_9;
	hipMalloc((void **)&d_var_16_9, sizeof(float *));
	
	float * h_var_17_0 = (float *)malloc(sizeof(float *));
	float * d_var_17_0;
	hipMalloc((void **)&d_var_17_0, sizeof(float *));
	
	float * h_var_17_1 = (float *)malloc(sizeof(float *));
	float * d_var_17_1;
	hipMalloc((void **)&d_var_17_1, sizeof(float *));
	
	float * h_var_17_2 = (float *)malloc(sizeof(float *));
	float * d_var_17_2;
	hipMalloc((void **)&d_var_17_2, sizeof(float *));
	
	float * h_var_17_3 = (float *)malloc(sizeof(float *));
	float * d_var_17_3;
	hipMalloc((void **)&d_var_17_3, sizeof(float *));
	
	float * h_var_17_4 = (float *)malloc(sizeof(float *));
	float * d_var_17_4;
	hipMalloc((void **)&d_var_17_4, sizeof(float *));
	
	float * h_var_17_5 = (float *)malloc(sizeof(float *));
	float * d_var_17_5;
	hipMalloc((void **)&d_var_17_5, sizeof(float *));
	
	float * h_var_17_6 = (float *)malloc(sizeof(float *));
	float * d_var_17_6;
	hipMalloc((void **)&d_var_17_6, sizeof(float *));
	
	float * h_var_17_7 = (float *)malloc(sizeof(float *));
	float * d_var_17_7;
	hipMalloc((void **)&d_var_17_7, sizeof(float *));
	
	float * h_var_17_8 = (float *)malloc(sizeof(float *));
	float * d_var_17_8;
	hipMalloc((void **)&d_var_17_8, sizeof(float *));
	
	float * h_var_17_9 = (float *)malloc(sizeof(float *));
	float * d_var_17_9;
	hipMalloc((void **)&d_var_17_9, sizeof(float *));
	
	float * h_var_18_0 = (float *)malloc(sizeof(float *));
	float * d_var_18_0;
	hipMalloc((void **)&d_var_18_0, sizeof(float *));
	
	float * h_var_18_1 = (float *)malloc(sizeof(float *));
	float * d_var_18_1;
	hipMalloc((void **)&d_var_18_1, sizeof(float *));
	
	float * h_var_18_2 = (float *)malloc(sizeof(float *));
	float * d_var_18_2;
	hipMalloc((void **)&d_var_18_2, sizeof(float *));
	
	float * h_var_18_3 = (float *)malloc(sizeof(float *));
	float * d_var_18_3;
	hipMalloc((void **)&d_var_18_3, sizeof(float *));
	
	float * h_var_18_4 = (float *)malloc(sizeof(float *));
	float * d_var_18_4;
	hipMalloc((void **)&d_var_18_4, sizeof(float *));
	
	float * h_var_18_5 = (float *)malloc(sizeof(float *));
	float * d_var_18_5;
	hipMalloc((void **)&d_var_18_5, sizeof(float *));
	
	float * h_var_18_6 = (float *)malloc(sizeof(float *));
	float * d_var_18_6;
	hipMalloc((void **)&d_var_18_6, sizeof(float *));
	
	float * h_var_18_7 = (float *)malloc(sizeof(float *));
	float * d_var_18_7;
	hipMalloc((void **)&d_var_18_7, sizeof(float *));
	
	float * h_var_18_8 = (float *)malloc(sizeof(float *));
	float * d_var_18_8;
	hipMalloc((void **)&d_var_18_8, sizeof(float *));
	
	float * h_var_18_9 = (float *)malloc(sizeof(float *));
	float * d_var_18_9;
	hipMalloc((void **)&d_var_18_9, sizeof(float *));
	
	float * h_var_19_0 = (float *)malloc(sizeof(float *));
	float * d_var_19_0;
	hipMalloc((void **)&d_var_19_0, sizeof(float *));
	
	float * h_var_19_1 = (float *)malloc(sizeof(float *));
	float * d_var_19_1;
	hipMalloc((void **)&d_var_19_1, sizeof(float *));
	
	float * h_var_19_2 = (float *)malloc(sizeof(float *));
	float * d_var_19_2;
	hipMalloc((void **)&d_var_19_2, sizeof(float *));
	
	float * h_var_19_3 = (float *)malloc(sizeof(float *));
	float * d_var_19_3;
	hipMalloc((void **)&d_var_19_3, sizeof(float *));
	
	float * h_var_19_4 = (float *)malloc(sizeof(float *));
	float * d_var_19_4;
	hipMalloc((void **)&d_var_19_4, sizeof(float *));
	
	float * h_var_19_5 = (float *)malloc(sizeof(float *));
	float * d_var_19_5;
	hipMalloc((void **)&d_var_19_5, sizeof(float *));
	
	float * h_var_19_6 = (float *)malloc(sizeof(float *));
	float * d_var_19_6;
	hipMalloc((void **)&d_var_19_6, sizeof(float *));
	
	float * h_var_19_7 = (float *)malloc(sizeof(float *));
	float * d_var_19_7;
	hipMalloc((void **)&d_var_19_7, sizeof(float *));
	
	float * h_var_19_8 = (float *)malloc(sizeof(float *));
	float * d_var_19_8;
	hipMalloc((void **)&d_var_19_8, sizeof(float *));
	
	float * h_var_19_9 = (float *)malloc(sizeof(float *));
	float * d_var_19_9;
	hipMalloc((void **)&d_var_19_9, sizeof(float *));
	
	float * h_var_20_0 = (float *)malloc(sizeof(float *));
	float * d_var_20_0;
	hipMalloc((void **)&d_var_20_0, sizeof(float *));
	
	float * h_var_20_1 = (float *)malloc(sizeof(float *));
	float * d_var_20_1;
	hipMalloc((void **)&d_var_20_1, sizeof(float *));
	
	float * h_var_20_2 = (float *)malloc(sizeof(float *));
	float * d_var_20_2;
	hipMalloc((void **)&d_var_20_2, sizeof(float *));
	
	float * h_var_20_3 = (float *)malloc(sizeof(float *));
	float * d_var_20_3;
	hipMalloc((void **)&d_var_20_3, sizeof(float *));
	
	float * h_var_20_4 = (float *)malloc(sizeof(float *));
	float * d_var_20_4;
	hipMalloc((void **)&d_var_20_4, sizeof(float *));
	
	float * h_var_20_5 = (float *)malloc(sizeof(float *));
	float * d_var_20_5;
	hipMalloc((void **)&d_var_20_5, sizeof(float *));
	
	float * h_var_20_6 = (float *)malloc(sizeof(float *));
	float * d_var_20_6;
	hipMalloc((void **)&d_var_20_6, sizeof(float *));
	
	float * h_var_20_7 = (float *)malloc(sizeof(float *));
	float * d_var_20_7;
	hipMalloc((void **)&d_var_20_7, sizeof(float *));
	
	float * h_var_20_8 = (float *)malloc(sizeof(float *));
	float * d_var_20_8;
	hipMalloc((void **)&d_var_20_8, sizeof(float *));
	
	float * h_var_20_9 = (float *)malloc(sizeof(float *));
	float * d_var_20_9;
	hipMalloc((void **)&d_var_20_9, sizeof(float *));
	
	float * h_var_21_0 = (float *)malloc(sizeof(float *));
	float * d_var_21_0;
	hipMalloc((void **)&d_var_21_0, sizeof(float *));
	
	float * h_var_21_1 = (float *)malloc(sizeof(float *));
	float * d_var_21_1;
	hipMalloc((void **)&d_var_21_1, sizeof(float *));
	
	float * h_var_21_2 = (float *)malloc(sizeof(float *));
	float * d_var_21_2;
	hipMalloc((void **)&d_var_21_2, sizeof(float *));
	
	float * h_var_21_3 = (float *)malloc(sizeof(float *));
	float * d_var_21_3;
	hipMalloc((void **)&d_var_21_3, sizeof(float *));
	
	float * h_var_21_4 = (float *)malloc(sizeof(float *));
	float * d_var_21_4;
	hipMalloc((void **)&d_var_21_4, sizeof(float *));
	
	float * h_var_21_5 = (float *)malloc(sizeof(float *));
	float * d_var_21_5;
	hipMalloc((void **)&d_var_21_5, sizeof(float *));
	
	float * h_var_21_6 = (float *)malloc(sizeof(float *));
	float * d_var_21_6;
	hipMalloc((void **)&d_var_21_6, sizeof(float *));
	
	float * h_var_21_7 = (float *)malloc(sizeof(float *));
	float * d_var_21_7;
	hipMalloc((void **)&d_var_21_7, sizeof(float *));
	
	float * h_var_21_8 = (float *)malloc(sizeof(float *));
	float * d_var_21_8;
	hipMalloc((void **)&d_var_21_8, sizeof(float *));
	
	float * h_var_21_9 = (float *)malloc(sizeof(float *));
	float * d_var_21_9;
	hipMalloc((void **)&d_var_21_9, sizeof(float *));
	
	float * h_var_22_0 = (float *)malloc(sizeof(float *));
	float * d_var_22_0;
	hipMalloc((void **)&d_var_22_0, sizeof(float *));
	
	float * h_var_22_1 = (float *)malloc(sizeof(float *));
	float * d_var_22_1;
	hipMalloc((void **)&d_var_22_1, sizeof(float *));
	
	float * h_var_22_2 = (float *)malloc(sizeof(float *));
	float * d_var_22_2;
	hipMalloc((void **)&d_var_22_2, sizeof(float *));
	
	float * h_var_22_3 = (float *)malloc(sizeof(float *));
	float * d_var_22_3;
	hipMalloc((void **)&d_var_22_3, sizeof(float *));
	
	float * h_var_22_4 = (float *)malloc(sizeof(float *));
	float * d_var_22_4;
	hipMalloc((void **)&d_var_22_4, sizeof(float *));
	
	float * h_var_22_5 = (float *)malloc(sizeof(float *));
	float * d_var_22_5;
	hipMalloc((void **)&d_var_22_5, sizeof(float *));
	
	float * h_var_22_6 = (float *)malloc(sizeof(float *));
	float * d_var_22_6;
	hipMalloc((void **)&d_var_22_6, sizeof(float *));
	
	float * h_var_22_7 = (float *)malloc(sizeof(float *));
	float * d_var_22_7;
	hipMalloc((void **)&d_var_22_7, sizeof(float *));
	
	float * h_var_22_8 = (float *)malloc(sizeof(float *));
	float * d_var_22_8;
	hipMalloc((void **)&d_var_22_8, sizeof(float *));
	
	float * h_var_22_9 = (float *)malloc(sizeof(float *));
	float * d_var_22_9;
	hipMalloc((void **)&d_var_22_9, sizeof(float *));
	
	float * h_var_23_0 = (float *)malloc(sizeof(float *));
	float * d_var_23_0;
	hipMalloc((void **)&d_var_23_0, sizeof(float *));
	
	float * h_var_23_1 = (float *)malloc(sizeof(float *));
	float * d_var_23_1;
	hipMalloc((void **)&d_var_23_1, sizeof(float *));
	
	float * h_var_23_2 = (float *)malloc(sizeof(float *));
	float * d_var_23_2;
	hipMalloc((void **)&d_var_23_2, sizeof(float *));
	
	float * h_var_23_3 = (float *)malloc(sizeof(float *));
	float * d_var_23_3;
	hipMalloc((void **)&d_var_23_3, sizeof(float *));
	
	float * h_var_23_4 = (float *)malloc(sizeof(float *));
	float * d_var_23_4;
	hipMalloc((void **)&d_var_23_4, sizeof(float *));
	
	float * h_var_23_5 = (float *)malloc(sizeof(float *));
	float * d_var_23_5;
	hipMalloc((void **)&d_var_23_5, sizeof(float *));
	
	float * h_var_23_6 = (float *)malloc(sizeof(float *));
	float * d_var_23_6;
	hipMalloc((void **)&d_var_23_6, sizeof(float *));
	
	float * h_var_23_7 = (float *)malloc(sizeof(float *));
	float * d_var_23_7;
	hipMalloc((void **)&d_var_23_7, sizeof(float *));
	
	float * h_var_23_8 = (float *)malloc(sizeof(float *));
	float * d_var_23_8;
	hipMalloc((void **)&d_var_23_8, sizeof(float *));
	
	float * h_var_23_9 = (float *)malloc(sizeof(float *));
	float * d_var_23_9;
	hipMalloc((void **)&d_var_23_9, sizeof(float *));
	
	float * h_var_24_0 = (float *)malloc(sizeof(float *));
	float * d_var_24_0;
	hipMalloc((void **)&d_var_24_0, sizeof(float *));
	
	float * h_var_24_1 = (float *)malloc(sizeof(float *));
	float * d_var_24_1;
	hipMalloc((void **)&d_var_24_1, sizeof(float *));
	
	float * h_var_24_2 = (float *)malloc(sizeof(float *));
	float * d_var_24_2;
	hipMalloc((void **)&d_var_24_2, sizeof(float *));
	
	float * h_var_24_3 = (float *)malloc(sizeof(float *));
	float * d_var_24_3;
	hipMalloc((void **)&d_var_24_3, sizeof(float *));
	
	float * h_var_24_4 = (float *)malloc(sizeof(float *));
	float * d_var_24_4;
	hipMalloc((void **)&d_var_24_4, sizeof(float *));
	
	float * h_var_24_5 = (float *)malloc(sizeof(float *));
	float * d_var_24_5;
	hipMalloc((void **)&d_var_24_5, sizeof(float *));
	
	float * h_var_24_6 = (float *)malloc(sizeof(float *));
	float * d_var_24_6;
	hipMalloc((void **)&d_var_24_6, sizeof(float *));
	
	float * h_var_24_7 = (float *)malloc(sizeof(float *));
	float * d_var_24_7;
	hipMalloc((void **)&d_var_24_7, sizeof(float *));
	
	float * h_var_24_8 = (float *)malloc(sizeof(float *));
	float * d_var_24_8;
	hipMalloc((void **)&d_var_24_8, sizeof(float *));
	
	float * h_var_24_9 = (float *)malloc(sizeof(float *));
	float * d_var_24_9;
	hipMalloc((void **)&d_var_24_9, sizeof(float *));
	
	float * h_var_25_0 = (float *)malloc(sizeof(float *));
	float * d_var_25_0;
	hipMalloc((void **)&d_var_25_0, sizeof(float *));
	
	float * h_var_25_1 = (float *)malloc(sizeof(float *));
	float * d_var_25_1;
	hipMalloc((void **)&d_var_25_1, sizeof(float *));
	
	float * h_var_25_2 = (float *)malloc(sizeof(float *));
	float * d_var_25_2;
	hipMalloc((void **)&d_var_25_2, sizeof(float *));
	
	float * h_var_25_3 = (float *)malloc(sizeof(float *));
	float * d_var_25_3;
	hipMalloc((void **)&d_var_25_3, sizeof(float *));
	
	float * h_var_25_4 = (float *)malloc(sizeof(float *));
	float * d_var_25_4;
	hipMalloc((void **)&d_var_25_4, sizeof(float *));
	
	float * h_var_25_5 = (float *)malloc(sizeof(float *));
	float * d_var_25_5;
	hipMalloc((void **)&d_var_25_5, sizeof(float *));
	
	float * h_var_25_6 = (float *)malloc(sizeof(float *));
	float * d_var_25_6;
	hipMalloc((void **)&d_var_25_6, sizeof(float *));
	
	float * h_var_25_7 = (float *)malloc(sizeof(float *));
	float * d_var_25_7;
	hipMalloc((void **)&d_var_25_7, sizeof(float *));
	
	float * h_var_25_8 = (float *)malloc(sizeof(float *));
	float * d_var_25_8;
	hipMalloc((void **)&d_var_25_8, sizeof(float *));
	
	float * h_var_25_9 = (float *)malloc(sizeof(float *));
	float * d_var_25_9;
	hipMalloc((void **)&d_var_25_9, sizeof(float *));
	
	float * h_var_26_0 = (float *)malloc(sizeof(float *));
	float * d_var_26_0;
	hipMalloc((void **)&d_var_26_0, sizeof(float *));
	
	float * h_var_26_1 = (float *)malloc(sizeof(float *));
	float * d_var_26_1;
	hipMalloc((void **)&d_var_26_1, sizeof(float *));
	
	float * h_var_26_2 = (float *)malloc(sizeof(float *));
	float * d_var_26_2;
	hipMalloc((void **)&d_var_26_2, sizeof(float *));
	
	float * h_var_26_3 = (float *)malloc(sizeof(float *));
	float * d_var_26_3;
	hipMalloc((void **)&d_var_26_3, sizeof(float *));
	
	float * h_var_26_4 = (float *)malloc(sizeof(float *));
	float * d_var_26_4;
	hipMalloc((void **)&d_var_26_4, sizeof(float *));
	
	float * h_var_26_5 = (float *)malloc(sizeof(float *));
	float * d_var_26_5;
	hipMalloc((void **)&d_var_26_5, sizeof(float *));
	
	float * h_var_26_6 = (float *)malloc(sizeof(float *));
	float * d_var_26_6;
	hipMalloc((void **)&d_var_26_6, sizeof(float *));
	
	float * h_var_26_7 = (float *)malloc(sizeof(float *));
	float * d_var_26_7;
	hipMalloc((void **)&d_var_26_7, sizeof(float *));
	
	float * h_var_26_8 = (float *)malloc(sizeof(float *));
	float * d_var_26_8;
	hipMalloc((void **)&d_var_26_8, sizeof(float *));
	
	float * h_var_26_9 = (float *)malloc(sizeof(float *));
	float * d_var_26_9;
	hipMalloc((void **)&d_var_26_9, sizeof(float *));
	
	float * h_var_27_0 = (float *)malloc(sizeof(float *));
	float * d_var_27_0;
	hipMalloc((void **)&d_var_27_0, sizeof(float *));
	
	float * h_var_27_1 = (float *)malloc(sizeof(float *));
	float * d_var_27_1;
	hipMalloc((void **)&d_var_27_1, sizeof(float *));
	
	float * h_var_27_2 = (float *)malloc(sizeof(float *));
	float * d_var_27_2;
	hipMalloc((void **)&d_var_27_2, sizeof(float *));
	
	float * h_var_27_3 = (float *)malloc(sizeof(float *));
	float * d_var_27_3;
	hipMalloc((void **)&d_var_27_3, sizeof(float *));
	
	float * h_var_27_4 = (float *)malloc(sizeof(float *));
	float * d_var_27_4;
	hipMalloc((void **)&d_var_27_4, sizeof(float *));
	
	float * h_var_27_5 = (float *)malloc(sizeof(float *));
	float * d_var_27_5;
	hipMalloc((void **)&d_var_27_5, sizeof(float *));
	
	float * h_var_27_6 = (float *)malloc(sizeof(float *));
	float * d_var_27_6;
	hipMalloc((void **)&d_var_27_6, sizeof(float *));
	
	float * h_var_27_7 = (float *)malloc(sizeof(float *));
	float * d_var_27_7;
	hipMalloc((void **)&d_var_27_7, sizeof(float *));
	
	float * h_var_27_8 = (float *)malloc(sizeof(float *));
	float * d_var_27_8;
	hipMalloc((void **)&d_var_27_8, sizeof(float *));
	
	float * h_var_27_9 = (float *)malloc(sizeof(float *));
	float * d_var_27_9;
	hipMalloc((void **)&d_var_27_9, sizeof(float *));
	
	float * h_var_28_0 = (float *)malloc(sizeof(float *));
	float * d_var_28_0;
	hipMalloc((void **)&d_var_28_0, sizeof(float *));
	
	float * h_var_28_1 = (float *)malloc(sizeof(float *));
	float * d_var_28_1;
	hipMalloc((void **)&d_var_28_1, sizeof(float *));
	
	float * h_var_28_2 = (float *)malloc(sizeof(float *));
	float * d_var_28_2;
	hipMalloc((void **)&d_var_28_2, sizeof(float *));
	
	float * h_var_28_3 = (float *)malloc(sizeof(float *));
	float * d_var_28_3;
	hipMalloc((void **)&d_var_28_3, sizeof(float *));
	
	float * h_var_28_4 = (float *)malloc(sizeof(float *));
	float * d_var_28_4;
	hipMalloc((void **)&d_var_28_4, sizeof(float *));
	
	float * h_var_28_5 = (float *)malloc(sizeof(float *));
	float * d_var_28_5;
	hipMalloc((void **)&d_var_28_5, sizeof(float *));
	
	float * h_var_28_6 = (float *)malloc(sizeof(float *));
	float * d_var_28_6;
	hipMalloc((void **)&d_var_28_6, sizeof(float *));
	
	float * h_var_28_7 = (float *)malloc(sizeof(float *));
	float * d_var_28_7;
	hipMalloc((void **)&d_var_28_7, sizeof(float *));
	
	float * h_var_28_8 = (float *)malloc(sizeof(float *));
	float * d_var_28_8;
	hipMalloc((void **)&d_var_28_8, sizeof(float *));
	
	float * h_var_28_9 = (float *)malloc(sizeof(float *));
	float * d_var_28_9;
	hipMalloc((void **)&d_var_28_9, sizeof(float *));
	
	float * h_var_29_0 = (float *)malloc(sizeof(float *));
	float * d_var_29_0;
	hipMalloc((void **)&d_var_29_0, sizeof(float *));
	
	float * h_var_29_1 = (float *)malloc(sizeof(float *));
	float * d_var_29_1;
	hipMalloc((void **)&d_var_29_1, sizeof(float *));
	
	float * h_var_29_2 = (float *)malloc(sizeof(float *));
	float * d_var_29_2;
	hipMalloc((void **)&d_var_29_2, sizeof(float *));
	
	float * h_var_29_3 = (float *)malloc(sizeof(float *));
	float * d_var_29_3;
	hipMalloc((void **)&d_var_29_3, sizeof(float *));
	
	float * h_var_29_4 = (float *)malloc(sizeof(float *));
	float * d_var_29_4;
	hipMalloc((void **)&d_var_29_4, sizeof(float *));
	
	float * h_var_29_5 = (float *)malloc(sizeof(float *));
	float * d_var_29_5;
	hipMalloc((void **)&d_var_29_5, sizeof(float *));
	
	float * h_var_29_6 = (float *)malloc(sizeof(float *));
	float * d_var_29_6;
	hipMalloc((void **)&d_var_29_6, sizeof(float *));
	
	float * h_var_29_7 = (float *)malloc(sizeof(float *));
	float * d_var_29_7;
	hipMalloc((void **)&d_var_29_7, sizeof(float *));
	
	float * h_var_29_8 = (float *)malloc(sizeof(float *));
	float * d_var_29_8;
	hipMalloc((void **)&d_var_29_8, sizeof(float *));
	
	float * h_var_29_9 = (float *)malloc(sizeof(float *));
	float * d_var_29_9;
	hipMalloc((void **)&d_var_29_9, sizeof(float *));
	
	float * h_var_30_0 = (float *)malloc(sizeof(float *));
	float * d_var_30_0;
	hipMalloc((void **)&d_var_30_0, sizeof(float *));
	
	float * h_var_30_1 = (float *)malloc(sizeof(float *));
	float * d_var_30_1;
	hipMalloc((void **)&d_var_30_1, sizeof(float *));
	
	float * h_var_30_2 = (float *)malloc(sizeof(float *));
	float * d_var_30_2;
	hipMalloc((void **)&d_var_30_2, sizeof(float *));
	
	float * h_var_30_3 = (float *)malloc(sizeof(float *));
	float * d_var_30_3;
	hipMalloc((void **)&d_var_30_3, sizeof(float *));
	
	float * h_var_30_4 = (float *)malloc(sizeof(float *));
	float * d_var_30_4;
	hipMalloc((void **)&d_var_30_4, sizeof(float *));
	
	float * h_var_30_5 = (float *)malloc(sizeof(float *));
	float * d_var_30_5;
	hipMalloc((void **)&d_var_30_5, sizeof(float *));
	
	float * h_var_30_6 = (float *)malloc(sizeof(float *));
	float * d_var_30_6;
	hipMalloc((void **)&d_var_30_6, sizeof(float *));
	
	float * h_var_30_7 = (float *)malloc(sizeof(float *));
	float * d_var_30_7;
	hipMalloc((void **)&d_var_30_7, sizeof(float *));
	
	float * h_var_30_8 = (float *)malloc(sizeof(float *));
	float * d_var_30_8;
	hipMalloc((void **)&d_var_30_8, sizeof(float *));
	
	float * h_var_30_9 = (float *)malloc(sizeof(float *));
	float * d_var_30_9;
	hipMalloc((void **)&d_var_30_9, sizeof(float *));
	
	float * h_var_31_0 = (float *)malloc(sizeof(float *));
	float * d_var_31_0;
	hipMalloc((void **)&d_var_31_0, sizeof(float *));
	
	float * h_var_31_1 = (float *)malloc(sizeof(float *));
	float * d_var_31_1;
	hipMalloc((void **)&d_var_31_1, sizeof(float *));
	
	float * h_var_31_2 = (float *)malloc(sizeof(float *));
	float * d_var_31_2;
	hipMalloc((void **)&d_var_31_2, sizeof(float *));
	
	float * h_var_31_3 = (float *)malloc(sizeof(float *));
	float * d_var_31_3;
	hipMalloc((void **)&d_var_31_3, sizeof(float *));
	
	float * h_var_31_4 = (float *)malloc(sizeof(float *));
	float * d_var_31_4;
	hipMalloc((void **)&d_var_31_4, sizeof(float *));
	
	float * h_var_31_5 = (float *)malloc(sizeof(float *));
	float * d_var_31_5;
	hipMalloc((void **)&d_var_31_5, sizeof(float *));
	
	float * h_var_31_6 = (float *)malloc(sizeof(float *));
	float * d_var_31_6;
	hipMalloc((void **)&d_var_31_6, sizeof(float *));
	
	float * h_var_31_7 = (float *)malloc(sizeof(float *));
	float * d_var_31_7;
	hipMalloc((void **)&d_var_31_7, sizeof(float *));
	
	float * h_var_31_8 = (float *)malloc(sizeof(float *));
	float * d_var_31_8;
	hipMalloc((void **)&d_var_31_8, sizeof(float *));
	
	float * h_var_31_9 = (float *)malloc(sizeof(float *));
	float * d_var_31_9;
	hipMalloc((void **)&d_var_31_9, sizeof(float *));
	
	float * h_var_32_0 = (float *)malloc(sizeof(float *));
	float * d_var_32_0;
	hipMalloc((void **)&d_var_32_0, sizeof(float *));
	
	float * h_var_32_1 = (float *)malloc(sizeof(float *));
	float * d_var_32_1;
	hipMalloc((void **)&d_var_32_1, sizeof(float *));
	
	float * h_var_32_2 = (float *)malloc(sizeof(float *));
	float * d_var_32_2;
	hipMalloc((void **)&d_var_32_2, sizeof(float *));
	
	float * h_var_32_3 = (float *)malloc(sizeof(float *));
	float * d_var_32_3;
	hipMalloc((void **)&d_var_32_3, sizeof(float *));
	
	float * h_var_32_4 = (float *)malloc(sizeof(float *));
	float * d_var_32_4;
	hipMalloc((void **)&d_var_32_4, sizeof(float *));
	
	float * h_var_32_5 = (float *)malloc(sizeof(float *));
	float * d_var_32_5;
	hipMalloc((void **)&d_var_32_5, sizeof(float *));
	
	float * h_var_32_6 = (float *)malloc(sizeof(float *));
	float * d_var_32_6;
	hipMalloc((void **)&d_var_32_6, sizeof(float *));
	
	float * h_var_32_7 = (float *)malloc(sizeof(float *));
	float * d_var_32_7;
	hipMalloc((void **)&d_var_32_7, sizeof(float *));
	
	float * h_var_32_8 = (float *)malloc(sizeof(float *));
	float * d_var_32_8;
	hipMalloc((void **)&d_var_32_8, sizeof(float *));
	
	float * h_var_32_9 = (float *)malloc(sizeof(float *));
	float * d_var_32_9;
	hipMalloc((void **)&d_var_32_9, sizeof(float *));
	
	float * h_var_33_0 = (float *)malloc(sizeof(float *));
	float * d_var_33_0;
	hipMalloc((void **)&d_var_33_0, sizeof(float *));
	
	float * h_var_33_1 = (float *)malloc(sizeof(float *));
	float * d_var_33_1;
	hipMalloc((void **)&d_var_33_1, sizeof(float *));
	
	float * h_var_33_2 = (float *)malloc(sizeof(float *));
	float * d_var_33_2;
	hipMalloc((void **)&d_var_33_2, sizeof(float *));
	
	float * h_var_33_3 = (float *)malloc(sizeof(float *));
	float * d_var_33_3;
	hipMalloc((void **)&d_var_33_3, sizeof(float *));
	
	float * h_var_33_4 = (float *)malloc(sizeof(float *));
	float * d_var_33_4;
	hipMalloc((void **)&d_var_33_4, sizeof(float *));
	
	float * h_var_33_5 = (float *)malloc(sizeof(float *));
	float * d_var_33_5;
	hipMalloc((void **)&d_var_33_5, sizeof(float *));
	
	float * h_var_33_6 = (float *)malloc(sizeof(float *));
	float * d_var_33_6;
	hipMalloc((void **)&d_var_33_6, sizeof(float *));
	
	float * h_var_33_7 = (float *)malloc(sizeof(float *));
	float * d_var_33_7;
	hipMalloc((void **)&d_var_33_7, sizeof(float *));
	
	float * h_var_33_8 = (float *)malloc(sizeof(float *));
	float * d_var_33_8;
	hipMalloc((void **)&d_var_33_8, sizeof(float *));
	
	float * h_var_33_9 = (float *)malloc(sizeof(float *));
	float * d_var_33_9;
	hipMalloc((void **)&d_var_33_9, sizeof(float *));
	
	float * h_var_34_0 = (float *)malloc(sizeof(float *));
	float * d_var_34_0;
	hipMalloc((void **)&d_var_34_0, sizeof(float *));
	
	float * h_var_34_1 = (float *)malloc(sizeof(float *));
	float * d_var_34_1;
	hipMalloc((void **)&d_var_34_1, sizeof(float *));
	
	float * h_var_34_2 = (float *)malloc(sizeof(float *));
	float * d_var_34_2;
	hipMalloc((void **)&d_var_34_2, sizeof(float *));
	
	float * h_var_34_3 = (float *)malloc(sizeof(float *));
	float * d_var_34_3;
	hipMalloc((void **)&d_var_34_3, sizeof(float *));
	
	float * h_var_34_4 = (float *)malloc(sizeof(float *));
	float * d_var_34_4;
	hipMalloc((void **)&d_var_34_4, sizeof(float *));
	
	float * h_var_34_5 = (float *)malloc(sizeof(float *));
	float * d_var_34_5;
	hipMalloc((void **)&d_var_34_5, sizeof(float *));
	
	float * h_var_34_6 = (float *)malloc(sizeof(float *));
	float * d_var_34_6;
	hipMalloc((void **)&d_var_34_6, sizeof(float *));
	
	float * h_var_34_7 = (float *)malloc(sizeof(float *));
	float * d_var_34_7;
	hipMalloc((void **)&d_var_34_7, sizeof(float *));
	
	float * h_var_34_8 = (float *)malloc(sizeof(float *));
	float * d_var_34_8;
	hipMalloc((void **)&d_var_34_8, sizeof(float *));
	
	float * h_var_34_9 = (float *)malloc(sizeof(float *));
	float * d_var_34_9;
	hipMalloc((void **)&d_var_34_9, sizeof(float *));
	
	float * h_var_35_0 = (float *)malloc(sizeof(float *));
	float * d_var_35_0;
	hipMalloc((void **)&d_var_35_0, sizeof(float *));
	
	float * h_var_35_1 = (float *)malloc(sizeof(float *));
	float * d_var_35_1;
	hipMalloc((void **)&d_var_35_1, sizeof(float *));
	
	float * h_var_35_2 = (float *)malloc(sizeof(float *));
	float * d_var_35_2;
	hipMalloc((void **)&d_var_35_2, sizeof(float *));
	
	float * h_var_35_3 = (float *)malloc(sizeof(float *));
	float * d_var_35_3;
	hipMalloc((void **)&d_var_35_3, sizeof(float *));
	
	float * h_var_35_4 = (float *)malloc(sizeof(float *));
	float * d_var_35_4;
	hipMalloc((void **)&d_var_35_4, sizeof(float *));
	
	float * h_var_35_5 = (float *)malloc(sizeof(float *));
	float * d_var_35_5;
	hipMalloc((void **)&d_var_35_5, sizeof(float *));
	
	float * h_var_35_6 = (float *)malloc(sizeof(float *));
	float * d_var_35_6;
	hipMalloc((void **)&d_var_35_6, sizeof(float *));
	
	float * h_var_35_7 = (float *)malloc(sizeof(float *));
	float * d_var_35_7;
	hipMalloc((void **)&d_var_35_7, sizeof(float *));
	
	float * h_var_35_8 = (float *)malloc(sizeof(float *));
	float * d_var_35_8;
	hipMalloc((void **)&d_var_35_8, sizeof(float *));
	
	float * h_var_35_9 = (float *)malloc(sizeof(float *));
	float * d_var_35_9;
	hipMalloc((void **)&d_var_35_9, sizeof(float *));
	
	float * h_var_36_0 = (float *)malloc(sizeof(float *));
	float * d_var_36_0;
	hipMalloc((void **)&d_var_36_0, sizeof(float *));
	
	float * h_var_36_1 = (float *)malloc(sizeof(float *));
	float * d_var_36_1;
	hipMalloc((void **)&d_var_36_1, sizeof(float *));
	
	float * h_var_36_2 = (float *)malloc(sizeof(float *));
	float * d_var_36_2;
	hipMalloc((void **)&d_var_36_2, sizeof(float *));
	
	float * h_var_36_3 = (float *)malloc(sizeof(float *));
	float * d_var_36_3;
	hipMalloc((void **)&d_var_36_3, sizeof(float *));
	
	float * h_var_36_4 = (float *)malloc(sizeof(float *));
	float * d_var_36_4;
	hipMalloc((void **)&d_var_36_4, sizeof(float *));
	
	float * h_var_36_5 = (float *)malloc(sizeof(float *));
	float * d_var_36_5;
	hipMalloc((void **)&d_var_36_5, sizeof(float *));
	
	float * h_var_36_6 = (float *)malloc(sizeof(float *));
	float * d_var_36_6;
	hipMalloc((void **)&d_var_36_6, sizeof(float *));
	
	float * h_var_36_7 = (float *)malloc(sizeof(float *));
	float * d_var_36_7;
	hipMalloc((void **)&d_var_36_7, sizeof(float *));
	
	float * h_var_36_8 = (float *)malloc(sizeof(float *));
	float * d_var_36_8;
	hipMalloc((void **)&d_var_36_8, sizeof(float *));
	
	float * h_var_36_9 = (float *)malloc(sizeof(float *));
	float * d_var_36_9;
	hipMalloc((void **)&d_var_36_9, sizeof(float *));
	
	float * h_var_37_0 = (float *)malloc(sizeof(float *));
	float * d_var_37_0;
	hipMalloc((void **)&d_var_37_0, sizeof(float *));
	
	float * h_var_37_1 = (float *)malloc(sizeof(float *));
	float * d_var_37_1;
	hipMalloc((void **)&d_var_37_1, sizeof(float *));
	
	float * h_var_37_2 = (float *)malloc(sizeof(float *));
	float * d_var_37_2;
	hipMalloc((void **)&d_var_37_2, sizeof(float *));
	
	float * h_var_37_3 = (float *)malloc(sizeof(float *));
	float * d_var_37_3;
	hipMalloc((void **)&d_var_37_3, sizeof(float *));
	
	float * h_var_37_4 = (float *)malloc(sizeof(float *));
	float * d_var_37_4;
	hipMalloc((void **)&d_var_37_4, sizeof(float *));
	
	float * h_var_37_5 = (float *)malloc(sizeof(float *));
	float * d_var_37_5;
	hipMalloc((void **)&d_var_37_5, sizeof(float *));
	
	float * h_var_37_6 = (float *)malloc(sizeof(float *));
	float * d_var_37_6;
	hipMalloc((void **)&d_var_37_6, sizeof(float *));
	
	float * h_var_37_7 = (float *)malloc(sizeof(float *));
	float * d_var_37_7;
	hipMalloc((void **)&d_var_37_7, sizeof(float *));
	
	float * h_var_37_8 = (float *)malloc(sizeof(float *));
	float * d_var_37_8;
	hipMalloc((void **)&d_var_37_8, sizeof(float *));
	
	float * h_var_37_9 = (float *)malloc(sizeof(float *));
	float * d_var_37_9;
	hipMalloc((void **)&d_var_37_9, sizeof(float *));
	
	float * h_var_38_0 = (float *)malloc(sizeof(float *));
	float * d_var_38_0;
	hipMalloc((void **)&d_var_38_0, sizeof(float *));
	
	float * h_var_38_1 = (float *)malloc(sizeof(float *));
	float * d_var_38_1;
	hipMalloc((void **)&d_var_38_1, sizeof(float *));
	
	float * h_var_38_2 = (float *)malloc(sizeof(float *));
	float * d_var_38_2;
	hipMalloc((void **)&d_var_38_2, sizeof(float *));
	
	float * h_var_38_3 = (float *)malloc(sizeof(float *));
	float * d_var_38_3;
	hipMalloc((void **)&d_var_38_3, sizeof(float *));
	
	float * h_var_38_4 = (float *)malloc(sizeof(float *));
	float * d_var_38_4;
	hipMalloc((void **)&d_var_38_4, sizeof(float *));
	
	float * h_var_38_5 = (float *)malloc(sizeof(float *));
	float * d_var_38_5;
	hipMalloc((void **)&d_var_38_5, sizeof(float *));
	
	float * h_var_38_6 = (float *)malloc(sizeof(float *));
	float * d_var_38_6;
	hipMalloc((void **)&d_var_38_6, sizeof(float *));
	
	float * h_var_38_7 = (float *)malloc(sizeof(float *));
	float * d_var_38_7;
	hipMalloc((void **)&d_var_38_7, sizeof(float *));
	
	float * h_var_38_8 = (float *)malloc(sizeof(float *));
	float * d_var_38_8;
	hipMalloc((void **)&d_var_38_8, sizeof(float *));
	
	float * h_var_38_9 = (float *)malloc(sizeof(float *));
	float * d_var_38_9;
	hipMalloc((void **)&d_var_38_9, sizeof(float *));
	
	float * h_var_39_0 = (float *)malloc(sizeof(float *));
	float * d_var_39_0;
	hipMalloc((void **)&d_var_39_0, sizeof(float *));
	
	float * h_var_39_1 = (float *)malloc(sizeof(float *));
	float * d_var_39_1;
	hipMalloc((void **)&d_var_39_1, sizeof(float *));
	
	float * h_var_39_2 = (float *)malloc(sizeof(float *));
	float * d_var_39_2;
	hipMalloc((void **)&d_var_39_2, sizeof(float *));
	
	float * h_var_39_3 = (float *)malloc(sizeof(float *));
	float * d_var_39_3;
	hipMalloc((void **)&d_var_39_3, sizeof(float *));
	
	float * h_var_39_4 = (float *)malloc(sizeof(float *));
	float * d_var_39_4;
	hipMalloc((void **)&d_var_39_4, sizeof(float *));
	
	float * h_var_39_5 = (float *)malloc(sizeof(float *));
	float * d_var_39_5;
	hipMalloc((void **)&d_var_39_5, sizeof(float *));
	
	float * h_var_39_6 = (float *)malloc(sizeof(float *));
	float * d_var_39_6;
	hipMalloc((void **)&d_var_39_6, sizeof(float *));
	
	float * h_var_39_7 = (float *)malloc(sizeof(float *));
	float * d_var_39_7;
	hipMalloc((void **)&d_var_39_7, sizeof(float *));
	
	float * h_var_39_8 = (float *)malloc(sizeof(float *));
	float * d_var_39_8;
	hipMalloc((void **)&d_var_39_8, sizeof(float *));
	
	float * h_var_39_9 = (float *)malloc(sizeof(float *));
	float * d_var_39_9;
	hipMalloc((void **)&d_var_39_9, sizeof(float *));
	
	float * h_var_40_0 = (float *)malloc(sizeof(float *));
	float * d_var_40_0;
	hipMalloc((void **)&d_var_40_0, sizeof(float *));
	
	float * h_var_40_1 = (float *)malloc(sizeof(float *));
	float * d_var_40_1;
	hipMalloc((void **)&d_var_40_1, sizeof(float *));
	
	float * h_var_40_2 = (float *)malloc(sizeof(float *));
	float * d_var_40_2;
	hipMalloc((void **)&d_var_40_2, sizeof(float *));
	
	float * h_var_40_3 = (float *)malloc(sizeof(float *));
	float * d_var_40_3;
	hipMalloc((void **)&d_var_40_3, sizeof(float *));
	
	float * h_var_40_4 = (float *)malloc(sizeof(float *));
	float * d_var_40_4;
	hipMalloc((void **)&d_var_40_4, sizeof(float *));
	
	float * h_var_40_5 = (float *)malloc(sizeof(float *));
	float * d_var_40_5;
	hipMalloc((void **)&d_var_40_5, sizeof(float *));
	
	float * h_var_40_6 = (float *)malloc(sizeof(float *));
	float * d_var_40_6;
	hipMalloc((void **)&d_var_40_6, sizeof(float *));
	
	float * h_var_40_7 = (float *)malloc(sizeof(float *));
	float * d_var_40_7;
	hipMalloc((void **)&d_var_40_7, sizeof(float *));
	
	float * h_var_40_8 = (float *)malloc(sizeof(float *));
	float * d_var_40_8;
	hipMalloc((void **)&d_var_40_8, sizeof(float *));
	
	float * h_var_40_9 = (float *)malloc(sizeof(float *));
	float * d_var_40_9;
	hipMalloc((void **)&d_var_40_9, sizeof(float *));
	
	float * h_var_41_0 = (float *)malloc(sizeof(float *));
	float * d_var_41_0;
	hipMalloc((void **)&d_var_41_0, sizeof(float *));
	
	float * h_var_41_1 = (float *)malloc(sizeof(float *));
	float * d_var_41_1;
	hipMalloc((void **)&d_var_41_1, sizeof(float *));
	
	float * h_var_41_2 = (float *)malloc(sizeof(float *));
	float * d_var_41_2;
	hipMalloc((void **)&d_var_41_2, sizeof(float *));
	
	float * h_var_41_3 = (float *)malloc(sizeof(float *));
	float * d_var_41_3;
	hipMalloc((void **)&d_var_41_3, sizeof(float *));
	
	float * h_var_41_4 = (float *)malloc(sizeof(float *));
	float * d_var_41_4;
	hipMalloc((void **)&d_var_41_4, sizeof(float *));
	
	float * h_var_41_5 = (float *)malloc(sizeof(float *));
	float * d_var_41_5;
	hipMalloc((void **)&d_var_41_5, sizeof(float *));
	
	float * h_var_41_6 = (float *)malloc(sizeof(float *));
	float * d_var_41_6;
	hipMalloc((void **)&d_var_41_6, sizeof(float *));
	
	float * h_var_41_7 = (float *)malloc(sizeof(float *));
	float * d_var_41_7;
	hipMalloc((void **)&d_var_41_7, sizeof(float *));
	
	float * h_var_41_8 = (float *)malloc(sizeof(float *));
	float * d_var_41_8;
	hipMalloc((void **)&d_var_41_8, sizeof(float *));
	
	float * h_var_41_9 = (float *)malloc(sizeof(float *));
	float * d_var_41_9;
	hipMalloc((void **)&d_var_41_9, sizeof(float *));
	
	float * h_var_42_0 = (float *)malloc(sizeof(float *));
	float * d_var_42_0;
	hipMalloc((void **)&d_var_42_0, sizeof(float *));
	
	float * h_var_42_1 = (float *)malloc(sizeof(float *));
	float * d_var_42_1;
	hipMalloc((void **)&d_var_42_1, sizeof(float *));
	
	float * h_var_42_2 = (float *)malloc(sizeof(float *));
	float * d_var_42_2;
	hipMalloc((void **)&d_var_42_2, sizeof(float *));
	
	float * h_var_42_3 = (float *)malloc(sizeof(float *));
	float * d_var_42_3;
	hipMalloc((void **)&d_var_42_3, sizeof(float *));
	
	float * h_var_42_4 = (float *)malloc(sizeof(float *));
	float * d_var_42_4;
	hipMalloc((void **)&d_var_42_4, sizeof(float *));
	
	float * h_var_42_5 = (float *)malloc(sizeof(float *));
	float * d_var_42_5;
	hipMalloc((void **)&d_var_42_5, sizeof(float *));
	
	float * h_var_42_6 = (float *)malloc(sizeof(float *));
	float * d_var_42_6;
	hipMalloc((void **)&d_var_42_6, sizeof(float *));
	
	float * h_var_42_7 = (float *)malloc(sizeof(float *));
	float * d_var_42_7;
	hipMalloc((void **)&d_var_42_7, sizeof(float *));
	
	float * h_var_42_8 = (float *)malloc(sizeof(float *));
	float * d_var_42_8;
	hipMalloc((void **)&d_var_42_8, sizeof(float *));
	
	float * h_var_42_9 = (float *)malloc(sizeof(float *));
	float * d_var_42_9;
	hipMalloc((void **)&d_var_42_9, sizeof(float *));
	
	float * h_var_43_0 = (float *)malloc(sizeof(float *));
	float * d_var_43_0;
	hipMalloc((void **)&d_var_43_0, sizeof(float *));
	
	float * h_var_43_1 = (float *)malloc(sizeof(float *));
	float * d_var_43_1;
	hipMalloc((void **)&d_var_43_1, sizeof(float *));
	
	float * h_var_43_2 = (float *)malloc(sizeof(float *));
	float * d_var_43_2;
	hipMalloc((void **)&d_var_43_2, sizeof(float *));
	
	float * h_var_43_3 = (float *)malloc(sizeof(float *));
	float * d_var_43_3;
	hipMalloc((void **)&d_var_43_3, sizeof(float *));
	
	float * h_var_43_4 = (float *)malloc(sizeof(float *));
	float * d_var_43_4;
	hipMalloc((void **)&d_var_43_4, sizeof(float *));
	
	float * h_var_43_5 = (float *)malloc(sizeof(float *));
	float * d_var_43_5;
	hipMalloc((void **)&d_var_43_5, sizeof(float *));
	
	float * h_var_43_6 = (float *)malloc(sizeof(float *));
	float * d_var_43_6;
	hipMalloc((void **)&d_var_43_6, sizeof(float *));
	
	float * h_var_43_7 = (float *)malloc(sizeof(float *));
	float * d_var_43_7;
	hipMalloc((void **)&d_var_43_7, sizeof(float *));
	
	float * h_var_43_8 = (float *)malloc(sizeof(float *));
	float * d_var_43_8;
	hipMalloc((void **)&d_var_43_8, sizeof(float *));
	
	float * h_var_43_9 = (float *)malloc(sizeof(float *));
	float * d_var_43_9;
	hipMalloc((void **)&d_var_43_9, sizeof(float *));
	
	float * h_var_44_0 = (float *)malloc(sizeof(float *));
	float * d_var_44_0;
	hipMalloc((void **)&d_var_44_0, sizeof(float *));
	
	float * h_var_44_1 = (float *)malloc(sizeof(float *));
	float * d_var_44_1;
	hipMalloc((void **)&d_var_44_1, sizeof(float *));
	
	float * h_var_44_2 = (float *)malloc(sizeof(float *));
	float * d_var_44_2;
	hipMalloc((void **)&d_var_44_2, sizeof(float *));
	
	float * h_var_44_3 = (float *)malloc(sizeof(float *));
	float * d_var_44_3;
	hipMalloc((void **)&d_var_44_3, sizeof(float *));
	
	float * h_var_44_4 = (float *)malloc(sizeof(float *));
	float * d_var_44_4;
	hipMalloc((void **)&d_var_44_4, sizeof(float *));
	
	float * h_var_44_5 = (float *)malloc(sizeof(float *));
	float * d_var_44_5;
	hipMalloc((void **)&d_var_44_5, sizeof(float *));
	
	float * h_var_44_6 = (float *)malloc(sizeof(float *));
	float * d_var_44_6;
	hipMalloc((void **)&d_var_44_6, sizeof(float *));
	
	float * h_var_44_7 = (float *)malloc(sizeof(float *));
	float * d_var_44_7;
	hipMalloc((void **)&d_var_44_7, sizeof(float *));
	
	float * h_var_44_8 = (float *)malloc(sizeof(float *));
	float * d_var_44_8;
	hipMalloc((void **)&d_var_44_8, sizeof(float *));
	
	float * h_var_44_9 = (float *)malloc(sizeof(float *));
	float * d_var_44_9;
	hipMalloc((void **)&d_var_44_9, sizeof(float *));
	
	float * h_var_45_0 = (float *)malloc(sizeof(float *));
	float * d_var_45_0;
	hipMalloc((void **)&d_var_45_0, sizeof(float *));
	
	float * h_var_45_1 = (float *)malloc(sizeof(float *));
	float * d_var_45_1;
	hipMalloc((void **)&d_var_45_1, sizeof(float *));
	
	float * h_var_45_2 = (float *)malloc(sizeof(float *));
	float * d_var_45_2;
	hipMalloc((void **)&d_var_45_2, sizeof(float *));
	
	float * h_var_45_3 = (float *)malloc(sizeof(float *));
	float * d_var_45_3;
	hipMalloc((void **)&d_var_45_3, sizeof(float *));
	
	float * h_var_45_4 = (float *)malloc(sizeof(float *));
	float * d_var_45_4;
	hipMalloc((void **)&d_var_45_4, sizeof(float *));
	
	float * h_var_45_5 = (float *)malloc(sizeof(float *));
	float * d_var_45_5;
	hipMalloc((void **)&d_var_45_5, sizeof(float *));
	
	float * h_var_45_6 = (float *)malloc(sizeof(float *));
	float * d_var_45_6;
	hipMalloc((void **)&d_var_45_6, sizeof(float *));
	
	float * h_var_45_7 = (float *)malloc(sizeof(float *));
	float * d_var_45_7;
	hipMalloc((void **)&d_var_45_7, sizeof(float *));
	
	float * h_var_45_8 = (float *)malloc(sizeof(float *));
	float * d_var_45_8;
	hipMalloc((void **)&d_var_45_8, sizeof(float *));
	
	float * h_var_45_9 = (float *)malloc(sizeof(float *));
	float * d_var_45_9;
	hipMalloc((void **)&d_var_45_9, sizeof(float *));
	
	float * h_var_46_0 = (float *)malloc(sizeof(float *));
	float * d_var_46_0;
	hipMalloc((void **)&d_var_46_0, sizeof(float *));
	
	float * h_var_46_1 = (float *)malloc(sizeof(float *));
	float * d_var_46_1;
	hipMalloc((void **)&d_var_46_1, sizeof(float *));
	
	float * h_var_46_2 = (float *)malloc(sizeof(float *));
	float * d_var_46_2;
	hipMalloc((void **)&d_var_46_2, sizeof(float *));
	
	float * h_var_46_3 = (float *)malloc(sizeof(float *));
	float * d_var_46_3;
	hipMalloc((void **)&d_var_46_3, sizeof(float *));
	
	float * h_var_46_4 = (float *)malloc(sizeof(float *));
	float * d_var_46_4;
	hipMalloc((void **)&d_var_46_4, sizeof(float *));
	
	float * h_var_46_5 = (float *)malloc(sizeof(float *));
	float * d_var_46_5;
	hipMalloc((void **)&d_var_46_5, sizeof(float *));
	
	float * h_var_46_6 = (float *)malloc(sizeof(float *));
	float * d_var_46_6;
	hipMalloc((void **)&d_var_46_6, sizeof(float *));
	
	float * h_var_46_7 = (float *)malloc(sizeof(float *));
	float * d_var_46_7;
	hipMalloc((void **)&d_var_46_7, sizeof(float *));
	
	float * h_var_46_8 = (float *)malloc(sizeof(float *));
	float * d_var_46_8;
	hipMalloc((void **)&d_var_46_8, sizeof(float *));
	
	float * h_var_46_9 = (float *)malloc(sizeof(float *));
	float * d_var_46_9;
	hipMalloc((void **)&d_var_46_9, sizeof(float *));
	
	float * h_var_47_0 = (float *)malloc(sizeof(float *));
	float * d_var_47_0;
	hipMalloc((void **)&d_var_47_0, sizeof(float *));
	
	float * h_var_47_1 = (float *)malloc(sizeof(float *));
	float * d_var_47_1;
	hipMalloc((void **)&d_var_47_1, sizeof(float *));
	
	float * h_var_47_2 = (float *)malloc(sizeof(float *));
	float * d_var_47_2;
	hipMalloc((void **)&d_var_47_2, sizeof(float *));
	
	float * h_var_47_3 = (float *)malloc(sizeof(float *));
	float * d_var_47_3;
	hipMalloc((void **)&d_var_47_3, sizeof(float *));
	
	float * h_var_47_4 = (float *)malloc(sizeof(float *));
	float * d_var_47_4;
	hipMalloc((void **)&d_var_47_4, sizeof(float *));
	
	float * h_var_47_5 = (float *)malloc(sizeof(float *));
	float * d_var_47_5;
	hipMalloc((void **)&d_var_47_5, sizeof(float *));
	
	float * h_var_47_6 = (float *)malloc(sizeof(float *));
	float * d_var_47_6;
	hipMalloc((void **)&d_var_47_6, sizeof(float *));
	
	float * h_var_47_7 = (float *)malloc(sizeof(float *));
	float * d_var_47_7;
	hipMalloc((void **)&d_var_47_7, sizeof(float *));
	
	float * h_var_47_8 = (float *)malloc(sizeof(float *));
	float * d_var_47_8;
	hipMalloc((void **)&d_var_47_8, sizeof(float *));
	
	float * h_var_47_9 = (float *)malloc(sizeof(float *));
	float * d_var_47_9;
	hipMalloc((void **)&d_var_47_9, sizeof(float *));
	
	float * h_var_48_0 = (float *)malloc(sizeof(float *));
	float * d_var_48_0;
	hipMalloc((void **)&d_var_48_0, sizeof(float *));
	
	float * h_var_48_1 = (float *)malloc(sizeof(float *));
	float * d_var_48_1;
	hipMalloc((void **)&d_var_48_1, sizeof(float *));
	
	float * h_var_48_2 = (float *)malloc(sizeof(float *));
	float * d_var_48_2;
	hipMalloc((void **)&d_var_48_2, sizeof(float *));
	
	float * h_var_48_3 = (float *)malloc(sizeof(float *));
	float * d_var_48_3;
	hipMalloc((void **)&d_var_48_3, sizeof(float *));
	
	float * h_var_48_4 = (float *)malloc(sizeof(float *));
	float * d_var_48_4;
	hipMalloc((void **)&d_var_48_4, sizeof(float *));
	
	float * h_var_48_5 = (float *)malloc(sizeof(float *));
	float * d_var_48_5;
	hipMalloc((void **)&d_var_48_5, sizeof(float *));
	
	float * h_var_48_6 = (float *)malloc(sizeof(float *));
	float * d_var_48_6;
	hipMalloc((void **)&d_var_48_6, sizeof(float *));
	
	float * h_var_48_7 = (float *)malloc(sizeof(float *));
	float * d_var_48_7;
	hipMalloc((void **)&d_var_48_7, sizeof(float *));
	
	float * h_var_48_8 = (float *)malloc(sizeof(float *));
	float * d_var_48_8;
	hipMalloc((void **)&d_var_48_8, sizeof(float *));
	
	float * h_var_48_9 = (float *)malloc(sizeof(float *));
	float * d_var_48_9;
	hipMalloc((void **)&d_var_48_9, sizeof(float *));
	
	float * h_var_49_0 = (float *)malloc(sizeof(float *));
	float * d_var_49_0;
	hipMalloc((void **)&d_var_49_0, sizeof(float *));
	
	float * h_var_49_1 = (float *)malloc(sizeof(float *));
	float * d_var_49_1;
	hipMalloc((void **)&d_var_49_1, sizeof(float *));
	
	float * h_var_49_2 = (float *)malloc(sizeof(float *));
	float * d_var_49_2;
	hipMalloc((void **)&d_var_49_2, sizeof(float *));
	
	float * h_var_49_3 = (float *)malloc(sizeof(float *));
	float * d_var_49_3;
	hipMalloc((void **)&d_var_49_3, sizeof(float *));
	
	float * h_var_49_4 = (float *)malloc(sizeof(float *));
	float * d_var_49_4;
	hipMalloc((void **)&d_var_49_4, sizeof(float *));
	
	float * h_var_49_5 = (float *)malloc(sizeof(float *));
	float * d_var_49_5;
	hipMalloc((void **)&d_var_49_5, sizeof(float *));
	
	float * h_var_49_6 = (float *)malloc(sizeof(float *));
	float * d_var_49_6;
	hipMalloc((void **)&d_var_49_6, sizeof(float *));
	
	float * h_var_49_7 = (float *)malloc(sizeof(float *));
	float * d_var_49_7;
	hipMalloc((void **)&d_var_49_7, sizeof(float *));
	
	float * h_var_49_8 = (float *)malloc(sizeof(float *));
	float * d_var_49_8;
	hipMalloc((void **)&d_var_49_8, sizeof(float *));
	
	float * h_var_49_9 = (float *)malloc(sizeof(float *));
	float * d_var_49_9;
	hipMalloc((void **)&d_var_49_9, sizeof(float *));
	
	float * h_var_50_0 = (float *)malloc(sizeof(float *));
	float * d_var_50_0;
	hipMalloc((void **)&d_var_50_0, sizeof(float *));
	
	float * h_var_50_1 = (float *)malloc(sizeof(float *));
	float * d_var_50_1;
	hipMalloc((void **)&d_var_50_1, sizeof(float *));
	
	float * h_var_50_2 = (float *)malloc(sizeof(float *));
	float * d_var_50_2;
	hipMalloc((void **)&d_var_50_2, sizeof(float *));
	
	float * h_var_50_3 = (float *)malloc(sizeof(float *));
	float * d_var_50_3;
	hipMalloc((void **)&d_var_50_3, sizeof(float *));
	
	float * h_var_50_4 = (float *)malloc(sizeof(float *));
	float * d_var_50_4;
	hipMalloc((void **)&d_var_50_4, sizeof(float *));
	
	float * h_var_50_5 = (float *)malloc(sizeof(float *));
	float * d_var_50_5;
	hipMalloc((void **)&d_var_50_5, sizeof(float *));
	
	float * h_var_50_6 = (float *)malloc(sizeof(float *));
	float * d_var_50_6;
	hipMalloc((void **)&d_var_50_6, sizeof(float *));
	
	float * h_var_50_7 = (float *)malloc(sizeof(float *));
	float * d_var_50_7;
	hipMalloc((void **)&d_var_50_7, sizeof(float *));
	
	float * h_var_50_8 = (float *)malloc(sizeof(float *));
	float * d_var_50_8;
	hipMalloc((void **)&d_var_50_8, sizeof(float *));
	
	float * h_var_50_9 = (float *)malloc(sizeof(float *));
	float * d_var_50_9;
	hipMalloc((void **)&d_var_50_9, sizeof(float *));
	
	float * h_var_51_0 = (float *)malloc(sizeof(float *));
	float * d_var_51_0;
	hipMalloc((void **)&d_var_51_0, sizeof(float *));
	
	float * h_var_51_1 = (float *)malloc(sizeof(float *));
	float * d_var_51_1;
	hipMalloc((void **)&d_var_51_1, sizeof(float *));
	
	float * h_var_51_2 = (float *)malloc(sizeof(float *));
	float * d_var_51_2;
	hipMalloc((void **)&d_var_51_2, sizeof(float *));
	
	float * h_var_51_3 = (float *)malloc(sizeof(float *));
	float * d_var_51_3;
	hipMalloc((void **)&d_var_51_3, sizeof(float *));
	
	float * h_var_51_4 = (float *)malloc(sizeof(float *));
	float * d_var_51_4;
	hipMalloc((void **)&d_var_51_4, sizeof(float *));
	
	float * h_var_51_5 = (float *)malloc(sizeof(float *));
	float * d_var_51_5;
	hipMalloc((void **)&d_var_51_5, sizeof(float *));
	
	float * h_var_51_6 = (float *)malloc(sizeof(float *));
	float * d_var_51_6;
	hipMalloc((void **)&d_var_51_6, sizeof(float *));
	
	float * h_var_51_7 = (float *)malloc(sizeof(float *));
	float * d_var_51_7;
	hipMalloc((void **)&d_var_51_7, sizeof(float *));
	
	float * h_var_51_8 = (float *)malloc(sizeof(float *));
	float * d_var_51_8;
	hipMalloc((void **)&d_var_51_8, sizeof(float *));
	
	float * h_var_51_9 = (float *)malloc(sizeof(float *));
	float * d_var_51_9;
	hipMalloc((void **)&d_var_51_9, sizeof(float *));
	
	float * h_var_52_0 = (float *)malloc(sizeof(float *));
	float * d_var_52_0;
	hipMalloc((void **)&d_var_52_0, sizeof(float *));
	
	float * h_var_52_1 = (float *)malloc(sizeof(float *));
	float * d_var_52_1;
	hipMalloc((void **)&d_var_52_1, sizeof(float *));
	
	float * h_var_52_2 = (float *)malloc(sizeof(float *));
	float * d_var_52_2;
	hipMalloc((void **)&d_var_52_2, sizeof(float *));
	
	float * h_var_52_3 = (float *)malloc(sizeof(float *));
	float * d_var_52_3;
	hipMalloc((void **)&d_var_52_3, sizeof(float *));
	
	float * h_var_52_4 = (float *)malloc(sizeof(float *));
	float * d_var_52_4;
	hipMalloc((void **)&d_var_52_4, sizeof(float *));
	
	float * h_var_52_5 = (float *)malloc(sizeof(float *));
	float * d_var_52_5;
	hipMalloc((void **)&d_var_52_5, sizeof(float *));
	
	float * h_var_52_6 = (float *)malloc(sizeof(float *));
	float * d_var_52_6;
	hipMalloc((void **)&d_var_52_6, sizeof(float *));
	
	float * h_var_52_7 = (float *)malloc(sizeof(float *));
	float * d_var_52_7;
	hipMalloc((void **)&d_var_52_7, sizeof(float *));
	
	float * h_var_52_8 = (float *)malloc(sizeof(float *));
	float * d_var_52_8;
	hipMalloc((void **)&d_var_52_8, sizeof(float *));
	
	float * h_var_52_9 = (float *)malloc(sizeof(float *));
	float * d_var_52_9;
	hipMalloc((void **)&d_var_52_9, sizeof(float *));
	
	float * h_var_53_0 = (float *)malloc(sizeof(float *));
	float * d_var_53_0;
	hipMalloc((void **)&d_var_53_0, sizeof(float *));
	
	float * h_var_53_1 = (float *)malloc(sizeof(float *));
	float * d_var_53_1;
	hipMalloc((void **)&d_var_53_1, sizeof(float *));
	
	float * h_var_53_2 = (float *)malloc(sizeof(float *));
	float * d_var_53_2;
	hipMalloc((void **)&d_var_53_2, sizeof(float *));
	
	float * h_var_53_3 = (float *)malloc(sizeof(float *));
	float * d_var_53_3;
	hipMalloc((void **)&d_var_53_3, sizeof(float *));
	
	float * h_var_53_4 = (float *)malloc(sizeof(float *));
	float * d_var_53_4;
	hipMalloc((void **)&d_var_53_4, sizeof(float *));
	
	float * h_var_53_5 = (float *)malloc(sizeof(float *));
	float * d_var_53_5;
	hipMalloc((void **)&d_var_53_5, sizeof(float *));
	
	float * h_var_53_6 = (float *)malloc(sizeof(float *));
	float * d_var_53_6;
	hipMalloc((void **)&d_var_53_6, sizeof(float *));
	
	float * h_var_53_7 = (float *)malloc(sizeof(float *));
	float * d_var_53_7;
	hipMalloc((void **)&d_var_53_7, sizeof(float *));
	
	float * h_var_53_8 = (float *)malloc(sizeof(float *));
	float * d_var_53_8;
	hipMalloc((void **)&d_var_53_8, sizeof(float *));
	
	float * h_var_53_9 = (float *)malloc(sizeof(float *));
	float * d_var_53_9;
	hipMalloc((void **)&d_var_53_9, sizeof(float *));
	
	float * h_var_54_0 = (float *)malloc(sizeof(float *));
	float * d_var_54_0;
	hipMalloc((void **)&d_var_54_0, sizeof(float *));
	
	float * h_var_54_1 = (float *)malloc(sizeof(float *));
	float * d_var_54_1;
	hipMalloc((void **)&d_var_54_1, sizeof(float *));
	
	float * h_var_54_2 = (float *)malloc(sizeof(float *));
	float * d_var_54_2;
	hipMalloc((void **)&d_var_54_2, sizeof(float *));
	
	float * h_var_54_3 = (float *)malloc(sizeof(float *));
	float * d_var_54_3;
	hipMalloc((void **)&d_var_54_3, sizeof(float *));
	
	float * h_var_54_4 = (float *)malloc(sizeof(float *));
	float * d_var_54_4;
	hipMalloc((void **)&d_var_54_4, sizeof(float *));
	
	float * h_var_54_5 = (float *)malloc(sizeof(float *));
	float * d_var_54_5;
	hipMalloc((void **)&d_var_54_5, sizeof(float *));
	
	float * h_var_54_6 = (float *)malloc(sizeof(float *));
	float * d_var_54_6;
	hipMalloc((void **)&d_var_54_6, sizeof(float *));
	
	float * h_var_54_7 = (float *)malloc(sizeof(float *));
	float * d_var_54_7;
	hipMalloc((void **)&d_var_54_7, sizeof(float *));
	
	float * h_var_54_8 = (float *)malloc(sizeof(float *));
	float * d_var_54_8;
	hipMalloc((void **)&d_var_54_8, sizeof(float *));
	
	float * h_var_54_9 = (float *)malloc(sizeof(float *));
	float * d_var_54_9;
	hipMalloc((void **)&d_var_54_9, sizeof(float *));
	
	float * h_var_55_0 = (float *)malloc(sizeof(float *));
	float * d_var_55_0;
	hipMalloc((void **)&d_var_55_0, sizeof(float *));
	
	float * h_var_55_1 = (float *)malloc(sizeof(float *));
	float * d_var_55_1;
	hipMalloc((void **)&d_var_55_1, sizeof(float *));
	
	float * h_var_55_2 = (float *)malloc(sizeof(float *));
	float * d_var_55_2;
	hipMalloc((void **)&d_var_55_2, sizeof(float *));
	
	float * h_var_55_3 = (float *)malloc(sizeof(float *));
	float * d_var_55_3;
	hipMalloc((void **)&d_var_55_3, sizeof(float *));
	
	float * h_var_55_4 = (float *)malloc(sizeof(float *));
	float * d_var_55_4;
	hipMalloc((void **)&d_var_55_4, sizeof(float *));
	
	float * h_var_55_5 = (float *)malloc(sizeof(float *));
	float * d_var_55_5;
	hipMalloc((void **)&d_var_55_5, sizeof(float *));
	
	float * h_var_55_6 = (float *)malloc(sizeof(float *));
	float * d_var_55_6;
	hipMalloc((void **)&d_var_55_6, sizeof(float *));
	
	float * h_var_55_7 = (float *)malloc(sizeof(float *));
	float * d_var_55_7;
	hipMalloc((void **)&d_var_55_7, sizeof(float *));
	
	float * h_var_55_8 = (float *)malloc(sizeof(float *));
	float * d_var_55_8;
	hipMalloc((void **)&d_var_55_8, sizeof(float *));
	
	float * h_var_55_9 = (float *)malloc(sizeof(float *));
	float * d_var_55_9;
	hipMalloc((void **)&d_var_55_9, sizeof(float *));
	
	float * h_var_56_0 = (float *)malloc(sizeof(float *));
	float * d_var_56_0;
	hipMalloc((void **)&d_var_56_0, sizeof(float *));
	
	float * h_var_56_1 = (float *)malloc(sizeof(float *));
	float * d_var_56_1;
	hipMalloc((void **)&d_var_56_1, sizeof(float *));
	
	float * h_var_56_2 = (float *)malloc(sizeof(float *));
	float * d_var_56_2;
	hipMalloc((void **)&d_var_56_2, sizeof(float *));
	
	float * h_var_56_3 = (float *)malloc(sizeof(float *));
	float * d_var_56_3;
	hipMalloc((void **)&d_var_56_3, sizeof(float *));
	
	float * h_var_56_4 = (float *)malloc(sizeof(float *));
	float * d_var_56_4;
	hipMalloc((void **)&d_var_56_4, sizeof(float *));
	
	float * h_var_56_5 = (float *)malloc(sizeof(float *));
	float * d_var_56_5;
	hipMalloc((void **)&d_var_56_5, sizeof(float *));
	
	float * h_var_56_6 = (float *)malloc(sizeof(float *));
	float * d_var_56_6;
	hipMalloc((void **)&d_var_56_6, sizeof(float *));
	
	float * h_var_56_7 = (float *)malloc(sizeof(float *));
	float * d_var_56_7;
	hipMalloc((void **)&d_var_56_7, sizeof(float *));
	
	float * h_var_56_8 = (float *)malloc(sizeof(float *));
	float * d_var_56_8;
	hipMalloc((void **)&d_var_56_8, sizeof(float *));
	
	float * h_var_56_9 = (float *)malloc(sizeof(float *));
	float * d_var_56_9;
	hipMalloc((void **)&d_var_56_9, sizeof(float *));
	
	float * h_var_57_0 = (float *)malloc(sizeof(float *));
	float * d_var_57_0;
	hipMalloc((void **)&d_var_57_0, sizeof(float *));
	
	float * h_var_57_1 = (float *)malloc(sizeof(float *));
	float * d_var_57_1;
	hipMalloc((void **)&d_var_57_1, sizeof(float *));
	
	float * h_var_57_2 = (float *)malloc(sizeof(float *));
	float * d_var_57_2;
	hipMalloc((void **)&d_var_57_2, sizeof(float *));
	
	float * h_var_57_3 = (float *)malloc(sizeof(float *));
	float * d_var_57_3;
	hipMalloc((void **)&d_var_57_3, sizeof(float *));
	
	float * h_var_57_4 = (float *)malloc(sizeof(float *));
	float * d_var_57_4;
	hipMalloc((void **)&d_var_57_4, sizeof(float *));
	
	float * h_var_57_5 = (float *)malloc(sizeof(float *));
	float * d_var_57_5;
	hipMalloc((void **)&d_var_57_5, sizeof(float *));
	
	float * h_var_57_6 = (float *)malloc(sizeof(float *));
	float * d_var_57_6;
	hipMalloc((void **)&d_var_57_6, sizeof(float *));
	
	float * h_var_57_7 = (float *)malloc(sizeof(float *));
	float * d_var_57_7;
	hipMalloc((void **)&d_var_57_7, sizeof(float *));
	
	float * h_var_57_8 = (float *)malloc(sizeof(float *));
	float * d_var_57_8;
	hipMalloc((void **)&d_var_57_8, sizeof(float *));
	
	float * h_var_57_9 = (float *)malloc(sizeof(float *));
	float * d_var_57_9;
	hipMalloc((void **)&d_var_57_9, sizeof(float *));
	
	float * h_var_58_0 = (float *)malloc(sizeof(float *));
	float * d_var_58_0;
	hipMalloc((void **)&d_var_58_0, sizeof(float *));
	
	float * h_var_58_1 = (float *)malloc(sizeof(float *));
	float * d_var_58_1;
	hipMalloc((void **)&d_var_58_1, sizeof(float *));
	
	float * h_var_58_2 = (float *)malloc(sizeof(float *));
	float * d_var_58_2;
	hipMalloc((void **)&d_var_58_2, sizeof(float *));
	
	float * h_var_58_3 = (float *)malloc(sizeof(float *));
	float * d_var_58_3;
	hipMalloc((void **)&d_var_58_3, sizeof(float *));
	
	float * h_var_58_4 = (float *)malloc(sizeof(float *));
	float * d_var_58_4;
	hipMalloc((void **)&d_var_58_4, sizeof(float *));
	
	float * h_var_58_5 = (float *)malloc(sizeof(float *));
	float * d_var_58_5;
	hipMalloc((void **)&d_var_58_5, sizeof(float *));
	
	float * h_var_58_6 = (float *)malloc(sizeof(float *));
	float * d_var_58_6;
	hipMalloc((void **)&d_var_58_6, sizeof(float *));
	
	float * h_var_58_7 = (float *)malloc(sizeof(float *));
	float * d_var_58_7;
	hipMalloc((void **)&d_var_58_7, sizeof(float *));
	
	float * h_var_58_8 = (float *)malloc(sizeof(float *));
	float * d_var_58_8;
	hipMalloc((void **)&d_var_58_8, sizeof(float *));
	
	float * h_var_58_9 = (float *)malloc(sizeof(float *));
	float * d_var_58_9;
	hipMalloc((void **)&d_var_58_9, sizeof(float *));
	
	float * h_var_59_0 = (float *)malloc(sizeof(float *));
	float * d_var_59_0;
	hipMalloc((void **)&d_var_59_0, sizeof(float *));
	
	float * h_var_59_1 = (float *)malloc(sizeof(float *));
	float * d_var_59_1;
	hipMalloc((void **)&d_var_59_1, sizeof(float *));
	
	float * h_var_59_2 = (float *)malloc(sizeof(float *));
	float * d_var_59_2;
	hipMalloc((void **)&d_var_59_2, sizeof(float *));
	
	float * h_var_59_3 = (float *)malloc(sizeof(float *));
	float * d_var_59_3;
	hipMalloc((void **)&d_var_59_3, sizeof(float *));
	
	float * h_var_59_4 = (float *)malloc(sizeof(float *));
	float * d_var_59_4;
	hipMalloc((void **)&d_var_59_4, sizeof(float *));
	
	float * h_var_59_5 = (float *)malloc(sizeof(float *));
	float * d_var_59_5;
	hipMalloc((void **)&d_var_59_5, sizeof(float *));
	
	float * h_var_59_6 = (float *)malloc(sizeof(float *));
	float * d_var_59_6;
	hipMalloc((void **)&d_var_59_6, sizeof(float *));
	
	float * h_var_59_7 = (float *)malloc(sizeof(float *));
	float * d_var_59_7;
	hipMalloc((void **)&d_var_59_7, sizeof(float *));
	
	float * h_var_59_8 = (float *)malloc(sizeof(float *));
	float * d_var_59_8;
	hipMalloc((void **)&d_var_59_8, sizeof(float *));
	
	float * h_var_59_9 = (float *)malloc(sizeof(float *));
	float * d_var_59_9;
	hipMalloc((void **)&d_var_59_9, sizeof(float *));
	
	float * h_var_60_0 = (float *)malloc(sizeof(float *));
	float * d_var_60_0;
	hipMalloc((void **)&d_var_60_0, sizeof(float *));
	
	float * h_var_60_1 = (float *)malloc(sizeof(float *));
	float * d_var_60_1;
	hipMalloc((void **)&d_var_60_1, sizeof(float *));
	
	float * h_var_60_2 = (float *)malloc(sizeof(float *));
	float * d_var_60_2;
	hipMalloc((void **)&d_var_60_2, sizeof(float *));
	
	float * h_var_60_3 = (float *)malloc(sizeof(float *));
	float * d_var_60_3;
	hipMalloc((void **)&d_var_60_3, sizeof(float *));
	
	float * h_var_60_4 = (float *)malloc(sizeof(float *));
	float * d_var_60_4;
	hipMalloc((void **)&d_var_60_4, sizeof(float *));
	
	float * h_var_60_5 = (float *)malloc(sizeof(float *));
	float * d_var_60_5;
	hipMalloc((void **)&d_var_60_5, sizeof(float *));
	
	float * h_var_60_6 = (float *)malloc(sizeof(float *));
	float * d_var_60_6;
	hipMalloc((void **)&d_var_60_6, sizeof(float *));
	
	float * h_var_60_7 = (float *)malloc(sizeof(float *));
	float * d_var_60_7;
	hipMalloc((void **)&d_var_60_7, sizeof(float *));
	
	float * h_var_60_8 = (float *)malloc(sizeof(float *));
	float * d_var_60_8;
	hipMalloc((void **)&d_var_60_8, sizeof(float *));
	
	float * h_var_60_9 = (float *)malloc(sizeof(float *));
	float * d_var_60_9;
	hipMalloc((void **)&d_var_60_9, sizeof(float *));
	
	float * h_var_61_0 = (float *)malloc(sizeof(float *));
	float * d_var_61_0;
	hipMalloc((void **)&d_var_61_0, sizeof(float *));
	
	float * h_var_61_1 = (float *)malloc(sizeof(float *));
	float * d_var_61_1;
	hipMalloc((void **)&d_var_61_1, sizeof(float *));
	
	float * h_var_61_2 = (float *)malloc(sizeof(float *));
	float * d_var_61_2;
	hipMalloc((void **)&d_var_61_2, sizeof(float *));
	
	float * h_var_61_3 = (float *)malloc(sizeof(float *));
	float * d_var_61_3;
	hipMalloc((void **)&d_var_61_3, sizeof(float *));
	
	float * h_var_61_4 = (float *)malloc(sizeof(float *));
	float * d_var_61_4;
	hipMalloc((void **)&d_var_61_4, sizeof(float *));
	
	float * h_var_61_5 = (float *)malloc(sizeof(float *));
	float * d_var_61_5;
	hipMalloc((void **)&d_var_61_5, sizeof(float *));
	
	float * h_var_61_6 = (float *)malloc(sizeof(float *));
	float * d_var_61_6;
	hipMalloc((void **)&d_var_61_6, sizeof(float *));
	
	float * h_var_61_7 = (float *)malloc(sizeof(float *));
	float * d_var_61_7;
	hipMalloc((void **)&d_var_61_7, sizeof(float *));
	
	float * h_var_61_8 = (float *)malloc(sizeof(float *));
	float * d_var_61_8;
	hipMalloc((void **)&d_var_61_8, sizeof(float *));
	
	float * h_var_61_9 = (float *)malloc(sizeof(float *));
	float * d_var_61_9;
	hipMalloc((void **)&d_var_61_9, sizeof(float *));
	
	float * h_var_62_0 = (float *)malloc(sizeof(float *));
	float * d_var_62_0;
	hipMalloc((void **)&d_var_62_0, sizeof(float *));
	
	float * h_var_62_1 = (float *)malloc(sizeof(float *));
	float * d_var_62_1;
	hipMalloc((void **)&d_var_62_1, sizeof(float *));
	
	float * h_var_62_2 = (float *)malloc(sizeof(float *));
	float * d_var_62_2;
	hipMalloc((void **)&d_var_62_2, sizeof(float *));
	
	float * h_var_62_3 = (float *)malloc(sizeof(float *));
	float * d_var_62_3;
	hipMalloc((void **)&d_var_62_3, sizeof(float *));
	
	float * h_var_62_4 = (float *)malloc(sizeof(float *));
	float * d_var_62_4;
	hipMalloc((void **)&d_var_62_4, sizeof(float *));
	
	float * h_var_62_5 = (float *)malloc(sizeof(float *));
	float * d_var_62_5;
	hipMalloc((void **)&d_var_62_5, sizeof(float *));
	
	float * h_var_62_6 = (float *)malloc(sizeof(float *));
	float * d_var_62_6;
	hipMalloc((void **)&d_var_62_6, sizeof(float *));
	
	float * h_var_62_7 = (float *)malloc(sizeof(float *));
	float * d_var_62_7;
	hipMalloc((void **)&d_var_62_7, sizeof(float *));
	
	float * h_var_62_8 = (float *)malloc(sizeof(float *));
	float * d_var_62_8;
	hipMalloc((void **)&d_var_62_8, sizeof(float *));
	
	float * h_var_62_9 = (float *)malloc(sizeof(float *));
	float * d_var_62_9;
	hipMalloc((void **)&d_var_62_9, sizeof(float *));
	
	float * h_var_63_0 = (float *)malloc(sizeof(float *));
	float * d_var_63_0;
	hipMalloc((void **)&d_var_63_0, sizeof(float *));
	
	float * h_var_63_1 = (float *)malloc(sizeof(float *));
	float * d_var_63_1;
	hipMalloc((void **)&d_var_63_1, sizeof(float *));
	
	float * h_var_63_2 = (float *)malloc(sizeof(float *));
	float * d_var_63_2;
	hipMalloc((void **)&d_var_63_2, sizeof(float *));
	
	float * h_var_63_3 = (float *)malloc(sizeof(float *));
	float * d_var_63_3;
	hipMalloc((void **)&d_var_63_3, sizeof(float *));
	
	float * h_var_63_4 = (float *)malloc(sizeof(float *));
	float * d_var_63_4;
	hipMalloc((void **)&d_var_63_4, sizeof(float *));
	
	float * h_var_63_5 = (float *)malloc(sizeof(float *));
	float * d_var_63_5;
	hipMalloc((void **)&d_var_63_5, sizeof(float *));
	
	float * h_var_63_6 = (float *)malloc(sizeof(float *));
	float * d_var_63_6;
	hipMalloc((void **)&d_var_63_6, sizeof(float *));
	
	float * h_var_63_7 = (float *)malloc(sizeof(float *));
	float * d_var_63_7;
	hipMalloc((void **)&d_var_63_7, sizeof(float *));
	
	float * h_var_63_8 = (float *)malloc(sizeof(float *));
	float * d_var_63_8;
	hipMalloc((void **)&d_var_63_8, sizeof(float *));
	
	float * h_var_63_9 = (float *)malloc(sizeof(float *));
	float * d_var_63_9;
	hipMalloc((void **)&d_var_63_9, sizeof(float *));
	
	float * h_var_64_0 = (float *)malloc(sizeof(float *));
	float * d_var_64_0;
	hipMalloc((void **)&d_var_64_0, sizeof(float *));
	
	float * h_var_64_1 = (float *)malloc(sizeof(float *));
	float * d_var_64_1;
	hipMalloc((void **)&d_var_64_1, sizeof(float *));
	
	float * h_var_64_2 = (float *)malloc(sizeof(float *));
	float * d_var_64_2;
	hipMalloc((void **)&d_var_64_2, sizeof(float *));
	
	float * h_var_64_3 = (float *)malloc(sizeof(float *));
	float * d_var_64_3;
	hipMalloc((void **)&d_var_64_3, sizeof(float *));
	
	float * h_var_64_4 = (float *)malloc(sizeof(float *));
	float * d_var_64_4;
	hipMalloc((void **)&d_var_64_4, sizeof(float *));
	
	float * h_var_64_5 = (float *)malloc(sizeof(float *));
	float * d_var_64_5;
	hipMalloc((void **)&d_var_64_5, sizeof(float *));
	
	float * h_var_64_6 = (float *)malloc(sizeof(float *));
	float * d_var_64_6;
	hipMalloc((void **)&d_var_64_6, sizeof(float *));
	
	float * h_var_64_7 = (float *)malloc(sizeof(float *));
	float * d_var_64_7;
	hipMalloc((void **)&d_var_64_7, sizeof(float *));
	
	float * h_var_64_8 = (float *)malloc(sizeof(float *));
	float * d_var_64_8;
	hipMalloc((void **)&d_var_64_8, sizeof(float *));
	
	float * h_var_64_9 = (float *)malloc(sizeof(float *));
	float * d_var_64_9;
	hipMalloc((void **)&d_var_64_9, sizeof(float *));
	
	float * h_var_65_0 = (float *)malloc(sizeof(float *));
	float * d_var_65_0;
	hipMalloc((void **)&d_var_65_0, sizeof(float *));
	
	float * h_var_65_1 = (float *)malloc(sizeof(float *));
	float * d_var_65_1;
	hipMalloc((void **)&d_var_65_1, sizeof(float *));
	
	float * h_var_65_2 = (float *)malloc(sizeof(float *));
	float * d_var_65_2;
	hipMalloc((void **)&d_var_65_2, sizeof(float *));
	
	float * h_var_65_3 = (float *)malloc(sizeof(float *));
	float * d_var_65_3;
	hipMalloc((void **)&d_var_65_3, sizeof(float *));
	
	float * h_var_65_4 = (float *)malloc(sizeof(float *));
	float * d_var_65_4;
	hipMalloc((void **)&d_var_65_4, sizeof(float *));
	
	float * h_var_65_5 = (float *)malloc(sizeof(float *));
	float * d_var_65_5;
	hipMalloc((void **)&d_var_65_5, sizeof(float *));
	
	float * h_var_65_6 = (float *)malloc(sizeof(float *));
	float * d_var_65_6;
	hipMalloc((void **)&d_var_65_6, sizeof(float *));
	
	float * h_var_65_7 = (float *)malloc(sizeof(float *));
	float * d_var_65_7;
	hipMalloc((void **)&d_var_65_7, sizeof(float *));
	
	float * h_var_65_8 = (float *)malloc(sizeof(float *));
	float * d_var_65_8;
	hipMalloc((void **)&d_var_65_8, sizeof(float *));
	
	float * h_var_65_9 = (float *)malloc(sizeof(float *));
	float * d_var_65_9;
	hipMalloc((void **)&d_var_65_9, sizeof(float *));
	
	float * h_var_66_0 = (float *)malloc(sizeof(float *));
	float * d_var_66_0;
	hipMalloc((void **)&d_var_66_0, sizeof(float *));
	
	float * h_var_66_1 = (float *)malloc(sizeof(float *));
	float * d_var_66_1;
	hipMalloc((void **)&d_var_66_1, sizeof(float *));
	
	float * h_var_66_2 = (float *)malloc(sizeof(float *));
	float * d_var_66_2;
	hipMalloc((void **)&d_var_66_2, sizeof(float *));
	
	float * h_var_66_3 = (float *)malloc(sizeof(float *));
	float * d_var_66_3;
	hipMalloc((void **)&d_var_66_3, sizeof(float *));
	
	float * h_var_66_4 = (float *)malloc(sizeof(float *));
	float * d_var_66_4;
	hipMalloc((void **)&d_var_66_4, sizeof(float *));
	
	float * h_var_66_5 = (float *)malloc(sizeof(float *));
	float * d_var_66_5;
	hipMalloc((void **)&d_var_66_5, sizeof(float *));
	
	float * h_var_66_6 = (float *)malloc(sizeof(float *));
	float * d_var_66_6;
	hipMalloc((void **)&d_var_66_6, sizeof(float *));
	
	float * h_var_66_7 = (float *)malloc(sizeof(float *));
	float * d_var_66_7;
	hipMalloc((void **)&d_var_66_7, sizeof(float *));
	
	float * h_var_66_8 = (float *)malloc(sizeof(float *));
	float * d_var_66_8;
	hipMalloc((void **)&d_var_66_8, sizeof(float *));
	
	float * h_var_66_9 = (float *)malloc(sizeof(float *));
	float * d_var_66_9;
	hipMalloc((void **)&d_var_66_9, sizeof(float *));
	
	float * h_var_67_0 = (float *)malloc(sizeof(float *));
	float * d_var_67_0;
	hipMalloc((void **)&d_var_67_0, sizeof(float *));
	
	float * h_var_67_1 = (float *)malloc(sizeof(float *));
	float * d_var_67_1;
	hipMalloc((void **)&d_var_67_1, sizeof(float *));
	
	float * h_var_67_2 = (float *)malloc(sizeof(float *));
	float * d_var_67_2;
	hipMalloc((void **)&d_var_67_2, sizeof(float *));
	
	float * h_var_67_3 = (float *)malloc(sizeof(float *));
	float * d_var_67_3;
	hipMalloc((void **)&d_var_67_3, sizeof(float *));
	
	float * h_var_67_4 = (float *)malloc(sizeof(float *));
	float * d_var_67_4;
	hipMalloc((void **)&d_var_67_4, sizeof(float *));
	
	float * h_var_67_5 = (float *)malloc(sizeof(float *));
	float * d_var_67_5;
	hipMalloc((void **)&d_var_67_5, sizeof(float *));
	
	float * h_var_67_6 = (float *)malloc(sizeof(float *));
	float * d_var_67_6;
	hipMalloc((void **)&d_var_67_6, sizeof(float *));
	
	float * h_var_67_7 = (float *)malloc(sizeof(float *));
	float * d_var_67_7;
	hipMalloc((void **)&d_var_67_7, sizeof(float *));
	
	float * h_var_67_8 = (float *)malloc(sizeof(float *));
	float * d_var_67_8;
	hipMalloc((void **)&d_var_67_8, sizeof(float *));
	
	float * h_var_67_9 = (float *)malloc(sizeof(float *));
	float * d_var_67_9;
	hipMalloc((void **)&d_var_67_9, sizeof(float *));
	
	float * h_var_68_0 = (float *)malloc(sizeof(float *));
	float * d_var_68_0;
	hipMalloc((void **)&d_var_68_0, sizeof(float *));
	
	float * h_var_68_1 = (float *)malloc(sizeof(float *));
	float * d_var_68_1;
	hipMalloc((void **)&d_var_68_1, sizeof(float *));
	
	float * h_var_68_2 = (float *)malloc(sizeof(float *));
	float * d_var_68_2;
	hipMalloc((void **)&d_var_68_2, sizeof(float *));
	
	float * h_var_68_3 = (float *)malloc(sizeof(float *));
	float * d_var_68_3;
	hipMalloc((void **)&d_var_68_3, sizeof(float *));
	
	float * h_var_68_4 = (float *)malloc(sizeof(float *));
	float * d_var_68_4;
	hipMalloc((void **)&d_var_68_4, sizeof(float *));
	
	float * h_var_68_5 = (float *)malloc(sizeof(float *));
	float * d_var_68_5;
	hipMalloc((void **)&d_var_68_5, sizeof(float *));
	
	float * h_var_68_6 = (float *)malloc(sizeof(float *));
	float * d_var_68_6;
	hipMalloc((void **)&d_var_68_6, sizeof(float *));
	
	float * h_var_68_7 = (float *)malloc(sizeof(float *));
	float * d_var_68_7;
	hipMalloc((void **)&d_var_68_7, sizeof(float *));
	
	float * h_var_68_8 = (float *)malloc(sizeof(float *));
	float * d_var_68_8;
	hipMalloc((void **)&d_var_68_8, sizeof(float *));
	
	float * h_var_68_9 = (float *)malloc(sizeof(float *));
	float * d_var_68_9;
	hipMalloc((void **)&d_var_68_9, sizeof(float *));
	
	float * h_var_69_0 = (float *)malloc(sizeof(float *));
	float * d_var_69_0;
	hipMalloc((void **)&d_var_69_0, sizeof(float *));
	
	float * h_var_69_1 = (float *)malloc(sizeof(float *));
	float * d_var_69_1;
	hipMalloc((void **)&d_var_69_1, sizeof(float *));
	
	float * h_var_69_2 = (float *)malloc(sizeof(float *));
	float * d_var_69_2;
	hipMalloc((void **)&d_var_69_2, sizeof(float *));
	
	float * h_var_69_3 = (float *)malloc(sizeof(float *));
	float * d_var_69_3;
	hipMalloc((void **)&d_var_69_3, sizeof(float *));
	
	float * h_var_69_4 = (float *)malloc(sizeof(float *));
	float * d_var_69_4;
	hipMalloc((void **)&d_var_69_4, sizeof(float *));
	
	float * h_var_69_5 = (float *)malloc(sizeof(float *));
	float * d_var_69_5;
	hipMalloc((void **)&d_var_69_5, sizeof(float *));
	
	float * h_var_69_6 = (float *)malloc(sizeof(float *));
	float * d_var_69_6;
	hipMalloc((void **)&d_var_69_6, sizeof(float *));
	
	float * h_var_69_7 = (float *)malloc(sizeof(float *));
	float * d_var_69_7;
	hipMalloc((void **)&d_var_69_7, sizeof(float *));
	
	float * h_var_69_8 = (float *)malloc(sizeof(float *));
	float * d_var_69_8;
	hipMalloc((void **)&d_var_69_8, sizeof(float *));
	
	float * h_var_69_9 = (float *)malloc(sizeof(float *));
	float * d_var_69_9;
	hipMalloc((void **)&d_var_69_9, sizeof(float *));
	
	float * h_var_70_0 = (float *)malloc(sizeof(float *));
	float * d_var_70_0;
	hipMalloc((void **)&d_var_70_0, sizeof(float *));
	
	float * h_var_70_1 = (float *)malloc(sizeof(float *));
	float * d_var_70_1;
	hipMalloc((void **)&d_var_70_1, sizeof(float *));
	
	float * h_var_70_2 = (float *)malloc(sizeof(float *));
	float * d_var_70_2;
	hipMalloc((void **)&d_var_70_2, sizeof(float *));
	
	float * h_var_70_3 = (float *)malloc(sizeof(float *));
	float * d_var_70_3;
	hipMalloc((void **)&d_var_70_3, sizeof(float *));
	
	float * h_var_70_4 = (float *)malloc(sizeof(float *));
	float * d_var_70_4;
	hipMalloc((void **)&d_var_70_4, sizeof(float *));
	
	float * h_var_70_5 = (float *)malloc(sizeof(float *));
	float * d_var_70_5;
	hipMalloc((void **)&d_var_70_5, sizeof(float *));
	
	float * h_var_70_6 = (float *)malloc(sizeof(float *));
	float * d_var_70_6;
	hipMalloc((void **)&d_var_70_6, sizeof(float *));
	
	float * h_var_70_7 = (float *)malloc(sizeof(float *));
	float * d_var_70_7;
	hipMalloc((void **)&d_var_70_7, sizeof(float *));
	
	float * h_var_70_8 = (float *)malloc(sizeof(float *));
	float * d_var_70_8;
	hipMalloc((void **)&d_var_70_8, sizeof(float *));
	
	float * h_var_70_9 = (float *)malloc(sizeof(float *));
	float * d_var_70_9;
	hipMalloc((void **)&d_var_70_9, sizeof(float *));
	
	float * h_var_71_0 = (float *)malloc(sizeof(float *));
	float * d_var_71_0;
	hipMalloc((void **)&d_var_71_0, sizeof(float *));
	
	float * h_var_71_1 = (float *)malloc(sizeof(float *));
	float * d_var_71_1;
	hipMalloc((void **)&d_var_71_1, sizeof(float *));
	
	float * h_var_71_2 = (float *)malloc(sizeof(float *));
	float * d_var_71_2;
	hipMalloc((void **)&d_var_71_2, sizeof(float *));
	
	float * h_var_71_3 = (float *)malloc(sizeof(float *));
	float * d_var_71_3;
	hipMalloc((void **)&d_var_71_3, sizeof(float *));
	
	float * h_var_71_4 = (float *)malloc(sizeof(float *));
	float * d_var_71_4;
	hipMalloc((void **)&d_var_71_4, sizeof(float *));
	
	float * h_var_71_5 = (float *)malloc(sizeof(float *));
	float * d_var_71_5;
	hipMalloc((void **)&d_var_71_5, sizeof(float *));
	
	float * h_var_71_6 = (float *)malloc(sizeof(float *));
	float * d_var_71_6;
	hipMalloc((void **)&d_var_71_6, sizeof(float *));
	
	float * h_var_71_7 = (float *)malloc(sizeof(float *));
	float * d_var_71_7;
	hipMalloc((void **)&d_var_71_7, sizeof(float *));
	
	float * h_var_71_8 = (float *)malloc(sizeof(float *));
	float * d_var_71_8;
	hipMalloc((void **)&d_var_71_8, sizeof(float *));
	
	float * h_var_71_9 = (float *)malloc(sizeof(float *));
	float * d_var_71_9;
	hipMalloc((void **)&d_var_71_9, sizeof(float *));
	
	float * h_var_72_0 = (float *)malloc(sizeof(float *));
	float * d_var_72_0;
	hipMalloc((void **)&d_var_72_0, sizeof(float *));
	
	float * h_var_72_1 = (float *)malloc(sizeof(float *));
	float * d_var_72_1;
	hipMalloc((void **)&d_var_72_1, sizeof(float *));
	
	float * h_var_72_2 = (float *)malloc(sizeof(float *));
	float * d_var_72_2;
	hipMalloc((void **)&d_var_72_2, sizeof(float *));
	
	float * h_var_72_3 = (float *)malloc(sizeof(float *));
	float * d_var_72_3;
	hipMalloc((void **)&d_var_72_3, sizeof(float *));
	
	float * h_var_72_4 = (float *)malloc(sizeof(float *));
	float * d_var_72_4;
	hipMalloc((void **)&d_var_72_4, sizeof(float *));
	
	float * h_var_72_5 = (float *)malloc(sizeof(float *));
	float * d_var_72_5;
	hipMalloc((void **)&d_var_72_5, sizeof(float *));
	
	float * h_var_72_6 = (float *)malloc(sizeof(float *));
	float * d_var_72_6;
	hipMalloc((void **)&d_var_72_6, sizeof(float *));
	
	float * h_var_72_7 = (float *)malloc(sizeof(float *));
	float * d_var_72_7;
	hipMalloc((void **)&d_var_72_7, sizeof(float *));
	
	float * h_var_72_8 = (float *)malloc(sizeof(float *));
	float * d_var_72_8;
	hipMalloc((void **)&d_var_72_8, sizeof(float *));
	
	float * h_var_72_9 = (float *)malloc(sizeof(float *));
	float * d_var_72_9;
	hipMalloc((void **)&d_var_72_9, sizeof(float *));
	
	float * h_var_73_0 = (float *)malloc(sizeof(float *));
	float * d_var_73_0;
	hipMalloc((void **)&d_var_73_0, sizeof(float *));
	
	float * h_var_73_1 = (float *)malloc(sizeof(float *));
	float * d_var_73_1;
	hipMalloc((void **)&d_var_73_1, sizeof(float *));
	
	float * h_var_73_2 = (float *)malloc(sizeof(float *));
	float * d_var_73_2;
	hipMalloc((void **)&d_var_73_2, sizeof(float *));
	
	float * h_var_73_3 = (float *)malloc(sizeof(float *));
	float * d_var_73_3;
	hipMalloc((void **)&d_var_73_3, sizeof(float *));
	
	float * h_var_73_4 = (float *)malloc(sizeof(float *));
	float * d_var_73_4;
	hipMalloc((void **)&d_var_73_4, sizeof(float *));
	
	float * h_var_73_5 = (float *)malloc(sizeof(float *));
	float * d_var_73_5;
	hipMalloc((void **)&d_var_73_5, sizeof(float *));
	
	float * h_var_73_6 = (float *)malloc(sizeof(float *));
	float * d_var_73_6;
	hipMalloc((void **)&d_var_73_6, sizeof(float *));
	
	float * h_var_73_7 = (float *)malloc(sizeof(float *));
	float * d_var_73_7;
	hipMalloc((void **)&d_var_73_7, sizeof(float *));
	
	float * h_var_73_8 = (float *)malloc(sizeof(float *));
	float * d_var_73_8;
	hipMalloc((void **)&d_var_73_8, sizeof(float *));
	
	float * h_var_73_9 = (float *)malloc(sizeof(float *));
	float * d_var_73_9;
	hipMalloc((void **)&d_var_73_9, sizeof(float *));
	
	float * h_var_74_0 = (float *)malloc(sizeof(float *));
	float * d_var_74_0;
	hipMalloc((void **)&d_var_74_0, sizeof(float *));
	
	float * h_var_74_1 = (float *)malloc(sizeof(float *));
	float * d_var_74_1;
	hipMalloc((void **)&d_var_74_1, sizeof(float *));
	
	float * h_var_74_2 = (float *)malloc(sizeof(float *));
	float * d_var_74_2;
	hipMalloc((void **)&d_var_74_2, sizeof(float *));
	
	float * h_var_74_3 = (float *)malloc(sizeof(float *));
	float * d_var_74_3;
	hipMalloc((void **)&d_var_74_3, sizeof(float *));
	
	float * h_var_74_4 = (float *)malloc(sizeof(float *));
	float * d_var_74_4;
	hipMalloc((void **)&d_var_74_4, sizeof(float *));
	
	float * h_var_74_5 = (float *)malloc(sizeof(float *));
	float * d_var_74_5;
	hipMalloc((void **)&d_var_74_5, sizeof(float *));
	
	float * h_var_74_6 = (float *)malloc(sizeof(float *));
	float * d_var_74_6;
	hipMalloc((void **)&d_var_74_6, sizeof(float *));
	
	float * h_var_74_7 = (float *)malloc(sizeof(float *));
	float * d_var_74_7;
	hipMalloc((void **)&d_var_74_7, sizeof(float *));
	
	float * h_var_74_8 = (float *)malloc(sizeof(float *));
	float * d_var_74_8;
	hipMalloc((void **)&d_var_74_8, sizeof(float *));
	
	float * h_var_74_9 = (float *)malloc(sizeof(float *));
	float * d_var_74_9;
	hipMalloc((void **)&d_var_74_9, sizeof(float *));
	
	float * h_var_75_0 = (float *)malloc(sizeof(float *));
	float * d_var_75_0;
	hipMalloc((void **)&d_var_75_0, sizeof(float *));
	
	float * h_var_75_1 = (float *)malloc(sizeof(float *));
	float * d_var_75_1;
	hipMalloc((void **)&d_var_75_1, sizeof(float *));
	
	float * h_var_75_2 = (float *)malloc(sizeof(float *));
	float * d_var_75_2;
	hipMalloc((void **)&d_var_75_2, sizeof(float *));
	
	float * h_var_75_3 = (float *)malloc(sizeof(float *));
	float * d_var_75_3;
	hipMalloc((void **)&d_var_75_3, sizeof(float *));
	
	float * h_var_75_4 = (float *)malloc(sizeof(float *));
	float * d_var_75_4;
	hipMalloc((void **)&d_var_75_4, sizeof(float *));
	
	float * h_var_75_5 = (float *)malloc(sizeof(float *));
	float * d_var_75_5;
	hipMalloc((void **)&d_var_75_5, sizeof(float *));
	
	float * h_var_75_6 = (float *)malloc(sizeof(float *));
	float * d_var_75_6;
	hipMalloc((void **)&d_var_75_6, sizeof(float *));
	
	float * h_var_75_7 = (float *)malloc(sizeof(float *));
	float * d_var_75_7;
	hipMalloc((void **)&d_var_75_7, sizeof(float *));
	
	float * h_var_75_8 = (float *)malloc(sizeof(float *));
	float * d_var_75_8;
	hipMalloc((void **)&d_var_75_8, sizeof(float *));
	
	float * h_var_75_9 = (float *)malloc(sizeof(float *));
	float * d_var_75_9;
	hipMalloc((void **)&d_var_75_9, sizeof(float *));
	
	float * h_var_76_0 = (float *)malloc(sizeof(float *));
	float * d_var_76_0;
	hipMalloc((void **)&d_var_76_0, sizeof(float *));
	
	float * h_var_76_1 = (float *)malloc(sizeof(float *));
	float * d_var_76_1;
	hipMalloc((void **)&d_var_76_1, sizeof(float *));
	
	float * h_var_76_2 = (float *)malloc(sizeof(float *));
	float * d_var_76_2;
	hipMalloc((void **)&d_var_76_2, sizeof(float *));
	
	float * h_var_76_3 = (float *)malloc(sizeof(float *));
	float * d_var_76_3;
	hipMalloc((void **)&d_var_76_3, sizeof(float *));
	
	float * h_var_76_4 = (float *)malloc(sizeof(float *));
	float * d_var_76_4;
	hipMalloc((void **)&d_var_76_4, sizeof(float *));
	
	float * h_var_76_5 = (float *)malloc(sizeof(float *));
	float * d_var_76_5;
	hipMalloc((void **)&d_var_76_5, sizeof(float *));
	
	float * h_var_76_6 = (float *)malloc(sizeof(float *));
	float * d_var_76_6;
	hipMalloc((void **)&d_var_76_6, sizeof(float *));
	
	float * h_var_76_7 = (float *)malloc(sizeof(float *));
	float * d_var_76_7;
	hipMalloc((void **)&d_var_76_7, sizeof(float *));
	
	float * h_var_76_8 = (float *)malloc(sizeof(float *));
	float * d_var_76_8;
	hipMalloc((void **)&d_var_76_8, sizeof(float *));
	
	float * h_var_76_9 = (float *)malloc(sizeof(float *));
	float * d_var_76_9;
	hipMalloc((void **)&d_var_76_9, sizeof(float *));
	
	float * h_var_77_0 = (float *)malloc(sizeof(float *));
	float * d_var_77_0;
	hipMalloc((void **)&d_var_77_0, sizeof(float *));
	
	float * h_var_77_1 = (float *)malloc(sizeof(float *));
	float * d_var_77_1;
	hipMalloc((void **)&d_var_77_1, sizeof(float *));
	
	float * h_var_77_2 = (float *)malloc(sizeof(float *));
	float * d_var_77_2;
	hipMalloc((void **)&d_var_77_2, sizeof(float *));
	
	float * h_var_77_3 = (float *)malloc(sizeof(float *));
	float * d_var_77_3;
	hipMalloc((void **)&d_var_77_3, sizeof(float *));
	
	float * h_var_77_4 = (float *)malloc(sizeof(float *));
	float * d_var_77_4;
	hipMalloc((void **)&d_var_77_4, sizeof(float *));
	
	float * h_var_77_5 = (float *)malloc(sizeof(float *));
	float * d_var_77_5;
	hipMalloc((void **)&d_var_77_5, sizeof(float *));
	
	float * h_var_77_6 = (float *)malloc(sizeof(float *));
	float * d_var_77_6;
	hipMalloc((void **)&d_var_77_6, sizeof(float *));
	
	float * h_var_77_7 = (float *)malloc(sizeof(float *));
	float * d_var_77_7;
	hipMalloc((void **)&d_var_77_7, sizeof(float *));
	
	float * h_var_77_8 = (float *)malloc(sizeof(float *));
	float * d_var_77_8;
	hipMalloc((void **)&d_var_77_8, sizeof(float *));
	
	float * h_var_77_9 = (float *)malloc(sizeof(float *));
	float * d_var_77_9;
	hipMalloc((void **)&d_var_77_9, sizeof(float *));
	
	float * h_var_78_0 = (float *)malloc(sizeof(float *));
	float * d_var_78_0;
	hipMalloc((void **)&d_var_78_0, sizeof(float *));
	
	float * h_var_78_1 = (float *)malloc(sizeof(float *));
	float * d_var_78_1;
	hipMalloc((void **)&d_var_78_1, sizeof(float *));
	
	float * h_var_78_2 = (float *)malloc(sizeof(float *));
	float * d_var_78_2;
	hipMalloc((void **)&d_var_78_2, sizeof(float *));
	
	float * h_var_78_3 = (float *)malloc(sizeof(float *));
	float * d_var_78_3;
	hipMalloc((void **)&d_var_78_3, sizeof(float *));
	
	float * h_var_78_4 = (float *)malloc(sizeof(float *));
	float * d_var_78_4;
	hipMalloc((void **)&d_var_78_4, sizeof(float *));
	
	float * h_var_78_5 = (float *)malloc(sizeof(float *));
	float * d_var_78_5;
	hipMalloc((void **)&d_var_78_5, sizeof(float *));
	
	float * h_var_78_6 = (float *)malloc(sizeof(float *));
	float * d_var_78_6;
	hipMalloc((void **)&d_var_78_6, sizeof(float *));
	
	float * h_var_78_7 = (float *)malloc(sizeof(float *));
	float * d_var_78_7;
	hipMalloc((void **)&d_var_78_7, sizeof(float *));
	
	float * h_var_78_8 = (float *)malloc(sizeof(float *));
	float * d_var_78_8;
	hipMalloc((void **)&d_var_78_8, sizeof(float *));
	
	float * h_var_78_9 = (float *)malloc(sizeof(float *));
	float * d_var_78_9;
	hipMalloc((void **)&d_var_78_9, sizeof(float *));
	
	float * h_var_79_0 = (float *)malloc(sizeof(float *));
	float * d_var_79_0;
	hipMalloc((void **)&d_var_79_0, sizeof(float *));
	
	float * h_var_79_1 = (float *)malloc(sizeof(float *));
	float * d_var_79_1;
	hipMalloc((void **)&d_var_79_1, sizeof(float *));
	
	float * h_var_79_2 = (float *)malloc(sizeof(float *));
	float * d_var_79_2;
	hipMalloc((void **)&d_var_79_2, sizeof(float *));
	
	float * h_var_79_3 = (float *)malloc(sizeof(float *));
	float * d_var_79_3;
	hipMalloc((void **)&d_var_79_3, sizeof(float *));
	
	float * h_var_79_4 = (float *)malloc(sizeof(float *));
	float * d_var_79_4;
	hipMalloc((void **)&d_var_79_4, sizeof(float *));
	
	float * h_var_79_5 = (float *)malloc(sizeof(float *));
	float * d_var_79_5;
	hipMalloc((void **)&d_var_79_5, sizeof(float *));
	
	float * h_var_79_6 = (float *)malloc(sizeof(float *));
	float * d_var_79_6;
	hipMalloc((void **)&d_var_79_6, sizeof(float *));
	
	float * h_var_79_7 = (float *)malloc(sizeof(float *));
	float * d_var_79_7;
	hipMalloc((void **)&d_var_79_7, sizeof(float *));
	
	float * h_var_79_8 = (float *)malloc(sizeof(float *));
	float * d_var_79_8;
	hipMalloc((void **)&d_var_79_8, sizeof(float *));
	
	float * h_var_79_9 = (float *)malloc(sizeof(float *));
	float * d_var_79_9;
	hipMalloc((void **)&d_var_79_9, sizeof(float *));
	

    // clang-format off
	
	kernel_0<<<10, 10>>>(d_var_0_0, d_var_0_1, d_var_0_2, d_var_0_3, d_var_0_4, d_var_0_5, d_var_0_6, d_var_0_7, d_var_0_8, d_var_0_9);
	
	kernel_1<<<10, 10>>>(d_var_1_0, d_var_1_1, d_var_1_2, d_var_1_3, d_var_1_4, d_var_1_5, d_var_1_6, d_var_1_7, d_var_1_8, d_var_1_9);
	
	kernel_2<<<10, 10>>>(d_var_2_0, d_var_2_1, d_var_2_2, d_var_2_3, d_var_2_4, d_var_2_5, d_var_2_6, d_var_2_7, d_var_2_8, d_var_2_9);
	
	kernel_3<<<10, 10>>>(d_var_3_0, d_var_3_1, d_var_3_2, d_var_3_3, d_var_3_4, d_var_3_5, d_var_3_6, d_var_3_7, d_var_3_8, d_var_3_9);
	
	kernel_4<<<10, 10>>>(d_var_4_0, d_var_4_1, d_var_4_2, d_var_4_3, d_var_4_4, d_var_4_5, d_var_4_6, d_var_4_7, d_var_4_8, d_var_4_9);
	
	kernel_5<<<10, 10>>>(d_var_5_0, d_var_5_1, d_var_5_2, d_var_5_3, d_var_5_4, d_var_5_5, d_var_5_6, d_var_5_7, d_var_5_8, d_var_5_9);
	
	kernel_6<<<10, 10>>>(d_var_6_0, d_var_6_1, d_var_6_2, d_var_6_3, d_var_6_4, d_var_6_5, d_var_6_6, d_var_6_7, d_var_6_8, d_var_6_9);
	
	kernel_7<<<10, 10>>>(d_var_7_0, d_var_7_1, d_var_7_2, d_var_7_3, d_var_7_4, d_var_7_5, d_var_7_6, d_var_7_7, d_var_7_8, d_var_7_9);
	
	kernel_8<<<10, 10>>>(d_var_8_0, d_var_8_1, d_var_8_2, d_var_8_3, d_var_8_4, d_var_8_5, d_var_8_6, d_var_8_7, d_var_8_8, d_var_8_9);
	
	kernel_9<<<10, 10>>>(d_var_9_0, d_var_9_1, d_var_9_2, d_var_9_3, d_var_9_4, d_var_9_5, d_var_9_6, d_var_9_7, d_var_9_8, d_var_9_9);
	
	kernel_10<<<10, 10>>>(d_var_10_0, d_var_10_1, d_var_10_2, d_var_10_3, d_var_10_4, d_var_10_5, d_var_10_6, d_var_10_7, d_var_10_8, d_var_10_9);
	
	kernel_11<<<10, 10>>>(d_var_11_0, d_var_11_1, d_var_11_2, d_var_11_3, d_var_11_4, d_var_11_5, d_var_11_6, d_var_11_7, d_var_11_8, d_var_11_9);
	
	kernel_12<<<10, 10>>>(d_var_12_0, d_var_12_1, d_var_12_2, d_var_12_3, d_var_12_4, d_var_12_5, d_var_12_6, d_var_12_7, d_var_12_8, d_var_12_9);
	
	kernel_13<<<10, 10>>>(d_var_13_0, d_var_13_1, d_var_13_2, d_var_13_3, d_var_13_4, d_var_13_5, d_var_13_6, d_var_13_7, d_var_13_8, d_var_13_9);
	
	kernel_14<<<10, 10>>>(d_var_14_0, d_var_14_1, d_var_14_2, d_var_14_3, d_var_14_4, d_var_14_5, d_var_14_6, d_var_14_7, d_var_14_8, d_var_14_9);
	
	kernel_15<<<10, 10>>>(d_var_15_0, d_var_15_1, d_var_15_2, d_var_15_3, d_var_15_4, d_var_15_5, d_var_15_6, d_var_15_7, d_var_15_8, d_var_15_9);
	
	kernel_16<<<10, 10>>>(d_var_16_0, d_var_16_1, d_var_16_2, d_var_16_3, d_var_16_4, d_var_16_5, d_var_16_6, d_var_16_7, d_var_16_8, d_var_16_9);
	
	kernel_17<<<10, 10>>>(d_var_17_0, d_var_17_1, d_var_17_2, d_var_17_3, d_var_17_4, d_var_17_5, d_var_17_6, d_var_17_7, d_var_17_8, d_var_17_9);
	
	kernel_18<<<10, 10>>>(d_var_18_0, d_var_18_1, d_var_18_2, d_var_18_3, d_var_18_4, d_var_18_5, d_var_18_6, d_var_18_7, d_var_18_8, d_var_18_9);
	
	kernel_19<<<10, 10>>>(d_var_19_0, d_var_19_1, d_var_19_2, d_var_19_3, d_var_19_4, d_var_19_5, d_var_19_6, d_var_19_7, d_var_19_8, d_var_19_9);
	
	kernel_20<<<10, 10>>>(d_var_20_0, d_var_20_1, d_var_20_2, d_var_20_3, d_var_20_4, d_var_20_5, d_var_20_6, d_var_20_7, d_var_20_8, d_var_20_9);
	
	kernel_21<<<10, 10>>>(d_var_21_0, d_var_21_1, d_var_21_2, d_var_21_3, d_var_21_4, d_var_21_5, d_var_21_6, d_var_21_7, d_var_21_8, d_var_21_9);
	
	kernel_22<<<10, 10>>>(d_var_22_0, d_var_22_1, d_var_22_2, d_var_22_3, d_var_22_4, d_var_22_5, d_var_22_6, d_var_22_7, d_var_22_8, d_var_22_9);
	
	kernel_23<<<10, 10>>>(d_var_23_0, d_var_23_1, d_var_23_2, d_var_23_3, d_var_23_4, d_var_23_5, d_var_23_6, d_var_23_7, d_var_23_8, d_var_23_9);
	
	kernel_24<<<10, 10>>>(d_var_24_0, d_var_24_1, d_var_24_2, d_var_24_3, d_var_24_4, d_var_24_5, d_var_24_6, d_var_24_7, d_var_24_8, d_var_24_9);
	
	kernel_25<<<10, 10>>>(d_var_25_0, d_var_25_1, d_var_25_2, d_var_25_3, d_var_25_4, d_var_25_5, d_var_25_6, d_var_25_7, d_var_25_8, d_var_25_9);
	
	kernel_26<<<10, 10>>>(d_var_26_0, d_var_26_1, d_var_26_2, d_var_26_3, d_var_26_4, d_var_26_5, d_var_26_6, d_var_26_7, d_var_26_8, d_var_26_9);
	
	kernel_27<<<10, 10>>>(d_var_27_0, d_var_27_1, d_var_27_2, d_var_27_3, d_var_27_4, d_var_27_5, d_var_27_6, d_var_27_7, d_var_27_8, d_var_27_9);
	
	kernel_28<<<10, 10>>>(d_var_28_0, d_var_28_1, d_var_28_2, d_var_28_3, d_var_28_4, d_var_28_5, d_var_28_6, d_var_28_7, d_var_28_8, d_var_28_9);
	
	kernel_29<<<10, 10>>>(d_var_29_0, d_var_29_1, d_var_29_2, d_var_29_3, d_var_29_4, d_var_29_5, d_var_29_6, d_var_29_7, d_var_29_8, d_var_29_9);
	
	kernel_30<<<10, 10>>>(d_var_30_0, d_var_30_1, d_var_30_2, d_var_30_3, d_var_30_4, d_var_30_5, d_var_30_6, d_var_30_7, d_var_30_8, d_var_30_9);
	
	kernel_31<<<10, 10>>>(d_var_31_0, d_var_31_1, d_var_31_2, d_var_31_3, d_var_31_4, d_var_31_5, d_var_31_6, d_var_31_7, d_var_31_8, d_var_31_9);
	
	kernel_32<<<10, 10>>>(d_var_32_0, d_var_32_1, d_var_32_2, d_var_32_3, d_var_32_4, d_var_32_5, d_var_32_6, d_var_32_7, d_var_32_8, d_var_32_9);
	
	kernel_33<<<10, 10>>>(d_var_33_0, d_var_33_1, d_var_33_2, d_var_33_3, d_var_33_4, d_var_33_5, d_var_33_6, d_var_33_7, d_var_33_8, d_var_33_9);
	
	kernel_34<<<10, 10>>>(d_var_34_0, d_var_34_1, d_var_34_2, d_var_34_3, d_var_34_4, d_var_34_5, d_var_34_6, d_var_34_7, d_var_34_8, d_var_34_9);
	
	kernel_35<<<10, 10>>>(d_var_35_0, d_var_35_1, d_var_35_2, d_var_35_3, d_var_35_4, d_var_35_5, d_var_35_6, d_var_35_7, d_var_35_8, d_var_35_9);
	
	kernel_36<<<10, 10>>>(d_var_36_0, d_var_36_1, d_var_36_2, d_var_36_3, d_var_36_4, d_var_36_5, d_var_36_6, d_var_36_7, d_var_36_8, d_var_36_9);
	
	kernel_37<<<10, 10>>>(d_var_37_0, d_var_37_1, d_var_37_2, d_var_37_3, d_var_37_4, d_var_37_5, d_var_37_6, d_var_37_7, d_var_37_8, d_var_37_9);
	
	kernel_38<<<10, 10>>>(d_var_38_0, d_var_38_1, d_var_38_2, d_var_38_3, d_var_38_4, d_var_38_5, d_var_38_6, d_var_38_7, d_var_38_8, d_var_38_9);
	
	kernel_39<<<10, 10>>>(d_var_39_0, d_var_39_1, d_var_39_2, d_var_39_3, d_var_39_4, d_var_39_5, d_var_39_6, d_var_39_7, d_var_39_8, d_var_39_9);
	
	kernel_40<<<10, 10>>>(d_var_40_0, d_var_40_1, d_var_40_2, d_var_40_3, d_var_40_4, d_var_40_5, d_var_40_6, d_var_40_7, d_var_40_8, d_var_40_9);
	
	kernel_41<<<10, 10>>>(d_var_41_0, d_var_41_1, d_var_41_2, d_var_41_3, d_var_41_4, d_var_41_5, d_var_41_6, d_var_41_7, d_var_41_8, d_var_41_9);
	
	kernel_42<<<10, 10>>>(d_var_42_0, d_var_42_1, d_var_42_2, d_var_42_3, d_var_42_4, d_var_42_5, d_var_42_6, d_var_42_7, d_var_42_8, d_var_42_9);
	
	kernel_43<<<10, 10>>>(d_var_43_0, d_var_43_1, d_var_43_2, d_var_43_3, d_var_43_4, d_var_43_5, d_var_43_6, d_var_43_7, d_var_43_8, d_var_43_9);
	
	kernel_44<<<10, 10>>>(d_var_44_0, d_var_44_1, d_var_44_2, d_var_44_3, d_var_44_4, d_var_44_5, d_var_44_6, d_var_44_7, d_var_44_8, d_var_44_9);
	
	kernel_45<<<10, 10>>>(d_var_45_0, d_var_45_1, d_var_45_2, d_var_45_3, d_var_45_4, d_var_45_5, d_var_45_6, d_var_45_7, d_var_45_8, d_var_45_9);
	
	kernel_46<<<10, 10>>>(d_var_46_0, d_var_46_1, d_var_46_2, d_var_46_3, d_var_46_4, d_var_46_5, d_var_46_6, d_var_46_7, d_var_46_8, d_var_46_9);
	
	kernel_47<<<10, 10>>>(d_var_47_0, d_var_47_1, d_var_47_2, d_var_47_3, d_var_47_4, d_var_47_5, d_var_47_6, d_var_47_7, d_var_47_8, d_var_47_9);
	
	kernel_48<<<10, 10>>>(d_var_48_0, d_var_48_1, d_var_48_2, d_var_48_3, d_var_48_4, d_var_48_5, d_var_48_6, d_var_48_7, d_var_48_8, d_var_48_9);
	
	kernel_49<<<10, 10>>>(d_var_49_0, d_var_49_1, d_var_49_2, d_var_49_3, d_var_49_4, d_var_49_5, d_var_49_6, d_var_49_7, d_var_49_8, d_var_49_9);
	
	kernel_50<<<10, 10>>>(d_var_50_0, d_var_50_1, d_var_50_2, d_var_50_3, d_var_50_4, d_var_50_5, d_var_50_6, d_var_50_7, d_var_50_8, d_var_50_9);
	
	kernel_51<<<10, 10>>>(d_var_51_0, d_var_51_1, d_var_51_2, d_var_51_3, d_var_51_4, d_var_51_5, d_var_51_6, d_var_51_7, d_var_51_8, d_var_51_9);
	
	kernel_52<<<10, 10>>>(d_var_52_0, d_var_52_1, d_var_52_2, d_var_52_3, d_var_52_4, d_var_52_5, d_var_52_6, d_var_52_7, d_var_52_8, d_var_52_9);
	
	kernel_53<<<10, 10>>>(d_var_53_0, d_var_53_1, d_var_53_2, d_var_53_3, d_var_53_4, d_var_53_5, d_var_53_6, d_var_53_7, d_var_53_8, d_var_53_9);
	
	kernel_54<<<10, 10>>>(d_var_54_0, d_var_54_1, d_var_54_2, d_var_54_3, d_var_54_4, d_var_54_5, d_var_54_6, d_var_54_7, d_var_54_8, d_var_54_9);
	
	kernel_55<<<10, 10>>>(d_var_55_0, d_var_55_1, d_var_55_2, d_var_55_3, d_var_55_4, d_var_55_5, d_var_55_6, d_var_55_7, d_var_55_8, d_var_55_9);
	
	kernel_56<<<10, 10>>>(d_var_56_0, d_var_56_1, d_var_56_2, d_var_56_3, d_var_56_4, d_var_56_5, d_var_56_6, d_var_56_7, d_var_56_8, d_var_56_9);
	
	kernel_57<<<10, 10>>>(d_var_57_0, d_var_57_1, d_var_57_2, d_var_57_3, d_var_57_4, d_var_57_5, d_var_57_6, d_var_57_7, d_var_57_8, d_var_57_9);
	
	kernel_58<<<10, 10>>>(d_var_58_0, d_var_58_1, d_var_58_2, d_var_58_3, d_var_58_4, d_var_58_5, d_var_58_6, d_var_58_7, d_var_58_8, d_var_58_9);
	
	kernel_59<<<10, 10>>>(d_var_59_0, d_var_59_1, d_var_59_2, d_var_59_3, d_var_59_4, d_var_59_5, d_var_59_6, d_var_59_7, d_var_59_8, d_var_59_9);
	
	kernel_60<<<10, 10>>>(d_var_60_0, d_var_60_1, d_var_60_2, d_var_60_3, d_var_60_4, d_var_60_5, d_var_60_6, d_var_60_7, d_var_60_8, d_var_60_9);
	
	kernel_61<<<10, 10>>>(d_var_61_0, d_var_61_1, d_var_61_2, d_var_61_3, d_var_61_4, d_var_61_5, d_var_61_6, d_var_61_7, d_var_61_8, d_var_61_9);
	
	kernel_62<<<10, 10>>>(d_var_62_0, d_var_62_1, d_var_62_2, d_var_62_3, d_var_62_4, d_var_62_5, d_var_62_6, d_var_62_7, d_var_62_8, d_var_62_9);
	
	kernel_63<<<10, 10>>>(d_var_63_0, d_var_63_1, d_var_63_2, d_var_63_3, d_var_63_4, d_var_63_5, d_var_63_6, d_var_63_7, d_var_63_8, d_var_63_9);
	
	kernel_64<<<10, 10>>>(d_var_64_0, d_var_64_1, d_var_64_2, d_var_64_3, d_var_64_4, d_var_64_5, d_var_64_6, d_var_64_7, d_var_64_8, d_var_64_9);
	
	kernel_65<<<10, 10>>>(d_var_65_0, d_var_65_1, d_var_65_2, d_var_65_3, d_var_65_4, d_var_65_5, d_var_65_6, d_var_65_7, d_var_65_8, d_var_65_9);
	
	kernel_66<<<10, 10>>>(d_var_66_0, d_var_66_1, d_var_66_2, d_var_66_3, d_var_66_4, d_var_66_5, d_var_66_6, d_var_66_7, d_var_66_8, d_var_66_9);
	
	kernel_67<<<10, 10>>>(d_var_67_0, d_var_67_1, d_var_67_2, d_var_67_3, d_var_67_4, d_var_67_5, d_var_67_6, d_var_67_7, d_var_67_8, d_var_67_9);
	
	kernel_68<<<10, 10>>>(d_var_68_0, d_var_68_1, d_var_68_2, d_var_68_3, d_var_68_4, d_var_68_5, d_var_68_6, d_var_68_7, d_var_68_8, d_var_68_9);
	
	kernel_69<<<10, 10>>>(d_var_69_0, d_var_69_1, d_var_69_2, d_var_69_3, d_var_69_4, d_var_69_5, d_var_69_6, d_var_69_7, d_var_69_8, d_var_69_9);
	
	kernel_70<<<10, 10>>>(d_var_70_0, d_var_70_1, d_var_70_2, d_var_70_3, d_var_70_4, d_var_70_5, d_var_70_6, d_var_70_7, d_var_70_8, d_var_70_9);
	
	kernel_71<<<10, 10>>>(d_var_71_0, d_var_71_1, d_var_71_2, d_var_71_3, d_var_71_4, d_var_71_5, d_var_71_6, d_var_71_7, d_var_71_8, d_var_71_9);
	
	kernel_72<<<10, 10>>>(d_var_72_0, d_var_72_1, d_var_72_2, d_var_72_3, d_var_72_4, d_var_72_5, d_var_72_6, d_var_72_7, d_var_72_8, d_var_72_9);
	
	kernel_73<<<10, 10>>>(d_var_73_0, d_var_73_1, d_var_73_2, d_var_73_3, d_var_73_4, d_var_73_5, d_var_73_6, d_var_73_7, d_var_73_8, d_var_73_9);
	
	kernel_74<<<10, 10>>>(d_var_74_0, d_var_74_1, d_var_74_2, d_var_74_3, d_var_74_4, d_var_74_5, d_var_74_6, d_var_74_7, d_var_74_8, d_var_74_9);
	
	kernel_75<<<10, 10>>>(d_var_75_0, d_var_75_1, d_var_75_2, d_var_75_3, d_var_75_4, d_var_75_5, d_var_75_6, d_var_75_7, d_var_75_8, d_var_75_9);
	
	kernel_76<<<10, 10>>>(d_var_76_0, d_var_76_1, d_var_76_2, d_var_76_3, d_var_76_4, d_var_76_5, d_var_76_6, d_var_76_7, d_var_76_8, d_var_76_9);
	
	kernel_77<<<10, 10>>>(d_var_77_0, d_var_77_1, d_var_77_2, d_var_77_3, d_var_77_4, d_var_77_5, d_var_77_6, d_var_77_7, d_var_77_8, d_var_77_9);
	
	kernel_78<<<10, 10>>>(d_var_78_0, d_var_78_1, d_var_78_2, d_var_78_3, d_var_78_4, d_var_78_5, d_var_78_6, d_var_78_7, d_var_78_8, d_var_78_9);
	
	kernel_79<<<10, 10>>>(d_var_79_0, d_var_79_1, d_var_79_2, d_var_79_3, d_var_79_4, d_var_79_5, d_var_79_6, d_var_79_7, d_var_79_8, d_var_79_9);
	
    // clang-format on

    printf("Done\n");
    return 0;
}
