/* AUTO-GENERATED, DO NOT CHANGE */

// To regenerate, run `make heavy-sample.cu` in this directory

#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void kernel_0(float * var_0_0, float * var_0_1, float * var_0_2, float * var_0_3, float * var_0_4, float * var_0_5, float * var_0_6, float * var_0_7, float * var_0_8, float * var_0_9, float * var_0_10, float * var_0_11, float * var_0_12, float * var_0_13, float * var_0_14, float * var_0_15, float * var_0_16, float * var_0_17, float * var_0_18, float * var_0_19) {
	__shared__ float myVar[1024];
	myVar[7] = 13.358492 * myVar[threadIdx.x];
	myVar[0] = 29.996864 * myVar[threadIdx.x];
	myVar[7] = 26.850372 * myVar[threadIdx.x];
	myVar[7] = 47.405648 * myVar[threadIdx.x];
	myVar[8] = 28.605543 * myVar[threadIdx.x];
	myVar[8] = 30.566722 * myVar[threadIdx.x];
	myVar[3] = 8.149732 * myVar[threadIdx.x];
	myVar[2] = 42.915701 * myVar[threadIdx.x];
	myVar[0] = 36.084747 * myVar[threadIdx.x];
	myVar[6] = 2.276106 * myVar[threadIdx.x];
	var_0_0[0] = myVar[0];
	var_0_1[1] = myVar[1];
	var_0_2[2] = myVar[2];
	var_0_3[3] = myVar[3];
	var_0_4[4] = myVar[4];
	var_0_5[5] = myVar[5];
	var_0_6[6] = myVar[6];
	var_0_7[7] = myVar[7];
	var_0_8[8] = myVar[8];
	var_0_9[9] = myVar[9];
	var_0_10[10] = myVar[10];
	var_0_11[11] = myVar[11];
	var_0_12[12] = myVar[12];
	var_0_13[13] = myVar[13];
	var_0_14[14] = myVar[14];
	var_0_15[15] = myVar[15];
	var_0_16[16] = myVar[16];
	var_0_17[17] = myVar[17];
	var_0_18[18] = myVar[18];
	var_0_19[19] = myVar[19];
	
}

__global__ void kernel_1(float * var_1_0, float * var_1_1, float * var_1_2, float * var_1_3, float * var_1_4, float * var_1_5, float * var_1_6, float * var_1_7, float * var_1_8, float * var_1_9, float * var_1_10, float * var_1_11, float * var_1_12, float * var_1_13, float * var_1_14, float * var_1_15, float * var_1_16, float * var_1_17, float * var_1_18, float * var_1_19) {
	__shared__ float myVar[1024];
	myVar[4] = 4.577338 * myVar[threadIdx.x];
	myVar[4] = 21.162567 * myVar[threadIdx.x];
	myVar[7] = 0.128262 * myVar[threadIdx.x];
	myVar[8] = 22.890511 * myVar[threadIdx.x];
	myVar[8] = 22.667310 * myVar[threadIdx.x];
	myVar[0] = 29.358554 * myVar[threadIdx.x];
	myVar[5] = 13.730931 * myVar[threadIdx.x];
	myVar[6] = 32.720741 * myVar[threadIdx.x];
	myVar[7] = 11.978346 * myVar[threadIdx.x];
	myVar[2] = 32.092827 * myVar[threadIdx.x];
	var_1_0[0] = myVar[0];
	var_1_1[1] = myVar[1];
	var_1_2[2] = myVar[2];
	var_1_3[3] = myVar[3];
	var_1_4[4] = myVar[4];
	var_1_5[5] = myVar[5];
	var_1_6[6] = myVar[6];
	var_1_7[7] = myVar[7];
	var_1_8[8] = myVar[8];
	var_1_9[9] = myVar[9];
	var_1_10[10] = myVar[10];
	var_1_11[11] = myVar[11];
	var_1_12[12] = myVar[12];
	var_1_13[13] = myVar[13];
	var_1_14[14] = myVar[14];
	var_1_15[15] = myVar[15];
	var_1_16[16] = myVar[16];
	var_1_17[17] = myVar[17];
	var_1_18[18] = myVar[18];
	var_1_19[19] = myVar[19];
	
}

__global__ void kernel_2(float * var_2_0, float * var_2_1, float * var_2_2, float * var_2_3, float * var_2_4, float * var_2_5, float * var_2_6, float * var_2_7, float * var_2_8, float * var_2_9, float * var_2_10, float * var_2_11, float * var_2_12, float * var_2_13, float * var_2_14, float * var_2_15, float * var_2_16, float * var_2_17, float * var_2_18, float * var_2_19) {
	__shared__ float myVar[1024];
	myVar[3] = 49.191664 * myVar[threadIdx.x];
	myVar[3] = 5.722245 * myVar[threadIdx.x];
	myVar[5] = 3.621897 * myVar[threadIdx.x];
	myVar[0] = 10.483091 * myVar[threadIdx.x];
	myVar[6] = 36.425846 * myVar[threadIdx.x];
	myVar[9] = 49.066374 * myVar[threadIdx.x];
	myVar[3] = 41.370231 * myVar[threadIdx.x];
	myVar[2] = 38.310331 * myVar[threadIdx.x];
	myVar[2] = 38.696066 * myVar[threadIdx.x];
	myVar[5] = 17.802057 * myVar[threadIdx.x];
	var_2_0[0] = myVar[0];
	var_2_1[1] = myVar[1];
	var_2_2[2] = myVar[2];
	var_2_3[3] = myVar[3];
	var_2_4[4] = myVar[4];
	var_2_5[5] = myVar[5];
	var_2_6[6] = myVar[6];
	var_2_7[7] = myVar[7];
	var_2_8[8] = myVar[8];
	var_2_9[9] = myVar[9];
	var_2_10[10] = myVar[10];
	var_2_11[11] = myVar[11];
	var_2_12[12] = myVar[12];
	var_2_13[13] = myVar[13];
	var_2_14[14] = myVar[14];
	var_2_15[15] = myVar[15];
	var_2_16[16] = myVar[16];
	var_2_17[17] = myVar[17];
	var_2_18[18] = myVar[18];
	var_2_19[19] = myVar[19];
	
}

__global__ void kernel_3(float * var_3_0, float * var_3_1, float * var_3_2, float * var_3_3, float * var_3_4, float * var_3_5, float * var_3_6, float * var_3_7, float * var_3_8, float * var_3_9, float * var_3_10, float * var_3_11, float * var_3_12, float * var_3_13, float * var_3_14, float * var_3_15, float * var_3_16, float * var_3_17, float * var_3_18, float * var_3_19) {
	__shared__ float myVar[1024];
	myVar[1] = 3.657881 * myVar[threadIdx.x];
	myVar[9] = 47.144202 * myVar[threadIdx.x];
	myVar[7] = 26.768394 * myVar[threadIdx.x];
	myVar[4] = 25.254215 * myVar[threadIdx.x];
	myVar[8] = 36.901003 * myVar[threadIdx.x];
	myVar[5] = 12.665010 * myVar[threadIdx.x];
	myVar[6] = 12.501012 * myVar[threadIdx.x];
	myVar[2] = 33.849199 * myVar[threadIdx.x];
	myVar[1] = 44.025130 * myVar[threadIdx.x];
	myVar[1] = 48.566763 * myVar[threadIdx.x];
	var_3_0[0] = myVar[0];
	var_3_1[1] = myVar[1];
	var_3_2[2] = myVar[2];
	var_3_3[3] = myVar[3];
	var_3_4[4] = myVar[4];
	var_3_5[5] = myVar[5];
	var_3_6[6] = myVar[6];
	var_3_7[7] = myVar[7];
	var_3_8[8] = myVar[8];
	var_3_9[9] = myVar[9];
	var_3_10[10] = myVar[10];
	var_3_11[11] = myVar[11];
	var_3_12[12] = myVar[12];
	var_3_13[13] = myVar[13];
	var_3_14[14] = myVar[14];
	var_3_15[15] = myVar[15];
	var_3_16[16] = myVar[16];
	var_3_17[17] = myVar[17];
	var_3_18[18] = myVar[18];
	var_3_19[19] = myVar[19];
	
}

__global__ void kernel_4(float * var_4_0, float * var_4_1, float * var_4_2, float * var_4_3, float * var_4_4, float * var_4_5, float * var_4_6, float * var_4_7, float * var_4_8, float * var_4_9, float * var_4_10, float * var_4_11, float * var_4_12, float * var_4_13, float * var_4_14, float * var_4_15, float * var_4_16, float * var_4_17, float * var_4_18, float * var_4_19) {
	__shared__ float myVar[1024];
	myVar[9] = 34.814125 * myVar[threadIdx.x];
	myVar[0] = 7.439284 * myVar[threadIdx.x];
	myVar[8] = 24.182738 * myVar[threadIdx.x];
	myVar[6] = 41.521589 * myVar[threadIdx.x];
	myVar[0] = 8.476383 * myVar[threadIdx.x];
	myVar[4] = 4.308362 * myVar[threadIdx.x];
	myVar[0] = 34.019957 * myVar[threadIdx.x];
	myVar[4] = 14.880842 * myVar[threadIdx.x];
	myVar[1] = 25.167021 * myVar[threadIdx.x];
	myVar[8] = 14.620295 * myVar[threadIdx.x];
	var_4_0[0] = myVar[0];
	var_4_1[1] = myVar[1];
	var_4_2[2] = myVar[2];
	var_4_3[3] = myVar[3];
	var_4_4[4] = myVar[4];
	var_4_5[5] = myVar[5];
	var_4_6[6] = myVar[6];
	var_4_7[7] = myVar[7];
	var_4_8[8] = myVar[8];
	var_4_9[9] = myVar[9];
	var_4_10[10] = myVar[10];
	var_4_11[11] = myVar[11];
	var_4_12[12] = myVar[12];
	var_4_13[13] = myVar[13];
	var_4_14[14] = myVar[14];
	var_4_15[15] = myVar[15];
	var_4_16[16] = myVar[16];
	var_4_17[17] = myVar[17];
	var_4_18[18] = myVar[18];
	var_4_19[19] = myVar[19];
	
}

__global__ void kernel_5(float * var_5_0, float * var_5_1, float * var_5_2, float * var_5_3, float * var_5_4, float * var_5_5, float * var_5_6, float * var_5_7, float * var_5_8, float * var_5_9, float * var_5_10, float * var_5_11, float * var_5_12, float * var_5_13, float * var_5_14, float * var_5_15, float * var_5_16, float * var_5_17, float * var_5_18, float * var_5_19) {
	__shared__ float myVar[1024];
	myVar[2] = 37.042397 * myVar[threadIdx.x];
	myVar[0] = 15.897506 * myVar[threadIdx.x];
	myVar[9] = 29.881857 * myVar[threadIdx.x];
	myVar[4] = 28.287586 * myVar[threadIdx.x];
	myVar[8] = 13.202707 * myVar[threadIdx.x];
	myVar[4] = 23.235843 * myVar[threadIdx.x];
	myVar[3] = 45.601563 * myVar[threadIdx.x];
	myVar[0] = 22.131163 * myVar[threadIdx.x];
	myVar[1] = 2.447910 * myVar[threadIdx.x];
	myVar[6] = 48.949264 * myVar[threadIdx.x];
	var_5_0[0] = myVar[0];
	var_5_1[1] = myVar[1];
	var_5_2[2] = myVar[2];
	var_5_3[3] = myVar[3];
	var_5_4[4] = myVar[4];
	var_5_5[5] = myVar[5];
	var_5_6[6] = myVar[6];
	var_5_7[7] = myVar[7];
	var_5_8[8] = myVar[8];
	var_5_9[9] = myVar[9];
	var_5_10[10] = myVar[10];
	var_5_11[11] = myVar[11];
	var_5_12[12] = myVar[12];
	var_5_13[13] = myVar[13];
	var_5_14[14] = myVar[14];
	var_5_15[15] = myVar[15];
	var_5_16[16] = myVar[16];
	var_5_17[17] = myVar[17];
	var_5_18[18] = myVar[18];
	var_5_19[19] = myVar[19];
	
}

__global__ void kernel_6(float * var_6_0, float * var_6_1, float * var_6_2, float * var_6_3, float * var_6_4, float * var_6_5, float * var_6_6, float * var_6_7, float * var_6_8, float * var_6_9, float * var_6_10, float * var_6_11, float * var_6_12, float * var_6_13, float * var_6_14, float * var_6_15, float * var_6_16, float * var_6_17, float * var_6_18, float * var_6_19) {
	__shared__ float myVar[1024];
	myVar[5] = 42.612716 * myVar[threadIdx.x];
	myVar[1] = 32.709412 * myVar[threadIdx.x];
	myVar[2] = 1.664703 * myVar[threadIdx.x];
	myVar[7] = 17.973638 * myVar[threadIdx.x];
	myVar[1] = 20.774570 * myVar[threadIdx.x];
	myVar[5] = 16.371010 * myVar[threadIdx.x];
	myVar[3] = 19.487553 * myVar[threadIdx.x];
	myVar[3] = 36.774980 * myVar[threadIdx.x];
	myVar[5] = 28.604668 * myVar[threadIdx.x];
	myVar[2] = 15.117050 * myVar[threadIdx.x];
	var_6_0[0] = myVar[0];
	var_6_1[1] = myVar[1];
	var_6_2[2] = myVar[2];
	var_6_3[3] = myVar[3];
	var_6_4[4] = myVar[4];
	var_6_5[5] = myVar[5];
	var_6_6[6] = myVar[6];
	var_6_7[7] = myVar[7];
	var_6_8[8] = myVar[8];
	var_6_9[9] = myVar[9];
	var_6_10[10] = myVar[10];
	var_6_11[11] = myVar[11];
	var_6_12[12] = myVar[12];
	var_6_13[13] = myVar[13];
	var_6_14[14] = myVar[14];
	var_6_15[15] = myVar[15];
	var_6_16[16] = myVar[16];
	var_6_17[17] = myVar[17];
	var_6_18[18] = myVar[18];
	var_6_19[19] = myVar[19];
	
}

__global__ void kernel_7(float * var_7_0, float * var_7_1, float * var_7_2, float * var_7_3, float * var_7_4, float * var_7_5, float * var_7_6, float * var_7_7, float * var_7_8, float * var_7_9, float * var_7_10, float * var_7_11, float * var_7_12, float * var_7_13, float * var_7_14, float * var_7_15, float * var_7_16, float * var_7_17, float * var_7_18, float * var_7_19) {
	__shared__ float myVar[1024];
	myVar[4] = 9.570039 * myVar[threadIdx.x];
	myVar[9] = 7.446622 * myVar[threadIdx.x];
	myVar[8] = 4.543599 * myVar[threadIdx.x];
	myVar[7] = 26.174945 * myVar[threadIdx.x];
	myVar[4] = 13.540677 * myVar[threadIdx.x];
	myVar[9] = 33.803400 * myVar[threadIdx.x];
	myVar[8] = 24.045981 * myVar[threadIdx.x];
	myVar[3] = 22.198640 * myVar[threadIdx.x];
	myVar[9] = 22.882328 * myVar[threadIdx.x];
	myVar[3] = 9.353756 * myVar[threadIdx.x];
	var_7_0[0] = myVar[0];
	var_7_1[1] = myVar[1];
	var_7_2[2] = myVar[2];
	var_7_3[3] = myVar[3];
	var_7_4[4] = myVar[4];
	var_7_5[5] = myVar[5];
	var_7_6[6] = myVar[6];
	var_7_7[7] = myVar[7];
	var_7_8[8] = myVar[8];
	var_7_9[9] = myVar[9];
	var_7_10[10] = myVar[10];
	var_7_11[11] = myVar[11];
	var_7_12[12] = myVar[12];
	var_7_13[13] = myVar[13];
	var_7_14[14] = myVar[14];
	var_7_15[15] = myVar[15];
	var_7_16[16] = myVar[16];
	var_7_17[17] = myVar[17];
	var_7_18[18] = myVar[18];
	var_7_19[19] = myVar[19];
	
}

__global__ void kernel_8(float * var_8_0, float * var_8_1, float * var_8_2, float * var_8_3, float * var_8_4, float * var_8_5, float * var_8_6, float * var_8_7, float * var_8_8, float * var_8_9, float * var_8_10, float * var_8_11, float * var_8_12, float * var_8_13, float * var_8_14, float * var_8_15, float * var_8_16, float * var_8_17, float * var_8_18, float * var_8_19) {
	__shared__ float myVar[1024];
	myVar[7] = 15.547479 * myVar[threadIdx.x];
	myVar[3] = 48.901650 * myVar[threadIdx.x];
	myVar[5] = 26.480024 * myVar[threadIdx.x];
	myVar[6] = 3.905452 * myVar[threadIdx.x];
	myVar[8] = 10.110801 * myVar[threadIdx.x];
	myVar[5] = 39.252634 * myVar[threadIdx.x];
	myVar[0] = 40.038305 * myVar[threadIdx.x];
	myVar[7] = 35.166664 * myVar[threadIdx.x];
	myVar[2] = 37.185844 * myVar[threadIdx.x];
	myVar[5] = 18.754616 * myVar[threadIdx.x];
	var_8_0[0] = myVar[0];
	var_8_1[1] = myVar[1];
	var_8_2[2] = myVar[2];
	var_8_3[3] = myVar[3];
	var_8_4[4] = myVar[4];
	var_8_5[5] = myVar[5];
	var_8_6[6] = myVar[6];
	var_8_7[7] = myVar[7];
	var_8_8[8] = myVar[8];
	var_8_9[9] = myVar[9];
	var_8_10[10] = myVar[10];
	var_8_11[11] = myVar[11];
	var_8_12[12] = myVar[12];
	var_8_13[13] = myVar[13];
	var_8_14[14] = myVar[14];
	var_8_15[15] = myVar[15];
	var_8_16[16] = myVar[16];
	var_8_17[17] = myVar[17];
	var_8_18[18] = myVar[18];
	var_8_19[19] = myVar[19];
	
}

__global__ void kernel_9(float * var_9_0, float * var_9_1, float * var_9_2, float * var_9_3, float * var_9_4, float * var_9_5, float * var_9_6, float * var_9_7, float * var_9_8, float * var_9_9, float * var_9_10, float * var_9_11, float * var_9_12, float * var_9_13, float * var_9_14, float * var_9_15, float * var_9_16, float * var_9_17, float * var_9_18, float * var_9_19) {
	__shared__ float myVar[1024];
	myVar[6] = 14.290958 * myVar[threadIdx.x];
	myVar[8] = 42.139091 * myVar[threadIdx.x];
	myVar[8] = 36.378596 * myVar[threadIdx.x];
	myVar[3] = 46.441771 * myVar[threadIdx.x];
	myVar[6] = 8.854676 * myVar[threadIdx.x];
	myVar[8] = 20.100396 * myVar[threadIdx.x];
	myVar[6] = 8.781566 * myVar[threadIdx.x];
	myVar[8] = 44.446736 * myVar[threadIdx.x];
	myVar[5] = 7.496880 * myVar[threadIdx.x];
	myVar[8] = 40.785167 * myVar[threadIdx.x];
	var_9_0[0] = myVar[0];
	var_9_1[1] = myVar[1];
	var_9_2[2] = myVar[2];
	var_9_3[3] = myVar[3];
	var_9_4[4] = myVar[4];
	var_9_5[5] = myVar[5];
	var_9_6[6] = myVar[6];
	var_9_7[7] = myVar[7];
	var_9_8[8] = myVar[8];
	var_9_9[9] = myVar[9];
	var_9_10[10] = myVar[10];
	var_9_11[11] = myVar[11];
	var_9_12[12] = myVar[12];
	var_9_13[13] = myVar[13];
	var_9_14[14] = myVar[14];
	var_9_15[15] = myVar[15];
	var_9_16[16] = myVar[16];
	var_9_17[17] = myVar[17];
	var_9_18[18] = myVar[18];
	var_9_19[19] = myVar[19];
	
}

__global__ void kernel_10(float * var_10_0, float * var_10_1, float * var_10_2, float * var_10_3, float * var_10_4, float * var_10_5, float * var_10_6, float * var_10_7, float * var_10_8, float * var_10_9, float * var_10_10, float * var_10_11, float * var_10_12, float * var_10_13, float * var_10_14, float * var_10_15, float * var_10_16, float * var_10_17, float * var_10_18, float * var_10_19) {
	__shared__ float myVar[1024];
	myVar[6] = 41.374924 * myVar[threadIdx.x];
	myVar[7] = 5.583937 * myVar[threadIdx.x];
	myVar[1] = 11.415106 * myVar[threadIdx.x];
	myVar[7] = 16.040032 * myVar[threadIdx.x];
	myVar[9] = 49.003678 * myVar[threadIdx.x];
	myVar[9] = 33.529291 * myVar[threadIdx.x];
	myVar[9] = 3.725300 * myVar[threadIdx.x];
	myVar[9] = 36.318961 * myVar[threadIdx.x];
	myVar[4] = 34.676808 * myVar[threadIdx.x];
	myVar[8] = 0.789490 * myVar[threadIdx.x];
	var_10_0[0] = myVar[0];
	var_10_1[1] = myVar[1];
	var_10_2[2] = myVar[2];
	var_10_3[3] = myVar[3];
	var_10_4[4] = myVar[4];
	var_10_5[5] = myVar[5];
	var_10_6[6] = myVar[6];
	var_10_7[7] = myVar[7];
	var_10_8[8] = myVar[8];
	var_10_9[9] = myVar[9];
	var_10_10[10] = myVar[10];
	var_10_11[11] = myVar[11];
	var_10_12[12] = myVar[12];
	var_10_13[13] = myVar[13];
	var_10_14[14] = myVar[14];
	var_10_15[15] = myVar[15];
	var_10_16[16] = myVar[16];
	var_10_17[17] = myVar[17];
	var_10_18[18] = myVar[18];
	var_10_19[19] = myVar[19];
	
}

__global__ void kernel_11(float * var_11_0, float * var_11_1, float * var_11_2, float * var_11_3, float * var_11_4, float * var_11_5, float * var_11_6, float * var_11_7, float * var_11_8, float * var_11_9, float * var_11_10, float * var_11_11, float * var_11_12, float * var_11_13, float * var_11_14, float * var_11_15, float * var_11_16, float * var_11_17, float * var_11_18, float * var_11_19) {
	__shared__ float myVar[1024];
	myVar[5] = 27.667212 * myVar[threadIdx.x];
	myVar[2] = 12.702892 * myVar[threadIdx.x];
	myVar[2] = 18.730612 * myVar[threadIdx.x];
	myVar[9] = 8.009668 * myVar[threadIdx.x];
	myVar[9] = 1.903979 * myVar[threadIdx.x];
	myVar[9] = 20.042565 * myVar[threadIdx.x];
	myVar[0] = 34.292619 * myVar[threadIdx.x];
	myVar[9] = 39.456481 * myVar[threadIdx.x];
	myVar[1] = 25.805260 * myVar[threadIdx.x];
	myVar[7] = 10.881045 * myVar[threadIdx.x];
	var_11_0[0] = myVar[0];
	var_11_1[1] = myVar[1];
	var_11_2[2] = myVar[2];
	var_11_3[3] = myVar[3];
	var_11_4[4] = myVar[4];
	var_11_5[5] = myVar[5];
	var_11_6[6] = myVar[6];
	var_11_7[7] = myVar[7];
	var_11_8[8] = myVar[8];
	var_11_9[9] = myVar[9];
	var_11_10[10] = myVar[10];
	var_11_11[11] = myVar[11];
	var_11_12[12] = myVar[12];
	var_11_13[13] = myVar[13];
	var_11_14[14] = myVar[14];
	var_11_15[15] = myVar[15];
	var_11_16[16] = myVar[16];
	var_11_17[17] = myVar[17];
	var_11_18[18] = myVar[18];
	var_11_19[19] = myVar[19];
	
}

__global__ void kernel_12(float * var_12_0, float * var_12_1, float * var_12_2, float * var_12_3, float * var_12_4, float * var_12_5, float * var_12_6, float * var_12_7, float * var_12_8, float * var_12_9, float * var_12_10, float * var_12_11, float * var_12_12, float * var_12_13, float * var_12_14, float * var_12_15, float * var_12_16, float * var_12_17, float * var_12_18, float * var_12_19) {
	__shared__ float myVar[1024];
	myVar[4] = 25.536327 * myVar[threadIdx.x];
	myVar[6] = 5.530078 * myVar[threadIdx.x];
	myVar[4] = 1.930924 * myVar[threadIdx.x];
	myVar[9] = 38.945469 * myVar[threadIdx.x];
	myVar[9] = 35.199808 * myVar[threadIdx.x];
	myVar[4] = 45.394917 * myVar[threadIdx.x];
	myVar[1] = 25.106426 * myVar[threadIdx.x];
	myVar[3] = 43.623141 * myVar[threadIdx.x];
	myVar[5] = 39.114022 * myVar[threadIdx.x];
	myVar[9] = 33.734836 * myVar[threadIdx.x];
	var_12_0[0] = myVar[0];
	var_12_1[1] = myVar[1];
	var_12_2[2] = myVar[2];
	var_12_3[3] = myVar[3];
	var_12_4[4] = myVar[4];
	var_12_5[5] = myVar[5];
	var_12_6[6] = myVar[6];
	var_12_7[7] = myVar[7];
	var_12_8[8] = myVar[8];
	var_12_9[9] = myVar[9];
	var_12_10[10] = myVar[10];
	var_12_11[11] = myVar[11];
	var_12_12[12] = myVar[12];
	var_12_13[13] = myVar[13];
	var_12_14[14] = myVar[14];
	var_12_15[15] = myVar[15];
	var_12_16[16] = myVar[16];
	var_12_17[17] = myVar[17];
	var_12_18[18] = myVar[18];
	var_12_19[19] = myVar[19];
	
}

__global__ void kernel_13(float * var_13_0, float * var_13_1, float * var_13_2, float * var_13_3, float * var_13_4, float * var_13_5, float * var_13_6, float * var_13_7, float * var_13_8, float * var_13_9, float * var_13_10, float * var_13_11, float * var_13_12, float * var_13_13, float * var_13_14, float * var_13_15, float * var_13_16, float * var_13_17, float * var_13_18, float * var_13_19) {
	__shared__ float myVar[1024];
	myVar[9] = 37.548477 * myVar[threadIdx.x];
	myVar[2] = 42.389733 * myVar[threadIdx.x];
	myVar[2] = 9.754293 * myVar[threadIdx.x];
	myVar[1] = 24.123587 * myVar[threadIdx.x];
	myVar[2] = 23.260944 * myVar[threadIdx.x];
	myVar[9] = 18.619336 * myVar[threadIdx.x];
	myVar[2] = 17.496481 * myVar[threadIdx.x];
	myVar[1] = 40.139043 * myVar[threadIdx.x];
	myVar[5] = 22.620901 * myVar[threadIdx.x];
	myVar[8] = 29.155860 * myVar[threadIdx.x];
	var_13_0[0] = myVar[0];
	var_13_1[1] = myVar[1];
	var_13_2[2] = myVar[2];
	var_13_3[3] = myVar[3];
	var_13_4[4] = myVar[4];
	var_13_5[5] = myVar[5];
	var_13_6[6] = myVar[6];
	var_13_7[7] = myVar[7];
	var_13_8[8] = myVar[8];
	var_13_9[9] = myVar[9];
	var_13_10[10] = myVar[10];
	var_13_11[11] = myVar[11];
	var_13_12[12] = myVar[12];
	var_13_13[13] = myVar[13];
	var_13_14[14] = myVar[14];
	var_13_15[15] = myVar[15];
	var_13_16[16] = myVar[16];
	var_13_17[17] = myVar[17];
	var_13_18[18] = myVar[18];
	var_13_19[19] = myVar[19];
	
}

__global__ void kernel_14(float * var_14_0, float * var_14_1, float * var_14_2, float * var_14_3, float * var_14_4, float * var_14_5, float * var_14_6, float * var_14_7, float * var_14_8, float * var_14_9, float * var_14_10, float * var_14_11, float * var_14_12, float * var_14_13, float * var_14_14, float * var_14_15, float * var_14_16, float * var_14_17, float * var_14_18, float * var_14_19) {
	__shared__ float myVar[1024];
	myVar[8] = 20.535383 * myVar[threadIdx.x];
	myVar[1] = 35.714406 * myVar[threadIdx.x];
	myVar[9] = 29.199587 * myVar[threadIdx.x];
	myVar[4] = 33.618211 * myVar[threadIdx.x];
	myVar[8] = 9.594275 * myVar[threadIdx.x];
	myVar[9] = 7.102031 * myVar[threadIdx.x];
	myVar[9] = 5.373745 * myVar[threadIdx.x];
	myVar[6] = 33.548882 * myVar[threadIdx.x];
	myVar[7] = 25.948322 * myVar[threadIdx.x];
	myVar[9] = 19.674437 * myVar[threadIdx.x];
	var_14_0[0] = myVar[0];
	var_14_1[1] = myVar[1];
	var_14_2[2] = myVar[2];
	var_14_3[3] = myVar[3];
	var_14_4[4] = myVar[4];
	var_14_5[5] = myVar[5];
	var_14_6[6] = myVar[6];
	var_14_7[7] = myVar[7];
	var_14_8[8] = myVar[8];
	var_14_9[9] = myVar[9];
	var_14_10[10] = myVar[10];
	var_14_11[11] = myVar[11];
	var_14_12[12] = myVar[12];
	var_14_13[13] = myVar[13];
	var_14_14[14] = myVar[14];
	var_14_15[15] = myVar[15];
	var_14_16[16] = myVar[16];
	var_14_17[17] = myVar[17];
	var_14_18[18] = myVar[18];
	var_14_19[19] = myVar[19];
	
}

__global__ void kernel_15(float * var_15_0, float * var_15_1, float * var_15_2, float * var_15_3, float * var_15_4, float * var_15_5, float * var_15_6, float * var_15_7, float * var_15_8, float * var_15_9, float * var_15_10, float * var_15_11, float * var_15_12, float * var_15_13, float * var_15_14, float * var_15_15, float * var_15_16, float * var_15_17, float * var_15_18, float * var_15_19) {
	__shared__ float myVar[1024];
	myVar[4] = 16.821004 * myVar[threadIdx.x];
	myVar[1] = 40.474564 * myVar[threadIdx.x];
	myVar[0] = 4.896632 * myVar[threadIdx.x];
	myVar[0] = 49.157457 * myVar[threadIdx.x];
	myVar[1] = 43.538341 * myVar[threadIdx.x];
	myVar[5] = 3.270043 * myVar[threadIdx.x];
	myVar[5] = 10.193788 * myVar[threadIdx.x];
	myVar[1] = 2.723051 * myVar[threadIdx.x];
	myVar[3] = 14.915672 * myVar[threadIdx.x];
	myVar[8] = 30.460698 * myVar[threadIdx.x];
	var_15_0[0] = myVar[0];
	var_15_1[1] = myVar[1];
	var_15_2[2] = myVar[2];
	var_15_3[3] = myVar[3];
	var_15_4[4] = myVar[4];
	var_15_5[5] = myVar[5];
	var_15_6[6] = myVar[6];
	var_15_7[7] = myVar[7];
	var_15_8[8] = myVar[8];
	var_15_9[9] = myVar[9];
	var_15_10[10] = myVar[10];
	var_15_11[11] = myVar[11];
	var_15_12[12] = myVar[12];
	var_15_13[13] = myVar[13];
	var_15_14[14] = myVar[14];
	var_15_15[15] = myVar[15];
	var_15_16[16] = myVar[16];
	var_15_17[17] = myVar[17];
	var_15_18[18] = myVar[18];
	var_15_19[19] = myVar[19];
	
}

__global__ void kernel_16(float * var_16_0, float * var_16_1, float * var_16_2, float * var_16_3, float * var_16_4, float * var_16_5, float * var_16_6, float * var_16_7, float * var_16_8, float * var_16_9, float * var_16_10, float * var_16_11, float * var_16_12, float * var_16_13, float * var_16_14, float * var_16_15, float * var_16_16, float * var_16_17, float * var_16_18, float * var_16_19) {
	__shared__ float myVar[1024];
	myVar[1] = 30.125380 * myVar[threadIdx.x];
	myVar[5] = 49.279292 * myVar[threadIdx.x];
	myVar[4] = 40.110493 * myVar[threadIdx.x];
	myVar[6] = 1.210120 * myVar[threadIdx.x];
	myVar[5] = 6.750555 * myVar[threadIdx.x];
	myVar[3] = 41.840500 * myVar[threadIdx.x];
	myVar[8] = 32.901153 * myVar[threadIdx.x];
	myVar[3] = 28.661612 * myVar[threadIdx.x];
	myVar[4] = 2.114233 * myVar[threadIdx.x];
	myVar[0] = 41.587771 * myVar[threadIdx.x];
	var_16_0[0] = myVar[0];
	var_16_1[1] = myVar[1];
	var_16_2[2] = myVar[2];
	var_16_3[3] = myVar[3];
	var_16_4[4] = myVar[4];
	var_16_5[5] = myVar[5];
	var_16_6[6] = myVar[6];
	var_16_7[7] = myVar[7];
	var_16_8[8] = myVar[8];
	var_16_9[9] = myVar[9];
	var_16_10[10] = myVar[10];
	var_16_11[11] = myVar[11];
	var_16_12[12] = myVar[12];
	var_16_13[13] = myVar[13];
	var_16_14[14] = myVar[14];
	var_16_15[15] = myVar[15];
	var_16_16[16] = myVar[16];
	var_16_17[17] = myVar[17];
	var_16_18[18] = myVar[18];
	var_16_19[19] = myVar[19];
	
}

__global__ void kernel_17(float * var_17_0, float * var_17_1, float * var_17_2, float * var_17_3, float * var_17_4, float * var_17_5, float * var_17_6, float * var_17_7, float * var_17_8, float * var_17_9, float * var_17_10, float * var_17_11, float * var_17_12, float * var_17_13, float * var_17_14, float * var_17_15, float * var_17_16, float * var_17_17, float * var_17_18, float * var_17_19) {
	__shared__ float myVar[1024];
	myVar[5] = 42.657163 * myVar[threadIdx.x];
	myVar[7] = 46.068280 * myVar[threadIdx.x];
	myVar[4] = 9.105916 * myVar[threadIdx.x];
	myVar[5] = 30.973747 * myVar[threadIdx.x];
	myVar[2] = 17.915047 * myVar[threadIdx.x];
	myVar[7] = 26.762380 * myVar[threadIdx.x];
	myVar[4] = 33.421623 * myVar[threadIdx.x];
	myVar[1] = 44.503851 * myVar[threadIdx.x];
	myVar[5] = 25.264571 * myVar[threadIdx.x];
	myVar[6] = 33.079366 * myVar[threadIdx.x];
	var_17_0[0] = myVar[0];
	var_17_1[1] = myVar[1];
	var_17_2[2] = myVar[2];
	var_17_3[3] = myVar[3];
	var_17_4[4] = myVar[4];
	var_17_5[5] = myVar[5];
	var_17_6[6] = myVar[6];
	var_17_7[7] = myVar[7];
	var_17_8[8] = myVar[8];
	var_17_9[9] = myVar[9];
	var_17_10[10] = myVar[10];
	var_17_11[11] = myVar[11];
	var_17_12[12] = myVar[12];
	var_17_13[13] = myVar[13];
	var_17_14[14] = myVar[14];
	var_17_15[15] = myVar[15];
	var_17_16[16] = myVar[16];
	var_17_17[17] = myVar[17];
	var_17_18[18] = myVar[18];
	var_17_19[19] = myVar[19];
	
}

__global__ void kernel_18(float * var_18_0, float * var_18_1, float * var_18_2, float * var_18_3, float * var_18_4, float * var_18_5, float * var_18_6, float * var_18_7, float * var_18_8, float * var_18_9, float * var_18_10, float * var_18_11, float * var_18_12, float * var_18_13, float * var_18_14, float * var_18_15, float * var_18_16, float * var_18_17, float * var_18_18, float * var_18_19) {
	__shared__ float myVar[1024];
	myVar[7] = 47.140548 * myVar[threadIdx.x];
	myVar[0] = 46.028238 * myVar[threadIdx.x];
	myVar[7] = 6.932360 * myVar[threadIdx.x];
	myVar[8] = 29.252615 * myVar[threadIdx.x];
	myVar[6] = 8.795276 * myVar[threadIdx.x];
	myVar[1] = 25.034969 * myVar[threadIdx.x];
	myVar[0] = 33.216461 * myVar[threadIdx.x];
	myVar[0] = 11.724631 * myVar[threadIdx.x];
	myVar[0] = 2.454614 * myVar[threadIdx.x];
	myVar[4] = 6.795019 * myVar[threadIdx.x];
	var_18_0[0] = myVar[0];
	var_18_1[1] = myVar[1];
	var_18_2[2] = myVar[2];
	var_18_3[3] = myVar[3];
	var_18_4[4] = myVar[4];
	var_18_5[5] = myVar[5];
	var_18_6[6] = myVar[6];
	var_18_7[7] = myVar[7];
	var_18_8[8] = myVar[8];
	var_18_9[9] = myVar[9];
	var_18_10[10] = myVar[10];
	var_18_11[11] = myVar[11];
	var_18_12[12] = myVar[12];
	var_18_13[13] = myVar[13];
	var_18_14[14] = myVar[14];
	var_18_15[15] = myVar[15];
	var_18_16[16] = myVar[16];
	var_18_17[17] = myVar[17];
	var_18_18[18] = myVar[18];
	var_18_19[19] = myVar[19];
	
}

__global__ void kernel_19(float * var_19_0, float * var_19_1, float * var_19_2, float * var_19_3, float * var_19_4, float * var_19_5, float * var_19_6, float * var_19_7, float * var_19_8, float * var_19_9, float * var_19_10, float * var_19_11, float * var_19_12, float * var_19_13, float * var_19_14, float * var_19_15, float * var_19_16, float * var_19_17, float * var_19_18, float * var_19_19) {
	__shared__ float myVar[1024];
	myVar[5] = 44.514315 * myVar[threadIdx.x];
	myVar[5] = 15.645175 * myVar[threadIdx.x];
	myVar[5] = 31.812628 * myVar[threadIdx.x];
	myVar[0] = 5.528910 * myVar[threadIdx.x];
	myVar[5] = 40.822349 * myVar[threadIdx.x];
	myVar[9] = 19.084764 * myVar[threadIdx.x];
	myVar[4] = 22.711739 * myVar[threadIdx.x];
	myVar[0] = 12.284228 * myVar[threadIdx.x];
	myVar[6] = 8.482777 * myVar[threadIdx.x];
	myVar[7] = 40.246930 * myVar[threadIdx.x];
	var_19_0[0] = myVar[0];
	var_19_1[1] = myVar[1];
	var_19_2[2] = myVar[2];
	var_19_3[3] = myVar[3];
	var_19_4[4] = myVar[4];
	var_19_5[5] = myVar[5];
	var_19_6[6] = myVar[6];
	var_19_7[7] = myVar[7];
	var_19_8[8] = myVar[8];
	var_19_9[9] = myVar[9];
	var_19_10[10] = myVar[10];
	var_19_11[11] = myVar[11];
	var_19_12[12] = myVar[12];
	var_19_13[13] = myVar[13];
	var_19_14[14] = myVar[14];
	var_19_15[15] = myVar[15];
	var_19_16[16] = myVar[16];
	var_19_17[17] = myVar[17];
	var_19_18[18] = myVar[18];
	var_19_19[19] = myVar[19];
	
}

__global__ void kernel_20(float * var_20_0, float * var_20_1, float * var_20_2, float * var_20_3, float * var_20_4, float * var_20_5, float * var_20_6, float * var_20_7, float * var_20_8, float * var_20_9, float * var_20_10, float * var_20_11, float * var_20_12, float * var_20_13, float * var_20_14, float * var_20_15, float * var_20_16, float * var_20_17, float * var_20_18, float * var_20_19) {
	__shared__ float myVar[1024];
	myVar[3] = 8.920762 * myVar[threadIdx.x];
	myVar[7] = 37.353696 * myVar[threadIdx.x];
	myVar[3] = 32.099913 * myVar[threadIdx.x];
	myVar[8] = 24.096614 * myVar[threadIdx.x];
	myVar[9] = 14.495044 * myVar[threadIdx.x];
	myVar[6] = 35.974989 * myVar[threadIdx.x];
	myVar[7] = 21.908123 * myVar[threadIdx.x];
	myVar[0] = 41.261727 * myVar[threadIdx.x];
	myVar[4] = 21.221434 * myVar[threadIdx.x];
	myVar[5] = 0.046441 * myVar[threadIdx.x];
	var_20_0[0] = myVar[0];
	var_20_1[1] = myVar[1];
	var_20_2[2] = myVar[2];
	var_20_3[3] = myVar[3];
	var_20_4[4] = myVar[4];
	var_20_5[5] = myVar[5];
	var_20_6[6] = myVar[6];
	var_20_7[7] = myVar[7];
	var_20_8[8] = myVar[8];
	var_20_9[9] = myVar[9];
	var_20_10[10] = myVar[10];
	var_20_11[11] = myVar[11];
	var_20_12[12] = myVar[12];
	var_20_13[13] = myVar[13];
	var_20_14[14] = myVar[14];
	var_20_15[15] = myVar[15];
	var_20_16[16] = myVar[16];
	var_20_17[17] = myVar[17];
	var_20_18[18] = myVar[18];
	var_20_19[19] = myVar[19];
	
}

__global__ void kernel_21(float * var_21_0, float * var_21_1, float * var_21_2, float * var_21_3, float * var_21_4, float * var_21_5, float * var_21_6, float * var_21_7, float * var_21_8, float * var_21_9, float * var_21_10, float * var_21_11, float * var_21_12, float * var_21_13, float * var_21_14, float * var_21_15, float * var_21_16, float * var_21_17, float * var_21_18, float * var_21_19) {
	__shared__ float myVar[1024];
	myVar[8] = 22.112567 * myVar[threadIdx.x];
	myVar[3] = 27.653067 * myVar[threadIdx.x];
	myVar[6] = 16.924127 * myVar[threadIdx.x];
	myVar[1] = 33.412665 * myVar[threadIdx.x];
	myVar[7] = 2.221997 * myVar[threadIdx.x];
	myVar[7] = 45.256766 * myVar[threadIdx.x];
	myVar[6] = 37.572069 * myVar[threadIdx.x];
	myVar[6] = 25.978406 * myVar[threadIdx.x];
	myVar[5] = 29.277489 * myVar[threadIdx.x];
	myVar[0] = 46.767500 * myVar[threadIdx.x];
	var_21_0[0] = myVar[0];
	var_21_1[1] = myVar[1];
	var_21_2[2] = myVar[2];
	var_21_3[3] = myVar[3];
	var_21_4[4] = myVar[4];
	var_21_5[5] = myVar[5];
	var_21_6[6] = myVar[6];
	var_21_7[7] = myVar[7];
	var_21_8[8] = myVar[8];
	var_21_9[9] = myVar[9];
	var_21_10[10] = myVar[10];
	var_21_11[11] = myVar[11];
	var_21_12[12] = myVar[12];
	var_21_13[13] = myVar[13];
	var_21_14[14] = myVar[14];
	var_21_15[15] = myVar[15];
	var_21_16[16] = myVar[16];
	var_21_17[17] = myVar[17];
	var_21_18[18] = myVar[18];
	var_21_19[19] = myVar[19];
	
}

__global__ void kernel_22(float * var_22_0, float * var_22_1, float * var_22_2, float * var_22_3, float * var_22_4, float * var_22_5, float * var_22_6, float * var_22_7, float * var_22_8, float * var_22_9, float * var_22_10, float * var_22_11, float * var_22_12, float * var_22_13, float * var_22_14, float * var_22_15, float * var_22_16, float * var_22_17, float * var_22_18, float * var_22_19) {
	__shared__ float myVar[1024];
	myVar[4] = 31.285070 * myVar[threadIdx.x];
	myVar[7] = 0.863628 * myVar[threadIdx.x];
	myVar[8] = 25.720817 * myVar[threadIdx.x];
	myVar[0] = 12.004407 * myVar[threadIdx.x];
	myVar[0] = 45.033315 * myVar[threadIdx.x];
	myVar[8] = 40.415036 * myVar[threadIdx.x];
	myVar[4] = 25.509417 * myVar[threadIdx.x];
	myVar[6] = 34.767809 * myVar[threadIdx.x];
	myVar[0] = 12.637889 * myVar[threadIdx.x];
	myVar[6] = 10.133237 * myVar[threadIdx.x];
	var_22_0[0] = myVar[0];
	var_22_1[1] = myVar[1];
	var_22_2[2] = myVar[2];
	var_22_3[3] = myVar[3];
	var_22_4[4] = myVar[4];
	var_22_5[5] = myVar[5];
	var_22_6[6] = myVar[6];
	var_22_7[7] = myVar[7];
	var_22_8[8] = myVar[8];
	var_22_9[9] = myVar[9];
	var_22_10[10] = myVar[10];
	var_22_11[11] = myVar[11];
	var_22_12[12] = myVar[12];
	var_22_13[13] = myVar[13];
	var_22_14[14] = myVar[14];
	var_22_15[15] = myVar[15];
	var_22_16[16] = myVar[16];
	var_22_17[17] = myVar[17];
	var_22_18[18] = myVar[18];
	var_22_19[19] = myVar[19];
	
}

__global__ void kernel_23(float * var_23_0, float * var_23_1, float * var_23_2, float * var_23_3, float * var_23_4, float * var_23_5, float * var_23_6, float * var_23_7, float * var_23_8, float * var_23_9, float * var_23_10, float * var_23_11, float * var_23_12, float * var_23_13, float * var_23_14, float * var_23_15, float * var_23_16, float * var_23_17, float * var_23_18, float * var_23_19) {
	__shared__ float myVar[1024];
	myVar[5] = 21.524582 * myVar[threadIdx.x];
	myVar[6] = 29.706542 * myVar[threadIdx.x];
	myVar[6] = 31.447616 * myVar[threadIdx.x];
	myVar[7] = 32.046018 * myVar[threadIdx.x];
	myVar[1] = 31.068229 * myVar[threadIdx.x];
	myVar[7] = 5.766474 * myVar[threadIdx.x];
	myVar[7] = 24.443588 * myVar[threadIdx.x];
	myVar[3] = 40.997058 * myVar[threadIdx.x];
	myVar[0] = 22.817930 * myVar[threadIdx.x];
	myVar[1] = 8.522397 * myVar[threadIdx.x];
	var_23_0[0] = myVar[0];
	var_23_1[1] = myVar[1];
	var_23_2[2] = myVar[2];
	var_23_3[3] = myVar[3];
	var_23_4[4] = myVar[4];
	var_23_5[5] = myVar[5];
	var_23_6[6] = myVar[6];
	var_23_7[7] = myVar[7];
	var_23_8[8] = myVar[8];
	var_23_9[9] = myVar[9];
	var_23_10[10] = myVar[10];
	var_23_11[11] = myVar[11];
	var_23_12[12] = myVar[12];
	var_23_13[13] = myVar[13];
	var_23_14[14] = myVar[14];
	var_23_15[15] = myVar[15];
	var_23_16[16] = myVar[16];
	var_23_17[17] = myVar[17];
	var_23_18[18] = myVar[18];
	var_23_19[19] = myVar[19];
	
}

__global__ void kernel_24(float * var_24_0, float * var_24_1, float * var_24_2, float * var_24_3, float * var_24_4, float * var_24_5, float * var_24_6, float * var_24_7, float * var_24_8, float * var_24_9, float * var_24_10, float * var_24_11, float * var_24_12, float * var_24_13, float * var_24_14, float * var_24_15, float * var_24_16, float * var_24_17, float * var_24_18, float * var_24_19) {
	__shared__ float myVar[1024];
	myVar[2] = 17.014051 * myVar[threadIdx.x];
	myVar[0] = 3.181015 * myVar[threadIdx.x];
	myVar[3] = 29.333601 * myVar[threadIdx.x];
	myVar[1] = 39.439184 * myVar[threadIdx.x];
	myVar[7] = 25.280476 * myVar[threadIdx.x];
	myVar[3] = 19.974617 * myVar[threadIdx.x];
	myVar[0] = 38.258641 * myVar[threadIdx.x];
	myVar[2] = 10.066041 * myVar[threadIdx.x];
	myVar[3] = 31.378324 * myVar[threadIdx.x];
	myVar[3] = 16.652854 * myVar[threadIdx.x];
	var_24_0[0] = myVar[0];
	var_24_1[1] = myVar[1];
	var_24_2[2] = myVar[2];
	var_24_3[3] = myVar[3];
	var_24_4[4] = myVar[4];
	var_24_5[5] = myVar[5];
	var_24_6[6] = myVar[6];
	var_24_7[7] = myVar[7];
	var_24_8[8] = myVar[8];
	var_24_9[9] = myVar[9];
	var_24_10[10] = myVar[10];
	var_24_11[11] = myVar[11];
	var_24_12[12] = myVar[12];
	var_24_13[13] = myVar[13];
	var_24_14[14] = myVar[14];
	var_24_15[15] = myVar[15];
	var_24_16[16] = myVar[16];
	var_24_17[17] = myVar[17];
	var_24_18[18] = myVar[18];
	var_24_19[19] = myVar[19];
	
}

__global__ void kernel_25(float * var_25_0, float * var_25_1, float * var_25_2, float * var_25_3, float * var_25_4, float * var_25_5, float * var_25_6, float * var_25_7, float * var_25_8, float * var_25_9, float * var_25_10, float * var_25_11, float * var_25_12, float * var_25_13, float * var_25_14, float * var_25_15, float * var_25_16, float * var_25_17, float * var_25_18, float * var_25_19) {
	__shared__ float myVar[1024];
	myVar[8] = 7.517236 * myVar[threadIdx.x];
	myVar[6] = 28.264222 * myVar[threadIdx.x];
	myVar[8] = 4.411520 * myVar[threadIdx.x];
	myVar[1] = 7.778072 * myVar[threadIdx.x];
	myVar[5] = 32.653238 * myVar[threadIdx.x];
	myVar[1] = 15.025972 * myVar[threadIdx.x];
	myVar[3] = 27.817416 * myVar[threadIdx.x];
	myVar[2] = 30.711862 * myVar[threadIdx.x];
	myVar[9] = 34.396261 * myVar[threadIdx.x];
	myVar[7] = 44.214172 * myVar[threadIdx.x];
	var_25_0[0] = myVar[0];
	var_25_1[1] = myVar[1];
	var_25_2[2] = myVar[2];
	var_25_3[3] = myVar[3];
	var_25_4[4] = myVar[4];
	var_25_5[5] = myVar[5];
	var_25_6[6] = myVar[6];
	var_25_7[7] = myVar[7];
	var_25_8[8] = myVar[8];
	var_25_9[9] = myVar[9];
	var_25_10[10] = myVar[10];
	var_25_11[11] = myVar[11];
	var_25_12[12] = myVar[12];
	var_25_13[13] = myVar[13];
	var_25_14[14] = myVar[14];
	var_25_15[15] = myVar[15];
	var_25_16[16] = myVar[16];
	var_25_17[17] = myVar[17];
	var_25_18[18] = myVar[18];
	var_25_19[19] = myVar[19];
	
}

__global__ void kernel_26(float * var_26_0, float * var_26_1, float * var_26_2, float * var_26_3, float * var_26_4, float * var_26_5, float * var_26_6, float * var_26_7, float * var_26_8, float * var_26_9, float * var_26_10, float * var_26_11, float * var_26_12, float * var_26_13, float * var_26_14, float * var_26_15, float * var_26_16, float * var_26_17, float * var_26_18, float * var_26_19) {
	__shared__ float myVar[1024];
	myVar[4] = 48.641031 * myVar[threadIdx.x];
	myVar[9] = 43.952846 * myVar[threadIdx.x];
	myVar[8] = 26.538523 * myVar[threadIdx.x];
	myVar[2] = 8.435853 * myVar[threadIdx.x];
	myVar[1] = 36.593866 * myVar[threadIdx.x];
	myVar[5] = 17.569830 * myVar[threadIdx.x];
	myVar[6] = 42.516282 * myVar[threadIdx.x];
	myVar[4] = 47.853700 * myVar[threadIdx.x];
	myVar[5] = 41.775186 * myVar[threadIdx.x];
	myVar[4] = 29.159559 * myVar[threadIdx.x];
	var_26_0[0] = myVar[0];
	var_26_1[1] = myVar[1];
	var_26_2[2] = myVar[2];
	var_26_3[3] = myVar[3];
	var_26_4[4] = myVar[4];
	var_26_5[5] = myVar[5];
	var_26_6[6] = myVar[6];
	var_26_7[7] = myVar[7];
	var_26_8[8] = myVar[8];
	var_26_9[9] = myVar[9];
	var_26_10[10] = myVar[10];
	var_26_11[11] = myVar[11];
	var_26_12[12] = myVar[12];
	var_26_13[13] = myVar[13];
	var_26_14[14] = myVar[14];
	var_26_15[15] = myVar[15];
	var_26_16[16] = myVar[16];
	var_26_17[17] = myVar[17];
	var_26_18[18] = myVar[18];
	var_26_19[19] = myVar[19];
	
}

__global__ void kernel_27(float * var_27_0, float * var_27_1, float * var_27_2, float * var_27_3, float * var_27_4, float * var_27_5, float * var_27_6, float * var_27_7, float * var_27_8, float * var_27_9, float * var_27_10, float * var_27_11, float * var_27_12, float * var_27_13, float * var_27_14, float * var_27_15, float * var_27_16, float * var_27_17, float * var_27_18, float * var_27_19) {
	__shared__ float myVar[1024];
	myVar[9] = 44.593467 * myVar[threadIdx.x];
	myVar[5] = 4.774136 * myVar[threadIdx.x];
	myVar[5] = 0.608903 * myVar[threadIdx.x];
	myVar[5] = 27.921276 * myVar[threadIdx.x];
	myVar[6] = 23.472995 * myVar[threadIdx.x];
	myVar[4] = 45.549882 * myVar[threadIdx.x];
	myVar[8] = 30.662462 * myVar[threadIdx.x];
	myVar[3] = 13.045959 * myVar[threadIdx.x];
	myVar[2] = 45.326496 * myVar[threadIdx.x];
	myVar[2] = 30.163615 * myVar[threadIdx.x];
	var_27_0[0] = myVar[0];
	var_27_1[1] = myVar[1];
	var_27_2[2] = myVar[2];
	var_27_3[3] = myVar[3];
	var_27_4[4] = myVar[4];
	var_27_5[5] = myVar[5];
	var_27_6[6] = myVar[6];
	var_27_7[7] = myVar[7];
	var_27_8[8] = myVar[8];
	var_27_9[9] = myVar[9];
	var_27_10[10] = myVar[10];
	var_27_11[11] = myVar[11];
	var_27_12[12] = myVar[12];
	var_27_13[13] = myVar[13];
	var_27_14[14] = myVar[14];
	var_27_15[15] = myVar[15];
	var_27_16[16] = myVar[16];
	var_27_17[17] = myVar[17];
	var_27_18[18] = myVar[18];
	var_27_19[19] = myVar[19];
	
}

__global__ void kernel_28(float * var_28_0, float * var_28_1, float * var_28_2, float * var_28_3, float * var_28_4, float * var_28_5, float * var_28_6, float * var_28_7, float * var_28_8, float * var_28_9, float * var_28_10, float * var_28_11, float * var_28_12, float * var_28_13, float * var_28_14, float * var_28_15, float * var_28_16, float * var_28_17, float * var_28_18, float * var_28_19) {
	__shared__ float myVar[1024];
	myVar[4] = 21.657435 * myVar[threadIdx.x];
	myVar[9] = 29.884629 * myVar[threadIdx.x];
	myVar[8] = 21.673294 * myVar[threadIdx.x];
	myVar[6] = 3.731673 * myVar[threadIdx.x];
	myVar[2] = 22.881731 * myVar[threadIdx.x];
	myVar[9] = 19.223812 * myVar[threadIdx.x];
	myVar[9] = 36.169111 * myVar[threadIdx.x];
	myVar[8] = 34.115668 * myVar[threadIdx.x];
	myVar[3] = 4.904825 * myVar[threadIdx.x];
	myVar[2] = 0.995835 * myVar[threadIdx.x];
	var_28_0[0] = myVar[0];
	var_28_1[1] = myVar[1];
	var_28_2[2] = myVar[2];
	var_28_3[3] = myVar[3];
	var_28_4[4] = myVar[4];
	var_28_5[5] = myVar[5];
	var_28_6[6] = myVar[6];
	var_28_7[7] = myVar[7];
	var_28_8[8] = myVar[8];
	var_28_9[9] = myVar[9];
	var_28_10[10] = myVar[10];
	var_28_11[11] = myVar[11];
	var_28_12[12] = myVar[12];
	var_28_13[13] = myVar[13];
	var_28_14[14] = myVar[14];
	var_28_15[15] = myVar[15];
	var_28_16[16] = myVar[16];
	var_28_17[17] = myVar[17];
	var_28_18[18] = myVar[18];
	var_28_19[19] = myVar[19];
	
}

__global__ void kernel_29(float * var_29_0, float * var_29_1, float * var_29_2, float * var_29_3, float * var_29_4, float * var_29_5, float * var_29_6, float * var_29_7, float * var_29_8, float * var_29_9, float * var_29_10, float * var_29_11, float * var_29_12, float * var_29_13, float * var_29_14, float * var_29_15, float * var_29_16, float * var_29_17, float * var_29_18, float * var_29_19) {
	__shared__ float myVar[1024];
	myVar[7] = 25.151348 * myVar[threadIdx.x];
	myVar[3] = 5.073383 * myVar[threadIdx.x];
	myVar[2] = 49.057405 * myVar[threadIdx.x];
	myVar[5] = 10.375361 * myVar[threadIdx.x];
	myVar[5] = 34.815777 * myVar[threadIdx.x];
	myVar[6] = 19.558450 * myVar[threadIdx.x];
	myVar[4] = 23.350865 * myVar[threadIdx.x];
	myVar[9] = 5.091217 * myVar[threadIdx.x];
	myVar[0] = 22.062347 * myVar[threadIdx.x];
	myVar[7] = 39.179353 * myVar[threadIdx.x];
	var_29_0[0] = myVar[0];
	var_29_1[1] = myVar[1];
	var_29_2[2] = myVar[2];
	var_29_3[3] = myVar[3];
	var_29_4[4] = myVar[4];
	var_29_5[5] = myVar[5];
	var_29_6[6] = myVar[6];
	var_29_7[7] = myVar[7];
	var_29_8[8] = myVar[8];
	var_29_9[9] = myVar[9];
	var_29_10[10] = myVar[10];
	var_29_11[11] = myVar[11];
	var_29_12[12] = myVar[12];
	var_29_13[13] = myVar[13];
	var_29_14[14] = myVar[14];
	var_29_15[15] = myVar[15];
	var_29_16[16] = myVar[16];
	var_29_17[17] = myVar[17];
	var_29_18[18] = myVar[18];
	var_29_19[19] = myVar[19];
	
}

__global__ void kernel_30(float * var_30_0, float * var_30_1, float * var_30_2, float * var_30_3, float * var_30_4, float * var_30_5, float * var_30_6, float * var_30_7, float * var_30_8, float * var_30_9, float * var_30_10, float * var_30_11, float * var_30_12, float * var_30_13, float * var_30_14, float * var_30_15, float * var_30_16, float * var_30_17, float * var_30_18, float * var_30_19) {
	__shared__ float myVar[1024];
	myVar[5] = 36.046011 * myVar[threadIdx.x];
	myVar[3] = 14.885873 * myVar[threadIdx.x];
	myVar[7] = 38.248691 * myVar[threadIdx.x];
	myVar[1] = 43.093572 * myVar[threadIdx.x];
	myVar[5] = 19.913632 * myVar[threadIdx.x];
	myVar[6] = 6.243649 * myVar[threadIdx.x];
	myVar[7] = 23.822577 * myVar[threadIdx.x];
	myVar[7] = 32.748343 * myVar[threadIdx.x];
	myVar[1] = 0.240827 * myVar[threadIdx.x];
	myVar[2] = 12.217483 * myVar[threadIdx.x];
	var_30_0[0] = myVar[0];
	var_30_1[1] = myVar[1];
	var_30_2[2] = myVar[2];
	var_30_3[3] = myVar[3];
	var_30_4[4] = myVar[4];
	var_30_5[5] = myVar[5];
	var_30_6[6] = myVar[6];
	var_30_7[7] = myVar[7];
	var_30_8[8] = myVar[8];
	var_30_9[9] = myVar[9];
	var_30_10[10] = myVar[10];
	var_30_11[11] = myVar[11];
	var_30_12[12] = myVar[12];
	var_30_13[13] = myVar[13];
	var_30_14[14] = myVar[14];
	var_30_15[15] = myVar[15];
	var_30_16[16] = myVar[16];
	var_30_17[17] = myVar[17];
	var_30_18[18] = myVar[18];
	var_30_19[19] = myVar[19];
	
}

__global__ void kernel_31(float * var_31_0, float * var_31_1, float * var_31_2, float * var_31_3, float * var_31_4, float * var_31_5, float * var_31_6, float * var_31_7, float * var_31_8, float * var_31_9, float * var_31_10, float * var_31_11, float * var_31_12, float * var_31_13, float * var_31_14, float * var_31_15, float * var_31_16, float * var_31_17, float * var_31_18, float * var_31_19) {
	__shared__ float myVar[1024];
	myVar[1] = 5.874735 * myVar[threadIdx.x];
	myVar[3] = 37.347456 * myVar[threadIdx.x];
	myVar[2] = 37.747604 * myVar[threadIdx.x];
	myVar[4] = 31.348833 * myVar[threadIdx.x];
	myVar[2] = 34.267739 * myVar[threadIdx.x];
	myVar[7] = 28.779658 * myVar[threadIdx.x];
	myVar[7] = 38.259609 * myVar[threadIdx.x];
	myVar[4] = 40.641591 * myVar[threadIdx.x];
	myVar[8] = 25.630688 * myVar[threadIdx.x];
	myVar[2] = 8.221034 * myVar[threadIdx.x];
	var_31_0[0] = myVar[0];
	var_31_1[1] = myVar[1];
	var_31_2[2] = myVar[2];
	var_31_3[3] = myVar[3];
	var_31_4[4] = myVar[4];
	var_31_5[5] = myVar[5];
	var_31_6[6] = myVar[6];
	var_31_7[7] = myVar[7];
	var_31_8[8] = myVar[8];
	var_31_9[9] = myVar[9];
	var_31_10[10] = myVar[10];
	var_31_11[11] = myVar[11];
	var_31_12[12] = myVar[12];
	var_31_13[13] = myVar[13];
	var_31_14[14] = myVar[14];
	var_31_15[15] = myVar[15];
	var_31_16[16] = myVar[16];
	var_31_17[17] = myVar[17];
	var_31_18[18] = myVar[18];
	var_31_19[19] = myVar[19];
	
}

__global__ void kernel_32(float * var_32_0, float * var_32_1, float * var_32_2, float * var_32_3, float * var_32_4, float * var_32_5, float * var_32_6, float * var_32_7, float * var_32_8, float * var_32_9, float * var_32_10, float * var_32_11, float * var_32_12, float * var_32_13, float * var_32_14, float * var_32_15, float * var_32_16, float * var_32_17, float * var_32_18, float * var_32_19) {
	__shared__ float myVar[1024];
	myVar[3] = 37.831150 * myVar[threadIdx.x];
	myVar[5] = 28.793004 * myVar[threadIdx.x];
	myVar[4] = 19.871804 * myVar[threadIdx.x];
	myVar[2] = 39.019470 * myVar[threadIdx.x];
	myVar[0] = 26.222847 * myVar[threadIdx.x];
	myVar[2] = 12.296851 * myVar[threadIdx.x];
	myVar[2] = 32.653467 * myVar[threadIdx.x];
	myVar[6] = 21.242219 * myVar[threadIdx.x];
	myVar[3] = 47.590289 * myVar[threadIdx.x];
	myVar[3] = 39.070860 * myVar[threadIdx.x];
	var_32_0[0] = myVar[0];
	var_32_1[1] = myVar[1];
	var_32_2[2] = myVar[2];
	var_32_3[3] = myVar[3];
	var_32_4[4] = myVar[4];
	var_32_5[5] = myVar[5];
	var_32_6[6] = myVar[6];
	var_32_7[7] = myVar[7];
	var_32_8[8] = myVar[8];
	var_32_9[9] = myVar[9];
	var_32_10[10] = myVar[10];
	var_32_11[11] = myVar[11];
	var_32_12[12] = myVar[12];
	var_32_13[13] = myVar[13];
	var_32_14[14] = myVar[14];
	var_32_15[15] = myVar[15];
	var_32_16[16] = myVar[16];
	var_32_17[17] = myVar[17];
	var_32_18[18] = myVar[18];
	var_32_19[19] = myVar[19];
	
}

__global__ void kernel_33(float * var_33_0, float * var_33_1, float * var_33_2, float * var_33_3, float * var_33_4, float * var_33_5, float * var_33_6, float * var_33_7, float * var_33_8, float * var_33_9, float * var_33_10, float * var_33_11, float * var_33_12, float * var_33_13, float * var_33_14, float * var_33_15, float * var_33_16, float * var_33_17, float * var_33_18, float * var_33_19) {
	__shared__ float myVar[1024];
	myVar[5] = 38.030299 * myVar[threadIdx.x];
	myVar[5] = 44.122182 * myVar[threadIdx.x];
	myVar[3] = 34.982739 * myVar[threadIdx.x];
	myVar[7] = 31.233982 * myVar[threadIdx.x];
	myVar[9] = 41.721021 * myVar[threadIdx.x];
	myVar[4] = 46.965980 * myVar[threadIdx.x];
	myVar[4] = 35.483172 * myVar[threadIdx.x];
	myVar[0] = 20.314613 * myVar[threadIdx.x];
	myVar[0] = 23.829995 * myVar[threadIdx.x];
	myVar[0] = 34.938981 * myVar[threadIdx.x];
	var_33_0[0] = myVar[0];
	var_33_1[1] = myVar[1];
	var_33_2[2] = myVar[2];
	var_33_3[3] = myVar[3];
	var_33_4[4] = myVar[4];
	var_33_5[5] = myVar[5];
	var_33_6[6] = myVar[6];
	var_33_7[7] = myVar[7];
	var_33_8[8] = myVar[8];
	var_33_9[9] = myVar[9];
	var_33_10[10] = myVar[10];
	var_33_11[11] = myVar[11];
	var_33_12[12] = myVar[12];
	var_33_13[13] = myVar[13];
	var_33_14[14] = myVar[14];
	var_33_15[15] = myVar[15];
	var_33_16[16] = myVar[16];
	var_33_17[17] = myVar[17];
	var_33_18[18] = myVar[18];
	var_33_19[19] = myVar[19];
	
}

__global__ void kernel_34(float * var_34_0, float * var_34_1, float * var_34_2, float * var_34_3, float * var_34_4, float * var_34_5, float * var_34_6, float * var_34_7, float * var_34_8, float * var_34_9, float * var_34_10, float * var_34_11, float * var_34_12, float * var_34_13, float * var_34_14, float * var_34_15, float * var_34_16, float * var_34_17, float * var_34_18, float * var_34_19) {
	__shared__ float myVar[1024];
	myVar[1] = 4.848555 * myVar[threadIdx.x];
	myVar[9] = 45.594005 * myVar[threadIdx.x];
	myVar[4] = 5.275042 * myVar[threadIdx.x];
	myVar[6] = 47.467082 * myVar[threadIdx.x];
	myVar[0] = 43.599883 * myVar[threadIdx.x];
	myVar[0] = 22.543226 * myVar[threadIdx.x];
	myVar[3] = 10.500562 * myVar[threadIdx.x];
	myVar[9] = 7.076448 * myVar[threadIdx.x];
	myVar[6] = 37.155668 * myVar[threadIdx.x];
	myVar[6] = 39.351688 * myVar[threadIdx.x];
	var_34_0[0] = myVar[0];
	var_34_1[1] = myVar[1];
	var_34_2[2] = myVar[2];
	var_34_3[3] = myVar[3];
	var_34_4[4] = myVar[4];
	var_34_5[5] = myVar[5];
	var_34_6[6] = myVar[6];
	var_34_7[7] = myVar[7];
	var_34_8[8] = myVar[8];
	var_34_9[9] = myVar[9];
	var_34_10[10] = myVar[10];
	var_34_11[11] = myVar[11];
	var_34_12[12] = myVar[12];
	var_34_13[13] = myVar[13];
	var_34_14[14] = myVar[14];
	var_34_15[15] = myVar[15];
	var_34_16[16] = myVar[16];
	var_34_17[17] = myVar[17];
	var_34_18[18] = myVar[18];
	var_34_19[19] = myVar[19];
	
}

__global__ void kernel_35(float * var_35_0, float * var_35_1, float * var_35_2, float * var_35_3, float * var_35_4, float * var_35_5, float * var_35_6, float * var_35_7, float * var_35_8, float * var_35_9, float * var_35_10, float * var_35_11, float * var_35_12, float * var_35_13, float * var_35_14, float * var_35_15, float * var_35_16, float * var_35_17, float * var_35_18, float * var_35_19) {
	__shared__ float myVar[1024];
	myVar[2] = 20.230487 * myVar[threadIdx.x];
	myVar[5] = 22.713707 * myVar[threadIdx.x];
	myVar[7] = 35.011226 * myVar[threadIdx.x];
	myVar[8] = 4.429307 * myVar[threadIdx.x];
	myVar[4] = 32.732229 * myVar[threadIdx.x];
	myVar[7] = 43.417915 * myVar[threadIdx.x];
	myVar[7] = 8.217305 * myVar[threadIdx.x];
	myVar[1] = 39.014612 * myVar[threadIdx.x];
	myVar[0] = 14.273283 * myVar[threadIdx.x];
	myVar[9] = 11.301961 * myVar[threadIdx.x];
	var_35_0[0] = myVar[0];
	var_35_1[1] = myVar[1];
	var_35_2[2] = myVar[2];
	var_35_3[3] = myVar[3];
	var_35_4[4] = myVar[4];
	var_35_5[5] = myVar[5];
	var_35_6[6] = myVar[6];
	var_35_7[7] = myVar[7];
	var_35_8[8] = myVar[8];
	var_35_9[9] = myVar[9];
	var_35_10[10] = myVar[10];
	var_35_11[11] = myVar[11];
	var_35_12[12] = myVar[12];
	var_35_13[13] = myVar[13];
	var_35_14[14] = myVar[14];
	var_35_15[15] = myVar[15];
	var_35_16[16] = myVar[16];
	var_35_17[17] = myVar[17];
	var_35_18[18] = myVar[18];
	var_35_19[19] = myVar[19];
	
}

__global__ void kernel_36(float * var_36_0, float * var_36_1, float * var_36_2, float * var_36_3, float * var_36_4, float * var_36_5, float * var_36_6, float * var_36_7, float * var_36_8, float * var_36_9, float * var_36_10, float * var_36_11, float * var_36_12, float * var_36_13, float * var_36_14, float * var_36_15, float * var_36_16, float * var_36_17, float * var_36_18, float * var_36_19) {
	__shared__ float myVar[1024];
	myVar[5] = 45.179389 * myVar[threadIdx.x];
	myVar[2] = 20.411022 * myVar[threadIdx.x];
	myVar[3] = 10.367868 * myVar[threadIdx.x];
	myVar[5] = 41.563758 * myVar[threadIdx.x];
	myVar[0] = 8.981381 * myVar[threadIdx.x];
	myVar[1] = 14.762939 * myVar[threadIdx.x];
	myVar[9] = 31.785718 * myVar[threadIdx.x];
	myVar[9] = 12.968729 * myVar[threadIdx.x];
	myVar[2] = 0.609264 * myVar[threadIdx.x];
	myVar[6] = 44.362367 * myVar[threadIdx.x];
	var_36_0[0] = myVar[0];
	var_36_1[1] = myVar[1];
	var_36_2[2] = myVar[2];
	var_36_3[3] = myVar[3];
	var_36_4[4] = myVar[4];
	var_36_5[5] = myVar[5];
	var_36_6[6] = myVar[6];
	var_36_7[7] = myVar[7];
	var_36_8[8] = myVar[8];
	var_36_9[9] = myVar[9];
	var_36_10[10] = myVar[10];
	var_36_11[11] = myVar[11];
	var_36_12[12] = myVar[12];
	var_36_13[13] = myVar[13];
	var_36_14[14] = myVar[14];
	var_36_15[15] = myVar[15];
	var_36_16[16] = myVar[16];
	var_36_17[17] = myVar[17];
	var_36_18[18] = myVar[18];
	var_36_19[19] = myVar[19];
	
}

__global__ void kernel_37(float * var_37_0, float * var_37_1, float * var_37_2, float * var_37_3, float * var_37_4, float * var_37_5, float * var_37_6, float * var_37_7, float * var_37_8, float * var_37_9, float * var_37_10, float * var_37_11, float * var_37_12, float * var_37_13, float * var_37_14, float * var_37_15, float * var_37_16, float * var_37_17, float * var_37_18, float * var_37_19) {
	__shared__ float myVar[1024];
	myVar[3] = 35.093701 * myVar[threadIdx.x];
	myVar[2] = 42.819633 * myVar[threadIdx.x];
	myVar[7] = 14.836877 * myVar[threadIdx.x];
	myVar[8] = 25.653325 * myVar[threadIdx.x];
	myVar[4] = 26.962135 * myVar[threadIdx.x];
	myVar[5] = 39.779576 * myVar[threadIdx.x];
	myVar[1] = 39.045629 * myVar[threadIdx.x];
	myVar[1] = 12.076513 * myVar[threadIdx.x];
	myVar[3] = 49.070441 * myVar[threadIdx.x];
	myVar[5] = 44.290816 * myVar[threadIdx.x];
	var_37_0[0] = myVar[0];
	var_37_1[1] = myVar[1];
	var_37_2[2] = myVar[2];
	var_37_3[3] = myVar[3];
	var_37_4[4] = myVar[4];
	var_37_5[5] = myVar[5];
	var_37_6[6] = myVar[6];
	var_37_7[7] = myVar[7];
	var_37_8[8] = myVar[8];
	var_37_9[9] = myVar[9];
	var_37_10[10] = myVar[10];
	var_37_11[11] = myVar[11];
	var_37_12[12] = myVar[12];
	var_37_13[13] = myVar[13];
	var_37_14[14] = myVar[14];
	var_37_15[15] = myVar[15];
	var_37_16[16] = myVar[16];
	var_37_17[17] = myVar[17];
	var_37_18[18] = myVar[18];
	var_37_19[19] = myVar[19];
	
}

__global__ void kernel_38(float * var_38_0, float * var_38_1, float * var_38_2, float * var_38_3, float * var_38_4, float * var_38_5, float * var_38_6, float * var_38_7, float * var_38_8, float * var_38_9, float * var_38_10, float * var_38_11, float * var_38_12, float * var_38_13, float * var_38_14, float * var_38_15, float * var_38_16, float * var_38_17, float * var_38_18, float * var_38_19) {
	__shared__ float myVar[1024];
	myVar[8] = 16.950942 * myVar[threadIdx.x];
	myVar[6] = 16.723500 * myVar[threadIdx.x];
	myVar[1] = 0.501504 * myVar[threadIdx.x];
	myVar[8] = 15.969000 * myVar[threadIdx.x];
	myVar[9] = 21.267720 * myVar[threadIdx.x];
	myVar[4] = 49.429729 * myVar[threadIdx.x];
	myVar[1] = 19.745463 * myVar[threadIdx.x];
	myVar[0] = 6.291363 * myVar[threadIdx.x];
	myVar[5] = 1.115146 * myVar[threadIdx.x];
	myVar[0] = 30.363812 * myVar[threadIdx.x];
	var_38_0[0] = myVar[0];
	var_38_1[1] = myVar[1];
	var_38_2[2] = myVar[2];
	var_38_3[3] = myVar[3];
	var_38_4[4] = myVar[4];
	var_38_5[5] = myVar[5];
	var_38_6[6] = myVar[6];
	var_38_7[7] = myVar[7];
	var_38_8[8] = myVar[8];
	var_38_9[9] = myVar[9];
	var_38_10[10] = myVar[10];
	var_38_11[11] = myVar[11];
	var_38_12[12] = myVar[12];
	var_38_13[13] = myVar[13];
	var_38_14[14] = myVar[14];
	var_38_15[15] = myVar[15];
	var_38_16[16] = myVar[16];
	var_38_17[17] = myVar[17];
	var_38_18[18] = myVar[18];
	var_38_19[19] = myVar[19];
	
}

__global__ void kernel_39(float * var_39_0, float * var_39_1, float * var_39_2, float * var_39_3, float * var_39_4, float * var_39_5, float * var_39_6, float * var_39_7, float * var_39_8, float * var_39_9, float * var_39_10, float * var_39_11, float * var_39_12, float * var_39_13, float * var_39_14, float * var_39_15, float * var_39_16, float * var_39_17, float * var_39_18, float * var_39_19) {
	__shared__ float myVar[1024];
	myVar[4] = 48.490515 * myVar[threadIdx.x];
	myVar[9] = 12.608231 * myVar[threadIdx.x];
	myVar[2] = 35.157848 * myVar[threadIdx.x];
	myVar[0] = 1.188302 * myVar[threadIdx.x];
	myVar[5] = 45.364279 * myVar[threadIdx.x];
	myVar[3] = 34.068832 * myVar[threadIdx.x];
	myVar[2] = 21.128017 * myVar[threadIdx.x];
	myVar[5] = 14.244563 * myVar[threadIdx.x];
	myVar[3] = 3.151376 * myVar[threadIdx.x];
	myVar[6] = 2.305583 * myVar[threadIdx.x];
	var_39_0[0] = myVar[0];
	var_39_1[1] = myVar[1];
	var_39_2[2] = myVar[2];
	var_39_3[3] = myVar[3];
	var_39_4[4] = myVar[4];
	var_39_5[5] = myVar[5];
	var_39_6[6] = myVar[6];
	var_39_7[7] = myVar[7];
	var_39_8[8] = myVar[8];
	var_39_9[9] = myVar[9];
	var_39_10[10] = myVar[10];
	var_39_11[11] = myVar[11];
	var_39_12[12] = myVar[12];
	var_39_13[13] = myVar[13];
	var_39_14[14] = myVar[14];
	var_39_15[15] = myVar[15];
	var_39_16[16] = myVar[16];
	var_39_17[17] = myVar[17];
	var_39_18[18] = myVar[18];
	var_39_19[19] = myVar[19];
	
}

__global__ void kernel_40(float * var_40_0, float * var_40_1, float * var_40_2, float * var_40_3, float * var_40_4, float * var_40_5, float * var_40_6, float * var_40_7, float * var_40_8, float * var_40_9, float * var_40_10, float * var_40_11, float * var_40_12, float * var_40_13, float * var_40_14, float * var_40_15, float * var_40_16, float * var_40_17, float * var_40_18, float * var_40_19) {
	__shared__ float myVar[1024];
	myVar[6] = 38.743765 * myVar[threadIdx.x];
	myVar[0] = 3.914831 * myVar[threadIdx.x];
	myVar[0] = 45.867457 * myVar[threadIdx.x];
	myVar[8] = 15.480244 * myVar[threadIdx.x];
	myVar[6] = 7.260518 * myVar[threadIdx.x];
	myVar[3] = 7.096993 * myVar[threadIdx.x];
	myVar[5] = 5.880275 * myVar[threadIdx.x];
	myVar[3] = 22.825388 * myVar[threadIdx.x];
	myVar[2] = 16.359728 * myVar[threadIdx.x];
	myVar[3] = 15.126936 * myVar[threadIdx.x];
	var_40_0[0] = myVar[0];
	var_40_1[1] = myVar[1];
	var_40_2[2] = myVar[2];
	var_40_3[3] = myVar[3];
	var_40_4[4] = myVar[4];
	var_40_5[5] = myVar[5];
	var_40_6[6] = myVar[6];
	var_40_7[7] = myVar[7];
	var_40_8[8] = myVar[8];
	var_40_9[9] = myVar[9];
	var_40_10[10] = myVar[10];
	var_40_11[11] = myVar[11];
	var_40_12[12] = myVar[12];
	var_40_13[13] = myVar[13];
	var_40_14[14] = myVar[14];
	var_40_15[15] = myVar[15];
	var_40_16[16] = myVar[16];
	var_40_17[17] = myVar[17];
	var_40_18[18] = myVar[18];
	var_40_19[19] = myVar[19];
	
}

__global__ void kernel_41(float * var_41_0, float * var_41_1, float * var_41_2, float * var_41_3, float * var_41_4, float * var_41_5, float * var_41_6, float * var_41_7, float * var_41_8, float * var_41_9, float * var_41_10, float * var_41_11, float * var_41_12, float * var_41_13, float * var_41_14, float * var_41_15, float * var_41_16, float * var_41_17, float * var_41_18, float * var_41_19) {
	__shared__ float myVar[1024];
	myVar[3] = 20.568324 * myVar[threadIdx.x];
	myVar[1] = 21.021676 * myVar[threadIdx.x];
	myVar[1] = 40.620338 * myVar[threadIdx.x];
	myVar[9] = 36.677789 * myVar[threadIdx.x];
	myVar[7] = 27.680906 * myVar[threadIdx.x];
	myVar[2] = 44.656317 * myVar[threadIdx.x];
	myVar[9] = 22.003000 * myVar[threadIdx.x];
	myVar[4] = 16.280718 * myVar[threadIdx.x];
	myVar[9] = 16.620876 * myVar[threadIdx.x];
	myVar[8] = 0.349996 * myVar[threadIdx.x];
	var_41_0[0] = myVar[0];
	var_41_1[1] = myVar[1];
	var_41_2[2] = myVar[2];
	var_41_3[3] = myVar[3];
	var_41_4[4] = myVar[4];
	var_41_5[5] = myVar[5];
	var_41_6[6] = myVar[6];
	var_41_7[7] = myVar[7];
	var_41_8[8] = myVar[8];
	var_41_9[9] = myVar[9];
	var_41_10[10] = myVar[10];
	var_41_11[11] = myVar[11];
	var_41_12[12] = myVar[12];
	var_41_13[13] = myVar[13];
	var_41_14[14] = myVar[14];
	var_41_15[15] = myVar[15];
	var_41_16[16] = myVar[16];
	var_41_17[17] = myVar[17];
	var_41_18[18] = myVar[18];
	var_41_19[19] = myVar[19];
	
}

__global__ void kernel_42(float * var_42_0, float * var_42_1, float * var_42_2, float * var_42_3, float * var_42_4, float * var_42_5, float * var_42_6, float * var_42_7, float * var_42_8, float * var_42_9, float * var_42_10, float * var_42_11, float * var_42_12, float * var_42_13, float * var_42_14, float * var_42_15, float * var_42_16, float * var_42_17, float * var_42_18, float * var_42_19) {
	__shared__ float myVar[1024];
	myVar[6] = 42.111438 * myVar[threadIdx.x];
	myVar[8] = 3.365143 * myVar[threadIdx.x];
	myVar[2] = 33.891024 * myVar[threadIdx.x];
	myVar[4] = 41.766899 * myVar[threadIdx.x];
	myVar[2] = 7.180711 * myVar[threadIdx.x];
	myVar[9] = 40.939054 * myVar[threadIdx.x];
	myVar[2] = 27.350946 * myVar[threadIdx.x];
	myVar[2] = 39.810918 * myVar[threadIdx.x];
	myVar[0] = 37.526751 * myVar[threadIdx.x];
	myVar[4] = 26.537539 * myVar[threadIdx.x];
	var_42_0[0] = myVar[0];
	var_42_1[1] = myVar[1];
	var_42_2[2] = myVar[2];
	var_42_3[3] = myVar[3];
	var_42_4[4] = myVar[4];
	var_42_5[5] = myVar[5];
	var_42_6[6] = myVar[6];
	var_42_7[7] = myVar[7];
	var_42_8[8] = myVar[8];
	var_42_9[9] = myVar[9];
	var_42_10[10] = myVar[10];
	var_42_11[11] = myVar[11];
	var_42_12[12] = myVar[12];
	var_42_13[13] = myVar[13];
	var_42_14[14] = myVar[14];
	var_42_15[15] = myVar[15];
	var_42_16[16] = myVar[16];
	var_42_17[17] = myVar[17];
	var_42_18[18] = myVar[18];
	var_42_19[19] = myVar[19];
	
}

__global__ void kernel_43(float * var_43_0, float * var_43_1, float * var_43_2, float * var_43_3, float * var_43_4, float * var_43_5, float * var_43_6, float * var_43_7, float * var_43_8, float * var_43_9, float * var_43_10, float * var_43_11, float * var_43_12, float * var_43_13, float * var_43_14, float * var_43_15, float * var_43_16, float * var_43_17, float * var_43_18, float * var_43_19) {
	__shared__ float myVar[1024];
	myVar[2] = 36.382057 * myVar[threadIdx.x];
	myVar[7] = 23.767897 * myVar[threadIdx.x];
	myVar[0] = 49.106766 * myVar[threadIdx.x];
	myVar[8] = 28.979968 * myVar[threadIdx.x];
	myVar[1] = 21.884905 * myVar[threadIdx.x];
	myVar[2] = 24.992381 * myVar[threadIdx.x];
	myVar[6] = 47.545519 * myVar[threadIdx.x];
	myVar[3] = 0.659092 * myVar[threadIdx.x];
	myVar[3] = 41.286335 * myVar[threadIdx.x];
	myVar[8] = 22.498220 * myVar[threadIdx.x];
	var_43_0[0] = myVar[0];
	var_43_1[1] = myVar[1];
	var_43_2[2] = myVar[2];
	var_43_3[3] = myVar[3];
	var_43_4[4] = myVar[4];
	var_43_5[5] = myVar[5];
	var_43_6[6] = myVar[6];
	var_43_7[7] = myVar[7];
	var_43_8[8] = myVar[8];
	var_43_9[9] = myVar[9];
	var_43_10[10] = myVar[10];
	var_43_11[11] = myVar[11];
	var_43_12[12] = myVar[12];
	var_43_13[13] = myVar[13];
	var_43_14[14] = myVar[14];
	var_43_15[15] = myVar[15];
	var_43_16[16] = myVar[16];
	var_43_17[17] = myVar[17];
	var_43_18[18] = myVar[18];
	var_43_19[19] = myVar[19];
	
}

__global__ void kernel_44(float * var_44_0, float * var_44_1, float * var_44_2, float * var_44_3, float * var_44_4, float * var_44_5, float * var_44_6, float * var_44_7, float * var_44_8, float * var_44_9, float * var_44_10, float * var_44_11, float * var_44_12, float * var_44_13, float * var_44_14, float * var_44_15, float * var_44_16, float * var_44_17, float * var_44_18, float * var_44_19) {
	__shared__ float myVar[1024];
	myVar[1] = 22.696796 * myVar[threadIdx.x];
	myVar[4] = 7.228871 * myVar[threadIdx.x];
	myVar[6] = 32.201492 * myVar[threadIdx.x];
	myVar[9] = 23.192476 * myVar[threadIdx.x];
	myVar[7] = 29.049439 * myVar[threadIdx.x];
	myVar[4] = 41.982081 * myVar[threadIdx.x];
	myVar[5] = 32.711130 * myVar[threadIdx.x];
	myVar[9] = 43.816981 * myVar[threadIdx.x];
	myVar[4] = 34.609433 * myVar[threadIdx.x];
	myVar[9] = 28.135683 * myVar[threadIdx.x];
	var_44_0[0] = myVar[0];
	var_44_1[1] = myVar[1];
	var_44_2[2] = myVar[2];
	var_44_3[3] = myVar[3];
	var_44_4[4] = myVar[4];
	var_44_5[5] = myVar[5];
	var_44_6[6] = myVar[6];
	var_44_7[7] = myVar[7];
	var_44_8[8] = myVar[8];
	var_44_9[9] = myVar[9];
	var_44_10[10] = myVar[10];
	var_44_11[11] = myVar[11];
	var_44_12[12] = myVar[12];
	var_44_13[13] = myVar[13];
	var_44_14[14] = myVar[14];
	var_44_15[15] = myVar[15];
	var_44_16[16] = myVar[16];
	var_44_17[17] = myVar[17];
	var_44_18[18] = myVar[18];
	var_44_19[19] = myVar[19];
	
}

__global__ void kernel_45(float * var_45_0, float * var_45_1, float * var_45_2, float * var_45_3, float * var_45_4, float * var_45_5, float * var_45_6, float * var_45_7, float * var_45_8, float * var_45_9, float * var_45_10, float * var_45_11, float * var_45_12, float * var_45_13, float * var_45_14, float * var_45_15, float * var_45_16, float * var_45_17, float * var_45_18, float * var_45_19) {
	__shared__ float myVar[1024];
	myVar[1] = 5.264738 * myVar[threadIdx.x];
	myVar[8] = 26.466403 * myVar[threadIdx.x];
	myVar[2] = 6.018990 * myVar[threadIdx.x];
	myVar[8] = 8.928184 * myVar[threadIdx.x];
	myVar[7] = 16.665097 * myVar[threadIdx.x];
	myVar[5] = 42.861306 * myVar[threadIdx.x];
	myVar[9] = 31.633341 * myVar[threadIdx.x];
	myVar[2] = 27.710102 * myVar[threadIdx.x];
	myVar[8] = 39.455016 * myVar[threadIdx.x];
	myVar[1] = 19.087801 * myVar[threadIdx.x];
	var_45_0[0] = myVar[0];
	var_45_1[1] = myVar[1];
	var_45_2[2] = myVar[2];
	var_45_3[3] = myVar[3];
	var_45_4[4] = myVar[4];
	var_45_5[5] = myVar[5];
	var_45_6[6] = myVar[6];
	var_45_7[7] = myVar[7];
	var_45_8[8] = myVar[8];
	var_45_9[9] = myVar[9];
	var_45_10[10] = myVar[10];
	var_45_11[11] = myVar[11];
	var_45_12[12] = myVar[12];
	var_45_13[13] = myVar[13];
	var_45_14[14] = myVar[14];
	var_45_15[15] = myVar[15];
	var_45_16[16] = myVar[16];
	var_45_17[17] = myVar[17];
	var_45_18[18] = myVar[18];
	var_45_19[19] = myVar[19];
	
}

__global__ void kernel_46(float * var_46_0, float * var_46_1, float * var_46_2, float * var_46_3, float * var_46_4, float * var_46_5, float * var_46_6, float * var_46_7, float * var_46_8, float * var_46_9, float * var_46_10, float * var_46_11, float * var_46_12, float * var_46_13, float * var_46_14, float * var_46_15, float * var_46_16, float * var_46_17, float * var_46_18, float * var_46_19) {
	__shared__ float myVar[1024];
	myVar[6] = 29.320805 * myVar[threadIdx.x];
	myVar[3] = 13.928934 * myVar[threadIdx.x];
	myVar[0] = 40.096413 * myVar[threadIdx.x];
	myVar[2] = 44.771034 * myVar[threadIdx.x];
	myVar[9] = 19.382410 * myVar[threadIdx.x];
	myVar[2] = 5.395163 * myVar[threadIdx.x];
	myVar[1] = 18.947185 * myVar[threadIdx.x];
	myVar[5] = 9.527862 * myVar[threadIdx.x];
	myVar[2] = 42.415643 * myVar[threadIdx.x];
	myVar[7] = 38.430310 * myVar[threadIdx.x];
	var_46_0[0] = myVar[0];
	var_46_1[1] = myVar[1];
	var_46_2[2] = myVar[2];
	var_46_3[3] = myVar[3];
	var_46_4[4] = myVar[4];
	var_46_5[5] = myVar[5];
	var_46_6[6] = myVar[6];
	var_46_7[7] = myVar[7];
	var_46_8[8] = myVar[8];
	var_46_9[9] = myVar[9];
	var_46_10[10] = myVar[10];
	var_46_11[11] = myVar[11];
	var_46_12[12] = myVar[12];
	var_46_13[13] = myVar[13];
	var_46_14[14] = myVar[14];
	var_46_15[15] = myVar[15];
	var_46_16[16] = myVar[16];
	var_46_17[17] = myVar[17];
	var_46_18[18] = myVar[18];
	var_46_19[19] = myVar[19];
	
}

__global__ void kernel_47(float * var_47_0, float * var_47_1, float * var_47_2, float * var_47_3, float * var_47_4, float * var_47_5, float * var_47_6, float * var_47_7, float * var_47_8, float * var_47_9, float * var_47_10, float * var_47_11, float * var_47_12, float * var_47_13, float * var_47_14, float * var_47_15, float * var_47_16, float * var_47_17, float * var_47_18, float * var_47_19) {
	__shared__ float myVar[1024];
	myVar[4] = 5.982213 * myVar[threadIdx.x];
	myVar[3] = 37.258891 * myVar[threadIdx.x];
	myVar[9] = 25.699217 * myVar[threadIdx.x];
	myVar[3] = 40.365154 * myVar[threadIdx.x];
	myVar[5] = 14.262341 * myVar[threadIdx.x];
	myVar[4] = 40.991364 * myVar[threadIdx.x];
	myVar[3] = 8.103409 * myVar[threadIdx.x];
	myVar[1] = 40.528052 * myVar[threadIdx.x];
	myVar[5] = 21.207895 * myVar[threadIdx.x];
	myVar[5] = 5.529244 * myVar[threadIdx.x];
	var_47_0[0] = myVar[0];
	var_47_1[1] = myVar[1];
	var_47_2[2] = myVar[2];
	var_47_3[3] = myVar[3];
	var_47_4[4] = myVar[4];
	var_47_5[5] = myVar[5];
	var_47_6[6] = myVar[6];
	var_47_7[7] = myVar[7];
	var_47_8[8] = myVar[8];
	var_47_9[9] = myVar[9];
	var_47_10[10] = myVar[10];
	var_47_11[11] = myVar[11];
	var_47_12[12] = myVar[12];
	var_47_13[13] = myVar[13];
	var_47_14[14] = myVar[14];
	var_47_15[15] = myVar[15];
	var_47_16[16] = myVar[16];
	var_47_17[17] = myVar[17];
	var_47_18[18] = myVar[18];
	var_47_19[19] = myVar[19];
	
}

__global__ void kernel_48(float * var_48_0, float * var_48_1, float * var_48_2, float * var_48_3, float * var_48_4, float * var_48_5, float * var_48_6, float * var_48_7, float * var_48_8, float * var_48_9, float * var_48_10, float * var_48_11, float * var_48_12, float * var_48_13, float * var_48_14, float * var_48_15, float * var_48_16, float * var_48_17, float * var_48_18, float * var_48_19) {
	__shared__ float myVar[1024];
	myVar[6] = 19.114624 * myVar[threadIdx.x];
	myVar[2] = 12.542276 * myVar[threadIdx.x];
	myVar[3] = 1.622599 * myVar[threadIdx.x];
	myVar[3] = 39.816754 * myVar[threadIdx.x];
	myVar[6] = 23.014758 * myVar[threadIdx.x];
	myVar[1] = 45.052447 * myVar[threadIdx.x];
	myVar[9] = 7.343654 * myVar[threadIdx.x];
	myVar[8] = 19.386823 * myVar[threadIdx.x];
	myVar[9] = 32.406551 * myVar[threadIdx.x];
	myVar[5] = 31.336219 * myVar[threadIdx.x];
	var_48_0[0] = myVar[0];
	var_48_1[1] = myVar[1];
	var_48_2[2] = myVar[2];
	var_48_3[3] = myVar[3];
	var_48_4[4] = myVar[4];
	var_48_5[5] = myVar[5];
	var_48_6[6] = myVar[6];
	var_48_7[7] = myVar[7];
	var_48_8[8] = myVar[8];
	var_48_9[9] = myVar[9];
	var_48_10[10] = myVar[10];
	var_48_11[11] = myVar[11];
	var_48_12[12] = myVar[12];
	var_48_13[13] = myVar[13];
	var_48_14[14] = myVar[14];
	var_48_15[15] = myVar[15];
	var_48_16[16] = myVar[16];
	var_48_17[17] = myVar[17];
	var_48_18[18] = myVar[18];
	var_48_19[19] = myVar[19];
	
}

__global__ void kernel_49(float * var_49_0, float * var_49_1, float * var_49_2, float * var_49_3, float * var_49_4, float * var_49_5, float * var_49_6, float * var_49_7, float * var_49_8, float * var_49_9, float * var_49_10, float * var_49_11, float * var_49_12, float * var_49_13, float * var_49_14, float * var_49_15, float * var_49_16, float * var_49_17, float * var_49_18, float * var_49_19) {
	__shared__ float myVar[1024];
	myVar[3] = 9.497627 * myVar[threadIdx.x];
	myVar[3] = 38.111004 * myVar[threadIdx.x];
	myVar[1] = 20.507056 * myVar[threadIdx.x];
	myVar[2] = 42.317821 * myVar[threadIdx.x];
	myVar[1] = 48.548658 * myVar[threadIdx.x];
	myVar[4] = 16.948474 * myVar[threadIdx.x];
	myVar[5] = 9.081909 * myVar[threadIdx.x];
	myVar[5] = 16.551171 * myVar[threadIdx.x];
	myVar[7] = 42.262653 * myVar[threadIdx.x];
	myVar[3] = 40.563470 * myVar[threadIdx.x];
	var_49_0[0] = myVar[0];
	var_49_1[1] = myVar[1];
	var_49_2[2] = myVar[2];
	var_49_3[3] = myVar[3];
	var_49_4[4] = myVar[4];
	var_49_5[5] = myVar[5];
	var_49_6[6] = myVar[6];
	var_49_7[7] = myVar[7];
	var_49_8[8] = myVar[8];
	var_49_9[9] = myVar[9];
	var_49_10[10] = myVar[10];
	var_49_11[11] = myVar[11];
	var_49_12[12] = myVar[12];
	var_49_13[13] = myVar[13];
	var_49_14[14] = myVar[14];
	var_49_15[15] = myVar[15];
	var_49_16[16] = myVar[16];
	var_49_17[17] = myVar[17];
	var_49_18[18] = myVar[18];
	var_49_19[19] = myVar[19];
	
}

__global__ void kernel_50(float * var_50_0, float * var_50_1, float * var_50_2, float * var_50_3, float * var_50_4, float * var_50_5, float * var_50_6, float * var_50_7, float * var_50_8, float * var_50_9, float * var_50_10, float * var_50_11, float * var_50_12, float * var_50_13, float * var_50_14, float * var_50_15, float * var_50_16, float * var_50_17, float * var_50_18, float * var_50_19) {
	__shared__ float myVar[1024];
	myVar[6] = 28.012459 * myVar[threadIdx.x];
	myVar[6] = 25.991875 * myVar[threadIdx.x];
	myVar[3] = 13.713698 * myVar[threadIdx.x];
	myVar[4] = 13.887336 * myVar[threadIdx.x];
	myVar[9] = 41.417406 * myVar[threadIdx.x];
	myVar[3] = 5.307227 * myVar[threadIdx.x];
	myVar[0] = 0.364875 * myVar[threadIdx.x];
	myVar[6] = 18.413486 * myVar[threadIdx.x];
	myVar[4] = 9.831031 * myVar[threadIdx.x];
	myVar[2] = 41.817142 * myVar[threadIdx.x];
	var_50_0[0] = myVar[0];
	var_50_1[1] = myVar[1];
	var_50_2[2] = myVar[2];
	var_50_3[3] = myVar[3];
	var_50_4[4] = myVar[4];
	var_50_5[5] = myVar[5];
	var_50_6[6] = myVar[6];
	var_50_7[7] = myVar[7];
	var_50_8[8] = myVar[8];
	var_50_9[9] = myVar[9];
	var_50_10[10] = myVar[10];
	var_50_11[11] = myVar[11];
	var_50_12[12] = myVar[12];
	var_50_13[13] = myVar[13];
	var_50_14[14] = myVar[14];
	var_50_15[15] = myVar[15];
	var_50_16[16] = myVar[16];
	var_50_17[17] = myVar[17];
	var_50_18[18] = myVar[18];
	var_50_19[19] = myVar[19];
	
}

__global__ void kernel_51(float * var_51_0, float * var_51_1, float * var_51_2, float * var_51_3, float * var_51_4, float * var_51_5, float * var_51_6, float * var_51_7, float * var_51_8, float * var_51_9, float * var_51_10, float * var_51_11, float * var_51_12, float * var_51_13, float * var_51_14, float * var_51_15, float * var_51_16, float * var_51_17, float * var_51_18, float * var_51_19) {
	__shared__ float myVar[1024];
	myVar[4] = 22.490023 * myVar[threadIdx.x];
	myVar[3] = 5.457193 * myVar[threadIdx.x];
	myVar[1] = 21.186693 * myVar[threadIdx.x];
	myVar[7] = 17.845219 * myVar[threadIdx.x];
	myVar[9] = 18.021808 * myVar[threadIdx.x];
	myVar[5] = 27.833284 * myVar[threadIdx.x];
	myVar[0] = 10.576937 * myVar[threadIdx.x];
	myVar[2] = 1.478394 * myVar[threadIdx.x];
	myVar[1] = 33.419164 * myVar[threadIdx.x];
	myVar[0] = 28.783970 * myVar[threadIdx.x];
	var_51_0[0] = myVar[0];
	var_51_1[1] = myVar[1];
	var_51_2[2] = myVar[2];
	var_51_3[3] = myVar[3];
	var_51_4[4] = myVar[4];
	var_51_5[5] = myVar[5];
	var_51_6[6] = myVar[6];
	var_51_7[7] = myVar[7];
	var_51_8[8] = myVar[8];
	var_51_9[9] = myVar[9];
	var_51_10[10] = myVar[10];
	var_51_11[11] = myVar[11];
	var_51_12[12] = myVar[12];
	var_51_13[13] = myVar[13];
	var_51_14[14] = myVar[14];
	var_51_15[15] = myVar[15];
	var_51_16[16] = myVar[16];
	var_51_17[17] = myVar[17];
	var_51_18[18] = myVar[18];
	var_51_19[19] = myVar[19];
	
}

__global__ void kernel_52(float * var_52_0, float * var_52_1, float * var_52_2, float * var_52_3, float * var_52_4, float * var_52_5, float * var_52_6, float * var_52_7, float * var_52_8, float * var_52_9, float * var_52_10, float * var_52_11, float * var_52_12, float * var_52_13, float * var_52_14, float * var_52_15, float * var_52_16, float * var_52_17, float * var_52_18, float * var_52_19) {
	__shared__ float myVar[1024];
	myVar[2] = 7.235504 * myVar[threadIdx.x];
	myVar[9] = 32.571032 * myVar[threadIdx.x];
	myVar[7] = 34.190828 * myVar[threadIdx.x];
	myVar[1] = 26.437176 * myVar[threadIdx.x];
	myVar[9] = 1.664443 * myVar[threadIdx.x];
	myVar[7] = 28.140066 * myVar[threadIdx.x];
	myVar[2] = 4.530189 * myVar[threadIdx.x];
	myVar[5] = 37.755731 * myVar[threadIdx.x];
	myVar[8] = 37.563695 * myVar[threadIdx.x];
	myVar[6] = 23.034860 * myVar[threadIdx.x];
	var_52_0[0] = myVar[0];
	var_52_1[1] = myVar[1];
	var_52_2[2] = myVar[2];
	var_52_3[3] = myVar[3];
	var_52_4[4] = myVar[4];
	var_52_5[5] = myVar[5];
	var_52_6[6] = myVar[6];
	var_52_7[7] = myVar[7];
	var_52_8[8] = myVar[8];
	var_52_9[9] = myVar[9];
	var_52_10[10] = myVar[10];
	var_52_11[11] = myVar[11];
	var_52_12[12] = myVar[12];
	var_52_13[13] = myVar[13];
	var_52_14[14] = myVar[14];
	var_52_15[15] = myVar[15];
	var_52_16[16] = myVar[16];
	var_52_17[17] = myVar[17];
	var_52_18[18] = myVar[18];
	var_52_19[19] = myVar[19];
	
}

__global__ void kernel_53(float * var_53_0, float * var_53_1, float * var_53_2, float * var_53_3, float * var_53_4, float * var_53_5, float * var_53_6, float * var_53_7, float * var_53_8, float * var_53_9, float * var_53_10, float * var_53_11, float * var_53_12, float * var_53_13, float * var_53_14, float * var_53_15, float * var_53_16, float * var_53_17, float * var_53_18, float * var_53_19) {
	__shared__ float myVar[1024];
	myVar[0] = 35.640143 * myVar[threadIdx.x];
	myVar[9] = 49.773588 * myVar[threadIdx.x];
	myVar[9] = 11.500338 * myVar[threadIdx.x];
	myVar[5] = 19.826921 * myVar[threadIdx.x];
	myVar[3] = 0.677061 * myVar[threadIdx.x];
	myVar[7] = 44.915955 * myVar[threadIdx.x];
	myVar[3] = 30.448106 * myVar[threadIdx.x];
	myVar[7] = 4.748795 * myVar[threadIdx.x];
	myVar[6] = 8.851894 * myVar[threadIdx.x];
	myVar[7] = 5.527967 * myVar[threadIdx.x];
	var_53_0[0] = myVar[0];
	var_53_1[1] = myVar[1];
	var_53_2[2] = myVar[2];
	var_53_3[3] = myVar[3];
	var_53_4[4] = myVar[4];
	var_53_5[5] = myVar[5];
	var_53_6[6] = myVar[6];
	var_53_7[7] = myVar[7];
	var_53_8[8] = myVar[8];
	var_53_9[9] = myVar[9];
	var_53_10[10] = myVar[10];
	var_53_11[11] = myVar[11];
	var_53_12[12] = myVar[12];
	var_53_13[13] = myVar[13];
	var_53_14[14] = myVar[14];
	var_53_15[15] = myVar[15];
	var_53_16[16] = myVar[16];
	var_53_17[17] = myVar[17];
	var_53_18[18] = myVar[18];
	var_53_19[19] = myVar[19];
	
}

__global__ void kernel_54(float * var_54_0, float * var_54_1, float * var_54_2, float * var_54_3, float * var_54_4, float * var_54_5, float * var_54_6, float * var_54_7, float * var_54_8, float * var_54_9, float * var_54_10, float * var_54_11, float * var_54_12, float * var_54_13, float * var_54_14, float * var_54_15, float * var_54_16, float * var_54_17, float * var_54_18, float * var_54_19) {
	__shared__ float myVar[1024];
	myVar[2] = 19.902653 * myVar[threadIdx.x];
	myVar[1] = 47.270573 * myVar[threadIdx.x];
	myVar[9] = 8.612166 * myVar[threadIdx.x];
	myVar[0] = 13.656687 * myVar[threadIdx.x];
	myVar[3] = 34.048088 * myVar[threadIdx.x];
	myVar[2] = 20.745672 * myVar[threadIdx.x];
	myVar[4] = 4.449649 * myVar[threadIdx.x];
	myVar[1] = 34.471773 * myVar[threadIdx.x];
	myVar[1] = 29.653638 * myVar[threadIdx.x];
	myVar[2] = 3.453407 * myVar[threadIdx.x];
	var_54_0[0] = myVar[0];
	var_54_1[1] = myVar[1];
	var_54_2[2] = myVar[2];
	var_54_3[3] = myVar[3];
	var_54_4[4] = myVar[4];
	var_54_5[5] = myVar[5];
	var_54_6[6] = myVar[6];
	var_54_7[7] = myVar[7];
	var_54_8[8] = myVar[8];
	var_54_9[9] = myVar[9];
	var_54_10[10] = myVar[10];
	var_54_11[11] = myVar[11];
	var_54_12[12] = myVar[12];
	var_54_13[13] = myVar[13];
	var_54_14[14] = myVar[14];
	var_54_15[15] = myVar[15];
	var_54_16[16] = myVar[16];
	var_54_17[17] = myVar[17];
	var_54_18[18] = myVar[18];
	var_54_19[19] = myVar[19];
	
}

__global__ void kernel_55(float * var_55_0, float * var_55_1, float * var_55_2, float * var_55_3, float * var_55_4, float * var_55_5, float * var_55_6, float * var_55_7, float * var_55_8, float * var_55_9, float * var_55_10, float * var_55_11, float * var_55_12, float * var_55_13, float * var_55_14, float * var_55_15, float * var_55_16, float * var_55_17, float * var_55_18, float * var_55_19) {
	__shared__ float myVar[1024];
	myVar[6] = 37.034285 * myVar[threadIdx.x];
	myVar[3] = 12.716498 * myVar[threadIdx.x];
	myVar[6] = 8.490564 * myVar[threadIdx.x];
	myVar[0] = 43.948535 * myVar[threadIdx.x];
	myVar[2] = 0.320381 * myVar[threadIdx.x];
	myVar[3] = 5.822891 * myVar[threadIdx.x];
	myVar[3] = 21.174635 * myVar[threadIdx.x];
	myVar[2] = 13.974690 * myVar[threadIdx.x];
	myVar[1] = 39.717704 * myVar[threadIdx.x];
	myVar[4] = 46.594515 * myVar[threadIdx.x];
	var_55_0[0] = myVar[0];
	var_55_1[1] = myVar[1];
	var_55_2[2] = myVar[2];
	var_55_3[3] = myVar[3];
	var_55_4[4] = myVar[4];
	var_55_5[5] = myVar[5];
	var_55_6[6] = myVar[6];
	var_55_7[7] = myVar[7];
	var_55_8[8] = myVar[8];
	var_55_9[9] = myVar[9];
	var_55_10[10] = myVar[10];
	var_55_11[11] = myVar[11];
	var_55_12[12] = myVar[12];
	var_55_13[13] = myVar[13];
	var_55_14[14] = myVar[14];
	var_55_15[15] = myVar[15];
	var_55_16[16] = myVar[16];
	var_55_17[17] = myVar[17];
	var_55_18[18] = myVar[18];
	var_55_19[19] = myVar[19];
	
}

__global__ void kernel_56(float * var_56_0, float * var_56_1, float * var_56_2, float * var_56_3, float * var_56_4, float * var_56_5, float * var_56_6, float * var_56_7, float * var_56_8, float * var_56_9, float * var_56_10, float * var_56_11, float * var_56_12, float * var_56_13, float * var_56_14, float * var_56_15, float * var_56_16, float * var_56_17, float * var_56_18, float * var_56_19) {
	__shared__ float myVar[1024];
	myVar[4] = 14.354722 * myVar[threadIdx.x];
	myVar[8] = 19.445647 * myVar[threadIdx.x];
	myVar[6] = 6.975940 * myVar[threadIdx.x];
	myVar[1] = 1.841344 * myVar[threadIdx.x];
	myVar[0] = 25.022314 * myVar[threadIdx.x];
	myVar[8] = 10.115960 * myVar[threadIdx.x];
	myVar[8] = 26.694995 * myVar[threadIdx.x];
	myVar[2] = 27.048145 * myVar[threadIdx.x];
	myVar[9] = 40.191317 * myVar[threadIdx.x];
	myVar[4] = 48.592553 * myVar[threadIdx.x];
	var_56_0[0] = myVar[0];
	var_56_1[1] = myVar[1];
	var_56_2[2] = myVar[2];
	var_56_3[3] = myVar[3];
	var_56_4[4] = myVar[4];
	var_56_5[5] = myVar[5];
	var_56_6[6] = myVar[6];
	var_56_7[7] = myVar[7];
	var_56_8[8] = myVar[8];
	var_56_9[9] = myVar[9];
	var_56_10[10] = myVar[10];
	var_56_11[11] = myVar[11];
	var_56_12[12] = myVar[12];
	var_56_13[13] = myVar[13];
	var_56_14[14] = myVar[14];
	var_56_15[15] = myVar[15];
	var_56_16[16] = myVar[16];
	var_56_17[17] = myVar[17];
	var_56_18[18] = myVar[18];
	var_56_19[19] = myVar[19];
	
}

__global__ void kernel_57(float * var_57_0, float * var_57_1, float * var_57_2, float * var_57_3, float * var_57_4, float * var_57_5, float * var_57_6, float * var_57_7, float * var_57_8, float * var_57_9, float * var_57_10, float * var_57_11, float * var_57_12, float * var_57_13, float * var_57_14, float * var_57_15, float * var_57_16, float * var_57_17, float * var_57_18, float * var_57_19) {
	__shared__ float myVar[1024];
	myVar[8] = 1.034066 * myVar[threadIdx.x];
	myVar[8] = 17.991719 * myVar[threadIdx.x];
	myVar[7] = 33.456486 * myVar[threadIdx.x];
	myVar[9] = 22.782415 * myVar[threadIdx.x];
	myVar[7] = 43.160783 * myVar[threadIdx.x];
	myVar[3] = 47.987453 * myVar[threadIdx.x];
	myVar[8] = 19.269042 * myVar[threadIdx.x];
	myVar[1] = 38.047578 * myVar[threadIdx.x];
	myVar[3] = 23.153168 * myVar[threadIdx.x];
	myVar[3] = 5.201573 * myVar[threadIdx.x];
	var_57_0[0] = myVar[0];
	var_57_1[1] = myVar[1];
	var_57_2[2] = myVar[2];
	var_57_3[3] = myVar[3];
	var_57_4[4] = myVar[4];
	var_57_5[5] = myVar[5];
	var_57_6[6] = myVar[6];
	var_57_7[7] = myVar[7];
	var_57_8[8] = myVar[8];
	var_57_9[9] = myVar[9];
	var_57_10[10] = myVar[10];
	var_57_11[11] = myVar[11];
	var_57_12[12] = myVar[12];
	var_57_13[13] = myVar[13];
	var_57_14[14] = myVar[14];
	var_57_15[15] = myVar[15];
	var_57_16[16] = myVar[16];
	var_57_17[17] = myVar[17];
	var_57_18[18] = myVar[18];
	var_57_19[19] = myVar[19];
	
}

__global__ void kernel_58(float * var_58_0, float * var_58_1, float * var_58_2, float * var_58_3, float * var_58_4, float * var_58_5, float * var_58_6, float * var_58_7, float * var_58_8, float * var_58_9, float * var_58_10, float * var_58_11, float * var_58_12, float * var_58_13, float * var_58_14, float * var_58_15, float * var_58_16, float * var_58_17, float * var_58_18, float * var_58_19) {
	__shared__ float myVar[1024];
	myVar[1] = 41.541838 * myVar[threadIdx.x];
	myVar[9] = 37.748709 * myVar[threadIdx.x];
	myVar[4] = 16.368320 * myVar[threadIdx.x];
	myVar[8] = 31.205332 * myVar[threadIdx.x];
	myVar[0] = 37.657988 * myVar[threadIdx.x];
	myVar[2] = 22.547574 * myVar[threadIdx.x];
	myVar[9] = 1.961046 * myVar[threadIdx.x];
	myVar[0] = 34.180845 * myVar[threadIdx.x];
	myVar[1] = 3.071738 * myVar[threadIdx.x];
	myVar[0] = 15.505802 * myVar[threadIdx.x];
	var_58_0[0] = myVar[0];
	var_58_1[1] = myVar[1];
	var_58_2[2] = myVar[2];
	var_58_3[3] = myVar[3];
	var_58_4[4] = myVar[4];
	var_58_5[5] = myVar[5];
	var_58_6[6] = myVar[6];
	var_58_7[7] = myVar[7];
	var_58_8[8] = myVar[8];
	var_58_9[9] = myVar[9];
	var_58_10[10] = myVar[10];
	var_58_11[11] = myVar[11];
	var_58_12[12] = myVar[12];
	var_58_13[13] = myVar[13];
	var_58_14[14] = myVar[14];
	var_58_15[15] = myVar[15];
	var_58_16[16] = myVar[16];
	var_58_17[17] = myVar[17];
	var_58_18[18] = myVar[18];
	var_58_19[19] = myVar[19];
	
}

__global__ void kernel_59(float * var_59_0, float * var_59_1, float * var_59_2, float * var_59_3, float * var_59_4, float * var_59_5, float * var_59_6, float * var_59_7, float * var_59_8, float * var_59_9, float * var_59_10, float * var_59_11, float * var_59_12, float * var_59_13, float * var_59_14, float * var_59_15, float * var_59_16, float * var_59_17, float * var_59_18, float * var_59_19) {
	__shared__ float myVar[1024];
	myVar[0] = 36.742748 * myVar[threadIdx.x];
	myVar[1] = 25.209210 * myVar[threadIdx.x];
	myVar[4] = 42.259580 * myVar[threadIdx.x];
	myVar[3] = 16.190616 * myVar[threadIdx.x];
	myVar[3] = 1.853530 * myVar[threadIdx.x];
	myVar[0] = 14.906392 * myVar[threadIdx.x];
	myVar[3] = 11.083243 * myVar[threadIdx.x];
	myVar[8] = 49.082613 * myVar[threadIdx.x];
	myVar[9] = 30.891738 * myVar[threadIdx.x];
	myVar[3] = 26.238603 * myVar[threadIdx.x];
	var_59_0[0] = myVar[0];
	var_59_1[1] = myVar[1];
	var_59_2[2] = myVar[2];
	var_59_3[3] = myVar[3];
	var_59_4[4] = myVar[4];
	var_59_5[5] = myVar[5];
	var_59_6[6] = myVar[6];
	var_59_7[7] = myVar[7];
	var_59_8[8] = myVar[8];
	var_59_9[9] = myVar[9];
	var_59_10[10] = myVar[10];
	var_59_11[11] = myVar[11];
	var_59_12[12] = myVar[12];
	var_59_13[13] = myVar[13];
	var_59_14[14] = myVar[14];
	var_59_15[15] = myVar[15];
	var_59_16[16] = myVar[16];
	var_59_17[17] = myVar[17];
	var_59_18[18] = myVar[18];
	var_59_19[19] = myVar[19];
	
}

__global__ void kernel_60(float * var_60_0, float * var_60_1, float * var_60_2, float * var_60_3, float * var_60_4, float * var_60_5, float * var_60_6, float * var_60_7, float * var_60_8, float * var_60_9, float * var_60_10, float * var_60_11, float * var_60_12, float * var_60_13, float * var_60_14, float * var_60_15, float * var_60_16, float * var_60_17, float * var_60_18, float * var_60_19) {
	__shared__ float myVar[1024];
	myVar[2] = 22.409327 * myVar[threadIdx.x];
	myVar[1] = 19.211186 * myVar[threadIdx.x];
	myVar[5] = 49.698355 * myVar[threadIdx.x];
	myVar[2] = 14.036316 * myVar[threadIdx.x];
	myVar[3] = 45.807214 * myVar[threadIdx.x];
	myVar[5] = 9.665548 * myVar[threadIdx.x];
	myVar[5] = 37.161092 * myVar[threadIdx.x];
	myVar[0] = 16.517638 * myVar[threadIdx.x];
	myVar[1] = 41.304760 * myVar[threadIdx.x];
	myVar[9] = 45.870091 * myVar[threadIdx.x];
	var_60_0[0] = myVar[0];
	var_60_1[1] = myVar[1];
	var_60_2[2] = myVar[2];
	var_60_3[3] = myVar[3];
	var_60_4[4] = myVar[4];
	var_60_5[5] = myVar[5];
	var_60_6[6] = myVar[6];
	var_60_7[7] = myVar[7];
	var_60_8[8] = myVar[8];
	var_60_9[9] = myVar[9];
	var_60_10[10] = myVar[10];
	var_60_11[11] = myVar[11];
	var_60_12[12] = myVar[12];
	var_60_13[13] = myVar[13];
	var_60_14[14] = myVar[14];
	var_60_15[15] = myVar[15];
	var_60_16[16] = myVar[16];
	var_60_17[17] = myVar[17];
	var_60_18[18] = myVar[18];
	var_60_19[19] = myVar[19];
	
}

__global__ void kernel_61(float * var_61_0, float * var_61_1, float * var_61_2, float * var_61_3, float * var_61_4, float * var_61_5, float * var_61_6, float * var_61_7, float * var_61_8, float * var_61_9, float * var_61_10, float * var_61_11, float * var_61_12, float * var_61_13, float * var_61_14, float * var_61_15, float * var_61_16, float * var_61_17, float * var_61_18, float * var_61_19) {
	__shared__ float myVar[1024];
	myVar[7] = 32.471986 * myVar[threadIdx.x];
	myVar[5] = 19.454519 * myVar[threadIdx.x];
	myVar[8] = 22.159774 * myVar[threadIdx.x];
	myVar[0] = 10.080446 * myVar[threadIdx.x];
	myVar[3] = 18.016182 * myVar[threadIdx.x];
	myVar[3] = 39.915484 * myVar[threadIdx.x];
	myVar[7] = 6.411692 * myVar[threadIdx.x];
	myVar[3] = 19.649969 * myVar[threadIdx.x];
	myVar[7] = 29.673918 * myVar[threadIdx.x];
	myVar[8] = 37.867687 * myVar[threadIdx.x];
	var_61_0[0] = myVar[0];
	var_61_1[1] = myVar[1];
	var_61_2[2] = myVar[2];
	var_61_3[3] = myVar[3];
	var_61_4[4] = myVar[4];
	var_61_5[5] = myVar[5];
	var_61_6[6] = myVar[6];
	var_61_7[7] = myVar[7];
	var_61_8[8] = myVar[8];
	var_61_9[9] = myVar[9];
	var_61_10[10] = myVar[10];
	var_61_11[11] = myVar[11];
	var_61_12[12] = myVar[12];
	var_61_13[13] = myVar[13];
	var_61_14[14] = myVar[14];
	var_61_15[15] = myVar[15];
	var_61_16[16] = myVar[16];
	var_61_17[17] = myVar[17];
	var_61_18[18] = myVar[18];
	var_61_19[19] = myVar[19];
	
}

__global__ void kernel_62(float * var_62_0, float * var_62_1, float * var_62_2, float * var_62_3, float * var_62_4, float * var_62_5, float * var_62_6, float * var_62_7, float * var_62_8, float * var_62_9, float * var_62_10, float * var_62_11, float * var_62_12, float * var_62_13, float * var_62_14, float * var_62_15, float * var_62_16, float * var_62_17, float * var_62_18, float * var_62_19) {
	__shared__ float myVar[1024];
	myVar[2] = 4.829668 * myVar[threadIdx.x];
	myVar[9] = 43.684248 * myVar[threadIdx.x];
	myVar[0] = 28.267032 * myVar[threadIdx.x];
	myVar[8] = 0.731295 * myVar[threadIdx.x];
	myVar[9] = 28.262118 * myVar[threadIdx.x];
	myVar[1] = 43.415830 * myVar[threadIdx.x];
	myVar[1] = 45.989398 * myVar[threadIdx.x];
	myVar[7] = 4.517703 * myVar[threadIdx.x];
	myVar[5] = 12.983002 * myVar[threadIdx.x];
	myVar[8] = 5.414934 * myVar[threadIdx.x];
	var_62_0[0] = myVar[0];
	var_62_1[1] = myVar[1];
	var_62_2[2] = myVar[2];
	var_62_3[3] = myVar[3];
	var_62_4[4] = myVar[4];
	var_62_5[5] = myVar[5];
	var_62_6[6] = myVar[6];
	var_62_7[7] = myVar[7];
	var_62_8[8] = myVar[8];
	var_62_9[9] = myVar[9];
	var_62_10[10] = myVar[10];
	var_62_11[11] = myVar[11];
	var_62_12[12] = myVar[12];
	var_62_13[13] = myVar[13];
	var_62_14[14] = myVar[14];
	var_62_15[15] = myVar[15];
	var_62_16[16] = myVar[16];
	var_62_17[17] = myVar[17];
	var_62_18[18] = myVar[18];
	var_62_19[19] = myVar[19];
	
}

__global__ void kernel_63(float * var_63_0, float * var_63_1, float * var_63_2, float * var_63_3, float * var_63_4, float * var_63_5, float * var_63_6, float * var_63_7, float * var_63_8, float * var_63_9, float * var_63_10, float * var_63_11, float * var_63_12, float * var_63_13, float * var_63_14, float * var_63_15, float * var_63_16, float * var_63_17, float * var_63_18, float * var_63_19) {
	__shared__ float myVar[1024];
	myVar[0] = 6.895080 * myVar[threadIdx.x];
	myVar[8] = 9.215553 * myVar[threadIdx.x];
	myVar[6] = 23.706782 * myVar[threadIdx.x];
	myVar[2] = 10.256461 * myVar[threadIdx.x];
	myVar[3] = 9.793091 * myVar[threadIdx.x];
	myVar[4] = 8.968549 * myVar[threadIdx.x];
	myVar[5] = 42.267950 * myVar[threadIdx.x];
	myVar[3] = 4.966480 * myVar[threadIdx.x];
	myVar[6] = 22.996148 * myVar[threadIdx.x];
	myVar[0] = 2.050984 * myVar[threadIdx.x];
	var_63_0[0] = myVar[0];
	var_63_1[1] = myVar[1];
	var_63_2[2] = myVar[2];
	var_63_3[3] = myVar[3];
	var_63_4[4] = myVar[4];
	var_63_5[5] = myVar[5];
	var_63_6[6] = myVar[6];
	var_63_7[7] = myVar[7];
	var_63_8[8] = myVar[8];
	var_63_9[9] = myVar[9];
	var_63_10[10] = myVar[10];
	var_63_11[11] = myVar[11];
	var_63_12[12] = myVar[12];
	var_63_13[13] = myVar[13];
	var_63_14[14] = myVar[14];
	var_63_15[15] = myVar[15];
	var_63_16[16] = myVar[16];
	var_63_17[17] = myVar[17];
	var_63_18[18] = myVar[18];
	var_63_19[19] = myVar[19];
	
}

__global__ void kernel_64(float * var_64_0, float * var_64_1, float * var_64_2, float * var_64_3, float * var_64_4, float * var_64_5, float * var_64_6, float * var_64_7, float * var_64_8, float * var_64_9, float * var_64_10, float * var_64_11, float * var_64_12, float * var_64_13, float * var_64_14, float * var_64_15, float * var_64_16, float * var_64_17, float * var_64_18, float * var_64_19) {
	__shared__ float myVar[1024];
	myVar[8] = 36.110596 * myVar[threadIdx.x];
	myVar[5] = 37.148806 * myVar[threadIdx.x];
	myVar[2] = 49.935301 * myVar[threadIdx.x];
	myVar[7] = 16.487061 * myVar[threadIdx.x];
	myVar[2] = 28.602171 * myVar[threadIdx.x];
	myVar[1] = 29.314083 * myVar[threadIdx.x];
	myVar[4] = 28.166662 * myVar[threadIdx.x];
	myVar[1] = 15.242463 * myVar[threadIdx.x];
	myVar[4] = 11.502013 * myVar[threadIdx.x];
	myVar[2] = 14.171394 * myVar[threadIdx.x];
	var_64_0[0] = myVar[0];
	var_64_1[1] = myVar[1];
	var_64_2[2] = myVar[2];
	var_64_3[3] = myVar[3];
	var_64_4[4] = myVar[4];
	var_64_5[5] = myVar[5];
	var_64_6[6] = myVar[6];
	var_64_7[7] = myVar[7];
	var_64_8[8] = myVar[8];
	var_64_9[9] = myVar[9];
	var_64_10[10] = myVar[10];
	var_64_11[11] = myVar[11];
	var_64_12[12] = myVar[12];
	var_64_13[13] = myVar[13];
	var_64_14[14] = myVar[14];
	var_64_15[15] = myVar[15];
	var_64_16[16] = myVar[16];
	var_64_17[17] = myVar[17];
	var_64_18[18] = myVar[18];
	var_64_19[19] = myVar[19];
	
}

__global__ void kernel_65(float * var_65_0, float * var_65_1, float * var_65_2, float * var_65_3, float * var_65_4, float * var_65_5, float * var_65_6, float * var_65_7, float * var_65_8, float * var_65_9, float * var_65_10, float * var_65_11, float * var_65_12, float * var_65_13, float * var_65_14, float * var_65_15, float * var_65_16, float * var_65_17, float * var_65_18, float * var_65_19) {
	__shared__ float myVar[1024];
	myVar[7] = 39.313026 * myVar[threadIdx.x];
	myVar[1] = 28.653257 * myVar[threadIdx.x];
	myVar[5] = 18.814101 * myVar[threadIdx.x];
	myVar[0] = 32.994610 * myVar[threadIdx.x];
	myVar[2] = 39.752569 * myVar[threadIdx.x];
	myVar[2] = 26.272723 * myVar[threadIdx.x];
	myVar[5] = 20.636778 * myVar[threadIdx.x];
	myVar[5] = 0.615425 * myVar[threadIdx.x];
	myVar[2] = 14.548679 * myVar[threadIdx.x];
	myVar[2] = 4.017999 * myVar[threadIdx.x];
	var_65_0[0] = myVar[0];
	var_65_1[1] = myVar[1];
	var_65_2[2] = myVar[2];
	var_65_3[3] = myVar[3];
	var_65_4[4] = myVar[4];
	var_65_5[5] = myVar[5];
	var_65_6[6] = myVar[6];
	var_65_7[7] = myVar[7];
	var_65_8[8] = myVar[8];
	var_65_9[9] = myVar[9];
	var_65_10[10] = myVar[10];
	var_65_11[11] = myVar[11];
	var_65_12[12] = myVar[12];
	var_65_13[13] = myVar[13];
	var_65_14[14] = myVar[14];
	var_65_15[15] = myVar[15];
	var_65_16[16] = myVar[16];
	var_65_17[17] = myVar[17];
	var_65_18[18] = myVar[18];
	var_65_19[19] = myVar[19];
	
}

__global__ void kernel_66(float * var_66_0, float * var_66_1, float * var_66_2, float * var_66_3, float * var_66_4, float * var_66_5, float * var_66_6, float * var_66_7, float * var_66_8, float * var_66_9, float * var_66_10, float * var_66_11, float * var_66_12, float * var_66_13, float * var_66_14, float * var_66_15, float * var_66_16, float * var_66_17, float * var_66_18, float * var_66_19) {
	__shared__ float myVar[1024];
	myVar[2] = 28.201326 * myVar[threadIdx.x];
	myVar[5] = 41.466212 * myVar[threadIdx.x];
	myVar[1] = 14.679026 * myVar[threadIdx.x];
	myVar[8] = 1.499039 * myVar[threadIdx.x];
	myVar[0] = 38.744664 * myVar[threadIdx.x];
	myVar[1] = 6.954847 * myVar[threadIdx.x];
	myVar[8] = 7.584151 * myVar[threadIdx.x];
	myVar[0] = 6.001016 * myVar[threadIdx.x];
	myVar[6] = 2.057322 * myVar[threadIdx.x];
	myVar[2] = 19.877629 * myVar[threadIdx.x];
	var_66_0[0] = myVar[0];
	var_66_1[1] = myVar[1];
	var_66_2[2] = myVar[2];
	var_66_3[3] = myVar[3];
	var_66_4[4] = myVar[4];
	var_66_5[5] = myVar[5];
	var_66_6[6] = myVar[6];
	var_66_7[7] = myVar[7];
	var_66_8[8] = myVar[8];
	var_66_9[9] = myVar[9];
	var_66_10[10] = myVar[10];
	var_66_11[11] = myVar[11];
	var_66_12[12] = myVar[12];
	var_66_13[13] = myVar[13];
	var_66_14[14] = myVar[14];
	var_66_15[15] = myVar[15];
	var_66_16[16] = myVar[16];
	var_66_17[17] = myVar[17];
	var_66_18[18] = myVar[18];
	var_66_19[19] = myVar[19];
	
}

__global__ void kernel_67(float * var_67_0, float * var_67_1, float * var_67_2, float * var_67_3, float * var_67_4, float * var_67_5, float * var_67_6, float * var_67_7, float * var_67_8, float * var_67_9, float * var_67_10, float * var_67_11, float * var_67_12, float * var_67_13, float * var_67_14, float * var_67_15, float * var_67_16, float * var_67_17, float * var_67_18, float * var_67_19) {
	__shared__ float myVar[1024];
	myVar[9] = 13.140397 * myVar[threadIdx.x];
	myVar[9] = 46.523538 * myVar[threadIdx.x];
	myVar[7] = 3.033182 * myVar[threadIdx.x];
	myVar[5] = 26.795130 * myVar[threadIdx.x];
	myVar[9] = 30.995732 * myVar[threadIdx.x];
	myVar[8] = 38.365689 * myVar[threadIdx.x];
	myVar[5] = 23.620947 * myVar[threadIdx.x];
	myVar[0] = 15.542619 * myVar[threadIdx.x];
	myVar[8] = 38.233037 * myVar[threadIdx.x];
	myVar[9] = 19.422601 * myVar[threadIdx.x];
	var_67_0[0] = myVar[0];
	var_67_1[1] = myVar[1];
	var_67_2[2] = myVar[2];
	var_67_3[3] = myVar[3];
	var_67_4[4] = myVar[4];
	var_67_5[5] = myVar[5];
	var_67_6[6] = myVar[6];
	var_67_7[7] = myVar[7];
	var_67_8[8] = myVar[8];
	var_67_9[9] = myVar[9];
	var_67_10[10] = myVar[10];
	var_67_11[11] = myVar[11];
	var_67_12[12] = myVar[12];
	var_67_13[13] = myVar[13];
	var_67_14[14] = myVar[14];
	var_67_15[15] = myVar[15];
	var_67_16[16] = myVar[16];
	var_67_17[17] = myVar[17];
	var_67_18[18] = myVar[18];
	var_67_19[19] = myVar[19];
	
}

__global__ void kernel_68(float * var_68_0, float * var_68_1, float * var_68_2, float * var_68_3, float * var_68_4, float * var_68_5, float * var_68_6, float * var_68_7, float * var_68_8, float * var_68_9, float * var_68_10, float * var_68_11, float * var_68_12, float * var_68_13, float * var_68_14, float * var_68_15, float * var_68_16, float * var_68_17, float * var_68_18, float * var_68_19) {
	__shared__ float myVar[1024];
	myVar[8] = 43.766329 * myVar[threadIdx.x];
	myVar[6] = 27.192649 * myVar[threadIdx.x];
	myVar[8] = 23.785973 * myVar[threadIdx.x];
	myVar[9] = 41.367051 * myVar[threadIdx.x];
	myVar[3] = 9.792684 * myVar[threadIdx.x];
	myVar[6] = 31.583700 * myVar[threadIdx.x];
	myVar[9] = 40.443538 * myVar[threadIdx.x];
	myVar[9] = 49.320981 * myVar[threadIdx.x];
	myVar[9] = 34.924821 * myVar[threadIdx.x];
	myVar[8] = 48.775670 * myVar[threadIdx.x];
	var_68_0[0] = myVar[0];
	var_68_1[1] = myVar[1];
	var_68_2[2] = myVar[2];
	var_68_3[3] = myVar[3];
	var_68_4[4] = myVar[4];
	var_68_5[5] = myVar[5];
	var_68_6[6] = myVar[6];
	var_68_7[7] = myVar[7];
	var_68_8[8] = myVar[8];
	var_68_9[9] = myVar[9];
	var_68_10[10] = myVar[10];
	var_68_11[11] = myVar[11];
	var_68_12[12] = myVar[12];
	var_68_13[13] = myVar[13];
	var_68_14[14] = myVar[14];
	var_68_15[15] = myVar[15];
	var_68_16[16] = myVar[16];
	var_68_17[17] = myVar[17];
	var_68_18[18] = myVar[18];
	var_68_19[19] = myVar[19];
	
}

__global__ void kernel_69(float * var_69_0, float * var_69_1, float * var_69_2, float * var_69_3, float * var_69_4, float * var_69_5, float * var_69_6, float * var_69_7, float * var_69_8, float * var_69_9, float * var_69_10, float * var_69_11, float * var_69_12, float * var_69_13, float * var_69_14, float * var_69_15, float * var_69_16, float * var_69_17, float * var_69_18, float * var_69_19) {
	__shared__ float myVar[1024];
	myVar[9] = 7.707027 * myVar[threadIdx.x];
	myVar[7] = 21.683945 * myVar[threadIdx.x];
	myVar[6] = 10.428954 * myVar[threadIdx.x];
	myVar[2] = 24.683253 * myVar[threadIdx.x];
	myVar[6] = 2.906943 * myVar[threadIdx.x];
	myVar[0] = 9.477013 * myVar[threadIdx.x];
	myVar[6] = 46.410466 * myVar[threadIdx.x];
	myVar[7] = 39.701940 * myVar[threadIdx.x];
	myVar[6] = 29.047030 * myVar[threadIdx.x];
	myVar[7] = 8.740714 * myVar[threadIdx.x];
	var_69_0[0] = myVar[0];
	var_69_1[1] = myVar[1];
	var_69_2[2] = myVar[2];
	var_69_3[3] = myVar[3];
	var_69_4[4] = myVar[4];
	var_69_5[5] = myVar[5];
	var_69_6[6] = myVar[6];
	var_69_7[7] = myVar[7];
	var_69_8[8] = myVar[8];
	var_69_9[9] = myVar[9];
	var_69_10[10] = myVar[10];
	var_69_11[11] = myVar[11];
	var_69_12[12] = myVar[12];
	var_69_13[13] = myVar[13];
	var_69_14[14] = myVar[14];
	var_69_15[15] = myVar[15];
	var_69_16[16] = myVar[16];
	var_69_17[17] = myVar[17];
	var_69_18[18] = myVar[18];
	var_69_19[19] = myVar[19];
	
}

__global__ void kernel_70(float * var_70_0, float * var_70_1, float * var_70_2, float * var_70_3, float * var_70_4, float * var_70_5, float * var_70_6, float * var_70_7, float * var_70_8, float * var_70_9, float * var_70_10, float * var_70_11, float * var_70_12, float * var_70_13, float * var_70_14, float * var_70_15, float * var_70_16, float * var_70_17, float * var_70_18, float * var_70_19) {
	__shared__ float myVar[1024];
	myVar[0] = 32.466671 * myVar[threadIdx.x];
	myVar[6] = 4.534523 * myVar[threadIdx.x];
	myVar[3] = 11.770629 * myVar[threadIdx.x];
	myVar[8] = 46.817250 * myVar[threadIdx.x];
	myVar[1] = 10.812564 * myVar[threadIdx.x];
	myVar[4] = 1.846516 * myVar[threadIdx.x];
	myVar[9] = 35.385064 * myVar[threadIdx.x];
	myVar[6] = 2.934052 * myVar[threadIdx.x];
	myVar[7] = 4.579234 * myVar[threadIdx.x];
	myVar[3] = 4.102144 * myVar[threadIdx.x];
	var_70_0[0] = myVar[0];
	var_70_1[1] = myVar[1];
	var_70_2[2] = myVar[2];
	var_70_3[3] = myVar[3];
	var_70_4[4] = myVar[4];
	var_70_5[5] = myVar[5];
	var_70_6[6] = myVar[6];
	var_70_7[7] = myVar[7];
	var_70_8[8] = myVar[8];
	var_70_9[9] = myVar[9];
	var_70_10[10] = myVar[10];
	var_70_11[11] = myVar[11];
	var_70_12[12] = myVar[12];
	var_70_13[13] = myVar[13];
	var_70_14[14] = myVar[14];
	var_70_15[15] = myVar[15];
	var_70_16[16] = myVar[16];
	var_70_17[17] = myVar[17];
	var_70_18[18] = myVar[18];
	var_70_19[19] = myVar[19];
	
}

__global__ void kernel_71(float * var_71_0, float * var_71_1, float * var_71_2, float * var_71_3, float * var_71_4, float * var_71_5, float * var_71_6, float * var_71_7, float * var_71_8, float * var_71_9, float * var_71_10, float * var_71_11, float * var_71_12, float * var_71_13, float * var_71_14, float * var_71_15, float * var_71_16, float * var_71_17, float * var_71_18, float * var_71_19) {
	__shared__ float myVar[1024];
	myVar[4] = 41.170601 * myVar[threadIdx.x];
	myVar[4] = 13.651976 * myVar[threadIdx.x];
	myVar[6] = 6.834099 * myVar[threadIdx.x];
	myVar[2] = 26.910055 * myVar[threadIdx.x];
	myVar[9] = 43.342289 * myVar[threadIdx.x];
	myVar[8] = 10.524127 * myVar[threadIdx.x];
	myVar[7] = 40.450485 * myVar[threadIdx.x];
	myVar[4] = 2.319119 * myVar[threadIdx.x];
	myVar[3] = 34.342359 * myVar[threadIdx.x];
	myVar[0] = 30.794013 * myVar[threadIdx.x];
	var_71_0[0] = myVar[0];
	var_71_1[1] = myVar[1];
	var_71_2[2] = myVar[2];
	var_71_3[3] = myVar[3];
	var_71_4[4] = myVar[4];
	var_71_5[5] = myVar[5];
	var_71_6[6] = myVar[6];
	var_71_7[7] = myVar[7];
	var_71_8[8] = myVar[8];
	var_71_9[9] = myVar[9];
	var_71_10[10] = myVar[10];
	var_71_11[11] = myVar[11];
	var_71_12[12] = myVar[12];
	var_71_13[13] = myVar[13];
	var_71_14[14] = myVar[14];
	var_71_15[15] = myVar[15];
	var_71_16[16] = myVar[16];
	var_71_17[17] = myVar[17];
	var_71_18[18] = myVar[18];
	var_71_19[19] = myVar[19];
	
}

__global__ void kernel_72(float * var_72_0, float * var_72_1, float * var_72_2, float * var_72_3, float * var_72_4, float * var_72_5, float * var_72_6, float * var_72_7, float * var_72_8, float * var_72_9, float * var_72_10, float * var_72_11, float * var_72_12, float * var_72_13, float * var_72_14, float * var_72_15, float * var_72_16, float * var_72_17, float * var_72_18, float * var_72_19) {
	__shared__ float myVar[1024];
	myVar[4] = 21.942838 * myVar[threadIdx.x];
	myVar[7] = 7.979056 * myVar[threadIdx.x];
	myVar[9] = 42.630924 * myVar[threadIdx.x];
	myVar[8] = 26.467586 * myVar[threadIdx.x];
	myVar[7] = 3.136713 * myVar[threadIdx.x];
	myVar[6] = 29.039205 * myVar[threadIdx.x];
	myVar[7] = 15.514938 * myVar[threadIdx.x];
	myVar[1] = 24.760805 * myVar[threadIdx.x];
	myVar[5] = 14.885079 * myVar[threadIdx.x];
	myVar[2] = 4.947861 * myVar[threadIdx.x];
	var_72_0[0] = myVar[0];
	var_72_1[1] = myVar[1];
	var_72_2[2] = myVar[2];
	var_72_3[3] = myVar[3];
	var_72_4[4] = myVar[4];
	var_72_5[5] = myVar[5];
	var_72_6[6] = myVar[6];
	var_72_7[7] = myVar[7];
	var_72_8[8] = myVar[8];
	var_72_9[9] = myVar[9];
	var_72_10[10] = myVar[10];
	var_72_11[11] = myVar[11];
	var_72_12[12] = myVar[12];
	var_72_13[13] = myVar[13];
	var_72_14[14] = myVar[14];
	var_72_15[15] = myVar[15];
	var_72_16[16] = myVar[16];
	var_72_17[17] = myVar[17];
	var_72_18[18] = myVar[18];
	var_72_19[19] = myVar[19];
	
}

__global__ void kernel_73(float * var_73_0, float * var_73_1, float * var_73_2, float * var_73_3, float * var_73_4, float * var_73_5, float * var_73_6, float * var_73_7, float * var_73_8, float * var_73_9, float * var_73_10, float * var_73_11, float * var_73_12, float * var_73_13, float * var_73_14, float * var_73_15, float * var_73_16, float * var_73_17, float * var_73_18, float * var_73_19) {
	__shared__ float myVar[1024];
	myVar[1] = 19.735558 * myVar[threadIdx.x];
	myVar[9] = 18.724792 * myVar[threadIdx.x];
	myVar[1] = 41.933294 * myVar[threadIdx.x];
	myVar[2] = 33.499581 * myVar[threadIdx.x];
	myVar[9] = 17.797468 * myVar[threadIdx.x];
	myVar[9] = 5.813515 * myVar[threadIdx.x];
	myVar[2] = 8.649189 * myVar[threadIdx.x];
	myVar[9] = 13.243289 * myVar[threadIdx.x];
	myVar[7] = 32.770060 * myVar[threadIdx.x];
	myVar[4] = 5.658495 * myVar[threadIdx.x];
	var_73_0[0] = myVar[0];
	var_73_1[1] = myVar[1];
	var_73_2[2] = myVar[2];
	var_73_3[3] = myVar[3];
	var_73_4[4] = myVar[4];
	var_73_5[5] = myVar[5];
	var_73_6[6] = myVar[6];
	var_73_7[7] = myVar[7];
	var_73_8[8] = myVar[8];
	var_73_9[9] = myVar[9];
	var_73_10[10] = myVar[10];
	var_73_11[11] = myVar[11];
	var_73_12[12] = myVar[12];
	var_73_13[13] = myVar[13];
	var_73_14[14] = myVar[14];
	var_73_15[15] = myVar[15];
	var_73_16[16] = myVar[16];
	var_73_17[17] = myVar[17];
	var_73_18[18] = myVar[18];
	var_73_19[19] = myVar[19];
	
}

__global__ void kernel_74(float * var_74_0, float * var_74_1, float * var_74_2, float * var_74_3, float * var_74_4, float * var_74_5, float * var_74_6, float * var_74_7, float * var_74_8, float * var_74_9, float * var_74_10, float * var_74_11, float * var_74_12, float * var_74_13, float * var_74_14, float * var_74_15, float * var_74_16, float * var_74_17, float * var_74_18, float * var_74_19) {
	__shared__ float myVar[1024];
	myVar[2] = 22.888231 * myVar[threadIdx.x];
	myVar[0] = 19.859960 * myVar[threadIdx.x];
	myVar[7] = 47.655515 * myVar[threadIdx.x];
	myVar[8] = 15.599701 * myVar[threadIdx.x];
	myVar[5] = 25.612808 * myVar[threadIdx.x];
	myVar[9] = 22.018348 * myVar[threadIdx.x];
	myVar[1] = 7.647660 * myVar[threadIdx.x];
	myVar[9] = 49.856509 * myVar[threadIdx.x];
	myVar[3] = 18.859440 * myVar[threadIdx.x];
	myVar[1] = 36.744651 * myVar[threadIdx.x];
	var_74_0[0] = myVar[0];
	var_74_1[1] = myVar[1];
	var_74_2[2] = myVar[2];
	var_74_3[3] = myVar[3];
	var_74_4[4] = myVar[4];
	var_74_5[5] = myVar[5];
	var_74_6[6] = myVar[6];
	var_74_7[7] = myVar[7];
	var_74_8[8] = myVar[8];
	var_74_9[9] = myVar[9];
	var_74_10[10] = myVar[10];
	var_74_11[11] = myVar[11];
	var_74_12[12] = myVar[12];
	var_74_13[13] = myVar[13];
	var_74_14[14] = myVar[14];
	var_74_15[15] = myVar[15];
	var_74_16[16] = myVar[16];
	var_74_17[17] = myVar[17];
	var_74_18[18] = myVar[18];
	var_74_19[19] = myVar[19];
	
}

__global__ void kernel_75(float * var_75_0, float * var_75_1, float * var_75_2, float * var_75_3, float * var_75_4, float * var_75_5, float * var_75_6, float * var_75_7, float * var_75_8, float * var_75_9, float * var_75_10, float * var_75_11, float * var_75_12, float * var_75_13, float * var_75_14, float * var_75_15, float * var_75_16, float * var_75_17, float * var_75_18, float * var_75_19) {
	__shared__ float myVar[1024];
	myVar[0] = 28.702369 * myVar[threadIdx.x];
	myVar[3] = 33.381361 * myVar[threadIdx.x];
	myVar[7] = 12.768239 * myVar[threadIdx.x];
	myVar[3] = 48.437948 * myVar[threadIdx.x];
	myVar[0] = 26.398561 * myVar[threadIdx.x];
	myVar[7] = 49.402374 * myVar[threadIdx.x];
	myVar[5] = 44.292255 * myVar[threadIdx.x];
	myVar[1] = 28.579125 * myVar[threadIdx.x];
	myVar[0] = 12.116632 * myVar[threadIdx.x];
	myVar[6] = 12.440771 * myVar[threadIdx.x];
	var_75_0[0] = myVar[0];
	var_75_1[1] = myVar[1];
	var_75_2[2] = myVar[2];
	var_75_3[3] = myVar[3];
	var_75_4[4] = myVar[4];
	var_75_5[5] = myVar[5];
	var_75_6[6] = myVar[6];
	var_75_7[7] = myVar[7];
	var_75_8[8] = myVar[8];
	var_75_9[9] = myVar[9];
	var_75_10[10] = myVar[10];
	var_75_11[11] = myVar[11];
	var_75_12[12] = myVar[12];
	var_75_13[13] = myVar[13];
	var_75_14[14] = myVar[14];
	var_75_15[15] = myVar[15];
	var_75_16[16] = myVar[16];
	var_75_17[17] = myVar[17];
	var_75_18[18] = myVar[18];
	var_75_19[19] = myVar[19];
	
}

__global__ void kernel_76(float * var_76_0, float * var_76_1, float * var_76_2, float * var_76_3, float * var_76_4, float * var_76_5, float * var_76_6, float * var_76_7, float * var_76_8, float * var_76_9, float * var_76_10, float * var_76_11, float * var_76_12, float * var_76_13, float * var_76_14, float * var_76_15, float * var_76_16, float * var_76_17, float * var_76_18, float * var_76_19) {
	__shared__ float myVar[1024];
	myVar[5] = 32.013261 * myVar[threadIdx.x];
	myVar[6] = 14.655572 * myVar[threadIdx.x];
	myVar[6] = 24.448350 * myVar[threadIdx.x];
	myVar[5] = 26.084300 * myVar[threadIdx.x];
	myVar[2] = 2.995143 * myVar[threadIdx.x];
	myVar[9] = 12.297336 * myVar[threadIdx.x];
	myVar[3] = 13.592696 * myVar[threadIdx.x];
	myVar[7] = 30.036508 * myVar[threadIdx.x];
	myVar[6] = 34.314084 * myVar[threadIdx.x];
	myVar[8] = 3.278087 * myVar[threadIdx.x];
	var_76_0[0] = myVar[0];
	var_76_1[1] = myVar[1];
	var_76_2[2] = myVar[2];
	var_76_3[3] = myVar[3];
	var_76_4[4] = myVar[4];
	var_76_5[5] = myVar[5];
	var_76_6[6] = myVar[6];
	var_76_7[7] = myVar[7];
	var_76_8[8] = myVar[8];
	var_76_9[9] = myVar[9];
	var_76_10[10] = myVar[10];
	var_76_11[11] = myVar[11];
	var_76_12[12] = myVar[12];
	var_76_13[13] = myVar[13];
	var_76_14[14] = myVar[14];
	var_76_15[15] = myVar[15];
	var_76_16[16] = myVar[16];
	var_76_17[17] = myVar[17];
	var_76_18[18] = myVar[18];
	var_76_19[19] = myVar[19];
	
}

__global__ void kernel_77(float * var_77_0, float * var_77_1, float * var_77_2, float * var_77_3, float * var_77_4, float * var_77_5, float * var_77_6, float * var_77_7, float * var_77_8, float * var_77_9, float * var_77_10, float * var_77_11, float * var_77_12, float * var_77_13, float * var_77_14, float * var_77_15, float * var_77_16, float * var_77_17, float * var_77_18, float * var_77_19) {
	__shared__ float myVar[1024];
	myVar[6] = 0.586754 * myVar[threadIdx.x];
	myVar[2] = 21.242784 * myVar[threadIdx.x];
	myVar[6] = 19.633715 * myVar[threadIdx.x];
	myVar[3] = 7.058163 * myVar[threadIdx.x];
	myVar[5] = 17.066796 * myVar[threadIdx.x];
	myVar[9] = 49.404883 * myVar[threadIdx.x];
	myVar[7] = 5.806399 * myVar[threadIdx.x];
	myVar[4] = 24.515104 * myVar[threadIdx.x];
	myVar[9] = 44.272751 * myVar[threadIdx.x];
	myVar[0] = 24.247686 * myVar[threadIdx.x];
	var_77_0[0] = myVar[0];
	var_77_1[1] = myVar[1];
	var_77_2[2] = myVar[2];
	var_77_3[3] = myVar[3];
	var_77_4[4] = myVar[4];
	var_77_5[5] = myVar[5];
	var_77_6[6] = myVar[6];
	var_77_7[7] = myVar[7];
	var_77_8[8] = myVar[8];
	var_77_9[9] = myVar[9];
	var_77_10[10] = myVar[10];
	var_77_11[11] = myVar[11];
	var_77_12[12] = myVar[12];
	var_77_13[13] = myVar[13];
	var_77_14[14] = myVar[14];
	var_77_15[15] = myVar[15];
	var_77_16[16] = myVar[16];
	var_77_17[17] = myVar[17];
	var_77_18[18] = myVar[18];
	var_77_19[19] = myVar[19];
	
}

__global__ void kernel_78(float * var_78_0, float * var_78_1, float * var_78_2, float * var_78_3, float * var_78_4, float * var_78_5, float * var_78_6, float * var_78_7, float * var_78_8, float * var_78_9, float * var_78_10, float * var_78_11, float * var_78_12, float * var_78_13, float * var_78_14, float * var_78_15, float * var_78_16, float * var_78_17, float * var_78_18, float * var_78_19) {
	__shared__ float myVar[1024];
	myVar[0] = 1.064089 * myVar[threadIdx.x];
	myVar[4] = 19.377686 * myVar[threadIdx.x];
	myVar[1] = 41.891596 * myVar[threadIdx.x];
	myVar[2] = 32.112862 * myVar[threadIdx.x];
	myVar[2] = 47.966346 * myVar[threadIdx.x];
	myVar[7] = 13.487199 * myVar[threadIdx.x];
	myVar[2] = 9.251429 * myVar[threadIdx.x];
	myVar[8] = 20.397124 * myVar[threadIdx.x];
	myVar[8] = 12.054116 * myVar[threadIdx.x];
	myVar[1] = 30.061273 * myVar[threadIdx.x];
	var_78_0[0] = myVar[0];
	var_78_1[1] = myVar[1];
	var_78_2[2] = myVar[2];
	var_78_3[3] = myVar[3];
	var_78_4[4] = myVar[4];
	var_78_5[5] = myVar[5];
	var_78_6[6] = myVar[6];
	var_78_7[7] = myVar[7];
	var_78_8[8] = myVar[8];
	var_78_9[9] = myVar[9];
	var_78_10[10] = myVar[10];
	var_78_11[11] = myVar[11];
	var_78_12[12] = myVar[12];
	var_78_13[13] = myVar[13];
	var_78_14[14] = myVar[14];
	var_78_15[15] = myVar[15];
	var_78_16[16] = myVar[16];
	var_78_17[17] = myVar[17];
	var_78_18[18] = myVar[18];
	var_78_19[19] = myVar[19];
	
}

__global__ void kernel_79(float * var_79_0, float * var_79_1, float * var_79_2, float * var_79_3, float * var_79_4, float * var_79_5, float * var_79_6, float * var_79_7, float * var_79_8, float * var_79_9, float * var_79_10, float * var_79_11, float * var_79_12, float * var_79_13, float * var_79_14, float * var_79_15, float * var_79_16, float * var_79_17, float * var_79_18, float * var_79_19) {
	__shared__ float myVar[1024];
	myVar[3] = 18.196824 * myVar[threadIdx.x];
	myVar[7] = 43.837447 * myVar[threadIdx.x];
	myVar[4] = 23.153390 * myVar[threadIdx.x];
	myVar[4] = 27.992319 * myVar[threadIdx.x];
	myVar[5] = 9.024027 * myVar[threadIdx.x];
	myVar[1] = 48.804173 * myVar[threadIdx.x];
	myVar[0] = 7.408689 * myVar[threadIdx.x];
	myVar[7] = 23.778323 * myVar[threadIdx.x];
	myVar[6] = 42.920944 * myVar[threadIdx.x];
	myVar[7] = 29.065020 * myVar[threadIdx.x];
	var_79_0[0] = myVar[0];
	var_79_1[1] = myVar[1];
	var_79_2[2] = myVar[2];
	var_79_3[3] = myVar[3];
	var_79_4[4] = myVar[4];
	var_79_5[5] = myVar[5];
	var_79_6[6] = myVar[6];
	var_79_7[7] = myVar[7];
	var_79_8[8] = myVar[8];
	var_79_9[9] = myVar[9];
	var_79_10[10] = myVar[10];
	var_79_11[11] = myVar[11];
	var_79_12[12] = myVar[12];
	var_79_13[13] = myVar[13];
	var_79_14[14] = myVar[14];
	var_79_15[15] = myVar[15];
	var_79_16[16] = myVar[16];
	var_79_17[17] = myVar[17];
	var_79_18[18] = myVar[18];
	var_79_19[19] = myVar[19];
	
}

__global__ void kernel_80(float * var_80_0, float * var_80_1, float * var_80_2, float * var_80_3, float * var_80_4, float * var_80_5, float * var_80_6, float * var_80_7, float * var_80_8, float * var_80_9, float * var_80_10, float * var_80_11, float * var_80_12, float * var_80_13, float * var_80_14, float * var_80_15, float * var_80_16, float * var_80_17, float * var_80_18, float * var_80_19) {
	__shared__ float myVar[1024];
	myVar[6] = 29.986274 * myVar[threadIdx.x];
	myVar[8] = 6.206306 * myVar[threadIdx.x];
	myVar[5] = 13.512669 * myVar[threadIdx.x];
	myVar[5] = 39.159626 * myVar[threadIdx.x];
	myVar[4] = 34.758343 * myVar[threadIdx.x];
	myVar[1] = 37.380999 * myVar[threadIdx.x];
	myVar[1] = 49.354852 * myVar[threadIdx.x];
	myVar[9] = 38.446218 * myVar[threadIdx.x];
	myVar[3] = 39.416359 * myVar[threadIdx.x];
	myVar[8] = 42.874535 * myVar[threadIdx.x];
	var_80_0[0] = myVar[0];
	var_80_1[1] = myVar[1];
	var_80_2[2] = myVar[2];
	var_80_3[3] = myVar[3];
	var_80_4[4] = myVar[4];
	var_80_5[5] = myVar[5];
	var_80_6[6] = myVar[6];
	var_80_7[7] = myVar[7];
	var_80_8[8] = myVar[8];
	var_80_9[9] = myVar[9];
	var_80_10[10] = myVar[10];
	var_80_11[11] = myVar[11];
	var_80_12[12] = myVar[12];
	var_80_13[13] = myVar[13];
	var_80_14[14] = myVar[14];
	var_80_15[15] = myVar[15];
	var_80_16[16] = myVar[16];
	var_80_17[17] = myVar[17];
	var_80_18[18] = myVar[18];
	var_80_19[19] = myVar[19];
	
}

__global__ void kernel_81(float * var_81_0, float * var_81_1, float * var_81_2, float * var_81_3, float * var_81_4, float * var_81_5, float * var_81_6, float * var_81_7, float * var_81_8, float * var_81_9, float * var_81_10, float * var_81_11, float * var_81_12, float * var_81_13, float * var_81_14, float * var_81_15, float * var_81_16, float * var_81_17, float * var_81_18, float * var_81_19) {
	__shared__ float myVar[1024];
	myVar[8] = 47.171056 * myVar[threadIdx.x];
	myVar[9] = 34.012280 * myVar[threadIdx.x];
	myVar[0] = 48.939174 * myVar[threadIdx.x];
	myVar[2] = 23.415897 * myVar[threadIdx.x];
	myVar[9] = 11.547523 * myVar[threadIdx.x];
	myVar[9] = 46.820279 * myVar[threadIdx.x];
	myVar[7] = 30.271263 * myVar[threadIdx.x];
	myVar[4] = 27.460999 * myVar[threadIdx.x];
	myVar[8] = 41.792915 * myVar[threadIdx.x];
	myVar[0] = 3.939068 * myVar[threadIdx.x];
	var_81_0[0] = myVar[0];
	var_81_1[1] = myVar[1];
	var_81_2[2] = myVar[2];
	var_81_3[3] = myVar[3];
	var_81_4[4] = myVar[4];
	var_81_5[5] = myVar[5];
	var_81_6[6] = myVar[6];
	var_81_7[7] = myVar[7];
	var_81_8[8] = myVar[8];
	var_81_9[9] = myVar[9];
	var_81_10[10] = myVar[10];
	var_81_11[11] = myVar[11];
	var_81_12[12] = myVar[12];
	var_81_13[13] = myVar[13];
	var_81_14[14] = myVar[14];
	var_81_15[15] = myVar[15];
	var_81_16[16] = myVar[16];
	var_81_17[17] = myVar[17];
	var_81_18[18] = myVar[18];
	var_81_19[19] = myVar[19];
	
}

__global__ void kernel_82(float * var_82_0, float * var_82_1, float * var_82_2, float * var_82_3, float * var_82_4, float * var_82_5, float * var_82_6, float * var_82_7, float * var_82_8, float * var_82_9, float * var_82_10, float * var_82_11, float * var_82_12, float * var_82_13, float * var_82_14, float * var_82_15, float * var_82_16, float * var_82_17, float * var_82_18, float * var_82_19) {
	__shared__ float myVar[1024];
	myVar[1] = 24.598735 * myVar[threadIdx.x];
	myVar[8] = 27.287896 * myVar[threadIdx.x];
	myVar[8] = 44.845122 * myVar[threadIdx.x];
	myVar[1] = 20.370291 * myVar[threadIdx.x];
	myVar[1] = 17.560660 * myVar[threadIdx.x];
	myVar[9] = 36.935529 * myVar[threadIdx.x];
	myVar[9] = 36.543273 * myVar[threadIdx.x];
	myVar[2] = 1.067267 * myVar[threadIdx.x];
	myVar[4] = 24.287814 * myVar[threadIdx.x];
	myVar[0] = 48.009908 * myVar[threadIdx.x];
	var_82_0[0] = myVar[0];
	var_82_1[1] = myVar[1];
	var_82_2[2] = myVar[2];
	var_82_3[3] = myVar[3];
	var_82_4[4] = myVar[4];
	var_82_5[5] = myVar[5];
	var_82_6[6] = myVar[6];
	var_82_7[7] = myVar[7];
	var_82_8[8] = myVar[8];
	var_82_9[9] = myVar[9];
	var_82_10[10] = myVar[10];
	var_82_11[11] = myVar[11];
	var_82_12[12] = myVar[12];
	var_82_13[13] = myVar[13];
	var_82_14[14] = myVar[14];
	var_82_15[15] = myVar[15];
	var_82_16[16] = myVar[16];
	var_82_17[17] = myVar[17];
	var_82_18[18] = myVar[18];
	var_82_19[19] = myVar[19];
	
}

__global__ void kernel_83(float * var_83_0, float * var_83_1, float * var_83_2, float * var_83_3, float * var_83_4, float * var_83_5, float * var_83_6, float * var_83_7, float * var_83_8, float * var_83_9, float * var_83_10, float * var_83_11, float * var_83_12, float * var_83_13, float * var_83_14, float * var_83_15, float * var_83_16, float * var_83_17, float * var_83_18, float * var_83_19) {
	__shared__ float myVar[1024];
	myVar[9] = 36.390445 * myVar[threadIdx.x];
	myVar[0] = 39.153191 * myVar[threadIdx.x];
	myVar[5] = 17.985216 * myVar[threadIdx.x];
	myVar[1] = 22.930816 * myVar[threadIdx.x];
	myVar[1] = 6.413215 * myVar[threadIdx.x];
	myVar[3] = 19.841041 * myVar[threadIdx.x];
	myVar[6] = 28.020421 * myVar[threadIdx.x];
	myVar[8] = 26.730542 * myVar[threadIdx.x];
	myVar[7] = 23.492608 * myVar[threadIdx.x];
	myVar[1] = 32.477826 * myVar[threadIdx.x];
	var_83_0[0] = myVar[0];
	var_83_1[1] = myVar[1];
	var_83_2[2] = myVar[2];
	var_83_3[3] = myVar[3];
	var_83_4[4] = myVar[4];
	var_83_5[5] = myVar[5];
	var_83_6[6] = myVar[6];
	var_83_7[7] = myVar[7];
	var_83_8[8] = myVar[8];
	var_83_9[9] = myVar[9];
	var_83_10[10] = myVar[10];
	var_83_11[11] = myVar[11];
	var_83_12[12] = myVar[12];
	var_83_13[13] = myVar[13];
	var_83_14[14] = myVar[14];
	var_83_15[15] = myVar[15];
	var_83_16[16] = myVar[16];
	var_83_17[17] = myVar[17];
	var_83_18[18] = myVar[18];
	var_83_19[19] = myVar[19];
	
}

__global__ void kernel_84(float * var_84_0, float * var_84_1, float * var_84_2, float * var_84_3, float * var_84_4, float * var_84_5, float * var_84_6, float * var_84_7, float * var_84_8, float * var_84_9, float * var_84_10, float * var_84_11, float * var_84_12, float * var_84_13, float * var_84_14, float * var_84_15, float * var_84_16, float * var_84_17, float * var_84_18, float * var_84_19) {
	__shared__ float myVar[1024];
	myVar[1] = 15.828457 * myVar[threadIdx.x];
	myVar[8] = 11.603154 * myVar[threadIdx.x];
	myVar[7] = 23.479446 * myVar[threadIdx.x];
	myVar[9] = 40.390499 * myVar[threadIdx.x];
	myVar[0] = 49.498116 * myVar[threadIdx.x];
	myVar[5] = 5.547645 * myVar[threadIdx.x];
	myVar[6] = 32.120135 * myVar[threadIdx.x];
	myVar[8] = 13.189183 * myVar[threadIdx.x];
	myVar[4] = 5.747827 * myVar[threadIdx.x];
	myVar[7] = 13.207244 * myVar[threadIdx.x];
	var_84_0[0] = myVar[0];
	var_84_1[1] = myVar[1];
	var_84_2[2] = myVar[2];
	var_84_3[3] = myVar[3];
	var_84_4[4] = myVar[4];
	var_84_5[5] = myVar[5];
	var_84_6[6] = myVar[6];
	var_84_7[7] = myVar[7];
	var_84_8[8] = myVar[8];
	var_84_9[9] = myVar[9];
	var_84_10[10] = myVar[10];
	var_84_11[11] = myVar[11];
	var_84_12[12] = myVar[12];
	var_84_13[13] = myVar[13];
	var_84_14[14] = myVar[14];
	var_84_15[15] = myVar[15];
	var_84_16[16] = myVar[16];
	var_84_17[17] = myVar[17];
	var_84_18[18] = myVar[18];
	var_84_19[19] = myVar[19];
	
}

__global__ void kernel_85(float * var_85_0, float * var_85_1, float * var_85_2, float * var_85_3, float * var_85_4, float * var_85_5, float * var_85_6, float * var_85_7, float * var_85_8, float * var_85_9, float * var_85_10, float * var_85_11, float * var_85_12, float * var_85_13, float * var_85_14, float * var_85_15, float * var_85_16, float * var_85_17, float * var_85_18, float * var_85_19) {
	__shared__ float myVar[1024];
	myVar[5] = 29.469157 * myVar[threadIdx.x];
	myVar[1] = 8.046192 * myVar[threadIdx.x];
	myVar[6] = 23.251429 * myVar[threadIdx.x];
	myVar[3] = 38.798927 * myVar[threadIdx.x];
	myVar[1] = 5.437214 * myVar[threadIdx.x];
	myVar[9] = 16.948765 * myVar[threadIdx.x];
	myVar[6] = 38.654682 * myVar[threadIdx.x];
	myVar[0] = 39.937615 * myVar[threadIdx.x];
	myVar[1] = 46.182269 * myVar[threadIdx.x];
	myVar[2] = 10.417832 * myVar[threadIdx.x];
	var_85_0[0] = myVar[0];
	var_85_1[1] = myVar[1];
	var_85_2[2] = myVar[2];
	var_85_3[3] = myVar[3];
	var_85_4[4] = myVar[4];
	var_85_5[5] = myVar[5];
	var_85_6[6] = myVar[6];
	var_85_7[7] = myVar[7];
	var_85_8[8] = myVar[8];
	var_85_9[9] = myVar[9];
	var_85_10[10] = myVar[10];
	var_85_11[11] = myVar[11];
	var_85_12[12] = myVar[12];
	var_85_13[13] = myVar[13];
	var_85_14[14] = myVar[14];
	var_85_15[15] = myVar[15];
	var_85_16[16] = myVar[16];
	var_85_17[17] = myVar[17];
	var_85_18[18] = myVar[18];
	var_85_19[19] = myVar[19];
	
}

__global__ void kernel_86(float * var_86_0, float * var_86_1, float * var_86_2, float * var_86_3, float * var_86_4, float * var_86_5, float * var_86_6, float * var_86_7, float * var_86_8, float * var_86_9, float * var_86_10, float * var_86_11, float * var_86_12, float * var_86_13, float * var_86_14, float * var_86_15, float * var_86_16, float * var_86_17, float * var_86_18, float * var_86_19) {
	__shared__ float myVar[1024];
	myVar[7] = 20.686764 * myVar[threadIdx.x];
	myVar[8] = 14.382144 * myVar[threadIdx.x];
	myVar[4] = 2.592520 * myVar[threadIdx.x];
	myVar[8] = 32.843433 * myVar[threadIdx.x];
	myVar[7] = 20.987655 * myVar[threadIdx.x];
	myVar[3] = 19.882539 * myVar[threadIdx.x];
	myVar[8] = 29.850287 * myVar[threadIdx.x];
	myVar[2] = 37.142193 * myVar[threadIdx.x];
	myVar[7] = 15.355836 * myVar[threadIdx.x];
	myVar[2] = 31.991174 * myVar[threadIdx.x];
	var_86_0[0] = myVar[0];
	var_86_1[1] = myVar[1];
	var_86_2[2] = myVar[2];
	var_86_3[3] = myVar[3];
	var_86_4[4] = myVar[4];
	var_86_5[5] = myVar[5];
	var_86_6[6] = myVar[6];
	var_86_7[7] = myVar[7];
	var_86_8[8] = myVar[8];
	var_86_9[9] = myVar[9];
	var_86_10[10] = myVar[10];
	var_86_11[11] = myVar[11];
	var_86_12[12] = myVar[12];
	var_86_13[13] = myVar[13];
	var_86_14[14] = myVar[14];
	var_86_15[15] = myVar[15];
	var_86_16[16] = myVar[16];
	var_86_17[17] = myVar[17];
	var_86_18[18] = myVar[18];
	var_86_19[19] = myVar[19];
	
}

__global__ void kernel_87(float * var_87_0, float * var_87_1, float * var_87_2, float * var_87_3, float * var_87_4, float * var_87_5, float * var_87_6, float * var_87_7, float * var_87_8, float * var_87_9, float * var_87_10, float * var_87_11, float * var_87_12, float * var_87_13, float * var_87_14, float * var_87_15, float * var_87_16, float * var_87_17, float * var_87_18, float * var_87_19) {
	__shared__ float myVar[1024];
	myVar[0] = 17.818198 * myVar[threadIdx.x];
	myVar[7] = 17.292375 * myVar[threadIdx.x];
	myVar[3] = 9.408437 * myVar[threadIdx.x];
	myVar[0] = 35.787921 * myVar[threadIdx.x];
	myVar[7] = 31.576850 * myVar[threadIdx.x];
	myVar[0] = 43.302253 * myVar[threadIdx.x];
	myVar[5] = 45.580790 * myVar[threadIdx.x];
	myVar[5] = 38.798559 * myVar[threadIdx.x];
	myVar[0] = 16.410893 * myVar[threadIdx.x];
	myVar[6] = 8.753178 * myVar[threadIdx.x];
	var_87_0[0] = myVar[0];
	var_87_1[1] = myVar[1];
	var_87_2[2] = myVar[2];
	var_87_3[3] = myVar[3];
	var_87_4[4] = myVar[4];
	var_87_5[5] = myVar[5];
	var_87_6[6] = myVar[6];
	var_87_7[7] = myVar[7];
	var_87_8[8] = myVar[8];
	var_87_9[9] = myVar[9];
	var_87_10[10] = myVar[10];
	var_87_11[11] = myVar[11];
	var_87_12[12] = myVar[12];
	var_87_13[13] = myVar[13];
	var_87_14[14] = myVar[14];
	var_87_15[15] = myVar[15];
	var_87_16[16] = myVar[16];
	var_87_17[17] = myVar[17];
	var_87_18[18] = myVar[18];
	var_87_19[19] = myVar[19];
	
}

__global__ void kernel_88(float * var_88_0, float * var_88_1, float * var_88_2, float * var_88_3, float * var_88_4, float * var_88_5, float * var_88_6, float * var_88_7, float * var_88_8, float * var_88_9, float * var_88_10, float * var_88_11, float * var_88_12, float * var_88_13, float * var_88_14, float * var_88_15, float * var_88_16, float * var_88_17, float * var_88_18, float * var_88_19) {
	__shared__ float myVar[1024];
	myVar[7] = 3.302447 * myVar[threadIdx.x];
	myVar[8] = 32.859062 * myVar[threadIdx.x];
	myVar[5] = 6.183310 * myVar[threadIdx.x];
	myVar[8] = 34.710643 * myVar[threadIdx.x];
	myVar[1] = 37.096195 * myVar[threadIdx.x];
	myVar[1] = 43.159629 * myVar[threadIdx.x];
	myVar[8] = 10.165780 * myVar[threadIdx.x];
	myVar[9] = 46.500045 * myVar[threadIdx.x];
	myVar[3] = 16.006992 * myVar[threadIdx.x];
	myVar[8] = 8.909091 * myVar[threadIdx.x];
	var_88_0[0] = myVar[0];
	var_88_1[1] = myVar[1];
	var_88_2[2] = myVar[2];
	var_88_3[3] = myVar[3];
	var_88_4[4] = myVar[4];
	var_88_5[5] = myVar[5];
	var_88_6[6] = myVar[6];
	var_88_7[7] = myVar[7];
	var_88_8[8] = myVar[8];
	var_88_9[9] = myVar[9];
	var_88_10[10] = myVar[10];
	var_88_11[11] = myVar[11];
	var_88_12[12] = myVar[12];
	var_88_13[13] = myVar[13];
	var_88_14[14] = myVar[14];
	var_88_15[15] = myVar[15];
	var_88_16[16] = myVar[16];
	var_88_17[17] = myVar[17];
	var_88_18[18] = myVar[18];
	var_88_19[19] = myVar[19];
	
}

__global__ void kernel_89(float * var_89_0, float * var_89_1, float * var_89_2, float * var_89_3, float * var_89_4, float * var_89_5, float * var_89_6, float * var_89_7, float * var_89_8, float * var_89_9, float * var_89_10, float * var_89_11, float * var_89_12, float * var_89_13, float * var_89_14, float * var_89_15, float * var_89_16, float * var_89_17, float * var_89_18, float * var_89_19) {
	__shared__ float myVar[1024];
	myVar[2] = 8.884512 * myVar[threadIdx.x];
	myVar[2] = 26.835709 * myVar[threadIdx.x];
	myVar[9] = 24.787851 * myVar[threadIdx.x];
	myVar[9] = 11.447755 * myVar[threadIdx.x];
	myVar[5] = 46.979558 * myVar[threadIdx.x];
	myVar[5] = 10.651160 * myVar[threadIdx.x];
	myVar[7] = 3.243080 * myVar[threadIdx.x];
	myVar[2] = 31.164741 * myVar[threadIdx.x];
	myVar[5] = 36.848732 * myVar[threadIdx.x];
	myVar[8] = 10.391745 * myVar[threadIdx.x];
	var_89_0[0] = myVar[0];
	var_89_1[1] = myVar[1];
	var_89_2[2] = myVar[2];
	var_89_3[3] = myVar[3];
	var_89_4[4] = myVar[4];
	var_89_5[5] = myVar[5];
	var_89_6[6] = myVar[6];
	var_89_7[7] = myVar[7];
	var_89_8[8] = myVar[8];
	var_89_9[9] = myVar[9];
	var_89_10[10] = myVar[10];
	var_89_11[11] = myVar[11];
	var_89_12[12] = myVar[12];
	var_89_13[13] = myVar[13];
	var_89_14[14] = myVar[14];
	var_89_15[15] = myVar[15];
	var_89_16[16] = myVar[16];
	var_89_17[17] = myVar[17];
	var_89_18[18] = myVar[18];
	var_89_19[19] = myVar[19];
	
}

__global__ void kernel_90(float * var_90_0, float * var_90_1, float * var_90_2, float * var_90_3, float * var_90_4, float * var_90_5, float * var_90_6, float * var_90_7, float * var_90_8, float * var_90_9, float * var_90_10, float * var_90_11, float * var_90_12, float * var_90_13, float * var_90_14, float * var_90_15, float * var_90_16, float * var_90_17, float * var_90_18, float * var_90_19) {
	__shared__ float myVar[1024];
	myVar[8] = 20.418554 * myVar[threadIdx.x];
	myVar[9] = 8.802054 * myVar[threadIdx.x];
	myVar[7] = 43.815346 * myVar[threadIdx.x];
	myVar[4] = 45.765129 * myVar[threadIdx.x];
	myVar[6] = 8.567715 * myVar[threadIdx.x];
	myVar[0] = 27.280536 * myVar[threadIdx.x];
	myVar[3] = 44.621018 * myVar[threadIdx.x];
	myVar[8] = 40.014008 * myVar[threadIdx.x];
	myVar[6] = 7.637198 * myVar[threadIdx.x];
	myVar[4] = 10.862993 * myVar[threadIdx.x];
	var_90_0[0] = myVar[0];
	var_90_1[1] = myVar[1];
	var_90_2[2] = myVar[2];
	var_90_3[3] = myVar[3];
	var_90_4[4] = myVar[4];
	var_90_5[5] = myVar[5];
	var_90_6[6] = myVar[6];
	var_90_7[7] = myVar[7];
	var_90_8[8] = myVar[8];
	var_90_9[9] = myVar[9];
	var_90_10[10] = myVar[10];
	var_90_11[11] = myVar[11];
	var_90_12[12] = myVar[12];
	var_90_13[13] = myVar[13];
	var_90_14[14] = myVar[14];
	var_90_15[15] = myVar[15];
	var_90_16[16] = myVar[16];
	var_90_17[17] = myVar[17];
	var_90_18[18] = myVar[18];
	var_90_19[19] = myVar[19];
	
}

__global__ void kernel_91(float * var_91_0, float * var_91_1, float * var_91_2, float * var_91_3, float * var_91_4, float * var_91_5, float * var_91_6, float * var_91_7, float * var_91_8, float * var_91_9, float * var_91_10, float * var_91_11, float * var_91_12, float * var_91_13, float * var_91_14, float * var_91_15, float * var_91_16, float * var_91_17, float * var_91_18, float * var_91_19) {
	__shared__ float myVar[1024];
	myVar[5] = 10.636806 * myVar[threadIdx.x];
	myVar[6] = 25.575829 * myVar[threadIdx.x];
	myVar[7] = 26.068188 * myVar[threadIdx.x];
	myVar[4] = 48.151045 * myVar[threadIdx.x];
	myVar[0] = 4.807278 * myVar[threadIdx.x];
	myVar[6] = 14.784279 * myVar[threadIdx.x];
	myVar[7] = 49.478457 * myVar[threadIdx.x];
	myVar[6] = 5.982118 * myVar[threadIdx.x];
	myVar[7] = 0.298905 * myVar[threadIdx.x];
	myVar[6] = 33.926429 * myVar[threadIdx.x];
	var_91_0[0] = myVar[0];
	var_91_1[1] = myVar[1];
	var_91_2[2] = myVar[2];
	var_91_3[3] = myVar[3];
	var_91_4[4] = myVar[4];
	var_91_5[5] = myVar[5];
	var_91_6[6] = myVar[6];
	var_91_7[7] = myVar[7];
	var_91_8[8] = myVar[8];
	var_91_9[9] = myVar[9];
	var_91_10[10] = myVar[10];
	var_91_11[11] = myVar[11];
	var_91_12[12] = myVar[12];
	var_91_13[13] = myVar[13];
	var_91_14[14] = myVar[14];
	var_91_15[15] = myVar[15];
	var_91_16[16] = myVar[16];
	var_91_17[17] = myVar[17];
	var_91_18[18] = myVar[18];
	var_91_19[19] = myVar[19];
	
}

__global__ void kernel_92(float * var_92_0, float * var_92_1, float * var_92_2, float * var_92_3, float * var_92_4, float * var_92_5, float * var_92_6, float * var_92_7, float * var_92_8, float * var_92_9, float * var_92_10, float * var_92_11, float * var_92_12, float * var_92_13, float * var_92_14, float * var_92_15, float * var_92_16, float * var_92_17, float * var_92_18, float * var_92_19) {
	__shared__ float myVar[1024];
	myVar[1] = 27.983791 * myVar[threadIdx.x];
	myVar[4] = 48.063810 * myVar[threadIdx.x];
	myVar[4] = 3.419796 * myVar[threadIdx.x];
	myVar[9] = 3.814167 * myVar[threadIdx.x];
	myVar[6] = 49.538755 * myVar[threadIdx.x];
	myVar[8] = 14.594070 * myVar[threadIdx.x];
	myVar[8] = 0.899012 * myVar[threadIdx.x];
	myVar[0] = 28.691193 * myVar[threadIdx.x];
	myVar[2] = 22.566337 * myVar[threadIdx.x];
	myVar[6] = 26.554205 * myVar[threadIdx.x];
	var_92_0[0] = myVar[0];
	var_92_1[1] = myVar[1];
	var_92_2[2] = myVar[2];
	var_92_3[3] = myVar[3];
	var_92_4[4] = myVar[4];
	var_92_5[5] = myVar[5];
	var_92_6[6] = myVar[6];
	var_92_7[7] = myVar[7];
	var_92_8[8] = myVar[8];
	var_92_9[9] = myVar[9];
	var_92_10[10] = myVar[10];
	var_92_11[11] = myVar[11];
	var_92_12[12] = myVar[12];
	var_92_13[13] = myVar[13];
	var_92_14[14] = myVar[14];
	var_92_15[15] = myVar[15];
	var_92_16[16] = myVar[16];
	var_92_17[17] = myVar[17];
	var_92_18[18] = myVar[18];
	var_92_19[19] = myVar[19];
	
}

__global__ void kernel_93(float * var_93_0, float * var_93_1, float * var_93_2, float * var_93_3, float * var_93_4, float * var_93_5, float * var_93_6, float * var_93_7, float * var_93_8, float * var_93_9, float * var_93_10, float * var_93_11, float * var_93_12, float * var_93_13, float * var_93_14, float * var_93_15, float * var_93_16, float * var_93_17, float * var_93_18, float * var_93_19) {
	__shared__ float myVar[1024];
	myVar[7] = 25.247132 * myVar[threadIdx.x];
	myVar[7] = 35.564602 * myVar[threadIdx.x];
	myVar[1] = 19.893224 * myVar[threadIdx.x];
	myVar[4] = 38.999653 * myVar[threadIdx.x];
	myVar[3] = 32.827499 * myVar[threadIdx.x];
	myVar[8] = 37.567465 * myVar[threadIdx.x];
	myVar[2] = 32.180819 * myVar[threadIdx.x];
	myVar[5] = 29.349211 * myVar[threadIdx.x];
	myVar[6] = 47.392912 * myVar[threadIdx.x];
	myVar[7] = 23.877738 * myVar[threadIdx.x];
	var_93_0[0] = myVar[0];
	var_93_1[1] = myVar[1];
	var_93_2[2] = myVar[2];
	var_93_3[3] = myVar[3];
	var_93_4[4] = myVar[4];
	var_93_5[5] = myVar[5];
	var_93_6[6] = myVar[6];
	var_93_7[7] = myVar[7];
	var_93_8[8] = myVar[8];
	var_93_9[9] = myVar[9];
	var_93_10[10] = myVar[10];
	var_93_11[11] = myVar[11];
	var_93_12[12] = myVar[12];
	var_93_13[13] = myVar[13];
	var_93_14[14] = myVar[14];
	var_93_15[15] = myVar[15];
	var_93_16[16] = myVar[16];
	var_93_17[17] = myVar[17];
	var_93_18[18] = myVar[18];
	var_93_19[19] = myVar[19];
	
}

__global__ void kernel_94(float * var_94_0, float * var_94_1, float * var_94_2, float * var_94_3, float * var_94_4, float * var_94_5, float * var_94_6, float * var_94_7, float * var_94_8, float * var_94_9, float * var_94_10, float * var_94_11, float * var_94_12, float * var_94_13, float * var_94_14, float * var_94_15, float * var_94_16, float * var_94_17, float * var_94_18, float * var_94_19) {
	__shared__ float myVar[1024];
	myVar[9] = 22.970190 * myVar[threadIdx.x];
	myVar[9] = 0.697030 * myVar[threadIdx.x];
	myVar[2] = 27.841112 * myVar[threadIdx.x];
	myVar[8] = 8.810656 * myVar[threadIdx.x];
	myVar[3] = 17.252632 * myVar[threadIdx.x];
	myVar[2] = 15.302823 * myVar[threadIdx.x];
	myVar[7] = 43.331670 * myVar[threadIdx.x];
	myVar[6] = 36.704199 * myVar[threadIdx.x];
	myVar[2] = 30.115334 * myVar[threadIdx.x];
	myVar[3] = 2.212627 * myVar[threadIdx.x];
	var_94_0[0] = myVar[0];
	var_94_1[1] = myVar[1];
	var_94_2[2] = myVar[2];
	var_94_3[3] = myVar[3];
	var_94_4[4] = myVar[4];
	var_94_5[5] = myVar[5];
	var_94_6[6] = myVar[6];
	var_94_7[7] = myVar[7];
	var_94_8[8] = myVar[8];
	var_94_9[9] = myVar[9];
	var_94_10[10] = myVar[10];
	var_94_11[11] = myVar[11];
	var_94_12[12] = myVar[12];
	var_94_13[13] = myVar[13];
	var_94_14[14] = myVar[14];
	var_94_15[15] = myVar[15];
	var_94_16[16] = myVar[16];
	var_94_17[17] = myVar[17];
	var_94_18[18] = myVar[18];
	var_94_19[19] = myVar[19];
	
}

__global__ void kernel_95(float * var_95_0, float * var_95_1, float * var_95_2, float * var_95_3, float * var_95_4, float * var_95_5, float * var_95_6, float * var_95_7, float * var_95_8, float * var_95_9, float * var_95_10, float * var_95_11, float * var_95_12, float * var_95_13, float * var_95_14, float * var_95_15, float * var_95_16, float * var_95_17, float * var_95_18, float * var_95_19) {
	__shared__ float myVar[1024];
	myVar[0] = 24.147069 * myVar[threadIdx.x];
	myVar[8] = 10.496619 * myVar[threadIdx.x];
	myVar[0] = 36.443158 * myVar[threadIdx.x];
	myVar[0] = 17.904213 * myVar[threadIdx.x];
	myVar[1] = 49.766844 * myVar[threadIdx.x];
	myVar[9] = 17.379044 * myVar[threadIdx.x];
	myVar[6] = 32.826730 * myVar[threadIdx.x];
	myVar[4] = 44.411972 * myVar[threadIdx.x];
	myVar[5] = 26.416494 * myVar[threadIdx.x];
	myVar[5] = 21.628386 * myVar[threadIdx.x];
	var_95_0[0] = myVar[0];
	var_95_1[1] = myVar[1];
	var_95_2[2] = myVar[2];
	var_95_3[3] = myVar[3];
	var_95_4[4] = myVar[4];
	var_95_5[5] = myVar[5];
	var_95_6[6] = myVar[6];
	var_95_7[7] = myVar[7];
	var_95_8[8] = myVar[8];
	var_95_9[9] = myVar[9];
	var_95_10[10] = myVar[10];
	var_95_11[11] = myVar[11];
	var_95_12[12] = myVar[12];
	var_95_13[13] = myVar[13];
	var_95_14[14] = myVar[14];
	var_95_15[15] = myVar[15];
	var_95_16[16] = myVar[16];
	var_95_17[17] = myVar[17];
	var_95_18[18] = myVar[18];
	var_95_19[19] = myVar[19];
	
}

__global__ void kernel_96(float * var_96_0, float * var_96_1, float * var_96_2, float * var_96_3, float * var_96_4, float * var_96_5, float * var_96_6, float * var_96_7, float * var_96_8, float * var_96_9, float * var_96_10, float * var_96_11, float * var_96_12, float * var_96_13, float * var_96_14, float * var_96_15, float * var_96_16, float * var_96_17, float * var_96_18, float * var_96_19) {
	__shared__ float myVar[1024];
	myVar[8] = 13.596222 * myVar[threadIdx.x];
	myVar[9] = 35.570629 * myVar[threadIdx.x];
	myVar[3] = 6.256915 * myVar[threadIdx.x];
	myVar[1] = 35.742467 * myVar[threadIdx.x];
	myVar[0] = 21.909653 * myVar[threadIdx.x];
	myVar[9] = 23.225823 * myVar[threadIdx.x];
	myVar[9] = 41.822623 * myVar[threadIdx.x];
	myVar[4] = 49.545857 * myVar[threadIdx.x];
	myVar[6] = 14.578220 * myVar[threadIdx.x];
	myVar[5] = 26.371621 * myVar[threadIdx.x];
	var_96_0[0] = myVar[0];
	var_96_1[1] = myVar[1];
	var_96_2[2] = myVar[2];
	var_96_3[3] = myVar[3];
	var_96_4[4] = myVar[4];
	var_96_5[5] = myVar[5];
	var_96_6[6] = myVar[6];
	var_96_7[7] = myVar[7];
	var_96_8[8] = myVar[8];
	var_96_9[9] = myVar[9];
	var_96_10[10] = myVar[10];
	var_96_11[11] = myVar[11];
	var_96_12[12] = myVar[12];
	var_96_13[13] = myVar[13];
	var_96_14[14] = myVar[14];
	var_96_15[15] = myVar[15];
	var_96_16[16] = myVar[16];
	var_96_17[17] = myVar[17];
	var_96_18[18] = myVar[18];
	var_96_19[19] = myVar[19];
	
}

__global__ void kernel_97(float * var_97_0, float * var_97_1, float * var_97_2, float * var_97_3, float * var_97_4, float * var_97_5, float * var_97_6, float * var_97_7, float * var_97_8, float * var_97_9, float * var_97_10, float * var_97_11, float * var_97_12, float * var_97_13, float * var_97_14, float * var_97_15, float * var_97_16, float * var_97_17, float * var_97_18, float * var_97_19) {
	__shared__ float myVar[1024];
	myVar[3] = 7.616703 * myVar[threadIdx.x];
	myVar[3] = 24.963788 * myVar[threadIdx.x];
	myVar[9] = 32.178852 * myVar[threadIdx.x];
	myVar[8] = 14.660428 * myVar[threadIdx.x];
	myVar[1] = 10.423802 * myVar[threadIdx.x];
	myVar[5] = 34.645808 * myVar[threadIdx.x];
	myVar[8] = 7.513570 * myVar[threadIdx.x];
	myVar[5] = 19.526371 * myVar[threadIdx.x];
	myVar[5] = 8.128375 * myVar[threadIdx.x];
	myVar[7] = 27.257317 * myVar[threadIdx.x];
	var_97_0[0] = myVar[0];
	var_97_1[1] = myVar[1];
	var_97_2[2] = myVar[2];
	var_97_3[3] = myVar[3];
	var_97_4[4] = myVar[4];
	var_97_5[5] = myVar[5];
	var_97_6[6] = myVar[6];
	var_97_7[7] = myVar[7];
	var_97_8[8] = myVar[8];
	var_97_9[9] = myVar[9];
	var_97_10[10] = myVar[10];
	var_97_11[11] = myVar[11];
	var_97_12[12] = myVar[12];
	var_97_13[13] = myVar[13];
	var_97_14[14] = myVar[14];
	var_97_15[15] = myVar[15];
	var_97_16[16] = myVar[16];
	var_97_17[17] = myVar[17];
	var_97_18[18] = myVar[18];
	var_97_19[19] = myVar[19];
	
}

__global__ void kernel_98(float * var_98_0, float * var_98_1, float * var_98_2, float * var_98_3, float * var_98_4, float * var_98_5, float * var_98_6, float * var_98_7, float * var_98_8, float * var_98_9, float * var_98_10, float * var_98_11, float * var_98_12, float * var_98_13, float * var_98_14, float * var_98_15, float * var_98_16, float * var_98_17, float * var_98_18, float * var_98_19) {
	__shared__ float myVar[1024];
	myVar[5] = 22.933224 * myVar[threadIdx.x];
	myVar[6] = 33.404707 * myVar[threadIdx.x];
	myVar[1] = 12.560293 * myVar[threadIdx.x];
	myVar[6] = 12.533825 * myVar[threadIdx.x];
	myVar[4] = 8.005213 * myVar[threadIdx.x];
	myVar[2] = 37.231474 * myVar[threadIdx.x];
	myVar[1] = 48.695888 * myVar[threadIdx.x];
	myVar[5] = 43.870667 * myVar[threadIdx.x];
	myVar[1] = 5.068447 * myVar[threadIdx.x];
	myVar[4] = 31.245133 * myVar[threadIdx.x];
	var_98_0[0] = myVar[0];
	var_98_1[1] = myVar[1];
	var_98_2[2] = myVar[2];
	var_98_3[3] = myVar[3];
	var_98_4[4] = myVar[4];
	var_98_5[5] = myVar[5];
	var_98_6[6] = myVar[6];
	var_98_7[7] = myVar[7];
	var_98_8[8] = myVar[8];
	var_98_9[9] = myVar[9];
	var_98_10[10] = myVar[10];
	var_98_11[11] = myVar[11];
	var_98_12[12] = myVar[12];
	var_98_13[13] = myVar[13];
	var_98_14[14] = myVar[14];
	var_98_15[15] = myVar[15];
	var_98_16[16] = myVar[16];
	var_98_17[17] = myVar[17];
	var_98_18[18] = myVar[18];
	var_98_19[19] = myVar[19];
	
}

__global__ void kernel_99(float * var_99_0, float * var_99_1, float * var_99_2, float * var_99_3, float * var_99_4, float * var_99_5, float * var_99_6, float * var_99_7, float * var_99_8, float * var_99_9, float * var_99_10, float * var_99_11, float * var_99_12, float * var_99_13, float * var_99_14, float * var_99_15, float * var_99_16, float * var_99_17, float * var_99_18, float * var_99_19) {
	__shared__ float myVar[1024];
	myVar[6] = 27.205712 * myVar[threadIdx.x];
	myVar[2] = 3.981201 * myVar[threadIdx.x];
	myVar[5] = 37.854242 * myVar[threadIdx.x];
	myVar[8] = 5.116412 * myVar[threadIdx.x];
	myVar[5] = 13.977419 * myVar[threadIdx.x];
	myVar[0] = 40.107187 * myVar[threadIdx.x];
	myVar[0] = 18.660288 * myVar[threadIdx.x];
	myVar[7] = 18.056329 * myVar[threadIdx.x];
	myVar[6] = 12.940238 * myVar[threadIdx.x];
	myVar[8] = 33.224260 * myVar[threadIdx.x];
	var_99_0[0] = myVar[0];
	var_99_1[1] = myVar[1];
	var_99_2[2] = myVar[2];
	var_99_3[3] = myVar[3];
	var_99_4[4] = myVar[4];
	var_99_5[5] = myVar[5];
	var_99_6[6] = myVar[6];
	var_99_7[7] = myVar[7];
	var_99_8[8] = myVar[8];
	var_99_9[9] = myVar[9];
	var_99_10[10] = myVar[10];
	var_99_11[11] = myVar[11];
	var_99_12[12] = myVar[12];
	var_99_13[13] = myVar[13];
	var_99_14[14] = myVar[14];
	var_99_15[15] = myVar[15];
	var_99_16[16] = myVar[16];
	var_99_17[17] = myVar[17];
	var_99_18[18] = myVar[18];
	var_99_19[19] = myVar[19];
	
}


int main(void) {
	
	float * h_var_0_0 = (float *)malloc(sizeof(float *));
	float * d_var_0_0;
	hipMalloc((void **)&d_var_0_0, sizeof(float *));
	
	float * h_var_0_1 = (float *)malloc(sizeof(float *));
	float * d_var_0_1;
	hipMalloc((void **)&d_var_0_1, sizeof(float *));
	
	float * h_var_0_2 = (float *)malloc(sizeof(float *));
	float * d_var_0_2;
	hipMalloc((void **)&d_var_0_2, sizeof(float *));
	
	float * h_var_0_3 = (float *)malloc(sizeof(float *));
	float * d_var_0_3;
	hipMalloc((void **)&d_var_0_3, sizeof(float *));
	
	float * h_var_0_4 = (float *)malloc(sizeof(float *));
	float * d_var_0_4;
	hipMalloc((void **)&d_var_0_4, sizeof(float *));
	
	float * h_var_0_5 = (float *)malloc(sizeof(float *));
	float * d_var_0_5;
	hipMalloc((void **)&d_var_0_5, sizeof(float *));
	
	float * h_var_0_6 = (float *)malloc(sizeof(float *));
	float * d_var_0_6;
	hipMalloc((void **)&d_var_0_6, sizeof(float *));
	
	float * h_var_0_7 = (float *)malloc(sizeof(float *));
	float * d_var_0_7;
	hipMalloc((void **)&d_var_0_7, sizeof(float *));
	
	float * h_var_0_8 = (float *)malloc(sizeof(float *));
	float * d_var_0_8;
	hipMalloc((void **)&d_var_0_8, sizeof(float *));
	
	float * h_var_0_9 = (float *)malloc(sizeof(float *));
	float * d_var_0_9;
	hipMalloc((void **)&d_var_0_9, sizeof(float *));
	
	float * h_var_0_10 = (float *)malloc(sizeof(float *));
	float * d_var_0_10;
	hipMalloc((void **)&d_var_0_10, sizeof(float *));
	
	float * h_var_0_11 = (float *)malloc(sizeof(float *));
	float * d_var_0_11;
	hipMalloc((void **)&d_var_0_11, sizeof(float *));
	
	float * h_var_0_12 = (float *)malloc(sizeof(float *));
	float * d_var_0_12;
	hipMalloc((void **)&d_var_0_12, sizeof(float *));
	
	float * h_var_0_13 = (float *)malloc(sizeof(float *));
	float * d_var_0_13;
	hipMalloc((void **)&d_var_0_13, sizeof(float *));
	
	float * h_var_0_14 = (float *)malloc(sizeof(float *));
	float * d_var_0_14;
	hipMalloc((void **)&d_var_0_14, sizeof(float *));
	
	float * h_var_0_15 = (float *)malloc(sizeof(float *));
	float * d_var_0_15;
	hipMalloc((void **)&d_var_0_15, sizeof(float *));
	
	float * h_var_0_16 = (float *)malloc(sizeof(float *));
	float * d_var_0_16;
	hipMalloc((void **)&d_var_0_16, sizeof(float *));
	
	float * h_var_0_17 = (float *)malloc(sizeof(float *));
	float * d_var_0_17;
	hipMalloc((void **)&d_var_0_17, sizeof(float *));
	
	float * h_var_0_18 = (float *)malloc(sizeof(float *));
	float * d_var_0_18;
	hipMalloc((void **)&d_var_0_18, sizeof(float *));
	
	float * h_var_0_19 = (float *)malloc(sizeof(float *));
	float * d_var_0_19;
	hipMalloc((void **)&d_var_0_19, sizeof(float *));
	
	float * h_var_1_0 = (float *)malloc(sizeof(float *));
	float * d_var_1_0;
	hipMalloc((void **)&d_var_1_0, sizeof(float *));
	
	float * h_var_1_1 = (float *)malloc(sizeof(float *));
	float * d_var_1_1;
	hipMalloc((void **)&d_var_1_1, sizeof(float *));
	
	float * h_var_1_2 = (float *)malloc(sizeof(float *));
	float * d_var_1_2;
	hipMalloc((void **)&d_var_1_2, sizeof(float *));
	
	float * h_var_1_3 = (float *)malloc(sizeof(float *));
	float * d_var_1_3;
	hipMalloc((void **)&d_var_1_3, sizeof(float *));
	
	float * h_var_1_4 = (float *)malloc(sizeof(float *));
	float * d_var_1_4;
	hipMalloc((void **)&d_var_1_4, sizeof(float *));
	
	float * h_var_1_5 = (float *)malloc(sizeof(float *));
	float * d_var_1_5;
	hipMalloc((void **)&d_var_1_5, sizeof(float *));
	
	float * h_var_1_6 = (float *)malloc(sizeof(float *));
	float * d_var_1_6;
	hipMalloc((void **)&d_var_1_6, sizeof(float *));
	
	float * h_var_1_7 = (float *)malloc(sizeof(float *));
	float * d_var_1_7;
	hipMalloc((void **)&d_var_1_7, sizeof(float *));
	
	float * h_var_1_8 = (float *)malloc(sizeof(float *));
	float * d_var_1_8;
	hipMalloc((void **)&d_var_1_8, sizeof(float *));
	
	float * h_var_1_9 = (float *)malloc(sizeof(float *));
	float * d_var_1_9;
	hipMalloc((void **)&d_var_1_9, sizeof(float *));
	
	float * h_var_1_10 = (float *)malloc(sizeof(float *));
	float * d_var_1_10;
	hipMalloc((void **)&d_var_1_10, sizeof(float *));
	
	float * h_var_1_11 = (float *)malloc(sizeof(float *));
	float * d_var_1_11;
	hipMalloc((void **)&d_var_1_11, sizeof(float *));
	
	float * h_var_1_12 = (float *)malloc(sizeof(float *));
	float * d_var_1_12;
	hipMalloc((void **)&d_var_1_12, sizeof(float *));
	
	float * h_var_1_13 = (float *)malloc(sizeof(float *));
	float * d_var_1_13;
	hipMalloc((void **)&d_var_1_13, sizeof(float *));
	
	float * h_var_1_14 = (float *)malloc(sizeof(float *));
	float * d_var_1_14;
	hipMalloc((void **)&d_var_1_14, sizeof(float *));
	
	float * h_var_1_15 = (float *)malloc(sizeof(float *));
	float * d_var_1_15;
	hipMalloc((void **)&d_var_1_15, sizeof(float *));
	
	float * h_var_1_16 = (float *)malloc(sizeof(float *));
	float * d_var_1_16;
	hipMalloc((void **)&d_var_1_16, sizeof(float *));
	
	float * h_var_1_17 = (float *)malloc(sizeof(float *));
	float * d_var_1_17;
	hipMalloc((void **)&d_var_1_17, sizeof(float *));
	
	float * h_var_1_18 = (float *)malloc(sizeof(float *));
	float * d_var_1_18;
	hipMalloc((void **)&d_var_1_18, sizeof(float *));
	
	float * h_var_1_19 = (float *)malloc(sizeof(float *));
	float * d_var_1_19;
	hipMalloc((void **)&d_var_1_19, sizeof(float *));
	
	float * h_var_2_0 = (float *)malloc(sizeof(float *));
	float * d_var_2_0;
	hipMalloc((void **)&d_var_2_0, sizeof(float *));
	
	float * h_var_2_1 = (float *)malloc(sizeof(float *));
	float * d_var_2_1;
	hipMalloc((void **)&d_var_2_1, sizeof(float *));
	
	float * h_var_2_2 = (float *)malloc(sizeof(float *));
	float * d_var_2_2;
	hipMalloc((void **)&d_var_2_2, sizeof(float *));
	
	float * h_var_2_3 = (float *)malloc(sizeof(float *));
	float * d_var_2_3;
	hipMalloc((void **)&d_var_2_3, sizeof(float *));
	
	float * h_var_2_4 = (float *)malloc(sizeof(float *));
	float * d_var_2_4;
	hipMalloc((void **)&d_var_2_4, sizeof(float *));
	
	float * h_var_2_5 = (float *)malloc(sizeof(float *));
	float * d_var_2_5;
	hipMalloc((void **)&d_var_2_5, sizeof(float *));
	
	float * h_var_2_6 = (float *)malloc(sizeof(float *));
	float * d_var_2_6;
	hipMalloc((void **)&d_var_2_6, sizeof(float *));
	
	float * h_var_2_7 = (float *)malloc(sizeof(float *));
	float * d_var_2_7;
	hipMalloc((void **)&d_var_2_7, sizeof(float *));
	
	float * h_var_2_8 = (float *)malloc(sizeof(float *));
	float * d_var_2_8;
	hipMalloc((void **)&d_var_2_8, sizeof(float *));
	
	float * h_var_2_9 = (float *)malloc(sizeof(float *));
	float * d_var_2_9;
	hipMalloc((void **)&d_var_2_9, sizeof(float *));
	
	float * h_var_2_10 = (float *)malloc(sizeof(float *));
	float * d_var_2_10;
	hipMalloc((void **)&d_var_2_10, sizeof(float *));
	
	float * h_var_2_11 = (float *)malloc(sizeof(float *));
	float * d_var_2_11;
	hipMalloc((void **)&d_var_2_11, sizeof(float *));
	
	float * h_var_2_12 = (float *)malloc(sizeof(float *));
	float * d_var_2_12;
	hipMalloc((void **)&d_var_2_12, sizeof(float *));
	
	float * h_var_2_13 = (float *)malloc(sizeof(float *));
	float * d_var_2_13;
	hipMalloc((void **)&d_var_2_13, sizeof(float *));
	
	float * h_var_2_14 = (float *)malloc(sizeof(float *));
	float * d_var_2_14;
	hipMalloc((void **)&d_var_2_14, sizeof(float *));
	
	float * h_var_2_15 = (float *)malloc(sizeof(float *));
	float * d_var_2_15;
	hipMalloc((void **)&d_var_2_15, sizeof(float *));
	
	float * h_var_2_16 = (float *)malloc(sizeof(float *));
	float * d_var_2_16;
	hipMalloc((void **)&d_var_2_16, sizeof(float *));
	
	float * h_var_2_17 = (float *)malloc(sizeof(float *));
	float * d_var_2_17;
	hipMalloc((void **)&d_var_2_17, sizeof(float *));
	
	float * h_var_2_18 = (float *)malloc(sizeof(float *));
	float * d_var_2_18;
	hipMalloc((void **)&d_var_2_18, sizeof(float *));
	
	float * h_var_2_19 = (float *)malloc(sizeof(float *));
	float * d_var_2_19;
	hipMalloc((void **)&d_var_2_19, sizeof(float *));
	
	float * h_var_3_0 = (float *)malloc(sizeof(float *));
	float * d_var_3_0;
	hipMalloc((void **)&d_var_3_0, sizeof(float *));
	
	float * h_var_3_1 = (float *)malloc(sizeof(float *));
	float * d_var_3_1;
	hipMalloc((void **)&d_var_3_1, sizeof(float *));
	
	float * h_var_3_2 = (float *)malloc(sizeof(float *));
	float * d_var_3_2;
	hipMalloc((void **)&d_var_3_2, sizeof(float *));
	
	float * h_var_3_3 = (float *)malloc(sizeof(float *));
	float * d_var_3_3;
	hipMalloc((void **)&d_var_3_3, sizeof(float *));
	
	float * h_var_3_4 = (float *)malloc(sizeof(float *));
	float * d_var_3_4;
	hipMalloc((void **)&d_var_3_4, sizeof(float *));
	
	float * h_var_3_5 = (float *)malloc(sizeof(float *));
	float * d_var_3_5;
	hipMalloc((void **)&d_var_3_5, sizeof(float *));
	
	float * h_var_3_6 = (float *)malloc(sizeof(float *));
	float * d_var_3_6;
	hipMalloc((void **)&d_var_3_6, sizeof(float *));
	
	float * h_var_3_7 = (float *)malloc(sizeof(float *));
	float * d_var_3_7;
	hipMalloc((void **)&d_var_3_7, sizeof(float *));
	
	float * h_var_3_8 = (float *)malloc(sizeof(float *));
	float * d_var_3_8;
	hipMalloc((void **)&d_var_3_8, sizeof(float *));
	
	float * h_var_3_9 = (float *)malloc(sizeof(float *));
	float * d_var_3_9;
	hipMalloc((void **)&d_var_3_9, sizeof(float *));
	
	float * h_var_3_10 = (float *)malloc(sizeof(float *));
	float * d_var_3_10;
	hipMalloc((void **)&d_var_3_10, sizeof(float *));
	
	float * h_var_3_11 = (float *)malloc(sizeof(float *));
	float * d_var_3_11;
	hipMalloc((void **)&d_var_3_11, sizeof(float *));
	
	float * h_var_3_12 = (float *)malloc(sizeof(float *));
	float * d_var_3_12;
	hipMalloc((void **)&d_var_3_12, sizeof(float *));
	
	float * h_var_3_13 = (float *)malloc(sizeof(float *));
	float * d_var_3_13;
	hipMalloc((void **)&d_var_3_13, sizeof(float *));
	
	float * h_var_3_14 = (float *)malloc(sizeof(float *));
	float * d_var_3_14;
	hipMalloc((void **)&d_var_3_14, sizeof(float *));
	
	float * h_var_3_15 = (float *)malloc(sizeof(float *));
	float * d_var_3_15;
	hipMalloc((void **)&d_var_3_15, sizeof(float *));
	
	float * h_var_3_16 = (float *)malloc(sizeof(float *));
	float * d_var_3_16;
	hipMalloc((void **)&d_var_3_16, sizeof(float *));
	
	float * h_var_3_17 = (float *)malloc(sizeof(float *));
	float * d_var_3_17;
	hipMalloc((void **)&d_var_3_17, sizeof(float *));
	
	float * h_var_3_18 = (float *)malloc(sizeof(float *));
	float * d_var_3_18;
	hipMalloc((void **)&d_var_3_18, sizeof(float *));
	
	float * h_var_3_19 = (float *)malloc(sizeof(float *));
	float * d_var_3_19;
	hipMalloc((void **)&d_var_3_19, sizeof(float *));
	
	float * h_var_4_0 = (float *)malloc(sizeof(float *));
	float * d_var_4_0;
	hipMalloc((void **)&d_var_4_0, sizeof(float *));
	
	float * h_var_4_1 = (float *)malloc(sizeof(float *));
	float * d_var_4_1;
	hipMalloc((void **)&d_var_4_1, sizeof(float *));
	
	float * h_var_4_2 = (float *)malloc(sizeof(float *));
	float * d_var_4_2;
	hipMalloc((void **)&d_var_4_2, sizeof(float *));
	
	float * h_var_4_3 = (float *)malloc(sizeof(float *));
	float * d_var_4_3;
	hipMalloc((void **)&d_var_4_3, sizeof(float *));
	
	float * h_var_4_4 = (float *)malloc(sizeof(float *));
	float * d_var_4_4;
	hipMalloc((void **)&d_var_4_4, sizeof(float *));
	
	float * h_var_4_5 = (float *)malloc(sizeof(float *));
	float * d_var_4_5;
	hipMalloc((void **)&d_var_4_5, sizeof(float *));
	
	float * h_var_4_6 = (float *)malloc(sizeof(float *));
	float * d_var_4_6;
	hipMalloc((void **)&d_var_4_6, sizeof(float *));
	
	float * h_var_4_7 = (float *)malloc(sizeof(float *));
	float * d_var_4_7;
	hipMalloc((void **)&d_var_4_7, sizeof(float *));
	
	float * h_var_4_8 = (float *)malloc(sizeof(float *));
	float * d_var_4_8;
	hipMalloc((void **)&d_var_4_8, sizeof(float *));
	
	float * h_var_4_9 = (float *)malloc(sizeof(float *));
	float * d_var_4_9;
	hipMalloc((void **)&d_var_4_9, sizeof(float *));
	
	float * h_var_4_10 = (float *)malloc(sizeof(float *));
	float * d_var_4_10;
	hipMalloc((void **)&d_var_4_10, sizeof(float *));
	
	float * h_var_4_11 = (float *)malloc(sizeof(float *));
	float * d_var_4_11;
	hipMalloc((void **)&d_var_4_11, sizeof(float *));
	
	float * h_var_4_12 = (float *)malloc(sizeof(float *));
	float * d_var_4_12;
	hipMalloc((void **)&d_var_4_12, sizeof(float *));
	
	float * h_var_4_13 = (float *)malloc(sizeof(float *));
	float * d_var_4_13;
	hipMalloc((void **)&d_var_4_13, sizeof(float *));
	
	float * h_var_4_14 = (float *)malloc(sizeof(float *));
	float * d_var_4_14;
	hipMalloc((void **)&d_var_4_14, sizeof(float *));
	
	float * h_var_4_15 = (float *)malloc(sizeof(float *));
	float * d_var_4_15;
	hipMalloc((void **)&d_var_4_15, sizeof(float *));
	
	float * h_var_4_16 = (float *)malloc(sizeof(float *));
	float * d_var_4_16;
	hipMalloc((void **)&d_var_4_16, sizeof(float *));
	
	float * h_var_4_17 = (float *)malloc(sizeof(float *));
	float * d_var_4_17;
	hipMalloc((void **)&d_var_4_17, sizeof(float *));
	
	float * h_var_4_18 = (float *)malloc(sizeof(float *));
	float * d_var_4_18;
	hipMalloc((void **)&d_var_4_18, sizeof(float *));
	
	float * h_var_4_19 = (float *)malloc(sizeof(float *));
	float * d_var_4_19;
	hipMalloc((void **)&d_var_4_19, sizeof(float *));
	
	float * h_var_5_0 = (float *)malloc(sizeof(float *));
	float * d_var_5_0;
	hipMalloc((void **)&d_var_5_0, sizeof(float *));
	
	float * h_var_5_1 = (float *)malloc(sizeof(float *));
	float * d_var_5_1;
	hipMalloc((void **)&d_var_5_1, sizeof(float *));
	
	float * h_var_5_2 = (float *)malloc(sizeof(float *));
	float * d_var_5_2;
	hipMalloc((void **)&d_var_5_2, sizeof(float *));
	
	float * h_var_5_3 = (float *)malloc(sizeof(float *));
	float * d_var_5_3;
	hipMalloc((void **)&d_var_5_3, sizeof(float *));
	
	float * h_var_5_4 = (float *)malloc(sizeof(float *));
	float * d_var_5_4;
	hipMalloc((void **)&d_var_5_4, sizeof(float *));
	
	float * h_var_5_5 = (float *)malloc(sizeof(float *));
	float * d_var_5_5;
	hipMalloc((void **)&d_var_5_5, sizeof(float *));
	
	float * h_var_5_6 = (float *)malloc(sizeof(float *));
	float * d_var_5_6;
	hipMalloc((void **)&d_var_5_6, sizeof(float *));
	
	float * h_var_5_7 = (float *)malloc(sizeof(float *));
	float * d_var_5_7;
	hipMalloc((void **)&d_var_5_7, sizeof(float *));
	
	float * h_var_5_8 = (float *)malloc(sizeof(float *));
	float * d_var_5_8;
	hipMalloc((void **)&d_var_5_8, sizeof(float *));
	
	float * h_var_5_9 = (float *)malloc(sizeof(float *));
	float * d_var_5_9;
	hipMalloc((void **)&d_var_5_9, sizeof(float *));
	
	float * h_var_5_10 = (float *)malloc(sizeof(float *));
	float * d_var_5_10;
	hipMalloc((void **)&d_var_5_10, sizeof(float *));
	
	float * h_var_5_11 = (float *)malloc(sizeof(float *));
	float * d_var_5_11;
	hipMalloc((void **)&d_var_5_11, sizeof(float *));
	
	float * h_var_5_12 = (float *)malloc(sizeof(float *));
	float * d_var_5_12;
	hipMalloc((void **)&d_var_5_12, sizeof(float *));
	
	float * h_var_5_13 = (float *)malloc(sizeof(float *));
	float * d_var_5_13;
	hipMalloc((void **)&d_var_5_13, sizeof(float *));
	
	float * h_var_5_14 = (float *)malloc(sizeof(float *));
	float * d_var_5_14;
	hipMalloc((void **)&d_var_5_14, sizeof(float *));
	
	float * h_var_5_15 = (float *)malloc(sizeof(float *));
	float * d_var_5_15;
	hipMalloc((void **)&d_var_5_15, sizeof(float *));
	
	float * h_var_5_16 = (float *)malloc(sizeof(float *));
	float * d_var_5_16;
	hipMalloc((void **)&d_var_5_16, sizeof(float *));
	
	float * h_var_5_17 = (float *)malloc(sizeof(float *));
	float * d_var_5_17;
	hipMalloc((void **)&d_var_5_17, sizeof(float *));
	
	float * h_var_5_18 = (float *)malloc(sizeof(float *));
	float * d_var_5_18;
	hipMalloc((void **)&d_var_5_18, sizeof(float *));
	
	float * h_var_5_19 = (float *)malloc(sizeof(float *));
	float * d_var_5_19;
	hipMalloc((void **)&d_var_5_19, sizeof(float *));
	
	float * h_var_6_0 = (float *)malloc(sizeof(float *));
	float * d_var_6_0;
	hipMalloc((void **)&d_var_6_0, sizeof(float *));
	
	float * h_var_6_1 = (float *)malloc(sizeof(float *));
	float * d_var_6_1;
	hipMalloc((void **)&d_var_6_1, sizeof(float *));
	
	float * h_var_6_2 = (float *)malloc(sizeof(float *));
	float * d_var_6_2;
	hipMalloc((void **)&d_var_6_2, sizeof(float *));
	
	float * h_var_6_3 = (float *)malloc(sizeof(float *));
	float * d_var_6_3;
	hipMalloc((void **)&d_var_6_3, sizeof(float *));
	
	float * h_var_6_4 = (float *)malloc(sizeof(float *));
	float * d_var_6_4;
	hipMalloc((void **)&d_var_6_4, sizeof(float *));
	
	float * h_var_6_5 = (float *)malloc(sizeof(float *));
	float * d_var_6_5;
	hipMalloc((void **)&d_var_6_5, sizeof(float *));
	
	float * h_var_6_6 = (float *)malloc(sizeof(float *));
	float * d_var_6_6;
	hipMalloc((void **)&d_var_6_6, sizeof(float *));
	
	float * h_var_6_7 = (float *)malloc(sizeof(float *));
	float * d_var_6_7;
	hipMalloc((void **)&d_var_6_7, sizeof(float *));
	
	float * h_var_6_8 = (float *)malloc(sizeof(float *));
	float * d_var_6_8;
	hipMalloc((void **)&d_var_6_8, sizeof(float *));
	
	float * h_var_6_9 = (float *)malloc(sizeof(float *));
	float * d_var_6_9;
	hipMalloc((void **)&d_var_6_9, sizeof(float *));
	
	float * h_var_6_10 = (float *)malloc(sizeof(float *));
	float * d_var_6_10;
	hipMalloc((void **)&d_var_6_10, sizeof(float *));
	
	float * h_var_6_11 = (float *)malloc(sizeof(float *));
	float * d_var_6_11;
	hipMalloc((void **)&d_var_6_11, sizeof(float *));
	
	float * h_var_6_12 = (float *)malloc(sizeof(float *));
	float * d_var_6_12;
	hipMalloc((void **)&d_var_6_12, sizeof(float *));
	
	float * h_var_6_13 = (float *)malloc(sizeof(float *));
	float * d_var_6_13;
	hipMalloc((void **)&d_var_6_13, sizeof(float *));
	
	float * h_var_6_14 = (float *)malloc(sizeof(float *));
	float * d_var_6_14;
	hipMalloc((void **)&d_var_6_14, sizeof(float *));
	
	float * h_var_6_15 = (float *)malloc(sizeof(float *));
	float * d_var_6_15;
	hipMalloc((void **)&d_var_6_15, sizeof(float *));
	
	float * h_var_6_16 = (float *)malloc(sizeof(float *));
	float * d_var_6_16;
	hipMalloc((void **)&d_var_6_16, sizeof(float *));
	
	float * h_var_6_17 = (float *)malloc(sizeof(float *));
	float * d_var_6_17;
	hipMalloc((void **)&d_var_6_17, sizeof(float *));
	
	float * h_var_6_18 = (float *)malloc(sizeof(float *));
	float * d_var_6_18;
	hipMalloc((void **)&d_var_6_18, sizeof(float *));
	
	float * h_var_6_19 = (float *)malloc(sizeof(float *));
	float * d_var_6_19;
	hipMalloc((void **)&d_var_6_19, sizeof(float *));
	
	float * h_var_7_0 = (float *)malloc(sizeof(float *));
	float * d_var_7_0;
	hipMalloc((void **)&d_var_7_0, sizeof(float *));
	
	float * h_var_7_1 = (float *)malloc(sizeof(float *));
	float * d_var_7_1;
	hipMalloc((void **)&d_var_7_1, sizeof(float *));
	
	float * h_var_7_2 = (float *)malloc(sizeof(float *));
	float * d_var_7_2;
	hipMalloc((void **)&d_var_7_2, sizeof(float *));
	
	float * h_var_7_3 = (float *)malloc(sizeof(float *));
	float * d_var_7_3;
	hipMalloc((void **)&d_var_7_3, sizeof(float *));
	
	float * h_var_7_4 = (float *)malloc(sizeof(float *));
	float * d_var_7_4;
	hipMalloc((void **)&d_var_7_4, sizeof(float *));
	
	float * h_var_7_5 = (float *)malloc(sizeof(float *));
	float * d_var_7_5;
	hipMalloc((void **)&d_var_7_5, sizeof(float *));
	
	float * h_var_7_6 = (float *)malloc(sizeof(float *));
	float * d_var_7_6;
	hipMalloc((void **)&d_var_7_6, sizeof(float *));
	
	float * h_var_7_7 = (float *)malloc(sizeof(float *));
	float * d_var_7_7;
	hipMalloc((void **)&d_var_7_7, sizeof(float *));
	
	float * h_var_7_8 = (float *)malloc(sizeof(float *));
	float * d_var_7_8;
	hipMalloc((void **)&d_var_7_8, sizeof(float *));
	
	float * h_var_7_9 = (float *)malloc(sizeof(float *));
	float * d_var_7_9;
	hipMalloc((void **)&d_var_7_9, sizeof(float *));
	
	float * h_var_7_10 = (float *)malloc(sizeof(float *));
	float * d_var_7_10;
	hipMalloc((void **)&d_var_7_10, sizeof(float *));
	
	float * h_var_7_11 = (float *)malloc(sizeof(float *));
	float * d_var_7_11;
	hipMalloc((void **)&d_var_7_11, sizeof(float *));
	
	float * h_var_7_12 = (float *)malloc(sizeof(float *));
	float * d_var_7_12;
	hipMalloc((void **)&d_var_7_12, sizeof(float *));
	
	float * h_var_7_13 = (float *)malloc(sizeof(float *));
	float * d_var_7_13;
	hipMalloc((void **)&d_var_7_13, sizeof(float *));
	
	float * h_var_7_14 = (float *)malloc(sizeof(float *));
	float * d_var_7_14;
	hipMalloc((void **)&d_var_7_14, sizeof(float *));
	
	float * h_var_7_15 = (float *)malloc(sizeof(float *));
	float * d_var_7_15;
	hipMalloc((void **)&d_var_7_15, sizeof(float *));
	
	float * h_var_7_16 = (float *)malloc(sizeof(float *));
	float * d_var_7_16;
	hipMalloc((void **)&d_var_7_16, sizeof(float *));
	
	float * h_var_7_17 = (float *)malloc(sizeof(float *));
	float * d_var_7_17;
	hipMalloc((void **)&d_var_7_17, sizeof(float *));
	
	float * h_var_7_18 = (float *)malloc(sizeof(float *));
	float * d_var_7_18;
	hipMalloc((void **)&d_var_7_18, sizeof(float *));
	
	float * h_var_7_19 = (float *)malloc(sizeof(float *));
	float * d_var_7_19;
	hipMalloc((void **)&d_var_7_19, sizeof(float *));
	
	float * h_var_8_0 = (float *)malloc(sizeof(float *));
	float * d_var_8_0;
	hipMalloc((void **)&d_var_8_0, sizeof(float *));
	
	float * h_var_8_1 = (float *)malloc(sizeof(float *));
	float * d_var_8_1;
	hipMalloc((void **)&d_var_8_1, sizeof(float *));
	
	float * h_var_8_2 = (float *)malloc(sizeof(float *));
	float * d_var_8_2;
	hipMalloc((void **)&d_var_8_2, sizeof(float *));
	
	float * h_var_8_3 = (float *)malloc(sizeof(float *));
	float * d_var_8_3;
	hipMalloc((void **)&d_var_8_3, sizeof(float *));
	
	float * h_var_8_4 = (float *)malloc(sizeof(float *));
	float * d_var_8_4;
	hipMalloc((void **)&d_var_8_4, sizeof(float *));
	
	float * h_var_8_5 = (float *)malloc(sizeof(float *));
	float * d_var_8_5;
	hipMalloc((void **)&d_var_8_5, sizeof(float *));
	
	float * h_var_8_6 = (float *)malloc(sizeof(float *));
	float * d_var_8_6;
	hipMalloc((void **)&d_var_8_6, sizeof(float *));
	
	float * h_var_8_7 = (float *)malloc(sizeof(float *));
	float * d_var_8_7;
	hipMalloc((void **)&d_var_8_7, sizeof(float *));
	
	float * h_var_8_8 = (float *)malloc(sizeof(float *));
	float * d_var_8_8;
	hipMalloc((void **)&d_var_8_8, sizeof(float *));
	
	float * h_var_8_9 = (float *)malloc(sizeof(float *));
	float * d_var_8_9;
	hipMalloc((void **)&d_var_8_9, sizeof(float *));
	
	float * h_var_8_10 = (float *)malloc(sizeof(float *));
	float * d_var_8_10;
	hipMalloc((void **)&d_var_8_10, sizeof(float *));
	
	float * h_var_8_11 = (float *)malloc(sizeof(float *));
	float * d_var_8_11;
	hipMalloc((void **)&d_var_8_11, sizeof(float *));
	
	float * h_var_8_12 = (float *)malloc(sizeof(float *));
	float * d_var_8_12;
	hipMalloc((void **)&d_var_8_12, sizeof(float *));
	
	float * h_var_8_13 = (float *)malloc(sizeof(float *));
	float * d_var_8_13;
	hipMalloc((void **)&d_var_8_13, sizeof(float *));
	
	float * h_var_8_14 = (float *)malloc(sizeof(float *));
	float * d_var_8_14;
	hipMalloc((void **)&d_var_8_14, sizeof(float *));
	
	float * h_var_8_15 = (float *)malloc(sizeof(float *));
	float * d_var_8_15;
	hipMalloc((void **)&d_var_8_15, sizeof(float *));
	
	float * h_var_8_16 = (float *)malloc(sizeof(float *));
	float * d_var_8_16;
	hipMalloc((void **)&d_var_8_16, sizeof(float *));
	
	float * h_var_8_17 = (float *)malloc(sizeof(float *));
	float * d_var_8_17;
	hipMalloc((void **)&d_var_8_17, sizeof(float *));
	
	float * h_var_8_18 = (float *)malloc(sizeof(float *));
	float * d_var_8_18;
	hipMalloc((void **)&d_var_8_18, sizeof(float *));
	
	float * h_var_8_19 = (float *)malloc(sizeof(float *));
	float * d_var_8_19;
	hipMalloc((void **)&d_var_8_19, sizeof(float *));
	
	float * h_var_9_0 = (float *)malloc(sizeof(float *));
	float * d_var_9_0;
	hipMalloc((void **)&d_var_9_0, sizeof(float *));
	
	float * h_var_9_1 = (float *)malloc(sizeof(float *));
	float * d_var_9_1;
	hipMalloc((void **)&d_var_9_1, sizeof(float *));
	
	float * h_var_9_2 = (float *)malloc(sizeof(float *));
	float * d_var_9_2;
	hipMalloc((void **)&d_var_9_2, sizeof(float *));
	
	float * h_var_9_3 = (float *)malloc(sizeof(float *));
	float * d_var_9_3;
	hipMalloc((void **)&d_var_9_3, sizeof(float *));
	
	float * h_var_9_4 = (float *)malloc(sizeof(float *));
	float * d_var_9_4;
	hipMalloc((void **)&d_var_9_4, sizeof(float *));
	
	float * h_var_9_5 = (float *)malloc(sizeof(float *));
	float * d_var_9_5;
	hipMalloc((void **)&d_var_9_5, sizeof(float *));
	
	float * h_var_9_6 = (float *)malloc(sizeof(float *));
	float * d_var_9_6;
	hipMalloc((void **)&d_var_9_6, sizeof(float *));
	
	float * h_var_9_7 = (float *)malloc(sizeof(float *));
	float * d_var_9_7;
	hipMalloc((void **)&d_var_9_7, sizeof(float *));
	
	float * h_var_9_8 = (float *)malloc(sizeof(float *));
	float * d_var_9_8;
	hipMalloc((void **)&d_var_9_8, sizeof(float *));
	
	float * h_var_9_9 = (float *)malloc(sizeof(float *));
	float * d_var_9_9;
	hipMalloc((void **)&d_var_9_9, sizeof(float *));
	
	float * h_var_9_10 = (float *)malloc(sizeof(float *));
	float * d_var_9_10;
	hipMalloc((void **)&d_var_9_10, sizeof(float *));
	
	float * h_var_9_11 = (float *)malloc(sizeof(float *));
	float * d_var_9_11;
	hipMalloc((void **)&d_var_9_11, sizeof(float *));
	
	float * h_var_9_12 = (float *)malloc(sizeof(float *));
	float * d_var_9_12;
	hipMalloc((void **)&d_var_9_12, sizeof(float *));
	
	float * h_var_9_13 = (float *)malloc(sizeof(float *));
	float * d_var_9_13;
	hipMalloc((void **)&d_var_9_13, sizeof(float *));
	
	float * h_var_9_14 = (float *)malloc(sizeof(float *));
	float * d_var_9_14;
	hipMalloc((void **)&d_var_9_14, sizeof(float *));
	
	float * h_var_9_15 = (float *)malloc(sizeof(float *));
	float * d_var_9_15;
	hipMalloc((void **)&d_var_9_15, sizeof(float *));
	
	float * h_var_9_16 = (float *)malloc(sizeof(float *));
	float * d_var_9_16;
	hipMalloc((void **)&d_var_9_16, sizeof(float *));
	
	float * h_var_9_17 = (float *)malloc(sizeof(float *));
	float * d_var_9_17;
	hipMalloc((void **)&d_var_9_17, sizeof(float *));
	
	float * h_var_9_18 = (float *)malloc(sizeof(float *));
	float * d_var_9_18;
	hipMalloc((void **)&d_var_9_18, sizeof(float *));
	
	float * h_var_9_19 = (float *)malloc(sizeof(float *));
	float * d_var_9_19;
	hipMalloc((void **)&d_var_9_19, sizeof(float *));
	
	float * h_var_10_0 = (float *)malloc(sizeof(float *));
	float * d_var_10_0;
	hipMalloc((void **)&d_var_10_0, sizeof(float *));
	
	float * h_var_10_1 = (float *)malloc(sizeof(float *));
	float * d_var_10_1;
	hipMalloc((void **)&d_var_10_1, sizeof(float *));
	
	float * h_var_10_2 = (float *)malloc(sizeof(float *));
	float * d_var_10_2;
	hipMalloc((void **)&d_var_10_2, sizeof(float *));
	
	float * h_var_10_3 = (float *)malloc(sizeof(float *));
	float * d_var_10_3;
	hipMalloc((void **)&d_var_10_3, sizeof(float *));
	
	float * h_var_10_4 = (float *)malloc(sizeof(float *));
	float * d_var_10_4;
	hipMalloc((void **)&d_var_10_4, sizeof(float *));
	
	float * h_var_10_5 = (float *)malloc(sizeof(float *));
	float * d_var_10_5;
	hipMalloc((void **)&d_var_10_5, sizeof(float *));
	
	float * h_var_10_6 = (float *)malloc(sizeof(float *));
	float * d_var_10_6;
	hipMalloc((void **)&d_var_10_6, sizeof(float *));
	
	float * h_var_10_7 = (float *)malloc(sizeof(float *));
	float * d_var_10_7;
	hipMalloc((void **)&d_var_10_7, sizeof(float *));
	
	float * h_var_10_8 = (float *)malloc(sizeof(float *));
	float * d_var_10_8;
	hipMalloc((void **)&d_var_10_8, sizeof(float *));
	
	float * h_var_10_9 = (float *)malloc(sizeof(float *));
	float * d_var_10_9;
	hipMalloc((void **)&d_var_10_9, sizeof(float *));
	
	float * h_var_10_10 = (float *)malloc(sizeof(float *));
	float * d_var_10_10;
	hipMalloc((void **)&d_var_10_10, sizeof(float *));
	
	float * h_var_10_11 = (float *)malloc(sizeof(float *));
	float * d_var_10_11;
	hipMalloc((void **)&d_var_10_11, sizeof(float *));
	
	float * h_var_10_12 = (float *)malloc(sizeof(float *));
	float * d_var_10_12;
	hipMalloc((void **)&d_var_10_12, sizeof(float *));
	
	float * h_var_10_13 = (float *)malloc(sizeof(float *));
	float * d_var_10_13;
	hipMalloc((void **)&d_var_10_13, sizeof(float *));
	
	float * h_var_10_14 = (float *)malloc(sizeof(float *));
	float * d_var_10_14;
	hipMalloc((void **)&d_var_10_14, sizeof(float *));
	
	float * h_var_10_15 = (float *)malloc(sizeof(float *));
	float * d_var_10_15;
	hipMalloc((void **)&d_var_10_15, sizeof(float *));
	
	float * h_var_10_16 = (float *)malloc(sizeof(float *));
	float * d_var_10_16;
	hipMalloc((void **)&d_var_10_16, sizeof(float *));
	
	float * h_var_10_17 = (float *)malloc(sizeof(float *));
	float * d_var_10_17;
	hipMalloc((void **)&d_var_10_17, sizeof(float *));
	
	float * h_var_10_18 = (float *)malloc(sizeof(float *));
	float * d_var_10_18;
	hipMalloc((void **)&d_var_10_18, sizeof(float *));
	
	float * h_var_10_19 = (float *)malloc(sizeof(float *));
	float * d_var_10_19;
	hipMalloc((void **)&d_var_10_19, sizeof(float *));
	
	float * h_var_11_0 = (float *)malloc(sizeof(float *));
	float * d_var_11_0;
	hipMalloc((void **)&d_var_11_0, sizeof(float *));
	
	float * h_var_11_1 = (float *)malloc(sizeof(float *));
	float * d_var_11_1;
	hipMalloc((void **)&d_var_11_1, sizeof(float *));
	
	float * h_var_11_2 = (float *)malloc(sizeof(float *));
	float * d_var_11_2;
	hipMalloc((void **)&d_var_11_2, sizeof(float *));
	
	float * h_var_11_3 = (float *)malloc(sizeof(float *));
	float * d_var_11_3;
	hipMalloc((void **)&d_var_11_3, sizeof(float *));
	
	float * h_var_11_4 = (float *)malloc(sizeof(float *));
	float * d_var_11_4;
	hipMalloc((void **)&d_var_11_4, sizeof(float *));
	
	float * h_var_11_5 = (float *)malloc(sizeof(float *));
	float * d_var_11_5;
	hipMalloc((void **)&d_var_11_5, sizeof(float *));
	
	float * h_var_11_6 = (float *)malloc(sizeof(float *));
	float * d_var_11_6;
	hipMalloc((void **)&d_var_11_6, sizeof(float *));
	
	float * h_var_11_7 = (float *)malloc(sizeof(float *));
	float * d_var_11_7;
	hipMalloc((void **)&d_var_11_7, sizeof(float *));
	
	float * h_var_11_8 = (float *)malloc(sizeof(float *));
	float * d_var_11_8;
	hipMalloc((void **)&d_var_11_8, sizeof(float *));
	
	float * h_var_11_9 = (float *)malloc(sizeof(float *));
	float * d_var_11_9;
	hipMalloc((void **)&d_var_11_9, sizeof(float *));
	
	float * h_var_11_10 = (float *)malloc(sizeof(float *));
	float * d_var_11_10;
	hipMalloc((void **)&d_var_11_10, sizeof(float *));
	
	float * h_var_11_11 = (float *)malloc(sizeof(float *));
	float * d_var_11_11;
	hipMalloc((void **)&d_var_11_11, sizeof(float *));
	
	float * h_var_11_12 = (float *)malloc(sizeof(float *));
	float * d_var_11_12;
	hipMalloc((void **)&d_var_11_12, sizeof(float *));
	
	float * h_var_11_13 = (float *)malloc(sizeof(float *));
	float * d_var_11_13;
	hipMalloc((void **)&d_var_11_13, sizeof(float *));
	
	float * h_var_11_14 = (float *)malloc(sizeof(float *));
	float * d_var_11_14;
	hipMalloc((void **)&d_var_11_14, sizeof(float *));
	
	float * h_var_11_15 = (float *)malloc(sizeof(float *));
	float * d_var_11_15;
	hipMalloc((void **)&d_var_11_15, sizeof(float *));
	
	float * h_var_11_16 = (float *)malloc(sizeof(float *));
	float * d_var_11_16;
	hipMalloc((void **)&d_var_11_16, sizeof(float *));
	
	float * h_var_11_17 = (float *)malloc(sizeof(float *));
	float * d_var_11_17;
	hipMalloc((void **)&d_var_11_17, sizeof(float *));
	
	float * h_var_11_18 = (float *)malloc(sizeof(float *));
	float * d_var_11_18;
	hipMalloc((void **)&d_var_11_18, sizeof(float *));
	
	float * h_var_11_19 = (float *)malloc(sizeof(float *));
	float * d_var_11_19;
	hipMalloc((void **)&d_var_11_19, sizeof(float *));
	
	float * h_var_12_0 = (float *)malloc(sizeof(float *));
	float * d_var_12_0;
	hipMalloc((void **)&d_var_12_0, sizeof(float *));
	
	float * h_var_12_1 = (float *)malloc(sizeof(float *));
	float * d_var_12_1;
	hipMalloc((void **)&d_var_12_1, sizeof(float *));
	
	float * h_var_12_2 = (float *)malloc(sizeof(float *));
	float * d_var_12_2;
	hipMalloc((void **)&d_var_12_2, sizeof(float *));
	
	float * h_var_12_3 = (float *)malloc(sizeof(float *));
	float * d_var_12_3;
	hipMalloc((void **)&d_var_12_3, sizeof(float *));
	
	float * h_var_12_4 = (float *)malloc(sizeof(float *));
	float * d_var_12_4;
	hipMalloc((void **)&d_var_12_4, sizeof(float *));
	
	float * h_var_12_5 = (float *)malloc(sizeof(float *));
	float * d_var_12_5;
	hipMalloc((void **)&d_var_12_5, sizeof(float *));
	
	float * h_var_12_6 = (float *)malloc(sizeof(float *));
	float * d_var_12_6;
	hipMalloc((void **)&d_var_12_6, sizeof(float *));
	
	float * h_var_12_7 = (float *)malloc(sizeof(float *));
	float * d_var_12_7;
	hipMalloc((void **)&d_var_12_7, sizeof(float *));
	
	float * h_var_12_8 = (float *)malloc(sizeof(float *));
	float * d_var_12_8;
	hipMalloc((void **)&d_var_12_8, sizeof(float *));
	
	float * h_var_12_9 = (float *)malloc(sizeof(float *));
	float * d_var_12_9;
	hipMalloc((void **)&d_var_12_9, sizeof(float *));
	
	float * h_var_12_10 = (float *)malloc(sizeof(float *));
	float * d_var_12_10;
	hipMalloc((void **)&d_var_12_10, sizeof(float *));
	
	float * h_var_12_11 = (float *)malloc(sizeof(float *));
	float * d_var_12_11;
	hipMalloc((void **)&d_var_12_11, sizeof(float *));
	
	float * h_var_12_12 = (float *)malloc(sizeof(float *));
	float * d_var_12_12;
	hipMalloc((void **)&d_var_12_12, sizeof(float *));
	
	float * h_var_12_13 = (float *)malloc(sizeof(float *));
	float * d_var_12_13;
	hipMalloc((void **)&d_var_12_13, sizeof(float *));
	
	float * h_var_12_14 = (float *)malloc(sizeof(float *));
	float * d_var_12_14;
	hipMalloc((void **)&d_var_12_14, sizeof(float *));
	
	float * h_var_12_15 = (float *)malloc(sizeof(float *));
	float * d_var_12_15;
	hipMalloc((void **)&d_var_12_15, sizeof(float *));
	
	float * h_var_12_16 = (float *)malloc(sizeof(float *));
	float * d_var_12_16;
	hipMalloc((void **)&d_var_12_16, sizeof(float *));
	
	float * h_var_12_17 = (float *)malloc(sizeof(float *));
	float * d_var_12_17;
	hipMalloc((void **)&d_var_12_17, sizeof(float *));
	
	float * h_var_12_18 = (float *)malloc(sizeof(float *));
	float * d_var_12_18;
	hipMalloc((void **)&d_var_12_18, sizeof(float *));
	
	float * h_var_12_19 = (float *)malloc(sizeof(float *));
	float * d_var_12_19;
	hipMalloc((void **)&d_var_12_19, sizeof(float *));
	
	float * h_var_13_0 = (float *)malloc(sizeof(float *));
	float * d_var_13_0;
	hipMalloc((void **)&d_var_13_0, sizeof(float *));
	
	float * h_var_13_1 = (float *)malloc(sizeof(float *));
	float * d_var_13_1;
	hipMalloc((void **)&d_var_13_1, sizeof(float *));
	
	float * h_var_13_2 = (float *)malloc(sizeof(float *));
	float * d_var_13_2;
	hipMalloc((void **)&d_var_13_2, sizeof(float *));
	
	float * h_var_13_3 = (float *)malloc(sizeof(float *));
	float * d_var_13_3;
	hipMalloc((void **)&d_var_13_3, sizeof(float *));
	
	float * h_var_13_4 = (float *)malloc(sizeof(float *));
	float * d_var_13_4;
	hipMalloc((void **)&d_var_13_4, sizeof(float *));
	
	float * h_var_13_5 = (float *)malloc(sizeof(float *));
	float * d_var_13_5;
	hipMalloc((void **)&d_var_13_5, sizeof(float *));
	
	float * h_var_13_6 = (float *)malloc(sizeof(float *));
	float * d_var_13_6;
	hipMalloc((void **)&d_var_13_6, sizeof(float *));
	
	float * h_var_13_7 = (float *)malloc(sizeof(float *));
	float * d_var_13_7;
	hipMalloc((void **)&d_var_13_7, sizeof(float *));
	
	float * h_var_13_8 = (float *)malloc(sizeof(float *));
	float * d_var_13_8;
	hipMalloc((void **)&d_var_13_8, sizeof(float *));
	
	float * h_var_13_9 = (float *)malloc(sizeof(float *));
	float * d_var_13_9;
	hipMalloc((void **)&d_var_13_9, sizeof(float *));
	
	float * h_var_13_10 = (float *)malloc(sizeof(float *));
	float * d_var_13_10;
	hipMalloc((void **)&d_var_13_10, sizeof(float *));
	
	float * h_var_13_11 = (float *)malloc(sizeof(float *));
	float * d_var_13_11;
	hipMalloc((void **)&d_var_13_11, sizeof(float *));
	
	float * h_var_13_12 = (float *)malloc(sizeof(float *));
	float * d_var_13_12;
	hipMalloc((void **)&d_var_13_12, sizeof(float *));
	
	float * h_var_13_13 = (float *)malloc(sizeof(float *));
	float * d_var_13_13;
	hipMalloc((void **)&d_var_13_13, sizeof(float *));
	
	float * h_var_13_14 = (float *)malloc(sizeof(float *));
	float * d_var_13_14;
	hipMalloc((void **)&d_var_13_14, sizeof(float *));
	
	float * h_var_13_15 = (float *)malloc(sizeof(float *));
	float * d_var_13_15;
	hipMalloc((void **)&d_var_13_15, sizeof(float *));
	
	float * h_var_13_16 = (float *)malloc(sizeof(float *));
	float * d_var_13_16;
	hipMalloc((void **)&d_var_13_16, sizeof(float *));
	
	float * h_var_13_17 = (float *)malloc(sizeof(float *));
	float * d_var_13_17;
	hipMalloc((void **)&d_var_13_17, sizeof(float *));
	
	float * h_var_13_18 = (float *)malloc(sizeof(float *));
	float * d_var_13_18;
	hipMalloc((void **)&d_var_13_18, sizeof(float *));
	
	float * h_var_13_19 = (float *)malloc(sizeof(float *));
	float * d_var_13_19;
	hipMalloc((void **)&d_var_13_19, sizeof(float *));
	
	float * h_var_14_0 = (float *)malloc(sizeof(float *));
	float * d_var_14_0;
	hipMalloc((void **)&d_var_14_0, sizeof(float *));
	
	float * h_var_14_1 = (float *)malloc(sizeof(float *));
	float * d_var_14_1;
	hipMalloc((void **)&d_var_14_1, sizeof(float *));
	
	float * h_var_14_2 = (float *)malloc(sizeof(float *));
	float * d_var_14_2;
	hipMalloc((void **)&d_var_14_2, sizeof(float *));
	
	float * h_var_14_3 = (float *)malloc(sizeof(float *));
	float * d_var_14_3;
	hipMalloc((void **)&d_var_14_3, sizeof(float *));
	
	float * h_var_14_4 = (float *)malloc(sizeof(float *));
	float * d_var_14_4;
	hipMalloc((void **)&d_var_14_4, sizeof(float *));
	
	float * h_var_14_5 = (float *)malloc(sizeof(float *));
	float * d_var_14_5;
	hipMalloc((void **)&d_var_14_5, sizeof(float *));
	
	float * h_var_14_6 = (float *)malloc(sizeof(float *));
	float * d_var_14_6;
	hipMalloc((void **)&d_var_14_6, sizeof(float *));
	
	float * h_var_14_7 = (float *)malloc(sizeof(float *));
	float * d_var_14_7;
	hipMalloc((void **)&d_var_14_7, sizeof(float *));
	
	float * h_var_14_8 = (float *)malloc(sizeof(float *));
	float * d_var_14_8;
	hipMalloc((void **)&d_var_14_8, sizeof(float *));
	
	float * h_var_14_9 = (float *)malloc(sizeof(float *));
	float * d_var_14_9;
	hipMalloc((void **)&d_var_14_9, sizeof(float *));
	
	float * h_var_14_10 = (float *)malloc(sizeof(float *));
	float * d_var_14_10;
	hipMalloc((void **)&d_var_14_10, sizeof(float *));
	
	float * h_var_14_11 = (float *)malloc(sizeof(float *));
	float * d_var_14_11;
	hipMalloc((void **)&d_var_14_11, sizeof(float *));
	
	float * h_var_14_12 = (float *)malloc(sizeof(float *));
	float * d_var_14_12;
	hipMalloc((void **)&d_var_14_12, sizeof(float *));
	
	float * h_var_14_13 = (float *)malloc(sizeof(float *));
	float * d_var_14_13;
	hipMalloc((void **)&d_var_14_13, sizeof(float *));
	
	float * h_var_14_14 = (float *)malloc(sizeof(float *));
	float * d_var_14_14;
	hipMalloc((void **)&d_var_14_14, sizeof(float *));
	
	float * h_var_14_15 = (float *)malloc(sizeof(float *));
	float * d_var_14_15;
	hipMalloc((void **)&d_var_14_15, sizeof(float *));
	
	float * h_var_14_16 = (float *)malloc(sizeof(float *));
	float * d_var_14_16;
	hipMalloc((void **)&d_var_14_16, sizeof(float *));
	
	float * h_var_14_17 = (float *)malloc(sizeof(float *));
	float * d_var_14_17;
	hipMalloc((void **)&d_var_14_17, sizeof(float *));
	
	float * h_var_14_18 = (float *)malloc(sizeof(float *));
	float * d_var_14_18;
	hipMalloc((void **)&d_var_14_18, sizeof(float *));
	
	float * h_var_14_19 = (float *)malloc(sizeof(float *));
	float * d_var_14_19;
	hipMalloc((void **)&d_var_14_19, sizeof(float *));
	
	float * h_var_15_0 = (float *)malloc(sizeof(float *));
	float * d_var_15_0;
	hipMalloc((void **)&d_var_15_0, sizeof(float *));
	
	float * h_var_15_1 = (float *)malloc(sizeof(float *));
	float * d_var_15_1;
	hipMalloc((void **)&d_var_15_1, sizeof(float *));
	
	float * h_var_15_2 = (float *)malloc(sizeof(float *));
	float * d_var_15_2;
	hipMalloc((void **)&d_var_15_2, sizeof(float *));
	
	float * h_var_15_3 = (float *)malloc(sizeof(float *));
	float * d_var_15_3;
	hipMalloc((void **)&d_var_15_3, sizeof(float *));
	
	float * h_var_15_4 = (float *)malloc(sizeof(float *));
	float * d_var_15_4;
	hipMalloc((void **)&d_var_15_4, sizeof(float *));
	
	float * h_var_15_5 = (float *)malloc(sizeof(float *));
	float * d_var_15_5;
	hipMalloc((void **)&d_var_15_5, sizeof(float *));
	
	float * h_var_15_6 = (float *)malloc(sizeof(float *));
	float * d_var_15_6;
	hipMalloc((void **)&d_var_15_6, sizeof(float *));
	
	float * h_var_15_7 = (float *)malloc(sizeof(float *));
	float * d_var_15_7;
	hipMalloc((void **)&d_var_15_7, sizeof(float *));
	
	float * h_var_15_8 = (float *)malloc(sizeof(float *));
	float * d_var_15_8;
	hipMalloc((void **)&d_var_15_8, sizeof(float *));
	
	float * h_var_15_9 = (float *)malloc(sizeof(float *));
	float * d_var_15_9;
	hipMalloc((void **)&d_var_15_9, sizeof(float *));
	
	float * h_var_15_10 = (float *)malloc(sizeof(float *));
	float * d_var_15_10;
	hipMalloc((void **)&d_var_15_10, sizeof(float *));
	
	float * h_var_15_11 = (float *)malloc(sizeof(float *));
	float * d_var_15_11;
	hipMalloc((void **)&d_var_15_11, sizeof(float *));
	
	float * h_var_15_12 = (float *)malloc(sizeof(float *));
	float * d_var_15_12;
	hipMalloc((void **)&d_var_15_12, sizeof(float *));
	
	float * h_var_15_13 = (float *)malloc(sizeof(float *));
	float * d_var_15_13;
	hipMalloc((void **)&d_var_15_13, sizeof(float *));
	
	float * h_var_15_14 = (float *)malloc(sizeof(float *));
	float * d_var_15_14;
	hipMalloc((void **)&d_var_15_14, sizeof(float *));
	
	float * h_var_15_15 = (float *)malloc(sizeof(float *));
	float * d_var_15_15;
	hipMalloc((void **)&d_var_15_15, sizeof(float *));
	
	float * h_var_15_16 = (float *)malloc(sizeof(float *));
	float * d_var_15_16;
	hipMalloc((void **)&d_var_15_16, sizeof(float *));
	
	float * h_var_15_17 = (float *)malloc(sizeof(float *));
	float * d_var_15_17;
	hipMalloc((void **)&d_var_15_17, sizeof(float *));
	
	float * h_var_15_18 = (float *)malloc(sizeof(float *));
	float * d_var_15_18;
	hipMalloc((void **)&d_var_15_18, sizeof(float *));
	
	float * h_var_15_19 = (float *)malloc(sizeof(float *));
	float * d_var_15_19;
	hipMalloc((void **)&d_var_15_19, sizeof(float *));
	
	float * h_var_16_0 = (float *)malloc(sizeof(float *));
	float * d_var_16_0;
	hipMalloc((void **)&d_var_16_0, sizeof(float *));
	
	float * h_var_16_1 = (float *)malloc(sizeof(float *));
	float * d_var_16_1;
	hipMalloc((void **)&d_var_16_1, sizeof(float *));
	
	float * h_var_16_2 = (float *)malloc(sizeof(float *));
	float * d_var_16_2;
	hipMalloc((void **)&d_var_16_2, sizeof(float *));
	
	float * h_var_16_3 = (float *)malloc(sizeof(float *));
	float * d_var_16_3;
	hipMalloc((void **)&d_var_16_3, sizeof(float *));
	
	float * h_var_16_4 = (float *)malloc(sizeof(float *));
	float * d_var_16_4;
	hipMalloc((void **)&d_var_16_4, sizeof(float *));
	
	float * h_var_16_5 = (float *)malloc(sizeof(float *));
	float * d_var_16_5;
	hipMalloc((void **)&d_var_16_5, sizeof(float *));
	
	float * h_var_16_6 = (float *)malloc(sizeof(float *));
	float * d_var_16_6;
	hipMalloc((void **)&d_var_16_6, sizeof(float *));
	
	float * h_var_16_7 = (float *)malloc(sizeof(float *));
	float * d_var_16_7;
	hipMalloc((void **)&d_var_16_7, sizeof(float *));
	
	float * h_var_16_8 = (float *)malloc(sizeof(float *));
	float * d_var_16_8;
	hipMalloc((void **)&d_var_16_8, sizeof(float *));
	
	float * h_var_16_9 = (float *)malloc(sizeof(float *));
	float * d_var_16_9;
	hipMalloc((void **)&d_var_16_9, sizeof(float *));
	
	float * h_var_16_10 = (float *)malloc(sizeof(float *));
	float * d_var_16_10;
	hipMalloc((void **)&d_var_16_10, sizeof(float *));
	
	float * h_var_16_11 = (float *)malloc(sizeof(float *));
	float * d_var_16_11;
	hipMalloc((void **)&d_var_16_11, sizeof(float *));
	
	float * h_var_16_12 = (float *)malloc(sizeof(float *));
	float * d_var_16_12;
	hipMalloc((void **)&d_var_16_12, sizeof(float *));
	
	float * h_var_16_13 = (float *)malloc(sizeof(float *));
	float * d_var_16_13;
	hipMalloc((void **)&d_var_16_13, sizeof(float *));
	
	float * h_var_16_14 = (float *)malloc(sizeof(float *));
	float * d_var_16_14;
	hipMalloc((void **)&d_var_16_14, sizeof(float *));
	
	float * h_var_16_15 = (float *)malloc(sizeof(float *));
	float * d_var_16_15;
	hipMalloc((void **)&d_var_16_15, sizeof(float *));
	
	float * h_var_16_16 = (float *)malloc(sizeof(float *));
	float * d_var_16_16;
	hipMalloc((void **)&d_var_16_16, sizeof(float *));
	
	float * h_var_16_17 = (float *)malloc(sizeof(float *));
	float * d_var_16_17;
	hipMalloc((void **)&d_var_16_17, sizeof(float *));
	
	float * h_var_16_18 = (float *)malloc(sizeof(float *));
	float * d_var_16_18;
	hipMalloc((void **)&d_var_16_18, sizeof(float *));
	
	float * h_var_16_19 = (float *)malloc(sizeof(float *));
	float * d_var_16_19;
	hipMalloc((void **)&d_var_16_19, sizeof(float *));
	
	float * h_var_17_0 = (float *)malloc(sizeof(float *));
	float * d_var_17_0;
	hipMalloc((void **)&d_var_17_0, sizeof(float *));
	
	float * h_var_17_1 = (float *)malloc(sizeof(float *));
	float * d_var_17_1;
	hipMalloc((void **)&d_var_17_1, sizeof(float *));
	
	float * h_var_17_2 = (float *)malloc(sizeof(float *));
	float * d_var_17_2;
	hipMalloc((void **)&d_var_17_2, sizeof(float *));
	
	float * h_var_17_3 = (float *)malloc(sizeof(float *));
	float * d_var_17_3;
	hipMalloc((void **)&d_var_17_3, sizeof(float *));
	
	float * h_var_17_4 = (float *)malloc(sizeof(float *));
	float * d_var_17_4;
	hipMalloc((void **)&d_var_17_4, sizeof(float *));
	
	float * h_var_17_5 = (float *)malloc(sizeof(float *));
	float * d_var_17_5;
	hipMalloc((void **)&d_var_17_5, sizeof(float *));
	
	float * h_var_17_6 = (float *)malloc(sizeof(float *));
	float * d_var_17_6;
	hipMalloc((void **)&d_var_17_6, sizeof(float *));
	
	float * h_var_17_7 = (float *)malloc(sizeof(float *));
	float * d_var_17_7;
	hipMalloc((void **)&d_var_17_7, sizeof(float *));
	
	float * h_var_17_8 = (float *)malloc(sizeof(float *));
	float * d_var_17_8;
	hipMalloc((void **)&d_var_17_8, sizeof(float *));
	
	float * h_var_17_9 = (float *)malloc(sizeof(float *));
	float * d_var_17_9;
	hipMalloc((void **)&d_var_17_9, sizeof(float *));
	
	float * h_var_17_10 = (float *)malloc(sizeof(float *));
	float * d_var_17_10;
	hipMalloc((void **)&d_var_17_10, sizeof(float *));
	
	float * h_var_17_11 = (float *)malloc(sizeof(float *));
	float * d_var_17_11;
	hipMalloc((void **)&d_var_17_11, sizeof(float *));
	
	float * h_var_17_12 = (float *)malloc(sizeof(float *));
	float * d_var_17_12;
	hipMalloc((void **)&d_var_17_12, sizeof(float *));
	
	float * h_var_17_13 = (float *)malloc(sizeof(float *));
	float * d_var_17_13;
	hipMalloc((void **)&d_var_17_13, sizeof(float *));
	
	float * h_var_17_14 = (float *)malloc(sizeof(float *));
	float * d_var_17_14;
	hipMalloc((void **)&d_var_17_14, sizeof(float *));
	
	float * h_var_17_15 = (float *)malloc(sizeof(float *));
	float * d_var_17_15;
	hipMalloc((void **)&d_var_17_15, sizeof(float *));
	
	float * h_var_17_16 = (float *)malloc(sizeof(float *));
	float * d_var_17_16;
	hipMalloc((void **)&d_var_17_16, sizeof(float *));
	
	float * h_var_17_17 = (float *)malloc(sizeof(float *));
	float * d_var_17_17;
	hipMalloc((void **)&d_var_17_17, sizeof(float *));
	
	float * h_var_17_18 = (float *)malloc(sizeof(float *));
	float * d_var_17_18;
	hipMalloc((void **)&d_var_17_18, sizeof(float *));
	
	float * h_var_17_19 = (float *)malloc(sizeof(float *));
	float * d_var_17_19;
	hipMalloc((void **)&d_var_17_19, sizeof(float *));
	
	float * h_var_18_0 = (float *)malloc(sizeof(float *));
	float * d_var_18_0;
	hipMalloc((void **)&d_var_18_0, sizeof(float *));
	
	float * h_var_18_1 = (float *)malloc(sizeof(float *));
	float * d_var_18_1;
	hipMalloc((void **)&d_var_18_1, sizeof(float *));
	
	float * h_var_18_2 = (float *)malloc(sizeof(float *));
	float * d_var_18_2;
	hipMalloc((void **)&d_var_18_2, sizeof(float *));
	
	float * h_var_18_3 = (float *)malloc(sizeof(float *));
	float * d_var_18_3;
	hipMalloc((void **)&d_var_18_3, sizeof(float *));
	
	float * h_var_18_4 = (float *)malloc(sizeof(float *));
	float * d_var_18_4;
	hipMalloc((void **)&d_var_18_4, sizeof(float *));
	
	float * h_var_18_5 = (float *)malloc(sizeof(float *));
	float * d_var_18_5;
	hipMalloc((void **)&d_var_18_5, sizeof(float *));
	
	float * h_var_18_6 = (float *)malloc(sizeof(float *));
	float * d_var_18_6;
	hipMalloc((void **)&d_var_18_6, sizeof(float *));
	
	float * h_var_18_7 = (float *)malloc(sizeof(float *));
	float * d_var_18_7;
	hipMalloc((void **)&d_var_18_7, sizeof(float *));
	
	float * h_var_18_8 = (float *)malloc(sizeof(float *));
	float * d_var_18_8;
	hipMalloc((void **)&d_var_18_8, sizeof(float *));
	
	float * h_var_18_9 = (float *)malloc(sizeof(float *));
	float * d_var_18_9;
	hipMalloc((void **)&d_var_18_9, sizeof(float *));
	
	float * h_var_18_10 = (float *)malloc(sizeof(float *));
	float * d_var_18_10;
	hipMalloc((void **)&d_var_18_10, sizeof(float *));
	
	float * h_var_18_11 = (float *)malloc(sizeof(float *));
	float * d_var_18_11;
	hipMalloc((void **)&d_var_18_11, sizeof(float *));
	
	float * h_var_18_12 = (float *)malloc(sizeof(float *));
	float * d_var_18_12;
	hipMalloc((void **)&d_var_18_12, sizeof(float *));
	
	float * h_var_18_13 = (float *)malloc(sizeof(float *));
	float * d_var_18_13;
	hipMalloc((void **)&d_var_18_13, sizeof(float *));
	
	float * h_var_18_14 = (float *)malloc(sizeof(float *));
	float * d_var_18_14;
	hipMalloc((void **)&d_var_18_14, sizeof(float *));
	
	float * h_var_18_15 = (float *)malloc(sizeof(float *));
	float * d_var_18_15;
	hipMalloc((void **)&d_var_18_15, sizeof(float *));
	
	float * h_var_18_16 = (float *)malloc(sizeof(float *));
	float * d_var_18_16;
	hipMalloc((void **)&d_var_18_16, sizeof(float *));
	
	float * h_var_18_17 = (float *)malloc(sizeof(float *));
	float * d_var_18_17;
	hipMalloc((void **)&d_var_18_17, sizeof(float *));
	
	float * h_var_18_18 = (float *)malloc(sizeof(float *));
	float * d_var_18_18;
	hipMalloc((void **)&d_var_18_18, sizeof(float *));
	
	float * h_var_18_19 = (float *)malloc(sizeof(float *));
	float * d_var_18_19;
	hipMalloc((void **)&d_var_18_19, sizeof(float *));
	
	float * h_var_19_0 = (float *)malloc(sizeof(float *));
	float * d_var_19_0;
	hipMalloc((void **)&d_var_19_0, sizeof(float *));
	
	float * h_var_19_1 = (float *)malloc(sizeof(float *));
	float * d_var_19_1;
	hipMalloc((void **)&d_var_19_1, sizeof(float *));
	
	float * h_var_19_2 = (float *)malloc(sizeof(float *));
	float * d_var_19_2;
	hipMalloc((void **)&d_var_19_2, sizeof(float *));
	
	float * h_var_19_3 = (float *)malloc(sizeof(float *));
	float * d_var_19_3;
	hipMalloc((void **)&d_var_19_3, sizeof(float *));
	
	float * h_var_19_4 = (float *)malloc(sizeof(float *));
	float * d_var_19_4;
	hipMalloc((void **)&d_var_19_4, sizeof(float *));
	
	float * h_var_19_5 = (float *)malloc(sizeof(float *));
	float * d_var_19_5;
	hipMalloc((void **)&d_var_19_5, sizeof(float *));
	
	float * h_var_19_6 = (float *)malloc(sizeof(float *));
	float * d_var_19_6;
	hipMalloc((void **)&d_var_19_6, sizeof(float *));
	
	float * h_var_19_7 = (float *)malloc(sizeof(float *));
	float * d_var_19_7;
	hipMalloc((void **)&d_var_19_7, sizeof(float *));
	
	float * h_var_19_8 = (float *)malloc(sizeof(float *));
	float * d_var_19_8;
	hipMalloc((void **)&d_var_19_8, sizeof(float *));
	
	float * h_var_19_9 = (float *)malloc(sizeof(float *));
	float * d_var_19_9;
	hipMalloc((void **)&d_var_19_9, sizeof(float *));
	
	float * h_var_19_10 = (float *)malloc(sizeof(float *));
	float * d_var_19_10;
	hipMalloc((void **)&d_var_19_10, sizeof(float *));
	
	float * h_var_19_11 = (float *)malloc(sizeof(float *));
	float * d_var_19_11;
	hipMalloc((void **)&d_var_19_11, sizeof(float *));
	
	float * h_var_19_12 = (float *)malloc(sizeof(float *));
	float * d_var_19_12;
	hipMalloc((void **)&d_var_19_12, sizeof(float *));
	
	float * h_var_19_13 = (float *)malloc(sizeof(float *));
	float * d_var_19_13;
	hipMalloc((void **)&d_var_19_13, sizeof(float *));
	
	float * h_var_19_14 = (float *)malloc(sizeof(float *));
	float * d_var_19_14;
	hipMalloc((void **)&d_var_19_14, sizeof(float *));
	
	float * h_var_19_15 = (float *)malloc(sizeof(float *));
	float * d_var_19_15;
	hipMalloc((void **)&d_var_19_15, sizeof(float *));
	
	float * h_var_19_16 = (float *)malloc(sizeof(float *));
	float * d_var_19_16;
	hipMalloc((void **)&d_var_19_16, sizeof(float *));
	
	float * h_var_19_17 = (float *)malloc(sizeof(float *));
	float * d_var_19_17;
	hipMalloc((void **)&d_var_19_17, sizeof(float *));
	
	float * h_var_19_18 = (float *)malloc(sizeof(float *));
	float * d_var_19_18;
	hipMalloc((void **)&d_var_19_18, sizeof(float *));
	
	float * h_var_19_19 = (float *)malloc(sizeof(float *));
	float * d_var_19_19;
	hipMalloc((void **)&d_var_19_19, sizeof(float *));
	
	float * h_var_20_0 = (float *)malloc(sizeof(float *));
	float * d_var_20_0;
	hipMalloc((void **)&d_var_20_0, sizeof(float *));
	
	float * h_var_20_1 = (float *)malloc(sizeof(float *));
	float * d_var_20_1;
	hipMalloc((void **)&d_var_20_1, sizeof(float *));
	
	float * h_var_20_2 = (float *)malloc(sizeof(float *));
	float * d_var_20_2;
	hipMalloc((void **)&d_var_20_2, sizeof(float *));
	
	float * h_var_20_3 = (float *)malloc(sizeof(float *));
	float * d_var_20_3;
	hipMalloc((void **)&d_var_20_3, sizeof(float *));
	
	float * h_var_20_4 = (float *)malloc(sizeof(float *));
	float * d_var_20_4;
	hipMalloc((void **)&d_var_20_4, sizeof(float *));
	
	float * h_var_20_5 = (float *)malloc(sizeof(float *));
	float * d_var_20_5;
	hipMalloc((void **)&d_var_20_5, sizeof(float *));
	
	float * h_var_20_6 = (float *)malloc(sizeof(float *));
	float * d_var_20_6;
	hipMalloc((void **)&d_var_20_6, sizeof(float *));
	
	float * h_var_20_7 = (float *)malloc(sizeof(float *));
	float * d_var_20_7;
	hipMalloc((void **)&d_var_20_7, sizeof(float *));
	
	float * h_var_20_8 = (float *)malloc(sizeof(float *));
	float * d_var_20_8;
	hipMalloc((void **)&d_var_20_8, sizeof(float *));
	
	float * h_var_20_9 = (float *)malloc(sizeof(float *));
	float * d_var_20_9;
	hipMalloc((void **)&d_var_20_9, sizeof(float *));
	
	float * h_var_20_10 = (float *)malloc(sizeof(float *));
	float * d_var_20_10;
	hipMalloc((void **)&d_var_20_10, sizeof(float *));
	
	float * h_var_20_11 = (float *)malloc(sizeof(float *));
	float * d_var_20_11;
	hipMalloc((void **)&d_var_20_11, sizeof(float *));
	
	float * h_var_20_12 = (float *)malloc(sizeof(float *));
	float * d_var_20_12;
	hipMalloc((void **)&d_var_20_12, sizeof(float *));
	
	float * h_var_20_13 = (float *)malloc(sizeof(float *));
	float * d_var_20_13;
	hipMalloc((void **)&d_var_20_13, sizeof(float *));
	
	float * h_var_20_14 = (float *)malloc(sizeof(float *));
	float * d_var_20_14;
	hipMalloc((void **)&d_var_20_14, sizeof(float *));
	
	float * h_var_20_15 = (float *)malloc(sizeof(float *));
	float * d_var_20_15;
	hipMalloc((void **)&d_var_20_15, sizeof(float *));
	
	float * h_var_20_16 = (float *)malloc(sizeof(float *));
	float * d_var_20_16;
	hipMalloc((void **)&d_var_20_16, sizeof(float *));
	
	float * h_var_20_17 = (float *)malloc(sizeof(float *));
	float * d_var_20_17;
	hipMalloc((void **)&d_var_20_17, sizeof(float *));
	
	float * h_var_20_18 = (float *)malloc(sizeof(float *));
	float * d_var_20_18;
	hipMalloc((void **)&d_var_20_18, sizeof(float *));
	
	float * h_var_20_19 = (float *)malloc(sizeof(float *));
	float * d_var_20_19;
	hipMalloc((void **)&d_var_20_19, sizeof(float *));
	
	float * h_var_21_0 = (float *)malloc(sizeof(float *));
	float * d_var_21_0;
	hipMalloc((void **)&d_var_21_0, sizeof(float *));
	
	float * h_var_21_1 = (float *)malloc(sizeof(float *));
	float * d_var_21_1;
	hipMalloc((void **)&d_var_21_1, sizeof(float *));
	
	float * h_var_21_2 = (float *)malloc(sizeof(float *));
	float * d_var_21_2;
	hipMalloc((void **)&d_var_21_2, sizeof(float *));
	
	float * h_var_21_3 = (float *)malloc(sizeof(float *));
	float * d_var_21_3;
	hipMalloc((void **)&d_var_21_3, sizeof(float *));
	
	float * h_var_21_4 = (float *)malloc(sizeof(float *));
	float * d_var_21_4;
	hipMalloc((void **)&d_var_21_4, sizeof(float *));
	
	float * h_var_21_5 = (float *)malloc(sizeof(float *));
	float * d_var_21_5;
	hipMalloc((void **)&d_var_21_5, sizeof(float *));
	
	float * h_var_21_6 = (float *)malloc(sizeof(float *));
	float * d_var_21_6;
	hipMalloc((void **)&d_var_21_6, sizeof(float *));
	
	float * h_var_21_7 = (float *)malloc(sizeof(float *));
	float * d_var_21_7;
	hipMalloc((void **)&d_var_21_7, sizeof(float *));
	
	float * h_var_21_8 = (float *)malloc(sizeof(float *));
	float * d_var_21_8;
	hipMalloc((void **)&d_var_21_8, sizeof(float *));
	
	float * h_var_21_9 = (float *)malloc(sizeof(float *));
	float * d_var_21_9;
	hipMalloc((void **)&d_var_21_9, sizeof(float *));
	
	float * h_var_21_10 = (float *)malloc(sizeof(float *));
	float * d_var_21_10;
	hipMalloc((void **)&d_var_21_10, sizeof(float *));
	
	float * h_var_21_11 = (float *)malloc(sizeof(float *));
	float * d_var_21_11;
	hipMalloc((void **)&d_var_21_11, sizeof(float *));
	
	float * h_var_21_12 = (float *)malloc(sizeof(float *));
	float * d_var_21_12;
	hipMalloc((void **)&d_var_21_12, sizeof(float *));
	
	float * h_var_21_13 = (float *)malloc(sizeof(float *));
	float * d_var_21_13;
	hipMalloc((void **)&d_var_21_13, sizeof(float *));
	
	float * h_var_21_14 = (float *)malloc(sizeof(float *));
	float * d_var_21_14;
	hipMalloc((void **)&d_var_21_14, sizeof(float *));
	
	float * h_var_21_15 = (float *)malloc(sizeof(float *));
	float * d_var_21_15;
	hipMalloc((void **)&d_var_21_15, sizeof(float *));
	
	float * h_var_21_16 = (float *)malloc(sizeof(float *));
	float * d_var_21_16;
	hipMalloc((void **)&d_var_21_16, sizeof(float *));
	
	float * h_var_21_17 = (float *)malloc(sizeof(float *));
	float * d_var_21_17;
	hipMalloc((void **)&d_var_21_17, sizeof(float *));
	
	float * h_var_21_18 = (float *)malloc(sizeof(float *));
	float * d_var_21_18;
	hipMalloc((void **)&d_var_21_18, sizeof(float *));
	
	float * h_var_21_19 = (float *)malloc(sizeof(float *));
	float * d_var_21_19;
	hipMalloc((void **)&d_var_21_19, sizeof(float *));
	
	float * h_var_22_0 = (float *)malloc(sizeof(float *));
	float * d_var_22_0;
	hipMalloc((void **)&d_var_22_0, sizeof(float *));
	
	float * h_var_22_1 = (float *)malloc(sizeof(float *));
	float * d_var_22_1;
	hipMalloc((void **)&d_var_22_1, sizeof(float *));
	
	float * h_var_22_2 = (float *)malloc(sizeof(float *));
	float * d_var_22_2;
	hipMalloc((void **)&d_var_22_2, sizeof(float *));
	
	float * h_var_22_3 = (float *)malloc(sizeof(float *));
	float * d_var_22_3;
	hipMalloc((void **)&d_var_22_3, sizeof(float *));
	
	float * h_var_22_4 = (float *)malloc(sizeof(float *));
	float * d_var_22_4;
	hipMalloc((void **)&d_var_22_4, sizeof(float *));
	
	float * h_var_22_5 = (float *)malloc(sizeof(float *));
	float * d_var_22_5;
	hipMalloc((void **)&d_var_22_5, sizeof(float *));
	
	float * h_var_22_6 = (float *)malloc(sizeof(float *));
	float * d_var_22_6;
	hipMalloc((void **)&d_var_22_6, sizeof(float *));
	
	float * h_var_22_7 = (float *)malloc(sizeof(float *));
	float * d_var_22_7;
	hipMalloc((void **)&d_var_22_7, sizeof(float *));
	
	float * h_var_22_8 = (float *)malloc(sizeof(float *));
	float * d_var_22_8;
	hipMalloc((void **)&d_var_22_8, sizeof(float *));
	
	float * h_var_22_9 = (float *)malloc(sizeof(float *));
	float * d_var_22_9;
	hipMalloc((void **)&d_var_22_9, sizeof(float *));
	
	float * h_var_22_10 = (float *)malloc(sizeof(float *));
	float * d_var_22_10;
	hipMalloc((void **)&d_var_22_10, sizeof(float *));
	
	float * h_var_22_11 = (float *)malloc(sizeof(float *));
	float * d_var_22_11;
	hipMalloc((void **)&d_var_22_11, sizeof(float *));
	
	float * h_var_22_12 = (float *)malloc(sizeof(float *));
	float * d_var_22_12;
	hipMalloc((void **)&d_var_22_12, sizeof(float *));
	
	float * h_var_22_13 = (float *)malloc(sizeof(float *));
	float * d_var_22_13;
	hipMalloc((void **)&d_var_22_13, sizeof(float *));
	
	float * h_var_22_14 = (float *)malloc(sizeof(float *));
	float * d_var_22_14;
	hipMalloc((void **)&d_var_22_14, sizeof(float *));
	
	float * h_var_22_15 = (float *)malloc(sizeof(float *));
	float * d_var_22_15;
	hipMalloc((void **)&d_var_22_15, sizeof(float *));
	
	float * h_var_22_16 = (float *)malloc(sizeof(float *));
	float * d_var_22_16;
	hipMalloc((void **)&d_var_22_16, sizeof(float *));
	
	float * h_var_22_17 = (float *)malloc(sizeof(float *));
	float * d_var_22_17;
	hipMalloc((void **)&d_var_22_17, sizeof(float *));
	
	float * h_var_22_18 = (float *)malloc(sizeof(float *));
	float * d_var_22_18;
	hipMalloc((void **)&d_var_22_18, sizeof(float *));
	
	float * h_var_22_19 = (float *)malloc(sizeof(float *));
	float * d_var_22_19;
	hipMalloc((void **)&d_var_22_19, sizeof(float *));
	
	float * h_var_23_0 = (float *)malloc(sizeof(float *));
	float * d_var_23_0;
	hipMalloc((void **)&d_var_23_0, sizeof(float *));
	
	float * h_var_23_1 = (float *)malloc(sizeof(float *));
	float * d_var_23_1;
	hipMalloc((void **)&d_var_23_1, sizeof(float *));
	
	float * h_var_23_2 = (float *)malloc(sizeof(float *));
	float * d_var_23_2;
	hipMalloc((void **)&d_var_23_2, sizeof(float *));
	
	float * h_var_23_3 = (float *)malloc(sizeof(float *));
	float * d_var_23_3;
	hipMalloc((void **)&d_var_23_3, sizeof(float *));
	
	float * h_var_23_4 = (float *)malloc(sizeof(float *));
	float * d_var_23_4;
	hipMalloc((void **)&d_var_23_4, sizeof(float *));
	
	float * h_var_23_5 = (float *)malloc(sizeof(float *));
	float * d_var_23_5;
	hipMalloc((void **)&d_var_23_5, sizeof(float *));
	
	float * h_var_23_6 = (float *)malloc(sizeof(float *));
	float * d_var_23_6;
	hipMalloc((void **)&d_var_23_6, sizeof(float *));
	
	float * h_var_23_7 = (float *)malloc(sizeof(float *));
	float * d_var_23_7;
	hipMalloc((void **)&d_var_23_7, sizeof(float *));
	
	float * h_var_23_8 = (float *)malloc(sizeof(float *));
	float * d_var_23_8;
	hipMalloc((void **)&d_var_23_8, sizeof(float *));
	
	float * h_var_23_9 = (float *)malloc(sizeof(float *));
	float * d_var_23_9;
	hipMalloc((void **)&d_var_23_9, sizeof(float *));
	
	float * h_var_23_10 = (float *)malloc(sizeof(float *));
	float * d_var_23_10;
	hipMalloc((void **)&d_var_23_10, sizeof(float *));
	
	float * h_var_23_11 = (float *)malloc(sizeof(float *));
	float * d_var_23_11;
	hipMalloc((void **)&d_var_23_11, sizeof(float *));
	
	float * h_var_23_12 = (float *)malloc(sizeof(float *));
	float * d_var_23_12;
	hipMalloc((void **)&d_var_23_12, sizeof(float *));
	
	float * h_var_23_13 = (float *)malloc(sizeof(float *));
	float * d_var_23_13;
	hipMalloc((void **)&d_var_23_13, sizeof(float *));
	
	float * h_var_23_14 = (float *)malloc(sizeof(float *));
	float * d_var_23_14;
	hipMalloc((void **)&d_var_23_14, sizeof(float *));
	
	float * h_var_23_15 = (float *)malloc(sizeof(float *));
	float * d_var_23_15;
	hipMalloc((void **)&d_var_23_15, sizeof(float *));
	
	float * h_var_23_16 = (float *)malloc(sizeof(float *));
	float * d_var_23_16;
	hipMalloc((void **)&d_var_23_16, sizeof(float *));
	
	float * h_var_23_17 = (float *)malloc(sizeof(float *));
	float * d_var_23_17;
	hipMalloc((void **)&d_var_23_17, sizeof(float *));
	
	float * h_var_23_18 = (float *)malloc(sizeof(float *));
	float * d_var_23_18;
	hipMalloc((void **)&d_var_23_18, sizeof(float *));
	
	float * h_var_23_19 = (float *)malloc(sizeof(float *));
	float * d_var_23_19;
	hipMalloc((void **)&d_var_23_19, sizeof(float *));
	
	float * h_var_24_0 = (float *)malloc(sizeof(float *));
	float * d_var_24_0;
	hipMalloc((void **)&d_var_24_0, sizeof(float *));
	
	float * h_var_24_1 = (float *)malloc(sizeof(float *));
	float * d_var_24_1;
	hipMalloc((void **)&d_var_24_1, sizeof(float *));
	
	float * h_var_24_2 = (float *)malloc(sizeof(float *));
	float * d_var_24_2;
	hipMalloc((void **)&d_var_24_2, sizeof(float *));
	
	float * h_var_24_3 = (float *)malloc(sizeof(float *));
	float * d_var_24_3;
	hipMalloc((void **)&d_var_24_3, sizeof(float *));
	
	float * h_var_24_4 = (float *)malloc(sizeof(float *));
	float * d_var_24_4;
	hipMalloc((void **)&d_var_24_4, sizeof(float *));
	
	float * h_var_24_5 = (float *)malloc(sizeof(float *));
	float * d_var_24_5;
	hipMalloc((void **)&d_var_24_5, sizeof(float *));
	
	float * h_var_24_6 = (float *)malloc(sizeof(float *));
	float * d_var_24_6;
	hipMalloc((void **)&d_var_24_6, sizeof(float *));
	
	float * h_var_24_7 = (float *)malloc(sizeof(float *));
	float * d_var_24_7;
	hipMalloc((void **)&d_var_24_7, sizeof(float *));
	
	float * h_var_24_8 = (float *)malloc(sizeof(float *));
	float * d_var_24_8;
	hipMalloc((void **)&d_var_24_8, sizeof(float *));
	
	float * h_var_24_9 = (float *)malloc(sizeof(float *));
	float * d_var_24_9;
	hipMalloc((void **)&d_var_24_9, sizeof(float *));
	
	float * h_var_24_10 = (float *)malloc(sizeof(float *));
	float * d_var_24_10;
	hipMalloc((void **)&d_var_24_10, sizeof(float *));
	
	float * h_var_24_11 = (float *)malloc(sizeof(float *));
	float * d_var_24_11;
	hipMalloc((void **)&d_var_24_11, sizeof(float *));
	
	float * h_var_24_12 = (float *)malloc(sizeof(float *));
	float * d_var_24_12;
	hipMalloc((void **)&d_var_24_12, sizeof(float *));
	
	float * h_var_24_13 = (float *)malloc(sizeof(float *));
	float * d_var_24_13;
	hipMalloc((void **)&d_var_24_13, sizeof(float *));
	
	float * h_var_24_14 = (float *)malloc(sizeof(float *));
	float * d_var_24_14;
	hipMalloc((void **)&d_var_24_14, sizeof(float *));
	
	float * h_var_24_15 = (float *)malloc(sizeof(float *));
	float * d_var_24_15;
	hipMalloc((void **)&d_var_24_15, sizeof(float *));
	
	float * h_var_24_16 = (float *)malloc(sizeof(float *));
	float * d_var_24_16;
	hipMalloc((void **)&d_var_24_16, sizeof(float *));
	
	float * h_var_24_17 = (float *)malloc(sizeof(float *));
	float * d_var_24_17;
	hipMalloc((void **)&d_var_24_17, sizeof(float *));
	
	float * h_var_24_18 = (float *)malloc(sizeof(float *));
	float * d_var_24_18;
	hipMalloc((void **)&d_var_24_18, sizeof(float *));
	
	float * h_var_24_19 = (float *)malloc(sizeof(float *));
	float * d_var_24_19;
	hipMalloc((void **)&d_var_24_19, sizeof(float *));
	
	float * h_var_25_0 = (float *)malloc(sizeof(float *));
	float * d_var_25_0;
	hipMalloc((void **)&d_var_25_0, sizeof(float *));
	
	float * h_var_25_1 = (float *)malloc(sizeof(float *));
	float * d_var_25_1;
	hipMalloc((void **)&d_var_25_1, sizeof(float *));
	
	float * h_var_25_2 = (float *)malloc(sizeof(float *));
	float * d_var_25_2;
	hipMalloc((void **)&d_var_25_2, sizeof(float *));
	
	float * h_var_25_3 = (float *)malloc(sizeof(float *));
	float * d_var_25_3;
	hipMalloc((void **)&d_var_25_3, sizeof(float *));
	
	float * h_var_25_4 = (float *)malloc(sizeof(float *));
	float * d_var_25_4;
	hipMalloc((void **)&d_var_25_4, sizeof(float *));
	
	float * h_var_25_5 = (float *)malloc(sizeof(float *));
	float * d_var_25_5;
	hipMalloc((void **)&d_var_25_5, sizeof(float *));
	
	float * h_var_25_6 = (float *)malloc(sizeof(float *));
	float * d_var_25_6;
	hipMalloc((void **)&d_var_25_6, sizeof(float *));
	
	float * h_var_25_7 = (float *)malloc(sizeof(float *));
	float * d_var_25_7;
	hipMalloc((void **)&d_var_25_7, sizeof(float *));
	
	float * h_var_25_8 = (float *)malloc(sizeof(float *));
	float * d_var_25_8;
	hipMalloc((void **)&d_var_25_8, sizeof(float *));
	
	float * h_var_25_9 = (float *)malloc(sizeof(float *));
	float * d_var_25_9;
	hipMalloc((void **)&d_var_25_9, sizeof(float *));
	
	float * h_var_25_10 = (float *)malloc(sizeof(float *));
	float * d_var_25_10;
	hipMalloc((void **)&d_var_25_10, sizeof(float *));
	
	float * h_var_25_11 = (float *)malloc(sizeof(float *));
	float * d_var_25_11;
	hipMalloc((void **)&d_var_25_11, sizeof(float *));
	
	float * h_var_25_12 = (float *)malloc(sizeof(float *));
	float * d_var_25_12;
	hipMalloc((void **)&d_var_25_12, sizeof(float *));
	
	float * h_var_25_13 = (float *)malloc(sizeof(float *));
	float * d_var_25_13;
	hipMalloc((void **)&d_var_25_13, sizeof(float *));
	
	float * h_var_25_14 = (float *)malloc(sizeof(float *));
	float * d_var_25_14;
	hipMalloc((void **)&d_var_25_14, sizeof(float *));
	
	float * h_var_25_15 = (float *)malloc(sizeof(float *));
	float * d_var_25_15;
	hipMalloc((void **)&d_var_25_15, sizeof(float *));
	
	float * h_var_25_16 = (float *)malloc(sizeof(float *));
	float * d_var_25_16;
	hipMalloc((void **)&d_var_25_16, sizeof(float *));
	
	float * h_var_25_17 = (float *)malloc(sizeof(float *));
	float * d_var_25_17;
	hipMalloc((void **)&d_var_25_17, sizeof(float *));
	
	float * h_var_25_18 = (float *)malloc(sizeof(float *));
	float * d_var_25_18;
	hipMalloc((void **)&d_var_25_18, sizeof(float *));
	
	float * h_var_25_19 = (float *)malloc(sizeof(float *));
	float * d_var_25_19;
	hipMalloc((void **)&d_var_25_19, sizeof(float *));
	
	float * h_var_26_0 = (float *)malloc(sizeof(float *));
	float * d_var_26_0;
	hipMalloc((void **)&d_var_26_0, sizeof(float *));
	
	float * h_var_26_1 = (float *)malloc(sizeof(float *));
	float * d_var_26_1;
	hipMalloc((void **)&d_var_26_1, sizeof(float *));
	
	float * h_var_26_2 = (float *)malloc(sizeof(float *));
	float * d_var_26_2;
	hipMalloc((void **)&d_var_26_2, sizeof(float *));
	
	float * h_var_26_3 = (float *)malloc(sizeof(float *));
	float * d_var_26_3;
	hipMalloc((void **)&d_var_26_3, sizeof(float *));
	
	float * h_var_26_4 = (float *)malloc(sizeof(float *));
	float * d_var_26_4;
	hipMalloc((void **)&d_var_26_4, sizeof(float *));
	
	float * h_var_26_5 = (float *)malloc(sizeof(float *));
	float * d_var_26_5;
	hipMalloc((void **)&d_var_26_5, sizeof(float *));
	
	float * h_var_26_6 = (float *)malloc(sizeof(float *));
	float * d_var_26_6;
	hipMalloc((void **)&d_var_26_6, sizeof(float *));
	
	float * h_var_26_7 = (float *)malloc(sizeof(float *));
	float * d_var_26_7;
	hipMalloc((void **)&d_var_26_7, sizeof(float *));
	
	float * h_var_26_8 = (float *)malloc(sizeof(float *));
	float * d_var_26_8;
	hipMalloc((void **)&d_var_26_8, sizeof(float *));
	
	float * h_var_26_9 = (float *)malloc(sizeof(float *));
	float * d_var_26_9;
	hipMalloc((void **)&d_var_26_9, sizeof(float *));
	
	float * h_var_26_10 = (float *)malloc(sizeof(float *));
	float * d_var_26_10;
	hipMalloc((void **)&d_var_26_10, sizeof(float *));
	
	float * h_var_26_11 = (float *)malloc(sizeof(float *));
	float * d_var_26_11;
	hipMalloc((void **)&d_var_26_11, sizeof(float *));
	
	float * h_var_26_12 = (float *)malloc(sizeof(float *));
	float * d_var_26_12;
	hipMalloc((void **)&d_var_26_12, sizeof(float *));
	
	float * h_var_26_13 = (float *)malloc(sizeof(float *));
	float * d_var_26_13;
	hipMalloc((void **)&d_var_26_13, sizeof(float *));
	
	float * h_var_26_14 = (float *)malloc(sizeof(float *));
	float * d_var_26_14;
	hipMalloc((void **)&d_var_26_14, sizeof(float *));
	
	float * h_var_26_15 = (float *)malloc(sizeof(float *));
	float * d_var_26_15;
	hipMalloc((void **)&d_var_26_15, sizeof(float *));
	
	float * h_var_26_16 = (float *)malloc(sizeof(float *));
	float * d_var_26_16;
	hipMalloc((void **)&d_var_26_16, sizeof(float *));
	
	float * h_var_26_17 = (float *)malloc(sizeof(float *));
	float * d_var_26_17;
	hipMalloc((void **)&d_var_26_17, sizeof(float *));
	
	float * h_var_26_18 = (float *)malloc(sizeof(float *));
	float * d_var_26_18;
	hipMalloc((void **)&d_var_26_18, sizeof(float *));
	
	float * h_var_26_19 = (float *)malloc(sizeof(float *));
	float * d_var_26_19;
	hipMalloc((void **)&d_var_26_19, sizeof(float *));
	
	float * h_var_27_0 = (float *)malloc(sizeof(float *));
	float * d_var_27_0;
	hipMalloc((void **)&d_var_27_0, sizeof(float *));
	
	float * h_var_27_1 = (float *)malloc(sizeof(float *));
	float * d_var_27_1;
	hipMalloc((void **)&d_var_27_1, sizeof(float *));
	
	float * h_var_27_2 = (float *)malloc(sizeof(float *));
	float * d_var_27_2;
	hipMalloc((void **)&d_var_27_2, sizeof(float *));
	
	float * h_var_27_3 = (float *)malloc(sizeof(float *));
	float * d_var_27_3;
	hipMalloc((void **)&d_var_27_3, sizeof(float *));
	
	float * h_var_27_4 = (float *)malloc(sizeof(float *));
	float * d_var_27_4;
	hipMalloc((void **)&d_var_27_4, sizeof(float *));
	
	float * h_var_27_5 = (float *)malloc(sizeof(float *));
	float * d_var_27_5;
	hipMalloc((void **)&d_var_27_5, sizeof(float *));
	
	float * h_var_27_6 = (float *)malloc(sizeof(float *));
	float * d_var_27_6;
	hipMalloc((void **)&d_var_27_6, sizeof(float *));
	
	float * h_var_27_7 = (float *)malloc(sizeof(float *));
	float * d_var_27_7;
	hipMalloc((void **)&d_var_27_7, sizeof(float *));
	
	float * h_var_27_8 = (float *)malloc(sizeof(float *));
	float * d_var_27_8;
	hipMalloc((void **)&d_var_27_8, sizeof(float *));
	
	float * h_var_27_9 = (float *)malloc(sizeof(float *));
	float * d_var_27_9;
	hipMalloc((void **)&d_var_27_9, sizeof(float *));
	
	float * h_var_27_10 = (float *)malloc(sizeof(float *));
	float * d_var_27_10;
	hipMalloc((void **)&d_var_27_10, sizeof(float *));
	
	float * h_var_27_11 = (float *)malloc(sizeof(float *));
	float * d_var_27_11;
	hipMalloc((void **)&d_var_27_11, sizeof(float *));
	
	float * h_var_27_12 = (float *)malloc(sizeof(float *));
	float * d_var_27_12;
	hipMalloc((void **)&d_var_27_12, sizeof(float *));
	
	float * h_var_27_13 = (float *)malloc(sizeof(float *));
	float * d_var_27_13;
	hipMalloc((void **)&d_var_27_13, sizeof(float *));
	
	float * h_var_27_14 = (float *)malloc(sizeof(float *));
	float * d_var_27_14;
	hipMalloc((void **)&d_var_27_14, sizeof(float *));
	
	float * h_var_27_15 = (float *)malloc(sizeof(float *));
	float * d_var_27_15;
	hipMalloc((void **)&d_var_27_15, sizeof(float *));
	
	float * h_var_27_16 = (float *)malloc(sizeof(float *));
	float * d_var_27_16;
	hipMalloc((void **)&d_var_27_16, sizeof(float *));
	
	float * h_var_27_17 = (float *)malloc(sizeof(float *));
	float * d_var_27_17;
	hipMalloc((void **)&d_var_27_17, sizeof(float *));
	
	float * h_var_27_18 = (float *)malloc(sizeof(float *));
	float * d_var_27_18;
	hipMalloc((void **)&d_var_27_18, sizeof(float *));
	
	float * h_var_27_19 = (float *)malloc(sizeof(float *));
	float * d_var_27_19;
	hipMalloc((void **)&d_var_27_19, sizeof(float *));
	
	float * h_var_28_0 = (float *)malloc(sizeof(float *));
	float * d_var_28_0;
	hipMalloc((void **)&d_var_28_0, sizeof(float *));
	
	float * h_var_28_1 = (float *)malloc(sizeof(float *));
	float * d_var_28_1;
	hipMalloc((void **)&d_var_28_1, sizeof(float *));
	
	float * h_var_28_2 = (float *)malloc(sizeof(float *));
	float * d_var_28_2;
	hipMalloc((void **)&d_var_28_2, sizeof(float *));
	
	float * h_var_28_3 = (float *)malloc(sizeof(float *));
	float * d_var_28_3;
	hipMalloc((void **)&d_var_28_3, sizeof(float *));
	
	float * h_var_28_4 = (float *)malloc(sizeof(float *));
	float * d_var_28_4;
	hipMalloc((void **)&d_var_28_4, sizeof(float *));
	
	float * h_var_28_5 = (float *)malloc(sizeof(float *));
	float * d_var_28_5;
	hipMalloc((void **)&d_var_28_5, sizeof(float *));
	
	float * h_var_28_6 = (float *)malloc(sizeof(float *));
	float * d_var_28_6;
	hipMalloc((void **)&d_var_28_6, sizeof(float *));
	
	float * h_var_28_7 = (float *)malloc(sizeof(float *));
	float * d_var_28_7;
	hipMalloc((void **)&d_var_28_7, sizeof(float *));
	
	float * h_var_28_8 = (float *)malloc(sizeof(float *));
	float * d_var_28_8;
	hipMalloc((void **)&d_var_28_8, sizeof(float *));
	
	float * h_var_28_9 = (float *)malloc(sizeof(float *));
	float * d_var_28_9;
	hipMalloc((void **)&d_var_28_9, sizeof(float *));
	
	float * h_var_28_10 = (float *)malloc(sizeof(float *));
	float * d_var_28_10;
	hipMalloc((void **)&d_var_28_10, sizeof(float *));
	
	float * h_var_28_11 = (float *)malloc(sizeof(float *));
	float * d_var_28_11;
	hipMalloc((void **)&d_var_28_11, sizeof(float *));
	
	float * h_var_28_12 = (float *)malloc(sizeof(float *));
	float * d_var_28_12;
	hipMalloc((void **)&d_var_28_12, sizeof(float *));
	
	float * h_var_28_13 = (float *)malloc(sizeof(float *));
	float * d_var_28_13;
	hipMalloc((void **)&d_var_28_13, sizeof(float *));
	
	float * h_var_28_14 = (float *)malloc(sizeof(float *));
	float * d_var_28_14;
	hipMalloc((void **)&d_var_28_14, sizeof(float *));
	
	float * h_var_28_15 = (float *)malloc(sizeof(float *));
	float * d_var_28_15;
	hipMalloc((void **)&d_var_28_15, sizeof(float *));
	
	float * h_var_28_16 = (float *)malloc(sizeof(float *));
	float * d_var_28_16;
	hipMalloc((void **)&d_var_28_16, sizeof(float *));
	
	float * h_var_28_17 = (float *)malloc(sizeof(float *));
	float * d_var_28_17;
	hipMalloc((void **)&d_var_28_17, sizeof(float *));
	
	float * h_var_28_18 = (float *)malloc(sizeof(float *));
	float * d_var_28_18;
	hipMalloc((void **)&d_var_28_18, sizeof(float *));
	
	float * h_var_28_19 = (float *)malloc(sizeof(float *));
	float * d_var_28_19;
	hipMalloc((void **)&d_var_28_19, sizeof(float *));
	
	float * h_var_29_0 = (float *)malloc(sizeof(float *));
	float * d_var_29_0;
	hipMalloc((void **)&d_var_29_0, sizeof(float *));
	
	float * h_var_29_1 = (float *)malloc(sizeof(float *));
	float * d_var_29_1;
	hipMalloc((void **)&d_var_29_1, sizeof(float *));
	
	float * h_var_29_2 = (float *)malloc(sizeof(float *));
	float * d_var_29_2;
	hipMalloc((void **)&d_var_29_2, sizeof(float *));
	
	float * h_var_29_3 = (float *)malloc(sizeof(float *));
	float * d_var_29_3;
	hipMalloc((void **)&d_var_29_3, sizeof(float *));
	
	float * h_var_29_4 = (float *)malloc(sizeof(float *));
	float * d_var_29_4;
	hipMalloc((void **)&d_var_29_4, sizeof(float *));
	
	float * h_var_29_5 = (float *)malloc(sizeof(float *));
	float * d_var_29_5;
	hipMalloc((void **)&d_var_29_5, sizeof(float *));
	
	float * h_var_29_6 = (float *)malloc(sizeof(float *));
	float * d_var_29_6;
	hipMalloc((void **)&d_var_29_6, sizeof(float *));
	
	float * h_var_29_7 = (float *)malloc(sizeof(float *));
	float * d_var_29_7;
	hipMalloc((void **)&d_var_29_7, sizeof(float *));
	
	float * h_var_29_8 = (float *)malloc(sizeof(float *));
	float * d_var_29_8;
	hipMalloc((void **)&d_var_29_8, sizeof(float *));
	
	float * h_var_29_9 = (float *)malloc(sizeof(float *));
	float * d_var_29_9;
	hipMalloc((void **)&d_var_29_9, sizeof(float *));
	
	float * h_var_29_10 = (float *)malloc(sizeof(float *));
	float * d_var_29_10;
	hipMalloc((void **)&d_var_29_10, sizeof(float *));
	
	float * h_var_29_11 = (float *)malloc(sizeof(float *));
	float * d_var_29_11;
	hipMalloc((void **)&d_var_29_11, sizeof(float *));
	
	float * h_var_29_12 = (float *)malloc(sizeof(float *));
	float * d_var_29_12;
	hipMalloc((void **)&d_var_29_12, sizeof(float *));
	
	float * h_var_29_13 = (float *)malloc(sizeof(float *));
	float * d_var_29_13;
	hipMalloc((void **)&d_var_29_13, sizeof(float *));
	
	float * h_var_29_14 = (float *)malloc(sizeof(float *));
	float * d_var_29_14;
	hipMalloc((void **)&d_var_29_14, sizeof(float *));
	
	float * h_var_29_15 = (float *)malloc(sizeof(float *));
	float * d_var_29_15;
	hipMalloc((void **)&d_var_29_15, sizeof(float *));
	
	float * h_var_29_16 = (float *)malloc(sizeof(float *));
	float * d_var_29_16;
	hipMalloc((void **)&d_var_29_16, sizeof(float *));
	
	float * h_var_29_17 = (float *)malloc(sizeof(float *));
	float * d_var_29_17;
	hipMalloc((void **)&d_var_29_17, sizeof(float *));
	
	float * h_var_29_18 = (float *)malloc(sizeof(float *));
	float * d_var_29_18;
	hipMalloc((void **)&d_var_29_18, sizeof(float *));
	
	float * h_var_29_19 = (float *)malloc(sizeof(float *));
	float * d_var_29_19;
	hipMalloc((void **)&d_var_29_19, sizeof(float *));
	
	float * h_var_30_0 = (float *)malloc(sizeof(float *));
	float * d_var_30_0;
	hipMalloc((void **)&d_var_30_0, sizeof(float *));
	
	float * h_var_30_1 = (float *)malloc(sizeof(float *));
	float * d_var_30_1;
	hipMalloc((void **)&d_var_30_1, sizeof(float *));
	
	float * h_var_30_2 = (float *)malloc(sizeof(float *));
	float * d_var_30_2;
	hipMalloc((void **)&d_var_30_2, sizeof(float *));
	
	float * h_var_30_3 = (float *)malloc(sizeof(float *));
	float * d_var_30_3;
	hipMalloc((void **)&d_var_30_3, sizeof(float *));
	
	float * h_var_30_4 = (float *)malloc(sizeof(float *));
	float * d_var_30_4;
	hipMalloc((void **)&d_var_30_4, sizeof(float *));
	
	float * h_var_30_5 = (float *)malloc(sizeof(float *));
	float * d_var_30_5;
	hipMalloc((void **)&d_var_30_5, sizeof(float *));
	
	float * h_var_30_6 = (float *)malloc(sizeof(float *));
	float * d_var_30_6;
	hipMalloc((void **)&d_var_30_6, sizeof(float *));
	
	float * h_var_30_7 = (float *)malloc(sizeof(float *));
	float * d_var_30_7;
	hipMalloc((void **)&d_var_30_7, sizeof(float *));
	
	float * h_var_30_8 = (float *)malloc(sizeof(float *));
	float * d_var_30_8;
	hipMalloc((void **)&d_var_30_8, sizeof(float *));
	
	float * h_var_30_9 = (float *)malloc(sizeof(float *));
	float * d_var_30_9;
	hipMalloc((void **)&d_var_30_9, sizeof(float *));
	
	float * h_var_30_10 = (float *)malloc(sizeof(float *));
	float * d_var_30_10;
	hipMalloc((void **)&d_var_30_10, sizeof(float *));
	
	float * h_var_30_11 = (float *)malloc(sizeof(float *));
	float * d_var_30_11;
	hipMalloc((void **)&d_var_30_11, sizeof(float *));
	
	float * h_var_30_12 = (float *)malloc(sizeof(float *));
	float * d_var_30_12;
	hipMalloc((void **)&d_var_30_12, sizeof(float *));
	
	float * h_var_30_13 = (float *)malloc(sizeof(float *));
	float * d_var_30_13;
	hipMalloc((void **)&d_var_30_13, sizeof(float *));
	
	float * h_var_30_14 = (float *)malloc(sizeof(float *));
	float * d_var_30_14;
	hipMalloc((void **)&d_var_30_14, sizeof(float *));
	
	float * h_var_30_15 = (float *)malloc(sizeof(float *));
	float * d_var_30_15;
	hipMalloc((void **)&d_var_30_15, sizeof(float *));
	
	float * h_var_30_16 = (float *)malloc(sizeof(float *));
	float * d_var_30_16;
	hipMalloc((void **)&d_var_30_16, sizeof(float *));
	
	float * h_var_30_17 = (float *)malloc(sizeof(float *));
	float * d_var_30_17;
	hipMalloc((void **)&d_var_30_17, sizeof(float *));
	
	float * h_var_30_18 = (float *)malloc(sizeof(float *));
	float * d_var_30_18;
	hipMalloc((void **)&d_var_30_18, sizeof(float *));
	
	float * h_var_30_19 = (float *)malloc(sizeof(float *));
	float * d_var_30_19;
	hipMalloc((void **)&d_var_30_19, sizeof(float *));
	
	float * h_var_31_0 = (float *)malloc(sizeof(float *));
	float * d_var_31_0;
	hipMalloc((void **)&d_var_31_0, sizeof(float *));
	
	float * h_var_31_1 = (float *)malloc(sizeof(float *));
	float * d_var_31_1;
	hipMalloc((void **)&d_var_31_1, sizeof(float *));
	
	float * h_var_31_2 = (float *)malloc(sizeof(float *));
	float * d_var_31_2;
	hipMalloc((void **)&d_var_31_2, sizeof(float *));
	
	float * h_var_31_3 = (float *)malloc(sizeof(float *));
	float * d_var_31_3;
	hipMalloc((void **)&d_var_31_3, sizeof(float *));
	
	float * h_var_31_4 = (float *)malloc(sizeof(float *));
	float * d_var_31_4;
	hipMalloc((void **)&d_var_31_4, sizeof(float *));
	
	float * h_var_31_5 = (float *)malloc(sizeof(float *));
	float * d_var_31_5;
	hipMalloc((void **)&d_var_31_5, sizeof(float *));
	
	float * h_var_31_6 = (float *)malloc(sizeof(float *));
	float * d_var_31_6;
	hipMalloc((void **)&d_var_31_6, sizeof(float *));
	
	float * h_var_31_7 = (float *)malloc(sizeof(float *));
	float * d_var_31_7;
	hipMalloc((void **)&d_var_31_7, sizeof(float *));
	
	float * h_var_31_8 = (float *)malloc(sizeof(float *));
	float * d_var_31_8;
	hipMalloc((void **)&d_var_31_8, sizeof(float *));
	
	float * h_var_31_9 = (float *)malloc(sizeof(float *));
	float * d_var_31_9;
	hipMalloc((void **)&d_var_31_9, sizeof(float *));
	
	float * h_var_31_10 = (float *)malloc(sizeof(float *));
	float * d_var_31_10;
	hipMalloc((void **)&d_var_31_10, sizeof(float *));
	
	float * h_var_31_11 = (float *)malloc(sizeof(float *));
	float * d_var_31_11;
	hipMalloc((void **)&d_var_31_11, sizeof(float *));
	
	float * h_var_31_12 = (float *)malloc(sizeof(float *));
	float * d_var_31_12;
	hipMalloc((void **)&d_var_31_12, sizeof(float *));
	
	float * h_var_31_13 = (float *)malloc(sizeof(float *));
	float * d_var_31_13;
	hipMalloc((void **)&d_var_31_13, sizeof(float *));
	
	float * h_var_31_14 = (float *)malloc(sizeof(float *));
	float * d_var_31_14;
	hipMalloc((void **)&d_var_31_14, sizeof(float *));
	
	float * h_var_31_15 = (float *)malloc(sizeof(float *));
	float * d_var_31_15;
	hipMalloc((void **)&d_var_31_15, sizeof(float *));
	
	float * h_var_31_16 = (float *)malloc(sizeof(float *));
	float * d_var_31_16;
	hipMalloc((void **)&d_var_31_16, sizeof(float *));
	
	float * h_var_31_17 = (float *)malloc(sizeof(float *));
	float * d_var_31_17;
	hipMalloc((void **)&d_var_31_17, sizeof(float *));
	
	float * h_var_31_18 = (float *)malloc(sizeof(float *));
	float * d_var_31_18;
	hipMalloc((void **)&d_var_31_18, sizeof(float *));
	
	float * h_var_31_19 = (float *)malloc(sizeof(float *));
	float * d_var_31_19;
	hipMalloc((void **)&d_var_31_19, sizeof(float *));
	
	float * h_var_32_0 = (float *)malloc(sizeof(float *));
	float * d_var_32_0;
	hipMalloc((void **)&d_var_32_0, sizeof(float *));
	
	float * h_var_32_1 = (float *)malloc(sizeof(float *));
	float * d_var_32_1;
	hipMalloc((void **)&d_var_32_1, sizeof(float *));
	
	float * h_var_32_2 = (float *)malloc(sizeof(float *));
	float * d_var_32_2;
	hipMalloc((void **)&d_var_32_2, sizeof(float *));
	
	float * h_var_32_3 = (float *)malloc(sizeof(float *));
	float * d_var_32_3;
	hipMalloc((void **)&d_var_32_3, sizeof(float *));
	
	float * h_var_32_4 = (float *)malloc(sizeof(float *));
	float * d_var_32_4;
	hipMalloc((void **)&d_var_32_4, sizeof(float *));
	
	float * h_var_32_5 = (float *)malloc(sizeof(float *));
	float * d_var_32_5;
	hipMalloc((void **)&d_var_32_5, sizeof(float *));
	
	float * h_var_32_6 = (float *)malloc(sizeof(float *));
	float * d_var_32_6;
	hipMalloc((void **)&d_var_32_6, sizeof(float *));
	
	float * h_var_32_7 = (float *)malloc(sizeof(float *));
	float * d_var_32_7;
	hipMalloc((void **)&d_var_32_7, sizeof(float *));
	
	float * h_var_32_8 = (float *)malloc(sizeof(float *));
	float * d_var_32_8;
	hipMalloc((void **)&d_var_32_8, sizeof(float *));
	
	float * h_var_32_9 = (float *)malloc(sizeof(float *));
	float * d_var_32_9;
	hipMalloc((void **)&d_var_32_9, sizeof(float *));
	
	float * h_var_32_10 = (float *)malloc(sizeof(float *));
	float * d_var_32_10;
	hipMalloc((void **)&d_var_32_10, sizeof(float *));
	
	float * h_var_32_11 = (float *)malloc(sizeof(float *));
	float * d_var_32_11;
	hipMalloc((void **)&d_var_32_11, sizeof(float *));
	
	float * h_var_32_12 = (float *)malloc(sizeof(float *));
	float * d_var_32_12;
	hipMalloc((void **)&d_var_32_12, sizeof(float *));
	
	float * h_var_32_13 = (float *)malloc(sizeof(float *));
	float * d_var_32_13;
	hipMalloc((void **)&d_var_32_13, sizeof(float *));
	
	float * h_var_32_14 = (float *)malloc(sizeof(float *));
	float * d_var_32_14;
	hipMalloc((void **)&d_var_32_14, sizeof(float *));
	
	float * h_var_32_15 = (float *)malloc(sizeof(float *));
	float * d_var_32_15;
	hipMalloc((void **)&d_var_32_15, sizeof(float *));
	
	float * h_var_32_16 = (float *)malloc(sizeof(float *));
	float * d_var_32_16;
	hipMalloc((void **)&d_var_32_16, sizeof(float *));
	
	float * h_var_32_17 = (float *)malloc(sizeof(float *));
	float * d_var_32_17;
	hipMalloc((void **)&d_var_32_17, sizeof(float *));
	
	float * h_var_32_18 = (float *)malloc(sizeof(float *));
	float * d_var_32_18;
	hipMalloc((void **)&d_var_32_18, sizeof(float *));
	
	float * h_var_32_19 = (float *)malloc(sizeof(float *));
	float * d_var_32_19;
	hipMalloc((void **)&d_var_32_19, sizeof(float *));
	
	float * h_var_33_0 = (float *)malloc(sizeof(float *));
	float * d_var_33_0;
	hipMalloc((void **)&d_var_33_0, sizeof(float *));
	
	float * h_var_33_1 = (float *)malloc(sizeof(float *));
	float * d_var_33_1;
	hipMalloc((void **)&d_var_33_1, sizeof(float *));
	
	float * h_var_33_2 = (float *)malloc(sizeof(float *));
	float * d_var_33_2;
	hipMalloc((void **)&d_var_33_2, sizeof(float *));
	
	float * h_var_33_3 = (float *)malloc(sizeof(float *));
	float * d_var_33_3;
	hipMalloc((void **)&d_var_33_3, sizeof(float *));
	
	float * h_var_33_4 = (float *)malloc(sizeof(float *));
	float * d_var_33_4;
	hipMalloc((void **)&d_var_33_4, sizeof(float *));
	
	float * h_var_33_5 = (float *)malloc(sizeof(float *));
	float * d_var_33_5;
	hipMalloc((void **)&d_var_33_5, sizeof(float *));
	
	float * h_var_33_6 = (float *)malloc(sizeof(float *));
	float * d_var_33_6;
	hipMalloc((void **)&d_var_33_6, sizeof(float *));
	
	float * h_var_33_7 = (float *)malloc(sizeof(float *));
	float * d_var_33_7;
	hipMalloc((void **)&d_var_33_7, sizeof(float *));
	
	float * h_var_33_8 = (float *)malloc(sizeof(float *));
	float * d_var_33_8;
	hipMalloc((void **)&d_var_33_8, sizeof(float *));
	
	float * h_var_33_9 = (float *)malloc(sizeof(float *));
	float * d_var_33_9;
	hipMalloc((void **)&d_var_33_9, sizeof(float *));
	
	float * h_var_33_10 = (float *)malloc(sizeof(float *));
	float * d_var_33_10;
	hipMalloc((void **)&d_var_33_10, sizeof(float *));
	
	float * h_var_33_11 = (float *)malloc(sizeof(float *));
	float * d_var_33_11;
	hipMalloc((void **)&d_var_33_11, sizeof(float *));
	
	float * h_var_33_12 = (float *)malloc(sizeof(float *));
	float * d_var_33_12;
	hipMalloc((void **)&d_var_33_12, sizeof(float *));
	
	float * h_var_33_13 = (float *)malloc(sizeof(float *));
	float * d_var_33_13;
	hipMalloc((void **)&d_var_33_13, sizeof(float *));
	
	float * h_var_33_14 = (float *)malloc(sizeof(float *));
	float * d_var_33_14;
	hipMalloc((void **)&d_var_33_14, sizeof(float *));
	
	float * h_var_33_15 = (float *)malloc(sizeof(float *));
	float * d_var_33_15;
	hipMalloc((void **)&d_var_33_15, sizeof(float *));
	
	float * h_var_33_16 = (float *)malloc(sizeof(float *));
	float * d_var_33_16;
	hipMalloc((void **)&d_var_33_16, sizeof(float *));
	
	float * h_var_33_17 = (float *)malloc(sizeof(float *));
	float * d_var_33_17;
	hipMalloc((void **)&d_var_33_17, sizeof(float *));
	
	float * h_var_33_18 = (float *)malloc(sizeof(float *));
	float * d_var_33_18;
	hipMalloc((void **)&d_var_33_18, sizeof(float *));
	
	float * h_var_33_19 = (float *)malloc(sizeof(float *));
	float * d_var_33_19;
	hipMalloc((void **)&d_var_33_19, sizeof(float *));
	
	float * h_var_34_0 = (float *)malloc(sizeof(float *));
	float * d_var_34_0;
	hipMalloc((void **)&d_var_34_0, sizeof(float *));
	
	float * h_var_34_1 = (float *)malloc(sizeof(float *));
	float * d_var_34_1;
	hipMalloc((void **)&d_var_34_1, sizeof(float *));
	
	float * h_var_34_2 = (float *)malloc(sizeof(float *));
	float * d_var_34_2;
	hipMalloc((void **)&d_var_34_2, sizeof(float *));
	
	float * h_var_34_3 = (float *)malloc(sizeof(float *));
	float * d_var_34_3;
	hipMalloc((void **)&d_var_34_3, sizeof(float *));
	
	float * h_var_34_4 = (float *)malloc(sizeof(float *));
	float * d_var_34_4;
	hipMalloc((void **)&d_var_34_4, sizeof(float *));
	
	float * h_var_34_5 = (float *)malloc(sizeof(float *));
	float * d_var_34_5;
	hipMalloc((void **)&d_var_34_5, sizeof(float *));
	
	float * h_var_34_6 = (float *)malloc(sizeof(float *));
	float * d_var_34_6;
	hipMalloc((void **)&d_var_34_6, sizeof(float *));
	
	float * h_var_34_7 = (float *)malloc(sizeof(float *));
	float * d_var_34_7;
	hipMalloc((void **)&d_var_34_7, sizeof(float *));
	
	float * h_var_34_8 = (float *)malloc(sizeof(float *));
	float * d_var_34_8;
	hipMalloc((void **)&d_var_34_8, sizeof(float *));
	
	float * h_var_34_9 = (float *)malloc(sizeof(float *));
	float * d_var_34_9;
	hipMalloc((void **)&d_var_34_9, sizeof(float *));
	
	float * h_var_34_10 = (float *)malloc(sizeof(float *));
	float * d_var_34_10;
	hipMalloc((void **)&d_var_34_10, sizeof(float *));
	
	float * h_var_34_11 = (float *)malloc(sizeof(float *));
	float * d_var_34_11;
	hipMalloc((void **)&d_var_34_11, sizeof(float *));
	
	float * h_var_34_12 = (float *)malloc(sizeof(float *));
	float * d_var_34_12;
	hipMalloc((void **)&d_var_34_12, sizeof(float *));
	
	float * h_var_34_13 = (float *)malloc(sizeof(float *));
	float * d_var_34_13;
	hipMalloc((void **)&d_var_34_13, sizeof(float *));
	
	float * h_var_34_14 = (float *)malloc(sizeof(float *));
	float * d_var_34_14;
	hipMalloc((void **)&d_var_34_14, sizeof(float *));
	
	float * h_var_34_15 = (float *)malloc(sizeof(float *));
	float * d_var_34_15;
	hipMalloc((void **)&d_var_34_15, sizeof(float *));
	
	float * h_var_34_16 = (float *)malloc(sizeof(float *));
	float * d_var_34_16;
	hipMalloc((void **)&d_var_34_16, sizeof(float *));
	
	float * h_var_34_17 = (float *)malloc(sizeof(float *));
	float * d_var_34_17;
	hipMalloc((void **)&d_var_34_17, sizeof(float *));
	
	float * h_var_34_18 = (float *)malloc(sizeof(float *));
	float * d_var_34_18;
	hipMalloc((void **)&d_var_34_18, sizeof(float *));
	
	float * h_var_34_19 = (float *)malloc(sizeof(float *));
	float * d_var_34_19;
	hipMalloc((void **)&d_var_34_19, sizeof(float *));
	
	float * h_var_35_0 = (float *)malloc(sizeof(float *));
	float * d_var_35_0;
	hipMalloc((void **)&d_var_35_0, sizeof(float *));
	
	float * h_var_35_1 = (float *)malloc(sizeof(float *));
	float * d_var_35_1;
	hipMalloc((void **)&d_var_35_1, sizeof(float *));
	
	float * h_var_35_2 = (float *)malloc(sizeof(float *));
	float * d_var_35_2;
	hipMalloc((void **)&d_var_35_2, sizeof(float *));
	
	float * h_var_35_3 = (float *)malloc(sizeof(float *));
	float * d_var_35_3;
	hipMalloc((void **)&d_var_35_3, sizeof(float *));
	
	float * h_var_35_4 = (float *)malloc(sizeof(float *));
	float * d_var_35_4;
	hipMalloc((void **)&d_var_35_4, sizeof(float *));
	
	float * h_var_35_5 = (float *)malloc(sizeof(float *));
	float * d_var_35_5;
	hipMalloc((void **)&d_var_35_5, sizeof(float *));
	
	float * h_var_35_6 = (float *)malloc(sizeof(float *));
	float * d_var_35_6;
	hipMalloc((void **)&d_var_35_6, sizeof(float *));
	
	float * h_var_35_7 = (float *)malloc(sizeof(float *));
	float * d_var_35_7;
	hipMalloc((void **)&d_var_35_7, sizeof(float *));
	
	float * h_var_35_8 = (float *)malloc(sizeof(float *));
	float * d_var_35_8;
	hipMalloc((void **)&d_var_35_8, sizeof(float *));
	
	float * h_var_35_9 = (float *)malloc(sizeof(float *));
	float * d_var_35_9;
	hipMalloc((void **)&d_var_35_9, sizeof(float *));
	
	float * h_var_35_10 = (float *)malloc(sizeof(float *));
	float * d_var_35_10;
	hipMalloc((void **)&d_var_35_10, sizeof(float *));
	
	float * h_var_35_11 = (float *)malloc(sizeof(float *));
	float * d_var_35_11;
	hipMalloc((void **)&d_var_35_11, sizeof(float *));
	
	float * h_var_35_12 = (float *)malloc(sizeof(float *));
	float * d_var_35_12;
	hipMalloc((void **)&d_var_35_12, sizeof(float *));
	
	float * h_var_35_13 = (float *)malloc(sizeof(float *));
	float * d_var_35_13;
	hipMalloc((void **)&d_var_35_13, sizeof(float *));
	
	float * h_var_35_14 = (float *)malloc(sizeof(float *));
	float * d_var_35_14;
	hipMalloc((void **)&d_var_35_14, sizeof(float *));
	
	float * h_var_35_15 = (float *)malloc(sizeof(float *));
	float * d_var_35_15;
	hipMalloc((void **)&d_var_35_15, sizeof(float *));
	
	float * h_var_35_16 = (float *)malloc(sizeof(float *));
	float * d_var_35_16;
	hipMalloc((void **)&d_var_35_16, sizeof(float *));
	
	float * h_var_35_17 = (float *)malloc(sizeof(float *));
	float * d_var_35_17;
	hipMalloc((void **)&d_var_35_17, sizeof(float *));
	
	float * h_var_35_18 = (float *)malloc(sizeof(float *));
	float * d_var_35_18;
	hipMalloc((void **)&d_var_35_18, sizeof(float *));
	
	float * h_var_35_19 = (float *)malloc(sizeof(float *));
	float * d_var_35_19;
	hipMalloc((void **)&d_var_35_19, sizeof(float *));
	
	float * h_var_36_0 = (float *)malloc(sizeof(float *));
	float * d_var_36_0;
	hipMalloc((void **)&d_var_36_0, sizeof(float *));
	
	float * h_var_36_1 = (float *)malloc(sizeof(float *));
	float * d_var_36_1;
	hipMalloc((void **)&d_var_36_1, sizeof(float *));
	
	float * h_var_36_2 = (float *)malloc(sizeof(float *));
	float * d_var_36_2;
	hipMalloc((void **)&d_var_36_2, sizeof(float *));
	
	float * h_var_36_3 = (float *)malloc(sizeof(float *));
	float * d_var_36_3;
	hipMalloc((void **)&d_var_36_3, sizeof(float *));
	
	float * h_var_36_4 = (float *)malloc(sizeof(float *));
	float * d_var_36_4;
	hipMalloc((void **)&d_var_36_4, sizeof(float *));
	
	float * h_var_36_5 = (float *)malloc(sizeof(float *));
	float * d_var_36_5;
	hipMalloc((void **)&d_var_36_5, sizeof(float *));
	
	float * h_var_36_6 = (float *)malloc(sizeof(float *));
	float * d_var_36_6;
	hipMalloc((void **)&d_var_36_6, sizeof(float *));
	
	float * h_var_36_7 = (float *)malloc(sizeof(float *));
	float * d_var_36_7;
	hipMalloc((void **)&d_var_36_7, sizeof(float *));
	
	float * h_var_36_8 = (float *)malloc(sizeof(float *));
	float * d_var_36_8;
	hipMalloc((void **)&d_var_36_8, sizeof(float *));
	
	float * h_var_36_9 = (float *)malloc(sizeof(float *));
	float * d_var_36_9;
	hipMalloc((void **)&d_var_36_9, sizeof(float *));
	
	float * h_var_36_10 = (float *)malloc(sizeof(float *));
	float * d_var_36_10;
	hipMalloc((void **)&d_var_36_10, sizeof(float *));
	
	float * h_var_36_11 = (float *)malloc(sizeof(float *));
	float * d_var_36_11;
	hipMalloc((void **)&d_var_36_11, sizeof(float *));
	
	float * h_var_36_12 = (float *)malloc(sizeof(float *));
	float * d_var_36_12;
	hipMalloc((void **)&d_var_36_12, sizeof(float *));
	
	float * h_var_36_13 = (float *)malloc(sizeof(float *));
	float * d_var_36_13;
	hipMalloc((void **)&d_var_36_13, sizeof(float *));
	
	float * h_var_36_14 = (float *)malloc(sizeof(float *));
	float * d_var_36_14;
	hipMalloc((void **)&d_var_36_14, sizeof(float *));
	
	float * h_var_36_15 = (float *)malloc(sizeof(float *));
	float * d_var_36_15;
	hipMalloc((void **)&d_var_36_15, sizeof(float *));
	
	float * h_var_36_16 = (float *)malloc(sizeof(float *));
	float * d_var_36_16;
	hipMalloc((void **)&d_var_36_16, sizeof(float *));
	
	float * h_var_36_17 = (float *)malloc(sizeof(float *));
	float * d_var_36_17;
	hipMalloc((void **)&d_var_36_17, sizeof(float *));
	
	float * h_var_36_18 = (float *)malloc(sizeof(float *));
	float * d_var_36_18;
	hipMalloc((void **)&d_var_36_18, sizeof(float *));
	
	float * h_var_36_19 = (float *)malloc(sizeof(float *));
	float * d_var_36_19;
	hipMalloc((void **)&d_var_36_19, sizeof(float *));
	
	float * h_var_37_0 = (float *)malloc(sizeof(float *));
	float * d_var_37_0;
	hipMalloc((void **)&d_var_37_0, sizeof(float *));
	
	float * h_var_37_1 = (float *)malloc(sizeof(float *));
	float * d_var_37_1;
	hipMalloc((void **)&d_var_37_1, sizeof(float *));
	
	float * h_var_37_2 = (float *)malloc(sizeof(float *));
	float * d_var_37_2;
	hipMalloc((void **)&d_var_37_2, sizeof(float *));
	
	float * h_var_37_3 = (float *)malloc(sizeof(float *));
	float * d_var_37_3;
	hipMalloc((void **)&d_var_37_3, sizeof(float *));
	
	float * h_var_37_4 = (float *)malloc(sizeof(float *));
	float * d_var_37_4;
	hipMalloc((void **)&d_var_37_4, sizeof(float *));
	
	float * h_var_37_5 = (float *)malloc(sizeof(float *));
	float * d_var_37_5;
	hipMalloc((void **)&d_var_37_5, sizeof(float *));
	
	float * h_var_37_6 = (float *)malloc(sizeof(float *));
	float * d_var_37_6;
	hipMalloc((void **)&d_var_37_6, sizeof(float *));
	
	float * h_var_37_7 = (float *)malloc(sizeof(float *));
	float * d_var_37_7;
	hipMalloc((void **)&d_var_37_7, sizeof(float *));
	
	float * h_var_37_8 = (float *)malloc(sizeof(float *));
	float * d_var_37_8;
	hipMalloc((void **)&d_var_37_8, sizeof(float *));
	
	float * h_var_37_9 = (float *)malloc(sizeof(float *));
	float * d_var_37_9;
	hipMalloc((void **)&d_var_37_9, sizeof(float *));
	
	float * h_var_37_10 = (float *)malloc(sizeof(float *));
	float * d_var_37_10;
	hipMalloc((void **)&d_var_37_10, sizeof(float *));
	
	float * h_var_37_11 = (float *)malloc(sizeof(float *));
	float * d_var_37_11;
	hipMalloc((void **)&d_var_37_11, sizeof(float *));
	
	float * h_var_37_12 = (float *)malloc(sizeof(float *));
	float * d_var_37_12;
	hipMalloc((void **)&d_var_37_12, sizeof(float *));
	
	float * h_var_37_13 = (float *)malloc(sizeof(float *));
	float * d_var_37_13;
	hipMalloc((void **)&d_var_37_13, sizeof(float *));
	
	float * h_var_37_14 = (float *)malloc(sizeof(float *));
	float * d_var_37_14;
	hipMalloc((void **)&d_var_37_14, sizeof(float *));
	
	float * h_var_37_15 = (float *)malloc(sizeof(float *));
	float * d_var_37_15;
	hipMalloc((void **)&d_var_37_15, sizeof(float *));
	
	float * h_var_37_16 = (float *)malloc(sizeof(float *));
	float * d_var_37_16;
	hipMalloc((void **)&d_var_37_16, sizeof(float *));
	
	float * h_var_37_17 = (float *)malloc(sizeof(float *));
	float * d_var_37_17;
	hipMalloc((void **)&d_var_37_17, sizeof(float *));
	
	float * h_var_37_18 = (float *)malloc(sizeof(float *));
	float * d_var_37_18;
	hipMalloc((void **)&d_var_37_18, sizeof(float *));
	
	float * h_var_37_19 = (float *)malloc(sizeof(float *));
	float * d_var_37_19;
	hipMalloc((void **)&d_var_37_19, sizeof(float *));
	
	float * h_var_38_0 = (float *)malloc(sizeof(float *));
	float * d_var_38_0;
	hipMalloc((void **)&d_var_38_0, sizeof(float *));
	
	float * h_var_38_1 = (float *)malloc(sizeof(float *));
	float * d_var_38_1;
	hipMalloc((void **)&d_var_38_1, sizeof(float *));
	
	float * h_var_38_2 = (float *)malloc(sizeof(float *));
	float * d_var_38_2;
	hipMalloc((void **)&d_var_38_2, sizeof(float *));
	
	float * h_var_38_3 = (float *)malloc(sizeof(float *));
	float * d_var_38_3;
	hipMalloc((void **)&d_var_38_3, sizeof(float *));
	
	float * h_var_38_4 = (float *)malloc(sizeof(float *));
	float * d_var_38_4;
	hipMalloc((void **)&d_var_38_4, sizeof(float *));
	
	float * h_var_38_5 = (float *)malloc(sizeof(float *));
	float * d_var_38_5;
	hipMalloc((void **)&d_var_38_5, sizeof(float *));
	
	float * h_var_38_6 = (float *)malloc(sizeof(float *));
	float * d_var_38_6;
	hipMalloc((void **)&d_var_38_6, sizeof(float *));
	
	float * h_var_38_7 = (float *)malloc(sizeof(float *));
	float * d_var_38_7;
	hipMalloc((void **)&d_var_38_7, sizeof(float *));
	
	float * h_var_38_8 = (float *)malloc(sizeof(float *));
	float * d_var_38_8;
	hipMalloc((void **)&d_var_38_8, sizeof(float *));
	
	float * h_var_38_9 = (float *)malloc(sizeof(float *));
	float * d_var_38_9;
	hipMalloc((void **)&d_var_38_9, sizeof(float *));
	
	float * h_var_38_10 = (float *)malloc(sizeof(float *));
	float * d_var_38_10;
	hipMalloc((void **)&d_var_38_10, sizeof(float *));
	
	float * h_var_38_11 = (float *)malloc(sizeof(float *));
	float * d_var_38_11;
	hipMalloc((void **)&d_var_38_11, sizeof(float *));
	
	float * h_var_38_12 = (float *)malloc(sizeof(float *));
	float * d_var_38_12;
	hipMalloc((void **)&d_var_38_12, sizeof(float *));
	
	float * h_var_38_13 = (float *)malloc(sizeof(float *));
	float * d_var_38_13;
	hipMalloc((void **)&d_var_38_13, sizeof(float *));
	
	float * h_var_38_14 = (float *)malloc(sizeof(float *));
	float * d_var_38_14;
	hipMalloc((void **)&d_var_38_14, sizeof(float *));
	
	float * h_var_38_15 = (float *)malloc(sizeof(float *));
	float * d_var_38_15;
	hipMalloc((void **)&d_var_38_15, sizeof(float *));
	
	float * h_var_38_16 = (float *)malloc(sizeof(float *));
	float * d_var_38_16;
	hipMalloc((void **)&d_var_38_16, sizeof(float *));
	
	float * h_var_38_17 = (float *)malloc(sizeof(float *));
	float * d_var_38_17;
	hipMalloc((void **)&d_var_38_17, sizeof(float *));
	
	float * h_var_38_18 = (float *)malloc(sizeof(float *));
	float * d_var_38_18;
	hipMalloc((void **)&d_var_38_18, sizeof(float *));
	
	float * h_var_38_19 = (float *)malloc(sizeof(float *));
	float * d_var_38_19;
	hipMalloc((void **)&d_var_38_19, sizeof(float *));
	
	float * h_var_39_0 = (float *)malloc(sizeof(float *));
	float * d_var_39_0;
	hipMalloc((void **)&d_var_39_0, sizeof(float *));
	
	float * h_var_39_1 = (float *)malloc(sizeof(float *));
	float * d_var_39_1;
	hipMalloc((void **)&d_var_39_1, sizeof(float *));
	
	float * h_var_39_2 = (float *)malloc(sizeof(float *));
	float * d_var_39_2;
	hipMalloc((void **)&d_var_39_2, sizeof(float *));
	
	float * h_var_39_3 = (float *)malloc(sizeof(float *));
	float * d_var_39_3;
	hipMalloc((void **)&d_var_39_3, sizeof(float *));
	
	float * h_var_39_4 = (float *)malloc(sizeof(float *));
	float * d_var_39_4;
	hipMalloc((void **)&d_var_39_4, sizeof(float *));
	
	float * h_var_39_5 = (float *)malloc(sizeof(float *));
	float * d_var_39_5;
	hipMalloc((void **)&d_var_39_5, sizeof(float *));
	
	float * h_var_39_6 = (float *)malloc(sizeof(float *));
	float * d_var_39_6;
	hipMalloc((void **)&d_var_39_6, sizeof(float *));
	
	float * h_var_39_7 = (float *)malloc(sizeof(float *));
	float * d_var_39_7;
	hipMalloc((void **)&d_var_39_7, sizeof(float *));
	
	float * h_var_39_8 = (float *)malloc(sizeof(float *));
	float * d_var_39_8;
	hipMalloc((void **)&d_var_39_8, sizeof(float *));
	
	float * h_var_39_9 = (float *)malloc(sizeof(float *));
	float * d_var_39_9;
	hipMalloc((void **)&d_var_39_9, sizeof(float *));
	
	float * h_var_39_10 = (float *)malloc(sizeof(float *));
	float * d_var_39_10;
	hipMalloc((void **)&d_var_39_10, sizeof(float *));
	
	float * h_var_39_11 = (float *)malloc(sizeof(float *));
	float * d_var_39_11;
	hipMalloc((void **)&d_var_39_11, sizeof(float *));
	
	float * h_var_39_12 = (float *)malloc(sizeof(float *));
	float * d_var_39_12;
	hipMalloc((void **)&d_var_39_12, sizeof(float *));
	
	float * h_var_39_13 = (float *)malloc(sizeof(float *));
	float * d_var_39_13;
	hipMalloc((void **)&d_var_39_13, sizeof(float *));
	
	float * h_var_39_14 = (float *)malloc(sizeof(float *));
	float * d_var_39_14;
	hipMalloc((void **)&d_var_39_14, sizeof(float *));
	
	float * h_var_39_15 = (float *)malloc(sizeof(float *));
	float * d_var_39_15;
	hipMalloc((void **)&d_var_39_15, sizeof(float *));
	
	float * h_var_39_16 = (float *)malloc(sizeof(float *));
	float * d_var_39_16;
	hipMalloc((void **)&d_var_39_16, sizeof(float *));
	
	float * h_var_39_17 = (float *)malloc(sizeof(float *));
	float * d_var_39_17;
	hipMalloc((void **)&d_var_39_17, sizeof(float *));
	
	float * h_var_39_18 = (float *)malloc(sizeof(float *));
	float * d_var_39_18;
	hipMalloc((void **)&d_var_39_18, sizeof(float *));
	
	float * h_var_39_19 = (float *)malloc(sizeof(float *));
	float * d_var_39_19;
	hipMalloc((void **)&d_var_39_19, sizeof(float *));
	
	float * h_var_40_0 = (float *)malloc(sizeof(float *));
	float * d_var_40_0;
	hipMalloc((void **)&d_var_40_0, sizeof(float *));
	
	float * h_var_40_1 = (float *)malloc(sizeof(float *));
	float * d_var_40_1;
	hipMalloc((void **)&d_var_40_1, sizeof(float *));
	
	float * h_var_40_2 = (float *)malloc(sizeof(float *));
	float * d_var_40_2;
	hipMalloc((void **)&d_var_40_2, sizeof(float *));
	
	float * h_var_40_3 = (float *)malloc(sizeof(float *));
	float * d_var_40_3;
	hipMalloc((void **)&d_var_40_3, sizeof(float *));
	
	float * h_var_40_4 = (float *)malloc(sizeof(float *));
	float * d_var_40_4;
	hipMalloc((void **)&d_var_40_4, sizeof(float *));
	
	float * h_var_40_5 = (float *)malloc(sizeof(float *));
	float * d_var_40_5;
	hipMalloc((void **)&d_var_40_5, sizeof(float *));
	
	float * h_var_40_6 = (float *)malloc(sizeof(float *));
	float * d_var_40_6;
	hipMalloc((void **)&d_var_40_6, sizeof(float *));
	
	float * h_var_40_7 = (float *)malloc(sizeof(float *));
	float * d_var_40_7;
	hipMalloc((void **)&d_var_40_7, sizeof(float *));
	
	float * h_var_40_8 = (float *)malloc(sizeof(float *));
	float * d_var_40_8;
	hipMalloc((void **)&d_var_40_8, sizeof(float *));
	
	float * h_var_40_9 = (float *)malloc(sizeof(float *));
	float * d_var_40_9;
	hipMalloc((void **)&d_var_40_9, sizeof(float *));
	
	float * h_var_40_10 = (float *)malloc(sizeof(float *));
	float * d_var_40_10;
	hipMalloc((void **)&d_var_40_10, sizeof(float *));
	
	float * h_var_40_11 = (float *)malloc(sizeof(float *));
	float * d_var_40_11;
	hipMalloc((void **)&d_var_40_11, sizeof(float *));
	
	float * h_var_40_12 = (float *)malloc(sizeof(float *));
	float * d_var_40_12;
	hipMalloc((void **)&d_var_40_12, sizeof(float *));
	
	float * h_var_40_13 = (float *)malloc(sizeof(float *));
	float * d_var_40_13;
	hipMalloc((void **)&d_var_40_13, sizeof(float *));
	
	float * h_var_40_14 = (float *)malloc(sizeof(float *));
	float * d_var_40_14;
	hipMalloc((void **)&d_var_40_14, sizeof(float *));
	
	float * h_var_40_15 = (float *)malloc(sizeof(float *));
	float * d_var_40_15;
	hipMalloc((void **)&d_var_40_15, sizeof(float *));
	
	float * h_var_40_16 = (float *)malloc(sizeof(float *));
	float * d_var_40_16;
	hipMalloc((void **)&d_var_40_16, sizeof(float *));
	
	float * h_var_40_17 = (float *)malloc(sizeof(float *));
	float * d_var_40_17;
	hipMalloc((void **)&d_var_40_17, sizeof(float *));
	
	float * h_var_40_18 = (float *)malloc(sizeof(float *));
	float * d_var_40_18;
	hipMalloc((void **)&d_var_40_18, sizeof(float *));
	
	float * h_var_40_19 = (float *)malloc(sizeof(float *));
	float * d_var_40_19;
	hipMalloc((void **)&d_var_40_19, sizeof(float *));
	
	float * h_var_41_0 = (float *)malloc(sizeof(float *));
	float * d_var_41_0;
	hipMalloc((void **)&d_var_41_0, sizeof(float *));
	
	float * h_var_41_1 = (float *)malloc(sizeof(float *));
	float * d_var_41_1;
	hipMalloc((void **)&d_var_41_1, sizeof(float *));
	
	float * h_var_41_2 = (float *)malloc(sizeof(float *));
	float * d_var_41_2;
	hipMalloc((void **)&d_var_41_2, sizeof(float *));
	
	float * h_var_41_3 = (float *)malloc(sizeof(float *));
	float * d_var_41_3;
	hipMalloc((void **)&d_var_41_3, sizeof(float *));
	
	float * h_var_41_4 = (float *)malloc(sizeof(float *));
	float * d_var_41_4;
	hipMalloc((void **)&d_var_41_4, sizeof(float *));
	
	float * h_var_41_5 = (float *)malloc(sizeof(float *));
	float * d_var_41_5;
	hipMalloc((void **)&d_var_41_5, sizeof(float *));
	
	float * h_var_41_6 = (float *)malloc(sizeof(float *));
	float * d_var_41_6;
	hipMalloc((void **)&d_var_41_6, sizeof(float *));
	
	float * h_var_41_7 = (float *)malloc(sizeof(float *));
	float * d_var_41_7;
	hipMalloc((void **)&d_var_41_7, sizeof(float *));
	
	float * h_var_41_8 = (float *)malloc(sizeof(float *));
	float * d_var_41_8;
	hipMalloc((void **)&d_var_41_8, sizeof(float *));
	
	float * h_var_41_9 = (float *)malloc(sizeof(float *));
	float * d_var_41_9;
	hipMalloc((void **)&d_var_41_9, sizeof(float *));
	
	float * h_var_41_10 = (float *)malloc(sizeof(float *));
	float * d_var_41_10;
	hipMalloc((void **)&d_var_41_10, sizeof(float *));
	
	float * h_var_41_11 = (float *)malloc(sizeof(float *));
	float * d_var_41_11;
	hipMalloc((void **)&d_var_41_11, sizeof(float *));
	
	float * h_var_41_12 = (float *)malloc(sizeof(float *));
	float * d_var_41_12;
	hipMalloc((void **)&d_var_41_12, sizeof(float *));
	
	float * h_var_41_13 = (float *)malloc(sizeof(float *));
	float * d_var_41_13;
	hipMalloc((void **)&d_var_41_13, sizeof(float *));
	
	float * h_var_41_14 = (float *)malloc(sizeof(float *));
	float * d_var_41_14;
	hipMalloc((void **)&d_var_41_14, sizeof(float *));
	
	float * h_var_41_15 = (float *)malloc(sizeof(float *));
	float * d_var_41_15;
	hipMalloc((void **)&d_var_41_15, sizeof(float *));
	
	float * h_var_41_16 = (float *)malloc(sizeof(float *));
	float * d_var_41_16;
	hipMalloc((void **)&d_var_41_16, sizeof(float *));
	
	float * h_var_41_17 = (float *)malloc(sizeof(float *));
	float * d_var_41_17;
	hipMalloc((void **)&d_var_41_17, sizeof(float *));
	
	float * h_var_41_18 = (float *)malloc(sizeof(float *));
	float * d_var_41_18;
	hipMalloc((void **)&d_var_41_18, sizeof(float *));
	
	float * h_var_41_19 = (float *)malloc(sizeof(float *));
	float * d_var_41_19;
	hipMalloc((void **)&d_var_41_19, sizeof(float *));
	
	float * h_var_42_0 = (float *)malloc(sizeof(float *));
	float * d_var_42_0;
	hipMalloc((void **)&d_var_42_0, sizeof(float *));
	
	float * h_var_42_1 = (float *)malloc(sizeof(float *));
	float * d_var_42_1;
	hipMalloc((void **)&d_var_42_1, sizeof(float *));
	
	float * h_var_42_2 = (float *)malloc(sizeof(float *));
	float * d_var_42_2;
	hipMalloc((void **)&d_var_42_2, sizeof(float *));
	
	float * h_var_42_3 = (float *)malloc(sizeof(float *));
	float * d_var_42_3;
	hipMalloc((void **)&d_var_42_3, sizeof(float *));
	
	float * h_var_42_4 = (float *)malloc(sizeof(float *));
	float * d_var_42_4;
	hipMalloc((void **)&d_var_42_4, sizeof(float *));
	
	float * h_var_42_5 = (float *)malloc(sizeof(float *));
	float * d_var_42_5;
	hipMalloc((void **)&d_var_42_5, sizeof(float *));
	
	float * h_var_42_6 = (float *)malloc(sizeof(float *));
	float * d_var_42_6;
	hipMalloc((void **)&d_var_42_6, sizeof(float *));
	
	float * h_var_42_7 = (float *)malloc(sizeof(float *));
	float * d_var_42_7;
	hipMalloc((void **)&d_var_42_7, sizeof(float *));
	
	float * h_var_42_8 = (float *)malloc(sizeof(float *));
	float * d_var_42_8;
	hipMalloc((void **)&d_var_42_8, sizeof(float *));
	
	float * h_var_42_9 = (float *)malloc(sizeof(float *));
	float * d_var_42_9;
	hipMalloc((void **)&d_var_42_9, sizeof(float *));
	
	float * h_var_42_10 = (float *)malloc(sizeof(float *));
	float * d_var_42_10;
	hipMalloc((void **)&d_var_42_10, sizeof(float *));
	
	float * h_var_42_11 = (float *)malloc(sizeof(float *));
	float * d_var_42_11;
	hipMalloc((void **)&d_var_42_11, sizeof(float *));
	
	float * h_var_42_12 = (float *)malloc(sizeof(float *));
	float * d_var_42_12;
	hipMalloc((void **)&d_var_42_12, sizeof(float *));
	
	float * h_var_42_13 = (float *)malloc(sizeof(float *));
	float * d_var_42_13;
	hipMalloc((void **)&d_var_42_13, sizeof(float *));
	
	float * h_var_42_14 = (float *)malloc(sizeof(float *));
	float * d_var_42_14;
	hipMalloc((void **)&d_var_42_14, sizeof(float *));
	
	float * h_var_42_15 = (float *)malloc(sizeof(float *));
	float * d_var_42_15;
	hipMalloc((void **)&d_var_42_15, sizeof(float *));
	
	float * h_var_42_16 = (float *)malloc(sizeof(float *));
	float * d_var_42_16;
	hipMalloc((void **)&d_var_42_16, sizeof(float *));
	
	float * h_var_42_17 = (float *)malloc(sizeof(float *));
	float * d_var_42_17;
	hipMalloc((void **)&d_var_42_17, sizeof(float *));
	
	float * h_var_42_18 = (float *)malloc(sizeof(float *));
	float * d_var_42_18;
	hipMalloc((void **)&d_var_42_18, sizeof(float *));
	
	float * h_var_42_19 = (float *)malloc(sizeof(float *));
	float * d_var_42_19;
	hipMalloc((void **)&d_var_42_19, sizeof(float *));
	
	float * h_var_43_0 = (float *)malloc(sizeof(float *));
	float * d_var_43_0;
	hipMalloc((void **)&d_var_43_0, sizeof(float *));
	
	float * h_var_43_1 = (float *)malloc(sizeof(float *));
	float * d_var_43_1;
	hipMalloc((void **)&d_var_43_1, sizeof(float *));
	
	float * h_var_43_2 = (float *)malloc(sizeof(float *));
	float * d_var_43_2;
	hipMalloc((void **)&d_var_43_2, sizeof(float *));
	
	float * h_var_43_3 = (float *)malloc(sizeof(float *));
	float * d_var_43_3;
	hipMalloc((void **)&d_var_43_3, sizeof(float *));
	
	float * h_var_43_4 = (float *)malloc(sizeof(float *));
	float * d_var_43_4;
	hipMalloc((void **)&d_var_43_4, sizeof(float *));
	
	float * h_var_43_5 = (float *)malloc(sizeof(float *));
	float * d_var_43_5;
	hipMalloc((void **)&d_var_43_5, sizeof(float *));
	
	float * h_var_43_6 = (float *)malloc(sizeof(float *));
	float * d_var_43_6;
	hipMalloc((void **)&d_var_43_6, sizeof(float *));
	
	float * h_var_43_7 = (float *)malloc(sizeof(float *));
	float * d_var_43_7;
	hipMalloc((void **)&d_var_43_7, sizeof(float *));
	
	float * h_var_43_8 = (float *)malloc(sizeof(float *));
	float * d_var_43_8;
	hipMalloc((void **)&d_var_43_8, sizeof(float *));
	
	float * h_var_43_9 = (float *)malloc(sizeof(float *));
	float * d_var_43_9;
	hipMalloc((void **)&d_var_43_9, sizeof(float *));
	
	float * h_var_43_10 = (float *)malloc(sizeof(float *));
	float * d_var_43_10;
	hipMalloc((void **)&d_var_43_10, sizeof(float *));
	
	float * h_var_43_11 = (float *)malloc(sizeof(float *));
	float * d_var_43_11;
	hipMalloc((void **)&d_var_43_11, sizeof(float *));
	
	float * h_var_43_12 = (float *)malloc(sizeof(float *));
	float * d_var_43_12;
	hipMalloc((void **)&d_var_43_12, sizeof(float *));
	
	float * h_var_43_13 = (float *)malloc(sizeof(float *));
	float * d_var_43_13;
	hipMalloc((void **)&d_var_43_13, sizeof(float *));
	
	float * h_var_43_14 = (float *)malloc(sizeof(float *));
	float * d_var_43_14;
	hipMalloc((void **)&d_var_43_14, sizeof(float *));
	
	float * h_var_43_15 = (float *)malloc(sizeof(float *));
	float * d_var_43_15;
	hipMalloc((void **)&d_var_43_15, sizeof(float *));
	
	float * h_var_43_16 = (float *)malloc(sizeof(float *));
	float * d_var_43_16;
	hipMalloc((void **)&d_var_43_16, sizeof(float *));
	
	float * h_var_43_17 = (float *)malloc(sizeof(float *));
	float * d_var_43_17;
	hipMalloc((void **)&d_var_43_17, sizeof(float *));
	
	float * h_var_43_18 = (float *)malloc(sizeof(float *));
	float * d_var_43_18;
	hipMalloc((void **)&d_var_43_18, sizeof(float *));
	
	float * h_var_43_19 = (float *)malloc(sizeof(float *));
	float * d_var_43_19;
	hipMalloc((void **)&d_var_43_19, sizeof(float *));
	
	float * h_var_44_0 = (float *)malloc(sizeof(float *));
	float * d_var_44_0;
	hipMalloc((void **)&d_var_44_0, sizeof(float *));
	
	float * h_var_44_1 = (float *)malloc(sizeof(float *));
	float * d_var_44_1;
	hipMalloc((void **)&d_var_44_1, sizeof(float *));
	
	float * h_var_44_2 = (float *)malloc(sizeof(float *));
	float * d_var_44_2;
	hipMalloc((void **)&d_var_44_2, sizeof(float *));
	
	float * h_var_44_3 = (float *)malloc(sizeof(float *));
	float * d_var_44_3;
	hipMalloc((void **)&d_var_44_3, sizeof(float *));
	
	float * h_var_44_4 = (float *)malloc(sizeof(float *));
	float * d_var_44_4;
	hipMalloc((void **)&d_var_44_4, sizeof(float *));
	
	float * h_var_44_5 = (float *)malloc(sizeof(float *));
	float * d_var_44_5;
	hipMalloc((void **)&d_var_44_5, sizeof(float *));
	
	float * h_var_44_6 = (float *)malloc(sizeof(float *));
	float * d_var_44_6;
	hipMalloc((void **)&d_var_44_6, sizeof(float *));
	
	float * h_var_44_7 = (float *)malloc(sizeof(float *));
	float * d_var_44_7;
	hipMalloc((void **)&d_var_44_7, sizeof(float *));
	
	float * h_var_44_8 = (float *)malloc(sizeof(float *));
	float * d_var_44_8;
	hipMalloc((void **)&d_var_44_8, sizeof(float *));
	
	float * h_var_44_9 = (float *)malloc(sizeof(float *));
	float * d_var_44_9;
	hipMalloc((void **)&d_var_44_9, sizeof(float *));
	
	float * h_var_44_10 = (float *)malloc(sizeof(float *));
	float * d_var_44_10;
	hipMalloc((void **)&d_var_44_10, sizeof(float *));
	
	float * h_var_44_11 = (float *)malloc(sizeof(float *));
	float * d_var_44_11;
	hipMalloc((void **)&d_var_44_11, sizeof(float *));
	
	float * h_var_44_12 = (float *)malloc(sizeof(float *));
	float * d_var_44_12;
	hipMalloc((void **)&d_var_44_12, sizeof(float *));
	
	float * h_var_44_13 = (float *)malloc(sizeof(float *));
	float * d_var_44_13;
	hipMalloc((void **)&d_var_44_13, sizeof(float *));
	
	float * h_var_44_14 = (float *)malloc(sizeof(float *));
	float * d_var_44_14;
	hipMalloc((void **)&d_var_44_14, sizeof(float *));
	
	float * h_var_44_15 = (float *)malloc(sizeof(float *));
	float * d_var_44_15;
	hipMalloc((void **)&d_var_44_15, sizeof(float *));
	
	float * h_var_44_16 = (float *)malloc(sizeof(float *));
	float * d_var_44_16;
	hipMalloc((void **)&d_var_44_16, sizeof(float *));
	
	float * h_var_44_17 = (float *)malloc(sizeof(float *));
	float * d_var_44_17;
	hipMalloc((void **)&d_var_44_17, sizeof(float *));
	
	float * h_var_44_18 = (float *)malloc(sizeof(float *));
	float * d_var_44_18;
	hipMalloc((void **)&d_var_44_18, sizeof(float *));
	
	float * h_var_44_19 = (float *)malloc(sizeof(float *));
	float * d_var_44_19;
	hipMalloc((void **)&d_var_44_19, sizeof(float *));
	
	float * h_var_45_0 = (float *)malloc(sizeof(float *));
	float * d_var_45_0;
	hipMalloc((void **)&d_var_45_0, sizeof(float *));
	
	float * h_var_45_1 = (float *)malloc(sizeof(float *));
	float * d_var_45_1;
	hipMalloc((void **)&d_var_45_1, sizeof(float *));
	
	float * h_var_45_2 = (float *)malloc(sizeof(float *));
	float * d_var_45_2;
	hipMalloc((void **)&d_var_45_2, sizeof(float *));
	
	float * h_var_45_3 = (float *)malloc(sizeof(float *));
	float * d_var_45_3;
	hipMalloc((void **)&d_var_45_3, sizeof(float *));
	
	float * h_var_45_4 = (float *)malloc(sizeof(float *));
	float * d_var_45_4;
	hipMalloc((void **)&d_var_45_4, sizeof(float *));
	
	float * h_var_45_5 = (float *)malloc(sizeof(float *));
	float * d_var_45_5;
	hipMalloc((void **)&d_var_45_5, sizeof(float *));
	
	float * h_var_45_6 = (float *)malloc(sizeof(float *));
	float * d_var_45_6;
	hipMalloc((void **)&d_var_45_6, sizeof(float *));
	
	float * h_var_45_7 = (float *)malloc(sizeof(float *));
	float * d_var_45_7;
	hipMalloc((void **)&d_var_45_7, sizeof(float *));
	
	float * h_var_45_8 = (float *)malloc(sizeof(float *));
	float * d_var_45_8;
	hipMalloc((void **)&d_var_45_8, sizeof(float *));
	
	float * h_var_45_9 = (float *)malloc(sizeof(float *));
	float * d_var_45_9;
	hipMalloc((void **)&d_var_45_9, sizeof(float *));
	
	float * h_var_45_10 = (float *)malloc(sizeof(float *));
	float * d_var_45_10;
	hipMalloc((void **)&d_var_45_10, sizeof(float *));
	
	float * h_var_45_11 = (float *)malloc(sizeof(float *));
	float * d_var_45_11;
	hipMalloc((void **)&d_var_45_11, sizeof(float *));
	
	float * h_var_45_12 = (float *)malloc(sizeof(float *));
	float * d_var_45_12;
	hipMalloc((void **)&d_var_45_12, sizeof(float *));
	
	float * h_var_45_13 = (float *)malloc(sizeof(float *));
	float * d_var_45_13;
	hipMalloc((void **)&d_var_45_13, sizeof(float *));
	
	float * h_var_45_14 = (float *)malloc(sizeof(float *));
	float * d_var_45_14;
	hipMalloc((void **)&d_var_45_14, sizeof(float *));
	
	float * h_var_45_15 = (float *)malloc(sizeof(float *));
	float * d_var_45_15;
	hipMalloc((void **)&d_var_45_15, sizeof(float *));
	
	float * h_var_45_16 = (float *)malloc(sizeof(float *));
	float * d_var_45_16;
	hipMalloc((void **)&d_var_45_16, sizeof(float *));
	
	float * h_var_45_17 = (float *)malloc(sizeof(float *));
	float * d_var_45_17;
	hipMalloc((void **)&d_var_45_17, sizeof(float *));
	
	float * h_var_45_18 = (float *)malloc(sizeof(float *));
	float * d_var_45_18;
	hipMalloc((void **)&d_var_45_18, sizeof(float *));
	
	float * h_var_45_19 = (float *)malloc(sizeof(float *));
	float * d_var_45_19;
	hipMalloc((void **)&d_var_45_19, sizeof(float *));
	
	float * h_var_46_0 = (float *)malloc(sizeof(float *));
	float * d_var_46_0;
	hipMalloc((void **)&d_var_46_0, sizeof(float *));
	
	float * h_var_46_1 = (float *)malloc(sizeof(float *));
	float * d_var_46_1;
	hipMalloc((void **)&d_var_46_1, sizeof(float *));
	
	float * h_var_46_2 = (float *)malloc(sizeof(float *));
	float * d_var_46_2;
	hipMalloc((void **)&d_var_46_2, sizeof(float *));
	
	float * h_var_46_3 = (float *)malloc(sizeof(float *));
	float * d_var_46_3;
	hipMalloc((void **)&d_var_46_3, sizeof(float *));
	
	float * h_var_46_4 = (float *)malloc(sizeof(float *));
	float * d_var_46_4;
	hipMalloc((void **)&d_var_46_4, sizeof(float *));
	
	float * h_var_46_5 = (float *)malloc(sizeof(float *));
	float * d_var_46_5;
	hipMalloc((void **)&d_var_46_5, sizeof(float *));
	
	float * h_var_46_6 = (float *)malloc(sizeof(float *));
	float * d_var_46_6;
	hipMalloc((void **)&d_var_46_6, sizeof(float *));
	
	float * h_var_46_7 = (float *)malloc(sizeof(float *));
	float * d_var_46_7;
	hipMalloc((void **)&d_var_46_7, sizeof(float *));
	
	float * h_var_46_8 = (float *)malloc(sizeof(float *));
	float * d_var_46_8;
	hipMalloc((void **)&d_var_46_8, sizeof(float *));
	
	float * h_var_46_9 = (float *)malloc(sizeof(float *));
	float * d_var_46_9;
	hipMalloc((void **)&d_var_46_9, sizeof(float *));
	
	float * h_var_46_10 = (float *)malloc(sizeof(float *));
	float * d_var_46_10;
	hipMalloc((void **)&d_var_46_10, sizeof(float *));
	
	float * h_var_46_11 = (float *)malloc(sizeof(float *));
	float * d_var_46_11;
	hipMalloc((void **)&d_var_46_11, sizeof(float *));
	
	float * h_var_46_12 = (float *)malloc(sizeof(float *));
	float * d_var_46_12;
	hipMalloc((void **)&d_var_46_12, sizeof(float *));
	
	float * h_var_46_13 = (float *)malloc(sizeof(float *));
	float * d_var_46_13;
	hipMalloc((void **)&d_var_46_13, sizeof(float *));
	
	float * h_var_46_14 = (float *)malloc(sizeof(float *));
	float * d_var_46_14;
	hipMalloc((void **)&d_var_46_14, sizeof(float *));
	
	float * h_var_46_15 = (float *)malloc(sizeof(float *));
	float * d_var_46_15;
	hipMalloc((void **)&d_var_46_15, sizeof(float *));
	
	float * h_var_46_16 = (float *)malloc(sizeof(float *));
	float * d_var_46_16;
	hipMalloc((void **)&d_var_46_16, sizeof(float *));
	
	float * h_var_46_17 = (float *)malloc(sizeof(float *));
	float * d_var_46_17;
	hipMalloc((void **)&d_var_46_17, sizeof(float *));
	
	float * h_var_46_18 = (float *)malloc(sizeof(float *));
	float * d_var_46_18;
	hipMalloc((void **)&d_var_46_18, sizeof(float *));
	
	float * h_var_46_19 = (float *)malloc(sizeof(float *));
	float * d_var_46_19;
	hipMalloc((void **)&d_var_46_19, sizeof(float *));
	
	float * h_var_47_0 = (float *)malloc(sizeof(float *));
	float * d_var_47_0;
	hipMalloc((void **)&d_var_47_0, sizeof(float *));
	
	float * h_var_47_1 = (float *)malloc(sizeof(float *));
	float * d_var_47_1;
	hipMalloc((void **)&d_var_47_1, sizeof(float *));
	
	float * h_var_47_2 = (float *)malloc(sizeof(float *));
	float * d_var_47_2;
	hipMalloc((void **)&d_var_47_2, sizeof(float *));
	
	float * h_var_47_3 = (float *)malloc(sizeof(float *));
	float * d_var_47_3;
	hipMalloc((void **)&d_var_47_3, sizeof(float *));
	
	float * h_var_47_4 = (float *)malloc(sizeof(float *));
	float * d_var_47_4;
	hipMalloc((void **)&d_var_47_4, sizeof(float *));
	
	float * h_var_47_5 = (float *)malloc(sizeof(float *));
	float * d_var_47_5;
	hipMalloc((void **)&d_var_47_5, sizeof(float *));
	
	float * h_var_47_6 = (float *)malloc(sizeof(float *));
	float * d_var_47_6;
	hipMalloc((void **)&d_var_47_6, sizeof(float *));
	
	float * h_var_47_7 = (float *)malloc(sizeof(float *));
	float * d_var_47_7;
	hipMalloc((void **)&d_var_47_7, sizeof(float *));
	
	float * h_var_47_8 = (float *)malloc(sizeof(float *));
	float * d_var_47_8;
	hipMalloc((void **)&d_var_47_8, sizeof(float *));
	
	float * h_var_47_9 = (float *)malloc(sizeof(float *));
	float * d_var_47_9;
	hipMalloc((void **)&d_var_47_9, sizeof(float *));
	
	float * h_var_47_10 = (float *)malloc(sizeof(float *));
	float * d_var_47_10;
	hipMalloc((void **)&d_var_47_10, sizeof(float *));
	
	float * h_var_47_11 = (float *)malloc(sizeof(float *));
	float * d_var_47_11;
	hipMalloc((void **)&d_var_47_11, sizeof(float *));
	
	float * h_var_47_12 = (float *)malloc(sizeof(float *));
	float * d_var_47_12;
	hipMalloc((void **)&d_var_47_12, sizeof(float *));
	
	float * h_var_47_13 = (float *)malloc(sizeof(float *));
	float * d_var_47_13;
	hipMalloc((void **)&d_var_47_13, sizeof(float *));
	
	float * h_var_47_14 = (float *)malloc(sizeof(float *));
	float * d_var_47_14;
	hipMalloc((void **)&d_var_47_14, sizeof(float *));
	
	float * h_var_47_15 = (float *)malloc(sizeof(float *));
	float * d_var_47_15;
	hipMalloc((void **)&d_var_47_15, sizeof(float *));
	
	float * h_var_47_16 = (float *)malloc(sizeof(float *));
	float * d_var_47_16;
	hipMalloc((void **)&d_var_47_16, sizeof(float *));
	
	float * h_var_47_17 = (float *)malloc(sizeof(float *));
	float * d_var_47_17;
	hipMalloc((void **)&d_var_47_17, sizeof(float *));
	
	float * h_var_47_18 = (float *)malloc(sizeof(float *));
	float * d_var_47_18;
	hipMalloc((void **)&d_var_47_18, sizeof(float *));
	
	float * h_var_47_19 = (float *)malloc(sizeof(float *));
	float * d_var_47_19;
	hipMalloc((void **)&d_var_47_19, sizeof(float *));
	
	float * h_var_48_0 = (float *)malloc(sizeof(float *));
	float * d_var_48_0;
	hipMalloc((void **)&d_var_48_0, sizeof(float *));
	
	float * h_var_48_1 = (float *)malloc(sizeof(float *));
	float * d_var_48_1;
	hipMalloc((void **)&d_var_48_1, sizeof(float *));
	
	float * h_var_48_2 = (float *)malloc(sizeof(float *));
	float * d_var_48_2;
	hipMalloc((void **)&d_var_48_2, sizeof(float *));
	
	float * h_var_48_3 = (float *)malloc(sizeof(float *));
	float * d_var_48_3;
	hipMalloc((void **)&d_var_48_3, sizeof(float *));
	
	float * h_var_48_4 = (float *)malloc(sizeof(float *));
	float * d_var_48_4;
	hipMalloc((void **)&d_var_48_4, sizeof(float *));
	
	float * h_var_48_5 = (float *)malloc(sizeof(float *));
	float * d_var_48_5;
	hipMalloc((void **)&d_var_48_5, sizeof(float *));
	
	float * h_var_48_6 = (float *)malloc(sizeof(float *));
	float * d_var_48_6;
	hipMalloc((void **)&d_var_48_6, sizeof(float *));
	
	float * h_var_48_7 = (float *)malloc(sizeof(float *));
	float * d_var_48_7;
	hipMalloc((void **)&d_var_48_7, sizeof(float *));
	
	float * h_var_48_8 = (float *)malloc(sizeof(float *));
	float * d_var_48_8;
	hipMalloc((void **)&d_var_48_8, sizeof(float *));
	
	float * h_var_48_9 = (float *)malloc(sizeof(float *));
	float * d_var_48_9;
	hipMalloc((void **)&d_var_48_9, sizeof(float *));
	
	float * h_var_48_10 = (float *)malloc(sizeof(float *));
	float * d_var_48_10;
	hipMalloc((void **)&d_var_48_10, sizeof(float *));
	
	float * h_var_48_11 = (float *)malloc(sizeof(float *));
	float * d_var_48_11;
	hipMalloc((void **)&d_var_48_11, sizeof(float *));
	
	float * h_var_48_12 = (float *)malloc(sizeof(float *));
	float * d_var_48_12;
	hipMalloc((void **)&d_var_48_12, sizeof(float *));
	
	float * h_var_48_13 = (float *)malloc(sizeof(float *));
	float * d_var_48_13;
	hipMalloc((void **)&d_var_48_13, sizeof(float *));
	
	float * h_var_48_14 = (float *)malloc(sizeof(float *));
	float * d_var_48_14;
	hipMalloc((void **)&d_var_48_14, sizeof(float *));
	
	float * h_var_48_15 = (float *)malloc(sizeof(float *));
	float * d_var_48_15;
	hipMalloc((void **)&d_var_48_15, sizeof(float *));
	
	float * h_var_48_16 = (float *)malloc(sizeof(float *));
	float * d_var_48_16;
	hipMalloc((void **)&d_var_48_16, sizeof(float *));
	
	float * h_var_48_17 = (float *)malloc(sizeof(float *));
	float * d_var_48_17;
	hipMalloc((void **)&d_var_48_17, sizeof(float *));
	
	float * h_var_48_18 = (float *)malloc(sizeof(float *));
	float * d_var_48_18;
	hipMalloc((void **)&d_var_48_18, sizeof(float *));
	
	float * h_var_48_19 = (float *)malloc(sizeof(float *));
	float * d_var_48_19;
	hipMalloc((void **)&d_var_48_19, sizeof(float *));
	
	float * h_var_49_0 = (float *)malloc(sizeof(float *));
	float * d_var_49_0;
	hipMalloc((void **)&d_var_49_0, sizeof(float *));
	
	float * h_var_49_1 = (float *)malloc(sizeof(float *));
	float * d_var_49_1;
	hipMalloc((void **)&d_var_49_1, sizeof(float *));
	
	float * h_var_49_2 = (float *)malloc(sizeof(float *));
	float * d_var_49_2;
	hipMalloc((void **)&d_var_49_2, sizeof(float *));
	
	float * h_var_49_3 = (float *)malloc(sizeof(float *));
	float * d_var_49_3;
	hipMalloc((void **)&d_var_49_3, sizeof(float *));
	
	float * h_var_49_4 = (float *)malloc(sizeof(float *));
	float * d_var_49_4;
	hipMalloc((void **)&d_var_49_4, sizeof(float *));
	
	float * h_var_49_5 = (float *)malloc(sizeof(float *));
	float * d_var_49_5;
	hipMalloc((void **)&d_var_49_5, sizeof(float *));
	
	float * h_var_49_6 = (float *)malloc(sizeof(float *));
	float * d_var_49_6;
	hipMalloc((void **)&d_var_49_6, sizeof(float *));
	
	float * h_var_49_7 = (float *)malloc(sizeof(float *));
	float * d_var_49_7;
	hipMalloc((void **)&d_var_49_7, sizeof(float *));
	
	float * h_var_49_8 = (float *)malloc(sizeof(float *));
	float * d_var_49_8;
	hipMalloc((void **)&d_var_49_8, sizeof(float *));
	
	float * h_var_49_9 = (float *)malloc(sizeof(float *));
	float * d_var_49_9;
	hipMalloc((void **)&d_var_49_9, sizeof(float *));
	
	float * h_var_49_10 = (float *)malloc(sizeof(float *));
	float * d_var_49_10;
	hipMalloc((void **)&d_var_49_10, sizeof(float *));
	
	float * h_var_49_11 = (float *)malloc(sizeof(float *));
	float * d_var_49_11;
	hipMalloc((void **)&d_var_49_11, sizeof(float *));
	
	float * h_var_49_12 = (float *)malloc(sizeof(float *));
	float * d_var_49_12;
	hipMalloc((void **)&d_var_49_12, sizeof(float *));
	
	float * h_var_49_13 = (float *)malloc(sizeof(float *));
	float * d_var_49_13;
	hipMalloc((void **)&d_var_49_13, sizeof(float *));
	
	float * h_var_49_14 = (float *)malloc(sizeof(float *));
	float * d_var_49_14;
	hipMalloc((void **)&d_var_49_14, sizeof(float *));
	
	float * h_var_49_15 = (float *)malloc(sizeof(float *));
	float * d_var_49_15;
	hipMalloc((void **)&d_var_49_15, sizeof(float *));
	
	float * h_var_49_16 = (float *)malloc(sizeof(float *));
	float * d_var_49_16;
	hipMalloc((void **)&d_var_49_16, sizeof(float *));
	
	float * h_var_49_17 = (float *)malloc(sizeof(float *));
	float * d_var_49_17;
	hipMalloc((void **)&d_var_49_17, sizeof(float *));
	
	float * h_var_49_18 = (float *)malloc(sizeof(float *));
	float * d_var_49_18;
	hipMalloc((void **)&d_var_49_18, sizeof(float *));
	
	float * h_var_49_19 = (float *)malloc(sizeof(float *));
	float * d_var_49_19;
	hipMalloc((void **)&d_var_49_19, sizeof(float *));
	
	float * h_var_50_0 = (float *)malloc(sizeof(float *));
	float * d_var_50_0;
	hipMalloc((void **)&d_var_50_0, sizeof(float *));
	
	float * h_var_50_1 = (float *)malloc(sizeof(float *));
	float * d_var_50_1;
	hipMalloc((void **)&d_var_50_1, sizeof(float *));
	
	float * h_var_50_2 = (float *)malloc(sizeof(float *));
	float * d_var_50_2;
	hipMalloc((void **)&d_var_50_2, sizeof(float *));
	
	float * h_var_50_3 = (float *)malloc(sizeof(float *));
	float * d_var_50_3;
	hipMalloc((void **)&d_var_50_3, sizeof(float *));
	
	float * h_var_50_4 = (float *)malloc(sizeof(float *));
	float * d_var_50_4;
	hipMalloc((void **)&d_var_50_4, sizeof(float *));
	
	float * h_var_50_5 = (float *)malloc(sizeof(float *));
	float * d_var_50_5;
	hipMalloc((void **)&d_var_50_5, sizeof(float *));
	
	float * h_var_50_6 = (float *)malloc(sizeof(float *));
	float * d_var_50_6;
	hipMalloc((void **)&d_var_50_6, sizeof(float *));
	
	float * h_var_50_7 = (float *)malloc(sizeof(float *));
	float * d_var_50_7;
	hipMalloc((void **)&d_var_50_7, sizeof(float *));
	
	float * h_var_50_8 = (float *)malloc(sizeof(float *));
	float * d_var_50_8;
	hipMalloc((void **)&d_var_50_8, sizeof(float *));
	
	float * h_var_50_9 = (float *)malloc(sizeof(float *));
	float * d_var_50_9;
	hipMalloc((void **)&d_var_50_9, sizeof(float *));
	
	float * h_var_50_10 = (float *)malloc(sizeof(float *));
	float * d_var_50_10;
	hipMalloc((void **)&d_var_50_10, sizeof(float *));
	
	float * h_var_50_11 = (float *)malloc(sizeof(float *));
	float * d_var_50_11;
	hipMalloc((void **)&d_var_50_11, sizeof(float *));
	
	float * h_var_50_12 = (float *)malloc(sizeof(float *));
	float * d_var_50_12;
	hipMalloc((void **)&d_var_50_12, sizeof(float *));
	
	float * h_var_50_13 = (float *)malloc(sizeof(float *));
	float * d_var_50_13;
	hipMalloc((void **)&d_var_50_13, sizeof(float *));
	
	float * h_var_50_14 = (float *)malloc(sizeof(float *));
	float * d_var_50_14;
	hipMalloc((void **)&d_var_50_14, sizeof(float *));
	
	float * h_var_50_15 = (float *)malloc(sizeof(float *));
	float * d_var_50_15;
	hipMalloc((void **)&d_var_50_15, sizeof(float *));
	
	float * h_var_50_16 = (float *)malloc(sizeof(float *));
	float * d_var_50_16;
	hipMalloc((void **)&d_var_50_16, sizeof(float *));
	
	float * h_var_50_17 = (float *)malloc(sizeof(float *));
	float * d_var_50_17;
	hipMalloc((void **)&d_var_50_17, sizeof(float *));
	
	float * h_var_50_18 = (float *)malloc(sizeof(float *));
	float * d_var_50_18;
	hipMalloc((void **)&d_var_50_18, sizeof(float *));
	
	float * h_var_50_19 = (float *)malloc(sizeof(float *));
	float * d_var_50_19;
	hipMalloc((void **)&d_var_50_19, sizeof(float *));
	
	float * h_var_51_0 = (float *)malloc(sizeof(float *));
	float * d_var_51_0;
	hipMalloc((void **)&d_var_51_0, sizeof(float *));
	
	float * h_var_51_1 = (float *)malloc(sizeof(float *));
	float * d_var_51_1;
	hipMalloc((void **)&d_var_51_1, sizeof(float *));
	
	float * h_var_51_2 = (float *)malloc(sizeof(float *));
	float * d_var_51_2;
	hipMalloc((void **)&d_var_51_2, sizeof(float *));
	
	float * h_var_51_3 = (float *)malloc(sizeof(float *));
	float * d_var_51_3;
	hipMalloc((void **)&d_var_51_3, sizeof(float *));
	
	float * h_var_51_4 = (float *)malloc(sizeof(float *));
	float * d_var_51_4;
	hipMalloc((void **)&d_var_51_4, sizeof(float *));
	
	float * h_var_51_5 = (float *)malloc(sizeof(float *));
	float * d_var_51_5;
	hipMalloc((void **)&d_var_51_5, sizeof(float *));
	
	float * h_var_51_6 = (float *)malloc(sizeof(float *));
	float * d_var_51_6;
	hipMalloc((void **)&d_var_51_6, sizeof(float *));
	
	float * h_var_51_7 = (float *)malloc(sizeof(float *));
	float * d_var_51_7;
	hipMalloc((void **)&d_var_51_7, sizeof(float *));
	
	float * h_var_51_8 = (float *)malloc(sizeof(float *));
	float * d_var_51_8;
	hipMalloc((void **)&d_var_51_8, sizeof(float *));
	
	float * h_var_51_9 = (float *)malloc(sizeof(float *));
	float * d_var_51_9;
	hipMalloc((void **)&d_var_51_9, sizeof(float *));
	
	float * h_var_51_10 = (float *)malloc(sizeof(float *));
	float * d_var_51_10;
	hipMalloc((void **)&d_var_51_10, sizeof(float *));
	
	float * h_var_51_11 = (float *)malloc(sizeof(float *));
	float * d_var_51_11;
	hipMalloc((void **)&d_var_51_11, sizeof(float *));
	
	float * h_var_51_12 = (float *)malloc(sizeof(float *));
	float * d_var_51_12;
	hipMalloc((void **)&d_var_51_12, sizeof(float *));
	
	float * h_var_51_13 = (float *)malloc(sizeof(float *));
	float * d_var_51_13;
	hipMalloc((void **)&d_var_51_13, sizeof(float *));
	
	float * h_var_51_14 = (float *)malloc(sizeof(float *));
	float * d_var_51_14;
	hipMalloc((void **)&d_var_51_14, sizeof(float *));
	
	float * h_var_51_15 = (float *)malloc(sizeof(float *));
	float * d_var_51_15;
	hipMalloc((void **)&d_var_51_15, sizeof(float *));
	
	float * h_var_51_16 = (float *)malloc(sizeof(float *));
	float * d_var_51_16;
	hipMalloc((void **)&d_var_51_16, sizeof(float *));
	
	float * h_var_51_17 = (float *)malloc(sizeof(float *));
	float * d_var_51_17;
	hipMalloc((void **)&d_var_51_17, sizeof(float *));
	
	float * h_var_51_18 = (float *)malloc(sizeof(float *));
	float * d_var_51_18;
	hipMalloc((void **)&d_var_51_18, sizeof(float *));
	
	float * h_var_51_19 = (float *)malloc(sizeof(float *));
	float * d_var_51_19;
	hipMalloc((void **)&d_var_51_19, sizeof(float *));
	
	float * h_var_52_0 = (float *)malloc(sizeof(float *));
	float * d_var_52_0;
	hipMalloc((void **)&d_var_52_0, sizeof(float *));
	
	float * h_var_52_1 = (float *)malloc(sizeof(float *));
	float * d_var_52_1;
	hipMalloc((void **)&d_var_52_1, sizeof(float *));
	
	float * h_var_52_2 = (float *)malloc(sizeof(float *));
	float * d_var_52_2;
	hipMalloc((void **)&d_var_52_2, sizeof(float *));
	
	float * h_var_52_3 = (float *)malloc(sizeof(float *));
	float * d_var_52_3;
	hipMalloc((void **)&d_var_52_3, sizeof(float *));
	
	float * h_var_52_4 = (float *)malloc(sizeof(float *));
	float * d_var_52_4;
	hipMalloc((void **)&d_var_52_4, sizeof(float *));
	
	float * h_var_52_5 = (float *)malloc(sizeof(float *));
	float * d_var_52_5;
	hipMalloc((void **)&d_var_52_5, sizeof(float *));
	
	float * h_var_52_6 = (float *)malloc(sizeof(float *));
	float * d_var_52_6;
	hipMalloc((void **)&d_var_52_6, sizeof(float *));
	
	float * h_var_52_7 = (float *)malloc(sizeof(float *));
	float * d_var_52_7;
	hipMalloc((void **)&d_var_52_7, sizeof(float *));
	
	float * h_var_52_8 = (float *)malloc(sizeof(float *));
	float * d_var_52_8;
	hipMalloc((void **)&d_var_52_8, sizeof(float *));
	
	float * h_var_52_9 = (float *)malloc(sizeof(float *));
	float * d_var_52_9;
	hipMalloc((void **)&d_var_52_9, sizeof(float *));
	
	float * h_var_52_10 = (float *)malloc(sizeof(float *));
	float * d_var_52_10;
	hipMalloc((void **)&d_var_52_10, sizeof(float *));
	
	float * h_var_52_11 = (float *)malloc(sizeof(float *));
	float * d_var_52_11;
	hipMalloc((void **)&d_var_52_11, sizeof(float *));
	
	float * h_var_52_12 = (float *)malloc(sizeof(float *));
	float * d_var_52_12;
	hipMalloc((void **)&d_var_52_12, sizeof(float *));
	
	float * h_var_52_13 = (float *)malloc(sizeof(float *));
	float * d_var_52_13;
	hipMalloc((void **)&d_var_52_13, sizeof(float *));
	
	float * h_var_52_14 = (float *)malloc(sizeof(float *));
	float * d_var_52_14;
	hipMalloc((void **)&d_var_52_14, sizeof(float *));
	
	float * h_var_52_15 = (float *)malloc(sizeof(float *));
	float * d_var_52_15;
	hipMalloc((void **)&d_var_52_15, sizeof(float *));
	
	float * h_var_52_16 = (float *)malloc(sizeof(float *));
	float * d_var_52_16;
	hipMalloc((void **)&d_var_52_16, sizeof(float *));
	
	float * h_var_52_17 = (float *)malloc(sizeof(float *));
	float * d_var_52_17;
	hipMalloc((void **)&d_var_52_17, sizeof(float *));
	
	float * h_var_52_18 = (float *)malloc(sizeof(float *));
	float * d_var_52_18;
	hipMalloc((void **)&d_var_52_18, sizeof(float *));
	
	float * h_var_52_19 = (float *)malloc(sizeof(float *));
	float * d_var_52_19;
	hipMalloc((void **)&d_var_52_19, sizeof(float *));
	
	float * h_var_53_0 = (float *)malloc(sizeof(float *));
	float * d_var_53_0;
	hipMalloc((void **)&d_var_53_0, sizeof(float *));
	
	float * h_var_53_1 = (float *)malloc(sizeof(float *));
	float * d_var_53_1;
	hipMalloc((void **)&d_var_53_1, sizeof(float *));
	
	float * h_var_53_2 = (float *)malloc(sizeof(float *));
	float * d_var_53_2;
	hipMalloc((void **)&d_var_53_2, sizeof(float *));
	
	float * h_var_53_3 = (float *)malloc(sizeof(float *));
	float * d_var_53_3;
	hipMalloc((void **)&d_var_53_3, sizeof(float *));
	
	float * h_var_53_4 = (float *)malloc(sizeof(float *));
	float * d_var_53_4;
	hipMalloc((void **)&d_var_53_4, sizeof(float *));
	
	float * h_var_53_5 = (float *)malloc(sizeof(float *));
	float * d_var_53_5;
	hipMalloc((void **)&d_var_53_5, sizeof(float *));
	
	float * h_var_53_6 = (float *)malloc(sizeof(float *));
	float * d_var_53_6;
	hipMalloc((void **)&d_var_53_6, sizeof(float *));
	
	float * h_var_53_7 = (float *)malloc(sizeof(float *));
	float * d_var_53_7;
	hipMalloc((void **)&d_var_53_7, sizeof(float *));
	
	float * h_var_53_8 = (float *)malloc(sizeof(float *));
	float * d_var_53_8;
	hipMalloc((void **)&d_var_53_8, sizeof(float *));
	
	float * h_var_53_9 = (float *)malloc(sizeof(float *));
	float * d_var_53_9;
	hipMalloc((void **)&d_var_53_9, sizeof(float *));
	
	float * h_var_53_10 = (float *)malloc(sizeof(float *));
	float * d_var_53_10;
	hipMalloc((void **)&d_var_53_10, sizeof(float *));
	
	float * h_var_53_11 = (float *)malloc(sizeof(float *));
	float * d_var_53_11;
	hipMalloc((void **)&d_var_53_11, sizeof(float *));
	
	float * h_var_53_12 = (float *)malloc(sizeof(float *));
	float * d_var_53_12;
	hipMalloc((void **)&d_var_53_12, sizeof(float *));
	
	float * h_var_53_13 = (float *)malloc(sizeof(float *));
	float * d_var_53_13;
	hipMalloc((void **)&d_var_53_13, sizeof(float *));
	
	float * h_var_53_14 = (float *)malloc(sizeof(float *));
	float * d_var_53_14;
	hipMalloc((void **)&d_var_53_14, sizeof(float *));
	
	float * h_var_53_15 = (float *)malloc(sizeof(float *));
	float * d_var_53_15;
	hipMalloc((void **)&d_var_53_15, sizeof(float *));
	
	float * h_var_53_16 = (float *)malloc(sizeof(float *));
	float * d_var_53_16;
	hipMalloc((void **)&d_var_53_16, sizeof(float *));
	
	float * h_var_53_17 = (float *)malloc(sizeof(float *));
	float * d_var_53_17;
	hipMalloc((void **)&d_var_53_17, sizeof(float *));
	
	float * h_var_53_18 = (float *)malloc(sizeof(float *));
	float * d_var_53_18;
	hipMalloc((void **)&d_var_53_18, sizeof(float *));
	
	float * h_var_53_19 = (float *)malloc(sizeof(float *));
	float * d_var_53_19;
	hipMalloc((void **)&d_var_53_19, sizeof(float *));
	
	float * h_var_54_0 = (float *)malloc(sizeof(float *));
	float * d_var_54_0;
	hipMalloc((void **)&d_var_54_0, sizeof(float *));
	
	float * h_var_54_1 = (float *)malloc(sizeof(float *));
	float * d_var_54_1;
	hipMalloc((void **)&d_var_54_1, sizeof(float *));
	
	float * h_var_54_2 = (float *)malloc(sizeof(float *));
	float * d_var_54_2;
	hipMalloc((void **)&d_var_54_2, sizeof(float *));
	
	float * h_var_54_3 = (float *)malloc(sizeof(float *));
	float * d_var_54_3;
	hipMalloc((void **)&d_var_54_3, sizeof(float *));
	
	float * h_var_54_4 = (float *)malloc(sizeof(float *));
	float * d_var_54_4;
	hipMalloc((void **)&d_var_54_4, sizeof(float *));
	
	float * h_var_54_5 = (float *)malloc(sizeof(float *));
	float * d_var_54_5;
	hipMalloc((void **)&d_var_54_5, sizeof(float *));
	
	float * h_var_54_6 = (float *)malloc(sizeof(float *));
	float * d_var_54_6;
	hipMalloc((void **)&d_var_54_6, sizeof(float *));
	
	float * h_var_54_7 = (float *)malloc(sizeof(float *));
	float * d_var_54_7;
	hipMalloc((void **)&d_var_54_7, sizeof(float *));
	
	float * h_var_54_8 = (float *)malloc(sizeof(float *));
	float * d_var_54_8;
	hipMalloc((void **)&d_var_54_8, sizeof(float *));
	
	float * h_var_54_9 = (float *)malloc(sizeof(float *));
	float * d_var_54_9;
	hipMalloc((void **)&d_var_54_9, sizeof(float *));
	
	float * h_var_54_10 = (float *)malloc(sizeof(float *));
	float * d_var_54_10;
	hipMalloc((void **)&d_var_54_10, sizeof(float *));
	
	float * h_var_54_11 = (float *)malloc(sizeof(float *));
	float * d_var_54_11;
	hipMalloc((void **)&d_var_54_11, sizeof(float *));
	
	float * h_var_54_12 = (float *)malloc(sizeof(float *));
	float * d_var_54_12;
	hipMalloc((void **)&d_var_54_12, sizeof(float *));
	
	float * h_var_54_13 = (float *)malloc(sizeof(float *));
	float * d_var_54_13;
	hipMalloc((void **)&d_var_54_13, sizeof(float *));
	
	float * h_var_54_14 = (float *)malloc(sizeof(float *));
	float * d_var_54_14;
	hipMalloc((void **)&d_var_54_14, sizeof(float *));
	
	float * h_var_54_15 = (float *)malloc(sizeof(float *));
	float * d_var_54_15;
	hipMalloc((void **)&d_var_54_15, sizeof(float *));
	
	float * h_var_54_16 = (float *)malloc(sizeof(float *));
	float * d_var_54_16;
	hipMalloc((void **)&d_var_54_16, sizeof(float *));
	
	float * h_var_54_17 = (float *)malloc(sizeof(float *));
	float * d_var_54_17;
	hipMalloc((void **)&d_var_54_17, sizeof(float *));
	
	float * h_var_54_18 = (float *)malloc(sizeof(float *));
	float * d_var_54_18;
	hipMalloc((void **)&d_var_54_18, sizeof(float *));
	
	float * h_var_54_19 = (float *)malloc(sizeof(float *));
	float * d_var_54_19;
	hipMalloc((void **)&d_var_54_19, sizeof(float *));
	
	float * h_var_55_0 = (float *)malloc(sizeof(float *));
	float * d_var_55_0;
	hipMalloc((void **)&d_var_55_0, sizeof(float *));
	
	float * h_var_55_1 = (float *)malloc(sizeof(float *));
	float * d_var_55_1;
	hipMalloc((void **)&d_var_55_1, sizeof(float *));
	
	float * h_var_55_2 = (float *)malloc(sizeof(float *));
	float * d_var_55_2;
	hipMalloc((void **)&d_var_55_2, sizeof(float *));
	
	float * h_var_55_3 = (float *)malloc(sizeof(float *));
	float * d_var_55_3;
	hipMalloc((void **)&d_var_55_3, sizeof(float *));
	
	float * h_var_55_4 = (float *)malloc(sizeof(float *));
	float * d_var_55_4;
	hipMalloc((void **)&d_var_55_4, sizeof(float *));
	
	float * h_var_55_5 = (float *)malloc(sizeof(float *));
	float * d_var_55_5;
	hipMalloc((void **)&d_var_55_5, sizeof(float *));
	
	float * h_var_55_6 = (float *)malloc(sizeof(float *));
	float * d_var_55_6;
	hipMalloc((void **)&d_var_55_6, sizeof(float *));
	
	float * h_var_55_7 = (float *)malloc(sizeof(float *));
	float * d_var_55_7;
	hipMalloc((void **)&d_var_55_7, sizeof(float *));
	
	float * h_var_55_8 = (float *)malloc(sizeof(float *));
	float * d_var_55_8;
	hipMalloc((void **)&d_var_55_8, sizeof(float *));
	
	float * h_var_55_9 = (float *)malloc(sizeof(float *));
	float * d_var_55_9;
	hipMalloc((void **)&d_var_55_9, sizeof(float *));
	
	float * h_var_55_10 = (float *)malloc(sizeof(float *));
	float * d_var_55_10;
	hipMalloc((void **)&d_var_55_10, sizeof(float *));
	
	float * h_var_55_11 = (float *)malloc(sizeof(float *));
	float * d_var_55_11;
	hipMalloc((void **)&d_var_55_11, sizeof(float *));
	
	float * h_var_55_12 = (float *)malloc(sizeof(float *));
	float * d_var_55_12;
	hipMalloc((void **)&d_var_55_12, sizeof(float *));
	
	float * h_var_55_13 = (float *)malloc(sizeof(float *));
	float * d_var_55_13;
	hipMalloc((void **)&d_var_55_13, sizeof(float *));
	
	float * h_var_55_14 = (float *)malloc(sizeof(float *));
	float * d_var_55_14;
	hipMalloc((void **)&d_var_55_14, sizeof(float *));
	
	float * h_var_55_15 = (float *)malloc(sizeof(float *));
	float * d_var_55_15;
	hipMalloc((void **)&d_var_55_15, sizeof(float *));
	
	float * h_var_55_16 = (float *)malloc(sizeof(float *));
	float * d_var_55_16;
	hipMalloc((void **)&d_var_55_16, sizeof(float *));
	
	float * h_var_55_17 = (float *)malloc(sizeof(float *));
	float * d_var_55_17;
	hipMalloc((void **)&d_var_55_17, sizeof(float *));
	
	float * h_var_55_18 = (float *)malloc(sizeof(float *));
	float * d_var_55_18;
	hipMalloc((void **)&d_var_55_18, sizeof(float *));
	
	float * h_var_55_19 = (float *)malloc(sizeof(float *));
	float * d_var_55_19;
	hipMalloc((void **)&d_var_55_19, sizeof(float *));
	
	float * h_var_56_0 = (float *)malloc(sizeof(float *));
	float * d_var_56_0;
	hipMalloc((void **)&d_var_56_0, sizeof(float *));
	
	float * h_var_56_1 = (float *)malloc(sizeof(float *));
	float * d_var_56_1;
	hipMalloc((void **)&d_var_56_1, sizeof(float *));
	
	float * h_var_56_2 = (float *)malloc(sizeof(float *));
	float * d_var_56_2;
	hipMalloc((void **)&d_var_56_2, sizeof(float *));
	
	float * h_var_56_3 = (float *)malloc(sizeof(float *));
	float * d_var_56_3;
	hipMalloc((void **)&d_var_56_3, sizeof(float *));
	
	float * h_var_56_4 = (float *)malloc(sizeof(float *));
	float * d_var_56_4;
	hipMalloc((void **)&d_var_56_4, sizeof(float *));
	
	float * h_var_56_5 = (float *)malloc(sizeof(float *));
	float * d_var_56_5;
	hipMalloc((void **)&d_var_56_5, sizeof(float *));
	
	float * h_var_56_6 = (float *)malloc(sizeof(float *));
	float * d_var_56_6;
	hipMalloc((void **)&d_var_56_6, sizeof(float *));
	
	float * h_var_56_7 = (float *)malloc(sizeof(float *));
	float * d_var_56_7;
	hipMalloc((void **)&d_var_56_7, sizeof(float *));
	
	float * h_var_56_8 = (float *)malloc(sizeof(float *));
	float * d_var_56_8;
	hipMalloc((void **)&d_var_56_8, sizeof(float *));
	
	float * h_var_56_9 = (float *)malloc(sizeof(float *));
	float * d_var_56_9;
	hipMalloc((void **)&d_var_56_9, sizeof(float *));
	
	float * h_var_56_10 = (float *)malloc(sizeof(float *));
	float * d_var_56_10;
	hipMalloc((void **)&d_var_56_10, sizeof(float *));
	
	float * h_var_56_11 = (float *)malloc(sizeof(float *));
	float * d_var_56_11;
	hipMalloc((void **)&d_var_56_11, sizeof(float *));
	
	float * h_var_56_12 = (float *)malloc(sizeof(float *));
	float * d_var_56_12;
	hipMalloc((void **)&d_var_56_12, sizeof(float *));
	
	float * h_var_56_13 = (float *)malloc(sizeof(float *));
	float * d_var_56_13;
	hipMalloc((void **)&d_var_56_13, sizeof(float *));
	
	float * h_var_56_14 = (float *)malloc(sizeof(float *));
	float * d_var_56_14;
	hipMalloc((void **)&d_var_56_14, sizeof(float *));
	
	float * h_var_56_15 = (float *)malloc(sizeof(float *));
	float * d_var_56_15;
	hipMalloc((void **)&d_var_56_15, sizeof(float *));
	
	float * h_var_56_16 = (float *)malloc(sizeof(float *));
	float * d_var_56_16;
	hipMalloc((void **)&d_var_56_16, sizeof(float *));
	
	float * h_var_56_17 = (float *)malloc(sizeof(float *));
	float * d_var_56_17;
	hipMalloc((void **)&d_var_56_17, sizeof(float *));
	
	float * h_var_56_18 = (float *)malloc(sizeof(float *));
	float * d_var_56_18;
	hipMalloc((void **)&d_var_56_18, sizeof(float *));
	
	float * h_var_56_19 = (float *)malloc(sizeof(float *));
	float * d_var_56_19;
	hipMalloc((void **)&d_var_56_19, sizeof(float *));
	
	float * h_var_57_0 = (float *)malloc(sizeof(float *));
	float * d_var_57_0;
	hipMalloc((void **)&d_var_57_0, sizeof(float *));
	
	float * h_var_57_1 = (float *)malloc(sizeof(float *));
	float * d_var_57_1;
	hipMalloc((void **)&d_var_57_1, sizeof(float *));
	
	float * h_var_57_2 = (float *)malloc(sizeof(float *));
	float * d_var_57_2;
	hipMalloc((void **)&d_var_57_2, sizeof(float *));
	
	float * h_var_57_3 = (float *)malloc(sizeof(float *));
	float * d_var_57_3;
	hipMalloc((void **)&d_var_57_3, sizeof(float *));
	
	float * h_var_57_4 = (float *)malloc(sizeof(float *));
	float * d_var_57_4;
	hipMalloc((void **)&d_var_57_4, sizeof(float *));
	
	float * h_var_57_5 = (float *)malloc(sizeof(float *));
	float * d_var_57_5;
	hipMalloc((void **)&d_var_57_5, sizeof(float *));
	
	float * h_var_57_6 = (float *)malloc(sizeof(float *));
	float * d_var_57_6;
	hipMalloc((void **)&d_var_57_6, sizeof(float *));
	
	float * h_var_57_7 = (float *)malloc(sizeof(float *));
	float * d_var_57_7;
	hipMalloc((void **)&d_var_57_7, sizeof(float *));
	
	float * h_var_57_8 = (float *)malloc(sizeof(float *));
	float * d_var_57_8;
	hipMalloc((void **)&d_var_57_8, sizeof(float *));
	
	float * h_var_57_9 = (float *)malloc(sizeof(float *));
	float * d_var_57_9;
	hipMalloc((void **)&d_var_57_9, sizeof(float *));
	
	float * h_var_57_10 = (float *)malloc(sizeof(float *));
	float * d_var_57_10;
	hipMalloc((void **)&d_var_57_10, sizeof(float *));
	
	float * h_var_57_11 = (float *)malloc(sizeof(float *));
	float * d_var_57_11;
	hipMalloc((void **)&d_var_57_11, sizeof(float *));
	
	float * h_var_57_12 = (float *)malloc(sizeof(float *));
	float * d_var_57_12;
	hipMalloc((void **)&d_var_57_12, sizeof(float *));
	
	float * h_var_57_13 = (float *)malloc(sizeof(float *));
	float * d_var_57_13;
	hipMalloc((void **)&d_var_57_13, sizeof(float *));
	
	float * h_var_57_14 = (float *)malloc(sizeof(float *));
	float * d_var_57_14;
	hipMalloc((void **)&d_var_57_14, sizeof(float *));
	
	float * h_var_57_15 = (float *)malloc(sizeof(float *));
	float * d_var_57_15;
	hipMalloc((void **)&d_var_57_15, sizeof(float *));
	
	float * h_var_57_16 = (float *)malloc(sizeof(float *));
	float * d_var_57_16;
	hipMalloc((void **)&d_var_57_16, sizeof(float *));
	
	float * h_var_57_17 = (float *)malloc(sizeof(float *));
	float * d_var_57_17;
	hipMalloc((void **)&d_var_57_17, sizeof(float *));
	
	float * h_var_57_18 = (float *)malloc(sizeof(float *));
	float * d_var_57_18;
	hipMalloc((void **)&d_var_57_18, sizeof(float *));
	
	float * h_var_57_19 = (float *)malloc(sizeof(float *));
	float * d_var_57_19;
	hipMalloc((void **)&d_var_57_19, sizeof(float *));
	
	float * h_var_58_0 = (float *)malloc(sizeof(float *));
	float * d_var_58_0;
	hipMalloc((void **)&d_var_58_0, sizeof(float *));
	
	float * h_var_58_1 = (float *)malloc(sizeof(float *));
	float * d_var_58_1;
	hipMalloc((void **)&d_var_58_1, sizeof(float *));
	
	float * h_var_58_2 = (float *)malloc(sizeof(float *));
	float * d_var_58_2;
	hipMalloc((void **)&d_var_58_2, sizeof(float *));
	
	float * h_var_58_3 = (float *)malloc(sizeof(float *));
	float * d_var_58_3;
	hipMalloc((void **)&d_var_58_3, sizeof(float *));
	
	float * h_var_58_4 = (float *)malloc(sizeof(float *));
	float * d_var_58_4;
	hipMalloc((void **)&d_var_58_4, sizeof(float *));
	
	float * h_var_58_5 = (float *)malloc(sizeof(float *));
	float * d_var_58_5;
	hipMalloc((void **)&d_var_58_5, sizeof(float *));
	
	float * h_var_58_6 = (float *)malloc(sizeof(float *));
	float * d_var_58_6;
	hipMalloc((void **)&d_var_58_6, sizeof(float *));
	
	float * h_var_58_7 = (float *)malloc(sizeof(float *));
	float * d_var_58_7;
	hipMalloc((void **)&d_var_58_7, sizeof(float *));
	
	float * h_var_58_8 = (float *)malloc(sizeof(float *));
	float * d_var_58_8;
	hipMalloc((void **)&d_var_58_8, sizeof(float *));
	
	float * h_var_58_9 = (float *)malloc(sizeof(float *));
	float * d_var_58_9;
	hipMalloc((void **)&d_var_58_9, sizeof(float *));
	
	float * h_var_58_10 = (float *)malloc(sizeof(float *));
	float * d_var_58_10;
	hipMalloc((void **)&d_var_58_10, sizeof(float *));
	
	float * h_var_58_11 = (float *)malloc(sizeof(float *));
	float * d_var_58_11;
	hipMalloc((void **)&d_var_58_11, sizeof(float *));
	
	float * h_var_58_12 = (float *)malloc(sizeof(float *));
	float * d_var_58_12;
	hipMalloc((void **)&d_var_58_12, sizeof(float *));
	
	float * h_var_58_13 = (float *)malloc(sizeof(float *));
	float * d_var_58_13;
	hipMalloc((void **)&d_var_58_13, sizeof(float *));
	
	float * h_var_58_14 = (float *)malloc(sizeof(float *));
	float * d_var_58_14;
	hipMalloc((void **)&d_var_58_14, sizeof(float *));
	
	float * h_var_58_15 = (float *)malloc(sizeof(float *));
	float * d_var_58_15;
	hipMalloc((void **)&d_var_58_15, sizeof(float *));
	
	float * h_var_58_16 = (float *)malloc(sizeof(float *));
	float * d_var_58_16;
	hipMalloc((void **)&d_var_58_16, sizeof(float *));
	
	float * h_var_58_17 = (float *)malloc(sizeof(float *));
	float * d_var_58_17;
	hipMalloc((void **)&d_var_58_17, sizeof(float *));
	
	float * h_var_58_18 = (float *)malloc(sizeof(float *));
	float * d_var_58_18;
	hipMalloc((void **)&d_var_58_18, sizeof(float *));
	
	float * h_var_58_19 = (float *)malloc(sizeof(float *));
	float * d_var_58_19;
	hipMalloc((void **)&d_var_58_19, sizeof(float *));
	
	float * h_var_59_0 = (float *)malloc(sizeof(float *));
	float * d_var_59_0;
	hipMalloc((void **)&d_var_59_0, sizeof(float *));
	
	float * h_var_59_1 = (float *)malloc(sizeof(float *));
	float * d_var_59_1;
	hipMalloc((void **)&d_var_59_1, sizeof(float *));
	
	float * h_var_59_2 = (float *)malloc(sizeof(float *));
	float * d_var_59_2;
	hipMalloc((void **)&d_var_59_2, sizeof(float *));
	
	float * h_var_59_3 = (float *)malloc(sizeof(float *));
	float * d_var_59_3;
	hipMalloc((void **)&d_var_59_3, sizeof(float *));
	
	float * h_var_59_4 = (float *)malloc(sizeof(float *));
	float * d_var_59_4;
	hipMalloc((void **)&d_var_59_4, sizeof(float *));
	
	float * h_var_59_5 = (float *)malloc(sizeof(float *));
	float * d_var_59_5;
	hipMalloc((void **)&d_var_59_5, sizeof(float *));
	
	float * h_var_59_6 = (float *)malloc(sizeof(float *));
	float * d_var_59_6;
	hipMalloc((void **)&d_var_59_6, sizeof(float *));
	
	float * h_var_59_7 = (float *)malloc(sizeof(float *));
	float * d_var_59_7;
	hipMalloc((void **)&d_var_59_7, sizeof(float *));
	
	float * h_var_59_8 = (float *)malloc(sizeof(float *));
	float * d_var_59_8;
	hipMalloc((void **)&d_var_59_8, sizeof(float *));
	
	float * h_var_59_9 = (float *)malloc(sizeof(float *));
	float * d_var_59_9;
	hipMalloc((void **)&d_var_59_9, sizeof(float *));
	
	float * h_var_59_10 = (float *)malloc(sizeof(float *));
	float * d_var_59_10;
	hipMalloc((void **)&d_var_59_10, sizeof(float *));
	
	float * h_var_59_11 = (float *)malloc(sizeof(float *));
	float * d_var_59_11;
	hipMalloc((void **)&d_var_59_11, sizeof(float *));
	
	float * h_var_59_12 = (float *)malloc(sizeof(float *));
	float * d_var_59_12;
	hipMalloc((void **)&d_var_59_12, sizeof(float *));
	
	float * h_var_59_13 = (float *)malloc(sizeof(float *));
	float * d_var_59_13;
	hipMalloc((void **)&d_var_59_13, sizeof(float *));
	
	float * h_var_59_14 = (float *)malloc(sizeof(float *));
	float * d_var_59_14;
	hipMalloc((void **)&d_var_59_14, sizeof(float *));
	
	float * h_var_59_15 = (float *)malloc(sizeof(float *));
	float * d_var_59_15;
	hipMalloc((void **)&d_var_59_15, sizeof(float *));
	
	float * h_var_59_16 = (float *)malloc(sizeof(float *));
	float * d_var_59_16;
	hipMalloc((void **)&d_var_59_16, sizeof(float *));
	
	float * h_var_59_17 = (float *)malloc(sizeof(float *));
	float * d_var_59_17;
	hipMalloc((void **)&d_var_59_17, sizeof(float *));
	
	float * h_var_59_18 = (float *)malloc(sizeof(float *));
	float * d_var_59_18;
	hipMalloc((void **)&d_var_59_18, sizeof(float *));
	
	float * h_var_59_19 = (float *)malloc(sizeof(float *));
	float * d_var_59_19;
	hipMalloc((void **)&d_var_59_19, sizeof(float *));
	
	float * h_var_60_0 = (float *)malloc(sizeof(float *));
	float * d_var_60_0;
	hipMalloc((void **)&d_var_60_0, sizeof(float *));
	
	float * h_var_60_1 = (float *)malloc(sizeof(float *));
	float * d_var_60_1;
	hipMalloc((void **)&d_var_60_1, sizeof(float *));
	
	float * h_var_60_2 = (float *)malloc(sizeof(float *));
	float * d_var_60_2;
	hipMalloc((void **)&d_var_60_2, sizeof(float *));
	
	float * h_var_60_3 = (float *)malloc(sizeof(float *));
	float * d_var_60_3;
	hipMalloc((void **)&d_var_60_3, sizeof(float *));
	
	float * h_var_60_4 = (float *)malloc(sizeof(float *));
	float * d_var_60_4;
	hipMalloc((void **)&d_var_60_4, sizeof(float *));
	
	float * h_var_60_5 = (float *)malloc(sizeof(float *));
	float * d_var_60_5;
	hipMalloc((void **)&d_var_60_5, sizeof(float *));
	
	float * h_var_60_6 = (float *)malloc(sizeof(float *));
	float * d_var_60_6;
	hipMalloc((void **)&d_var_60_6, sizeof(float *));
	
	float * h_var_60_7 = (float *)malloc(sizeof(float *));
	float * d_var_60_7;
	hipMalloc((void **)&d_var_60_7, sizeof(float *));
	
	float * h_var_60_8 = (float *)malloc(sizeof(float *));
	float * d_var_60_8;
	hipMalloc((void **)&d_var_60_8, sizeof(float *));
	
	float * h_var_60_9 = (float *)malloc(sizeof(float *));
	float * d_var_60_9;
	hipMalloc((void **)&d_var_60_9, sizeof(float *));
	
	float * h_var_60_10 = (float *)malloc(sizeof(float *));
	float * d_var_60_10;
	hipMalloc((void **)&d_var_60_10, sizeof(float *));
	
	float * h_var_60_11 = (float *)malloc(sizeof(float *));
	float * d_var_60_11;
	hipMalloc((void **)&d_var_60_11, sizeof(float *));
	
	float * h_var_60_12 = (float *)malloc(sizeof(float *));
	float * d_var_60_12;
	hipMalloc((void **)&d_var_60_12, sizeof(float *));
	
	float * h_var_60_13 = (float *)malloc(sizeof(float *));
	float * d_var_60_13;
	hipMalloc((void **)&d_var_60_13, sizeof(float *));
	
	float * h_var_60_14 = (float *)malloc(sizeof(float *));
	float * d_var_60_14;
	hipMalloc((void **)&d_var_60_14, sizeof(float *));
	
	float * h_var_60_15 = (float *)malloc(sizeof(float *));
	float * d_var_60_15;
	hipMalloc((void **)&d_var_60_15, sizeof(float *));
	
	float * h_var_60_16 = (float *)malloc(sizeof(float *));
	float * d_var_60_16;
	hipMalloc((void **)&d_var_60_16, sizeof(float *));
	
	float * h_var_60_17 = (float *)malloc(sizeof(float *));
	float * d_var_60_17;
	hipMalloc((void **)&d_var_60_17, sizeof(float *));
	
	float * h_var_60_18 = (float *)malloc(sizeof(float *));
	float * d_var_60_18;
	hipMalloc((void **)&d_var_60_18, sizeof(float *));
	
	float * h_var_60_19 = (float *)malloc(sizeof(float *));
	float * d_var_60_19;
	hipMalloc((void **)&d_var_60_19, sizeof(float *));
	
	float * h_var_61_0 = (float *)malloc(sizeof(float *));
	float * d_var_61_0;
	hipMalloc((void **)&d_var_61_0, sizeof(float *));
	
	float * h_var_61_1 = (float *)malloc(sizeof(float *));
	float * d_var_61_1;
	hipMalloc((void **)&d_var_61_1, sizeof(float *));
	
	float * h_var_61_2 = (float *)malloc(sizeof(float *));
	float * d_var_61_2;
	hipMalloc((void **)&d_var_61_2, sizeof(float *));
	
	float * h_var_61_3 = (float *)malloc(sizeof(float *));
	float * d_var_61_3;
	hipMalloc((void **)&d_var_61_3, sizeof(float *));
	
	float * h_var_61_4 = (float *)malloc(sizeof(float *));
	float * d_var_61_4;
	hipMalloc((void **)&d_var_61_4, sizeof(float *));
	
	float * h_var_61_5 = (float *)malloc(sizeof(float *));
	float * d_var_61_5;
	hipMalloc((void **)&d_var_61_5, sizeof(float *));
	
	float * h_var_61_6 = (float *)malloc(sizeof(float *));
	float * d_var_61_6;
	hipMalloc((void **)&d_var_61_6, sizeof(float *));
	
	float * h_var_61_7 = (float *)malloc(sizeof(float *));
	float * d_var_61_7;
	hipMalloc((void **)&d_var_61_7, sizeof(float *));
	
	float * h_var_61_8 = (float *)malloc(sizeof(float *));
	float * d_var_61_8;
	hipMalloc((void **)&d_var_61_8, sizeof(float *));
	
	float * h_var_61_9 = (float *)malloc(sizeof(float *));
	float * d_var_61_9;
	hipMalloc((void **)&d_var_61_9, sizeof(float *));
	
	float * h_var_61_10 = (float *)malloc(sizeof(float *));
	float * d_var_61_10;
	hipMalloc((void **)&d_var_61_10, sizeof(float *));
	
	float * h_var_61_11 = (float *)malloc(sizeof(float *));
	float * d_var_61_11;
	hipMalloc((void **)&d_var_61_11, sizeof(float *));
	
	float * h_var_61_12 = (float *)malloc(sizeof(float *));
	float * d_var_61_12;
	hipMalloc((void **)&d_var_61_12, sizeof(float *));
	
	float * h_var_61_13 = (float *)malloc(sizeof(float *));
	float * d_var_61_13;
	hipMalloc((void **)&d_var_61_13, sizeof(float *));
	
	float * h_var_61_14 = (float *)malloc(sizeof(float *));
	float * d_var_61_14;
	hipMalloc((void **)&d_var_61_14, sizeof(float *));
	
	float * h_var_61_15 = (float *)malloc(sizeof(float *));
	float * d_var_61_15;
	hipMalloc((void **)&d_var_61_15, sizeof(float *));
	
	float * h_var_61_16 = (float *)malloc(sizeof(float *));
	float * d_var_61_16;
	hipMalloc((void **)&d_var_61_16, sizeof(float *));
	
	float * h_var_61_17 = (float *)malloc(sizeof(float *));
	float * d_var_61_17;
	hipMalloc((void **)&d_var_61_17, sizeof(float *));
	
	float * h_var_61_18 = (float *)malloc(sizeof(float *));
	float * d_var_61_18;
	hipMalloc((void **)&d_var_61_18, sizeof(float *));
	
	float * h_var_61_19 = (float *)malloc(sizeof(float *));
	float * d_var_61_19;
	hipMalloc((void **)&d_var_61_19, sizeof(float *));
	
	float * h_var_62_0 = (float *)malloc(sizeof(float *));
	float * d_var_62_0;
	hipMalloc((void **)&d_var_62_0, sizeof(float *));
	
	float * h_var_62_1 = (float *)malloc(sizeof(float *));
	float * d_var_62_1;
	hipMalloc((void **)&d_var_62_1, sizeof(float *));
	
	float * h_var_62_2 = (float *)malloc(sizeof(float *));
	float * d_var_62_2;
	hipMalloc((void **)&d_var_62_2, sizeof(float *));
	
	float * h_var_62_3 = (float *)malloc(sizeof(float *));
	float * d_var_62_3;
	hipMalloc((void **)&d_var_62_3, sizeof(float *));
	
	float * h_var_62_4 = (float *)malloc(sizeof(float *));
	float * d_var_62_4;
	hipMalloc((void **)&d_var_62_4, sizeof(float *));
	
	float * h_var_62_5 = (float *)malloc(sizeof(float *));
	float * d_var_62_5;
	hipMalloc((void **)&d_var_62_5, sizeof(float *));
	
	float * h_var_62_6 = (float *)malloc(sizeof(float *));
	float * d_var_62_6;
	hipMalloc((void **)&d_var_62_6, sizeof(float *));
	
	float * h_var_62_7 = (float *)malloc(sizeof(float *));
	float * d_var_62_7;
	hipMalloc((void **)&d_var_62_7, sizeof(float *));
	
	float * h_var_62_8 = (float *)malloc(sizeof(float *));
	float * d_var_62_8;
	hipMalloc((void **)&d_var_62_8, sizeof(float *));
	
	float * h_var_62_9 = (float *)malloc(sizeof(float *));
	float * d_var_62_9;
	hipMalloc((void **)&d_var_62_9, sizeof(float *));
	
	float * h_var_62_10 = (float *)malloc(sizeof(float *));
	float * d_var_62_10;
	hipMalloc((void **)&d_var_62_10, sizeof(float *));
	
	float * h_var_62_11 = (float *)malloc(sizeof(float *));
	float * d_var_62_11;
	hipMalloc((void **)&d_var_62_11, sizeof(float *));
	
	float * h_var_62_12 = (float *)malloc(sizeof(float *));
	float * d_var_62_12;
	hipMalloc((void **)&d_var_62_12, sizeof(float *));
	
	float * h_var_62_13 = (float *)malloc(sizeof(float *));
	float * d_var_62_13;
	hipMalloc((void **)&d_var_62_13, sizeof(float *));
	
	float * h_var_62_14 = (float *)malloc(sizeof(float *));
	float * d_var_62_14;
	hipMalloc((void **)&d_var_62_14, sizeof(float *));
	
	float * h_var_62_15 = (float *)malloc(sizeof(float *));
	float * d_var_62_15;
	hipMalloc((void **)&d_var_62_15, sizeof(float *));
	
	float * h_var_62_16 = (float *)malloc(sizeof(float *));
	float * d_var_62_16;
	hipMalloc((void **)&d_var_62_16, sizeof(float *));
	
	float * h_var_62_17 = (float *)malloc(sizeof(float *));
	float * d_var_62_17;
	hipMalloc((void **)&d_var_62_17, sizeof(float *));
	
	float * h_var_62_18 = (float *)malloc(sizeof(float *));
	float * d_var_62_18;
	hipMalloc((void **)&d_var_62_18, sizeof(float *));
	
	float * h_var_62_19 = (float *)malloc(sizeof(float *));
	float * d_var_62_19;
	hipMalloc((void **)&d_var_62_19, sizeof(float *));
	
	float * h_var_63_0 = (float *)malloc(sizeof(float *));
	float * d_var_63_0;
	hipMalloc((void **)&d_var_63_0, sizeof(float *));
	
	float * h_var_63_1 = (float *)malloc(sizeof(float *));
	float * d_var_63_1;
	hipMalloc((void **)&d_var_63_1, sizeof(float *));
	
	float * h_var_63_2 = (float *)malloc(sizeof(float *));
	float * d_var_63_2;
	hipMalloc((void **)&d_var_63_2, sizeof(float *));
	
	float * h_var_63_3 = (float *)malloc(sizeof(float *));
	float * d_var_63_3;
	hipMalloc((void **)&d_var_63_3, sizeof(float *));
	
	float * h_var_63_4 = (float *)malloc(sizeof(float *));
	float * d_var_63_4;
	hipMalloc((void **)&d_var_63_4, sizeof(float *));
	
	float * h_var_63_5 = (float *)malloc(sizeof(float *));
	float * d_var_63_5;
	hipMalloc((void **)&d_var_63_5, sizeof(float *));
	
	float * h_var_63_6 = (float *)malloc(sizeof(float *));
	float * d_var_63_6;
	hipMalloc((void **)&d_var_63_6, sizeof(float *));
	
	float * h_var_63_7 = (float *)malloc(sizeof(float *));
	float * d_var_63_7;
	hipMalloc((void **)&d_var_63_7, sizeof(float *));
	
	float * h_var_63_8 = (float *)malloc(sizeof(float *));
	float * d_var_63_8;
	hipMalloc((void **)&d_var_63_8, sizeof(float *));
	
	float * h_var_63_9 = (float *)malloc(sizeof(float *));
	float * d_var_63_9;
	hipMalloc((void **)&d_var_63_9, sizeof(float *));
	
	float * h_var_63_10 = (float *)malloc(sizeof(float *));
	float * d_var_63_10;
	hipMalloc((void **)&d_var_63_10, sizeof(float *));
	
	float * h_var_63_11 = (float *)malloc(sizeof(float *));
	float * d_var_63_11;
	hipMalloc((void **)&d_var_63_11, sizeof(float *));
	
	float * h_var_63_12 = (float *)malloc(sizeof(float *));
	float * d_var_63_12;
	hipMalloc((void **)&d_var_63_12, sizeof(float *));
	
	float * h_var_63_13 = (float *)malloc(sizeof(float *));
	float * d_var_63_13;
	hipMalloc((void **)&d_var_63_13, sizeof(float *));
	
	float * h_var_63_14 = (float *)malloc(sizeof(float *));
	float * d_var_63_14;
	hipMalloc((void **)&d_var_63_14, sizeof(float *));
	
	float * h_var_63_15 = (float *)malloc(sizeof(float *));
	float * d_var_63_15;
	hipMalloc((void **)&d_var_63_15, sizeof(float *));
	
	float * h_var_63_16 = (float *)malloc(sizeof(float *));
	float * d_var_63_16;
	hipMalloc((void **)&d_var_63_16, sizeof(float *));
	
	float * h_var_63_17 = (float *)malloc(sizeof(float *));
	float * d_var_63_17;
	hipMalloc((void **)&d_var_63_17, sizeof(float *));
	
	float * h_var_63_18 = (float *)malloc(sizeof(float *));
	float * d_var_63_18;
	hipMalloc((void **)&d_var_63_18, sizeof(float *));
	
	float * h_var_63_19 = (float *)malloc(sizeof(float *));
	float * d_var_63_19;
	hipMalloc((void **)&d_var_63_19, sizeof(float *));
	
	float * h_var_64_0 = (float *)malloc(sizeof(float *));
	float * d_var_64_0;
	hipMalloc((void **)&d_var_64_0, sizeof(float *));
	
	float * h_var_64_1 = (float *)malloc(sizeof(float *));
	float * d_var_64_1;
	hipMalloc((void **)&d_var_64_1, sizeof(float *));
	
	float * h_var_64_2 = (float *)malloc(sizeof(float *));
	float * d_var_64_2;
	hipMalloc((void **)&d_var_64_2, sizeof(float *));
	
	float * h_var_64_3 = (float *)malloc(sizeof(float *));
	float * d_var_64_3;
	hipMalloc((void **)&d_var_64_3, sizeof(float *));
	
	float * h_var_64_4 = (float *)malloc(sizeof(float *));
	float * d_var_64_4;
	hipMalloc((void **)&d_var_64_4, sizeof(float *));
	
	float * h_var_64_5 = (float *)malloc(sizeof(float *));
	float * d_var_64_5;
	hipMalloc((void **)&d_var_64_5, sizeof(float *));
	
	float * h_var_64_6 = (float *)malloc(sizeof(float *));
	float * d_var_64_6;
	hipMalloc((void **)&d_var_64_6, sizeof(float *));
	
	float * h_var_64_7 = (float *)malloc(sizeof(float *));
	float * d_var_64_7;
	hipMalloc((void **)&d_var_64_7, sizeof(float *));
	
	float * h_var_64_8 = (float *)malloc(sizeof(float *));
	float * d_var_64_8;
	hipMalloc((void **)&d_var_64_8, sizeof(float *));
	
	float * h_var_64_9 = (float *)malloc(sizeof(float *));
	float * d_var_64_9;
	hipMalloc((void **)&d_var_64_9, sizeof(float *));
	
	float * h_var_64_10 = (float *)malloc(sizeof(float *));
	float * d_var_64_10;
	hipMalloc((void **)&d_var_64_10, sizeof(float *));
	
	float * h_var_64_11 = (float *)malloc(sizeof(float *));
	float * d_var_64_11;
	hipMalloc((void **)&d_var_64_11, sizeof(float *));
	
	float * h_var_64_12 = (float *)malloc(sizeof(float *));
	float * d_var_64_12;
	hipMalloc((void **)&d_var_64_12, sizeof(float *));
	
	float * h_var_64_13 = (float *)malloc(sizeof(float *));
	float * d_var_64_13;
	hipMalloc((void **)&d_var_64_13, sizeof(float *));
	
	float * h_var_64_14 = (float *)malloc(sizeof(float *));
	float * d_var_64_14;
	hipMalloc((void **)&d_var_64_14, sizeof(float *));
	
	float * h_var_64_15 = (float *)malloc(sizeof(float *));
	float * d_var_64_15;
	hipMalloc((void **)&d_var_64_15, sizeof(float *));
	
	float * h_var_64_16 = (float *)malloc(sizeof(float *));
	float * d_var_64_16;
	hipMalloc((void **)&d_var_64_16, sizeof(float *));
	
	float * h_var_64_17 = (float *)malloc(sizeof(float *));
	float * d_var_64_17;
	hipMalloc((void **)&d_var_64_17, sizeof(float *));
	
	float * h_var_64_18 = (float *)malloc(sizeof(float *));
	float * d_var_64_18;
	hipMalloc((void **)&d_var_64_18, sizeof(float *));
	
	float * h_var_64_19 = (float *)malloc(sizeof(float *));
	float * d_var_64_19;
	hipMalloc((void **)&d_var_64_19, sizeof(float *));
	
	float * h_var_65_0 = (float *)malloc(sizeof(float *));
	float * d_var_65_0;
	hipMalloc((void **)&d_var_65_0, sizeof(float *));
	
	float * h_var_65_1 = (float *)malloc(sizeof(float *));
	float * d_var_65_1;
	hipMalloc((void **)&d_var_65_1, sizeof(float *));
	
	float * h_var_65_2 = (float *)malloc(sizeof(float *));
	float * d_var_65_2;
	hipMalloc((void **)&d_var_65_2, sizeof(float *));
	
	float * h_var_65_3 = (float *)malloc(sizeof(float *));
	float * d_var_65_3;
	hipMalloc((void **)&d_var_65_3, sizeof(float *));
	
	float * h_var_65_4 = (float *)malloc(sizeof(float *));
	float * d_var_65_4;
	hipMalloc((void **)&d_var_65_4, sizeof(float *));
	
	float * h_var_65_5 = (float *)malloc(sizeof(float *));
	float * d_var_65_5;
	hipMalloc((void **)&d_var_65_5, sizeof(float *));
	
	float * h_var_65_6 = (float *)malloc(sizeof(float *));
	float * d_var_65_6;
	hipMalloc((void **)&d_var_65_6, sizeof(float *));
	
	float * h_var_65_7 = (float *)malloc(sizeof(float *));
	float * d_var_65_7;
	hipMalloc((void **)&d_var_65_7, sizeof(float *));
	
	float * h_var_65_8 = (float *)malloc(sizeof(float *));
	float * d_var_65_8;
	hipMalloc((void **)&d_var_65_8, sizeof(float *));
	
	float * h_var_65_9 = (float *)malloc(sizeof(float *));
	float * d_var_65_9;
	hipMalloc((void **)&d_var_65_9, sizeof(float *));
	
	float * h_var_65_10 = (float *)malloc(sizeof(float *));
	float * d_var_65_10;
	hipMalloc((void **)&d_var_65_10, sizeof(float *));
	
	float * h_var_65_11 = (float *)malloc(sizeof(float *));
	float * d_var_65_11;
	hipMalloc((void **)&d_var_65_11, sizeof(float *));
	
	float * h_var_65_12 = (float *)malloc(sizeof(float *));
	float * d_var_65_12;
	hipMalloc((void **)&d_var_65_12, sizeof(float *));
	
	float * h_var_65_13 = (float *)malloc(sizeof(float *));
	float * d_var_65_13;
	hipMalloc((void **)&d_var_65_13, sizeof(float *));
	
	float * h_var_65_14 = (float *)malloc(sizeof(float *));
	float * d_var_65_14;
	hipMalloc((void **)&d_var_65_14, sizeof(float *));
	
	float * h_var_65_15 = (float *)malloc(sizeof(float *));
	float * d_var_65_15;
	hipMalloc((void **)&d_var_65_15, sizeof(float *));
	
	float * h_var_65_16 = (float *)malloc(sizeof(float *));
	float * d_var_65_16;
	hipMalloc((void **)&d_var_65_16, sizeof(float *));
	
	float * h_var_65_17 = (float *)malloc(sizeof(float *));
	float * d_var_65_17;
	hipMalloc((void **)&d_var_65_17, sizeof(float *));
	
	float * h_var_65_18 = (float *)malloc(sizeof(float *));
	float * d_var_65_18;
	hipMalloc((void **)&d_var_65_18, sizeof(float *));
	
	float * h_var_65_19 = (float *)malloc(sizeof(float *));
	float * d_var_65_19;
	hipMalloc((void **)&d_var_65_19, sizeof(float *));
	
	float * h_var_66_0 = (float *)malloc(sizeof(float *));
	float * d_var_66_0;
	hipMalloc((void **)&d_var_66_0, sizeof(float *));
	
	float * h_var_66_1 = (float *)malloc(sizeof(float *));
	float * d_var_66_1;
	hipMalloc((void **)&d_var_66_1, sizeof(float *));
	
	float * h_var_66_2 = (float *)malloc(sizeof(float *));
	float * d_var_66_2;
	hipMalloc((void **)&d_var_66_2, sizeof(float *));
	
	float * h_var_66_3 = (float *)malloc(sizeof(float *));
	float * d_var_66_3;
	hipMalloc((void **)&d_var_66_3, sizeof(float *));
	
	float * h_var_66_4 = (float *)malloc(sizeof(float *));
	float * d_var_66_4;
	hipMalloc((void **)&d_var_66_4, sizeof(float *));
	
	float * h_var_66_5 = (float *)malloc(sizeof(float *));
	float * d_var_66_5;
	hipMalloc((void **)&d_var_66_5, sizeof(float *));
	
	float * h_var_66_6 = (float *)malloc(sizeof(float *));
	float * d_var_66_6;
	hipMalloc((void **)&d_var_66_6, sizeof(float *));
	
	float * h_var_66_7 = (float *)malloc(sizeof(float *));
	float * d_var_66_7;
	hipMalloc((void **)&d_var_66_7, sizeof(float *));
	
	float * h_var_66_8 = (float *)malloc(sizeof(float *));
	float * d_var_66_8;
	hipMalloc((void **)&d_var_66_8, sizeof(float *));
	
	float * h_var_66_9 = (float *)malloc(sizeof(float *));
	float * d_var_66_9;
	hipMalloc((void **)&d_var_66_9, sizeof(float *));
	
	float * h_var_66_10 = (float *)malloc(sizeof(float *));
	float * d_var_66_10;
	hipMalloc((void **)&d_var_66_10, sizeof(float *));
	
	float * h_var_66_11 = (float *)malloc(sizeof(float *));
	float * d_var_66_11;
	hipMalloc((void **)&d_var_66_11, sizeof(float *));
	
	float * h_var_66_12 = (float *)malloc(sizeof(float *));
	float * d_var_66_12;
	hipMalloc((void **)&d_var_66_12, sizeof(float *));
	
	float * h_var_66_13 = (float *)malloc(sizeof(float *));
	float * d_var_66_13;
	hipMalloc((void **)&d_var_66_13, sizeof(float *));
	
	float * h_var_66_14 = (float *)malloc(sizeof(float *));
	float * d_var_66_14;
	hipMalloc((void **)&d_var_66_14, sizeof(float *));
	
	float * h_var_66_15 = (float *)malloc(sizeof(float *));
	float * d_var_66_15;
	hipMalloc((void **)&d_var_66_15, sizeof(float *));
	
	float * h_var_66_16 = (float *)malloc(sizeof(float *));
	float * d_var_66_16;
	hipMalloc((void **)&d_var_66_16, sizeof(float *));
	
	float * h_var_66_17 = (float *)malloc(sizeof(float *));
	float * d_var_66_17;
	hipMalloc((void **)&d_var_66_17, sizeof(float *));
	
	float * h_var_66_18 = (float *)malloc(sizeof(float *));
	float * d_var_66_18;
	hipMalloc((void **)&d_var_66_18, sizeof(float *));
	
	float * h_var_66_19 = (float *)malloc(sizeof(float *));
	float * d_var_66_19;
	hipMalloc((void **)&d_var_66_19, sizeof(float *));
	
	float * h_var_67_0 = (float *)malloc(sizeof(float *));
	float * d_var_67_0;
	hipMalloc((void **)&d_var_67_0, sizeof(float *));
	
	float * h_var_67_1 = (float *)malloc(sizeof(float *));
	float * d_var_67_1;
	hipMalloc((void **)&d_var_67_1, sizeof(float *));
	
	float * h_var_67_2 = (float *)malloc(sizeof(float *));
	float * d_var_67_2;
	hipMalloc((void **)&d_var_67_2, sizeof(float *));
	
	float * h_var_67_3 = (float *)malloc(sizeof(float *));
	float * d_var_67_3;
	hipMalloc((void **)&d_var_67_3, sizeof(float *));
	
	float * h_var_67_4 = (float *)malloc(sizeof(float *));
	float * d_var_67_4;
	hipMalloc((void **)&d_var_67_4, sizeof(float *));
	
	float * h_var_67_5 = (float *)malloc(sizeof(float *));
	float * d_var_67_5;
	hipMalloc((void **)&d_var_67_5, sizeof(float *));
	
	float * h_var_67_6 = (float *)malloc(sizeof(float *));
	float * d_var_67_6;
	hipMalloc((void **)&d_var_67_6, sizeof(float *));
	
	float * h_var_67_7 = (float *)malloc(sizeof(float *));
	float * d_var_67_7;
	hipMalloc((void **)&d_var_67_7, sizeof(float *));
	
	float * h_var_67_8 = (float *)malloc(sizeof(float *));
	float * d_var_67_8;
	hipMalloc((void **)&d_var_67_8, sizeof(float *));
	
	float * h_var_67_9 = (float *)malloc(sizeof(float *));
	float * d_var_67_9;
	hipMalloc((void **)&d_var_67_9, sizeof(float *));
	
	float * h_var_67_10 = (float *)malloc(sizeof(float *));
	float * d_var_67_10;
	hipMalloc((void **)&d_var_67_10, sizeof(float *));
	
	float * h_var_67_11 = (float *)malloc(sizeof(float *));
	float * d_var_67_11;
	hipMalloc((void **)&d_var_67_11, sizeof(float *));
	
	float * h_var_67_12 = (float *)malloc(sizeof(float *));
	float * d_var_67_12;
	hipMalloc((void **)&d_var_67_12, sizeof(float *));
	
	float * h_var_67_13 = (float *)malloc(sizeof(float *));
	float * d_var_67_13;
	hipMalloc((void **)&d_var_67_13, sizeof(float *));
	
	float * h_var_67_14 = (float *)malloc(sizeof(float *));
	float * d_var_67_14;
	hipMalloc((void **)&d_var_67_14, sizeof(float *));
	
	float * h_var_67_15 = (float *)malloc(sizeof(float *));
	float * d_var_67_15;
	hipMalloc((void **)&d_var_67_15, sizeof(float *));
	
	float * h_var_67_16 = (float *)malloc(sizeof(float *));
	float * d_var_67_16;
	hipMalloc((void **)&d_var_67_16, sizeof(float *));
	
	float * h_var_67_17 = (float *)malloc(sizeof(float *));
	float * d_var_67_17;
	hipMalloc((void **)&d_var_67_17, sizeof(float *));
	
	float * h_var_67_18 = (float *)malloc(sizeof(float *));
	float * d_var_67_18;
	hipMalloc((void **)&d_var_67_18, sizeof(float *));
	
	float * h_var_67_19 = (float *)malloc(sizeof(float *));
	float * d_var_67_19;
	hipMalloc((void **)&d_var_67_19, sizeof(float *));
	
	float * h_var_68_0 = (float *)malloc(sizeof(float *));
	float * d_var_68_0;
	hipMalloc((void **)&d_var_68_0, sizeof(float *));
	
	float * h_var_68_1 = (float *)malloc(sizeof(float *));
	float * d_var_68_1;
	hipMalloc((void **)&d_var_68_1, sizeof(float *));
	
	float * h_var_68_2 = (float *)malloc(sizeof(float *));
	float * d_var_68_2;
	hipMalloc((void **)&d_var_68_2, sizeof(float *));
	
	float * h_var_68_3 = (float *)malloc(sizeof(float *));
	float * d_var_68_3;
	hipMalloc((void **)&d_var_68_3, sizeof(float *));
	
	float * h_var_68_4 = (float *)malloc(sizeof(float *));
	float * d_var_68_4;
	hipMalloc((void **)&d_var_68_4, sizeof(float *));
	
	float * h_var_68_5 = (float *)malloc(sizeof(float *));
	float * d_var_68_5;
	hipMalloc((void **)&d_var_68_5, sizeof(float *));
	
	float * h_var_68_6 = (float *)malloc(sizeof(float *));
	float * d_var_68_6;
	hipMalloc((void **)&d_var_68_6, sizeof(float *));
	
	float * h_var_68_7 = (float *)malloc(sizeof(float *));
	float * d_var_68_7;
	hipMalloc((void **)&d_var_68_7, sizeof(float *));
	
	float * h_var_68_8 = (float *)malloc(sizeof(float *));
	float * d_var_68_8;
	hipMalloc((void **)&d_var_68_8, sizeof(float *));
	
	float * h_var_68_9 = (float *)malloc(sizeof(float *));
	float * d_var_68_9;
	hipMalloc((void **)&d_var_68_9, sizeof(float *));
	
	float * h_var_68_10 = (float *)malloc(sizeof(float *));
	float * d_var_68_10;
	hipMalloc((void **)&d_var_68_10, sizeof(float *));
	
	float * h_var_68_11 = (float *)malloc(sizeof(float *));
	float * d_var_68_11;
	hipMalloc((void **)&d_var_68_11, sizeof(float *));
	
	float * h_var_68_12 = (float *)malloc(sizeof(float *));
	float * d_var_68_12;
	hipMalloc((void **)&d_var_68_12, sizeof(float *));
	
	float * h_var_68_13 = (float *)malloc(sizeof(float *));
	float * d_var_68_13;
	hipMalloc((void **)&d_var_68_13, sizeof(float *));
	
	float * h_var_68_14 = (float *)malloc(sizeof(float *));
	float * d_var_68_14;
	hipMalloc((void **)&d_var_68_14, sizeof(float *));
	
	float * h_var_68_15 = (float *)malloc(sizeof(float *));
	float * d_var_68_15;
	hipMalloc((void **)&d_var_68_15, sizeof(float *));
	
	float * h_var_68_16 = (float *)malloc(sizeof(float *));
	float * d_var_68_16;
	hipMalloc((void **)&d_var_68_16, sizeof(float *));
	
	float * h_var_68_17 = (float *)malloc(sizeof(float *));
	float * d_var_68_17;
	hipMalloc((void **)&d_var_68_17, sizeof(float *));
	
	float * h_var_68_18 = (float *)malloc(sizeof(float *));
	float * d_var_68_18;
	hipMalloc((void **)&d_var_68_18, sizeof(float *));
	
	float * h_var_68_19 = (float *)malloc(sizeof(float *));
	float * d_var_68_19;
	hipMalloc((void **)&d_var_68_19, sizeof(float *));
	
	float * h_var_69_0 = (float *)malloc(sizeof(float *));
	float * d_var_69_0;
	hipMalloc((void **)&d_var_69_0, sizeof(float *));
	
	float * h_var_69_1 = (float *)malloc(sizeof(float *));
	float * d_var_69_1;
	hipMalloc((void **)&d_var_69_1, sizeof(float *));
	
	float * h_var_69_2 = (float *)malloc(sizeof(float *));
	float * d_var_69_2;
	hipMalloc((void **)&d_var_69_2, sizeof(float *));
	
	float * h_var_69_3 = (float *)malloc(sizeof(float *));
	float * d_var_69_3;
	hipMalloc((void **)&d_var_69_3, sizeof(float *));
	
	float * h_var_69_4 = (float *)malloc(sizeof(float *));
	float * d_var_69_4;
	hipMalloc((void **)&d_var_69_4, sizeof(float *));
	
	float * h_var_69_5 = (float *)malloc(sizeof(float *));
	float * d_var_69_5;
	hipMalloc((void **)&d_var_69_5, sizeof(float *));
	
	float * h_var_69_6 = (float *)malloc(sizeof(float *));
	float * d_var_69_6;
	hipMalloc((void **)&d_var_69_6, sizeof(float *));
	
	float * h_var_69_7 = (float *)malloc(sizeof(float *));
	float * d_var_69_7;
	hipMalloc((void **)&d_var_69_7, sizeof(float *));
	
	float * h_var_69_8 = (float *)malloc(sizeof(float *));
	float * d_var_69_8;
	hipMalloc((void **)&d_var_69_8, sizeof(float *));
	
	float * h_var_69_9 = (float *)malloc(sizeof(float *));
	float * d_var_69_9;
	hipMalloc((void **)&d_var_69_9, sizeof(float *));
	
	float * h_var_69_10 = (float *)malloc(sizeof(float *));
	float * d_var_69_10;
	hipMalloc((void **)&d_var_69_10, sizeof(float *));
	
	float * h_var_69_11 = (float *)malloc(sizeof(float *));
	float * d_var_69_11;
	hipMalloc((void **)&d_var_69_11, sizeof(float *));
	
	float * h_var_69_12 = (float *)malloc(sizeof(float *));
	float * d_var_69_12;
	hipMalloc((void **)&d_var_69_12, sizeof(float *));
	
	float * h_var_69_13 = (float *)malloc(sizeof(float *));
	float * d_var_69_13;
	hipMalloc((void **)&d_var_69_13, sizeof(float *));
	
	float * h_var_69_14 = (float *)malloc(sizeof(float *));
	float * d_var_69_14;
	hipMalloc((void **)&d_var_69_14, sizeof(float *));
	
	float * h_var_69_15 = (float *)malloc(sizeof(float *));
	float * d_var_69_15;
	hipMalloc((void **)&d_var_69_15, sizeof(float *));
	
	float * h_var_69_16 = (float *)malloc(sizeof(float *));
	float * d_var_69_16;
	hipMalloc((void **)&d_var_69_16, sizeof(float *));
	
	float * h_var_69_17 = (float *)malloc(sizeof(float *));
	float * d_var_69_17;
	hipMalloc((void **)&d_var_69_17, sizeof(float *));
	
	float * h_var_69_18 = (float *)malloc(sizeof(float *));
	float * d_var_69_18;
	hipMalloc((void **)&d_var_69_18, sizeof(float *));
	
	float * h_var_69_19 = (float *)malloc(sizeof(float *));
	float * d_var_69_19;
	hipMalloc((void **)&d_var_69_19, sizeof(float *));
	
	float * h_var_70_0 = (float *)malloc(sizeof(float *));
	float * d_var_70_0;
	hipMalloc((void **)&d_var_70_0, sizeof(float *));
	
	float * h_var_70_1 = (float *)malloc(sizeof(float *));
	float * d_var_70_1;
	hipMalloc((void **)&d_var_70_1, sizeof(float *));
	
	float * h_var_70_2 = (float *)malloc(sizeof(float *));
	float * d_var_70_2;
	hipMalloc((void **)&d_var_70_2, sizeof(float *));
	
	float * h_var_70_3 = (float *)malloc(sizeof(float *));
	float * d_var_70_3;
	hipMalloc((void **)&d_var_70_3, sizeof(float *));
	
	float * h_var_70_4 = (float *)malloc(sizeof(float *));
	float * d_var_70_4;
	hipMalloc((void **)&d_var_70_4, sizeof(float *));
	
	float * h_var_70_5 = (float *)malloc(sizeof(float *));
	float * d_var_70_5;
	hipMalloc((void **)&d_var_70_5, sizeof(float *));
	
	float * h_var_70_6 = (float *)malloc(sizeof(float *));
	float * d_var_70_6;
	hipMalloc((void **)&d_var_70_6, sizeof(float *));
	
	float * h_var_70_7 = (float *)malloc(sizeof(float *));
	float * d_var_70_7;
	hipMalloc((void **)&d_var_70_7, sizeof(float *));
	
	float * h_var_70_8 = (float *)malloc(sizeof(float *));
	float * d_var_70_8;
	hipMalloc((void **)&d_var_70_8, sizeof(float *));
	
	float * h_var_70_9 = (float *)malloc(sizeof(float *));
	float * d_var_70_9;
	hipMalloc((void **)&d_var_70_9, sizeof(float *));
	
	float * h_var_70_10 = (float *)malloc(sizeof(float *));
	float * d_var_70_10;
	hipMalloc((void **)&d_var_70_10, sizeof(float *));
	
	float * h_var_70_11 = (float *)malloc(sizeof(float *));
	float * d_var_70_11;
	hipMalloc((void **)&d_var_70_11, sizeof(float *));
	
	float * h_var_70_12 = (float *)malloc(sizeof(float *));
	float * d_var_70_12;
	hipMalloc((void **)&d_var_70_12, sizeof(float *));
	
	float * h_var_70_13 = (float *)malloc(sizeof(float *));
	float * d_var_70_13;
	hipMalloc((void **)&d_var_70_13, sizeof(float *));
	
	float * h_var_70_14 = (float *)malloc(sizeof(float *));
	float * d_var_70_14;
	hipMalloc((void **)&d_var_70_14, sizeof(float *));
	
	float * h_var_70_15 = (float *)malloc(sizeof(float *));
	float * d_var_70_15;
	hipMalloc((void **)&d_var_70_15, sizeof(float *));
	
	float * h_var_70_16 = (float *)malloc(sizeof(float *));
	float * d_var_70_16;
	hipMalloc((void **)&d_var_70_16, sizeof(float *));
	
	float * h_var_70_17 = (float *)malloc(sizeof(float *));
	float * d_var_70_17;
	hipMalloc((void **)&d_var_70_17, sizeof(float *));
	
	float * h_var_70_18 = (float *)malloc(sizeof(float *));
	float * d_var_70_18;
	hipMalloc((void **)&d_var_70_18, sizeof(float *));
	
	float * h_var_70_19 = (float *)malloc(sizeof(float *));
	float * d_var_70_19;
	hipMalloc((void **)&d_var_70_19, sizeof(float *));
	
	float * h_var_71_0 = (float *)malloc(sizeof(float *));
	float * d_var_71_0;
	hipMalloc((void **)&d_var_71_0, sizeof(float *));
	
	float * h_var_71_1 = (float *)malloc(sizeof(float *));
	float * d_var_71_1;
	hipMalloc((void **)&d_var_71_1, sizeof(float *));
	
	float * h_var_71_2 = (float *)malloc(sizeof(float *));
	float * d_var_71_2;
	hipMalloc((void **)&d_var_71_2, sizeof(float *));
	
	float * h_var_71_3 = (float *)malloc(sizeof(float *));
	float * d_var_71_3;
	hipMalloc((void **)&d_var_71_3, sizeof(float *));
	
	float * h_var_71_4 = (float *)malloc(sizeof(float *));
	float * d_var_71_4;
	hipMalloc((void **)&d_var_71_4, sizeof(float *));
	
	float * h_var_71_5 = (float *)malloc(sizeof(float *));
	float * d_var_71_5;
	hipMalloc((void **)&d_var_71_5, sizeof(float *));
	
	float * h_var_71_6 = (float *)malloc(sizeof(float *));
	float * d_var_71_6;
	hipMalloc((void **)&d_var_71_6, sizeof(float *));
	
	float * h_var_71_7 = (float *)malloc(sizeof(float *));
	float * d_var_71_7;
	hipMalloc((void **)&d_var_71_7, sizeof(float *));
	
	float * h_var_71_8 = (float *)malloc(sizeof(float *));
	float * d_var_71_8;
	hipMalloc((void **)&d_var_71_8, sizeof(float *));
	
	float * h_var_71_9 = (float *)malloc(sizeof(float *));
	float * d_var_71_9;
	hipMalloc((void **)&d_var_71_9, sizeof(float *));
	
	float * h_var_71_10 = (float *)malloc(sizeof(float *));
	float * d_var_71_10;
	hipMalloc((void **)&d_var_71_10, sizeof(float *));
	
	float * h_var_71_11 = (float *)malloc(sizeof(float *));
	float * d_var_71_11;
	hipMalloc((void **)&d_var_71_11, sizeof(float *));
	
	float * h_var_71_12 = (float *)malloc(sizeof(float *));
	float * d_var_71_12;
	hipMalloc((void **)&d_var_71_12, sizeof(float *));
	
	float * h_var_71_13 = (float *)malloc(sizeof(float *));
	float * d_var_71_13;
	hipMalloc((void **)&d_var_71_13, sizeof(float *));
	
	float * h_var_71_14 = (float *)malloc(sizeof(float *));
	float * d_var_71_14;
	hipMalloc((void **)&d_var_71_14, sizeof(float *));
	
	float * h_var_71_15 = (float *)malloc(sizeof(float *));
	float * d_var_71_15;
	hipMalloc((void **)&d_var_71_15, sizeof(float *));
	
	float * h_var_71_16 = (float *)malloc(sizeof(float *));
	float * d_var_71_16;
	hipMalloc((void **)&d_var_71_16, sizeof(float *));
	
	float * h_var_71_17 = (float *)malloc(sizeof(float *));
	float * d_var_71_17;
	hipMalloc((void **)&d_var_71_17, sizeof(float *));
	
	float * h_var_71_18 = (float *)malloc(sizeof(float *));
	float * d_var_71_18;
	hipMalloc((void **)&d_var_71_18, sizeof(float *));
	
	float * h_var_71_19 = (float *)malloc(sizeof(float *));
	float * d_var_71_19;
	hipMalloc((void **)&d_var_71_19, sizeof(float *));
	
	float * h_var_72_0 = (float *)malloc(sizeof(float *));
	float * d_var_72_0;
	hipMalloc((void **)&d_var_72_0, sizeof(float *));
	
	float * h_var_72_1 = (float *)malloc(sizeof(float *));
	float * d_var_72_1;
	hipMalloc((void **)&d_var_72_1, sizeof(float *));
	
	float * h_var_72_2 = (float *)malloc(sizeof(float *));
	float * d_var_72_2;
	hipMalloc((void **)&d_var_72_2, sizeof(float *));
	
	float * h_var_72_3 = (float *)malloc(sizeof(float *));
	float * d_var_72_3;
	hipMalloc((void **)&d_var_72_3, sizeof(float *));
	
	float * h_var_72_4 = (float *)malloc(sizeof(float *));
	float * d_var_72_4;
	hipMalloc((void **)&d_var_72_4, sizeof(float *));
	
	float * h_var_72_5 = (float *)malloc(sizeof(float *));
	float * d_var_72_5;
	hipMalloc((void **)&d_var_72_5, sizeof(float *));
	
	float * h_var_72_6 = (float *)malloc(sizeof(float *));
	float * d_var_72_6;
	hipMalloc((void **)&d_var_72_6, sizeof(float *));
	
	float * h_var_72_7 = (float *)malloc(sizeof(float *));
	float * d_var_72_7;
	hipMalloc((void **)&d_var_72_7, sizeof(float *));
	
	float * h_var_72_8 = (float *)malloc(sizeof(float *));
	float * d_var_72_8;
	hipMalloc((void **)&d_var_72_8, sizeof(float *));
	
	float * h_var_72_9 = (float *)malloc(sizeof(float *));
	float * d_var_72_9;
	hipMalloc((void **)&d_var_72_9, sizeof(float *));
	
	float * h_var_72_10 = (float *)malloc(sizeof(float *));
	float * d_var_72_10;
	hipMalloc((void **)&d_var_72_10, sizeof(float *));
	
	float * h_var_72_11 = (float *)malloc(sizeof(float *));
	float * d_var_72_11;
	hipMalloc((void **)&d_var_72_11, sizeof(float *));
	
	float * h_var_72_12 = (float *)malloc(sizeof(float *));
	float * d_var_72_12;
	hipMalloc((void **)&d_var_72_12, sizeof(float *));
	
	float * h_var_72_13 = (float *)malloc(sizeof(float *));
	float * d_var_72_13;
	hipMalloc((void **)&d_var_72_13, sizeof(float *));
	
	float * h_var_72_14 = (float *)malloc(sizeof(float *));
	float * d_var_72_14;
	hipMalloc((void **)&d_var_72_14, sizeof(float *));
	
	float * h_var_72_15 = (float *)malloc(sizeof(float *));
	float * d_var_72_15;
	hipMalloc((void **)&d_var_72_15, sizeof(float *));
	
	float * h_var_72_16 = (float *)malloc(sizeof(float *));
	float * d_var_72_16;
	hipMalloc((void **)&d_var_72_16, sizeof(float *));
	
	float * h_var_72_17 = (float *)malloc(sizeof(float *));
	float * d_var_72_17;
	hipMalloc((void **)&d_var_72_17, sizeof(float *));
	
	float * h_var_72_18 = (float *)malloc(sizeof(float *));
	float * d_var_72_18;
	hipMalloc((void **)&d_var_72_18, sizeof(float *));
	
	float * h_var_72_19 = (float *)malloc(sizeof(float *));
	float * d_var_72_19;
	hipMalloc((void **)&d_var_72_19, sizeof(float *));
	
	float * h_var_73_0 = (float *)malloc(sizeof(float *));
	float * d_var_73_0;
	hipMalloc((void **)&d_var_73_0, sizeof(float *));
	
	float * h_var_73_1 = (float *)malloc(sizeof(float *));
	float * d_var_73_1;
	hipMalloc((void **)&d_var_73_1, sizeof(float *));
	
	float * h_var_73_2 = (float *)malloc(sizeof(float *));
	float * d_var_73_2;
	hipMalloc((void **)&d_var_73_2, sizeof(float *));
	
	float * h_var_73_3 = (float *)malloc(sizeof(float *));
	float * d_var_73_3;
	hipMalloc((void **)&d_var_73_3, sizeof(float *));
	
	float * h_var_73_4 = (float *)malloc(sizeof(float *));
	float * d_var_73_4;
	hipMalloc((void **)&d_var_73_4, sizeof(float *));
	
	float * h_var_73_5 = (float *)malloc(sizeof(float *));
	float * d_var_73_5;
	hipMalloc((void **)&d_var_73_5, sizeof(float *));
	
	float * h_var_73_6 = (float *)malloc(sizeof(float *));
	float * d_var_73_6;
	hipMalloc((void **)&d_var_73_6, sizeof(float *));
	
	float * h_var_73_7 = (float *)malloc(sizeof(float *));
	float * d_var_73_7;
	hipMalloc((void **)&d_var_73_7, sizeof(float *));
	
	float * h_var_73_8 = (float *)malloc(sizeof(float *));
	float * d_var_73_8;
	hipMalloc((void **)&d_var_73_8, sizeof(float *));
	
	float * h_var_73_9 = (float *)malloc(sizeof(float *));
	float * d_var_73_9;
	hipMalloc((void **)&d_var_73_9, sizeof(float *));
	
	float * h_var_73_10 = (float *)malloc(sizeof(float *));
	float * d_var_73_10;
	hipMalloc((void **)&d_var_73_10, sizeof(float *));
	
	float * h_var_73_11 = (float *)malloc(sizeof(float *));
	float * d_var_73_11;
	hipMalloc((void **)&d_var_73_11, sizeof(float *));
	
	float * h_var_73_12 = (float *)malloc(sizeof(float *));
	float * d_var_73_12;
	hipMalloc((void **)&d_var_73_12, sizeof(float *));
	
	float * h_var_73_13 = (float *)malloc(sizeof(float *));
	float * d_var_73_13;
	hipMalloc((void **)&d_var_73_13, sizeof(float *));
	
	float * h_var_73_14 = (float *)malloc(sizeof(float *));
	float * d_var_73_14;
	hipMalloc((void **)&d_var_73_14, sizeof(float *));
	
	float * h_var_73_15 = (float *)malloc(sizeof(float *));
	float * d_var_73_15;
	hipMalloc((void **)&d_var_73_15, sizeof(float *));
	
	float * h_var_73_16 = (float *)malloc(sizeof(float *));
	float * d_var_73_16;
	hipMalloc((void **)&d_var_73_16, sizeof(float *));
	
	float * h_var_73_17 = (float *)malloc(sizeof(float *));
	float * d_var_73_17;
	hipMalloc((void **)&d_var_73_17, sizeof(float *));
	
	float * h_var_73_18 = (float *)malloc(sizeof(float *));
	float * d_var_73_18;
	hipMalloc((void **)&d_var_73_18, sizeof(float *));
	
	float * h_var_73_19 = (float *)malloc(sizeof(float *));
	float * d_var_73_19;
	hipMalloc((void **)&d_var_73_19, sizeof(float *));
	
	float * h_var_74_0 = (float *)malloc(sizeof(float *));
	float * d_var_74_0;
	hipMalloc((void **)&d_var_74_0, sizeof(float *));
	
	float * h_var_74_1 = (float *)malloc(sizeof(float *));
	float * d_var_74_1;
	hipMalloc((void **)&d_var_74_1, sizeof(float *));
	
	float * h_var_74_2 = (float *)malloc(sizeof(float *));
	float * d_var_74_2;
	hipMalloc((void **)&d_var_74_2, sizeof(float *));
	
	float * h_var_74_3 = (float *)malloc(sizeof(float *));
	float * d_var_74_3;
	hipMalloc((void **)&d_var_74_3, sizeof(float *));
	
	float * h_var_74_4 = (float *)malloc(sizeof(float *));
	float * d_var_74_4;
	hipMalloc((void **)&d_var_74_4, sizeof(float *));
	
	float * h_var_74_5 = (float *)malloc(sizeof(float *));
	float * d_var_74_5;
	hipMalloc((void **)&d_var_74_5, sizeof(float *));
	
	float * h_var_74_6 = (float *)malloc(sizeof(float *));
	float * d_var_74_6;
	hipMalloc((void **)&d_var_74_6, sizeof(float *));
	
	float * h_var_74_7 = (float *)malloc(sizeof(float *));
	float * d_var_74_7;
	hipMalloc((void **)&d_var_74_7, sizeof(float *));
	
	float * h_var_74_8 = (float *)malloc(sizeof(float *));
	float * d_var_74_8;
	hipMalloc((void **)&d_var_74_8, sizeof(float *));
	
	float * h_var_74_9 = (float *)malloc(sizeof(float *));
	float * d_var_74_9;
	hipMalloc((void **)&d_var_74_9, sizeof(float *));
	
	float * h_var_74_10 = (float *)malloc(sizeof(float *));
	float * d_var_74_10;
	hipMalloc((void **)&d_var_74_10, sizeof(float *));
	
	float * h_var_74_11 = (float *)malloc(sizeof(float *));
	float * d_var_74_11;
	hipMalloc((void **)&d_var_74_11, sizeof(float *));
	
	float * h_var_74_12 = (float *)malloc(sizeof(float *));
	float * d_var_74_12;
	hipMalloc((void **)&d_var_74_12, sizeof(float *));
	
	float * h_var_74_13 = (float *)malloc(sizeof(float *));
	float * d_var_74_13;
	hipMalloc((void **)&d_var_74_13, sizeof(float *));
	
	float * h_var_74_14 = (float *)malloc(sizeof(float *));
	float * d_var_74_14;
	hipMalloc((void **)&d_var_74_14, sizeof(float *));
	
	float * h_var_74_15 = (float *)malloc(sizeof(float *));
	float * d_var_74_15;
	hipMalloc((void **)&d_var_74_15, sizeof(float *));
	
	float * h_var_74_16 = (float *)malloc(sizeof(float *));
	float * d_var_74_16;
	hipMalloc((void **)&d_var_74_16, sizeof(float *));
	
	float * h_var_74_17 = (float *)malloc(sizeof(float *));
	float * d_var_74_17;
	hipMalloc((void **)&d_var_74_17, sizeof(float *));
	
	float * h_var_74_18 = (float *)malloc(sizeof(float *));
	float * d_var_74_18;
	hipMalloc((void **)&d_var_74_18, sizeof(float *));
	
	float * h_var_74_19 = (float *)malloc(sizeof(float *));
	float * d_var_74_19;
	hipMalloc((void **)&d_var_74_19, sizeof(float *));
	
	float * h_var_75_0 = (float *)malloc(sizeof(float *));
	float * d_var_75_0;
	hipMalloc((void **)&d_var_75_0, sizeof(float *));
	
	float * h_var_75_1 = (float *)malloc(sizeof(float *));
	float * d_var_75_1;
	hipMalloc((void **)&d_var_75_1, sizeof(float *));
	
	float * h_var_75_2 = (float *)malloc(sizeof(float *));
	float * d_var_75_2;
	hipMalloc((void **)&d_var_75_2, sizeof(float *));
	
	float * h_var_75_3 = (float *)malloc(sizeof(float *));
	float * d_var_75_3;
	hipMalloc((void **)&d_var_75_3, sizeof(float *));
	
	float * h_var_75_4 = (float *)malloc(sizeof(float *));
	float * d_var_75_4;
	hipMalloc((void **)&d_var_75_4, sizeof(float *));
	
	float * h_var_75_5 = (float *)malloc(sizeof(float *));
	float * d_var_75_5;
	hipMalloc((void **)&d_var_75_5, sizeof(float *));
	
	float * h_var_75_6 = (float *)malloc(sizeof(float *));
	float * d_var_75_6;
	hipMalloc((void **)&d_var_75_6, sizeof(float *));
	
	float * h_var_75_7 = (float *)malloc(sizeof(float *));
	float * d_var_75_7;
	hipMalloc((void **)&d_var_75_7, sizeof(float *));
	
	float * h_var_75_8 = (float *)malloc(sizeof(float *));
	float * d_var_75_8;
	hipMalloc((void **)&d_var_75_8, sizeof(float *));
	
	float * h_var_75_9 = (float *)malloc(sizeof(float *));
	float * d_var_75_9;
	hipMalloc((void **)&d_var_75_9, sizeof(float *));
	
	float * h_var_75_10 = (float *)malloc(sizeof(float *));
	float * d_var_75_10;
	hipMalloc((void **)&d_var_75_10, sizeof(float *));
	
	float * h_var_75_11 = (float *)malloc(sizeof(float *));
	float * d_var_75_11;
	hipMalloc((void **)&d_var_75_11, sizeof(float *));
	
	float * h_var_75_12 = (float *)malloc(sizeof(float *));
	float * d_var_75_12;
	hipMalloc((void **)&d_var_75_12, sizeof(float *));
	
	float * h_var_75_13 = (float *)malloc(sizeof(float *));
	float * d_var_75_13;
	hipMalloc((void **)&d_var_75_13, sizeof(float *));
	
	float * h_var_75_14 = (float *)malloc(sizeof(float *));
	float * d_var_75_14;
	hipMalloc((void **)&d_var_75_14, sizeof(float *));
	
	float * h_var_75_15 = (float *)malloc(sizeof(float *));
	float * d_var_75_15;
	hipMalloc((void **)&d_var_75_15, sizeof(float *));
	
	float * h_var_75_16 = (float *)malloc(sizeof(float *));
	float * d_var_75_16;
	hipMalloc((void **)&d_var_75_16, sizeof(float *));
	
	float * h_var_75_17 = (float *)malloc(sizeof(float *));
	float * d_var_75_17;
	hipMalloc((void **)&d_var_75_17, sizeof(float *));
	
	float * h_var_75_18 = (float *)malloc(sizeof(float *));
	float * d_var_75_18;
	hipMalloc((void **)&d_var_75_18, sizeof(float *));
	
	float * h_var_75_19 = (float *)malloc(sizeof(float *));
	float * d_var_75_19;
	hipMalloc((void **)&d_var_75_19, sizeof(float *));
	
	float * h_var_76_0 = (float *)malloc(sizeof(float *));
	float * d_var_76_0;
	hipMalloc((void **)&d_var_76_0, sizeof(float *));
	
	float * h_var_76_1 = (float *)malloc(sizeof(float *));
	float * d_var_76_1;
	hipMalloc((void **)&d_var_76_1, sizeof(float *));
	
	float * h_var_76_2 = (float *)malloc(sizeof(float *));
	float * d_var_76_2;
	hipMalloc((void **)&d_var_76_2, sizeof(float *));
	
	float * h_var_76_3 = (float *)malloc(sizeof(float *));
	float * d_var_76_3;
	hipMalloc((void **)&d_var_76_3, sizeof(float *));
	
	float * h_var_76_4 = (float *)malloc(sizeof(float *));
	float * d_var_76_4;
	hipMalloc((void **)&d_var_76_4, sizeof(float *));
	
	float * h_var_76_5 = (float *)malloc(sizeof(float *));
	float * d_var_76_5;
	hipMalloc((void **)&d_var_76_5, sizeof(float *));
	
	float * h_var_76_6 = (float *)malloc(sizeof(float *));
	float * d_var_76_6;
	hipMalloc((void **)&d_var_76_6, sizeof(float *));
	
	float * h_var_76_7 = (float *)malloc(sizeof(float *));
	float * d_var_76_7;
	hipMalloc((void **)&d_var_76_7, sizeof(float *));
	
	float * h_var_76_8 = (float *)malloc(sizeof(float *));
	float * d_var_76_8;
	hipMalloc((void **)&d_var_76_8, sizeof(float *));
	
	float * h_var_76_9 = (float *)malloc(sizeof(float *));
	float * d_var_76_9;
	hipMalloc((void **)&d_var_76_9, sizeof(float *));
	
	float * h_var_76_10 = (float *)malloc(sizeof(float *));
	float * d_var_76_10;
	hipMalloc((void **)&d_var_76_10, sizeof(float *));
	
	float * h_var_76_11 = (float *)malloc(sizeof(float *));
	float * d_var_76_11;
	hipMalloc((void **)&d_var_76_11, sizeof(float *));
	
	float * h_var_76_12 = (float *)malloc(sizeof(float *));
	float * d_var_76_12;
	hipMalloc((void **)&d_var_76_12, sizeof(float *));
	
	float * h_var_76_13 = (float *)malloc(sizeof(float *));
	float * d_var_76_13;
	hipMalloc((void **)&d_var_76_13, sizeof(float *));
	
	float * h_var_76_14 = (float *)malloc(sizeof(float *));
	float * d_var_76_14;
	hipMalloc((void **)&d_var_76_14, sizeof(float *));
	
	float * h_var_76_15 = (float *)malloc(sizeof(float *));
	float * d_var_76_15;
	hipMalloc((void **)&d_var_76_15, sizeof(float *));
	
	float * h_var_76_16 = (float *)malloc(sizeof(float *));
	float * d_var_76_16;
	hipMalloc((void **)&d_var_76_16, sizeof(float *));
	
	float * h_var_76_17 = (float *)malloc(sizeof(float *));
	float * d_var_76_17;
	hipMalloc((void **)&d_var_76_17, sizeof(float *));
	
	float * h_var_76_18 = (float *)malloc(sizeof(float *));
	float * d_var_76_18;
	hipMalloc((void **)&d_var_76_18, sizeof(float *));
	
	float * h_var_76_19 = (float *)malloc(sizeof(float *));
	float * d_var_76_19;
	hipMalloc((void **)&d_var_76_19, sizeof(float *));
	
	float * h_var_77_0 = (float *)malloc(sizeof(float *));
	float * d_var_77_0;
	hipMalloc((void **)&d_var_77_0, sizeof(float *));
	
	float * h_var_77_1 = (float *)malloc(sizeof(float *));
	float * d_var_77_1;
	hipMalloc((void **)&d_var_77_1, sizeof(float *));
	
	float * h_var_77_2 = (float *)malloc(sizeof(float *));
	float * d_var_77_2;
	hipMalloc((void **)&d_var_77_2, sizeof(float *));
	
	float * h_var_77_3 = (float *)malloc(sizeof(float *));
	float * d_var_77_3;
	hipMalloc((void **)&d_var_77_3, sizeof(float *));
	
	float * h_var_77_4 = (float *)malloc(sizeof(float *));
	float * d_var_77_4;
	hipMalloc((void **)&d_var_77_4, sizeof(float *));
	
	float * h_var_77_5 = (float *)malloc(sizeof(float *));
	float * d_var_77_5;
	hipMalloc((void **)&d_var_77_5, sizeof(float *));
	
	float * h_var_77_6 = (float *)malloc(sizeof(float *));
	float * d_var_77_6;
	hipMalloc((void **)&d_var_77_6, sizeof(float *));
	
	float * h_var_77_7 = (float *)malloc(sizeof(float *));
	float * d_var_77_7;
	hipMalloc((void **)&d_var_77_7, sizeof(float *));
	
	float * h_var_77_8 = (float *)malloc(sizeof(float *));
	float * d_var_77_8;
	hipMalloc((void **)&d_var_77_8, sizeof(float *));
	
	float * h_var_77_9 = (float *)malloc(sizeof(float *));
	float * d_var_77_9;
	hipMalloc((void **)&d_var_77_9, sizeof(float *));
	
	float * h_var_77_10 = (float *)malloc(sizeof(float *));
	float * d_var_77_10;
	hipMalloc((void **)&d_var_77_10, sizeof(float *));
	
	float * h_var_77_11 = (float *)malloc(sizeof(float *));
	float * d_var_77_11;
	hipMalloc((void **)&d_var_77_11, sizeof(float *));
	
	float * h_var_77_12 = (float *)malloc(sizeof(float *));
	float * d_var_77_12;
	hipMalloc((void **)&d_var_77_12, sizeof(float *));
	
	float * h_var_77_13 = (float *)malloc(sizeof(float *));
	float * d_var_77_13;
	hipMalloc((void **)&d_var_77_13, sizeof(float *));
	
	float * h_var_77_14 = (float *)malloc(sizeof(float *));
	float * d_var_77_14;
	hipMalloc((void **)&d_var_77_14, sizeof(float *));
	
	float * h_var_77_15 = (float *)malloc(sizeof(float *));
	float * d_var_77_15;
	hipMalloc((void **)&d_var_77_15, sizeof(float *));
	
	float * h_var_77_16 = (float *)malloc(sizeof(float *));
	float * d_var_77_16;
	hipMalloc((void **)&d_var_77_16, sizeof(float *));
	
	float * h_var_77_17 = (float *)malloc(sizeof(float *));
	float * d_var_77_17;
	hipMalloc((void **)&d_var_77_17, sizeof(float *));
	
	float * h_var_77_18 = (float *)malloc(sizeof(float *));
	float * d_var_77_18;
	hipMalloc((void **)&d_var_77_18, sizeof(float *));
	
	float * h_var_77_19 = (float *)malloc(sizeof(float *));
	float * d_var_77_19;
	hipMalloc((void **)&d_var_77_19, sizeof(float *));
	
	float * h_var_78_0 = (float *)malloc(sizeof(float *));
	float * d_var_78_0;
	hipMalloc((void **)&d_var_78_0, sizeof(float *));
	
	float * h_var_78_1 = (float *)malloc(sizeof(float *));
	float * d_var_78_1;
	hipMalloc((void **)&d_var_78_1, sizeof(float *));
	
	float * h_var_78_2 = (float *)malloc(sizeof(float *));
	float * d_var_78_2;
	hipMalloc((void **)&d_var_78_2, sizeof(float *));
	
	float * h_var_78_3 = (float *)malloc(sizeof(float *));
	float * d_var_78_3;
	hipMalloc((void **)&d_var_78_3, sizeof(float *));
	
	float * h_var_78_4 = (float *)malloc(sizeof(float *));
	float * d_var_78_4;
	hipMalloc((void **)&d_var_78_4, sizeof(float *));
	
	float * h_var_78_5 = (float *)malloc(sizeof(float *));
	float * d_var_78_5;
	hipMalloc((void **)&d_var_78_5, sizeof(float *));
	
	float * h_var_78_6 = (float *)malloc(sizeof(float *));
	float * d_var_78_6;
	hipMalloc((void **)&d_var_78_6, sizeof(float *));
	
	float * h_var_78_7 = (float *)malloc(sizeof(float *));
	float * d_var_78_7;
	hipMalloc((void **)&d_var_78_7, sizeof(float *));
	
	float * h_var_78_8 = (float *)malloc(sizeof(float *));
	float * d_var_78_8;
	hipMalloc((void **)&d_var_78_8, sizeof(float *));
	
	float * h_var_78_9 = (float *)malloc(sizeof(float *));
	float * d_var_78_9;
	hipMalloc((void **)&d_var_78_9, sizeof(float *));
	
	float * h_var_78_10 = (float *)malloc(sizeof(float *));
	float * d_var_78_10;
	hipMalloc((void **)&d_var_78_10, sizeof(float *));
	
	float * h_var_78_11 = (float *)malloc(sizeof(float *));
	float * d_var_78_11;
	hipMalloc((void **)&d_var_78_11, sizeof(float *));
	
	float * h_var_78_12 = (float *)malloc(sizeof(float *));
	float * d_var_78_12;
	hipMalloc((void **)&d_var_78_12, sizeof(float *));
	
	float * h_var_78_13 = (float *)malloc(sizeof(float *));
	float * d_var_78_13;
	hipMalloc((void **)&d_var_78_13, sizeof(float *));
	
	float * h_var_78_14 = (float *)malloc(sizeof(float *));
	float * d_var_78_14;
	hipMalloc((void **)&d_var_78_14, sizeof(float *));
	
	float * h_var_78_15 = (float *)malloc(sizeof(float *));
	float * d_var_78_15;
	hipMalloc((void **)&d_var_78_15, sizeof(float *));
	
	float * h_var_78_16 = (float *)malloc(sizeof(float *));
	float * d_var_78_16;
	hipMalloc((void **)&d_var_78_16, sizeof(float *));
	
	float * h_var_78_17 = (float *)malloc(sizeof(float *));
	float * d_var_78_17;
	hipMalloc((void **)&d_var_78_17, sizeof(float *));
	
	float * h_var_78_18 = (float *)malloc(sizeof(float *));
	float * d_var_78_18;
	hipMalloc((void **)&d_var_78_18, sizeof(float *));
	
	float * h_var_78_19 = (float *)malloc(sizeof(float *));
	float * d_var_78_19;
	hipMalloc((void **)&d_var_78_19, sizeof(float *));
	
	float * h_var_79_0 = (float *)malloc(sizeof(float *));
	float * d_var_79_0;
	hipMalloc((void **)&d_var_79_0, sizeof(float *));
	
	float * h_var_79_1 = (float *)malloc(sizeof(float *));
	float * d_var_79_1;
	hipMalloc((void **)&d_var_79_1, sizeof(float *));
	
	float * h_var_79_2 = (float *)malloc(sizeof(float *));
	float * d_var_79_2;
	hipMalloc((void **)&d_var_79_2, sizeof(float *));
	
	float * h_var_79_3 = (float *)malloc(sizeof(float *));
	float * d_var_79_3;
	hipMalloc((void **)&d_var_79_3, sizeof(float *));
	
	float * h_var_79_4 = (float *)malloc(sizeof(float *));
	float * d_var_79_4;
	hipMalloc((void **)&d_var_79_4, sizeof(float *));
	
	float * h_var_79_5 = (float *)malloc(sizeof(float *));
	float * d_var_79_5;
	hipMalloc((void **)&d_var_79_5, sizeof(float *));
	
	float * h_var_79_6 = (float *)malloc(sizeof(float *));
	float * d_var_79_6;
	hipMalloc((void **)&d_var_79_6, sizeof(float *));
	
	float * h_var_79_7 = (float *)malloc(sizeof(float *));
	float * d_var_79_7;
	hipMalloc((void **)&d_var_79_7, sizeof(float *));
	
	float * h_var_79_8 = (float *)malloc(sizeof(float *));
	float * d_var_79_8;
	hipMalloc((void **)&d_var_79_8, sizeof(float *));
	
	float * h_var_79_9 = (float *)malloc(sizeof(float *));
	float * d_var_79_9;
	hipMalloc((void **)&d_var_79_9, sizeof(float *));
	
	float * h_var_79_10 = (float *)malloc(sizeof(float *));
	float * d_var_79_10;
	hipMalloc((void **)&d_var_79_10, sizeof(float *));
	
	float * h_var_79_11 = (float *)malloc(sizeof(float *));
	float * d_var_79_11;
	hipMalloc((void **)&d_var_79_11, sizeof(float *));
	
	float * h_var_79_12 = (float *)malloc(sizeof(float *));
	float * d_var_79_12;
	hipMalloc((void **)&d_var_79_12, sizeof(float *));
	
	float * h_var_79_13 = (float *)malloc(sizeof(float *));
	float * d_var_79_13;
	hipMalloc((void **)&d_var_79_13, sizeof(float *));
	
	float * h_var_79_14 = (float *)malloc(sizeof(float *));
	float * d_var_79_14;
	hipMalloc((void **)&d_var_79_14, sizeof(float *));
	
	float * h_var_79_15 = (float *)malloc(sizeof(float *));
	float * d_var_79_15;
	hipMalloc((void **)&d_var_79_15, sizeof(float *));
	
	float * h_var_79_16 = (float *)malloc(sizeof(float *));
	float * d_var_79_16;
	hipMalloc((void **)&d_var_79_16, sizeof(float *));
	
	float * h_var_79_17 = (float *)malloc(sizeof(float *));
	float * d_var_79_17;
	hipMalloc((void **)&d_var_79_17, sizeof(float *));
	
	float * h_var_79_18 = (float *)malloc(sizeof(float *));
	float * d_var_79_18;
	hipMalloc((void **)&d_var_79_18, sizeof(float *));
	
	float * h_var_79_19 = (float *)malloc(sizeof(float *));
	float * d_var_79_19;
	hipMalloc((void **)&d_var_79_19, sizeof(float *));
	
	float * h_var_80_0 = (float *)malloc(sizeof(float *));
	float * d_var_80_0;
	hipMalloc((void **)&d_var_80_0, sizeof(float *));
	
	float * h_var_80_1 = (float *)malloc(sizeof(float *));
	float * d_var_80_1;
	hipMalloc((void **)&d_var_80_1, sizeof(float *));
	
	float * h_var_80_2 = (float *)malloc(sizeof(float *));
	float * d_var_80_2;
	hipMalloc((void **)&d_var_80_2, sizeof(float *));
	
	float * h_var_80_3 = (float *)malloc(sizeof(float *));
	float * d_var_80_3;
	hipMalloc((void **)&d_var_80_3, sizeof(float *));
	
	float * h_var_80_4 = (float *)malloc(sizeof(float *));
	float * d_var_80_4;
	hipMalloc((void **)&d_var_80_4, sizeof(float *));
	
	float * h_var_80_5 = (float *)malloc(sizeof(float *));
	float * d_var_80_5;
	hipMalloc((void **)&d_var_80_5, sizeof(float *));
	
	float * h_var_80_6 = (float *)malloc(sizeof(float *));
	float * d_var_80_6;
	hipMalloc((void **)&d_var_80_6, sizeof(float *));
	
	float * h_var_80_7 = (float *)malloc(sizeof(float *));
	float * d_var_80_7;
	hipMalloc((void **)&d_var_80_7, sizeof(float *));
	
	float * h_var_80_8 = (float *)malloc(sizeof(float *));
	float * d_var_80_8;
	hipMalloc((void **)&d_var_80_8, sizeof(float *));
	
	float * h_var_80_9 = (float *)malloc(sizeof(float *));
	float * d_var_80_9;
	hipMalloc((void **)&d_var_80_9, sizeof(float *));
	
	float * h_var_80_10 = (float *)malloc(sizeof(float *));
	float * d_var_80_10;
	hipMalloc((void **)&d_var_80_10, sizeof(float *));
	
	float * h_var_80_11 = (float *)malloc(sizeof(float *));
	float * d_var_80_11;
	hipMalloc((void **)&d_var_80_11, sizeof(float *));
	
	float * h_var_80_12 = (float *)malloc(sizeof(float *));
	float * d_var_80_12;
	hipMalloc((void **)&d_var_80_12, sizeof(float *));
	
	float * h_var_80_13 = (float *)malloc(sizeof(float *));
	float * d_var_80_13;
	hipMalloc((void **)&d_var_80_13, sizeof(float *));
	
	float * h_var_80_14 = (float *)malloc(sizeof(float *));
	float * d_var_80_14;
	hipMalloc((void **)&d_var_80_14, sizeof(float *));
	
	float * h_var_80_15 = (float *)malloc(sizeof(float *));
	float * d_var_80_15;
	hipMalloc((void **)&d_var_80_15, sizeof(float *));
	
	float * h_var_80_16 = (float *)malloc(sizeof(float *));
	float * d_var_80_16;
	hipMalloc((void **)&d_var_80_16, sizeof(float *));
	
	float * h_var_80_17 = (float *)malloc(sizeof(float *));
	float * d_var_80_17;
	hipMalloc((void **)&d_var_80_17, sizeof(float *));
	
	float * h_var_80_18 = (float *)malloc(sizeof(float *));
	float * d_var_80_18;
	hipMalloc((void **)&d_var_80_18, sizeof(float *));
	
	float * h_var_80_19 = (float *)malloc(sizeof(float *));
	float * d_var_80_19;
	hipMalloc((void **)&d_var_80_19, sizeof(float *));
	
	float * h_var_81_0 = (float *)malloc(sizeof(float *));
	float * d_var_81_0;
	hipMalloc((void **)&d_var_81_0, sizeof(float *));
	
	float * h_var_81_1 = (float *)malloc(sizeof(float *));
	float * d_var_81_1;
	hipMalloc((void **)&d_var_81_1, sizeof(float *));
	
	float * h_var_81_2 = (float *)malloc(sizeof(float *));
	float * d_var_81_2;
	hipMalloc((void **)&d_var_81_2, sizeof(float *));
	
	float * h_var_81_3 = (float *)malloc(sizeof(float *));
	float * d_var_81_3;
	hipMalloc((void **)&d_var_81_3, sizeof(float *));
	
	float * h_var_81_4 = (float *)malloc(sizeof(float *));
	float * d_var_81_4;
	hipMalloc((void **)&d_var_81_4, sizeof(float *));
	
	float * h_var_81_5 = (float *)malloc(sizeof(float *));
	float * d_var_81_5;
	hipMalloc((void **)&d_var_81_5, sizeof(float *));
	
	float * h_var_81_6 = (float *)malloc(sizeof(float *));
	float * d_var_81_6;
	hipMalloc((void **)&d_var_81_6, sizeof(float *));
	
	float * h_var_81_7 = (float *)malloc(sizeof(float *));
	float * d_var_81_7;
	hipMalloc((void **)&d_var_81_7, sizeof(float *));
	
	float * h_var_81_8 = (float *)malloc(sizeof(float *));
	float * d_var_81_8;
	hipMalloc((void **)&d_var_81_8, sizeof(float *));
	
	float * h_var_81_9 = (float *)malloc(sizeof(float *));
	float * d_var_81_9;
	hipMalloc((void **)&d_var_81_9, sizeof(float *));
	
	float * h_var_81_10 = (float *)malloc(sizeof(float *));
	float * d_var_81_10;
	hipMalloc((void **)&d_var_81_10, sizeof(float *));
	
	float * h_var_81_11 = (float *)malloc(sizeof(float *));
	float * d_var_81_11;
	hipMalloc((void **)&d_var_81_11, sizeof(float *));
	
	float * h_var_81_12 = (float *)malloc(sizeof(float *));
	float * d_var_81_12;
	hipMalloc((void **)&d_var_81_12, sizeof(float *));
	
	float * h_var_81_13 = (float *)malloc(sizeof(float *));
	float * d_var_81_13;
	hipMalloc((void **)&d_var_81_13, sizeof(float *));
	
	float * h_var_81_14 = (float *)malloc(sizeof(float *));
	float * d_var_81_14;
	hipMalloc((void **)&d_var_81_14, sizeof(float *));
	
	float * h_var_81_15 = (float *)malloc(sizeof(float *));
	float * d_var_81_15;
	hipMalloc((void **)&d_var_81_15, sizeof(float *));
	
	float * h_var_81_16 = (float *)malloc(sizeof(float *));
	float * d_var_81_16;
	hipMalloc((void **)&d_var_81_16, sizeof(float *));
	
	float * h_var_81_17 = (float *)malloc(sizeof(float *));
	float * d_var_81_17;
	hipMalloc((void **)&d_var_81_17, sizeof(float *));
	
	float * h_var_81_18 = (float *)malloc(sizeof(float *));
	float * d_var_81_18;
	hipMalloc((void **)&d_var_81_18, sizeof(float *));
	
	float * h_var_81_19 = (float *)malloc(sizeof(float *));
	float * d_var_81_19;
	hipMalloc((void **)&d_var_81_19, sizeof(float *));
	
	float * h_var_82_0 = (float *)malloc(sizeof(float *));
	float * d_var_82_0;
	hipMalloc((void **)&d_var_82_0, sizeof(float *));
	
	float * h_var_82_1 = (float *)malloc(sizeof(float *));
	float * d_var_82_1;
	hipMalloc((void **)&d_var_82_1, sizeof(float *));
	
	float * h_var_82_2 = (float *)malloc(sizeof(float *));
	float * d_var_82_2;
	hipMalloc((void **)&d_var_82_2, sizeof(float *));
	
	float * h_var_82_3 = (float *)malloc(sizeof(float *));
	float * d_var_82_3;
	hipMalloc((void **)&d_var_82_3, sizeof(float *));
	
	float * h_var_82_4 = (float *)malloc(sizeof(float *));
	float * d_var_82_4;
	hipMalloc((void **)&d_var_82_4, sizeof(float *));
	
	float * h_var_82_5 = (float *)malloc(sizeof(float *));
	float * d_var_82_5;
	hipMalloc((void **)&d_var_82_5, sizeof(float *));
	
	float * h_var_82_6 = (float *)malloc(sizeof(float *));
	float * d_var_82_6;
	hipMalloc((void **)&d_var_82_6, sizeof(float *));
	
	float * h_var_82_7 = (float *)malloc(sizeof(float *));
	float * d_var_82_7;
	hipMalloc((void **)&d_var_82_7, sizeof(float *));
	
	float * h_var_82_8 = (float *)malloc(sizeof(float *));
	float * d_var_82_8;
	hipMalloc((void **)&d_var_82_8, sizeof(float *));
	
	float * h_var_82_9 = (float *)malloc(sizeof(float *));
	float * d_var_82_9;
	hipMalloc((void **)&d_var_82_9, sizeof(float *));
	
	float * h_var_82_10 = (float *)malloc(sizeof(float *));
	float * d_var_82_10;
	hipMalloc((void **)&d_var_82_10, sizeof(float *));
	
	float * h_var_82_11 = (float *)malloc(sizeof(float *));
	float * d_var_82_11;
	hipMalloc((void **)&d_var_82_11, sizeof(float *));
	
	float * h_var_82_12 = (float *)malloc(sizeof(float *));
	float * d_var_82_12;
	hipMalloc((void **)&d_var_82_12, sizeof(float *));
	
	float * h_var_82_13 = (float *)malloc(sizeof(float *));
	float * d_var_82_13;
	hipMalloc((void **)&d_var_82_13, sizeof(float *));
	
	float * h_var_82_14 = (float *)malloc(sizeof(float *));
	float * d_var_82_14;
	hipMalloc((void **)&d_var_82_14, sizeof(float *));
	
	float * h_var_82_15 = (float *)malloc(sizeof(float *));
	float * d_var_82_15;
	hipMalloc((void **)&d_var_82_15, sizeof(float *));
	
	float * h_var_82_16 = (float *)malloc(sizeof(float *));
	float * d_var_82_16;
	hipMalloc((void **)&d_var_82_16, sizeof(float *));
	
	float * h_var_82_17 = (float *)malloc(sizeof(float *));
	float * d_var_82_17;
	hipMalloc((void **)&d_var_82_17, sizeof(float *));
	
	float * h_var_82_18 = (float *)malloc(sizeof(float *));
	float * d_var_82_18;
	hipMalloc((void **)&d_var_82_18, sizeof(float *));
	
	float * h_var_82_19 = (float *)malloc(sizeof(float *));
	float * d_var_82_19;
	hipMalloc((void **)&d_var_82_19, sizeof(float *));
	
	float * h_var_83_0 = (float *)malloc(sizeof(float *));
	float * d_var_83_0;
	hipMalloc((void **)&d_var_83_0, sizeof(float *));
	
	float * h_var_83_1 = (float *)malloc(sizeof(float *));
	float * d_var_83_1;
	hipMalloc((void **)&d_var_83_1, sizeof(float *));
	
	float * h_var_83_2 = (float *)malloc(sizeof(float *));
	float * d_var_83_2;
	hipMalloc((void **)&d_var_83_2, sizeof(float *));
	
	float * h_var_83_3 = (float *)malloc(sizeof(float *));
	float * d_var_83_3;
	hipMalloc((void **)&d_var_83_3, sizeof(float *));
	
	float * h_var_83_4 = (float *)malloc(sizeof(float *));
	float * d_var_83_4;
	hipMalloc((void **)&d_var_83_4, sizeof(float *));
	
	float * h_var_83_5 = (float *)malloc(sizeof(float *));
	float * d_var_83_5;
	hipMalloc((void **)&d_var_83_5, sizeof(float *));
	
	float * h_var_83_6 = (float *)malloc(sizeof(float *));
	float * d_var_83_6;
	hipMalloc((void **)&d_var_83_6, sizeof(float *));
	
	float * h_var_83_7 = (float *)malloc(sizeof(float *));
	float * d_var_83_7;
	hipMalloc((void **)&d_var_83_7, sizeof(float *));
	
	float * h_var_83_8 = (float *)malloc(sizeof(float *));
	float * d_var_83_8;
	hipMalloc((void **)&d_var_83_8, sizeof(float *));
	
	float * h_var_83_9 = (float *)malloc(sizeof(float *));
	float * d_var_83_9;
	hipMalloc((void **)&d_var_83_9, sizeof(float *));
	
	float * h_var_83_10 = (float *)malloc(sizeof(float *));
	float * d_var_83_10;
	hipMalloc((void **)&d_var_83_10, sizeof(float *));
	
	float * h_var_83_11 = (float *)malloc(sizeof(float *));
	float * d_var_83_11;
	hipMalloc((void **)&d_var_83_11, sizeof(float *));
	
	float * h_var_83_12 = (float *)malloc(sizeof(float *));
	float * d_var_83_12;
	hipMalloc((void **)&d_var_83_12, sizeof(float *));
	
	float * h_var_83_13 = (float *)malloc(sizeof(float *));
	float * d_var_83_13;
	hipMalloc((void **)&d_var_83_13, sizeof(float *));
	
	float * h_var_83_14 = (float *)malloc(sizeof(float *));
	float * d_var_83_14;
	hipMalloc((void **)&d_var_83_14, sizeof(float *));
	
	float * h_var_83_15 = (float *)malloc(sizeof(float *));
	float * d_var_83_15;
	hipMalloc((void **)&d_var_83_15, sizeof(float *));
	
	float * h_var_83_16 = (float *)malloc(sizeof(float *));
	float * d_var_83_16;
	hipMalloc((void **)&d_var_83_16, sizeof(float *));
	
	float * h_var_83_17 = (float *)malloc(sizeof(float *));
	float * d_var_83_17;
	hipMalloc((void **)&d_var_83_17, sizeof(float *));
	
	float * h_var_83_18 = (float *)malloc(sizeof(float *));
	float * d_var_83_18;
	hipMalloc((void **)&d_var_83_18, sizeof(float *));
	
	float * h_var_83_19 = (float *)malloc(sizeof(float *));
	float * d_var_83_19;
	hipMalloc((void **)&d_var_83_19, sizeof(float *));
	
	float * h_var_84_0 = (float *)malloc(sizeof(float *));
	float * d_var_84_0;
	hipMalloc((void **)&d_var_84_0, sizeof(float *));
	
	float * h_var_84_1 = (float *)malloc(sizeof(float *));
	float * d_var_84_1;
	hipMalloc((void **)&d_var_84_1, sizeof(float *));
	
	float * h_var_84_2 = (float *)malloc(sizeof(float *));
	float * d_var_84_2;
	hipMalloc((void **)&d_var_84_2, sizeof(float *));
	
	float * h_var_84_3 = (float *)malloc(sizeof(float *));
	float * d_var_84_3;
	hipMalloc((void **)&d_var_84_3, sizeof(float *));
	
	float * h_var_84_4 = (float *)malloc(sizeof(float *));
	float * d_var_84_4;
	hipMalloc((void **)&d_var_84_4, sizeof(float *));
	
	float * h_var_84_5 = (float *)malloc(sizeof(float *));
	float * d_var_84_5;
	hipMalloc((void **)&d_var_84_5, sizeof(float *));
	
	float * h_var_84_6 = (float *)malloc(sizeof(float *));
	float * d_var_84_6;
	hipMalloc((void **)&d_var_84_6, sizeof(float *));
	
	float * h_var_84_7 = (float *)malloc(sizeof(float *));
	float * d_var_84_7;
	hipMalloc((void **)&d_var_84_7, sizeof(float *));
	
	float * h_var_84_8 = (float *)malloc(sizeof(float *));
	float * d_var_84_8;
	hipMalloc((void **)&d_var_84_8, sizeof(float *));
	
	float * h_var_84_9 = (float *)malloc(sizeof(float *));
	float * d_var_84_9;
	hipMalloc((void **)&d_var_84_9, sizeof(float *));
	
	float * h_var_84_10 = (float *)malloc(sizeof(float *));
	float * d_var_84_10;
	hipMalloc((void **)&d_var_84_10, sizeof(float *));
	
	float * h_var_84_11 = (float *)malloc(sizeof(float *));
	float * d_var_84_11;
	hipMalloc((void **)&d_var_84_11, sizeof(float *));
	
	float * h_var_84_12 = (float *)malloc(sizeof(float *));
	float * d_var_84_12;
	hipMalloc((void **)&d_var_84_12, sizeof(float *));
	
	float * h_var_84_13 = (float *)malloc(sizeof(float *));
	float * d_var_84_13;
	hipMalloc((void **)&d_var_84_13, sizeof(float *));
	
	float * h_var_84_14 = (float *)malloc(sizeof(float *));
	float * d_var_84_14;
	hipMalloc((void **)&d_var_84_14, sizeof(float *));
	
	float * h_var_84_15 = (float *)malloc(sizeof(float *));
	float * d_var_84_15;
	hipMalloc((void **)&d_var_84_15, sizeof(float *));
	
	float * h_var_84_16 = (float *)malloc(sizeof(float *));
	float * d_var_84_16;
	hipMalloc((void **)&d_var_84_16, sizeof(float *));
	
	float * h_var_84_17 = (float *)malloc(sizeof(float *));
	float * d_var_84_17;
	hipMalloc((void **)&d_var_84_17, sizeof(float *));
	
	float * h_var_84_18 = (float *)malloc(sizeof(float *));
	float * d_var_84_18;
	hipMalloc((void **)&d_var_84_18, sizeof(float *));
	
	float * h_var_84_19 = (float *)malloc(sizeof(float *));
	float * d_var_84_19;
	hipMalloc((void **)&d_var_84_19, sizeof(float *));
	
	float * h_var_85_0 = (float *)malloc(sizeof(float *));
	float * d_var_85_0;
	hipMalloc((void **)&d_var_85_0, sizeof(float *));
	
	float * h_var_85_1 = (float *)malloc(sizeof(float *));
	float * d_var_85_1;
	hipMalloc((void **)&d_var_85_1, sizeof(float *));
	
	float * h_var_85_2 = (float *)malloc(sizeof(float *));
	float * d_var_85_2;
	hipMalloc((void **)&d_var_85_2, sizeof(float *));
	
	float * h_var_85_3 = (float *)malloc(sizeof(float *));
	float * d_var_85_3;
	hipMalloc((void **)&d_var_85_3, sizeof(float *));
	
	float * h_var_85_4 = (float *)malloc(sizeof(float *));
	float * d_var_85_4;
	hipMalloc((void **)&d_var_85_4, sizeof(float *));
	
	float * h_var_85_5 = (float *)malloc(sizeof(float *));
	float * d_var_85_5;
	hipMalloc((void **)&d_var_85_5, sizeof(float *));
	
	float * h_var_85_6 = (float *)malloc(sizeof(float *));
	float * d_var_85_6;
	hipMalloc((void **)&d_var_85_6, sizeof(float *));
	
	float * h_var_85_7 = (float *)malloc(sizeof(float *));
	float * d_var_85_7;
	hipMalloc((void **)&d_var_85_7, sizeof(float *));
	
	float * h_var_85_8 = (float *)malloc(sizeof(float *));
	float * d_var_85_8;
	hipMalloc((void **)&d_var_85_8, sizeof(float *));
	
	float * h_var_85_9 = (float *)malloc(sizeof(float *));
	float * d_var_85_9;
	hipMalloc((void **)&d_var_85_9, sizeof(float *));
	
	float * h_var_85_10 = (float *)malloc(sizeof(float *));
	float * d_var_85_10;
	hipMalloc((void **)&d_var_85_10, sizeof(float *));
	
	float * h_var_85_11 = (float *)malloc(sizeof(float *));
	float * d_var_85_11;
	hipMalloc((void **)&d_var_85_11, sizeof(float *));
	
	float * h_var_85_12 = (float *)malloc(sizeof(float *));
	float * d_var_85_12;
	hipMalloc((void **)&d_var_85_12, sizeof(float *));
	
	float * h_var_85_13 = (float *)malloc(sizeof(float *));
	float * d_var_85_13;
	hipMalloc((void **)&d_var_85_13, sizeof(float *));
	
	float * h_var_85_14 = (float *)malloc(sizeof(float *));
	float * d_var_85_14;
	hipMalloc((void **)&d_var_85_14, sizeof(float *));
	
	float * h_var_85_15 = (float *)malloc(sizeof(float *));
	float * d_var_85_15;
	hipMalloc((void **)&d_var_85_15, sizeof(float *));
	
	float * h_var_85_16 = (float *)malloc(sizeof(float *));
	float * d_var_85_16;
	hipMalloc((void **)&d_var_85_16, sizeof(float *));
	
	float * h_var_85_17 = (float *)malloc(sizeof(float *));
	float * d_var_85_17;
	hipMalloc((void **)&d_var_85_17, sizeof(float *));
	
	float * h_var_85_18 = (float *)malloc(sizeof(float *));
	float * d_var_85_18;
	hipMalloc((void **)&d_var_85_18, sizeof(float *));
	
	float * h_var_85_19 = (float *)malloc(sizeof(float *));
	float * d_var_85_19;
	hipMalloc((void **)&d_var_85_19, sizeof(float *));
	
	float * h_var_86_0 = (float *)malloc(sizeof(float *));
	float * d_var_86_0;
	hipMalloc((void **)&d_var_86_0, sizeof(float *));
	
	float * h_var_86_1 = (float *)malloc(sizeof(float *));
	float * d_var_86_1;
	hipMalloc((void **)&d_var_86_1, sizeof(float *));
	
	float * h_var_86_2 = (float *)malloc(sizeof(float *));
	float * d_var_86_2;
	hipMalloc((void **)&d_var_86_2, sizeof(float *));
	
	float * h_var_86_3 = (float *)malloc(sizeof(float *));
	float * d_var_86_3;
	hipMalloc((void **)&d_var_86_3, sizeof(float *));
	
	float * h_var_86_4 = (float *)malloc(sizeof(float *));
	float * d_var_86_4;
	hipMalloc((void **)&d_var_86_4, sizeof(float *));
	
	float * h_var_86_5 = (float *)malloc(sizeof(float *));
	float * d_var_86_5;
	hipMalloc((void **)&d_var_86_5, sizeof(float *));
	
	float * h_var_86_6 = (float *)malloc(sizeof(float *));
	float * d_var_86_6;
	hipMalloc((void **)&d_var_86_6, sizeof(float *));
	
	float * h_var_86_7 = (float *)malloc(sizeof(float *));
	float * d_var_86_7;
	hipMalloc((void **)&d_var_86_7, sizeof(float *));
	
	float * h_var_86_8 = (float *)malloc(sizeof(float *));
	float * d_var_86_8;
	hipMalloc((void **)&d_var_86_8, sizeof(float *));
	
	float * h_var_86_9 = (float *)malloc(sizeof(float *));
	float * d_var_86_9;
	hipMalloc((void **)&d_var_86_9, sizeof(float *));
	
	float * h_var_86_10 = (float *)malloc(sizeof(float *));
	float * d_var_86_10;
	hipMalloc((void **)&d_var_86_10, sizeof(float *));
	
	float * h_var_86_11 = (float *)malloc(sizeof(float *));
	float * d_var_86_11;
	hipMalloc((void **)&d_var_86_11, sizeof(float *));
	
	float * h_var_86_12 = (float *)malloc(sizeof(float *));
	float * d_var_86_12;
	hipMalloc((void **)&d_var_86_12, sizeof(float *));
	
	float * h_var_86_13 = (float *)malloc(sizeof(float *));
	float * d_var_86_13;
	hipMalloc((void **)&d_var_86_13, sizeof(float *));
	
	float * h_var_86_14 = (float *)malloc(sizeof(float *));
	float * d_var_86_14;
	hipMalloc((void **)&d_var_86_14, sizeof(float *));
	
	float * h_var_86_15 = (float *)malloc(sizeof(float *));
	float * d_var_86_15;
	hipMalloc((void **)&d_var_86_15, sizeof(float *));
	
	float * h_var_86_16 = (float *)malloc(sizeof(float *));
	float * d_var_86_16;
	hipMalloc((void **)&d_var_86_16, sizeof(float *));
	
	float * h_var_86_17 = (float *)malloc(sizeof(float *));
	float * d_var_86_17;
	hipMalloc((void **)&d_var_86_17, sizeof(float *));
	
	float * h_var_86_18 = (float *)malloc(sizeof(float *));
	float * d_var_86_18;
	hipMalloc((void **)&d_var_86_18, sizeof(float *));
	
	float * h_var_86_19 = (float *)malloc(sizeof(float *));
	float * d_var_86_19;
	hipMalloc((void **)&d_var_86_19, sizeof(float *));
	
	float * h_var_87_0 = (float *)malloc(sizeof(float *));
	float * d_var_87_0;
	hipMalloc((void **)&d_var_87_0, sizeof(float *));
	
	float * h_var_87_1 = (float *)malloc(sizeof(float *));
	float * d_var_87_1;
	hipMalloc((void **)&d_var_87_1, sizeof(float *));
	
	float * h_var_87_2 = (float *)malloc(sizeof(float *));
	float * d_var_87_2;
	hipMalloc((void **)&d_var_87_2, sizeof(float *));
	
	float * h_var_87_3 = (float *)malloc(sizeof(float *));
	float * d_var_87_3;
	hipMalloc((void **)&d_var_87_3, sizeof(float *));
	
	float * h_var_87_4 = (float *)malloc(sizeof(float *));
	float * d_var_87_4;
	hipMalloc((void **)&d_var_87_4, sizeof(float *));
	
	float * h_var_87_5 = (float *)malloc(sizeof(float *));
	float * d_var_87_5;
	hipMalloc((void **)&d_var_87_5, sizeof(float *));
	
	float * h_var_87_6 = (float *)malloc(sizeof(float *));
	float * d_var_87_6;
	hipMalloc((void **)&d_var_87_6, sizeof(float *));
	
	float * h_var_87_7 = (float *)malloc(sizeof(float *));
	float * d_var_87_7;
	hipMalloc((void **)&d_var_87_7, sizeof(float *));
	
	float * h_var_87_8 = (float *)malloc(sizeof(float *));
	float * d_var_87_8;
	hipMalloc((void **)&d_var_87_8, sizeof(float *));
	
	float * h_var_87_9 = (float *)malloc(sizeof(float *));
	float * d_var_87_9;
	hipMalloc((void **)&d_var_87_9, sizeof(float *));
	
	float * h_var_87_10 = (float *)malloc(sizeof(float *));
	float * d_var_87_10;
	hipMalloc((void **)&d_var_87_10, sizeof(float *));
	
	float * h_var_87_11 = (float *)malloc(sizeof(float *));
	float * d_var_87_11;
	hipMalloc((void **)&d_var_87_11, sizeof(float *));
	
	float * h_var_87_12 = (float *)malloc(sizeof(float *));
	float * d_var_87_12;
	hipMalloc((void **)&d_var_87_12, sizeof(float *));
	
	float * h_var_87_13 = (float *)malloc(sizeof(float *));
	float * d_var_87_13;
	hipMalloc((void **)&d_var_87_13, sizeof(float *));
	
	float * h_var_87_14 = (float *)malloc(sizeof(float *));
	float * d_var_87_14;
	hipMalloc((void **)&d_var_87_14, sizeof(float *));
	
	float * h_var_87_15 = (float *)malloc(sizeof(float *));
	float * d_var_87_15;
	hipMalloc((void **)&d_var_87_15, sizeof(float *));
	
	float * h_var_87_16 = (float *)malloc(sizeof(float *));
	float * d_var_87_16;
	hipMalloc((void **)&d_var_87_16, sizeof(float *));
	
	float * h_var_87_17 = (float *)malloc(sizeof(float *));
	float * d_var_87_17;
	hipMalloc((void **)&d_var_87_17, sizeof(float *));
	
	float * h_var_87_18 = (float *)malloc(sizeof(float *));
	float * d_var_87_18;
	hipMalloc((void **)&d_var_87_18, sizeof(float *));
	
	float * h_var_87_19 = (float *)malloc(sizeof(float *));
	float * d_var_87_19;
	hipMalloc((void **)&d_var_87_19, sizeof(float *));
	
	float * h_var_88_0 = (float *)malloc(sizeof(float *));
	float * d_var_88_0;
	hipMalloc((void **)&d_var_88_0, sizeof(float *));
	
	float * h_var_88_1 = (float *)malloc(sizeof(float *));
	float * d_var_88_1;
	hipMalloc((void **)&d_var_88_1, sizeof(float *));
	
	float * h_var_88_2 = (float *)malloc(sizeof(float *));
	float * d_var_88_2;
	hipMalloc((void **)&d_var_88_2, sizeof(float *));
	
	float * h_var_88_3 = (float *)malloc(sizeof(float *));
	float * d_var_88_3;
	hipMalloc((void **)&d_var_88_3, sizeof(float *));
	
	float * h_var_88_4 = (float *)malloc(sizeof(float *));
	float * d_var_88_4;
	hipMalloc((void **)&d_var_88_4, sizeof(float *));
	
	float * h_var_88_5 = (float *)malloc(sizeof(float *));
	float * d_var_88_5;
	hipMalloc((void **)&d_var_88_5, sizeof(float *));
	
	float * h_var_88_6 = (float *)malloc(sizeof(float *));
	float * d_var_88_6;
	hipMalloc((void **)&d_var_88_6, sizeof(float *));
	
	float * h_var_88_7 = (float *)malloc(sizeof(float *));
	float * d_var_88_7;
	hipMalloc((void **)&d_var_88_7, sizeof(float *));
	
	float * h_var_88_8 = (float *)malloc(sizeof(float *));
	float * d_var_88_8;
	hipMalloc((void **)&d_var_88_8, sizeof(float *));
	
	float * h_var_88_9 = (float *)malloc(sizeof(float *));
	float * d_var_88_9;
	hipMalloc((void **)&d_var_88_9, sizeof(float *));
	
	float * h_var_88_10 = (float *)malloc(sizeof(float *));
	float * d_var_88_10;
	hipMalloc((void **)&d_var_88_10, sizeof(float *));
	
	float * h_var_88_11 = (float *)malloc(sizeof(float *));
	float * d_var_88_11;
	hipMalloc((void **)&d_var_88_11, sizeof(float *));
	
	float * h_var_88_12 = (float *)malloc(sizeof(float *));
	float * d_var_88_12;
	hipMalloc((void **)&d_var_88_12, sizeof(float *));
	
	float * h_var_88_13 = (float *)malloc(sizeof(float *));
	float * d_var_88_13;
	hipMalloc((void **)&d_var_88_13, sizeof(float *));
	
	float * h_var_88_14 = (float *)malloc(sizeof(float *));
	float * d_var_88_14;
	hipMalloc((void **)&d_var_88_14, sizeof(float *));
	
	float * h_var_88_15 = (float *)malloc(sizeof(float *));
	float * d_var_88_15;
	hipMalloc((void **)&d_var_88_15, sizeof(float *));
	
	float * h_var_88_16 = (float *)malloc(sizeof(float *));
	float * d_var_88_16;
	hipMalloc((void **)&d_var_88_16, sizeof(float *));
	
	float * h_var_88_17 = (float *)malloc(sizeof(float *));
	float * d_var_88_17;
	hipMalloc((void **)&d_var_88_17, sizeof(float *));
	
	float * h_var_88_18 = (float *)malloc(sizeof(float *));
	float * d_var_88_18;
	hipMalloc((void **)&d_var_88_18, sizeof(float *));
	
	float * h_var_88_19 = (float *)malloc(sizeof(float *));
	float * d_var_88_19;
	hipMalloc((void **)&d_var_88_19, sizeof(float *));
	
	float * h_var_89_0 = (float *)malloc(sizeof(float *));
	float * d_var_89_0;
	hipMalloc((void **)&d_var_89_0, sizeof(float *));
	
	float * h_var_89_1 = (float *)malloc(sizeof(float *));
	float * d_var_89_1;
	hipMalloc((void **)&d_var_89_1, sizeof(float *));
	
	float * h_var_89_2 = (float *)malloc(sizeof(float *));
	float * d_var_89_2;
	hipMalloc((void **)&d_var_89_2, sizeof(float *));
	
	float * h_var_89_3 = (float *)malloc(sizeof(float *));
	float * d_var_89_3;
	hipMalloc((void **)&d_var_89_3, sizeof(float *));
	
	float * h_var_89_4 = (float *)malloc(sizeof(float *));
	float * d_var_89_4;
	hipMalloc((void **)&d_var_89_4, sizeof(float *));
	
	float * h_var_89_5 = (float *)malloc(sizeof(float *));
	float * d_var_89_5;
	hipMalloc((void **)&d_var_89_5, sizeof(float *));
	
	float * h_var_89_6 = (float *)malloc(sizeof(float *));
	float * d_var_89_6;
	hipMalloc((void **)&d_var_89_6, sizeof(float *));
	
	float * h_var_89_7 = (float *)malloc(sizeof(float *));
	float * d_var_89_7;
	hipMalloc((void **)&d_var_89_7, sizeof(float *));
	
	float * h_var_89_8 = (float *)malloc(sizeof(float *));
	float * d_var_89_8;
	hipMalloc((void **)&d_var_89_8, sizeof(float *));
	
	float * h_var_89_9 = (float *)malloc(sizeof(float *));
	float * d_var_89_9;
	hipMalloc((void **)&d_var_89_9, sizeof(float *));
	
	float * h_var_89_10 = (float *)malloc(sizeof(float *));
	float * d_var_89_10;
	hipMalloc((void **)&d_var_89_10, sizeof(float *));
	
	float * h_var_89_11 = (float *)malloc(sizeof(float *));
	float * d_var_89_11;
	hipMalloc((void **)&d_var_89_11, sizeof(float *));
	
	float * h_var_89_12 = (float *)malloc(sizeof(float *));
	float * d_var_89_12;
	hipMalloc((void **)&d_var_89_12, sizeof(float *));
	
	float * h_var_89_13 = (float *)malloc(sizeof(float *));
	float * d_var_89_13;
	hipMalloc((void **)&d_var_89_13, sizeof(float *));
	
	float * h_var_89_14 = (float *)malloc(sizeof(float *));
	float * d_var_89_14;
	hipMalloc((void **)&d_var_89_14, sizeof(float *));
	
	float * h_var_89_15 = (float *)malloc(sizeof(float *));
	float * d_var_89_15;
	hipMalloc((void **)&d_var_89_15, sizeof(float *));
	
	float * h_var_89_16 = (float *)malloc(sizeof(float *));
	float * d_var_89_16;
	hipMalloc((void **)&d_var_89_16, sizeof(float *));
	
	float * h_var_89_17 = (float *)malloc(sizeof(float *));
	float * d_var_89_17;
	hipMalloc((void **)&d_var_89_17, sizeof(float *));
	
	float * h_var_89_18 = (float *)malloc(sizeof(float *));
	float * d_var_89_18;
	hipMalloc((void **)&d_var_89_18, sizeof(float *));
	
	float * h_var_89_19 = (float *)malloc(sizeof(float *));
	float * d_var_89_19;
	hipMalloc((void **)&d_var_89_19, sizeof(float *));
	
	float * h_var_90_0 = (float *)malloc(sizeof(float *));
	float * d_var_90_0;
	hipMalloc((void **)&d_var_90_0, sizeof(float *));
	
	float * h_var_90_1 = (float *)malloc(sizeof(float *));
	float * d_var_90_1;
	hipMalloc((void **)&d_var_90_1, sizeof(float *));
	
	float * h_var_90_2 = (float *)malloc(sizeof(float *));
	float * d_var_90_2;
	hipMalloc((void **)&d_var_90_2, sizeof(float *));
	
	float * h_var_90_3 = (float *)malloc(sizeof(float *));
	float * d_var_90_3;
	hipMalloc((void **)&d_var_90_3, sizeof(float *));
	
	float * h_var_90_4 = (float *)malloc(sizeof(float *));
	float * d_var_90_4;
	hipMalloc((void **)&d_var_90_4, sizeof(float *));
	
	float * h_var_90_5 = (float *)malloc(sizeof(float *));
	float * d_var_90_5;
	hipMalloc((void **)&d_var_90_5, sizeof(float *));
	
	float * h_var_90_6 = (float *)malloc(sizeof(float *));
	float * d_var_90_6;
	hipMalloc((void **)&d_var_90_6, sizeof(float *));
	
	float * h_var_90_7 = (float *)malloc(sizeof(float *));
	float * d_var_90_7;
	hipMalloc((void **)&d_var_90_7, sizeof(float *));
	
	float * h_var_90_8 = (float *)malloc(sizeof(float *));
	float * d_var_90_8;
	hipMalloc((void **)&d_var_90_8, sizeof(float *));
	
	float * h_var_90_9 = (float *)malloc(sizeof(float *));
	float * d_var_90_9;
	hipMalloc((void **)&d_var_90_9, sizeof(float *));
	
	float * h_var_90_10 = (float *)malloc(sizeof(float *));
	float * d_var_90_10;
	hipMalloc((void **)&d_var_90_10, sizeof(float *));
	
	float * h_var_90_11 = (float *)malloc(sizeof(float *));
	float * d_var_90_11;
	hipMalloc((void **)&d_var_90_11, sizeof(float *));
	
	float * h_var_90_12 = (float *)malloc(sizeof(float *));
	float * d_var_90_12;
	hipMalloc((void **)&d_var_90_12, sizeof(float *));
	
	float * h_var_90_13 = (float *)malloc(sizeof(float *));
	float * d_var_90_13;
	hipMalloc((void **)&d_var_90_13, sizeof(float *));
	
	float * h_var_90_14 = (float *)malloc(sizeof(float *));
	float * d_var_90_14;
	hipMalloc((void **)&d_var_90_14, sizeof(float *));
	
	float * h_var_90_15 = (float *)malloc(sizeof(float *));
	float * d_var_90_15;
	hipMalloc((void **)&d_var_90_15, sizeof(float *));
	
	float * h_var_90_16 = (float *)malloc(sizeof(float *));
	float * d_var_90_16;
	hipMalloc((void **)&d_var_90_16, sizeof(float *));
	
	float * h_var_90_17 = (float *)malloc(sizeof(float *));
	float * d_var_90_17;
	hipMalloc((void **)&d_var_90_17, sizeof(float *));
	
	float * h_var_90_18 = (float *)malloc(sizeof(float *));
	float * d_var_90_18;
	hipMalloc((void **)&d_var_90_18, sizeof(float *));
	
	float * h_var_90_19 = (float *)malloc(sizeof(float *));
	float * d_var_90_19;
	hipMalloc((void **)&d_var_90_19, sizeof(float *));
	
	float * h_var_91_0 = (float *)malloc(sizeof(float *));
	float * d_var_91_0;
	hipMalloc((void **)&d_var_91_0, sizeof(float *));
	
	float * h_var_91_1 = (float *)malloc(sizeof(float *));
	float * d_var_91_1;
	hipMalloc((void **)&d_var_91_1, sizeof(float *));
	
	float * h_var_91_2 = (float *)malloc(sizeof(float *));
	float * d_var_91_2;
	hipMalloc((void **)&d_var_91_2, sizeof(float *));
	
	float * h_var_91_3 = (float *)malloc(sizeof(float *));
	float * d_var_91_3;
	hipMalloc((void **)&d_var_91_3, sizeof(float *));
	
	float * h_var_91_4 = (float *)malloc(sizeof(float *));
	float * d_var_91_4;
	hipMalloc((void **)&d_var_91_4, sizeof(float *));
	
	float * h_var_91_5 = (float *)malloc(sizeof(float *));
	float * d_var_91_5;
	hipMalloc((void **)&d_var_91_5, sizeof(float *));
	
	float * h_var_91_6 = (float *)malloc(sizeof(float *));
	float * d_var_91_6;
	hipMalloc((void **)&d_var_91_6, sizeof(float *));
	
	float * h_var_91_7 = (float *)malloc(sizeof(float *));
	float * d_var_91_7;
	hipMalloc((void **)&d_var_91_7, sizeof(float *));
	
	float * h_var_91_8 = (float *)malloc(sizeof(float *));
	float * d_var_91_8;
	hipMalloc((void **)&d_var_91_8, sizeof(float *));
	
	float * h_var_91_9 = (float *)malloc(sizeof(float *));
	float * d_var_91_9;
	hipMalloc((void **)&d_var_91_9, sizeof(float *));
	
	float * h_var_91_10 = (float *)malloc(sizeof(float *));
	float * d_var_91_10;
	hipMalloc((void **)&d_var_91_10, sizeof(float *));
	
	float * h_var_91_11 = (float *)malloc(sizeof(float *));
	float * d_var_91_11;
	hipMalloc((void **)&d_var_91_11, sizeof(float *));
	
	float * h_var_91_12 = (float *)malloc(sizeof(float *));
	float * d_var_91_12;
	hipMalloc((void **)&d_var_91_12, sizeof(float *));
	
	float * h_var_91_13 = (float *)malloc(sizeof(float *));
	float * d_var_91_13;
	hipMalloc((void **)&d_var_91_13, sizeof(float *));
	
	float * h_var_91_14 = (float *)malloc(sizeof(float *));
	float * d_var_91_14;
	hipMalloc((void **)&d_var_91_14, sizeof(float *));
	
	float * h_var_91_15 = (float *)malloc(sizeof(float *));
	float * d_var_91_15;
	hipMalloc((void **)&d_var_91_15, sizeof(float *));
	
	float * h_var_91_16 = (float *)malloc(sizeof(float *));
	float * d_var_91_16;
	hipMalloc((void **)&d_var_91_16, sizeof(float *));
	
	float * h_var_91_17 = (float *)malloc(sizeof(float *));
	float * d_var_91_17;
	hipMalloc((void **)&d_var_91_17, sizeof(float *));
	
	float * h_var_91_18 = (float *)malloc(sizeof(float *));
	float * d_var_91_18;
	hipMalloc((void **)&d_var_91_18, sizeof(float *));
	
	float * h_var_91_19 = (float *)malloc(sizeof(float *));
	float * d_var_91_19;
	hipMalloc((void **)&d_var_91_19, sizeof(float *));
	
	float * h_var_92_0 = (float *)malloc(sizeof(float *));
	float * d_var_92_0;
	hipMalloc((void **)&d_var_92_0, sizeof(float *));
	
	float * h_var_92_1 = (float *)malloc(sizeof(float *));
	float * d_var_92_1;
	hipMalloc((void **)&d_var_92_1, sizeof(float *));
	
	float * h_var_92_2 = (float *)malloc(sizeof(float *));
	float * d_var_92_2;
	hipMalloc((void **)&d_var_92_2, sizeof(float *));
	
	float * h_var_92_3 = (float *)malloc(sizeof(float *));
	float * d_var_92_3;
	hipMalloc((void **)&d_var_92_3, sizeof(float *));
	
	float * h_var_92_4 = (float *)malloc(sizeof(float *));
	float * d_var_92_4;
	hipMalloc((void **)&d_var_92_4, sizeof(float *));
	
	float * h_var_92_5 = (float *)malloc(sizeof(float *));
	float * d_var_92_5;
	hipMalloc((void **)&d_var_92_5, sizeof(float *));
	
	float * h_var_92_6 = (float *)malloc(sizeof(float *));
	float * d_var_92_6;
	hipMalloc((void **)&d_var_92_6, sizeof(float *));
	
	float * h_var_92_7 = (float *)malloc(sizeof(float *));
	float * d_var_92_7;
	hipMalloc((void **)&d_var_92_7, sizeof(float *));
	
	float * h_var_92_8 = (float *)malloc(sizeof(float *));
	float * d_var_92_8;
	hipMalloc((void **)&d_var_92_8, sizeof(float *));
	
	float * h_var_92_9 = (float *)malloc(sizeof(float *));
	float * d_var_92_9;
	hipMalloc((void **)&d_var_92_9, sizeof(float *));
	
	float * h_var_92_10 = (float *)malloc(sizeof(float *));
	float * d_var_92_10;
	hipMalloc((void **)&d_var_92_10, sizeof(float *));
	
	float * h_var_92_11 = (float *)malloc(sizeof(float *));
	float * d_var_92_11;
	hipMalloc((void **)&d_var_92_11, sizeof(float *));
	
	float * h_var_92_12 = (float *)malloc(sizeof(float *));
	float * d_var_92_12;
	hipMalloc((void **)&d_var_92_12, sizeof(float *));
	
	float * h_var_92_13 = (float *)malloc(sizeof(float *));
	float * d_var_92_13;
	hipMalloc((void **)&d_var_92_13, sizeof(float *));
	
	float * h_var_92_14 = (float *)malloc(sizeof(float *));
	float * d_var_92_14;
	hipMalloc((void **)&d_var_92_14, sizeof(float *));
	
	float * h_var_92_15 = (float *)malloc(sizeof(float *));
	float * d_var_92_15;
	hipMalloc((void **)&d_var_92_15, sizeof(float *));
	
	float * h_var_92_16 = (float *)malloc(sizeof(float *));
	float * d_var_92_16;
	hipMalloc((void **)&d_var_92_16, sizeof(float *));
	
	float * h_var_92_17 = (float *)malloc(sizeof(float *));
	float * d_var_92_17;
	hipMalloc((void **)&d_var_92_17, sizeof(float *));
	
	float * h_var_92_18 = (float *)malloc(sizeof(float *));
	float * d_var_92_18;
	hipMalloc((void **)&d_var_92_18, sizeof(float *));
	
	float * h_var_92_19 = (float *)malloc(sizeof(float *));
	float * d_var_92_19;
	hipMalloc((void **)&d_var_92_19, sizeof(float *));
	
	float * h_var_93_0 = (float *)malloc(sizeof(float *));
	float * d_var_93_0;
	hipMalloc((void **)&d_var_93_0, sizeof(float *));
	
	float * h_var_93_1 = (float *)malloc(sizeof(float *));
	float * d_var_93_1;
	hipMalloc((void **)&d_var_93_1, sizeof(float *));
	
	float * h_var_93_2 = (float *)malloc(sizeof(float *));
	float * d_var_93_2;
	hipMalloc((void **)&d_var_93_2, sizeof(float *));
	
	float * h_var_93_3 = (float *)malloc(sizeof(float *));
	float * d_var_93_3;
	hipMalloc((void **)&d_var_93_3, sizeof(float *));
	
	float * h_var_93_4 = (float *)malloc(sizeof(float *));
	float * d_var_93_4;
	hipMalloc((void **)&d_var_93_4, sizeof(float *));
	
	float * h_var_93_5 = (float *)malloc(sizeof(float *));
	float * d_var_93_5;
	hipMalloc((void **)&d_var_93_5, sizeof(float *));
	
	float * h_var_93_6 = (float *)malloc(sizeof(float *));
	float * d_var_93_6;
	hipMalloc((void **)&d_var_93_6, sizeof(float *));
	
	float * h_var_93_7 = (float *)malloc(sizeof(float *));
	float * d_var_93_7;
	hipMalloc((void **)&d_var_93_7, sizeof(float *));
	
	float * h_var_93_8 = (float *)malloc(sizeof(float *));
	float * d_var_93_8;
	hipMalloc((void **)&d_var_93_8, sizeof(float *));
	
	float * h_var_93_9 = (float *)malloc(sizeof(float *));
	float * d_var_93_9;
	hipMalloc((void **)&d_var_93_9, sizeof(float *));
	
	float * h_var_93_10 = (float *)malloc(sizeof(float *));
	float * d_var_93_10;
	hipMalloc((void **)&d_var_93_10, sizeof(float *));
	
	float * h_var_93_11 = (float *)malloc(sizeof(float *));
	float * d_var_93_11;
	hipMalloc((void **)&d_var_93_11, sizeof(float *));
	
	float * h_var_93_12 = (float *)malloc(sizeof(float *));
	float * d_var_93_12;
	hipMalloc((void **)&d_var_93_12, sizeof(float *));
	
	float * h_var_93_13 = (float *)malloc(sizeof(float *));
	float * d_var_93_13;
	hipMalloc((void **)&d_var_93_13, sizeof(float *));
	
	float * h_var_93_14 = (float *)malloc(sizeof(float *));
	float * d_var_93_14;
	hipMalloc((void **)&d_var_93_14, sizeof(float *));
	
	float * h_var_93_15 = (float *)malloc(sizeof(float *));
	float * d_var_93_15;
	hipMalloc((void **)&d_var_93_15, sizeof(float *));
	
	float * h_var_93_16 = (float *)malloc(sizeof(float *));
	float * d_var_93_16;
	hipMalloc((void **)&d_var_93_16, sizeof(float *));
	
	float * h_var_93_17 = (float *)malloc(sizeof(float *));
	float * d_var_93_17;
	hipMalloc((void **)&d_var_93_17, sizeof(float *));
	
	float * h_var_93_18 = (float *)malloc(sizeof(float *));
	float * d_var_93_18;
	hipMalloc((void **)&d_var_93_18, sizeof(float *));
	
	float * h_var_93_19 = (float *)malloc(sizeof(float *));
	float * d_var_93_19;
	hipMalloc((void **)&d_var_93_19, sizeof(float *));
	
	float * h_var_94_0 = (float *)malloc(sizeof(float *));
	float * d_var_94_0;
	hipMalloc((void **)&d_var_94_0, sizeof(float *));
	
	float * h_var_94_1 = (float *)malloc(sizeof(float *));
	float * d_var_94_1;
	hipMalloc((void **)&d_var_94_1, sizeof(float *));
	
	float * h_var_94_2 = (float *)malloc(sizeof(float *));
	float * d_var_94_2;
	hipMalloc((void **)&d_var_94_2, sizeof(float *));
	
	float * h_var_94_3 = (float *)malloc(sizeof(float *));
	float * d_var_94_3;
	hipMalloc((void **)&d_var_94_3, sizeof(float *));
	
	float * h_var_94_4 = (float *)malloc(sizeof(float *));
	float * d_var_94_4;
	hipMalloc((void **)&d_var_94_4, sizeof(float *));
	
	float * h_var_94_5 = (float *)malloc(sizeof(float *));
	float * d_var_94_5;
	hipMalloc((void **)&d_var_94_5, sizeof(float *));
	
	float * h_var_94_6 = (float *)malloc(sizeof(float *));
	float * d_var_94_6;
	hipMalloc((void **)&d_var_94_6, sizeof(float *));
	
	float * h_var_94_7 = (float *)malloc(sizeof(float *));
	float * d_var_94_7;
	hipMalloc((void **)&d_var_94_7, sizeof(float *));
	
	float * h_var_94_8 = (float *)malloc(sizeof(float *));
	float * d_var_94_8;
	hipMalloc((void **)&d_var_94_8, sizeof(float *));
	
	float * h_var_94_9 = (float *)malloc(sizeof(float *));
	float * d_var_94_9;
	hipMalloc((void **)&d_var_94_9, sizeof(float *));
	
	float * h_var_94_10 = (float *)malloc(sizeof(float *));
	float * d_var_94_10;
	hipMalloc((void **)&d_var_94_10, sizeof(float *));
	
	float * h_var_94_11 = (float *)malloc(sizeof(float *));
	float * d_var_94_11;
	hipMalloc((void **)&d_var_94_11, sizeof(float *));
	
	float * h_var_94_12 = (float *)malloc(sizeof(float *));
	float * d_var_94_12;
	hipMalloc((void **)&d_var_94_12, sizeof(float *));
	
	float * h_var_94_13 = (float *)malloc(sizeof(float *));
	float * d_var_94_13;
	hipMalloc((void **)&d_var_94_13, sizeof(float *));
	
	float * h_var_94_14 = (float *)malloc(sizeof(float *));
	float * d_var_94_14;
	hipMalloc((void **)&d_var_94_14, sizeof(float *));
	
	float * h_var_94_15 = (float *)malloc(sizeof(float *));
	float * d_var_94_15;
	hipMalloc((void **)&d_var_94_15, sizeof(float *));
	
	float * h_var_94_16 = (float *)malloc(sizeof(float *));
	float * d_var_94_16;
	hipMalloc((void **)&d_var_94_16, sizeof(float *));
	
	float * h_var_94_17 = (float *)malloc(sizeof(float *));
	float * d_var_94_17;
	hipMalloc((void **)&d_var_94_17, sizeof(float *));
	
	float * h_var_94_18 = (float *)malloc(sizeof(float *));
	float * d_var_94_18;
	hipMalloc((void **)&d_var_94_18, sizeof(float *));
	
	float * h_var_94_19 = (float *)malloc(sizeof(float *));
	float * d_var_94_19;
	hipMalloc((void **)&d_var_94_19, sizeof(float *));
	
	float * h_var_95_0 = (float *)malloc(sizeof(float *));
	float * d_var_95_0;
	hipMalloc((void **)&d_var_95_0, sizeof(float *));
	
	float * h_var_95_1 = (float *)malloc(sizeof(float *));
	float * d_var_95_1;
	hipMalloc((void **)&d_var_95_1, sizeof(float *));
	
	float * h_var_95_2 = (float *)malloc(sizeof(float *));
	float * d_var_95_2;
	hipMalloc((void **)&d_var_95_2, sizeof(float *));
	
	float * h_var_95_3 = (float *)malloc(sizeof(float *));
	float * d_var_95_3;
	hipMalloc((void **)&d_var_95_3, sizeof(float *));
	
	float * h_var_95_4 = (float *)malloc(sizeof(float *));
	float * d_var_95_4;
	hipMalloc((void **)&d_var_95_4, sizeof(float *));
	
	float * h_var_95_5 = (float *)malloc(sizeof(float *));
	float * d_var_95_5;
	hipMalloc((void **)&d_var_95_5, sizeof(float *));
	
	float * h_var_95_6 = (float *)malloc(sizeof(float *));
	float * d_var_95_6;
	hipMalloc((void **)&d_var_95_6, sizeof(float *));
	
	float * h_var_95_7 = (float *)malloc(sizeof(float *));
	float * d_var_95_7;
	hipMalloc((void **)&d_var_95_7, sizeof(float *));
	
	float * h_var_95_8 = (float *)malloc(sizeof(float *));
	float * d_var_95_8;
	hipMalloc((void **)&d_var_95_8, sizeof(float *));
	
	float * h_var_95_9 = (float *)malloc(sizeof(float *));
	float * d_var_95_9;
	hipMalloc((void **)&d_var_95_9, sizeof(float *));
	
	float * h_var_95_10 = (float *)malloc(sizeof(float *));
	float * d_var_95_10;
	hipMalloc((void **)&d_var_95_10, sizeof(float *));
	
	float * h_var_95_11 = (float *)malloc(sizeof(float *));
	float * d_var_95_11;
	hipMalloc((void **)&d_var_95_11, sizeof(float *));
	
	float * h_var_95_12 = (float *)malloc(sizeof(float *));
	float * d_var_95_12;
	hipMalloc((void **)&d_var_95_12, sizeof(float *));
	
	float * h_var_95_13 = (float *)malloc(sizeof(float *));
	float * d_var_95_13;
	hipMalloc((void **)&d_var_95_13, sizeof(float *));
	
	float * h_var_95_14 = (float *)malloc(sizeof(float *));
	float * d_var_95_14;
	hipMalloc((void **)&d_var_95_14, sizeof(float *));
	
	float * h_var_95_15 = (float *)malloc(sizeof(float *));
	float * d_var_95_15;
	hipMalloc((void **)&d_var_95_15, sizeof(float *));
	
	float * h_var_95_16 = (float *)malloc(sizeof(float *));
	float * d_var_95_16;
	hipMalloc((void **)&d_var_95_16, sizeof(float *));
	
	float * h_var_95_17 = (float *)malloc(sizeof(float *));
	float * d_var_95_17;
	hipMalloc((void **)&d_var_95_17, sizeof(float *));
	
	float * h_var_95_18 = (float *)malloc(sizeof(float *));
	float * d_var_95_18;
	hipMalloc((void **)&d_var_95_18, sizeof(float *));
	
	float * h_var_95_19 = (float *)malloc(sizeof(float *));
	float * d_var_95_19;
	hipMalloc((void **)&d_var_95_19, sizeof(float *));
	
	float * h_var_96_0 = (float *)malloc(sizeof(float *));
	float * d_var_96_0;
	hipMalloc((void **)&d_var_96_0, sizeof(float *));
	
	float * h_var_96_1 = (float *)malloc(sizeof(float *));
	float * d_var_96_1;
	hipMalloc((void **)&d_var_96_1, sizeof(float *));
	
	float * h_var_96_2 = (float *)malloc(sizeof(float *));
	float * d_var_96_2;
	hipMalloc((void **)&d_var_96_2, sizeof(float *));
	
	float * h_var_96_3 = (float *)malloc(sizeof(float *));
	float * d_var_96_3;
	hipMalloc((void **)&d_var_96_3, sizeof(float *));
	
	float * h_var_96_4 = (float *)malloc(sizeof(float *));
	float * d_var_96_4;
	hipMalloc((void **)&d_var_96_4, sizeof(float *));
	
	float * h_var_96_5 = (float *)malloc(sizeof(float *));
	float * d_var_96_5;
	hipMalloc((void **)&d_var_96_5, sizeof(float *));
	
	float * h_var_96_6 = (float *)malloc(sizeof(float *));
	float * d_var_96_6;
	hipMalloc((void **)&d_var_96_6, sizeof(float *));
	
	float * h_var_96_7 = (float *)malloc(sizeof(float *));
	float * d_var_96_7;
	hipMalloc((void **)&d_var_96_7, sizeof(float *));
	
	float * h_var_96_8 = (float *)malloc(sizeof(float *));
	float * d_var_96_8;
	hipMalloc((void **)&d_var_96_8, sizeof(float *));
	
	float * h_var_96_9 = (float *)malloc(sizeof(float *));
	float * d_var_96_9;
	hipMalloc((void **)&d_var_96_9, sizeof(float *));
	
	float * h_var_96_10 = (float *)malloc(sizeof(float *));
	float * d_var_96_10;
	hipMalloc((void **)&d_var_96_10, sizeof(float *));
	
	float * h_var_96_11 = (float *)malloc(sizeof(float *));
	float * d_var_96_11;
	hipMalloc((void **)&d_var_96_11, sizeof(float *));
	
	float * h_var_96_12 = (float *)malloc(sizeof(float *));
	float * d_var_96_12;
	hipMalloc((void **)&d_var_96_12, sizeof(float *));
	
	float * h_var_96_13 = (float *)malloc(sizeof(float *));
	float * d_var_96_13;
	hipMalloc((void **)&d_var_96_13, sizeof(float *));
	
	float * h_var_96_14 = (float *)malloc(sizeof(float *));
	float * d_var_96_14;
	hipMalloc((void **)&d_var_96_14, sizeof(float *));
	
	float * h_var_96_15 = (float *)malloc(sizeof(float *));
	float * d_var_96_15;
	hipMalloc((void **)&d_var_96_15, sizeof(float *));
	
	float * h_var_96_16 = (float *)malloc(sizeof(float *));
	float * d_var_96_16;
	hipMalloc((void **)&d_var_96_16, sizeof(float *));
	
	float * h_var_96_17 = (float *)malloc(sizeof(float *));
	float * d_var_96_17;
	hipMalloc((void **)&d_var_96_17, sizeof(float *));
	
	float * h_var_96_18 = (float *)malloc(sizeof(float *));
	float * d_var_96_18;
	hipMalloc((void **)&d_var_96_18, sizeof(float *));
	
	float * h_var_96_19 = (float *)malloc(sizeof(float *));
	float * d_var_96_19;
	hipMalloc((void **)&d_var_96_19, sizeof(float *));
	
	float * h_var_97_0 = (float *)malloc(sizeof(float *));
	float * d_var_97_0;
	hipMalloc((void **)&d_var_97_0, sizeof(float *));
	
	float * h_var_97_1 = (float *)malloc(sizeof(float *));
	float * d_var_97_1;
	hipMalloc((void **)&d_var_97_1, sizeof(float *));
	
	float * h_var_97_2 = (float *)malloc(sizeof(float *));
	float * d_var_97_2;
	hipMalloc((void **)&d_var_97_2, sizeof(float *));
	
	float * h_var_97_3 = (float *)malloc(sizeof(float *));
	float * d_var_97_3;
	hipMalloc((void **)&d_var_97_3, sizeof(float *));
	
	float * h_var_97_4 = (float *)malloc(sizeof(float *));
	float * d_var_97_4;
	hipMalloc((void **)&d_var_97_4, sizeof(float *));
	
	float * h_var_97_5 = (float *)malloc(sizeof(float *));
	float * d_var_97_5;
	hipMalloc((void **)&d_var_97_5, sizeof(float *));
	
	float * h_var_97_6 = (float *)malloc(sizeof(float *));
	float * d_var_97_6;
	hipMalloc((void **)&d_var_97_6, sizeof(float *));
	
	float * h_var_97_7 = (float *)malloc(sizeof(float *));
	float * d_var_97_7;
	hipMalloc((void **)&d_var_97_7, sizeof(float *));
	
	float * h_var_97_8 = (float *)malloc(sizeof(float *));
	float * d_var_97_8;
	hipMalloc((void **)&d_var_97_8, sizeof(float *));
	
	float * h_var_97_9 = (float *)malloc(sizeof(float *));
	float * d_var_97_9;
	hipMalloc((void **)&d_var_97_9, sizeof(float *));
	
	float * h_var_97_10 = (float *)malloc(sizeof(float *));
	float * d_var_97_10;
	hipMalloc((void **)&d_var_97_10, sizeof(float *));
	
	float * h_var_97_11 = (float *)malloc(sizeof(float *));
	float * d_var_97_11;
	hipMalloc((void **)&d_var_97_11, sizeof(float *));
	
	float * h_var_97_12 = (float *)malloc(sizeof(float *));
	float * d_var_97_12;
	hipMalloc((void **)&d_var_97_12, sizeof(float *));
	
	float * h_var_97_13 = (float *)malloc(sizeof(float *));
	float * d_var_97_13;
	hipMalloc((void **)&d_var_97_13, sizeof(float *));
	
	float * h_var_97_14 = (float *)malloc(sizeof(float *));
	float * d_var_97_14;
	hipMalloc((void **)&d_var_97_14, sizeof(float *));
	
	float * h_var_97_15 = (float *)malloc(sizeof(float *));
	float * d_var_97_15;
	hipMalloc((void **)&d_var_97_15, sizeof(float *));
	
	float * h_var_97_16 = (float *)malloc(sizeof(float *));
	float * d_var_97_16;
	hipMalloc((void **)&d_var_97_16, sizeof(float *));
	
	float * h_var_97_17 = (float *)malloc(sizeof(float *));
	float * d_var_97_17;
	hipMalloc((void **)&d_var_97_17, sizeof(float *));
	
	float * h_var_97_18 = (float *)malloc(sizeof(float *));
	float * d_var_97_18;
	hipMalloc((void **)&d_var_97_18, sizeof(float *));
	
	float * h_var_97_19 = (float *)malloc(sizeof(float *));
	float * d_var_97_19;
	hipMalloc((void **)&d_var_97_19, sizeof(float *));
	
	float * h_var_98_0 = (float *)malloc(sizeof(float *));
	float * d_var_98_0;
	hipMalloc((void **)&d_var_98_0, sizeof(float *));
	
	float * h_var_98_1 = (float *)malloc(sizeof(float *));
	float * d_var_98_1;
	hipMalloc((void **)&d_var_98_1, sizeof(float *));
	
	float * h_var_98_2 = (float *)malloc(sizeof(float *));
	float * d_var_98_2;
	hipMalloc((void **)&d_var_98_2, sizeof(float *));
	
	float * h_var_98_3 = (float *)malloc(sizeof(float *));
	float * d_var_98_3;
	hipMalloc((void **)&d_var_98_3, sizeof(float *));
	
	float * h_var_98_4 = (float *)malloc(sizeof(float *));
	float * d_var_98_4;
	hipMalloc((void **)&d_var_98_4, sizeof(float *));
	
	float * h_var_98_5 = (float *)malloc(sizeof(float *));
	float * d_var_98_5;
	hipMalloc((void **)&d_var_98_5, sizeof(float *));
	
	float * h_var_98_6 = (float *)malloc(sizeof(float *));
	float * d_var_98_6;
	hipMalloc((void **)&d_var_98_6, sizeof(float *));
	
	float * h_var_98_7 = (float *)malloc(sizeof(float *));
	float * d_var_98_7;
	hipMalloc((void **)&d_var_98_7, sizeof(float *));
	
	float * h_var_98_8 = (float *)malloc(sizeof(float *));
	float * d_var_98_8;
	hipMalloc((void **)&d_var_98_8, sizeof(float *));
	
	float * h_var_98_9 = (float *)malloc(sizeof(float *));
	float * d_var_98_9;
	hipMalloc((void **)&d_var_98_9, sizeof(float *));
	
	float * h_var_98_10 = (float *)malloc(sizeof(float *));
	float * d_var_98_10;
	hipMalloc((void **)&d_var_98_10, sizeof(float *));
	
	float * h_var_98_11 = (float *)malloc(sizeof(float *));
	float * d_var_98_11;
	hipMalloc((void **)&d_var_98_11, sizeof(float *));
	
	float * h_var_98_12 = (float *)malloc(sizeof(float *));
	float * d_var_98_12;
	hipMalloc((void **)&d_var_98_12, sizeof(float *));
	
	float * h_var_98_13 = (float *)malloc(sizeof(float *));
	float * d_var_98_13;
	hipMalloc((void **)&d_var_98_13, sizeof(float *));
	
	float * h_var_98_14 = (float *)malloc(sizeof(float *));
	float * d_var_98_14;
	hipMalloc((void **)&d_var_98_14, sizeof(float *));
	
	float * h_var_98_15 = (float *)malloc(sizeof(float *));
	float * d_var_98_15;
	hipMalloc((void **)&d_var_98_15, sizeof(float *));
	
	float * h_var_98_16 = (float *)malloc(sizeof(float *));
	float * d_var_98_16;
	hipMalloc((void **)&d_var_98_16, sizeof(float *));
	
	float * h_var_98_17 = (float *)malloc(sizeof(float *));
	float * d_var_98_17;
	hipMalloc((void **)&d_var_98_17, sizeof(float *));
	
	float * h_var_98_18 = (float *)malloc(sizeof(float *));
	float * d_var_98_18;
	hipMalloc((void **)&d_var_98_18, sizeof(float *));
	
	float * h_var_98_19 = (float *)malloc(sizeof(float *));
	float * d_var_98_19;
	hipMalloc((void **)&d_var_98_19, sizeof(float *));
	
	float * h_var_99_0 = (float *)malloc(sizeof(float *));
	float * d_var_99_0;
	hipMalloc((void **)&d_var_99_0, sizeof(float *));
	
	float * h_var_99_1 = (float *)malloc(sizeof(float *));
	float * d_var_99_1;
	hipMalloc((void **)&d_var_99_1, sizeof(float *));
	
	float * h_var_99_2 = (float *)malloc(sizeof(float *));
	float * d_var_99_2;
	hipMalloc((void **)&d_var_99_2, sizeof(float *));
	
	float * h_var_99_3 = (float *)malloc(sizeof(float *));
	float * d_var_99_3;
	hipMalloc((void **)&d_var_99_3, sizeof(float *));
	
	float * h_var_99_4 = (float *)malloc(sizeof(float *));
	float * d_var_99_4;
	hipMalloc((void **)&d_var_99_4, sizeof(float *));
	
	float * h_var_99_5 = (float *)malloc(sizeof(float *));
	float * d_var_99_5;
	hipMalloc((void **)&d_var_99_5, sizeof(float *));
	
	float * h_var_99_6 = (float *)malloc(sizeof(float *));
	float * d_var_99_6;
	hipMalloc((void **)&d_var_99_6, sizeof(float *));
	
	float * h_var_99_7 = (float *)malloc(sizeof(float *));
	float * d_var_99_7;
	hipMalloc((void **)&d_var_99_7, sizeof(float *));
	
	float * h_var_99_8 = (float *)malloc(sizeof(float *));
	float * d_var_99_8;
	hipMalloc((void **)&d_var_99_8, sizeof(float *));
	
	float * h_var_99_9 = (float *)malloc(sizeof(float *));
	float * d_var_99_9;
	hipMalloc((void **)&d_var_99_9, sizeof(float *));
	
	float * h_var_99_10 = (float *)malloc(sizeof(float *));
	float * d_var_99_10;
	hipMalloc((void **)&d_var_99_10, sizeof(float *));
	
	float * h_var_99_11 = (float *)malloc(sizeof(float *));
	float * d_var_99_11;
	hipMalloc((void **)&d_var_99_11, sizeof(float *));
	
	float * h_var_99_12 = (float *)malloc(sizeof(float *));
	float * d_var_99_12;
	hipMalloc((void **)&d_var_99_12, sizeof(float *));
	
	float * h_var_99_13 = (float *)malloc(sizeof(float *));
	float * d_var_99_13;
	hipMalloc((void **)&d_var_99_13, sizeof(float *));
	
	float * h_var_99_14 = (float *)malloc(sizeof(float *));
	float * d_var_99_14;
	hipMalloc((void **)&d_var_99_14, sizeof(float *));
	
	float * h_var_99_15 = (float *)malloc(sizeof(float *));
	float * d_var_99_15;
	hipMalloc((void **)&d_var_99_15, sizeof(float *));
	
	float * h_var_99_16 = (float *)malloc(sizeof(float *));
	float * d_var_99_16;
	hipMalloc((void **)&d_var_99_16, sizeof(float *));
	
	float * h_var_99_17 = (float *)malloc(sizeof(float *));
	float * d_var_99_17;
	hipMalloc((void **)&d_var_99_17, sizeof(float *));
	
	float * h_var_99_18 = (float *)malloc(sizeof(float *));
	float * d_var_99_18;
	hipMalloc((void **)&d_var_99_18, sizeof(float *));
	
	float * h_var_99_19 = (float *)malloc(sizeof(float *));
	float * d_var_99_19;
	hipMalloc((void **)&d_var_99_19, sizeof(float *));
	

    // clang-format off
	
	kernel_0<<<10, 10>>>(d_var_0_0, d_var_0_1, d_var_0_2, d_var_0_3, d_var_0_4, d_var_0_5, d_var_0_6, d_var_0_7, d_var_0_8, d_var_0_9, d_var_0_10, d_var_0_11, d_var_0_12, d_var_0_13, d_var_0_14, d_var_0_15, d_var_0_16, d_var_0_17, d_var_0_18, d_var_0_19);
	
	kernel_1<<<10, 10>>>(d_var_1_0, d_var_1_1, d_var_1_2, d_var_1_3, d_var_1_4, d_var_1_5, d_var_1_6, d_var_1_7, d_var_1_8, d_var_1_9, d_var_1_10, d_var_1_11, d_var_1_12, d_var_1_13, d_var_1_14, d_var_1_15, d_var_1_16, d_var_1_17, d_var_1_18, d_var_1_19);
	
	kernel_2<<<10, 10>>>(d_var_2_0, d_var_2_1, d_var_2_2, d_var_2_3, d_var_2_4, d_var_2_5, d_var_2_6, d_var_2_7, d_var_2_8, d_var_2_9, d_var_2_10, d_var_2_11, d_var_2_12, d_var_2_13, d_var_2_14, d_var_2_15, d_var_2_16, d_var_2_17, d_var_2_18, d_var_2_19);
	
	kernel_3<<<10, 10>>>(d_var_3_0, d_var_3_1, d_var_3_2, d_var_3_3, d_var_3_4, d_var_3_5, d_var_3_6, d_var_3_7, d_var_3_8, d_var_3_9, d_var_3_10, d_var_3_11, d_var_3_12, d_var_3_13, d_var_3_14, d_var_3_15, d_var_3_16, d_var_3_17, d_var_3_18, d_var_3_19);
	
	kernel_4<<<10, 10>>>(d_var_4_0, d_var_4_1, d_var_4_2, d_var_4_3, d_var_4_4, d_var_4_5, d_var_4_6, d_var_4_7, d_var_4_8, d_var_4_9, d_var_4_10, d_var_4_11, d_var_4_12, d_var_4_13, d_var_4_14, d_var_4_15, d_var_4_16, d_var_4_17, d_var_4_18, d_var_4_19);
	
	kernel_5<<<10, 10>>>(d_var_5_0, d_var_5_1, d_var_5_2, d_var_5_3, d_var_5_4, d_var_5_5, d_var_5_6, d_var_5_7, d_var_5_8, d_var_5_9, d_var_5_10, d_var_5_11, d_var_5_12, d_var_5_13, d_var_5_14, d_var_5_15, d_var_5_16, d_var_5_17, d_var_5_18, d_var_5_19);
	
	kernel_6<<<10, 10>>>(d_var_6_0, d_var_6_1, d_var_6_2, d_var_6_3, d_var_6_4, d_var_6_5, d_var_6_6, d_var_6_7, d_var_6_8, d_var_6_9, d_var_6_10, d_var_6_11, d_var_6_12, d_var_6_13, d_var_6_14, d_var_6_15, d_var_6_16, d_var_6_17, d_var_6_18, d_var_6_19);
	
	kernel_7<<<10, 10>>>(d_var_7_0, d_var_7_1, d_var_7_2, d_var_7_3, d_var_7_4, d_var_7_5, d_var_7_6, d_var_7_7, d_var_7_8, d_var_7_9, d_var_7_10, d_var_7_11, d_var_7_12, d_var_7_13, d_var_7_14, d_var_7_15, d_var_7_16, d_var_7_17, d_var_7_18, d_var_7_19);
	
	kernel_8<<<10, 10>>>(d_var_8_0, d_var_8_1, d_var_8_2, d_var_8_3, d_var_8_4, d_var_8_5, d_var_8_6, d_var_8_7, d_var_8_8, d_var_8_9, d_var_8_10, d_var_8_11, d_var_8_12, d_var_8_13, d_var_8_14, d_var_8_15, d_var_8_16, d_var_8_17, d_var_8_18, d_var_8_19);
	
	kernel_9<<<10, 10>>>(d_var_9_0, d_var_9_1, d_var_9_2, d_var_9_3, d_var_9_4, d_var_9_5, d_var_9_6, d_var_9_7, d_var_9_8, d_var_9_9, d_var_9_10, d_var_9_11, d_var_9_12, d_var_9_13, d_var_9_14, d_var_9_15, d_var_9_16, d_var_9_17, d_var_9_18, d_var_9_19);
	
	kernel_10<<<10, 10>>>(d_var_10_0, d_var_10_1, d_var_10_2, d_var_10_3, d_var_10_4, d_var_10_5, d_var_10_6, d_var_10_7, d_var_10_8, d_var_10_9, d_var_10_10, d_var_10_11, d_var_10_12, d_var_10_13, d_var_10_14, d_var_10_15, d_var_10_16, d_var_10_17, d_var_10_18, d_var_10_19);
	
	kernel_11<<<10, 10>>>(d_var_11_0, d_var_11_1, d_var_11_2, d_var_11_3, d_var_11_4, d_var_11_5, d_var_11_6, d_var_11_7, d_var_11_8, d_var_11_9, d_var_11_10, d_var_11_11, d_var_11_12, d_var_11_13, d_var_11_14, d_var_11_15, d_var_11_16, d_var_11_17, d_var_11_18, d_var_11_19);
	
	kernel_12<<<10, 10>>>(d_var_12_0, d_var_12_1, d_var_12_2, d_var_12_3, d_var_12_4, d_var_12_5, d_var_12_6, d_var_12_7, d_var_12_8, d_var_12_9, d_var_12_10, d_var_12_11, d_var_12_12, d_var_12_13, d_var_12_14, d_var_12_15, d_var_12_16, d_var_12_17, d_var_12_18, d_var_12_19);
	
	kernel_13<<<10, 10>>>(d_var_13_0, d_var_13_1, d_var_13_2, d_var_13_3, d_var_13_4, d_var_13_5, d_var_13_6, d_var_13_7, d_var_13_8, d_var_13_9, d_var_13_10, d_var_13_11, d_var_13_12, d_var_13_13, d_var_13_14, d_var_13_15, d_var_13_16, d_var_13_17, d_var_13_18, d_var_13_19);
	
	kernel_14<<<10, 10>>>(d_var_14_0, d_var_14_1, d_var_14_2, d_var_14_3, d_var_14_4, d_var_14_5, d_var_14_6, d_var_14_7, d_var_14_8, d_var_14_9, d_var_14_10, d_var_14_11, d_var_14_12, d_var_14_13, d_var_14_14, d_var_14_15, d_var_14_16, d_var_14_17, d_var_14_18, d_var_14_19);
	
	kernel_15<<<10, 10>>>(d_var_15_0, d_var_15_1, d_var_15_2, d_var_15_3, d_var_15_4, d_var_15_5, d_var_15_6, d_var_15_7, d_var_15_8, d_var_15_9, d_var_15_10, d_var_15_11, d_var_15_12, d_var_15_13, d_var_15_14, d_var_15_15, d_var_15_16, d_var_15_17, d_var_15_18, d_var_15_19);
	
	kernel_16<<<10, 10>>>(d_var_16_0, d_var_16_1, d_var_16_2, d_var_16_3, d_var_16_4, d_var_16_5, d_var_16_6, d_var_16_7, d_var_16_8, d_var_16_9, d_var_16_10, d_var_16_11, d_var_16_12, d_var_16_13, d_var_16_14, d_var_16_15, d_var_16_16, d_var_16_17, d_var_16_18, d_var_16_19);
	
	kernel_17<<<10, 10>>>(d_var_17_0, d_var_17_1, d_var_17_2, d_var_17_3, d_var_17_4, d_var_17_5, d_var_17_6, d_var_17_7, d_var_17_8, d_var_17_9, d_var_17_10, d_var_17_11, d_var_17_12, d_var_17_13, d_var_17_14, d_var_17_15, d_var_17_16, d_var_17_17, d_var_17_18, d_var_17_19);
	
	kernel_18<<<10, 10>>>(d_var_18_0, d_var_18_1, d_var_18_2, d_var_18_3, d_var_18_4, d_var_18_5, d_var_18_6, d_var_18_7, d_var_18_8, d_var_18_9, d_var_18_10, d_var_18_11, d_var_18_12, d_var_18_13, d_var_18_14, d_var_18_15, d_var_18_16, d_var_18_17, d_var_18_18, d_var_18_19);
	
	kernel_19<<<10, 10>>>(d_var_19_0, d_var_19_1, d_var_19_2, d_var_19_3, d_var_19_4, d_var_19_5, d_var_19_6, d_var_19_7, d_var_19_8, d_var_19_9, d_var_19_10, d_var_19_11, d_var_19_12, d_var_19_13, d_var_19_14, d_var_19_15, d_var_19_16, d_var_19_17, d_var_19_18, d_var_19_19);
	
	kernel_20<<<10, 10>>>(d_var_20_0, d_var_20_1, d_var_20_2, d_var_20_3, d_var_20_4, d_var_20_5, d_var_20_6, d_var_20_7, d_var_20_8, d_var_20_9, d_var_20_10, d_var_20_11, d_var_20_12, d_var_20_13, d_var_20_14, d_var_20_15, d_var_20_16, d_var_20_17, d_var_20_18, d_var_20_19);
	
	kernel_21<<<10, 10>>>(d_var_21_0, d_var_21_1, d_var_21_2, d_var_21_3, d_var_21_4, d_var_21_5, d_var_21_6, d_var_21_7, d_var_21_8, d_var_21_9, d_var_21_10, d_var_21_11, d_var_21_12, d_var_21_13, d_var_21_14, d_var_21_15, d_var_21_16, d_var_21_17, d_var_21_18, d_var_21_19);
	
	kernel_22<<<10, 10>>>(d_var_22_0, d_var_22_1, d_var_22_2, d_var_22_3, d_var_22_4, d_var_22_5, d_var_22_6, d_var_22_7, d_var_22_8, d_var_22_9, d_var_22_10, d_var_22_11, d_var_22_12, d_var_22_13, d_var_22_14, d_var_22_15, d_var_22_16, d_var_22_17, d_var_22_18, d_var_22_19);
	
	kernel_23<<<10, 10>>>(d_var_23_0, d_var_23_1, d_var_23_2, d_var_23_3, d_var_23_4, d_var_23_5, d_var_23_6, d_var_23_7, d_var_23_8, d_var_23_9, d_var_23_10, d_var_23_11, d_var_23_12, d_var_23_13, d_var_23_14, d_var_23_15, d_var_23_16, d_var_23_17, d_var_23_18, d_var_23_19);
	
	kernel_24<<<10, 10>>>(d_var_24_0, d_var_24_1, d_var_24_2, d_var_24_3, d_var_24_4, d_var_24_5, d_var_24_6, d_var_24_7, d_var_24_8, d_var_24_9, d_var_24_10, d_var_24_11, d_var_24_12, d_var_24_13, d_var_24_14, d_var_24_15, d_var_24_16, d_var_24_17, d_var_24_18, d_var_24_19);
	
	kernel_25<<<10, 10>>>(d_var_25_0, d_var_25_1, d_var_25_2, d_var_25_3, d_var_25_4, d_var_25_5, d_var_25_6, d_var_25_7, d_var_25_8, d_var_25_9, d_var_25_10, d_var_25_11, d_var_25_12, d_var_25_13, d_var_25_14, d_var_25_15, d_var_25_16, d_var_25_17, d_var_25_18, d_var_25_19);
	
	kernel_26<<<10, 10>>>(d_var_26_0, d_var_26_1, d_var_26_2, d_var_26_3, d_var_26_4, d_var_26_5, d_var_26_6, d_var_26_7, d_var_26_8, d_var_26_9, d_var_26_10, d_var_26_11, d_var_26_12, d_var_26_13, d_var_26_14, d_var_26_15, d_var_26_16, d_var_26_17, d_var_26_18, d_var_26_19);
	
	kernel_27<<<10, 10>>>(d_var_27_0, d_var_27_1, d_var_27_2, d_var_27_3, d_var_27_4, d_var_27_5, d_var_27_6, d_var_27_7, d_var_27_8, d_var_27_9, d_var_27_10, d_var_27_11, d_var_27_12, d_var_27_13, d_var_27_14, d_var_27_15, d_var_27_16, d_var_27_17, d_var_27_18, d_var_27_19);
	
	kernel_28<<<10, 10>>>(d_var_28_0, d_var_28_1, d_var_28_2, d_var_28_3, d_var_28_4, d_var_28_5, d_var_28_6, d_var_28_7, d_var_28_8, d_var_28_9, d_var_28_10, d_var_28_11, d_var_28_12, d_var_28_13, d_var_28_14, d_var_28_15, d_var_28_16, d_var_28_17, d_var_28_18, d_var_28_19);
	
	kernel_29<<<10, 10>>>(d_var_29_0, d_var_29_1, d_var_29_2, d_var_29_3, d_var_29_4, d_var_29_5, d_var_29_6, d_var_29_7, d_var_29_8, d_var_29_9, d_var_29_10, d_var_29_11, d_var_29_12, d_var_29_13, d_var_29_14, d_var_29_15, d_var_29_16, d_var_29_17, d_var_29_18, d_var_29_19);
	
	kernel_30<<<10, 10>>>(d_var_30_0, d_var_30_1, d_var_30_2, d_var_30_3, d_var_30_4, d_var_30_5, d_var_30_6, d_var_30_7, d_var_30_8, d_var_30_9, d_var_30_10, d_var_30_11, d_var_30_12, d_var_30_13, d_var_30_14, d_var_30_15, d_var_30_16, d_var_30_17, d_var_30_18, d_var_30_19);
	
	kernel_31<<<10, 10>>>(d_var_31_0, d_var_31_1, d_var_31_2, d_var_31_3, d_var_31_4, d_var_31_5, d_var_31_6, d_var_31_7, d_var_31_8, d_var_31_9, d_var_31_10, d_var_31_11, d_var_31_12, d_var_31_13, d_var_31_14, d_var_31_15, d_var_31_16, d_var_31_17, d_var_31_18, d_var_31_19);
	
	kernel_32<<<10, 10>>>(d_var_32_0, d_var_32_1, d_var_32_2, d_var_32_3, d_var_32_4, d_var_32_5, d_var_32_6, d_var_32_7, d_var_32_8, d_var_32_9, d_var_32_10, d_var_32_11, d_var_32_12, d_var_32_13, d_var_32_14, d_var_32_15, d_var_32_16, d_var_32_17, d_var_32_18, d_var_32_19);
	
	kernel_33<<<10, 10>>>(d_var_33_0, d_var_33_1, d_var_33_2, d_var_33_3, d_var_33_4, d_var_33_5, d_var_33_6, d_var_33_7, d_var_33_8, d_var_33_9, d_var_33_10, d_var_33_11, d_var_33_12, d_var_33_13, d_var_33_14, d_var_33_15, d_var_33_16, d_var_33_17, d_var_33_18, d_var_33_19);
	
	kernel_34<<<10, 10>>>(d_var_34_0, d_var_34_1, d_var_34_2, d_var_34_3, d_var_34_4, d_var_34_5, d_var_34_6, d_var_34_7, d_var_34_8, d_var_34_9, d_var_34_10, d_var_34_11, d_var_34_12, d_var_34_13, d_var_34_14, d_var_34_15, d_var_34_16, d_var_34_17, d_var_34_18, d_var_34_19);
	
	kernel_35<<<10, 10>>>(d_var_35_0, d_var_35_1, d_var_35_2, d_var_35_3, d_var_35_4, d_var_35_5, d_var_35_6, d_var_35_7, d_var_35_8, d_var_35_9, d_var_35_10, d_var_35_11, d_var_35_12, d_var_35_13, d_var_35_14, d_var_35_15, d_var_35_16, d_var_35_17, d_var_35_18, d_var_35_19);
	
	kernel_36<<<10, 10>>>(d_var_36_0, d_var_36_1, d_var_36_2, d_var_36_3, d_var_36_4, d_var_36_5, d_var_36_6, d_var_36_7, d_var_36_8, d_var_36_9, d_var_36_10, d_var_36_11, d_var_36_12, d_var_36_13, d_var_36_14, d_var_36_15, d_var_36_16, d_var_36_17, d_var_36_18, d_var_36_19);
	
	kernel_37<<<10, 10>>>(d_var_37_0, d_var_37_1, d_var_37_2, d_var_37_3, d_var_37_4, d_var_37_5, d_var_37_6, d_var_37_7, d_var_37_8, d_var_37_9, d_var_37_10, d_var_37_11, d_var_37_12, d_var_37_13, d_var_37_14, d_var_37_15, d_var_37_16, d_var_37_17, d_var_37_18, d_var_37_19);
	
	kernel_38<<<10, 10>>>(d_var_38_0, d_var_38_1, d_var_38_2, d_var_38_3, d_var_38_4, d_var_38_5, d_var_38_6, d_var_38_7, d_var_38_8, d_var_38_9, d_var_38_10, d_var_38_11, d_var_38_12, d_var_38_13, d_var_38_14, d_var_38_15, d_var_38_16, d_var_38_17, d_var_38_18, d_var_38_19);
	
	kernel_39<<<10, 10>>>(d_var_39_0, d_var_39_1, d_var_39_2, d_var_39_3, d_var_39_4, d_var_39_5, d_var_39_6, d_var_39_7, d_var_39_8, d_var_39_9, d_var_39_10, d_var_39_11, d_var_39_12, d_var_39_13, d_var_39_14, d_var_39_15, d_var_39_16, d_var_39_17, d_var_39_18, d_var_39_19);
	
	kernel_40<<<10, 10>>>(d_var_40_0, d_var_40_1, d_var_40_2, d_var_40_3, d_var_40_4, d_var_40_5, d_var_40_6, d_var_40_7, d_var_40_8, d_var_40_9, d_var_40_10, d_var_40_11, d_var_40_12, d_var_40_13, d_var_40_14, d_var_40_15, d_var_40_16, d_var_40_17, d_var_40_18, d_var_40_19);
	
	kernel_41<<<10, 10>>>(d_var_41_0, d_var_41_1, d_var_41_2, d_var_41_3, d_var_41_4, d_var_41_5, d_var_41_6, d_var_41_7, d_var_41_8, d_var_41_9, d_var_41_10, d_var_41_11, d_var_41_12, d_var_41_13, d_var_41_14, d_var_41_15, d_var_41_16, d_var_41_17, d_var_41_18, d_var_41_19);
	
	kernel_42<<<10, 10>>>(d_var_42_0, d_var_42_1, d_var_42_2, d_var_42_3, d_var_42_4, d_var_42_5, d_var_42_6, d_var_42_7, d_var_42_8, d_var_42_9, d_var_42_10, d_var_42_11, d_var_42_12, d_var_42_13, d_var_42_14, d_var_42_15, d_var_42_16, d_var_42_17, d_var_42_18, d_var_42_19);
	
	kernel_43<<<10, 10>>>(d_var_43_0, d_var_43_1, d_var_43_2, d_var_43_3, d_var_43_4, d_var_43_5, d_var_43_6, d_var_43_7, d_var_43_8, d_var_43_9, d_var_43_10, d_var_43_11, d_var_43_12, d_var_43_13, d_var_43_14, d_var_43_15, d_var_43_16, d_var_43_17, d_var_43_18, d_var_43_19);
	
	kernel_44<<<10, 10>>>(d_var_44_0, d_var_44_1, d_var_44_2, d_var_44_3, d_var_44_4, d_var_44_5, d_var_44_6, d_var_44_7, d_var_44_8, d_var_44_9, d_var_44_10, d_var_44_11, d_var_44_12, d_var_44_13, d_var_44_14, d_var_44_15, d_var_44_16, d_var_44_17, d_var_44_18, d_var_44_19);
	
	kernel_45<<<10, 10>>>(d_var_45_0, d_var_45_1, d_var_45_2, d_var_45_3, d_var_45_4, d_var_45_5, d_var_45_6, d_var_45_7, d_var_45_8, d_var_45_9, d_var_45_10, d_var_45_11, d_var_45_12, d_var_45_13, d_var_45_14, d_var_45_15, d_var_45_16, d_var_45_17, d_var_45_18, d_var_45_19);
	
	kernel_46<<<10, 10>>>(d_var_46_0, d_var_46_1, d_var_46_2, d_var_46_3, d_var_46_4, d_var_46_5, d_var_46_6, d_var_46_7, d_var_46_8, d_var_46_9, d_var_46_10, d_var_46_11, d_var_46_12, d_var_46_13, d_var_46_14, d_var_46_15, d_var_46_16, d_var_46_17, d_var_46_18, d_var_46_19);
	
	kernel_47<<<10, 10>>>(d_var_47_0, d_var_47_1, d_var_47_2, d_var_47_3, d_var_47_4, d_var_47_5, d_var_47_6, d_var_47_7, d_var_47_8, d_var_47_9, d_var_47_10, d_var_47_11, d_var_47_12, d_var_47_13, d_var_47_14, d_var_47_15, d_var_47_16, d_var_47_17, d_var_47_18, d_var_47_19);
	
	kernel_48<<<10, 10>>>(d_var_48_0, d_var_48_1, d_var_48_2, d_var_48_3, d_var_48_4, d_var_48_5, d_var_48_6, d_var_48_7, d_var_48_8, d_var_48_9, d_var_48_10, d_var_48_11, d_var_48_12, d_var_48_13, d_var_48_14, d_var_48_15, d_var_48_16, d_var_48_17, d_var_48_18, d_var_48_19);
	
	kernel_49<<<10, 10>>>(d_var_49_0, d_var_49_1, d_var_49_2, d_var_49_3, d_var_49_4, d_var_49_5, d_var_49_6, d_var_49_7, d_var_49_8, d_var_49_9, d_var_49_10, d_var_49_11, d_var_49_12, d_var_49_13, d_var_49_14, d_var_49_15, d_var_49_16, d_var_49_17, d_var_49_18, d_var_49_19);
	
	kernel_50<<<10, 10>>>(d_var_50_0, d_var_50_1, d_var_50_2, d_var_50_3, d_var_50_4, d_var_50_5, d_var_50_6, d_var_50_7, d_var_50_8, d_var_50_9, d_var_50_10, d_var_50_11, d_var_50_12, d_var_50_13, d_var_50_14, d_var_50_15, d_var_50_16, d_var_50_17, d_var_50_18, d_var_50_19);
	
	kernel_51<<<10, 10>>>(d_var_51_0, d_var_51_1, d_var_51_2, d_var_51_3, d_var_51_4, d_var_51_5, d_var_51_6, d_var_51_7, d_var_51_8, d_var_51_9, d_var_51_10, d_var_51_11, d_var_51_12, d_var_51_13, d_var_51_14, d_var_51_15, d_var_51_16, d_var_51_17, d_var_51_18, d_var_51_19);
	
	kernel_52<<<10, 10>>>(d_var_52_0, d_var_52_1, d_var_52_2, d_var_52_3, d_var_52_4, d_var_52_5, d_var_52_6, d_var_52_7, d_var_52_8, d_var_52_9, d_var_52_10, d_var_52_11, d_var_52_12, d_var_52_13, d_var_52_14, d_var_52_15, d_var_52_16, d_var_52_17, d_var_52_18, d_var_52_19);
	
	kernel_53<<<10, 10>>>(d_var_53_0, d_var_53_1, d_var_53_2, d_var_53_3, d_var_53_4, d_var_53_5, d_var_53_6, d_var_53_7, d_var_53_8, d_var_53_9, d_var_53_10, d_var_53_11, d_var_53_12, d_var_53_13, d_var_53_14, d_var_53_15, d_var_53_16, d_var_53_17, d_var_53_18, d_var_53_19);
	
	kernel_54<<<10, 10>>>(d_var_54_0, d_var_54_1, d_var_54_2, d_var_54_3, d_var_54_4, d_var_54_5, d_var_54_6, d_var_54_7, d_var_54_8, d_var_54_9, d_var_54_10, d_var_54_11, d_var_54_12, d_var_54_13, d_var_54_14, d_var_54_15, d_var_54_16, d_var_54_17, d_var_54_18, d_var_54_19);
	
	kernel_55<<<10, 10>>>(d_var_55_0, d_var_55_1, d_var_55_2, d_var_55_3, d_var_55_4, d_var_55_5, d_var_55_6, d_var_55_7, d_var_55_8, d_var_55_9, d_var_55_10, d_var_55_11, d_var_55_12, d_var_55_13, d_var_55_14, d_var_55_15, d_var_55_16, d_var_55_17, d_var_55_18, d_var_55_19);
	
	kernel_56<<<10, 10>>>(d_var_56_0, d_var_56_1, d_var_56_2, d_var_56_3, d_var_56_4, d_var_56_5, d_var_56_6, d_var_56_7, d_var_56_8, d_var_56_9, d_var_56_10, d_var_56_11, d_var_56_12, d_var_56_13, d_var_56_14, d_var_56_15, d_var_56_16, d_var_56_17, d_var_56_18, d_var_56_19);
	
	kernel_57<<<10, 10>>>(d_var_57_0, d_var_57_1, d_var_57_2, d_var_57_3, d_var_57_4, d_var_57_5, d_var_57_6, d_var_57_7, d_var_57_8, d_var_57_9, d_var_57_10, d_var_57_11, d_var_57_12, d_var_57_13, d_var_57_14, d_var_57_15, d_var_57_16, d_var_57_17, d_var_57_18, d_var_57_19);
	
	kernel_58<<<10, 10>>>(d_var_58_0, d_var_58_1, d_var_58_2, d_var_58_3, d_var_58_4, d_var_58_5, d_var_58_6, d_var_58_7, d_var_58_8, d_var_58_9, d_var_58_10, d_var_58_11, d_var_58_12, d_var_58_13, d_var_58_14, d_var_58_15, d_var_58_16, d_var_58_17, d_var_58_18, d_var_58_19);
	
	kernel_59<<<10, 10>>>(d_var_59_0, d_var_59_1, d_var_59_2, d_var_59_3, d_var_59_4, d_var_59_5, d_var_59_6, d_var_59_7, d_var_59_8, d_var_59_9, d_var_59_10, d_var_59_11, d_var_59_12, d_var_59_13, d_var_59_14, d_var_59_15, d_var_59_16, d_var_59_17, d_var_59_18, d_var_59_19);
	
	kernel_60<<<10, 10>>>(d_var_60_0, d_var_60_1, d_var_60_2, d_var_60_3, d_var_60_4, d_var_60_5, d_var_60_6, d_var_60_7, d_var_60_8, d_var_60_9, d_var_60_10, d_var_60_11, d_var_60_12, d_var_60_13, d_var_60_14, d_var_60_15, d_var_60_16, d_var_60_17, d_var_60_18, d_var_60_19);
	
	kernel_61<<<10, 10>>>(d_var_61_0, d_var_61_1, d_var_61_2, d_var_61_3, d_var_61_4, d_var_61_5, d_var_61_6, d_var_61_7, d_var_61_8, d_var_61_9, d_var_61_10, d_var_61_11, d_var_61_12, d_var_61_13, d_var_61_14, d_var_61_15, d_var_61_16, d_var_61_17, d_var_61_18, d_var_61_19);
	
	kernel_62<<<10, 10>>>(d_var_62_0, d_var_62_1, d_var_62_2, d_var_62_3, d_var_62_4, d_var_62_5, d_var_62_6, d_var_62_7, d_var_62_8, d_var_62_9, d_var_62_10, d_var_62_11, d_var_62_12, d_var_62_13, d_var_62_14, d_var_62_15, d_var_62_16, d_var_62_17, d_var_62_18, d_var_62_19);
	
	kernel_63<<<10, 10>>>(d_var_63_0, d_var_63_1, d_var_63_2, d_var_63_3, d_var_63_4, d_var_63_5, d_var_63_6, d_var_63_7, d_var_63_8, d_var_63_9, d_var_63_10, d_var_63_11, d_var_63_12, d_var_63_13, d_var_63_14, d_var_63_15, d_var_63_16, d_var_63_17, d_var_63_18, d_var_63_19);
	
	kernel_64<<<10, 10>>>(d_var_64_0, d_var_64_1, d_var_64_2, d_var_64_3, d_var_64_4, d_var_64_5, d_var_64_6, d_var_64_7, d_var_64_8, d_var_64_9, d_var_64_10, d_var_64_11, d_var_64_12, d_var_64_13, d_var_64_14, d_var_64_15, d_var_64_16, d_var_64_17, d_var_64_18, d_var_64_19);
	
	kernel_65<<<10, 10>>>(d_var_65_0, d_var_65_1, d_var_65_2, d_var_65_3, d_var_65_4, d_var_65_5, d_var_65_6, d_var_65_7, d_var_65_8, d_var_65_9, d_var_65_10, d_var_65_11, d_var_65_12, d_var_65_13, d_var_65_14, d_var_65_15, d_var_65_16, d_var_65_17, d_var_65_18, d_var_65_19);
	
	kernel_66<<<10, 10>>>(d_var_66_0, d_var_66_1, d_var_66_2, d_var_66_3, d_var_66_4, d_var_66_5, d_var_66_6, d_var_66_7, d_var_66_8, d_var_66_9, d_var_66_10, d_var_66_11, d_var_66_12, d_var_66_13, d_var_66_14, d_var_66_15, d_var_66_16, d_var_66_17, d_var_66_18, d_var_66_19);
	
	kernel_67<<<10, 10>>>(d_var_67_0, d_var_67_1, d_var_67_2, d_var_67_3, d_var_67_4, d_var_67_5, d_var_67_6, d_var_67_7, d_var_67_8, d_var_67_9, d_var_67_10, d_var_67_11, d_var_67_12, d_var_67_13, d_var_67_14, d_var_67_15, d_var_67_16, d_var_67_17, d_var_67_18, d_var_67_19);
	
	kernel_68<<<10, 10>>>(d_var_68_0, d_var_68_1, d_var_68_2, d_var_68_3, d_var_68_4, d_var_68_5, d_var_68_6, d_var_68_7, d_var_68_8, d_var_68_9, d_var_68_10, d_var_68_11, d_var_68_12, d_var_68_13, d_var_68_14, d_var_68_15, d_var_68_16, d_var_68_17, d_var_68_18, d_var_68_19);
	
	kernel_69<<<10, 10>>>(d_var_69_0, d_var_69_1, d_var_69_2, d_var_69_3, d_var_69_4, d_var_69_5, d_var_69_6, d_var_69_7, d_var_69_8, d_var_69_9, d_var_69_10, d_var_69_11, d_var_69_12, d_var_69_13, d_var_69_14, d_var_69_15, d_var_69_16, d_var_69_17, d_var_69_18, d_var_69_19);
	
	kernel_70<<<10, 10>>>(d_var_70_0, d_var_70_1, d_var_70_2, d_var_70_3, d_var_70_4, d_var_70_5, d_var_70_6, d_var_70_7, d_var_70_8, d_var_70_9, d_var_70_10, d_var_70_11, d_var_70_12, d_var_70_13, d_var_70_14, d_var_70_15, d_var_70_16, d_var_70_17, d_var_70_18, d_var_70_19);
	
	kernel_71<<<10, 10>>>(d_var_71_0, d_var_71_1, d_var_71_2, d_var_71_3, d_var_71_4, d_var_71_5, d_var_71_6, d_var_71_7, d_var_71_8, d_var_71_9, d_var_71_10, d_var_71_11, d_var_71_12, d_var_71_13, d_var_71_14, d_var_71_15, d_var_71_16, d_var_71_17, d_var_71_18, d_var_71_19);
	
	kernel_72<<<10, 10>>>(d_var_72_0, d_var_72_1, d_var_72_2, d_var_72_3, d_var_72_4, d_var_72_5, d_var_72_6, d_var_72_7, d_var_72_8, d_var_72_9, d_var_72_10, d_var_72_11, d_var_72_12, d_var_72_13, d_var_72_14, d_var_72_15, d_var_72_16, d_var_72_17, d_var_72_18, d_var_72_19);
	
	kernel_73<<<10, 10>>>(d_var_73_0, d_var_73_1, d_var_73_2, d_var_73_3, d_var_73_4, d_var_73_5, d_var_73_6, d_var_73_7, d_var_73_8, d_var_73_9, d_var_73_10, d_var_73_11, d_var_73_12, d_var_73_13, d_var_73_14, d_var_73_15, d_var_73_16, d_var_73_17, d_var_73_18, d_var_73_19);
	
	kernel_74<<<10, 10>>>(d_var_74_0, d_var_74_1, d_var_74_2, d_var_74_3, d_var_74_4, d_var_74_5, d_var_74_6, d_var_74_7, d_var_74_8, d_var_74_9, d_var_74_10, d_var_74_11, d_var_74_12, d_var_74_13, d_var_74_14, d_var_74_15, d_var_74_16, d_var_74_17, d_var_74_18, d_var_74_19);
	
	kernel_75<<<10, 10>>>(d_var_75_0, d_var_75_1, d_var_75_2, d_var_75_3, d_var_75_4, d_var_75_5, d_var_75_6, d_var_75_7, d_var_75_8, d_var_75_9, d_var_75_10, d_var_75_11, d_var_75_12, d_var_75_13, d_var_75_14, d_var_75_15, d_var_75_16, d_var_75_17, d_var_75_18, d_var_75_19);
	
	kernel_76<<<10, 10>>>(d_var_76_0, d_var_76_1, d_var_76_2, d_var_76_3, d_var_76_4, d_var_76_5, d_var_76_6, d_var_76_7, d_var_76_8, d_var_76_9, d_var_76_10, d_var_76_11, d_var_76_12, d_var_76_13, d_var_76_14, d_var_76_15, d_var_76_16, d_var_76_17, d_var_76_18, d_var_76_19);
	
	kernel_77<<<10, 10>>>(d_var_77_0, d_var_77_1, d_var_77_2, d_var_77_3, d_var_77_4, d_var_77_5, d_var_77_6, d_var_77_7, d_var_77_8, d_var_77_9, d_var_77_10, d_var_77_11, d_var_77_12, d_var_77_13, d_var_77_14, d_var_77_15, d_var_77_16, d_var_77_17, d_var_77_18, d_var_77_19);
	
	kernel_78<<<10, 10>>>(d_var_78_0, d_var_78_1, d_var_78_2, d_var_78_3, d_var_78_4, d_var_78_5, d_var_78_6, d_var_78_7, d_var_78_8, d_var_78_9, d_var_78_10, d_var_78_11, d_var_78_12, d_var_78_13, d_var_78_14, d_var_78_15, d_var_78_16, d_var_78_17, d_var_78_18, d_var_78_19);
	
	kernel_79<<<10, 10>>>(d_var_79_0, d_var_79_1, d_var_79_2, d_var_79_3, d_var_79_4, d_var_79_5, d_var_79_6, d_var_79_7, d_var_79_8, d_var_79_9, d_var_79_10, d_var_79_11, d_var_79_12, d_var_79_13, d_var_79_14, d_var_79_15, d_var_79_16, d_var_79_17, d_var_79_18, d_var_79_19);
	
	kernel_80<<<10, 10>>>(d_var_80_0, d_var_80_1, d_var_80_2, d_var_80_3, d_var_80_4, d_var_80_5, d_var_80_6, d_var_80_7, d_var_80_8, d_var_80_9, d_var_80_10, d_var_80_11, d_var_80_12, d_var_80_13, d_var_80_14, d_var_80_15, d_var_80_16, d_var_80_17, d_var_80_18, d_var_80_19);
	
	kernel_81<<<10, 10>>>(d_var_81_0, d_var_81_1, d_var_81_2, d_var_81_3, d_var_81_4, d_var_81_5, d_var_81_6, d_var_81_7, d_var_81_8, d_var_81_9, d_var_81_10, d_var_81_11, d_var_81_12, d_var_81_13, d_var_81_14, d_var_81_15, d_var_81_16, d_var_81_17, d_var_81_18, d_var_81_19);
	
	kernel_82<<<10, 10>>>(d_var_82_0, d_var_82_1, d_var_82_2, d_var_82_3, d_var_82_4, d_var_82_5, d_var_82_6, d_var_82_7, d_var_82_8, d_var_82_9, d_var_82_10, d_var_82_11, d_var_82_12, d_var_82_13, d_var_82_14, d_var_82_15, d_var_82_16, d_var_82_17, d_var_82_18, d_var_82_19);
	
	kernel_83<<<10, 10>>>(d_var_83_0, d_var_83_1, d_var_83_2, d_var_83_3, d_var_83_4, d_var_83_5, d_var_83_6, d_var_83_7, d_var_83_8, d_var_83_9, d_var_83_10, d_var_83_11, d_var_83_12, d_var_83_13, d_var_83_14, d_var_83_15, d_var_83_16, d_var_83_17, d_var_83_18, d_var_83_19);
	
	kernel_84<<<10, 10>>>(d_var_84_0, d_var_84_1, d_var_84_2, d_var_84_3, d_var_84_4, d_var_84_5, d_var_84_6, d_var_84_7, d_var_84_8, d_var_84_9, d_var_84_10, d_var_84_11, d_var_84_12, d_var_84_13, d_var_84_14, d_var_84_15, d_var_84_16, d_var_84_17, d_var_84_18, d_var_84_19);
	
	kernel_85<<<10, 10>>>(d_var_85_0, d_var_85_1, d_var_85_2, d_var_85_3, d_var_85_4, d_var_85_5, d_var_85_6, d_var_85_7, d_var_85_8, d_var_85_9, d_var_85_10, d_var_85_11, d_var_85_12, d_var_85_13, d_var_85_14, d_var_85_15, d_var_85_16, d_var_85_17, d_var_85_18, d_var_85_19);
	
	kernel_86<<<10, 10>>>(d_var_86_0, d_var_86_1, d_var_86_2, d_var_86_3, d_var_86_4, d_var_86_5, d_var_86_6, d_var_86_7, d_var_86_8, d_var_86_9, d_var_86_10, d_var_86_11, d_var_86_12, d_var_86_13, d_var_86_14, d_var_86_15, d_var_86_16, d_var_86_17, d_var_86_18, d_var_86_19);
	
	kernel_87<<<10, 10>>>(d_var_87_0, d_var_87_1, d_var_87_2, d_var_87_3, d_var_87_4, d_var_87_5, d_var_87_6, d_var_87_7, d_var_87_8, d_var_87_9, d_var_87_10, d_var_87_11, d_var_87_12, d_var_87_13, d_var_87_14, d_var_87_15, d_var_87_16, d_var_87_17, d_var_87_18, d_var_87_19);
	
	kernel_88<<<10, 10>>>(d_var_88_0, d_var_88_1, d_var_88_2, d_var_88_3, d_var_88_4, d_var_88_5, d_var_88_6, d_var_88_7, d_var_88_8, d_var_88_9, d_var_88_10, d_var_88_11, d_var_88_12, d_var_88_13, d_var_88_14, d_var_88_15, d_var_88_16, d_var_88_17, d_var_88_18, d_var_88_19);
	
	kernel_89<<<10, 10>>>(d_var_89_0, d_var_89_1, d_var_89_2, d_var_89_3, d_var_89_4, d_var_89_5, d_var_89_6, d_var_89_7, d_var_89_8, d_var_89_9, d_var_89_10, d_var_89_11, d_var_89_12, d_var_89_13, d_var_89_14, d_var_89_15, d_var_89_16, d_var_89_17, d_var_89_18, d_var_89_19);
	
	kernel_90<<<10, 10>>>(d_var_90_0, d_var_90_1, d_var_90_2, d_var_90_3, d_var_90_4, d_var_90_5, d_var_90_6, d_var_90_7, d_var_90_8, d_var_90_9, d_var_90_10, d_var_90_11, d_var_90_12, d_var_90_13, d_var_90_14, d_var_90_15, d_var_90_16, d_var_90_17, d_var_90_18, d_var_90_19);
	
	kernel_91<<<10, 10>>>(d_var_91_0, d_var_91_1, d_var_91_2, d_var_91_3, d_var_91_4, d_var_91_5, d_var_91_6, d_var_91_7, d_var_91_8, d_var_91_9, d_var_91_10, d_var_91_11, d_var_91_12, d_var_91_13, d_var_91_14, d_var_91_15, d_var_91_16, d_var_91_17, d_var_91_18, d_var_91_19);
	
	kernel_92<<<10, 10>>>(d_var_92_0, d_var_92_1, d_var_92_2, d_var_92_3, d_var_92_4, d_var_92_5, d_var_92_6, d_var_92_7, d_var_92_8, d_var_92_9, d_var_92_10, d_var_92_11, d_var_92_12, d_var_92_13, d_var_92_14, d_var_92_15, d_var_92_16, d_var_92_17, d_var_92_18, d_var_92_19);
	
	kernel_93<<<10, 10>>>(d_var_93_0, d_var_93_1, d_var_93_2, d_var_93_3, d_var_93_4, d_var_93_5, d_var_93_6, d_var_93_7, d_var_93_8, d_var_93_9, d_var_93_10, d_var_93_11, d_var_93_12, d_var_93_13, d_var_93_14, d_var_93_15, d_var_93_16, d_var_93_17, d_var_93_18, d_var_93_19);
	
	kernel_94<<<10, 10>>>(d_var_94_0, d_var_94_1, d_var_94_2, d_var_94_3, d_var_94_4, d_var_94_5, d_var_94_6, d_var_94_7, d_var_94_8, d_var_94_9, d_var_94_10, d_var_94_11, d_var_94_12, d_var_94_13, d_var_94_14, d_var_94_15, d_var_94_16, d_var_94_17, d_var_94_18, d_var_94_19);
	
	kernel_95<<<10, 10>>>(d_var_95_0, d_var_95_1, d_var_95_2, d_var_95_3, d_var_95_4, d_var_95_5, d_var_95_6, d_var_95_7, d_var_95_8, d_var_95_9, d_var_95_10, d_var_95_11, d_var_95_12, d_var_95_13, d_var_95_14, d_var_95_15, d_var_95_16, d_var_95_17, d_var_95_18, d_var_95_19);
	
	kernel_96<<<10, 10>>>(d_var_96_0, d_var_96_1, d_var_96_2, d_var_96_3, d_var_96_4, d_var_96_5, d_var_96_6, d_var_96_7, d_var_96_8, d_var_96_9, d_var_96_10, d_var_96_11, d_var_96_12, d_var_96_13, d_var_96_14, d_var_96_15, d_var_96_16, d_var_96_17, d_var_96_18, d_var_96_19);
	
	kernel_97<<<10, 10>>>(d_var_97_0, d_var_97_1, d_var_97_2, d_var_97_3, d_var_97_4, d_var_97_5, d_var_97_6, d_var_97_7, d_var_97_8, d_var_97_9, d_var_97_10, d_var_97_11, d_var_97_12, d_var_97_13, d_var_97_14, d_var_97_15, d_var_97_16, d_var_97_17, d_var_97_18, d_var_97_19);
	
	kernel_98<<<10, 10>>>(d_var_98_0, d_var_98_1, d_var_98_2, d_var_98_3, d_var_98_4, d_var_98_5, d_var_98_6, d_var_98_7, d_var_98_8, d_var_98_9, d_var_98_10, d_var_98_11, d_var_98_12, d_var_98_13, d_var_98_14, d_var_98_15, d_var_98_16, d_var_98_17, d_var_98_18, d_var_98_19);
	
	kernel_99<<<10, 10>>>(d_var_99_0, d_var_99_1, d_var_99_2, d_var_99_3, d_var_99_4, d_var_99_5, d_var_99_6, d_var_99_7, d_var_99_8, d_var_99_9, d_var_99_10, d_var_99_11, d_var_99_12, d_var_99_13, d_var_99_14, d_var_99_15, d_var_99_16, d_var_99_17, d_var_99_18, d_var_99_19);
	
    // clang-format on

    printf("Done\n");
    return 0;
}
