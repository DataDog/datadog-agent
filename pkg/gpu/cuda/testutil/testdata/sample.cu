/* Sample program that does nothing, just build a binary that we will inspect */

#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void kernel1(float *A, int n) {
    A[0] = 0;
}

__global__ void kernel2(float *A, int n) {
    __shared__ char globalArray[256];

    for (int i = 0; i < n; i++) {
        globalArray[threadIdx.x] = A[i];
    }

    A[0] = globalArray[threadIdx.x];
}

int main(void) {
    float *h_A = (float *)malloc(sizeof(float));

    float *d_A;
    hipMalloc((void **)&d_A, 1);

    // clang-format off
    kernel1<<<10, 10>>>(d_A, 100);
    kernel2<<<10, 10>>>(d_A, 100);
    // clang-format on

    printf("Done\n");
    return 0;
}
